#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/


#include "test_common.h"
#include "transformer_engine/logging.h"
#include "transformer_engine/transformer_engine.h"
#include <gtest/gtest.h>
#include <algorithm>
#include <memory>
#include <random>

namespace test {

std::vector<DType> all_fp_types = {DType::kFloat32,
                                   DType::kFloat16,
                                   DType::kBFloat16,
                                   DType::kFloat8E5M2,
                                   DType::kFloat8E4M3};

bool areShapesEqual(const NVTEShape &s1, const NVTEShape &s2) {
  if (s1.ndim != s2.ndim) return false;

  for (size_t i = 0; i < s1.ndim; ++i) {
    if (s1.data[i] != s2.data[i]) return false;
  }

  return true;
}

size_t typeToSize(DType type) {
  TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(type, T,
  {
      return TypeInfo<T>::size;
  });
}

const std::string &typeName(DType type) {
  static const std::unordered_map<DType, std::string> name_map = {
    {DType::kByte, "byte"},
    {DType::kInt32, "int32"},
    {DType::kFloat32, "float32"},
    {DType::kFloat16, "float16"},
    {DType::kBFloat16, "bfloat16"},
    {DType::kFloat8E4M3, "float8e4m3"},
    {DType::kFloat8E5M2, "float8e5m2"}};
  return name_map.at(type);
}

size_t product(const NVTEShape &shape) {
    size_t ret = 1;
    for (size_t i = 0; i < shape.ndim; ++i) {
      ret *= shape.data[i];
    }
    return ret;
}

Tensor::Tensor(const NVTEShape &shape, const DType type) {
    size_t s = typeToSize(type);
    size_t total_size = product(shape) * s;
    void *dptr = nullptr;
    cpu_data_ = nullptr;
    if (total_size != 0) {
        hipMalloc((void**)&dptr, total_size);  // NOLINT(*)
        hipMemset(dptr, 0, total_size);
        cpu_data_ = std::make_unique<unsigned char[]>(total_size);
    }
    tensor_ = TensorWrapper(dptr, shape, type);
}

void Tensor::to_cpu() const {
  const NVTEShape s = tensor_.shape();
  const size_t size = product(s) * typeToSize(tensor_.dtype());
  hipMemcpy(cpu_data_.get(), tensor_.dptr(), size, hipMemcpyDeviceToHost);
}

void Tensor::from_cpu() const {
  const NVTEShape s = tensor_.shape();
  const size_t size = product(s) * typeToSize(tensor_.dtype());
  hipMemcpy(tensor_.dptr(), cpu_data_.get(), size, hipMemcpyHostToDevice);
}

using std::to_string;

template <typename T>
std::string to_string(const std::vector<T> &v) {
  std::string s = "[";
  for (const auto x : v) {
    s += to_string(x) + ", ";
  }
  s.pop_back();
  s.pop_back();
  return s + "]";
}

std::vector<size_t> unravel(const size_t i, const NVTEShape &shape) {
  std::vector<size_t> ret;
  size_t current_i = i;
  for (size_t current = shape.ndim - 1;
       current > 0;
       --current) {
    ret.push_back(current_i % shape.data[current]);
    current_i /= shape.data[current];
  }
  ret.push_back(current_i);
  std::reverse(ret.begin(), ret.end());
  return ret;
}

void compareResults(const std::string &name, const Tensor &test, const void *ref,
                    double atol, double rtol) {
  test.to_cpu();
  const size_t N = product(test.shape());
  TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(test.dtype(), T,
    const T *test_data = test.cpu_dptr<T>();
    const T *ref_data = reinterpret_cast<const T*>(ref);
    for (size_t i = 0; i < N; ++i) {
      double t = static_cast<double>(test_data[i]);
      double r = static_cast<double>(ref_data[i]);
      bool mismatch = fabs(t - r) > atol && (r == 0 || fabs((t - r) / r) > rtol);
      /* For Float32 the floating point comparison is enough to error out */
      bool assertion = mismatch && test.dtype() == DType::kFloat32;
      if (mismatch && !assertion) {
        /* Check if it is just a failure of round to nearest choosing different
           side of the real value */
        const double mean = (t + r) / 2;
        const double mean_p = mean >= 0 ? mean * (1 + 1e-6) : mean * (1 - 1e-6);
        const double mean_m = mean >= 0 ? mean * (1 - 1e-6) : mean * (1 + 1e-6);
        const double cast_mean_p = static_cast<double>(static_cast<T>(mean_p));
        const double cast_mean_m = static_cast<double>(static_cast<T>(mean_m));
        assertion = !(cast_mean_m == std::min(t,r) && cast_mean_p == std::max(t,r));
      }
      ASSERT_FALSE(assertion) << "Error in tensor " << name << std::endl
                              << "Mismatch at place " << to_string(unravel(i, test.shape()))
                              << " (" << std::to_string(i) << "): " << t << " vs " << r;

    }
  );
}

std::pair<double, double> getTolerances(const DType type) {
  switch(type) {
    case DType::kFloat32:
      return {1e-6, 5e-6};
    case DType::kFloat16:
      return {1e-5, 1e-3};
    case DType::kBFloat16:
      return {1e-5, 1e-2};
    case DType::kFloat8E4M3:
    case DType::kFloat8E5M2:
      return {1e-2, 1e-2};
    default:
      NVTE_CHECK("Invalid type!");
  }
  return {0, 0};
}

void fillUniform(const Tensor &t) {
  const size_t size = product(t.shape());
  static std::mt19937 gen(12345);
  std::uniform_real_distribution<> dis(-2.0, 1.0);
  TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(t.dtype(), T, {
      T *data = t.cpu_dptr<T>();
      for (size_t i = 0; i < size; ++i) {
          data[i] = T(dis(gen));
      }
  });
  t.from_cpu();
}

}  // namespace test
