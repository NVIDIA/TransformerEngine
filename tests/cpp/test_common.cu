#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/


#include "test_common.h"

#include <algorithm>
#include <memory>
#include <random>
#include <cassert>
#include <cmath>
#include <string>

#include <gtest/gtest.h>
#include <omp.h>

#include <transformer_engine/transformer_engine.h>
#include "util/logging.h"

namespace test {

size_t create_seed_from_tensor_name(const std::string& tensor_name) {
  auto full_name = std::string(testing::UnitTest::GetInstance()->current_test_info()->name()) +
                   "/" + tensor_name;
  return std::hash<std::string>{}(full_name);
}

std::vector<DType> all_fp_types = {DType::kFloat32,
                                   DType::kFloat16,
                                   DType::kBFloat16,
                                   DType::kFloat8E5M2,
                                   DType::kFloat8E4M3};

bool areShapesEqual(const NVTEShape &s1, const NVTEShape &s2) {
  if (s1.ndim != s2.ndim) return false;

  for (size_t i = 0; i < s1.ndim; ++i) {
    if (s1.data[i] != s2.data[i]) return false;
  }

  return true;
}

size_t typeToSize(DType type) {
  TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(type, T,
  {
      return TypeInfo<T>::size;
  });
}

const std::string &typeName(DType type) {
  static const std::unordered_map<DType, std::string> name_map = {
    {DType::kByte, "byte"},
    {DType::kInt32, "int32"},
    {DType::kInt64, "int64"},
    {DType::kFloat32, "float32"},
    {DType::kFloat16, "float16"},
    {DType::kBFloat16, "bfloat16"},
    {DType::kFloat8E4M3, "float8e4m3"},
    {DType::kFloat8E5M2, "float8e5m2"},
    {DType::kFloat8E8M0, "float8e8m0"}};
  return name_map.at(type);
}

const std::string& caseName(InputsFillCase type) {
  static const std::unordered_map<InputsFillCase, std::string> name_map = {
    {InputsFillCase::uniform, "uniform"},
    {InputsFillCase::zeros, "zeros"},
    {InputsFillCase::zero_to_minNorm, "zero_to_minNorm"},
    {InputsFillCase::minNorm_to_maxNorm, "minNorm_to_maxNorm"},
    {InputsFillCase::maxNorm_to_inf, "maxNorm_to_inf"}};
  return name_map.at(type);
}

size_t product(const NVTEShape &shape, size_t begin, size_t end) {
    size_t ret = 1;
    NVTE_CHECK(end <= shape.ndim);
    for (size_t i = begin; i < end; ++i) {
      ret *= shape.data[i];
    }
    return ret;
}
size_t product(const NVTEShape &shape) {
  return product(shape, 0, shape.ndim);
}
size_t product(const std::vector<size_t> shape, size_t begin, size_t end) {
    size_t ret = 1;
    NVTE_CHECK(end <= shape.size());
    for (size_t i = begin; i < end; ++i) {
      ret *= shape[i];
    }
    return ret;
}

size_t product(const std::vector<size_t>& shape) {
  return product(shape, 0, shape.size());
}

size_t DIVUP(const size_t &x, const size_t &y){
  return (((x) + ((y)-1)) / (y));
}

inline bool is_tensor_scaling(const NVTEScalingMode &mode) {
  return mode == NVTE_DELAYED_TENSOR_SCALING;
}

struct scale_inv_meta {
  std::vector<size_t> shape;
  DType type;
  size_t type_size;
};

NVTEShape convertShape(const std::vector<size_t>& shape) {
  return {shape.data(), shape.size()};
}

std::pair<scale_inv_meta, scale_inv_meta> get_scales(const NVTEShape& shape,
                                                     const NVTEScalingMode scaling_mode) {
  if (scaling_mode == NVTE_DELAYED_TENSOR_SCALING) {
    scale_inv_meta ret;
    ret.shape = {1};
    ret.type = DType::kFloat32;
    ret.type_size = sizeof(float);
    return {ret, ret};
  }
  if (scaling_mode == NVTE_MXFP8_1D_SCALING) {
    std::vector<size_t> shape_vec;
    for (size_t i = 0; i < shape.ndim; ++i) {
      shape_vec.push_back(shape.data[i]);
    }
    size_t first_dim = first_dimension(shape_vec);
    size_t last_dim = last_dimension(shape_vec);

    scale_inv_meta ret_rowwise, ret_colwise;

    auto block_alignment = std::vector<size_t>{128ul,4ul};
    {
      auto alignment = block_alignment[0];
      auto scale_dim_0 = DIVUP(DIVUP(first_dim,
                                     static_cast<size_t>(1)),
                               alignment) * alignment;
      alignment = block_alignment[1];
      auto scale_dim_1 = DIVUP(DIVUP(last_dim,
                                     static_cast<size_t>(32)),
                               alignment) * alignment;
      ret_rowwise.shape = {scale_dim_0, scale_dim_1};
    }
    {
      auto alignment = block_alignment[1];
      auto scale_dim_0 = DIVUP(DIVUP(first_dim,
                                     static_cast<size_t>(32)),
                               alignment) * alignment;
      alignment = block_alignment[0];
      auto scale_dim_1 = DIVUP(DIVUP(last_dim,
                                     static_cast<size_t>(1)),
                               alignment) * alignment;
      ret_colwise.shape = {scale_dim_0, scale_dim_1};
    }
    ret_rowwise.type = DType::kFloat8E8M0;
    ret_colwise.type = DType::kFloat8E8M0;
    ret_rowwise.type_size = sizeof(uint8_t);
    ret_colwise.type_size = sizeof(uint8_t);

    return {ret_rowwise, ret_colwise};
  }

  NVTE_ERROR("Invalid scaling mode!");
}

Tensor::Tensor(const std::string& name,
               const NVTEShape &shape, const DType type,
               const bool rowwise, const bool columnwise,
               const NVTEScalingMode &scaling_mode) {
  name_ = name;
  const size_t seed = create_seed_from_tensor_name(name);
  gen_.seed(seed);
  rowwise_ = rowwise;
  columnwise_ = columnwise;
  size_t s = typeToSize(type);
  size_t total_size = product(shape) * s;
  void *dptr_rowwise = nullptr;
  void *dptr_columnwise = nullptr;
  cpu_data_rowwise_ = nullptr;
  cpu_data_columnwise_ = nullptr;
  amax_cpu_data_ = nullptr;
  scale_cpu_data_ = nullptr;
  rowwise_scale_inv_cpu_data_ = nullptr;
  columnwise_scale_inv_cpu_data_ = nullptr;
  float *amax = nullptr, *scale = nullptr;
  float *rowwise_scale_inv = nullptr, *columnwise_scale_inv = nullptr;
  if (columnwise) {
    NVTE_CHECK(shape.ndim >= 2);
  }
  std::vector<size_t> normalized_shape_v = {product(shape, 0, shape.ndim - 1),
                                            shape.data[shape.ndim - 1]};
  NVTEShape normalized_shape = convertShape(normalized_shape_v);

  std::vector<size_t> columnwise_shape_vec;
  if (scaling_mode == NVTE_DELAYED_TENSOR_SCALING) {
    // Transpose when tensor scaling
    columnwise_shape_vec.emplace_back(shape.data[shape.ndim - 1]);
    for (size_t i = 0; i < shape.ndim - 1; ++i) {
      columnwise_shape_vec.emplace_back(shape.data[i]);
    }
  } else {
    // Same shape for MX
    for (size_t i = 0; i < shape.ndim; ++i) {
      columnwise_shape_vec.emplace_back(shape.data[i]);
    }
  }
  const NVTEShape columnwise_shape{columnwise_shape_vec.data(), columnwise_shape_vec.size()};

  tensor_ = TensorWrapper(scaling_mode);

  if (total_size != 0) {
    if (rowwise) {
      hipMalloc((void**)&dptr_rowwise, total_size);  // NOLINT(*)
      hipMemset(dptr_rowwise, 0, total_size);
      cpu_data_rowwise_ = std::make_unique<unsigned char[]>(total_size);
      std::fill_n(cpu_data_rowwise_.get(), total_size, 0);
    }
    if (columnwise) {
      hipMalloc((void**)&dptr_columnwise, total_size);  // NOLINT(*)
      hipMemset(dptr_columnwise, 0, total_size);
      cpu_data_columnwise_ = std::make_unique<unsigned char[]>(total_size);
      std::fill_n(cpu_data_columnwise_.get(), total_size, 0);
    }
  }
  tensor_.set_rowwise_data(dptr_rowwise, type, shape);
  tensor_.set_columnwise_data(dptr_columnwise, type, columnwise_shape);

  if (isFp8Type(type)) {
    if (is_tensor_scaling(scaling_mode)) {
      hipMalloc((void**)&amax, sizeof(float));  // NOLINT(*)
      hipMemset(amax, 0, sizeof(float));
      hipMalloc((void**)&scale, sizeof(float));  // NOLINT(*)
      hipMemset(scale, 0, sizeof(float));
      amax_cpu_data_ = std::make_shared<float>(0);
      scale_cpu_data_ = std::make_shared<float>(0);
      tensor_.set_amax(amax, DType::kFloat32, std::vector<size_t>{1});
      tensor_.set_scale(scale, DType::kFloat32, std::vector<size_t>{1});
      hipMalloc((void**)&rowwise_scale_inv, sizeof(float));  // NOLINT(*)
      if (rowwise) {
        tensor_.set_rowwise_scale_inv(rowwise_scale_inv, DType::kFloat32,
                                      std::vector<size_t>{1});
        rowwise_scale_inv_cpu_data_ = std::make_unique<unsigned char[]>(sizeof(float));
        std::fill_n(rowwise_scale_inv_cpu_data_.get(), sizeof(float), 0);
      }
      if (columnwise) {
        tensor_.set_columnwise_scale_inv(rowwise_scale_inv, DType::kFloat32,
                                         std::vector<size_t>{1});
        columnwise_scale_inv_cpu_data_ = std::make_unique<unsigned char[]>(sizeof(float));
        std::fill_n(columnwise_scale_inv_cpu_data_.get(), sizeof(float), 0);
      }
    } else {
      auto [rowwise_scale_meta, colwise_scale_meta] = get_scales(normalized_shape,
                                                                 tensor_.scaling_mode());
      auto rowwise_scale_size = product(rowwise_scale_meta.shape) * rowwise_scale_meta.type_size;
      auto columnwise_scale_size = product(colwise_scale_meta.shape) * colwise_scale_meta.type_size;
      auto scale_shape = rowwise_scale_meta.shape;
      auto columnwise_scale_shape = colwise_scale_meta.shape;
      if (rowwise) {
        hipMalloc((void**)&rowwise_scale_inv, rowwise_scale_size);  // NOLINT(*)
        hipMemset(rowwise_scale_inv, 0, rowwise_scale_size);
        rowwise_scale_inv_cpu_data_ = std::make_unique<unsigned char[]>(rowwise_scale_size);
        std::fill_n(rowwise_scale_inv_cpu_data_.get(), rowwise_scale_size, 0);
        tensor_.set_rowwise_scale_inv(rowwise_scale_inv, DType::kFloat8E8M0, scale_shape);
      }
      if (columnwise) {
        hipMalloc((void**)&columnwise_scale_inv, columnwise_scale_size);  // NOLINT(*)
        hipMemset(columnwise_scale_inv, 0, columnwise_scale_size);
        columnwise_scale_inv_cpu_data_ = std::make_unique<unsigned char[]>(columnwise_scale_size);
        std::fill_n(columnwise_scale_inv_cpu_data_.get(), columnwise_scale_size, 0);
        tensor_.set_columnwise_scale_inv(columnwise_scale_inv, DType::kFloat8E8M0, columnwise_scale_shape);
      }
    }
  }
}

void Tensor::to_cpu() const {
  const NVTEShape s = tensor_.shape();
  const size_t size = product(s) * typeToSize(tensor_.dtype());
  if (rowwise_) {
    hipMemcpy(cpu_data_rowwise_.get(),
               tensor_.get_rowwise_data().data_ptr,
               size,
               hipMemcpyDeviceToHost);
  }
  if (columnwise_) {
    hipMemcpy(cpu_data_columnwise_.get(),
               tensor_.get_columnwise_data().data_ptr,
               size,
               hipMemcpyDeviceToHost);
  }
  if (isFp8Type(dtype())) {
    if (is_tensor_scaling(tensor_.scaling_mode())) {
      hipMemcpy(amax_cpu_data_.get(),
                 tensor_.amax(),
                 sizeof(float),
                 hipMemcpyDeviceToHost);
      hipMemcpy(scale_cpu_data_.get(),
                 tensor_.scale(),
                 sizeof(float),
                 hipMemcpyDeviceToHost);
    }
    auto [rowwise_scale_meta, colwise_scale_meta] = get_scales(s, tensor_.scaling_mode());
    if (rowwise_) {
      auto scale_size = product(rowwise_scale_meta.shape) * rowwise_scale_meta.type_size;
      hipMemcpy(rowwise_scale_inv_cpu_data_.get(),
                 tensor_.get_rowwise_scale_inv().data_ptr,
                 scale_size,
                 hipMemcpyDeviceToHost);
    }
    if (columnwise_) {
      auto scale_size = product(colwise_scale_meta.shape) * colwise_scale_meta.type_size;
      hipMemcpy(columnwise_scale_inv_cpu_data_.get(),
                 tensor_.get_columnwise_scale_inv().data_ptr,
                 scale_size,
                 hipMemcpyDeviceToHost);
    }
  }
}

void Tensor::from_cpu() const {
  const NVTEShape s = tensor_.shape();
  const size_t size = product(s) * typeToSize(tensor_.dtype());
  if (rowwise_) {
    hipMemcpy(tensor_.get_rowwise_data().data_ptr,
               cpu_data_rowwise_.get(), size, hipMemcpyHostToDevice);
  }
  if (columnwise_) {
    hipMemcpy(tensor_.get_columnwise_data().data_ptr,
               cpu_data_columnwise_.get(), size, hipMemcpyHostToDevice);
  }
  if (isFp8Type(dtype())) {
    if (is_tensor_scaling(tensor_.scaling_mode())) {
      hipMemcpy(tensor_.amax(), amax_cpu_data_.get(), sizeof(float),
                 hipMemcpyHostToDevice);
      hipMemcpy(tensor_.scale(), scale_cpu_data_.get(), sizeof(float),
                 hipMemcpyHostToDevice);
    }
    auto [rowwise_scale_meta, colwise_scale_meta] = get_scales(s, tensor_.scaling_mode());
    if (rowwise_) {
      auto scale_size = product(rowwise_scale_meta.shape) * rowwise_scale_meta.type_size;
      hipMemcpy(tensor_.get_rowwise_scale_inv().data_ptr,
                 rowwise_scale_inv_cpu_data_.get(), scale_size,
                 hipMemcpyHostToDevice);
    }
    if (columnwise_) {
      auto scale_size = product(colwise_scale_meta.shape) * colwise_scale_meta.type_size;
      hipMemcpy(tensor_.get_columnwise_scale_inv().data_ptr,
                 columnwise_scale_inv_cpu_data_.get(), scale_size,
                 hipMemcpyHostToDevice);
    }
  }
}

void Tensor::set_scale(float scale) {
  if (isFp8Type(dtype())) {
    NVTE_CHECK(scale_cpu_data_);
  if (is_tensor_scaling(tensor_.scaling_mode())) {
      *scale_cpu_data_ = scale;
      from_cpu();
    }
  }
}

void Tensor::set_scale_inv(float scale_inv) {
  if (isFp8Type(dtype())) {
    if (rowwise_) {
      NVTE_CHECK(rowwise_scale_inv_cpu_data_);
    }
    if (columnwise_) {
      NVTE_CHECK(columnwise_scale_inv_cpu_data_);
    }
    auto [rowwise_scale_meta, colwise_scale_meta] = get_scales(tensor_.shape(), tensor_.scaling_mode());
    if (rowwise_) {
      auto num_scales = product(rowwise_scale_meta.shape);
      if (num_scales == 1){
        rowwise_cpu_scale_inv_ptr<float>()[0] = scale_inv;
      } else{
        std::uniform_int_distribution<uint8_t> dis(0, 127);
        auto* scale_inv_ptr = rowwise_cpu_scale_inv_ptr<uint8_t>();
        for (size_t i = 0; i < num_scales; i++){
          scale_inv_ptr[i] = dis(gen_);
        }
      }
    }
    if (columnwise_) {
      auto num_scales = product(colwise_scale_meta.shape);
      if (num_scales == 1){
        columnwise_cpu_scale_inv_ptr<float>()[0] = scale_inv;
      } else{
        std::uniform_int_distribution<uint8_t> dis(0, 127);
        auto* scale_inv_ptr = columnwise_cpu_scale_inv_ptr<uint8_t>();
        for (size_t i = 0; i < num_scales; i++){
          scale_inv_ptr[i] = dis(gen_);
        }
      }
    }
    from_cpu();
  }
}

void Tensor::shareFP8Meta(const Tensor &other) {
  if(isFp8Type(dtype()) && isFp8Type(other.dtype())) {
    auto new_tensor = TensorWrapper(other.tensor_.scaling_mode());
    auto my_rowwise_data = tensor_.get_rowwise_data();
    new_tensor.set_rowwise_data(my_rowwise_data.data_ptr,
                                static_cast<DType>(my_rowwise_data.dtype),
                                my_rowwise_data.shape);
    auto my_columnwise_data = tensor_.get_columnwise_data();
    new_tensor.set_columnwise_data(my_columnwise_data.data_ptr,
                                   static_cast<DType>(my_columnwise_data.dtype),
                                   my_columnwise_data.shape);
    auto other_amax = other.tensor_.get_amax();
    new_tensor.set_amax(other_amax.data_ptr,
                        static_cast<DType>(other_amax.dtype),
                        other_amax.shape);
    auto other_scale = other.tensor_.get_scale();
    new_tensor.set_scale(other_scale.data_ptr,
                         static_cast<DType>(other_scale.dtype),
                         other_scale.shape);
    auto other_row_scale_inv = other.tensor_.get_rowwise_scale_inv();
    new_tensor.set_rowwise_scale_inv(other_row_scale_inv.data_ptr,
                                     static_cast<DType>(other_row_scale_inv.dtype),
                                     other_row_scale_inv.shape);
    auto other_col_scale_inv = other.tensor_.get_columnwise_scale_inv();
    new_tensor.set_columnwise_scale_inv(other_col_scale_inv.data_ptr,
                                        static_cast<DType>(other_col_scale_inv.dtype),
                                        other_col_scale_inv.shape);
    tensor_ = std::move(new_tensor);
    to_cpu();
  }
}

using std::to_string;

template <typename T>
std::string to_string(const std::vector<T> &v) {
  std::string s = "[";
  for (const auto x : v) {
    s += to_string(x) + ", ";
  }
  s.pop_back();
  s.pop_back();
  return s + "]";
}

std::vector<size_t> unravel(const size_t i, const NVTEShape &shape) {
  std::vector<size_t> ret;
  size_t current_i = i;
  for (size_t current = shape.ndim - 1;
       current > 0;
       --current) {
    ret.push_back(current_i % shape.data[current]);
    current_i /= shape.data[current];
  }
  ret.push_back(current_i);
  std::reverse(ret.begin(), ret.end());
  return ret;
}

void compareResults_sequential(const std::string &name, const Tensor &test,
                               const void *ref, const bool rowwise,
                               double atol, double rtol, bool if_on_gpus) {
  if (if_on_gpus) test.to_cpu();
  const auto& shape = rowwise ? test.rowwise_shape() : test.columnwise_shape();
  const size_t N = product(shape);
  TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(test.dtype(), T,
    const T *test_data = rowwise ? test.rowwise_cpu_dptr<T>() : test.columnwise_cpu_dptr<T>();
    const T *ref_data = reinterpret_cast<const T*>(ref);
    for (size_t i = 0; i < N; ++i) {
      double t = static_cast<double>(test_data[i]);
      double r = static_cast<double>(ref_data[i]);
      bool mismatch = fabs(t - r) > atol && (r == 0 || fabs((t - r) / r) > rtol);
      /* For Float32 the floating point comparison is enough to error out */
      bool assertion = mismatch && test.dtype() == DType::kFloat32;
      if (mismatch && !assertion) {
        /* Check if it is just a failure of round to nearest choosing different
           side of the real value */
        const double mean = (t + r) / 2;
        const double mean_p = mean >= 0 ? mean * (1 + 1e-6) : mean * (1 - 1e-6);
        const double mean_m = mean >= 0 ? mean * (1 - 1e-6) : mean * (1 + 1e-6);
        const double cast_mean_p = static_cast<double>(static_cast<T>(mean_p));
        const double cast_mean_m = static_cast<double>(static_cast<T>(mean_m));
        assertion = !(cast_mean_m == std::min(t,r) && cast_mean_p == std::max(t,r));
      }
      std::string direction = rowwise ? "rowwise" : "columnwise";
      ASSERT_FALSE(assertion) << "Error in tensor " << name << " in "
                              << direction << " direction." << std::endl
                              << "Mismatch at place " << to_string(unravel(i, shape))
                              << " (" << std::to_string(i) << "): " << t << " vs " << r;
    }
  );
}

template <typename T>
static size_t getFirstMismatchIdx(const DType data_type, const T* test_data, const T* ref_data,
                                  const size_t N, const double atol, const double rtol) {
  int first_mismatch_idx = N;

  bool is_mismatch_found = false;
  #pragma omp parallel for schedule(static) firstprivate(is_mismatch_found) \
    reduction(min: first_mismatch_idx) proc_bind(spread)
  for (size_t i = 0; i < N; ++i) {
    if (is_mismatch_found) {    // early escape of the omp thread
      continue;
    }

    double t = static_cast<double>(test_data[i]);
    double r = static_cast<double>(ref_data[i]);

    bool mismatch = fabs(t - r) > atol && (r == 0 || fabs((t - r) / r) > rtol);
    /* For Float32 the floating point comparison is enough to error out */
    bool assertion = mismatch && (data_type == DType::kFloat32);
    if (mismatch && !assertion) {
      /* Check if it is just a failure of round to nearest choosing different
          side of the real value */
      const double mean = (t + r) / 2;
      const double mean_p = mean >= 0 ? mean * (1 + 1e-6) : mean * (1 - 1e-6);
      const double mean_m = mean >= 0 ? mean * (1 - 1e-6) : mean * (1 + 1e-6);
      const double cast_mean_p = static_cast<double>(static_cast<T>(mean_p));
      const double cast_mean_m = static_cast<double>(static_cast<T>(mean_m));
      assertion = !(cast_mean_m == std::min(t,r) && cast_mean_p == std::max(t,r));
    }
    if (assertion && i < first_mismatch_idx) {
      first_mismatch_idx = i;
      is_mismatch_found = true;
    }
  }
  return first_mismatch_idx;
}

void compareResults_parallel(const std::string &name, const Tensor &test, const void *ref,
                             const bool rowwise, double atol, double rtol, bool if_on_gpus) {
  if (if_on_gpus) test.to_cpu();
  const auto& shape = rowwise ? test.rowwise_shape() : test.columnwise_shape();
  const size_t N = product(shape);
  TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(test.dtype(), T,
    const T *test_data = rowwise ? test.rowwise_cpu_dptr<T>() : test.columnwise_cpu_dptr<T>();
    const T *ref_data = reinterpret_cast<const T*>(ref);

    const size_t i = getFirstMismatchIdx<T>(test.dtype(), test_data, ref_data, N, atol, rtol);
    if (i != N) {
      const double t = static_cast<double>(test_data[i]);
      const double r = static_cast<double>(ref_data[i]);
      std::string direction = rowwise ? "rowwise" : "columnwise";
      ASSERT_FALSE(true) << "Error in tensor " << name << " in "
                         << direction << " direction." << std::endl
                         << "Mismatch at place " << to_string(unravel(i, shape))
                         << " (" << std::to_string(i) << "): " << t << " vs " << r;
    }
  );
}

void compareResults(const std::string &name, const Tensor &test, const void *ref,
                    const bool rowwise, double atol, double rtol, bool if_on_gpus) {
  constexpr bool sequential = false;
  if constexpr (sequential) {
    compareResults_sequential(name, test, ref, rowwise, atol, rtol, if_on_gpus);
  } else {
    compareResults_parallel(name, test, ref, rowwise, atol, rtol, if_on_gpus);
  }
}

void compareResults(const std::string &name, const float test, const float ref,
                    double atol, double rtol) {
  double t = static_cast<double>(test);
  double r = static_cast<double>(ref);
  bool mismatch = fabs(t - r) > atol && (r == 0 || fabs((t - r) / r) > rtol);
  ASSERT_FALSE(mismatch) << "Error in " << name << std::endl
                         << "Mismatch: " << t << " vs " << r;

}


void compareResults(const std::string &name, const uint8_t *test, const uint8_t *ref,
                    size_t N, float mismatch_rate_tol) {
  size_t max_mismatches = std::ceil(N * mismatch_rate_tol);
  size_t n_mismatches = 0;
  std::vector<size_t> mismatch_indices;
  for (int i = 0; i < N; i++){
    bool mismatch = test[i] != ref[i];
    if (mismatch){
      n_mismatches++;
      mismatch_indices.push_back(i);
    }
    if (n_mismatches > max_mismatches){
      std::cout << "Error in " << name << std::endl;
      for (auto &index : mismatch_indices)
        std::cout << "Mismatch at (" << index << "):" << static_cast<int>(test[i]) << " vs "
        << static_cast<int>(ref[i]) << std::endl;
      GTEST_FAIL() << n_mismatches << " mismatche(s) which is more than mismatch tol.";
    }
  }
}

void compare_e8m0_scaling_factors(const std::string &name, const uint8_t *test, const uint8_t *ref,
                                  const size_t row_blocks, const size_t col_blocks, const size_t stride)
{
  for (int i = 0; i < row_blocks; ++i) {
    for (int j = 0; j < col_blocks; ++j) {
      const int idx = i * stride + j;
      ASSERT_FALSE(test[idx] != ref[idx]) << "Error in " << name << std::endl
        << "Mismatch: " << static_cast<int>(test[idx]) << " vs "
        << static_cast<int>(ref[idx]) << " at index " << idx;
    }
  }
}

void compare_e8m0_scaling_factors(const std::string &name, const uint8_t *test, const uint8_t *ref,
                                  const size_t N)
{
  for (int i = 0; i < N; i++) {
    ASSERT_FALSE(test[i] != ref[i]) << "Error in " << name << std::endl
      << "Mismatch: " << static_cast<int>(test[i]) << " vs "
      << static_cast<int>(ref[i]) << " at index " << i;
  }
}

std::pair<double, double> getTolerances(const DType type) {
  switch(type) {
    case DType::kFloat32:
      return {1e-6, 5e-6};
    case DType::kFloat16:
      return {1e-5, 1e-3};
    case DType::kBFloat16:
      return {1e-5, 1e-2};
    case DType::kFloat8E4M3:
    case DType::kFloat8E5M2:
    case DType::kFloat8E8M0:
      return {1e-2, 1e-2};
    default:
      NVTE_CHECK("Invalid type!");
  }
  return {0, 0};
}

template <typename T>
void generate_data_uniformly(T* data, const size_t size, std::mt19937* gen) {
  #pragma omp parallel proc_bind(spread)
  {
    std::mt19937 gen_local = *gen;
    gen_local.discard(omp_get_thread_num() * 599);
    std::uniform_real_distribution<> dis(-2.0, 1.0);
    #pragma omp for schedule(static)
    for (size_t i = 0; i < size; ++i) {
      data[i] = static_cast<T>(dis(gen_local));
    }
  }
  gen->discard(size);
}

void fillUniform(Tensor *t) {
  if (t->rowwise()) {
    const size_t size = product(t->rowwise_shape());
    TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(t->dtype(), T,
      {
        T *data = t->rowwise_cpu_dptr<T>();
        generate_data_uniformly(data, size, &(t->gen()));
      }
    );
  } else {
    const size_t size = product(t->columnwise_shape());
    TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(t->dtype(), T,
      {
        T *data = t->columnwise_cpu_dptr<T>();
        generate_data_uniformly(data, size, &(t->gen()));
      }
    );
  }
  std::uniform_real_distribution<> dis(-2.0, 1.0);
  t->set_scale_inv(dis(t->gen()));
  t->from_cpu();
}

template<typename InputEncoding, InputsFillCase Case>
void fillCase_special(Tensor *t) {
  const size_t size = product(t->rowwise_shape());
  const size_t rows = t->rowwise_shape().data[0];
  const size_t cols = t->rowwise_shape().data[1];

  if constexpr (Case == InputsFillCase::zeros) {
    TRANSFORMER_ENGINE_TYPE_SWITCH_FP16_FP32_ONLY(t->dtype(), InputType, {
      InputType *data = t->rowwise_cpu_dptr<InputType>();
      for (size_t i = 0; i < size; ++i) {
        data[i] = static_cast<InputType>(0);
      }
    });
  } else {
    double minAbs = -2.0;
    double maxAbs =  1.0;
    if constexpr (Case != InputsFillCase::uniform) {
      minAbs = Quantized_Limits<InputEncoding>::ranges[Case];
      maxAbs = Quantized_Limits<InputEncoding>::ranges[Case + 1];
    }
    std::uniform_real_distribution<> dis(minAbs, maxAbs);
    std::uniform_real_distribution<> dis_sign(-1.0, 1.0);
    TRANSFORMER_ENGINE_TYPE_SWITCH_FP16_FP32_ONLY(t->dtype(), InputType, {
      InputType *data = t->rowwise_cpu_dptr<InputType>();
      for (size_t i = 0; i < rows; ++i) {
        for (size_t j = 0; j < cols; ++j) {
          const size_t idx = i * cols + j;
          const bool is_negative = (dis_sign(t->gen()) < 0.0);
          double val = dis(t->gen());
          if (is_negative) {
            val = -val;
          }
          data[idx] = static_cast<InputType>(val);
        }
      }
    });
  }
  t->set_scale_inv(1.0);
  t->from_cpu();
}

template <typename InputEncoding>
void fillCase(Tensor *t, const InputsFillCase fill_case) {
  switch (fill_case) {
    case InputsFillCase::uniform:
        fillCase_special<InputEncoding, InputsFillCase::uniform>(t); break;
    case InputsFillCase::zeros:
        fillCase_special<InputEncoding, InputsFillCase::zeros>(t); break;
    case InputsFillCase::zero_to_minNorm:
        fillCase_special<InputEncoding, InputsFillCase::zero_to_minNorm>(t); break;
    case InputsFillCase::minNorm_to_maxNorm:
        fillCase_special<InputEncoding, InputsFillCase::minNorm_to_maxNorm>(t); break;
    case InputsFillCase::maxNorm_to_inf:
        fillCase_special<InputEncoding, InputsFillCase::maxNorm_to_inf>(t); break;
  }
}

template void fillCase<fp8e4m3>(Tensor *t, const InputsFillCase fill_case);
template void fillCase<fp8e5m2>(Tensor *t, const InputsFillCase fill_case);
template void fillCase<fp32>(Tensor *t, const InputsFillCase fill_case);

void setRandomScale(Tensor *t) {
  std::uniform_real_distribution<> dis(-2.0, 1.0);
  const float scale = dis(t->gen());
  t->set_scale(scale);
}

void setRandomScaleInv(Tensor *t) {
  std::uniform_real_distribution<> dis(-2.0, 1.0);
  const float scale_inv = dis(t->gen());
  t->set_scale_inv(scale_inv);
}

bool isFp8Type(DType type) {
    return type == DType::kFloat8E4M3 || type == DType::kFloat8E5M2 || type == DType::kFloat8E8M0;
}

int32_t getDeviceComputeCapability()
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    return 10 * deviceProp.major + deviceProp.minor;
}

size_t first_dimension(const std::vector<size_t> &shape) {
  if (shape.size() == 0) return 1;
  if (shape.size() == 1) return 1;
  return product(shape, 0, shape.size() - 1);
}

size_t last_dimension(const std::vector<size_t> &shape) {
  if (shape.size() == 0) return 1;
  return shape[shape.size() - 1];
}

std::array<size_t, 4> get_scale_tensor_dims(const size_t rows,
                                            const size_t cols,
                                            const size_t block_size_rows,
                                            const size_t block_size_cols) {
    const bool is_rowwise = (block_size_rows == 1) && (block_size_cols == 32);

    const size_t alignment_Y = is_rowwise
                               ? scale_tensor_alignment_Y_rowwise
                               : scale_tensor_alignment_Y_colwise;
    const size_t alignment_X = is_rowwise
                               ? scale_tensor_alignment_X_rowwise
                               : scale_tensor_alignment_X_colwise;

    const size_t unpadded_blocks_Y = divide_round_up(rows, block_size_rows);
    const size_t unpadded_blocks_X = divide_round_up(cols, block_size_cols);

    const size_t blocks_Y = round_up_to_nearest_multiple(unpadded_blocks_Y, alignment_Y);
    const size_t blocks_X = round_up_to_nearest_multiple(unpadded_blocks_X, alignment_X);
    return {unpadded_blocks_Y, unpadded_blocks_X, blocks_Y, blocks_X};
}

}  // namespace test
