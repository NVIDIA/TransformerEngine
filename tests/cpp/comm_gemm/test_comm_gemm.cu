#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <mpi.h>
#include <transformer_engine/comm_gemm.h>
#include <transformer_engine/gemm.h>
#include <transformer_engine/transformer_engine.h>

#include <iostream>
#include <limits>
#include <random>
#include <sstream>
#include <string>
#include <vector>

#include "common.h"

using transformer_engine::DType;
using transformer_engine::SimpleTensor;
using transformer_engine::Tensor;
using transformer_engine::TypeInfo;

#define CHECK_MPI(expr)                                              \
  do {                                                               \
    int err = (expr);                                                \
    if (err != MPI_SUCCESS) {                                        \
      char err_str[MPI_MAX_ERROR_STRING + 1]{};                      \
      int _len{};                                                    \
      MPI_Error_string(err, err_str, &_len);                         \
      EXPECT_TRUE(false) << "MPI error: " << err << ": " << err_str; \
    }                                                                \
  } while (false)

int main(int argc, char* argv[]) {
  ::testing::InitGoogleTest(&argc, argv);
  CHECK_MPI(MPI_Init(&argc, &argv));
  auto ret = RUN_ALL_TESTS();
  CHECK_MPI(MPI_Finalize());
  return ret;
}

template <typename T>
std::vector<T> CopyMatrix(const std::vector<T>& data, size_t mstart, size_t nstart, size_t msize,
                          size_t nsize, size_t ld) {
  std::vector<T> ret(msize * nsize);
  size_t dst = 0;
  for (size_t j = nstart; j < nstart + nsize; ++j) {
    for (size_t i = mstart; i < mstart + msize; ++i) {
      ret[dst++] = data[j * ld + i];
    }
  }
  return ret;
}

template <typename T>
SimpleTensor MakeSimple(const std::vector<size_t> dims, const std::vector<T>& values) {
  void* dptr{};
  NVTE_CHECK_CUDA(hipMalloc(&dptr, values.size() * sizeof values[0]));
  NVTE_CHECK_CUDA(
      hipMemcpy(dptr, values.data(), values.size() * sizeof values[0], hipMemcpyDefault));
  return {dptr, dims, TypeInfo<T>::dtype};
}

struct TensorHolder {
  template <typename T>
  static TensorHolder Make(size_t m, size_t n, float scale) {
    TensorHolder ret;
    ret.t.data = MakeSimple({n, m}, std::vector<T>(m * n));
    ret.t.amax = MakeSimple<float>({1}, {0.0f});
    ret.t.scale = MakeSimple<float>({1}, {scale});
    ret.t.scale_inv = MakeSimple<float>({1}, {1.0f / scale});
    return ret;
  }

  template <typename T>
  static TensorHolder MakeFromData(const std::vector<T>& data, size_t mstart, size_t nstart,
                                   size_t msize, size_t nsize, size_t ld, float scale) {
    auto values = CopyMatrix(data, mstart, nstart, msize, nsize, ld);
    TensorHolder ret;
    ret.t.data = MakeSimple({nsize, msize}, values);
    ret.t.amax = MakeSimple<float>({1}, {0.0f});
    ret.t.scale = MakeSimple<float>({1}, {scale});
    ret.t.scale_inv = MakeSimple<float>({1}, {1.0f / scale});
    return ret;
  }

  Tensor t;

  ~TensorHolder() {
    hipFree(t.data.dptr);
    hipFree(t.amax.dptr);
    hipFree(t.scale.dptr);
    hipFree(t.scale_inv.dptr);
  }
};

template <typename T>
float GetScale(float amax) {
  if constexpr (sizeof(T) > 1) return 1.0;
  return static_cast<float>(static_cast<T>(std::numeric_limits<float>::max())) / amax;
}

struct Params {
  DType a_type;
  DType b_type;
  DType d_type;
  bool transa;
  bool transb;
  size_t m;
  size_t n;
  size_t k;
  float tol;
};

class CommGemmFixure : public ::testing::TestWithParam<Params> {
 protected:
  CommGemmFixure() {
    CHECK_MPI(MPI_Comm_size(MPI_COMM_WORLD, &nranks_));
    CHECK_MPI(MPI_Comm_rank(MPI_COMM_WORLD, &rank_));
    int local_device = rank_;
    NVTE_CHECK_CUDA(hipSetDevice(rank_));
    ctx_ = nvte_comm_gemm_ctx_create(nranks_, rank_, local_device);
  }
  ~CommGemmFixure() { nvte_comm_gemm_ctx_destroy(ctx_); }

  struct PatternDims {
    int64_t a_rows_start;
    int64_t a_rows_num;
    int64_t a_cols_start;
    int64_t a_cols_num;
    int64_t b_rows_start;
    int64_t b_rows_num;
    int64_t b_cols_start;
    int64_t b_cols_num;
    int64_t d_rows_start;
    int64_t d_rows_num;
    int64_t d_cols_start;
    int64_t d_cols_num;
  };

  virtual PatternDims DistributeTensors(int64_t m, int64_t n, int64_t k) = 0;

  virtual void CommGemm(int64_t m, int64_t n, int64_t k, const NVTETensor a, const NVTETensor b,
                        const NVTETensor d, const NVTETensor bias, const NVTETensor pre_act_out,
                        bool transa, bool transb, bool grad, bool accumulate, int comm_sm_count,
                        hipStream_t stream) = 0;

  template <typename AType, typename BType, typename DType, typename BiasType>
  void Run(bool transa, bool transb, size_t m, size_t n, size_t k, float tol) {
    hipStream_t stream{};
    NVTE_CHECK_CUDA(hipStreamCreate(&stream));

    constexpr float MAX_IN = 1.0;
    std::mt19937 rng(12);
    std::uniform_real_distribution<float> dist(0.0, MAX_IN);

    float a_scale = GetScale<AType>(MAX_IN);
    float b_scale = GetScale<BType>(MAX_IN);
    float d_scale = GetScale<DType>(MAX_IN * MAX_IN * k);
    float bias_scale = GetScale<BiasType>(MAX_IN);

    std::vector<AType> adata(m * k);
    std::generate(adata.begin(), adata.end(),
                  [&rng, &dist, a_scale] { return static_cast<AType>(dist(rng) * a_scale); });
    std::vector<BType> bdata(k * n);
    std::generate(bdata.begin(), bdata.end(),
                  [&rng, &dist, b_scale] { return static_cast<BType>(dist(rng) * b_scale); });
    std::vector<BiasType> biasdata(m * n);
    std::generate(biasdata.begin(), biasdata.end(), [&rng, &dist, bias_scale] {
      return static_cast<BiasType>(dist(rng) * bias_scale);
    });

    auto ga = transa ? TensorHolder::MakeFromData<AType>(adata, 0, 0, k, m, k, a_scale)
                     : TensorHolder::MakeFromData<AType>(adata, 0, 0, m, k, m, a_scale);
    auto gb = transb ? TensorHolder::MakeFromData<BType>(bdata, 0, 0, n, k, n, b_scale)
                     : TensorHolder::MakeFromData<BType>(bdata, 0, 0, k, n, k, b_scale);
    auto gbias = TensorHolder::MakeFromData<BiasType>(biasdata, 0, 0, m, n, m, bias_scale);
    auto gd = TensorHolder::Make<DType>(m, n, d_scale);

    auto dims = DistributeTensors(m, n, k);
    auto a = transa
                 ? TensorHolder::MakeFromData<AType>(adata, dims.a_rows_start, dims.a_cols_start,
                                                     dims.a_rows_num, dims.a_cols_num, k, a_scale)
                 : TensorHolder::MakeFromData<AType>(adata, dims.a_cols_start, dims.a_rows_start,
                                                     dims.a_cols_num, dims.a_rows_num, m, a_scale);
    auto b = transb
                 ? TensorHolder::MakeFromData<BType>(bdata, dims.b_cols_start, dims.b_rows_start,
                                                     dims.b_cols_num, dims.b_rows_num, n, b_scale)
                 : TensorHolder::MakeFromData<BType>(bdata, dims.b_rows_start, dims.b_cols_start,
                                                     dims.b_rows_num, dims.b_cols_num, k, b_scale);
    auto bias =
        TensorHolder::MakeFromData<BiasType>(biasdata, dims.d_rows_start, dims.d_cols_start,
                                             dims.d_rows_num, dims.d_cols_num, m, bias_scale);
    auto d = TensorHolder::Make<DType>(dims.d_rows_num, dims.d_cols_num, d_scale);

    Tensor pre_act_out;
    bool grad = false;
    bool accumulate = false;
    CommGemm(m, n, k, &a.t, &b.t, &d.t, &bias, &pre_act_out, transa, transb, grad, accumulate,
             0 /*comm_sm_count*/, stream);
    auto workspace = TensorHolder::Make<uint8_t>(1, 32 << 20, 1.0);
    nvte_cublas_gemm(&ga.t, &gb.t, &gd.t, &bias, &pre_act_out, transa, transb, grad, &workspace.t,
                     accumulate, false /* use_split_accumulator */, 0 /* math_sm_count */, stream);
    NVTE_CHECK_CUDA(hipStreamSynchronize(stream));
    NVTE_CHECK_CUDA(hipStreamDestroy(stream));
    std::vector<DType> out(dims.d_rows_num * dims.d_cols_num);
    NVTE_CHECK_CUDA(
        hipMemcpy(out.data(), d.t.data.dptr, out.size() * sizeof out[0], hipMemcpyDefault));
    std::vector<DType> out_golden_global(m * n);
    NVTE_CHECK_CUDA(hipMemcpy(out_golden_global.data(), gd.t.data.dptr,
                               out_golden_global.size() * sizeof out_golden_global[0],
                               hipMemcpyDefault));

    auto out_golden = CopyMatrix(out_golden_global, dims.d_rows_start, dims.d_cols_start,
                                 dims.d_rows_num, dims.d_cols_num, m);
    NVTE_CHECK(out.size() == out_golden.size());
    for (size_t i = 0; i < out.size(); ++i) {
      if (rand() % 100 < 3) {
        // std::cerr << "== at " << rank_ << ": " << i << ": " << static_cast<float>(out[i]) << " "
        //           << static_cast<float>(out_golden[i]) << std::endl;
      }
      EXPECT_NEAR(static_cast<float>(out[i]), static_cast<float>(out_golden[i]), tol * k);
    }
  }

  CommGemmCtx* ctx_{};
  int nranks_{};
  int rank_{};
};

struct AgGemm : public CommGemmFixure {
  PatternDims DistributeTensors(int64_t m, int64_t n, int64_t k) override {
    auto a_cols_num = nvte_comm_gemm_numroc(ctx_, m);
    auto b_cols_num = nvte_comm_gemm_numroc(ctx_, n);

    int64_t a_cols_start{};
    int64_t b_cols_start{};
    MPI_Exscan(&a_cols_num, &a_cols_start, 1, MPI_INT64_T, MPI_SUM, MPI_COMM_WORLD);
    MPI_Exscan(&b_cols_num, &b_cols_start, 1, MPI_INT64_T, MPI_SUM, MPI_COMM_WORLD);

    return PatternDims{
        .a_rows_start = 0,
        .a_rows_num = k,
        .a_cols_start = a_cols_start,
        .a_cols_num = a_cols_num,
        .b_rows_start = 0,
        .b_rows_num = k,
        .b_cols_start = b_cols_start,
        .b_cols_num = b_cols_num,
        .d_rows_start = a_cols_start,
        .d_rows_num = a_cols_num,
        .d_cols_start = 0,
        .d_cols_num = n,
    };
  }

  void CommGemm(int64_t m, int64_t n, int64_t k, const NVTETensor a, const NVTETensor b,
                const NVTETensor d, const NVTETensor bias, const NVTETensor pre_act_out,
                bool transa, bool transb, bool grad, bool accumulate, int comm_sm_count,
                hipStream_t stream) override {
    nvte_all_gather_gemm(ctx_, m, n, k, a, b, d, bias, pre_act_out, transa, transb, grad,
                         accumulate, comm_sm_count, stream);
  }
};

struct GemmRs : public CommGemmFixure {
  PatternDims DistributeTensors(int64_t m, int64_t n, int64_t k) override {
    auto rows_num = nvte_comm_gemm_numroc(ctx_, k);
    auto d_cols_num = nvte_comm_gemm_numroc(ctx_, n);

    int64_t rows_start{};
    int64_t d_cols_start{};
    MPI_Exscan(&rows_num, &rows_start, 1, MPI_INT64_T, MPI_SUM, MPI_COMM_WORLD);
    MPI_Exscan(&d_cols_num, &d_cols_start, 1, MPI_INT64_T, MPI_SUM, MPI_COMM_WORLD);

    return PatternDims{
        .a_rows_start = rows_start,
        .a_rows_num = rows_num,
        .a_cols_start = 0,
        .a_cols_num = m,
        .b_rows_start = rows_start,
        .b_rows_num = rows_num,
        .b_cols_start = 0,
        .b_cols_num = n,
        .d_rows_start = 0,
        .d_rows_num = m,
        .d_cols_start = d_cols_start,
        .d_cols_num = d_cols_num,
    };
  }

  void CommGemm(int64_t m, int64_t n, int64_t k, const NVTETensor a, const NVTETensor b,
                const NVTETensor d, const NVTETensor bias, const NVTETensor pre_act_out,
                bool transa, bool transb, bool grad, bool accumulate, int comm_sm_count,
                hipStream_t stream) override {
    nvte_gemm_reduce_scatter(ctx_, m, n, k, a, b, d, bias, pre_act_out, transa, transb, grad,
                             accumulate, comm_sm_count, stream);
  }
};

struct GemmAr : public CommGemmFixure {
  PatternDims DistributeTensors(int64_t m, int64_t n, int64_t k) override {
    auto rows_num = nvte_comm_gemm_numroc(ctx_, k);

    int64_t rows_start{};
    MPI_Exscan(&rows_num, &rows_start, 1, MPI_INT64_T, MPI_SUM, MPI_COMM_WORLD);

    return PatternDims{
        .a_rows_start = rows_start,
        .a_rows_num = rows_num,
        .a_cols_start = 0,
        .a_cols_num = m,
        .b_rows_start = rows_start,
        .b_rows_num = rows_num,
        .b_cols_start = 0,
        .b_cols_num = n,
        .d_rows_start = 0,
        .d_rows_num = m,
        .d_cols_start = 0,
        .d_cols_num = n,
    };
  }

  void CommGemm(int64_t m, int64_t n, int64_t k, const NVTETensor a, const NVTETensor b,
                const NVTETensor d, const NVTETensor bias, const NVTETensor pre_act_out,
                bool transa, bool transb, bool grad, bool accumulate, int comm_sm_count,
                hipStream_t stream) override {
    nvte_gemm_all_reduce(ctx_, m, n, k, a, b, d, bias, pre_act_out, transa, transb, grad,
                         accumulate, comm_sm_count, stream);
  }
};

TEST_P(AgGemm, Gemm) {
  auto [a_type, b_type, d_type, transa, transb, m, n, k, tol] = GetParam();
  TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(
      a_type, AType,
      TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(
          b_type, BType,
          TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(
              d_type, DType, Run<AType, BType, DType, DType>(transa, transb, m, n, k, tol);)));
}

TEST_P(GemmRs, Gemm) {
  auto [a_type, b_type, d_type, transa, transb, m, n, k, tol] = GetParam();
  TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(
      a_type, AType,
      TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(
          b_type, BType,
          TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(
              d_type, DType, Run<AType, BType, DType, DType>(transa, transb, m, n, k, tol);)));
}

TEST_P(GemmAr, Gemm) {
  auto [a_type, b_type, d_type, transa, transb, m, n, k, tol] = GetParam();
  TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(
      a_type, AType,
      TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(
          b_type, BType,
          TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(
              d_type, DType, Run<AType, BType, DType, DType>(transa, transb, m, n, k, tol);)));
}

std::string ParamSuffix(const testing::TestParamInfo<Params>& info) {
  const auto [a_type, b_type, d_type, transa, transb, m, n, k, _tol] = info.param;
  std::ostringstream ss;
  ss << to_string(a_type) << "_" << to_string(b_type) << "_" << to_string(d_type) << "_"
     << "_" << (transa ? "T" : "N") << (transb ? "T" : "N") << "_" << m << "x" << n << "x" << k;
  return ss.str();
}

INSTANTIATE_TEST_SUITE_P(AgGemm, AgGemm,
                         testing::Values(Params{DType::kFloat16, DType::kFloat16, DType::kFloat16,
                                                false, false, 256, 128, 64, 5e-2},
                                         Params{DType::kFloat16, DType::kFloat16, DType::kFloat16,
                                                false, true, 256, 128, 64, 5e-2},
                                         Params{DType::kFloat16, DType::kFloat16, DType::kFloat16,
                                                true, false, 256, 128, 64, 5e-2},
                                         Params{DType::kBFloat16, DType::kBFloat16,
                                                DType::kBFloat16, false, false, 256, 128, 64, 5e-2},
                                         Params{DType::kBFloat16, DType::kBFloat16,
                                                DType::kBFloat16, false, true, 256, 128, 64, 5e-2},
                                         Params{DType::kBFloat16, DType::kBFloat16,
                                                DType::kBFloat16, true, false, 256, 128, 64, 5e-2},
                                         Params{DType::kFloat8E4M3, DType::kFloat8E4M3,
                                                DType::kFloat16, true, false, 256, 128, 64, 5e-2},
                                         Params{DType::kFloat8E4M3, DType::kFloat8E5M2,
                                                DType::kFloat16, true, false, 256, 128, 64, 5e-2},
                                         Params{DType::kFloat8E5M2, DType::kFloat8E4M3,
                                                DType::kFloat16, true, false, 256, 128, 64, 5e-2}),
                         &ParamSuffix);

INSTANTIATE_TEST_SUITE_P(GemmRs, GemmRs,
                         testing::Values(Params{DType::kFloat16, DType::kFloat16, DType::kFloat16,
                                                false, false, 64, 128, 256, 5e-2},
                                         Params{DType::kFloat16, DType::kFloat16, DType::kFloat16,
                                                false, true, 64, 128, 256, 5e-1},
                                         Params{DType::kFloat16, DType::kFloat16, DType::kFloat16,
                                                true, false, 64, 128, 256, 5e-2},
                                         Params{DType::kBFloat16, DType::kBFloat16,
                                                DType::kBFloat16, false, false, 64, 128, 256, 5e-1},
                                         Params{DType::kBFloat16, DType::kBFloat16,
                                                DType::kBFloat16, false, true, 64, 128, 256, 5e-1},
                                         Params{DType::kBFloat16, DType::kBFloat16,
                                                DType::kBFloat16, true, false, 64, 128, 256, 5e-1},
                                         Params{DType::kFloat8E4M3, DType::kFloat8E4M3,
                                                DType::kFloat16, true, false, 64, 128, 256, 5e-2},
                                         Params{DType::kFloat8E4M3, DType::kFloat8E5M2,
                                                DType::kFloat16, true, false, 64, 128, 256, 5e-2},
                                         Params{DType::kFloat8E5M2, DType::kFloat8E4M3,
                                                DType::kFloat16, true, false, 64, 128, 256, 5e-2}),
                         &ParamSuffix);

INSTANTIATE_TEST_SUITE_P(
    GemmAr, GemmAr,
    testing::Values(Params{DType::kFloat16, DType::kFloat16, DType::kFloat16, false, false, 64,
                           64 * 4, 64 * 4, 1e-4},
                    Params{DType::kFloat16, DType::kFloat16, DType::kFloat16, false, true, 64,
                           64 * 4, 64 * 4, 1e-4},
                    Params{DType::kFloat16, DType::kFloat16, DType::kFloat16, true, false, 64,
                           64 * 4, 64 * 4, 1e-4},
                    Params{DType::kBFloat16, DType::kBFloat16, DType::kBFloat16, false, false, 64,
                           64 * 4, 64 * 4, 1e-4},
                    Params{DType::kBFloat16, DType::kBFloat16, DType::kBFloat16, false, true, 64,
                           64 * 4, 64 * 4, 1e-4},
                    Params{DType::kBFloat16, DType::kBFloat16, DType::kBFloat16, true, false, 64,
                           64 * 4, 64 * 4, 1e-4},
                    Params{DType::kFloat8E4M3, DType::kFloat8E4M3, DType::kBFloat16, true, false,
                           64, 64 * 4, 64 * 4, 1e-4}),
    &ParamSuffix);
