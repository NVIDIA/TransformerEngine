/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <cstring>
#include <iomanip>
#include <iostream>
#include <memory>
#include <random>
#include <vector>
#include <cstdio>

#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <transformer_engine/padding.h>
#include "../test_common.h"

using namespace transformer_engine;

namespace {

template <typename InputType, typename OutputType>
void compute_ref(const std::vector<std::vector<InputType>>& input_list,
                 std::vector<std::vector<OutputType>>& output_list,
                 const std::vector<size_t>& height_list,
                 const std::vector<size_t>& width_list,
                 const std::vector<int>& padded_height_list) {
  using compute_t = float;
  for (size_t tensor_id = 0; tensor_id < input_list.size(); ++tensor_id) {
    const auto& input = input_list[tensor_id];
    auto& output = output_list[tensor_id];
    const size_t height = height_list[tensor_id];
    const size_t width = width_list[tensor_id];
    const size_t padded_height = padded_height_list[tensor_id];

    for (size_t i = 0; i < padded_height; ++i) {
      if (i < height) {
        for (size_t j = 0; j < width; ++j) {
          const compute_t x = static_cast<compute_t>(input[i * width + j]);
          const OutputType y = static_cast<OutputType>(x);
          output[i * width + j] = y;
        }
      } else {
        for (size_t j = 0; j < width; ++j) {
          output[i * width + j] = static_cast<OutputType>(0.f);
        }
      }
    }
  }
}

template <typename InputType, typename OutputType>
void performTest() {
  using namespace test;

  const DType itype = TypeInfo<InputType>::dtype;
  const DType otype = TypeInfo<OutputType>::dtype;
  const std::vector<std::pair<size_t, size_t>> tensor_dims = {{1,1},
                                                              {1,768},
                                                              {768,1},
                                                              {768,768},
                                                              {43,43},
                                                              {43,256},
                                                              {256,43},
                                                              {256,256}};
  const size_t num_tensors = tensor_dims.size();
  constexpr int align = 16;

  // Buffers for Transformer Engine implementation
  std::vector<Tensor> input_list, output_list, output_t_list;

  // Buffers for reference implementation
  std::vector<std::vector<InputType>> ref_input_list;
  std::vector<std::vector<OutputType>> ref_output_list;
  std::vector<size_t> ref_height_list(num_tensors), ref_width_list(num_tensors);
  std::vector<int> ref_padded_height_list(num_tensors);

  // Initialize buffers
  for (size_t tensor_id = 0; tensor_id < num_tensors; ++tensor_id) {
    const size_t height = tensor_dims[tensor_id].first;
    const size_t width = tensor_dims[tensor_id].second;
    const size_t padded_height = (height + align - 1) / align * align;
    input_list.emplace_back(Tensor({ height, width }, itype));
    output_list.emplace_back(Tensor({ padded_height, width }, otype));

    auto& input = input_list.back();
    auto& output = output_list.back();
    fillUniform(&input);
    setRandomScale(&output);

    ref_input_list.emplace_back(height*width);
    ref_output_list.emplace_back(padded_height*width);

    std::copy(input.cpu_dptr<InputType>(),
              input.cpu_dptr<InputType>() + height * width,
              ref_input_list.back().begin());
    ref_height_list[tensor_id] = height;
    ref_width_list[tensor_id] = width;
    ref_padded_height_list[tensor_id] = padded_height;
  }

  // Transformer Engine implementation
  auto make_nvte_vector = [](std::vector<Tensor>& tensor_list)
    -> std::vector<NVTETensor> {
    std::vector<NVTETensor> nvte_tensor_list;
    for (auto& tensor : tensor_list) {
      nvte_tensor_list.emplace_back(tensor.data());
    }
    return nvte_tensor_list;
  };
  nvte_multi_padding(num_tensors,
                                make_nvte_vector(input_list).data(),
                                make_nvte_vector(output_list).data(),
                                ref_padded_height_list.data(),
                                0);
  hipDeviceSynchronize();
  auto err = hipGetLastError();
  ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);

  // Reference implementation
  compute_ref<InputType, OutputType>(ref_input_list,
                                     ref_output_list,
                                     ref_height_list,
                                     ref_width_list,
                                     ref_padded_height_list);

  // Check correctness
  for (size_t tensor_id = 0; tensor_id < num_tensors; ++tensor_id) {
    auto [atol, rtol] = getTolerances(otype);
    compareResults("output",
                   output_list[tensor_id],
                   ref_output_list[tensor_id].data(),
                   atol, rtol);
  }
}

}  // namespace

class MultiPaddingTestSuite
  : public ::testing::TestWithParam<
                                               transformer_engine::DType> {};

TEST_P(MultiPaddingTestSuite, TestMultiPaddingTranspose) {
  using namespace transformer_engine;
  using namespace test;

  const DType input_type = GetParam();
  const DType output_type = input_type;

  TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(input_type, InputType,
    TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(output_type, OutputType,
      performTest<InputType, OutputType>();
    );
  );
}


INSTANTIATE_TEST_SUITE_P(
  OperatorTest,
  MultiPaddingTestSuite,
  ::testing::ValuesIn(test::all_fp_types),
  [](const testing::TestParamInfo<MultiPaddingTestSuite::ParamType>& info) {
    std::string name = test::typeName(info.param);
    return name;
  });
