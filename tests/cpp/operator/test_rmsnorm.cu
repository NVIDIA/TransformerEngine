#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <transformer_engine/rmsnorm.h>
#include <transformer_engine/transformer_engine.h>
#include <cmath>
#include <cstring>
#include <iomanip>
#include <iostream>
#include <memory>
#include <random>
#include "../test_common.h"

using namespace transformer_engine;
using namespace test;

namespace {

template <typename InputType>
void compute_ref_stats(const InputType *data, float *rsigma, const size_t N, const size_t H,
                       const double epsilon) {
  using compute_t = float;
  for (size_t i = 0; i < N; ++i) {
    compute_t sum = 0;
    for (size_t j = 0; j < H; ++j) {
      compute_t current = static_cast<compute_t>(data[i * H + j]);
      sum += (current) * (current);
    }
    sum = sum / H;
    compute_t rs = rsqrtf(sum + epsilon);
    rsigma[i] = rs;
  }
}

template <typename InputType, typename OutputType>
void compute_ref_output(const InputType *data, const InputType *gamma, OutputType *output,
                        const float *rsigma, const size_t N, const size_t H, float *amax,
                        float scale) {
  using compute_t = float;
  compute_t current_max = -1e100;
  for (size_t i = 0; i < N; ++i) {
    for (size_t j = 0; j < H; ++j) {
      compute_t current = static_cast<compute_t>(data[i * H + j]);
      compute_t tmp = current * rsigma[i] * static_cast<compute_t>(gamma[j]);
      output[i * H + j] = static_cast<OutputType>(tmp * scale);
      current_max = fmaxf(current_max, fabsf(tmp));
    }
  }
  *amax = current_max;
}

template <typename InputType, typename OutputType>
void compute_ref_backward(const OutputType *output_grad, const InputType *data, const float *rsigma,
                          const InputType *gamma, InputType *data_grad, InputType *gamma_grad,
                          const size_t N, const size_t H) {
  using compute_t = float;
  std::vector<compute_t> dgamma(H, 0.f);

  for (size_t i = 0; i < N; ++i) {
    // Reductions
    compute_t mdyy = 0;
    for (size_t j = 0; j < H; ++j) {
      const compute_t x = static_cast<compute_t>(data[i * H + j]);
      const compute_t y = x * rsigma[i];
      const compute_t g = static_cast<compute_t>(gamma[j]);
      const compute_t dz = static_cast<compute_t>(output_grad[i * H + j]);
      const compute_t dy = g * dz;
      dgamma[j] += y * dz;
      mdyy += dy * y;
    }
    mdyy /= H;

    // Input grads
    for (size_t j = 0; j < H; ++j) {
      const compute_t x = static_cast<compute_t>(data[i * H + j]);
      const compute_t y = x * rsigma[i];
      const compute_t g = static_cast<compute_t>(gamma[j]);
      const compute_t dz = static_cast<compute_t>(output_grad[i * H + j]);
      const compute_t dy = g * dz;
      const compute_t dx = rsigma[i] * (dy - mdyy * y);
      data_grad[i * H + j] = static_cast<InputType>(dx);
    }
  }

  // Weight grads
  for (size_t j = 0; j < H; ++j) {
    gamma_grad[j] = static_cast<InputType>(dgamma[j]);
  }
}

template <typename InputType, typename OutputType>
void performTest(const size_t N, const size_t H) {
  if (sizeof(InputType) < sizeof(OutputType)) {
    GTEST_SKIP() << "RMSNorm kernel does not support OutputType > InputType";
    return;
  }
  using WeightType = InputType;
  DType itype = TypeInfo<InputType>::dtype;
  DType wtype = TypeInfo<WeightType>::dtype;
  DType otype = TypeInfo<OutputType>::dtype;

  if ((itype == DType::kBFloat16 && otype == DType::kFloat16) ||
      (itype == DType::kFloat16 && otype == DType::kBFloat16)) {
    GTEST_SKIP() << "RMSNorm kernel does not support mixing Float16 and BFloat16";
    return;
  }

  Tensor input({N, H}, itype);
  Tensor z({N, H}, otype);
  Tensor gamma({H}, wtype);
  Tensor rsigma({N}, DType::kFloat32);
  Tensor dz({N, H}, wtype);
  Tensor dx({N, H}, itype);
  Tensor dgamma({H}, wtype);
  Tensor workspace, barrier, dgamma_part;

  fillUniform(&input);
  fillUniform(&gamma);
  fillUniform(&dz);
  setRandomScale(&z);

  std::unique_ptr<OutputType[]> ref_output = std::make_unique<OutputType[]>(N * H);
  std::unique_ptr<float[]> ref_rsigma = std::make_unique<float[]>(N);
  std::unique_ptr<InputType[]> ref_dx = std::make_unique<InputType[]>(N * H);
  std::unique_ptr<WeightType[]> ref_dgamma = std::make_unique<InputType[]>(H);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  // Forward kernel
  float epsilon = 1e-5;
  nvte_rmsnorm_fwd(input.data(), gamma.data(), epsilon, z.data(), rsigma.data(), 0,
                   prop.multiProcessorCount, workspace.data(), barrier.data());
  workspace = Tensor(workspace.shape(), workspace.dtype());
  barrier = Tensor(barrier.shape(), barrier.dtype());
  nvte_rmsnorm_fwd(input.data(), gamma.data(), epsilon, z.data(), rsigma.data(), 0,
                   prop.multiProcessorCount, workspace.data(), barrier.data());

  // Backward kernel
  nvte_rmsnorm_bwd(dz.data(), input.data(), rsigma.data(), gamma.data(), dx.data(), dgamma.data(),
                   dgamma_part.data(), 0, prop.multiProcessorCount, workspace.data(),
                   barrier.data());
  workspace = Tensor(workspace.shape(), workspace.dtype());
  barrier = Tensor(barrier.shape(), barrier.dtype());
  dgamma_part = Tensor(dgamma_part.shape(), dgamma_part.dtype());
  nvte_rmsnorm_bwd(dz.data(), input.data(), rsigma.data(), gamma.data(), dx.data(), dgamma.data(),
                   dgamma_part.data(), 0, prop.multiProcessorCount, workspace.data(),
                   barrier.data());

  // Reference implementations
  // use the GPU stats to tighten the tolerances
  rsigma.to_cpu();
  float ref_amax;
  compute_ref_stats(input.cpu_dptr<InputType>(), ref_rsigma.get(), N, H, epsilon);
  float ref_scale = isFp8Type(otype) ? z.scale() : 1.f;
  compute_ref_output(input.cpu_dptr<InputType>(), gamma.cpu_dptr<WeightType>(), ref_output.get(),
                     rsigma.cpu_dptr<float>(), N, H, &ref_amax, ref_scale);
  compute_ref_backward(dz.cpu_dptr<WeightType>(), input.cpu_dptr<InputType>(),
                       rsigma.cpu_dptr<float>(), gamma.cpu_dptr<WeightType>(), ref_dx.get(),
                       ref_dgamma.get(), N, H);

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);

  auto [atol_amax, rtol_amax] = getTolerances(DType::kFloat32);
  if (isFp8Type(otype)) {
    compareResults("amax", z.amax(), ref_amax, atol_amax, rtol_amax);
  }

  auto [atol_stats, rtol_stats] = getTolerances(DType::kFloat32);
  rtol_stats = 5e-5;
  compareResults("rsigma", rsigma, ref_rsigma.get(), atol_stats, rtol_stats);

  auto [atol, rtol] = getTolerances(otype);
  atol = 1e-8;
  compareResults("output", z, ref_output.get(), atol, rtol);

  double atol_bwd = 5e-6;
  double rtol_bwd = 1e-4;
  compareResults("dx", dx, ref_dx.get(), atol_bwd, rtol_bwd);
  compareResults("dgamma", dgamma, ref_dgamma.get(), atol_bwd, rtol_bwd);
}

std::vector<std::pair<size_t, size_t>> test_cases = {
    {2048, 4096}, {768, 2048}, {256, 1024}, {128, 768}, {64, 512}, {173, 409},  // Primes 40, 80
    {71, 3571},                                                                 // Primes 20, 500
    {29, 17389}};                                                               // Primes 10, 2000

}  // namespace

class RMSNormTestSuite
    : public ::testing::TestWithParam<std::tuple<
          transformer_engine::DType, transformer_engine::DType, std::pair<size_t, size_t>>> {};

TEST_P(RMSNormTestSuite, TestRMSNorm) {
  using namespace transformer_engine;
  using namespace test;

  const DType input_type = std::get<0>(GetParam());
  const DType output_type = std::get<1>(GetParam());
  const auto size = std::get<2>(GetParam());

  TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(
      input_type, InputType,
      TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(
          output_type, OutputType, performTest<InputType, OutputType>(size.first, size.second);););
}

INSTANTIATE_TEST_SUITE_P(OperatorTest, RMSNormTestSuite,
                         ::testing::Combine(::testing::Values(DType::kFloat32, DType::kBFloat16,
                                                              DType::kFloat16),
                                            ::testing::Values(DType::kFloat32, DType::kBFloat16,
                                                              DType::kFloat16, DType::kFloat8E4M3),
                                            ::testing::ValuesIn(test_cases)),
                         [](const testing::TestParamInfo<RMSNormTestSuite::ParamType> &info) {
                           std::string name = test::typeName(std::get<0>(info.param)) + "X" +
                                              test::typeName(std::get<1>(info.param)) + "X" +
                                              std::to_string(std::get<2>(info.param).first) + "X" +
                                              std::to_string(std::get<2>(info.param).second);
                           return name;
                         });
