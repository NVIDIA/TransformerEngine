#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <transformer_engine/softmax.h>
#include "../test_common.h"

using namespace transformer_engine;

namespace {

template <typename Type>
void compute_single_head_fwd(
  Type *softmax_out,
  const Type *data_in,
  const float scaling_factor,
  const int rows,
  const int cols)
{
  using compute_t = float;

  for (int i = 0; i < rows; ++i) {
    size_t offset = i * cols;

    const int masked_elements = i + cols - rows + 1;
    compute_t max_value = static_cast<compute_t>(-10'000.f);
    for (int j = 0; j < masked_elements; ++j) {
      compute_t tmp = scaling_factor * static_cast<compute_t>(data_in[offset + j]);
      softmax_out[offset + j] = static_cast<Type>(tmp);
      max_value = std::max(max_value, tmp);
    }

    compute_t accumulator = static_cast<compute_t>(0.f);
    for (int j = 0; j < masked_elements; ++j) {
      compute_t tmp = std::exp(static_cast<compute_t>(softmax_out[offset + j]) - max_value);
      softmax_out[offset + j] = static_cast<Type>(tmp);
      accumulator += tmp;
    }

    for (int j = 0; j < cols; ++j) {
      if (j < masked_elements) {
        compute_t tmp = static_cast<compute_t>(softmax_out[offset + j]) / accumulator;
        softmax_out[offset + j] = static_cast<Type>(tmp);
      } else {
        softmax_out[offset + j] = static_cast<Type>(0.f);
      }
    }
  }
}

template <typename Type>
void compute_single_head_bwd(
  Type *grad_out,
  const Type *grad_in,
  const Type *softmax_in,
  const float scaling_factor,
  const int batches,
  const int heads,
  const int rows,
  const int cols)
{
  using compute_t = float;

  for (int i = 0; i < rows; ++i) {
    size_t offset = i * cols;

    const int masked_elements = i + cols - rows + 1;
    compute_t accumulator = static_cast<compute_t>(0.f);
    for (int j = 0; j < masked_elements; ++j) {
      compute_t tmp = static_cast<compute_t>(softmax_in[offset + j])
                      * static_cast<compute_t>(grad_in[offset + j]);
      grad_out[offset + j] = static_cast<Type>(tmp);
      accumulator += tmp;
    }

    for (int j = 0; j < cols; ++j) {
      if (j < masked_elements) {
        compute_t tmp = static_cast<compute_t>(grad_out[offset + j])
                        - static_cast<compute_t>(softmax_in[offset + j]) * accumulator;
        grad_out[offset + j] = static_cast<Type>(scaling_factor * tmp);
      } else {
        grad_out[offset + j] = static_cast<Type>(0.f);
      }
    }
  }
}

template <typename Type>
void compute_fwd_ref(
  Type *softmax_out,
  const Type *data_in,
  const float scaling_factor,
  const int batches,
  const int heads,
  const int rows,
  const int cols)
{
  using compute_t = float;
  size_t head_size = rows * cols;
  size_t batch_size = heads * head_size;

  for (int b = 0; b < batches; ++b) {
    for (int h = 0; h < heads; ++h) {
      size_t offset = b * batch_size + h * head_size;
      compute_single_head_fwd(
          softmax_out + offset, data_in + offset, scaling_factor, rows, cols);
    }
  }
}

template <typename Type>
void compute_bwd_ref(
  Type *grad_out,
  const Type *grad_in,
  const Type *softmax_in,
  const float scaling_factor,
  const int batches,
  const int heads,
  const int rows,
  const int cols)
{
  using compute_t = float;
  size_t head_size = rows * cols;
  size_t batch_size = heads * head_size;

  for (int b = 0; b < batches; ++b) {
    for (int h = 0; h < heads; ++h) {
      size_t offset = b * batch_size + h * head_size;
      compute_single_head_bwd(grad_out + offset, grad_in + offset, softmax_in + offset,
                              scaling_factor, batches, heads, rows, cols);
    }
  }
}


// Query Sequence Length = rows
// Key Sequence Length = cols
template <typename Type>
void performTest(
  const size_t batches,
  const size_t heads,
  const size_t rows,
  const size_t cols,
  float scaling_factor)
{
  using namespace test;

  DType itype = TypeInfo<Type>::dtype;

  Tensor data_in({ batches, heads, rows, cols }, itype);
  Tensor softmax_out({ batches, heads, rows, cols }, itype);
  Tensor softmax_in({ batches, heads, rows, cols }, itype);
  Tensor grads_in({ batches, heads, rows, cols }, itype);
  Tensor grads_out({ batches, heads, rows, cols }, itype);

  const size_t elements_total = batches * heads * rows * cols;
  std::unique_ptr<Type[]> softmax_out_ref = std::make_unique<Type[]>(elements_total);
  std::unique_ptr<Type[]> grads_out_ref = std::make_unique<Type[]>(elements_total);

  fillUniform(&data_in);
  fillUniform(&softmax_in);
  fillUniform(&grads_in);

  nvte_scaled_aligned_causal_masked_softmax_forward(
      data_in.data(), softmax_out.data(), scaling_factor, 0);
  nvte_scaled_aligned_causal_masked_softmax_backward(
      grads_in.data(), softmax_in.data(), grads_out.data(), scaling_factor, 0);


  // Reference implementations
  compute_fwd_ref(softmax_out_ref.get(), data_in.cpu_dptr<Type>(),
                  scaling_factor, batches, heads, rows, cols);
  compute_bwd_ref(grads_out_ref.get(), grads_in.cpu_dptr<Type>(), softmax_in.cpu_dptr<Type>(),
                  scaling_factor, batches, heads, rows, cols);

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);
  auto [atol, rtol] = getTolerances(itype);
  compareResults("softmax_fwd", softmax_out, softmax_out_ref.get(), atol, rtol);
  compareResults("softmax_bwd", grads_out, grads_out_ref.get(), atol, rtol);
}

// [Batches, Attention Heads, Query Sequence Length, Key Sequence Length, Scaling Factor]
std::vector<std::tuple<size_t, size_t, size_t, size_t, float>> test_cases = {
    {   1,    1,     1,    16,  -1.0f},
    {   1,    2,    17,    32,   0.8f},
    {   2,    1,    37,   112,   1.0f},
    {   2,    4,   127,   128,  -0.2f},
    {   8,    6,   128,   256,   1.3f},
    {   1,    4,   270,   256,   0.8f},
    {   2,    2,   512,   512,  -1.5f},
    {   1,    2,   819,  1024,   2.1f}};

}  // namespace

class CausalSoftmaxTestSuite
    : public ::testing::TestWithParam<std::tuple<
        transformer_engine::DType,
        std::tuple<size_t, size_t, size_t, size_t, float>>> {};

TEST_P(CausalSoftmaxTestSuite, TestCausalSoftmax) {
  using namespace transformer_engine;
  using namespace test;

  const DType input_type = std::get<0>(GetParam());
  const auto size = std::get<1>(GetParam());

  const size_t batches = std::get<0>(size);
  const size_t heads = std::get<1>(size);
  const size_t query_seq_len = std::get<2>(size);
  const size_t key_seq_len = std::get<3>(size);
  const float scaling_factor = std::get<4>(size);

  TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(input_type, InputType,
    performTest<InputType>(batches, heads, query_seq_len, key_seq_len, scaling_factor);
  );
}


INSTANTIATE_TEST_SUITE_P(
  OperatorTest,
  CausalSoftmaxTestSuite,
  ::testing::Combine(
      ::testing::Values(DType::kFloat16, DType::kBFloat16),
      ::testing::ValuesIn(test_cases)),
  [](const testing::TestParamInfo<CausalSoftmaxTestSuite::ParamType>& info) {
    const auto size = std::get<1>(info.param);
    const size_t batches = std::get<0>(size);
    const size_t heads = std::get<1>(size);
    const size_t query_seq_len = std::get<2>(size);
    const size_t key_seq_len = std::get<3>(size);

    std::string scaling_factor = std::to_string(std::get<4>(size));
    for (char& c : scaling_factor) {
      if (c == '-') { c = 'N'; }
      if (c == '.') { c = 'p'; }
    }

    std::string name = test::typeName(std::get<0>(info.param)) + "X" +
                       std::to_string(batches) + "X" +
                       std::to_string(heads) + "X" +
                       std::to_string(query_seq_len) + "X" +
                       std::to_string(key_seq_len) + "X" +
                       scaling_factor;
    return name;
  });
