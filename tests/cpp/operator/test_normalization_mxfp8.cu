#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <cmath>
#include <cstring>
#include <memory>
#include <map>
#include <iomanip>
#include <iostream>
#include <random>

#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <transformer_engine/normalization.h>
#include <transformer_engine/transformer_engine.h>
#include "../test_common.h"

using namespace transformer_engine;
using namespace test;

namespace {

using fp8e8m0 = byte;

enum NormType {
  LayerNorm,
  RMSNorm
};

std::map<NormType, std::string> normToString = {
  {NormType::LayerNorm, "LayerNorm"},
  {NormType::RMSNorm, "RMSNorm"}
};

template <typename InputType, typename ScaleType, typename OutputType>
void dequantize_1x_kernel(InputType* input_ptr, ScaleType* scale_ptr, OutputType* output_ptr,
  size_t rows, size_t cols, size_t scaling_mode_x, size_t scaling_mode_y){

  const size_t block_size_Y = scaling_mode_x;   // mind the mapping Y <-- x
  const size_t block_size_X = scaling_mode_y;   //              and X <-- y
  const size_t tile_size_Y = std::max(32lu, block_size_Y);
  const size_t tile_size_X = std::max(64lu, block_size_X);
  const size_t tiles_num_Y = (rows + tile_size_Y - 1) / tile_size_Y;
  const size_t tiles_num_X = (cols + tile_size_X - 1) / tile_size_X;
  const size_t blocks_per_tile_Y = tile_size_Y / block_size_Y;
  const size_t blocks_per_tile_X = tile_size_X / block_size_X;
  const size_t blocks_per_row = (cols + block_size_X - 1) / block_size_X;

  #pragma omp parallel for proc_bind(spread) schedule(static)
  for (size_t t = 0; t < tiles_num_Y * tiles_num_X; ++t) {
      const size_t tile_Y = t / tiles_num_X;
      const size_t tile_X = t % tiles_num_X;
      const size_t tile_offset_Y = tile_Y * tile_size_Y;
      const size_t tile_offset_X = tile_X * tile_size_X;

      for (size_t ii = 0; ii < blocks_per_tile_Y; ++ii) {
          const size_t block_idx_Y = tile_Y * blocks_per_tile_Y + ii;
          const size_t block_offset_Y = ii * block_size_Y;
          const size_t i_min = tile_offset_Y + block_offset_Y;
          const size_t i_max = std::min(i_min + block_size_Y, rows);

          for (size_t jj = 0; jj < blocks_per_tile_X; ++jj) {
              const size_t block_idx_X = tile_X * blocks_per_tile_X + jj;
              const size_t block_offset_X = jj * block_size_X;
              const size_t j_min = tile_offset_X + block_offset_X;
              const size_t j_max = std::min(j_min + block_size_X, cols);

              const size_t mx_scale_idx = block_idx_Y * blocks_per_row + block_idx_X;

              // TODO: padded SFs i.e. (4,128)
              const float scale_inv = exp2f(static_cast<float>(scale_ptr[mx_scale_idx]) - FP32_EXPONENT_BIAS);
              for (size_t i = i_min; i < i_max; ++i) {
                  for (size_t j = j_min; j < j_max; ++j) {
                    const size_t idx = i * cols + j;
                    const float elem = static_cast<float>(input_ptr[idx]);
                    output_ptr[idx] = static_cast<float>(elem * scale_inv);
                  }
              }
          }
      }
  }
}

template <typename InputType, typename ScaleType>
void dequantize_2x(Tensor& input, Tensor& output, bool is_training)
{
  input.to_cpu();
  auto scaling_mode = input.scaling_mode();
  assert(input.rowwise_shape().ndim == 2);
  assert(input.columnwise_shape().ndim == 2);

  dequantize_1x_kernel(input.rowwise_cpu_dptr<InputType>(),
                       input.rowwise_cpu_scale_inv_ptr<ScaleType>(),
                       output.rowwise_cpu_dptr<float>(),
                       input.rowwise_shape().data[0], input.rowwise_shape().data[1],
                       1, 32);
  if (is_training)
    dequantize_1x_kernel(input.columnwise_cpu_dptr<InputType>(),
                         input.columnwise_cpu_scale_inv_ptr<ScaleType>(),
                         output.columnwise_cpu_dptr<float>(),
                         input.columnwise_shape().data[0], input.columnwise_shape().data[1],
                         32, 1);
}

template <typename InputType>
void compute_ref_stats(NormType norm_type,
                       const InputType *data, float *mu, float *rsigma,
                       const size_t N, const size_t H, const double epsilon){
  using compute_t = float;

  #pragma omp parallel for proc_bind(spread)
  for (size_t i = 0; i < N; ++i) {
    compute_t sum = 0;
    for (size_t j = 0; j < H; ++j) {
      sum += static_cast<compute_t>(data[i * H + j]);
    }
    compute_t m;
    if (norm_type == LayerNorm){
      mu[i] = sum / H;
      m = mu[i];
    } else { m = 0;}

    compute_t sum_sq = 0;
    for (size_t j = 0; j < H; ++j) {
      compute_t current = static_cast<compute_t>(data[i * H + j]);
      sum_sq += (current - m) * (current - m);
    }
    rsigma[i] = rsqrtf((sum_sq / H) + epsilon);
  }
}

template <typename InputType, typename OutputType>
void compute_ref_output(NormType norm_type,
                        const InputType *data, const InputType *gamma, const InputType *beta,
                        const float *mu, const float *rsigma,
                        const size_t N, const size_t H,
                        OutputType* output,
                        const bool zero_centered_gamma){
  using compute_t = float;

  #pragma omp parallel for proc_bind(spread)
  for (size_t i = 0; i < N; ++i) {
    for (size_t j = 0; j < H; ++j) {
      compute_t current = static_cast<compute_t>(data[i * H + j]);
      compute_t g = static_cast<compute_t>(gamma[j]);
      if (zero_centered_gamma) {
        g += 1.0;
      }

      compute_t tmp;
      if (norm_type == LayerNorm) {
        tmp = (current - mu[i]) * rsigma[i] * g + static_cast<compute_t>(beta[j]);
      } else { // RMSNorm
        tmp = current * rsigma[i] * g;
      }

      output[i * H + j] = tmp;
    }
  }
}

template <typename InputType, typename OutputType>
void performTest(const size_t N, const size_t H, const bool zero_centered_gamma, NormType norm_type, bool is_training) {

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  if (getDeviceComputeCapability() < blackwellComputeCapability) {
    GTEST_SKIP();
  }

  using WeightType = InputType;
  DType itype = TypeInfo<InputType>::dtype;
  DType wtype = TypeInfo<WeightType>::dtype;
  DType otype = TypeInfo<OutputType>::dtype;

  Tensor input("input", { N, H }, itype);
  Tensor z("z", { N, H }, otype, true, is_training, NVTE_MXFP8_1D_SCALING);
  Tensor gamma("gamma", { H }, wtype);
  Tensor beta("beta", { H }, wtype);
  Tensor mu("mu", { N }, DType::kFloat32);
  Tensor rsigma("rsigma", { N }, DType::kFloat32);
  Tensor workspace;


  fillUniform(&input);
  fillUniform(&gamma);
  fillUniform(&beta);

  // Forward kernel
  float epsilon = 1e-5;
  if (norm_type == NormType::LayerNorm){
    nvte_layernorm_fwd(input.data(), gamma.data(), beta.data(), epsilon,
                       z.data(), mu.data(), rsigma.data(), workspace.data(),
                       prop.multiProcessorCount, zero_centered_gamma,
                       0);
    workspace = Tensor("workspace", workspace.rowwise_shape(), workspace.dtype());
    nvte_layernorm_fwd(input.data(), gamma.data(), beta.data(), epsilon,
                       z.data(), mu.data(), rsigma.data(), workspace.data(),
                       prop.multiProcessorCount, zero_centered_gamma,
                       0);
  } else {
    nvte_rmsnorm_fwd(input.data(), gamma.data(), epsilon,
                     z.data(), rsigma.data(), workspace.data(),
                     prop.multiProcessorCount, zero_centered_gamma,
                     0);

    workspace = Tensor("workspace", workspace.rowwise_shape(), workspace.dtype());
    nvte_rmsnorm_fwd(input.data(), gamma.data(), epsilon,
                     z.data(), rsigma.data(), workspace.data(),
                     prop.multiProcessorCount, zero_centered_gamma,
                     0);
  }

  Tensor dequantized_output("dequantized_output", { N, H }, DType::kFloat32, true, true);

  dequantize_2x<OutputType, fp8e8m0>(z, dequantized_output, is_training);

  // Reference implementations
  std::unique_ptr<float[]> ref_mu = std::make_unique<float[]>(N);
  std::unique_ptr<float[]> ref_rsigma = std::make_unique<float[]>(N);
  std::unique_ptr<float[]> ref_output = std::make_unique<float[]>(N * H);


  compute_ref_stats(norm_type, input.rowwise_cpu_dptr<InputType>(), ref_mu.get(),
                    ref_rsigma.get(), N, H, epsilon);
  // use the GPU stats to tighten the tolerances
  float *ref_mu_ptr, *ref_rsigma_ptr;
  if (is_training){
    mu.to_cpu();
    rsigma.to_cpu();
    ref_mu_ptr = mu.rowwise_cpu_dptr<float>();
    ref_rsigma_ptr = rsigma.rowwise_cpu_dptr<float>();
  } else {
    ref_mu_ptr = ref_mu.get();
    ref_rsigma_ptr = ref_rsigma.get();
  }
  compute_ref_output(norm_type, input.rowwise_cpu_dptr<InputType>(),
                     gamma.rowwise_cpu_dptr<WeightType>(),
                     beta.rowwise_cpu_dptr<WeightType>(),
                     ref_mu_ptr,
                     ref_rsigma_ptr,
                     N, H,
                     ref_output.get(),
                     zero_centered_gamma);

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);

  auto [atol_stats, rtol_stats] = getTolerances(DType::kFloat32);
  rtol_stats = 5e-5;
  if (is_training){
    compareResults("mu", mu, ref_mu.get(), true, atol_stats, rtol_stats);
    compareResults("rsigma", rsigma, ref_rsigma.get(), true, atol_stats, rtol_stats);
  }

  float atol, rtol;
  if (otype == DType::kFloat8E5M2){
    atol = 1.25e-1;
    rtol = 1.25e-1;
  } else if (otype == DType::kFloat8E4M3){
    if (itype == DType::kBFloat16){
      atol = 7e-2;
      rtol = 7e-2;
    } else {
      atol = 6.25e-2;
      rtol = 6.25e-2;
    }
  }
  compareResults("output_rowwise", dequantized_output, ref_output.get(), true, atol, rtol, false);
  if (is_training)
    compareResults("output_colwise", dequantized_output, ref_output.get(), false, atol, rtol, false);
}

std::vector<std::pair<size_t, size_t>> test_cases = {
  {32, 32},
  {768, 2304},
  {2048, 12288},
};

std::vector<NormType> norms = {
  NormType::LayerNorm,
  NormType::RMSNorm
};

}  // namespace

class MxNormTestSuite : public ::testing::TestWithParam< std::tuple<NormType,
                                                                    transformer_engine::DType,
                                                                    transformer_engine::DType,
                                                                    std::pair<size_t, size_t>,
                                                                    bool, bool>> {};

TEST_P(MxNormTestSuite, TestMxNorm) {
  using namespace transformer_engine;
  using namespace test;

  const NormType norm_type = std::get<0>(GetParam());
  const DType input_type = std::get<1>(GetParam());
  const DType output_type = std::get<2>(GetParam());
  const auto size = std::get<3>(GetParam());
  const bool zero_centered_gamma = std::get<4>(GetParam());
  const bool is_training = std::get<5>(GetParam());

  TRANSFORMER_ENGINE_TYPE_SWITCH_FP16_FP32_ONLY(input_type, InputType,
    TRANSFORMER_ENGINE_TYPE_SWITCH_FP8_ONLY(output_type, OutputType,
      performTest<InputType, OutputType>(size.first, size.second, zero_centered_gamma, norm_type, is_training);
    );
  );
}

INSTANTIATE_TEST_SUITE_P(
  OperatorTest,
  MxNormTestSuite,
  ::testing::Combine(
    ::testing::Values(NormType::LayerNorm, NormType::RMSNorm),
    ::testing::Values(DType::kFloat32, DType::kBFloat16, DType::kFloat16),
    ::testing::Values(DType::kFloat8E5M2, DType::kFloat8E4M3),
    ::testing::ValuesIn(test_cases),
    ::testing::Values(true, false),
    ::testing::Values(true, false)),
  [](const testing::TestParamInfo<MxNormTestSuite::ParamType>& info) {
    std::string name = normToString.at(std::get<0>(info.param)) + "_" +
      test::typeName(std::get<1>(info.param)) + "X" +
      test::typeName(std::get<2>(info.param)) + "X" +
      std::to_string(std::get<3>(info.param).first) + "X" +
      std::to_string(std::get<3>(info.param).second) + "X" +
      std::to_string(std::get<4>(info.param)) + "out" +
      std::to_string(int(std::get<5>(info.param)) + 1) + "x";
    return name;
  });
