#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <transformer_engine/transpose.h>
#include <transformer_engine/logging.h>
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <memory>
#include <iostream>
#include <iomanip>
#include <random>
#include <cstring>
#include "../test_common.h"

using namespace transformer_engine;

namespace {

template <typename InputType, typename OutputType>
void compute_ref(const InputType *data, OutputType *output_c, OutputType *output_t,
                 const size_t N, const size_t H,
                 float *amax, float scale) {
  using compute_t = float;
  compute_t current_max = -1e100;
  for (size_t i = 0; i < N; ++i) {
    for (size_t j = 0; j < H; ++j) {
      compute_t current = static_cast<compute_t>(data[i * H + j]);
      current_max = fmaxf(current_max, fabsf(current));
      output_c[i * H + j] = OutputType(scale * current);
      output_t[j * N + i] = OutputType(scale * current);
    }
  }
  *amax = current_max;
}

template <typename InputType, typename OutputType>
void performTest(const size_t N, const size_t H) {
  using namespace test;

  DType itype = TypeInfo<InputType>::dtype;
  DType otype = TypeInfo<OutputType>::dtype;

  Tensor input({ N, H }, itype);
  Tensor output_c({ N, H }, otype);
  Tensor output_t({ H, N }, otype);

  std::unique_ptr<OutputType[]> ref_output_c = std::make_unique<OutputType[]>(N * H);
  std::unique_ptr<OutputType[]> ref_output_t = std::make_unique<OutputType[]>(N * H);

  fillUniform(&input);
  setRandomScale(&output_c);
  output_t.shareFP8Meta(output_c);

  nvte_cast_transpose(input.data(), output_c.data(), output_t.data(), 0);

  float ref_amax;
  compute_ref<InputType, OutputType>(input.cpu_dptr<InputType>(), ref_output_c.get(),
                                     ref_output_t.get(), N, H, &ref_amax,
                                     output_c.scale());

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);
  if (isFp8Type(otype)) {
    auto [atol_amax, rtol_amax] = getTolerances(DType::kFloat32);
    compareResults("amax", output_c.amax(), ref_amax, atol_amax, rtol_amax);
  }
  auto [atol, rtol] = getTolerances(otype);
  compareResults("output_c", output_c, ref_output_c.get(), atol, rtol);
  compareResults("output_t", output_t, ref_output_t.get(), atol, rtol);
}

std::vector<std::pair<size_t, size_t>> test_cases = {{2048, 12288},
                                                     {768, 1024},
                                                     {256, 65536},
                                                     {65536, 128},
                                                     {256, 256},
                                                     {120, 2080},
                                                     {8, 8}};
}  // namespace

class CTTestSuite : public ::testing::TestWithParam<std::tuple<transformer_engine::DType,
                                                               transformer_engine::DType,
                                                               std::pair<size_t, size_t>>> {};

TEST_P(CTTestSuite, TestCastTranspose) {
  using namespace transformer_engine;
  using namespace test;

  const DType input_type = std::get<0>(GetParam());
  const DType output_type = std::get<1>(GetParam());
  const auto size = std::get<2>(GetParam());

  TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(input_type, InputType,
    TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(output_type, OutputType,
      performTest<InputType, OutputType>(size.first, size.second);
    );
  );
}



INSTANTIATE_TEST_SUITE_P(
  OperatorTest,
  CTTestSuite,
  ::testing::Combine(
      ::testing::Values(DType::kFloat32, DType::kBFloat16, DType::kFloat16),
      ::testing::ValuesIn(test::all_fp_types),
      ::testing::ValuesIn(test_cases)),
  [](const testing::TestParamInfo<CTTestSuite::ParamType>& info) {
    std::string name = test::typeName(std::get<0>(info.param)) + "X" +
                       test::typeName(std::get<1>(info.param)) + "X" +
                       std::to_string(std::get<2>(info.param).first) + "X" +
                       std::to_string(std::get<2>(info.param).second);
    return name;
  });
