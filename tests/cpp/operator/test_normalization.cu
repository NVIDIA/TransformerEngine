#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <cmath>
#include <cstring>
#include <memory>
#include <iomanip>
#include <iostream>
#include <random>
#include <stdlib.h>

#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <transformer_engine/normalization.h>
#include <transformer_engine/transformer_engine.h>
#include "../test_common.h"

using namespace transformer_engine;
using namespace test;

namespace {

enum NormType {
  LayerNorm,
  RMSNorm
};

std::map<NormType, std::string> normToString = {
  {NormType::LayerNorm, "LayerNorm"},
  {NormType::RMSNorm, "RmsNorm"}
};

template <typename InputType>
void compute_ref_stats(NormType norm_type,
                       const InputType *data, float *mu, float *rsigma,
                       const size_t N, const size_t H, const double epsilon){
  using compute_t = float;
  compute_t current, m;
  for (size_t i = 0; i < N; ++i) {
    compute_t sum = 0;
    for (size_t j = 0; j < H; ++j) {
      sum += static_cast<compute_t>(data[i * H + j]);
    }
    if (norm_type == LayerNorm){
      mu[i] = sum / H;
      m = mu[i];
    } else { m = 0;}

    compute_t sum_sq = 0;
    for (size_t j = 0; j < H; ++j) {
      current = static_cast<compute_t>(data[i * H + j]);
      sum_sq += (current - m) * (current - m);
    }
    rsigma[i] = rsqrtf((sum_sq / H) + epsilon);
  }
}

// For now, cudnn does static_cast<compute_t>(gamma + static_cast<input_t>(1.0))
// This will be changed in the future release
template <typename InputType>
inline auto compute_gamma(InputType gamma, const bool zero_centered_gamma, const bool use_cudnn){

  using compute_t = float;
  if constexpr (std::is_same_v<InputType, fp8e5m2> || std::is_same_v<InputType, fp8e4m3>){
    compute_t g = static_cast<compute_t>(gamma);
    if (zero_centered_gamma) {
      g += static_cast<compute_t>(1.f);
    }
    return g;
  } else {
    if (use_cudnn){
      compute_t g = static_cast<compute_t>(0.f);
      InputType gi = gamma;
      if (zero_centered_gamma) {
        gi = gi + static_cast<InputType>(1.f);
      }
      g = static_cast<compute_t>(gi);
      return g;
    } else {
      compute_t g = static_cast<compute_t>(gamma);
      if (zero_centered_gamma) {
        g += static_cast<compute_t>(1.f);
      }
      return g;
    }
  }
}

template <typename InputType, typename OutputType>
void compute_ref_output(NormType norm_type,
                        const InputType *data, const InputType *gamma, const InputType *beta,
                        OutputType* output,
                        const float *mu, const float *rsigma,
                        const size_t N, const size_t H,
                        float *amax, float scale, const bool zero_centered_gamma, const bool use_cudnn) {
  using compute_t = float;
  compute_t current_max = -1e100;
  for (size_t i = 0; i < N; ++i) {
    for (size_t j = 0; j < H; ++j) {
      compute_t current = static_cast<compute_t>(data[i * H + j]);
      compute_t g = compute_gamma(gamma[j], zero_centered_gamma, use_cudnn);

      compute_t tmp;
      if (norm_type == LayerNorm) {
        tmp = (current - mu[i]) * rsigma[i] * g + static_cast<compute_t>(beta[j]);
      } else { // RMSNorm
        tmp = current * rsigma[i] * g;
      }

      output[i * H + j] = static_cast<OutputType>(tmp * scale);
      current_max = fmaxf(current_max, fabsf(tmp));
    }
  }
  *amax = current_max;
}


template <typename InputType, typename OutputType>
void compute_ref_backward(const NormType norm_type, const OutputType *output_grad, const InputType *data,
                          const float *mu, const float *rsigma,
                          const InputType *gamma,
                          InputType *data_grad,
                          InputType *gamma_grad, InputType *beta_grad,
                          const size_t N, const size_t H,
                          const bool zero_centered_gamma, const bool use_cudnn) {
  using compute_t = float;
  std::vector<compute_t> dgamma(H, 0.f);
  std::vector<compute_t> dbeta(H, 0.f);

  for (size_t i = 0 ; i < N; ++i) {
    // Reductions
    auto local_mu = (norm_type == LayerNorm) ? mu[i] : 0.;
    compute_t mdy = 0, mdyy = 0;
    for (size_t j = 0; j < H; ++j) {
      const compute_t x = static_cast<compute_t>(data[i * H + j]);
      const compute_t y = (x - local_mu) * rsigma[i];
      compute_t g = compute_gamma(gamma[j], zero_centered_gamma, use_cudnn);
      const compute_t dz = static_cast<compute_t>(output_grad[i * H + j]);
      const compute_t dy = g * dz;
      dgamma[j] += y * dz;
      if (norm_type == LayerNorm) {
        dbeta[j] += dz;
        mdy += dy;
      }
      mdyy += dy * y;
    }
    mdy /= H;
    mdyy /= H;

    // Input grads
    for (size_t j = 0; j < H; ++j) {
      const compute_t x = static_cast<compute_t>(data[i * H + j]);
      const compute_t y = (x - local_mu) * rsigma[i];
      compute_t g = compute_gamma(gamma[j], zero_centered_gamma, use_cudnn);
      const compute_t dz = static_cast<compute_t>(output_grad[i * H + j]);
      const compute_t dy = g * dz;
      const compute_t dx = rsigma[i] * (dy - mdyy * y - mdy);
      data_grad[i * H + j] = static_cast<InputType>(dx);
    }
  }

  // Weight grads
  for (size_t j = 0; j < H; ++j) gamma_grad[j] = static_cast<InputType>(dgamma[j]);
  if (norm_type == LayerNorm) for (size_t j = 0; j < H; ++j) beta_grad[j] = static_cast<InputType>(dbeta[j]);
}

template <typename InputType, typename OutputType>
void performTest(const size_t N, const size_t H, const bool zero_centered_gamma,
                 NormType norm_type, bool use_cudnn) {
  if (sizeof(InputType) < sizeof(OutputType)) {
    GTEST_SKIP() << "LN kernel does not support OutputType > InputType";
    return;
  }
  using WeightType = InputType;
  DType itype = TypeInfo<InputType>::dtype;
  DType wtype = TypeInfo<WeightType>::dtype;
  DType otype = TypeInfo<OutputType>::dtype;

  if ((itype == DType::kBFloat16 && otype == DType::kFloat16) ||
      (itype == DType::kFloat16 && otype == DType::kBFloat16)) {
    GTEST_SKIP() << "LN kernel does not support mixing Float16 and BFloat16";
    return;
  }

  Tensor input({ N, H }, itype);
  Tensor z({ N, H }, otype);
  Tensor gamma({ H }, wtype);
  Tensor beta({ H }, wtype);
  Tensor mu({ N }, DType::kFloat32);
  Tensor rsigma({ N }, DType::kFloat32);
  Tensor dz({ N, H }, wtype);
  Tensor dx({ N, H }, itype);
  Tensor dgamma({ H }, wtype);
  Tensor dbeta({ H }, wtype);
  Tensor workspace_fwd, workspace_bwd;

  fillUniform(&input);
  fillUniform(&gamma);
  fillUniform(&beta);
  setRandomScale(&z);
  fillUniform(&dz);

  std::unique_ptr<OutputType[]> ref_output = std::make_unique<OutputType[]>(N * H);
  std::unique_ptr<float[]> ref_mu = std::make_unique<float[]>(N);
  std::unique_ptr<float[]> ref_rsigma = std::make_unique<float[]>(N);
  std::unique_ptr<InputType[]> ref_dx = std::make_unique<InputType[]>(N * H);
  std::unique_ptr<WeightType[]> ref_dgamma = std::make_unique<InputType[]>(H);
  std::unique_ptr<WeightType[]> ref_dbeta = std::make_unique<InputType[]>(H);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  if (use_cudnn){
    nvte_enable_cudnn_norm_fwd(true);
    nvte_enable_cudnn_norm_bwd(true);
  }

  // Forward kernel
  float epsilon = 1e-5;
  if (norm_type == LayerNorm){
    nvte_layernorm_fwd(input.data(), gamma.data(), beta.data(), epsilon,
                       z.data(), mu.data(), rsigma.data(), workspace_fwd.data(),
                       prop.multiProcessorCount, zero_centered_gamma, 0);
    workspace_fwd = Tensor(workspace_fwd.shape(), workspace_fwd.dtype());
    nvte_layernorm_fwd(input.data(), gamma.data(), beta.data(), epsilon,
                       z.data(), mu.data(), rsigma.data(), workspace_fwd.data(),
                       prop.multiProcessorCount, zero_centered_gamma, 0);

    nvte_layernorm_bwd(dz.data(), input.data(),
                       mu.data(), rsigma.data(), gamma.data(),
                       dx.data(), dgamma.data(), dbeta.data(),
                       workspace_bwd.data(),
                       prop.multiProcessorCount, zero_centered_gamma, 0);
    workspace_bwd = Tensor(workspace_bwd.shape(), workspace_bwd.dtype());
    nvte_layernorm_bwd(dz.data(), input.data(),
                       mu.data(), rsigma.data(), gamma.data(),
                       dx.data(), dgamma.data(), dbeta.data(),
                       workspace_bwd.data(),
                       prop.multiProcessorCount, zero_centered_gamma, 0);
  } else {
    nvte_rmsnorm_fwd(input.data(), gamma.data(), epsilon,
                     z.data(), rsigma.data(), workspace_fwd.data(),
                     prop.multiProcessorCount, zero_centered_gamma, 0);
    workspace_fwd = Tensor(workspace_fwd.shape(), workspace_fwd.dtype());
    nvte_rmsnorm_fwd(input.data(), gamma.data(), epsilon,
                     z.data(), rsigma.data(), workspace_fwd.data(),
                     prop.multiProcessorCount, zero_centered_gamma, 0);

    nvte_rmsnorm_bwd(dz.data(), input.data(), rsigma.data(), gamma.data(),
                     dx.data(), dgamma.data(),
                     workspace_bwd.data(),
                     prop.multiProcessorCount, zero_centered_gamma, 0);
    workspace_bwd = Tensor(workspace_bwd.shape(), workspace_bwd.dtype());
    nvte_rmsnorm_bwd(dz.data(), input.data(), rsigma.data(), gamma.data(),
                     dx.data(), dgamma.data(),
                     workspace_bwd.data(),
                     prop.multiProcessorCount, zero_centered_gamma, 0);
  }

  if (use_cudnn){
    nvte_enable_cudnn_norm_fwd(false);
    nvte_enable_cudnn_norm_bwd(false);
  }

  // Reference implementations
  // use the GPU stats to tighten the tolerances
  mu.to_cpu();
  rsigma.to_cpu();
  float ref_amax;
  compute_ref_stats(norm_type, input.cpu_dptr<InputType>(), ref_mu.get(),
                    ref_rsigma.get(), N, H, epsilon);
  float ref_scale = isFp8Type(otype) ? z.scale() : 1.f;
  compute_ref_output(norm_type, input.cpu_dptr<InputType>(),
                     gamma.cpu_dptr<WeightType>(),
                     beta.cpu_dptr<WeightType>(),
                     ref_output.get(),
                     mu.cpu_dptr<float>(),
                     rsigma.cpu_dptr<float>(),
                     N, H,
                     &ref_amax,
                     ref_scale,
                     zero_centered_gamma,
                     use_cudnn);
  compute_ref_backward(norm_type, dz.cpu_dptr<WeightType>(), input.cpu_dptr<InputType>(),
                       mu.cpu_dptr<float>(), rsigma.cpu_dptr<float>(),
                       gamma.cpu_dptr<WeightType>(),
                       ref_dx.get(), ref_dgamma.get(), ref_dbeta.get(),
                       N, H, zero_centered_gamma,
                       use_cudnn);

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);

  auto [atol_amax, rtol_amax] = getTolerances(DType::kFloat32);
  if (isFp8Type(otype)) {
    compareResults("amax", z.amax(), ref_amax, atol_amax, rtol_amax);
    float ref_scale_inv = 1.f / z.scale();
    compareResults("scale_inv", z.scale_inv(), ref_scale_inv, atol_amax, rtol_amax);
  }

  auto [atol_stats, rtol_stats] = getTolerances(DType::kFloat32);
  rtol_stats = 5e-5;
  compareResults("mu", mu, ref_mu.get(), atol_stats, rtol_stats);
  compareResults("rsigma", rsigma, ref_rsigma.get(), atol_stats, rtol_stats);

  auto [atol, rtol] = getTolerances(otype);
  if (otype == DType::kFloat32) {
    atol = 5e-7;
  }
  compareResults("output", z, ref_output.get(), atol, rtol);

  double atol_bwd = 5e-4;
  double rtol_bwd = 5e-4;
  compareResults("dx", dx, ref_dx.get(), atol_bwd, rtol_bwd);
  compareResults("dgamma", dgamma, ref_dgamma.get(), atol_bwd, rtol_bwd);
  compareResults("dbeta", dbeta, ref_dbeta.get(), atol_bwd, rtol_bwd);
}

std::vector<std::pair<size_t, size_t>> test_cases = {
  {71, 229},
  {29, 541},
  {768, 6144},
  {2048, 12288},
};

}  // namespace

class NormTestSuite : public ::testing::TestWithParam<std::tuple<bool,
NormType,
transformer_engine::DType,
                                                               transformer_engine::DType,
                                                               std::pair<size_t, size_t>,
                                                               bool>> {};

TEST_P(NormTestSuite, TestNorm) {
    using namespace transformer_engine;
    using namespace test;

  const bool use_cudnn = std::get<0>(GetParam());
  const NormType norm_type = std::get<1>(GetParam());
    const DType input_type = std::get<2>(GetParam());
    const DType output_type = std::get<3>(GetParam());
    const auto size = std::get<4>(GetParam());
    const bool zero_centered_gamma = std::get<5>(GetParam());

    TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(input_type, InputType,
      TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(output_type, OutputType,
        performTest<InputType, OutputType>(size.first, size.second, zero_centered_gamma, norm_type, use_cudnn);
      );
    );
}

INSTANTIATE_TEST_SUITE_P(
    OperatorTest,
    NormTestSuite,
    ::testing::Combine(
        ::testing::Values(false), //TODO: enabling tests for cudnn backend
        ::testing::Values(NormType::LayerNorm, NormType::RMSNorm),
        ::testing::Values(DType::kFloat32, DType::kBFloat16, DType::kFloat16),
        ::testing::Values(DType::kFloat32, DType::kBFloat16, DType::kFloat16, DType::kFloat8E4M3),
        ::testing::ValuesIn(test_cases),
        ::testing::Values(false, true)),
    [](const testing::TestParamInfo<NormTestSuite::ParamType>& info) {
    auto backend = std::get<0>(info.param) == false ? "Te" : "Cudnn";
std::string name =
  backend +
  normToString.at(std::get<1>(info.param)) + "_" +
  test::typeName(std::get<2>(info.param)) + "X" +
  test::typeName(std::get<3>(info.param)) + "X" +
  std::to_string(std::get<4>(info.param).first) + "X" +
  std::to_string(std::get<4>(info.param).second) + "X" +
  std::to_string(std::get<5>(info.param));
      return name;
    });
