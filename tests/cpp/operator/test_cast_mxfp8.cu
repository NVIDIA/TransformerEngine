#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <transformer_engine/cast.h>
#include <transformer_engine/activation.h>
#include "../test_common.h"
#include "transformer_engine/transformer_engine.h"

using namespace transformer_engine;
using namespace test;

namespace {

enum ProcessingMethod {
    CAST_ONLY,
    CAST_DBIAS,
    CAST_DBIAS_DACT,
    CAST_DACT,
    CAST_ACT
};

enum ActivationType {
    Identity,
    GeLU,
    SiLU,
    ReLU,
    QGeLU,
    SReLU
};

template <typename InputType, typename OutputType>
void compute_ref(const ProcessingMethod processing_method,
                 float (*OP)(const float),
                 const bool rowwise,
                 const bool colwise,
                 const InputType* input,
                 const InputType* grad,
                 OutputType* output_rowwise,
                 OutputType* output_colwise,
                 fp8e8m0* output_scales_rowwise,
                 fp8e8m0* output_scales_colwise,
                 InputType* output_dbias,
                 const size_t rows,
                 const size_t cols,
                 const size_t scales_stride_rowwise,
                 const size_t scales_stride_colwise)
{
    const size_t tile_size_Y = 32;
    const size_t tile_size_X = 32;
    const size_t tiles_num_Y = (rows + tile_size_Y - 1) / tile_size_Y;
    const size_t tiles_num_X = (cols + tile_size_X - 1) / tile_size_X;

    std::vector<float> output_dbias_fp32(cols, 0);
    #pragma omp parallel proc_bind(spread)
    {
        // Buffers to cache intermediate computations
        std::vector<float> cache_buffer(tile_size_Y * tile_size_X);

        std::vector<float> thread_dbias(cols, 0);
        #pragma omp for schedule(static)
        for (size_t t = 0; t < tiles_num_Y * tiles_num_X; ++t) {
            const size_t tile_Y = t / tiles_num_X;
            const size_t tile_X = t % tiles_num_X;
            const size_t tile_offset_Y = tile_Y * tile_size_Y;
            const size_t tile_offset_X = tile_X * tile_size_X;

            const size_t i_min = tile_offset_Y;
            const size_t i_max = std::min(i_min + tile_size_Y, rows);

            const size_t j_min = tile_offset_X;
            const size_t j_max = std::min(j_min + tile_size_X, cols);

            // Cache computations
            for (size_t i = i_min; i < i_max; ++i) {
                for (size_t j = j_min; j < j_max; ++j) {

                    const size_t idx = i * cols + j;
                    const size_t cache_idx = (i - i_min) * tile_size_X + (j - j_min);

                    float elt = static_cast<float>(input[idx]);
                    if (processing_method == ProcessingMethod::CAST_DBIAS) {
                        // grad is the input
                        elt = static_cast<float>(grad[idx]);
                    }
                    if (processing_method != ProcessingMethod::CAST_ONLY
                        && processing_method != ProcessingMethod::CAST_DBIAS) {
                        elt = OP(elt);
                    }
                    if (processing_method == ProcessingMethod::CAST_DACT ||
                        processing_method == ProcessingMethod::CAST_DBIAS_DACT) {
                        elt *= static_cast<float>(grad[idx]);
                    }
                    thread_dbias[j] += elt;

                    // Numerical truncation: after downcast to InputType (BF16/FP16), upcast it back to FP32
                    elt = static_cast<float>(static_cast<InputType>(elt));

                    cache_buffer[cache_idx] = elt;
                    if (isinf(elt) || isnan(elt)) {
                        continue;
                    }
                }
            }

            if (rowwise) {
                for (size_t i = i_min; i < i_max; ++i) {
                    float block_amax = 0.0f;

                    for (size_t j = j_min; j < j_max; ++j) {
                        const size_t cache_idx = (i - i_min) * tile_size_X + (j - j_min);
                        block_amax = std::max(block_amax, std::abs(cache_buffer[cache_idx]));
                    }

                    const fp8e8m0 biased_exponent = float_to_e8m0(block_amax * Quantized_Limits<OutputType>::max_reciprocal());
                    const size_t scale_idx = i * scales_stride_rowwise + tile_X;
                    output_scales_rowwise[scale_idx] = biased_exponent;
                    const float scale_reciprocal = exp2f_rcp(biased_exponent);

                    for (size_t j = j_min; j < j_max; ++j) {
                        const size_t idx = i * cols + j;
                        const size_t cache_idx = (i - i_min) * tile_size_X + (j - j_min);
                        output_rowwise[idx] = static_cast<OutputType>(cache_buffer[cache_idx] * scale_reciprocal);
                    }
                }
            }
            if (colwise) {
                for (size_t j = j_min; j < j_max; ++j) {
                    float block_amax = 0.0f;

                    for (size_t i = i_min; i < i_max; ++i) {
                        const size_t cache_idx = (i - i_min) * tile_size_X + (j - j_min);
                        block_amax = std::max(block_amax, std::abs(cache_buffer[cache_idx]));
                    }

                    const fp8e8m0 biased_exponent = float_to_e8m0(block_amax * Quantized_Limits<OutputType>::max_reciprocal());
                    const size_t scale_idx = tile_Y * scales_stride_colwise + j;
                    output_scales_colwise[scale_idx] = biased_exponent;
                    const float scale_reciprocal = exp2f_rcp(biased_exponent);

                    for (size_t i = i_min; i < i_max; ++i) {
                        const size_t idx = i * cols + j;
                        const size_t cache_idx = (i - i_min) * tile_size_X + (j - j_min);
                        output_colwise[idx] = static_cast<OutputType>(cache_buffer[cache_idx] * scale_reciprocal);
                    }
                }
            }
        }
        #pragma omp critical
        {
            for (size_t j = 0; j < cols; ++j) {
                output_dbias_fp32[j] += thread_dbias[j];
            }
        }
    }
    for (size_t j = 0; j < cols; ++j) {
        output_dbias[j] = static_cast<InputType>(output_dbias_fp32[j]);
    }
}

/**
 * Scaling along single dimension (either rows or columns)
 * Produces one set of output data and the corresponding data of the fused operation (dbias):
 * 1) Scaled rows + row-wise scaling factors
 *       OR
 * 2) Scaled columns + column-wise scaling factors
 */

template <typename InputType, typename OutputType>
void performTest_x1(const ProcessingMethod processing_method,
                    float (*OP)(const float),
                    const std::vector<size_t>& shape,
                    const bool rowwise,
                    const bool colwise,
                    InputsFillCase fill_case) {
    using namespace test;
    using EncodingType = fp32;
    DType itype = TypeInfo<InputType>::dtype;
    DType otype = TypeInfo<OutputType>::dtype;

    const size_t rows = first_dimension(shape);
    const size_t cols = last_dimension(shape);

    if (shape.size() < 2 && colwise) {
      GTEST_SKIP();
    }

    const size_t block_size_rows = rowwise ? 1 : 32;
    const size_t block_size_cols = colwise ? 1 : 32;

    const std::array<size_t,4> scale_dims = get_scale_tensor_dims(rows, cols, block_size_rows,
                                                                  block_size_cols);

    const size_t unpadded_blocks_Y = scale_dims[0];
    const size_t unpadded_blocks_X = scale_dims[1];
    const size_t blocks_Y = scale_dims[2];
    const size_t blocks_X = scale_dims[3];
    const size_t scales_stride = blocks_X;

    Tensor input("input", shape, itype);
    Tensor grad("grad", shape, itype);
    Tensor output_c("output_c", shape, otype, rowwise, colwise, NVTE_MXFP8_1D_SCALING);
    Tensor output_dbias("output_dbias", std::vector<size_t>{ cols }, itype);

    std::unique_ptr<OutputType[]> ref_output_c = std::make_unique<OutputType[]>(rows * cols);
    std::unique_ptr<InputType[]> ref_output_dbias = std::make_unique<InputType[]>(cols);
    std::unique_ptr<fp8e8m0[]> ref_output_scales = std::make_unique<fp8e8m0[]>(blocks_Y * blocks_X);

    fillCase<EncodingType>(&input, fill_case);
    fillUniform(&grad);

    Tensor workspace;
    switch (processing_method) {
        case ProcessingMethod::CAST_ONLY: {
            nvte_quantize(input.data(), output_c.data(), 0);
            break;
        }
        case ProcessingMethod::CAST_DBIAS: {
            nvte_quantize_dbias(grad.data(),
                                output_c.data(),
                                output_dbias.data(),
                                workspace.data(),
                                0);
            workspace = Tensor("workspace", workspace.rowwise_shape(), workspace.dtype());

            nvte_quantize_dbias(grad.data(),
                                output_c.data(),
                                output_dbias.data(),
                                workspace.data(),
                                0);
            break;
        }
        case ProcessingMethod::CAST_DBIAS_DACT: {
            auto nvte_quantize_dbias_dact = &nvte_quantize_dbias_dgelu;
            if (OP == &dsilu)       { nvte_quantize_dbias_dact = &nvte_quantize_dbias_dsilu; }
            else if (OP == &drelu)  { nvte_quantize_dbias_dact = &nvte_quantize_dbias_drelu; }
            else if (OP == &dqgelu) { nvte_quantize_dbias_dact = &nvte_quantize_dbias_dqgelu; }
            else if (OP == &dsrelu) { nvte_quantize_dbias_dact = &nvte_quantize_dbias_dsrelu; }

            nvte_quantize_dbias_dact(grad.data(),
                                     input.data(),
                                     output_c.data(),
                                     output_dbias.data(),
                                     workspace.data(),
                                     0);
            workspace = Tensor("workspace", workspace.rowwise_shape(), workspace.dtype());

            nvte_quantize_dbias_dact(grad.data(),
                                     input.data(),
                                     output_c.data(),
                                     output_dbias.data(),
                                     workspace.data(),
                                     0);
            break;
        }
        case ProcessingMethod::CAST_DACT: {
            auto nvte_dact = &nvte_dgelu;
            if (OP == &dsilu)       { nvte_dact = &nvte_dsilu; }
            else if (OP == &drelu)  { nvte_dact = &nvte_drelu; }
            else if (OP == &dqgelu) { nvte_dact = &nvte_dqgelu; }
            else if (OP == &dsrelu) { nvte_dact = &nvte_dsrelu; }

            nvte_dact(grad.data(), input.data(), output_c.data(), 0);
            break;
        }
        case ProcessingMethod::CAST_ACT: {
            auto nvte_act = &nvte_gelu;
            if (OP == &silu)       { nvte_act = &nvte_silu; }
            else if (OP == &relu)  { nvte_act = &nvte_relu; }
            else if (OP == &qgelu) { nvte_act = &nvte_qgelu; }
            else if (OP == &srelu) { nvte_act = &nvte_srelu; }

            nvte_act(input.data(), output_c.data(), 0);
            break;
        }
    }

    hipDeviceSynchronize();
    auto err = hipGetLastError();
    ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);

    compute_ref<InputType, OutputType>(processing_method,
                                       OP,
                                       rowwise,
                                       colwise,
                                       input.rowwise_cpu_dptr<InputType>(),
                                       grad.rowwise_cpu_dptr<InputType>(),
                                       ref_output_c.get(),
                                       ref_output_c.get(),
                                       ref_output_scales.get(),
                                       ref_output_scales.get(),
                                       ref_output_dbias.get(),
                                       rows,
                                       cols,
                                       scales_stride,
                                       scales_stride);

    const uint8_t * const gpu_scales_ptr = rowwise
                                           ? output_c.rowwise_cpu_scale_inv_ptr<fp8e8m0>()
                                           : output_c.columnwise_cpu_scale_inv_ptr<fp8e8m0>();

    const size_t scale_diff_abs_tolerance = 0;
    const double abs_tolerable_mismatches_limit = 0.0;
    const double rel_tolerable_mismatches_limit = 0.0;

    size_t mismatches_scales = 0;

    compare_scaling_factors("scales", gpu_scales_ptr, ref_output_scales.get(),
                            unpadded_blocks_Y, unpadded_blocks_X, scales_stride,
                            mismatches_scales,
                            scale_diff_abs_tolerance,
                            abs_tolerable_mismatches_limit,
                            rel_tolerable_mismatches_limit);

    const size_t mismatches_elts = 32 * mismatches_scales;
    auto [atol, rtol] = getTolerances(otype);
    compareResults("output_c", output_c, ref_output_c.get(), rowwise, atol, rtol, true, mismatches_elts);

    if (processing_method == ProcessingMethod::CAST_DBIAS
        || processing_method == ProcessingMethod::CAST_DBIAS_DACT)
    {
        auto [atol_dbias, rtol_dbias] = getTolerances(itype);
        if (itype == DType::kFloat32) {
            atol_dbias = 1e-4;
            rtol_dbias *= sqrt(static_cast<double>(rows)) ;
        } else {
            rtol_dbias *= 4;
        }
        compareResults("output_dbias", output_dbias, ref_output_dbias.get(), true, atol_dbias, rtol_dbias);
    }
}

/**
 * Scaling along both dimensions (rows and columns)
 * Produces two sets of scaled output data and the corresponding data of the fused operation (dbias):
 * 1) Scaled rows + row-wise scaling factors
 *      AND
 * 2) Scaled columns + column-wise scaling factors
 */
template <typename InputType, typename OutputType>
void performTest_x2(const ProcessingMethod processing_method,
                    float (*OP)(const float),
                    const std::vector<size_t>& shape,
                    const size_t block_size_rows,
                    const size_t block_size_cols,
                    InputsFillCase fill_case) {
    using namespace test;
    using EncodingType = fp32;
    DType itype = TypeInfo<InputType>::dtype;
    DType otype = TypeInfo<OutputType>::dtype;

    if (shape.size() < 2) {
      GTEST_SKIP();
    }

    const size_t rows = first_dimension(shape);
    const size_t cols = last_dimension(shape);

    const std::array<size_t,4> scale_dims_rowwise = get_scale_tensor_dims(rows, cols, 1, 32);
    const std::array<size_t,4> scale_dims_colwise = get_scale_tensor_dims(rows, cols, 32, 1);

    const size_t unpadded_blocks_Y_rowwise = scale_dims_rowwise[0];
    const size_t unpadded_blocks_X_rowwise = scale_dims_rowwise[1];
    const size_t blocks_Y_rowwise = scale_dims_rowwise[2];
    const size_t blocks_X_rowwise = scale_dims_rowwise[3];
    const size_t scales_stride_rowwise = blocks_X_rowwise;

    const size_t unpadded_blocks_Y_colwise = scale_dims_colwise[0];
    const size_t unpadded_blocks_X_colwise = scale_dims_colwise[1];
    const size_t blocks_Y_colwise = scale_dims_colwise[2];
    const size_t blocks_X_colwise = scale_dims_colwise[3];
    const size_t scales_stride_colwise = blocks_X_colwise;

    Tensor input("input", shape, itype);
    Tensor grad("grad", shape, itype);
    Tensor output("output", shape, otype, true, true, NVTE_MXFP8_1D_SCALING);
    Tensor output_dbias("output_dbias", std::vector<size_t>{ cols }, itype);

    std::unique_ptr<OutputType[]> ref_output_c_rowwise = std::make_unique<OutputType[]>(rows * cols);
    std::unique_ptr<OutputType[]> ref_output_c_colwise = std::make_unique<OutputType[]>(rows * cols);
    std::unique_ptr<fp8e8m0[]> ref_scales_rowwise = std::make_unique<fp8e8m0[]>(blocks_Y_rowwise * blocks_X_rowwise);
    std::unique_ptr<fp8e8m0[]> ref_scales_colwise = std::make_unique<fp8e8m0[]>(blocks_Y_colwise * blocks_X_colwise);
    std::unique_ptr<InputType[]> ref_output_dbias = std::make_unique<InputType[]>(cols);

    fillCase<EncodingType>(&input, fill_case);
    fillUniform(&grad);

    Tensor workspace;
    switch (processing_method) {
        case ProcessingMethod::CAST_ONLY: {
            nvte_quantize(input.data(), output.data(), 0);
            break;
        }
        case ProcessingMethod::CAST_DBIAS: {
            nvte_quantize_dbias(grad.data(),
                                output.data(),
                                output_dbias.data(),
                                workspace.data(),
                                0);
            workspace = Tensor("workspace", workspace.rowwise_shape(), workspace.dtype());

            nvte_quantize_dbias(grad.data(),
                                output.data(),
                                output_dbias.data(),
                                workspace.data(),
                                0);
            break;
        }
        case ProcessingMethod::CAST_DBIAS_DACT: {
            auto nvte_quantize_dbias_dact = &nvte_quantize_dbias_dgelu;
            if (OP == &dsilu)       { nvte_quantize_dbias_dact = &nvte_quantize_dbias_dsilu; }
            else if (OP == &drelu)  { nvte_quantize_dbias_dact = &nvte_quantize_dbias_drelu; }
            else if (OP == &dqgelu) { nvte_quantize_dbias_dact = &nvte_quantize_dbias_dqgelu; }
            else if (OP == &dsrelu) { nvte_quantize_dbias_dact = &nvte_quantize_dbias_dsrelu; }

            nvte_quantize_dbias_dact(grad.data(),
                                     input.data(),
                                     output.data(),
                                     output_dbias.data(),
                                     workspace.data(),
                                     0);
            workspace = Tensor("workspace", workspace.rowwise_shape(), workspace.dtype());

            nvte_quantize_dbias_dact(grad.data(),
                                     input.data(),
                                     output.data(),
                                     output_dbias.data(),
                                     workspace.data(),
                                     0);
            break;
        }
        case ProcessingMethod::CAST_DACT: {
            auto nvte_dact = &nvte_dgelu;
            if (OP == &dsilu)       { nvte_dact = &nvte_dsilu; }
            else if (OP == &drelu)  { nvte_dact = &nvte_drelu; }
            else if (OP == &dqgelu) { nvte_dact = &nvte_dqgelu; }
            else if (OP == &dsrelu) { nvte_dact = &nvte_dsrelu; }

            nvte_dact(grad.data(), input.data(), output.data(), 0);
            break;
        }
        case ProcessingMethod::CAST_ACT: {
            auto nvte_act = &nvte_gelu;
            if (OP == &silu)       { nvte_act = &nvte_silu; }
            else if (OP == &relu)  { nvte_act = &nvte_relu; }
            else if (OP == &qgelu) { nvte_act = &nvte_qgelu; }
            else if (OP == &srelu) { nvte_act = &nvte_srelu; }

            nvte_act(input.data(), output.data(), 0);
            break;
        }
    }

    hipDeviceSynchronize();
    auto err = hipGetLastError();
    ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);

    compute_ref<InputType, OutputType>(processing_method,
                                       OP,
                                       true,
                                       true,
                                       input.rowwise_cpu_dptr<InputType>(),
                                       grad.rowwise_cpu_dptr<InputType>(),
                                       ref_output_c_rowwise.get(),
                                       ref_output_c_colwise.get(),
                                       ref_scales_rowwise.get(),
                                       ref_scales_colwise.get(),
                                       ref_output_dbias.get(),
                                       rows,
                                       cols,
                                       scales_stride_rowwise,
                                       scales_stride_colwise);

    const size_t scale_diff_abs_tolerance = 0;
    const double abs_tolerable_mismatches_limit = 0.0;
    const double rel_tolerable_mismatches_limit = 0.0;

    size_t mismatches_scales_rowwise = 0;
    compare_scaling_factors("scales_rowwise", output.rowwise_cpu_scale_inv_ptr<fp8e8m0>(),
                            ref_scales_rowwise.get(), unpadded_blocks_Y_rowwise,
                            unpadded_blocks_X_rowwise, scales_stride_rowwise,
                            mismatches_scales_rowwise,
                            scale_diff_abs_tolerance,
                            abs_tolerable_mismatches_limit,
                            rel_tolerable_mismatches_limit);

    size_t mismatches_scales_colwise = 0;
    compare_scaling_factors("scales_colwise", output.columnwise_cpu_scale_inv_ptr<fp8e8m0>(),
                            ref_scales_colwise.get(), unpadded_blocks_Y_colwise,
                            unpadded_blocks_X_colwise, scales_stride_colwise,
                            mismatches_scales_colwise,
                            scale_diff_abs_tolerance,
                            abs_tolerable_mismatches_limit,
                            rel_tolerable_mismatches_limit);

    const size_t mismatches_elts_rowwise = 32 * mismatches_scales_rowwise;
    const size_t mismatches_elts_colwise = 32 * mismatches_scales_colwise;

    auto [atol, rtol] = getTolerances(otype);
    compareResults("output_c_rowwise", output, ref_output_c_rowwise.get(), true, atol, rtol, true, mismatches_elts_rowwise);
    compareResults("output_c_colwise", output, ref_output_c_colwise.get(), false, atol, rtol, true, mismatches_elts_colwise);

    if (processing_method == ProcessingMethod::CAST_DBIAS
        || processing_method == ProcessingMethod::CAST_DBIAS_DACT)
    {
        auto [atol_dbias, rtol_dbias] = getTolerances(itype);
        if (itype == DType::kFloat32) {
            atol_dbias = 1e-4;
            rtol_dbias *= sqrt(static_cast<double>(rows)) ;
        } else {
            rtol_dbias *= 4;
        }
        compareResults("output_dbias", output_dbias, ref_output_dbias.get(), true, atol_dbias, rtol_dbias);
    }
}

std::vector<std::vector<size_t>> matrix_sizes = {
    {1, 16},
    {16, 48},
    {65, 96},
    {128, 128},
    {256, 256},
    {993, 512},
    {511, 6144},
    {8192, 128},
    {2048, 160},
    {577, 1632},
    {1024},
    {8, 32, 1024},
    {16, 8, 4, 512},
};

std::vector<std::pair<size_t, size_t>> block_sizes = {
    {1, 32},
    {32, 1},
    {32, 32},
};

std::vector<InputsFillCase> input_scenarios = {
    InputsFillCase::uniform,
    // InputsFillCase::zeros,
    // InputsFillCase::zero_to_minNorm,
    // InputsFillCase::minNorm_to_maxNorm,
    // InputsFillCase::maxNorm_to_inf
};

std::vector<ProcessingMethod> processing_methods = {
    ProcessingMethod::CAST_ONLY,
    ProcessingMethod::CAST_DBIAS,
    ProcessingMethod::CAST_DBIAS_DACT,
    ProcessingMethod::CAST_DACT,
    ProcessingMethod::CAST_ACT,
};

// Only GeLU activation tests are supported
std::vector<ActivationType> Activation_types = {
    ActivationType::Identity,
    ActivationType::GeLU,
    // ActivationType::SiLU,
    // ActivationType::ReLU,
    // ActivationType::QGeLU,
    // ActivationType::SReLU,
};

}  // namespace

class FusedCastMXFP8TestSuite : public ::testing::TestWithParam
    <std::tuple<ProcessingMethod,
                ActivationType,
                std::vector<size_t>,
                std::pair<size_t, size_t>,
                transformer_engine::DType,
                transformer_engine::DType,
                InputsFillCase>> {};

TEST_P(FusedCastMXFP8TestSuite, TestFusedCastMXFP8) {
    // Skip tests for pre-Blackwell architectures
    if (getDeviceComputeCapability() < blackwellComputeCapability) {
        GTEST_SKIP();
    }

    using namespace transformer_engine;
    using namespace test;

    const ProcessingMethod processing_method = std::get<0>(GetParam());
    const ActivationType Act_type = std::get<1>(GetParam());
    const auto matrix_size = std::get<2>(GetParam());
    const auto block_size = std::get<3>(GetParam());
    const DType input_type = std::get<4>(GetParam());
    const DType output_type = std::get<5>(GetParam());
    const InputsFillCase fill_case = std::get<6>(GetParam());

    // Skips non Act tests if the Activation type is not an identity
    if ((processing_method == ProcessingMethod::CAST_ONLY || processing_method == ProcessingMethod::CAST_DBIAS)
        && Act_type != ActivationType::Identity) {
        GTEST_SKIP();
    }
    // Skips Act tests if the Activation is an identity
    if ((processing_method == ProcessingMethod::CAST_DBIAS_DACT
        || processing_method == ProcessingMethod::CAST_DACT
        || processing_method == ProcessingMethod::CAST_ACT) && (Act_type == ActivationType::Identity)) {
        GTEST_SKIP();
    }

    const bool rowwise = block_size.second != 1;
    const bool colwise = block_size.first != 1;
    if (processing_method == ProcessingMethod::CAST_ACT) {
        // Forward activations
        auto OP = &identity;
        switch (Act_type) {
            case ActivationType::GeLU: OP = &gelu; break;
            case ActivationType::SiLU: OP = &silu; break;
            case ActivationType::ReLU: OP = &relu; break;
            case ActivationType::QGeLU: OP = &qgelu; break;
            case ActivationType::SReLU: OP = &srelu; break;
        }

        TRANSFORMER_ENGINE_TYPE_SWITCH_FP16_FP32_ONLY(input_type, InputType,
            TRANSFORMER_ENGINE_TYPE_SWITCH_FP8_ONLY(output_type, OutputType,
                if (block_size.first == 1 || block_size.second == 1) {
                    performTest_x1<InputType, OutputType>(
                        processing_method, OP, matrix_size,
                        rowwise, colwise, fill_case);
                } else {
                    performTest_x2<InputType, OutputType>(
                        processing_method, OP, matrix_size,
                        block_size.first, block_size.second, fill_case);
                }
            );
        );
    } else {
        auto OP = &identity;
        switch (Act_type) {
            case ActivationType::GeLU: OP = &dgelu; break;
            case ActivationType::SiLU: OP = &dsilu; break;
            case ActivationType::ReLU: OP = &drelu; break;
            case ActivationType::QGeLU: OP = &dqgelu; break;
            case ActivationType::SReLU: OP = &dsrelu; break;
        }
        TRANSFORMER_ENGINE_TYPE_SWITCH_FP16_FP32_ONLY(input_type, InputType,
            TRANSFORMER_ENGINE_TYPE_SWITCH_FP8_ONLY(output_type, OutputType,
                if (block_size.first == 1 || block_size.second == 1) {
                    performTest_x1<InputType, OutputType>(
                        processing_method, OP, matrix_size,
                        rowwise, colwise, fill_case);
                } else {
                    performTest_x2<InputType, OutputType>(
                        processing_method, OP, matrix_size,
                        block_size.first, block_size.second, fill_case);
                }
            );
        );
    }
}

std::string to_string(const ProcessingMethod method) {
    switch (method) {
        case ProcessingMethod::CAST_ONLY:       return "CAST_ONLY";
        case ProcessingMethod::CAST_DBIAS:      return "CAST_DBIAS";
        case ProcessingMethod::CAST_DBIAS_DACT: return "CAST_DBIAS_DACT";
        case ProcessingMethod::CAST_DACT:       return "CAST_DACT";
        case ProcessingMethod::CAST_ACT:        return "CAST_ACT";
        default: return "";
    }
}

std::string to_string(const ActivationType Act_type) {
    switch (Act_type) {
        case ActivationType::Identity:  return "Identity";
        case ActivationType::GeLU:      return "GeLU";
        case ActivationType::SiLU:      return "SiLU";
        case ActivationType::ReLU:      return "ReLU";
        case ActivationType::QGeLU:     return "QGeLU";
        case ActivationType::SReLU:     return "SReLU";
        default: return "";
    }
}

INSTANTIATE_TEST_SUITE_P(
    OperatorTest,
    FusedCastMXFP8TestSuite,
    ::testing::Combine(
        ::testing::ValuesIn(processing_methods),
        ::testing::ValuesIn(Activation_types),
        ::testing::ValuesIn(matrix_sizes),
        ::testing::ValuesIn(block_sizes),
        ::testing::Values(DType::kFloat32, DType::kBFloat16, DType::kFloat16),
        ::testing::Values(DType::kFloat8E4M3, DType::kFloat8E5M2),
        ::testing::ValuesIn(input_scenarios)),
    [](const testing::TestParamInfo<FusedCastMXFP8TestSuite::ParamType>& info) {
        std::string name = to_string(std::get<0>(info.param)) + "X" +
                           to_string(std::get<1>(info.param));
      const auto& shape = std::get<2>(info.param);
      for ( const auto& s: shape) {
        name += "X" + std::to_string(s);
      }
      name += "X" + std::to_string(std::get<3>(info.param).first) +
              "X" + std::to_string(std::get<3>(info.param).second) +
              "X" + test::typeName(std::get<4>(info.param)) +
              "X" + test::typeName(std::get<5>(info.param)) +
              "X" + test::caseName(std::get<6>(info.param));
        return name;
    });
