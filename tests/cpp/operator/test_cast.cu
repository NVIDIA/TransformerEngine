#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <cstring>
#include <iomanip>
#include <iostream>
#include <memory>
#include <random>

#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <transformer_engine/cast.h>
#include "../test_common.h"

using namespace transformer_engine;

namespace {

template <typename InputType, typename OutputType>
void compute_ref(const InputType *data, OutputType *output_c,
                 const size_t size,
                 float *amax, float scale) {
  using compute_t = float;
  compute_t current_max = -1e100;
  for (size_t i = 0; i < size; ++i) {
      compute_t current = static_cast<compute_t>(data[i]);
      current_max = fmaxf(current_max, fabsf(current));
      output_c[i] = OutputType(scale * current);
  }
  *amax = current_max;
}

template <typename InputType, typename OutputType>
void performTest(const std::vector<size_t>& shape) {
  using namespace test;

  const size_t full_size = product(shape);

  DType itype = TypeInfo<InputType>::dtype;
  DType otype = TypeInfo<OutputType>::dtype;

  Tensor input(shape, itype);
  Tensor output_c(shape, otype);

  std::unique_ptr<OutputType[]> ref_output_c = std::make_unique<OutputType[]>(full_size);

  fillUniform(&input);
  setRandomScale(&output_c);

  nvte_quantize(input.data(), output_c.data(), 0);

  float ref_amax;
  compute_ref<InputType, OutputType>(input.rowwise_cpu_dptr<InputType>(), ref_output_c.get(),
                                     full_size, &ref_amax, output_c.scale());

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);
  if (isFp8Type(otype)) {
    auto [atol_amax, rtol_amax] = getTolerances(DType::kFloat32);
    compareResults("amax", output_c.amax(), ref_amax, atol_amax, rtol_amax);
    float ref_scale_inv = 1.f / output_c.scale();
    compareResults("scale_inv", output_c.rowwise_scale_inv(), ref_scale_inv, atol_amax, rtol_amax);
  }
  auto [atol, rtol] = getTolerances(otype);
  compareResults("output_c", output_c, ref_output_c.get(), true, atol, rtol);
}

std::vector<std::vector<size_t>> test_cases = {
  {16},
  {16000},
  {128, 128},
  {256, 256},
  {768, 1024},
  {256, 65536},
  {2048, 12288},
  {65536, 128},
  {65536, 160},
  {16384, 1616},
  {1, 128},
  {1, 1296},
  {1, 16},
  {5, 160},
  {5, 4, 3, 160},
  {217, 256},
};
}  // namespace

class CastTestSuite : public ::testing::TestWithParam<std::tuple<transformer_engine::DType,
                                                                 transformer_engine::DType,
                                                                 std::vector<size_t>>> {};

TEST_P(CastTestSuite, TestCast) {
  using namespace transformer_engine;
  using namespace test;

  const DType input_type = std::get<0>(GetParam());
  const DType output_type = std::get<1>(GetParam());
  const auto size = std::get<2>(GetParam());

  TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(input_type, InputType,
    TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(output_type, OutputType,
      performTest<InputType, OutputType>(size);
    );
  );
}



INSTANTIATE_TEST_SUITE_P(
  OperatorTest,
  CastTestSuite,
  ::testing::Combine(
      ::testing::Values(DType::kFloat32, DType::kBFloat16, DType::kFloat16),
      ::testing::Values(DType::kFloat8E4M3, DType::kFloat8E5M2),
      ::testing::ValuesIn(test_cases)),
  [](const testing::TestParamInfo<CastTestSuite::ParamType>& info) {
    std::string name = test::typeName(std::get<0>(info.param)) + "X" +
                       test::typeName(std::get<1>(info.param));
    const auto& shape = std::get<2>(info.param);
    for ( const auto& s: shape) {
      name += "X" + std::to_string(s);
    }
    return name;
  });
