#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <cmath>
#include <cstring>
#include <memory>
#include <iomanip>
#include <iostream>
#include <random>

#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <transformer_engine/cast.h>
#include "../test_common.h"

using namespace transformer_engine;
using namespace test;

namespace {

template <typename IT, typename OT, typename CT>
void compute_ref_cast_dbias_dgelu(const IT *input,
                                  const IT *gelu_input,
                                  const CT scale,
                                  OT *output_c,
                                  CT *amax_h,
                                  IT *dbias,
                                  const size_t N,
                                  const size_t H) {
  CT amax  = 0.;

  std::vector<CT> acc_dbias(H, 0.);

  for (size_t i = 0; i < N; i++) {
    for (size_t j = 0; j < H; j++) {
      CT in_elt = static_cast<CT>(input[i * H + j]);
      const CT gelu_in = static_cast<CT>(gelu_input[i * H + j]);

      const CT elt = in_elt * static_cast<float>(dgelu(static_cast<float>(gelu_in)));
      const CT elt_abs = std::abs(elt);

      // update amax
      if (elt_abs > amax) {
        amax = elt_abs;
      }

      output_c[i * H + j] = static_cast<OT>(scale * elt);

      // dbias
      acc_dbias[j] += elt;
    }
  }

  *amax_h = amax;

  for (size_t i = 0; i < H; i++) {
    dbias[i] = static_cast<IT>(acc_dbias[i]);
  }
}

template <typename IType, typename OType>
void performTest(const std::vector<size_t>& shape) {
  using namespace test;
  using CType = fp32;

  DType itype = TypeInfo<IType>::dtype;
  DType otype = TypeInfo<OType>::dtype;

  const size_t N = first_dimension(shape);
  const size_t H = last_dimension(shape);

  Tensor input(shape, itype);
  Tensor gelu_input(shape, itype);

  Tensor output_c(shape, otype);
  // dbias has the same data type with "output grad"
  Tensor dbias({H}, itype);

  fillUniform(&input);
  fillUniform(&gelu_input);
  setRandomScale(&output_c);

  std::unique_ptr<OType[]> ref_output_c = std::make_unique<OType[]>(N*H);
  std::unique_ptr<IType[]> ref_output_dbias = std::make_unique<IType[]>(H);

  CType ref_amax;
  compute_ref_cast_dbias_dgelu(input.rowwise_cpu_dptr<IType>(),
                               gelu_input.rowwise_cpu_dptr<IType>(),
                               output_c.scale(),
                               ref_output_c.get(),
                               &ref_amax,
                               ref_output_dbias.get(),
                               N, H);

  Tensor workspace;

  nvte_quantize_dbias_dgelu(input.data(),
                            gelu_input.data(),
                            output_c.data(),
                            dbias.data(),
                            workspace.data(),
                            0);

  workspace = Tensor(workspace.rowwise_shape(), workspace.dtype());


  nvte_quantize_dbias_dgelu(input.data(),
                            gelu_input.data(),
                            output_c.data(),
                            dbias.data(),
                            workspace.data(),
                            0);

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);

  if (isFp8Type(otype)) {
    auto [atol_amax, rtol_amax] = getTolerances(DType::kFloat32);
    compareResults("amax", output_c.amax(), ref_amax, atol_amax, rtol_amax);
    float ref_scale_inv = 1.f / output_c.scale();
    compareResults("scale_inv", output_c.rowwise_scale_inv(), ref_scale_inv, atol_amax, rtol_amax);
  }

  auto [atol, rtol] = getTolerances(otype);
  compareResults("output_c", output_c, ref_output_c.get(), true, atol, rtol);

  auto [atol_dbias, rtol_dbias] = getTolerances(itype);
  rtol_dbias *= 4;
  compareResults("output_dbias", dbias, ref_output_dbias.get(), true, atol_dbias, rtol_dbias);
}

std::vector<std::vector<size_t>> test_cases = {
  {128, 128},
  {256, 256},
  {768, 1024},
  {256, 65536},
  {2048, 12288},
  {65536, 128},
  {65536, 160},
  {16384, 1616},
  {1, 128},
  {1, 1296},
  {1, 16},
  {5, 160},
  {5, 4, 3, 160},
  {217, 256},
};

}  // namespace;


class CastDBiasDGeluTestSuite : public ::testing::TestWithParam<std::tuple<transformer_engine::DType,
                                                                           transformer_engine::DType,
                                                                           std::vector<size_t>>> {};

TEST_P(CastDBiasDGeluTestSuite, TestCastDBiasDgelu) {
    using namespace transformer_engine;
    using namespace test;
    // Skip tests for pre-Blackwell architectures
    if (getDeviceComputeCapability() < blackwellComputeCapability) {
        GTEST_SKIP();
    }

    const DType input_type = std::get<0>(GetParam());
    const DType output_type = std::get<1>(GetParam());
    const auto size = std::get<2>(GetParam());

    TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(input_type, InputType,
      TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(output_type, OutputType,
        performTest<InputType, OutputType>(size);
      );
    );
}

INSTANTIATE_TEST_SUITE_P(
    OperatorTest,
    CastDBiasDGeluTestSuite,
    ::testing::Combine(
        ::testing::Values(DType::kFloat32, DType::kBFloat16, DType::kFloat16),
        ::testing::Values(DType::kFloat8E4M3, DType::kFloat8E5M2),
        ::testing::ValuesIn(test_cases)),
    [](const testing::TestParamInfo<CastDBiasDGeluTestSuite::ParamType>& info) {
      std::string name = test::typeName(std::get<0>(info.param)) + "X" +
                         test::typeName(std::get<1>(info.param));
      const auto& shape = std::get<2>(info.param);
      for ( const auto& s: shape) {
        name += "X" + std::to_string(s);
      }
      return name;
    });
