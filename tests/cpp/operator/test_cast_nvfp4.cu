#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_fp4.h>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <transformer_engine/cast.h>
#include <transformer_engine/activation.h>
#include "../test_common.h"
#include "transformer_engine/transformer_engine.h"

using namespace transformer_engine;
using namespace test;

namespace {

enum ScalingType {
    ROWWISE = 0,
    COLWISE = 1,
    BIDIMENSIONAL = 2
};

enum ActivationType {
    Identity,
    GeLU,
    SiLU,
    ReLU,
    QGeLU,
    SReLU
};

float2 cvt_fp4x2_to_float2(fp4e2m1x2 fp4_pair) {
    const __half2_raw raw_truncated_to_fp4e2m1_pair =
        __hip_cvt_fp4x2_to_halfraw2(*reinterpret_cast<__hip_fp4x2_storage_t*>(&fp4_pair), __HIP_E2M1);

    const __half2 truncated_to_fp4e2m1_pair(raw_truncated_to_fp4e2m1_pair);
    const float truncated_to_fp4e2m1_x = static_cast<float>(truncated_to_fp4e2m1_pair.x);
    const float truncated_to_fp4e2m1_y = static_cast<float>(truncated_to_fp4e2m1_pair.y);
    return {truncated_to_fp4e2m1_x, truncated_to_fp4e2m1_y};
}

template <typename InputType, typename OutputType>
void compute_ref(const bool rowwise,
                 const bool colwise,
                 float (*OP)(const float),
                 const InputType* input,
                 fp4e2m1x2* output_rowwise_nvfp4,
                 OutputType* output_colwise_mxfp8,
                 fp8e4m3* scales_rowwise_nvfp4,
                 fp8e8m0* scales_colwise_mxfp8,
                 const float nvfp4_second_stage_scale,
                 const size_t rows,
                 const size_t cols,
                 const size_t scales_stride_rowwise,
                 const size_t scales_stride_colwise)
{
    const size_t tile_size_Y = 32;
    const size_t tile_size_X = 16;
    const size_t tiles_num_Y = (rows + tile_size_Y - 1) / tile_size_Y;
    const size_t tiles_num_X = (cols + tile_size_X - 1) / tile_size_X;

    // Compute a global encoding/decoding scaling factor for all S_dec_b 
    const float S_enc = 1.0f / nvfp4_second_stage_scale;

    printf("CPU S_enc: %f\n", S_enc);

    #pragma omp parallel proc_bind(spread)
    {
        // Buffers to cache intermediate computations
        std::vector<float> cache_buffer(tile_size_Y * tile_size_X);

        #pragma omp for schedule(static)
        for (size_t t = 0; t < tiles_num_Y * tiles_num_X; ++t) {
            const size_t tile_Y = t / tiles_num_X;
            const size_t tile_X = t % tiles_num_X;
            const size_t tile_offset_Y = tile_Y * tile_size_Y;
            const size_t tile_offset_X = tile_X * tile_size_X;

            const size_t i_min = tile_offset_Y;
            const size_t i_max = std::min(i_min + tile_size_Y, rows);

            const size_t j_min = tile_offset_X;
            const size_t j_max = std::min(j_min + tile_size_X, cols);

            // Cache computations
            for (size_t i = i_min; i < i_max; ++i) {
                for (size_t j = j_min; j < j_max; ++j) {
                    const int idx = i * cols + j;
                    const int cache_idx = (i - i_min) * tile_size_X + (j - j_min);

                    const float input_elt = static_cast<float>(input[idx]);
                    const float act_elt = OP(input_elt);


                    // Numerical truncation: after downcast to InputType (BF16/FP16), upcast it back to FP32
                    const float elt = static_cast<float>(static_cast<InputType>(act_elt));

                    cache_buffer[cache_idx] = elt;
                    // printf("Idx: %d Input: %f, Act: %f, Truncated: %f\n", idx, input_elt, act_elt, elt);
                    if (isinf(elt) || isnan(elt)) {
                        continue;
                    }
                }
                // printf("--------------------------------------------------------------------------\n\n");
            }

            if (rowwise) {
                for (size_t i = i_min; i < i_max; ++i) {
                    float block_amax = 0.0f;

                    for (size_t j = j_min; j < j_max; ++j) {
                        const int cache_idx = (i - i_min) * tile_size_X + (j - j_min);
                        block_amax = std::max(block_amax, std::abs(cache_buffer[cache_idx]));
                    }

                    // 2. Compute E4M3 scaling factor
                    // Compute per-block encoding/decoding scaling factor
                    const float S_dec_b = block_amax / 6.0f;

                    // Scale & Store per-block decoding scaling factor
                    const fp8e4m3 S_dec_b_fp8 = static_cast<fp8e4m3>(S_dec_b * S_enc);

                    // Compute "correct" per-block encoding scaling factor
                    const float S_enc_b_fp8 = S_enc / static_cast<float>(S_dec_b_fp8);

                    const int scale_idx = i * scales_stride_rowwise + tile_X;
                    scales_rowwise_nvfp4[scale_idx] = S_dec_b_fp8;
                    const float scale_reciprocal = S_enc_b_fp8;

                    // printf("Scale Reciprocal: %f\n", scale_reciprocal);
                    for (size_t j = j_min; j < j_max; j += 2) {
                        const int idx_pair = (i * cols + j) / 2;
                        const int cache_idx_x = (i - i_min) * tile_size_X + (j     - j_min);
                        const int cache_idx_y = (i - i_min) * tile_size_X + (j + 1 - j_min);
                        const float cached_x = cache_buffer[cache_idx_x];
                        const float cached_y = cache_buffer[cache_idx_y];
                        const float scaled_elt_x = cached_x * scale_reciprocal;
                        const float scaled_elt_y = cached_y * scale_reciprocal;
                        const float2 scaled_elt_pair = {scaled_elt_x, scaled_elt_y};

                        fp4e2m1x2 casted_to_e2m1_pair(scaled_elt_pair);
                        output_rowwise_nvfp4[idx_pair] = casted_to_e2m1_pair;

                        const float2 truncated_pair = cvt_fp4x2_to_float2(casted_to_e2m1_pair);
                        // printf("Idx: %d Cached: %f, Scaled: %f, Truncated to E2M1: %f\n", cache_idx_x, cached_x, scaled_elt_x, truncated_pair.x);
                        // printf("Idx: %d Cached: %f, Scaled: %f, Truncated to E2M1: %f\n", cache_idx_y, cached_y, scaled_elt_y, truncated_pair.y);
                    }
                    // printf("--------------------------------------------------------------------------\n\n");
                }
            }
            if (colwise) {
                for (size_t j = j_min; j < j_max; ++j) {
                    float block_amax = 0.0f;

                    for (size_t i = i_min; i < i_max; ++i) {
                        const int cache_idx = (i - i_min) * tile_size_X + (j - j_min);
                        block_amax = std::max(block_amax, std::abs(cache_buffer[cache_idx]));
                    }

                    const fp8e8m0 biased_exponent = float_to_e8m0(block_amax * Quantized_Limits<OutputType>::max_reciprocal());
                    const int scale_idx = tile_Y * scales_stride_colwise + j;
                    scales_colwise_mxfp8[scale_idx] = biased_exponent;
                    const float scale_reciprocal = exp2f_rcp(biased_exponent);

                    for (size_t i = i_min; i < i_max; ++i) {
                        const int idx = i * cols + j;
                        const int cache_idx = (i - i_min) * tile_size_X + (j - j_min);
                        output_colwise_mxfp8[idx] = static_cast<OutputType>(cache_buffer[cache_idx] * scale_reciprocal);
                    }
                }
            }
        }
    }
}


void compareResults_nvfp4(const std::string &name, const Tensor &test,
                          const void *ref, const bool rowwise,
                          double atol = 1e-5, double rtol = 1e-8, bool if_on_gpus = true) {

  const std::string direction = rowwise ? "rowwise" : "colwise";

  if (if_on_gpus) test.to_cpu();
  const auto& shape = rowwise ? test.rowwise_shape() : test.columnwise_shape();
  const size_t N = product(shape);
  const fp4e2m1 *test_data = rowwise
                             ? test.rowwise_cpu_dptr<fp4e2m1>()
                             : test.columnwise_cpu_dptr<fp4e2m1>();
  const fp4e2m1 *ref_data = reinterpret_cast<const fp4e2m1*>(ref);
  for (size_t i = 0; i < N; i += 2) {

    const __hip_fp4x2_storage_t* test_raw_storage = reinterpret_cast<const __hip_fp4x2_storage_t*>(&test_data[i/2]);
    const __hip_fp4x2_storage_t* ref_raw_storage = reinterpret_cast<const __hip_fp4x2_storage_t*>(&ref_data[i/2]);

    const __half2_raw test_data_pair_raw = __hip_cvt_fp4x2_to_halfraw2(*test_raw_storage, __HIP_E2M1);
    const __half2_raw ref_data_pair_raw = __hip_cvt_fp4x2_to_halfraw2(*ref_raw_storage, __HIP_E2M1);

    const __half2 test_data_pair(test_data_pair_raw);
    const __half2 ref_data_pair(ref_data_pair_raw);

    for (int k = 0; k < 2; ++k) {
        const double t = static_cast<double>(k == 0 ? test_data_pair.x : test_data_pair.y);
        const double r = static_cast<double>(k == 0 ? ref_data_pair.x : ref_data_pair.y);

        bool mismatch = fabs(t - r) > atol && (r == 0 || fabs((t - r) / r) > rtol);
        /* For Float32 the floating point comparison is enough to error out */
        bool assertion = mismatch && test.dtype() == DType::kFloat32;
        if (mismatch && !assertion) {
          /* Check if it is just a failure of round to nearest choosing different
             side of the real value */
          const double mean = (t + r) / 2;
          const double mean_p = mean >= 0 ? mean * (1 + 1e-6) : mean * (1 - 1e-6);
          const double mean_m = mean >= 0 ? mean * (1 - 1e-6) : mean * (1 + 1e-6);
          const double cast_mean_p = static_cast<double>(static_cast<fp4e2m1>(mean_p));
          const double cast_mean_m = static_cast<double>(static_cast<fp4e2m1>(mean_m));
          assertion = !(cast_mean_m == std::min(t,r) && cast_mean_p == std::max(t,r));
        }
        // printf("%3lu GPU: %6.2f    CPU: %6.2f\n", i + k, t, r);
        if (assertion) {
            ASSERT_FALSE(assertion) << "Error in tensor " << name << " in "
                                    << direction << " direction." << std::endl
                                    << "Mismatch at place " 
                                    << " (" << std::to_string(i + k) << "): "
                                    << t << " vs " << r;
        }
    }
  }
}


/**
 * Scaling along selected dimensions (rows and/or columns)
 * Produces sets of output data:
 * 1) NVFP4 Scaled rows + E4M3 row-wise scaling factors
 *       AND/OR
 * 2) MXFP8 Scaled columns + E8M0 column-wise scaling factors
 */

template <typename InputType, typename OutputType>
void performTest(float (*OP)(const float),
                 const std::vector<size_t>& shape,
                 const bool rowwise,
                 const bool colwise,
                 InputsFillCase fill_case) {
    using namespace test;
    using EncodingType = fp32;
    DType itype = TypeInfo<InputType>::dtype;
    DType otype = rowwise ? TypeInfo<fp4e2m1>::dtype : TypeInfo<OutputType>::dtype;

    const size_t rows = first_dimension(shape);
    const size_t cols = last_dimension(shape);

    const std::array<size_t,4> scale_dims_rowwise = get_scale_tensor_dims(rows, cols, 1, 16);
    const std::array<size_t,4> scale_dims_colwise = get_scale_tensor_dims(rows, cols, 32, 1);

    const size_t unpadded_blocks_Y_rowwise = scale_dims_rowwise[0];
    const size_t unpadded_blocks_X_rowwise = scale_dims_rowwise[1];
    const size_t blocks_Y_rowwise = scale_dims_rowwise[2];
    const size_t blocks_X_rowwise = scale_dims_rowwise[3];
    const size_t scales_stride_rowwise = blocks_X_rowwise;

    const size_t unpadded_blocks_Y_colwise = scale_dims_colwise[0];
    const size_t unpadded_blocks_X_colwise = scale_dims_colwise[1];
    const size_t blocks_Y_colwise = scale_dims_colwise[2];
    const size_t blocks_X_colwise = scale_dims_colwise[3];
    const size_t scales_stride_colwise = blocks_X_colwise;

    Tensor input("input", shape, itype);
    Tensor output("output", shape, otype, rowwise, colwise, NVTE_FWD_NVFP4_BWD_MXFP8_SCALING);

    std::unique_ptr<fp4e2m1x2[]> ref_output_nvfp4;
    std::unique_ptr<OutputType[]> ref_output_mxfp8;
    std::unique_ptr<fp8e4m3[]> ref_scales_nvfp4;
    std::unique_ptr<fp8e8m0[]> ref_scales_mxfp8;

    if (rowwise) {
        ref_output_nvfp4 = std::make_unique<fp4e2m1x2[]>(rows * cols / 2);
        ref_scales_nvfp4 = std::make_unique<fp8e4m3[]>(blocks_Y_rowwise * blocks_X_rowwise);
    }
    if (colwise) {
        ref_output_mxfp8 = std::make_unique<OutputType[]>(rows * cols);
        ref_scales_mxfp8 = std::make_unique<fp8e8m0[]>(blocks_Y_colwise * blocks_X_colwise);
    }

    fillCase<EncodingType>(&input, fill_case);
    setRandomScale(&output);

    auto nvte_quantize_operation = &nvte_quantize;
    if (OP == &gelu)       { nvte_quantize_operation = &nvte_gelu; }
    else if (OP == &silu)  { nvte_quantize_operation = &nvte_silu; }
    else if (OP == &relu)  { nvte_quantize_operation = &nvte_relu; }
    else if (OP == &qgelu) { nvte_quantize_operation = &nvte_qgelu; }
    else if (OP == &srelu) { nvte_quantize_operation = &nvte_srelu; }

    nvte_quantize_operation(input.data(), output.data(), 0);

    hipDeviceSynchronize();
    auto err = hipGetLastError();
    ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);

    compute_ref<InputType, OutputType>(rowwise,
                                       colwise,
                                       OP,
                                       input.rowwise_cpu_dptr<InputType>(),
                                       ref_output_nvfp4.get(),
                                       ref_output_mxfp8.get(),
                                       ref_scales_nvfp4.get(),
                                       ref_scales_mxfp8.get(),
                                       output.scale(),
                                       rows,
                                       cols,
                                       scales_stride_rowwise,
                                       scales_stride_colwise);

    if (rowwise) {
        const double atol = 0.05;
        const double rtol = 0.1;
        const bool rowwise_true = true;
        compareResults_nvfp4("output_nvfp4", output, ref_output_nvfp4.get(), rowwise_true, atol, rtol);
    
        size_t scale_mismatches_num = 0;
        compare_scaling_factors("rowwise_scales_E4M3", output.rowwise_cpu_scale_inv_ptr<fp8e4m3>(),
                                ref_scales_nvfp4.get(),
                                unpadded_blocks_Y_rowwise, unpadded_blocks_X_rowwise, scales_stride_rowwise,
                                scale_mismatches_num);
        printf("scale_mismatches_num: %lu \n", scale_mismatches_num);
    }
    if (colwise) {
        const size_t scale_diff_abs_tolerance = 0;
        const double abs_tolerable_mismatches_limit = 0.0;
        const double rel_tolerable_mismatches_limit = 0.0;

        size_t mismatches_scales = 0;
        compare_scaling_factors("colwise_scales_E8M0", output.columnwise_cpu_scale_inv_ptr<fp8e8m0>(),
                                ref_scales_mxfp8.get(),
                                unpadded_blocks_Y_colwise, unpadded_blocks_X_colwise, scales_stride_colwise,
                                mismatches_scales,
                                scale_diff_abs_tolerance,
                                abs_tolerable_mismatches_limit,
                                rel_tolerable_mismatches_limit);

        const size_t mismatches_elts_limit = 32 * mismatches_scales;
        auto [atol, rtol] = getTolerances(otype);
        const bool rowwise_false = false;
        compareResults("output_mxfp8", output, ref_output_mxfp8.get(), rowwise_false, atol, rtol, true, mismatches_elts_limit);
    }
}

std::vector<std::vector<size_t>> matrix_sizes = {
    {1, 32},
    // {1, 32},
    // {16, 48},    
    // {65, 96},
    // {128, 128},
    // {256, 256},
    // {993, 512},
    // {256, 65536},
    // {2048, 6144},
    // {16384, 128},
    // {32768, 160},
    // {4096, 1632},
    // {1024},
    // {8, 32, 1024},
    // {16, 8, 4, 512},
    // {1024, 16384},
    // {4096, 13312},
};

std::vector<ScalingType> scaling_case = {
    ScalingType::ROWWISE,           // Row-wise NVFP4{1, 16}
    // ScalingType::COLWISE,           // Column-wise MXFP8 {32, 1},
    // ScalingType::BIDIMENSIONAL      // {32, 16} Row-wise NVFP4 AND Column-wise MXFP8
};

std::vector<InputsFillCase> input_scenarios = {
    InputsFillCase::uniform,
    // InputsFillCase::zeros,
    // InputsFillCase::zero_to_minNorm,
    // InputsFillCase::minNorm_to_maxNorm,
    // InputsFillCase::maxNorm_to_inf
};

// Only GeLU activation tests are supported
std::vector<ActivationType> Activation_types = {
    ActivationType::Identity,
    // ActivationType::GeLU,
    // ActivationType::SiLU,
    // ActivationType::ReLU,
    // ActivationType::QGeLU,
    // ActivationType::SReLU,
};

}  // namespace

class FusedCastNVFP4TestSuite : public ::testing::TestWithParam
    <std::tuple<ActivationType,
                std::vector<size_t>,
                ScalingType,
                transformer_engine::DType,
                transformer_engine::DType,
                InputsFillCase>> {};

TEST_P(FusedCastNVFP4TestSuite, TestFusedCastNVFP4) {
    // Skip tests for pre-Blackwell architectures
    if (getDeviceComputeCapability() < blackwellComputeCapability) {
        GTEST_SKIP();
    }

    using namespace transformer_engine;
    using namespace test;

    const ActivationType Act_type = std::get<0>(GetParam());
    const auto tensor_dims = std::get<1>(GetParam());
    const ScalingType scaling_case = std::get<2>(GetParam());
    const DType input_type = std::get<3>(GetParam());
    const DType output_type = std::get<4>(GetParam());
    const InputsFillCase fill_case = std::get<5>(GetParam());

    const bool rowwise = (scaling_case == ScalingType::ROWWISE) || (scaling_case == ScalingType::BIDIMENSIONAL);
    const bool colwise = (scaling_case == ScalingType::COLWISE) || (scaling_case == ScalingType::BIDIMENSIONAL);

    // Skip tests with colwise scaling, if the input tensor is 1D  
    if (tensor_dims.size() < 2 && colwise) {
        GTEST_SKIP();
    }

    // Forward activations
    auto OP = &identity;
    switch (Act_type) {
        case ActivationType::GeLU: OP = &gelu; break;
        case ActivationType::SiLU: OP = &silu; break;
        case ActivationType::ReLU: OP = &relu; break;
        case ActivationType::QGeLU: OP = &qgelu; break;
        case ActivationType::SReLU: OP = &srelu; break;
    }

    TRANSFORMER_ENGINE_TYPE_SWITCH_FP16_FP32_ONLY(input_type, InputType,
        TRANSFORMER_ENGINE_TYPE_SWITCH_FP8_ONLY(output_type, OutputType,
            performTest<InputType, OutputType>(OP, tensor_dims, rowwise, colwise, fill_case);
        );
    );
}

std::string to_string(const ActivationType Act_type) {
    switch (Act_type) {
        case ActivationType::Identity:  return "CAST_ONLY";
        case ActivationType::GeLU:      return "GeLU";
        case ActivationType::SiLU:      return "SiLU";
        case ActivationType::ReLU:      return "ReLU";
        case ActivationType::QGeLU:     return "QGeLU";
        case ActivationType::SReLU:     return "SReLU";
        default: return "";
    }
}

std::string to_string(const ScalingType scaling_type) {
    switch (scaling_type) {
        case ScalingType::ROWWISE:       return "ROWWISE_NVFP4_1x16";
        case ScalingType::COLWISE:       return "COLWISE_MXFP8_32x1";
        case ScalingType::BIDIMENSIONAL: return "BIDIMENSIONAL_32x16";
        default: return "";
    }
}

INSTANTIATE_TEST_SUITE_P(
    OperatorTest,
    FusedCastNVFP4TestSuite,
    ::testing::Combine(
        ::testing::ValuesIn(Activation_types),
        ::testing::ValuesIn(matrix_sizes),
        ::testing::ValuesIn(scaling_case),
        // ::testing::Values(DType::kFloat32, DType::kBFloat16, DType::kFloat16),
        // ::testing::Values(DType::kFloat8E4M3, DType::kFloat8E5M2),
        ::testing::Values(DType::kBFloat16),
        ::testing::Values(DType::kFloat8E4M3),
        ::testing::ValuesIn(input_scenarios)),
    [](const testing::TestParamInfo<FusedCastNVFP4TestSuite::ParamType>& info) {
        std::string name = to_string(std::get<0>(info.param));
      const auto& shape = std::get<1>(info.param);
      for ( const auto& s: shape) {
        name += "X" + std::to_string(s);
      }
      name += "X" + to_string(std::get<2>(info.param)) +
              "X" + test::typeName(std::get<3>(info.param)) +
              "X" + test::typeName(std::get<4>(info.param)) +
              "X" + test::caseName(std::get<5>(info.param));
        return name;
    });
