#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <hip/hip_bf16.h>
#include <cuda_pipeline.h>
#include <hip/hip_runtime.h>
#include <transformer_engine/hadamard_transform.h>

#include <cuda/barrier>

#include "common/common.h"
#include "common/util/ptx.cuh"
#include "common/utils.cuh"

namespace transformer_engine {
namespace {

constexpr int kThreadsPerWarp = 32;
constexpr float k16x16HadamardScale = 0.25f;

template <bool kTranspose>
__device__ __forceinline__ void ldmatrix_x4_m8n8_shared_b16(uint32_t& a0, uint32_t& a1,
                                                            uint32_t& a2, uint32_t& a3,
                                                            void* addr) {
  auto smem_addr = static_cast<uint32_t>(__cvta_generic_to_shared(addr));
  if constexpr (kTranspose) {
    asm volatile("ldmatrix.sync.aligned.x4.trans.m8n8.shared.b16 {%0,%1,%2,%3}, [%4];\n"
                 : "=r"(a0), "=r"(a1), "=r"(a2), "=r"(a3)
                 : "r"(smem_addr));
  } else {
    asm volatile("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0,%1,%2,%3}, [%4];\n"
                 : "=r"(a0), "=r"(a1), "=r"(a2), "=r"(a3)
                 : "r"(smem_addr));
  }
}

template <bool kTranspose>
__device__ __forceinline__ void load_matrix_16x16_from_shared(uint32_t& a0, uint32_t& a1,
                                                              uint32_t& a2, uint32_t& a3,
                                                              void* addr, uint32_t stride) {
  if constexpr (kTranspose) {
    asm volatile(
        "wmma.load.a.sync.aligned.col.m16n16k16.shared::cta.bf16 "
        "{%0,%1,%2,%3}, [%4], %5;\n"
        : "=r"(a0), "=r"(a1), "=r"(a2), "=r"(a3)
        : "l"(addr), "r"(stride));
  } else {
    asm volatile(
        "wmma.load.a.sync.aligned.row.m16n16k16.shared::cta.bf16 "
        "{%0,%1,%2,%3}, [%4], %5;\n"
        : "=r"(a0), "=r"(a1), "=r"(a2), "=r"(a3)
        : "l"(addr), "r"(stride));
  }
}

template <bool kTranspose>
__device__ __forceinline__ void store_matrix_16x16_to_global(uint32_t& a0, uint32_t& a1,
                                                             uint32_t& a2, uint32_t& a3, void* addr,
                                                             uint32_t stride) {
  if constexpr (kTranspose) {
    asm volatile("wmma.store.d.sync.aligned.col.m16n16k16.global.f16 [%0], {%1, %2, %3, %4}, %5;\n"
                 :
                 : "l"(addr), "r"(a0), "r"(a1), "r"(a2), "r"(a3), "r"(stride));
  } else {
    asm volatile("wmma.store.d.sync.aligned.row.m16n16k16.global.f16 [%0], {%1, %2, %3, %4}, %5;\n"
                 :
                 : "l"(addr), "r"(a0), "r"(a1), "r"(a2), "r"(a3), "r"(stride));
  }
}

__device__ __forceinline__ void matrix_transpose_m8_n8_b16_inplace(uint32_t& a0) {
  asm volatile(
      "movmatrix.sync.aligned.m8n8.trans.b16 "
      "%0, %1;\n\t"
      : "=r"(a0)
      : "r"(a0));
}

__device__ __forceinline__ void unpack_max_of_packed_bf16(uint32_t& packed_bf16, float& float_dst) {
  __hip_bfloat162 bf16x2 = *reinterpret_cast<__hip_bfloat162*>(&packed_bf16);
  float f_a = __bfloat162float(bf16x2.x);
  float f_b = __bfloat162float(bf16x2.y);
  asm volatile("max.xorsign.abs.f32 %0, %1, %2;\n\t" : "=f"(float_dst) : "f"(f_a), "f"(f_b));
  float_dst = fabsf(float_dst);
}

template <bool kCalculateAmax>
__device__ __forceinline__ void mma_m16_n16_k16_b16_b16_b16_noacc(
    uint32_t& a0, uint32_t& a1, uint32_t& a2, uint32_t& a3, uint32_t& b0, uint32_t& b1,
    uint32_t& b2, uint32_t& b3, uint32_t& c0, uint32_t& c1, uint32_t& c2, uint32_t& c3,
    uint32_t& amax_result) {
  uint32_t zero = 0;
  uint32_t temp0, temp1, temp2, temp3, temp4, temp5, temp6, temp7;
  asm volatile(
      "wmma.mma.sync.aligned.row.row.m16n16k16.f32.bf16.bf16.f32 \n"
      "{%0, %1, %2, %3, %4, %5, %6, %7}, \n"
      "{%8, %9, %10, %11}, \n"
      "{%12, %13, %14, %15}, \n"
      "{%16, %17, %18, %19, %20, %21, %22, %23};\n\t"
      : "=r"(temp0), "=r"(temp1), "=r"(temp2), "=r"(temp3), "=r"(temp4), "=r"(temp5), "=r"(temp6),
        "=r"(temp7)
      : "r"(a0), "r"(a1), "r"(a2), "r"(a3), "r"(b0), "r"(b1), "r"(b2), "r"(b3), "r"(zero),
        "r"(zero), "r"(zero), "r"(zero), "r"(zero), "r"(zero), "r"(zero), "r"(zero));
  asm volatile("cvt.rn.bf16x2.f32 %0, %1, %2;\n\t" : "=r"(c0) : "r"(temp1), "r"(temp0));
  asm volatile("cvt.rn.bf16x2.f32 %0, %1, %2;\n\t" : "=r"(c1) : "r"(temp3), "r"(temp2));
  asm volatile("cvt.rn.bf16x2.f32 %0, %1, %2;\n\t" : "=r"(c2) : "r"(temp5), "r"(temp4));
  asm volatile("cvt.rn.bf16x2.f32 %0, %1, %2;\n\t" : "=r"(c3) : "r"(temp7), "r"(temp6));
  if constexpr (kCalculateAmax) {
    uint32_t max_even;
    uint32_t max_odd;
    // Reduction tree to amax(abs(result)) into bf16x2 reg outparam.
    asm volatile("max.xorsign.abs.bf16x2 %0, %1, %2;\n\t" : "=r"(max_even) : "r"(c0), "r"(c2));
    asm volatile("max.xorsign.abs.bf16x2 %0, %1, %2;\n\t" : "=r"(max_odd) : "r"(c1), "r"(c3));
    // N.B. mma is only called up to once per thread for identity and transpose respectively, so
    // we don't have to accumulate into amax_result and can directly store into it.
    asm volatile("max.xorsign.abs.bf16x2 %0, %1, %2;\n\t"
                 : "=r"(amax_result)
                 : "r"(max_even), "r"(max_odd));
  }
}

template <bool kReturnIdentity, bool kReturnTransposed, bool kInverseHadamardIdentity,
          bool kInverseHadamardTransposed>
__device__ __forceinline__ void get_hadamard_matrix_fragment(uint32_t* had_frag_i,
                                                             uint16_t random_sign_mask,
                                                             uint32_t* had_frag_t,
                                                             uint16_t random_sign_mask_t) {
  int32_t tid = threadIdx.x % 32;  // Local tid
  float temp_i[2];
  float temp_t[2];
#pragma unroll
  for (int i = 0; i < 2; i++) {
    // i is the vertical fragment index.
    // For a 16x16 matrix matrix fragment, 4 threads fill a fragment of 8 BF16 vals.
    uint32_t r = i * 8 + tid / 4;

#pragma unroll
    for (int j = 0; j < 2; j++) {
#pragma unroll
      for (int k = 0; k < 2; k++) {
        // k is column position [0, 1] within a quad of 2 BF16s  stored together in 32 bits.
        // j is the column fragment idx selecting between even and odd fragments.
        // j increments 8 columns by switching fragments.
        uint32_t c = j * 8 + k + tid % 4 * 2;
        // 1 -> -1.0f, 0 -> 1.0f
        int32_t base_sign = __popc(r & c);
        if constexpr (kReturnIdentity) {
          int32_t sign_i;
          // Because tensor cores want the dot product dimension,
          // contiguous, the regular, non-inverse hadamard swaps
          // signs of columns and rows for inverse. In a simple reference,
          // x.reshape(-1, 16) @ sign @ H16, this would be opposite but
          // (sign @ H16) is transposed in this fragment.
          if constexpr (kInverseHadamardIdentity) {
            sign_i = ((random_sign_mask >> r) ^ base_sign);
          } else {
            sign_i = ((random_sign_mask >> c) ^ base_sign);
          }
          temp_i[k] = copysignf(k16x16HadamardScale, __int_as_float(sign_i << 31));
        }
        if constexpr (kReturnTransposed) {
          int32_t sign_t;
          if constexpr (kInverseHadamardTransposed) {
            sign_t = ((random_sign_mask_t >> r) ^ base_sign);
          } else {
            sign_t = ((random_sign_mask_t >> c) ^ base_sign);
          }
          temp_t[k] = copysignf(k16x16HadamardScale, __int_as_float(sign_t << 31));
        }
      }

      if constexpr (kReturnIdentity) {
        asm volatile("cvt.rn.bf16x2.f32 %0, %1, %2;\n\t"
                     : "=r"(had_frag_i[i * 2 + j])
                     : "f"(temp_i[1]), "f"(temp_i[0]));
      }
      if constexpr (kReturnTransposed) {
        asm volatile("cvt.rn.bf16x2.f32 %0, %1, %2;\n\t"
                     : "=r"(had_frag_t[i * 2 + j])
                     : "f"(temp_t[1]), "f"(temp_t[0]));
      }
    }
  }
}

__device__ __forceinline__ uint32_t swizzle_128B_atom_32B(uint32_t gmem_row_idx,
                                                          uint32_t gmem_col_idx) {
  uint32_t smem_row_idx = gmem_row_idx;
  uint32_t xor_factor = (smem_row_idx * 2) % 8;
  uint32_t smem_col_idx = gmem_col_idx ^ xor_factor;
  return smem_row_idx * 8 + smem_col_idx;
}

template <typename IType, int kHadamardDimension, int BUFF_DIM_Y, int BUFF_DIM_X,
          bool kReturnPreRhtAmax, bool kReturnIdentityAmax, bool kReturnTransposedAmax>
__device__ __forceinline__ void ComputeKernel(uint32_t b_frag_i[4], uint32_t b_frag_t[4],
                                              IType* in_sh_ptr, uint32_t& local_pre_rht_amax_reg,
                                              uint32_t& local_amax_reg,
                                              uint32_t& local_amax_t_reg) {
  uint32_t a_frag[4];  // A matrix fragment
  uint32_t c_frag[4];  // Result fragment

  int warp_id = threadIdx.x / kThreadsPerWarp;
  int local_rank = (threadIdx.x % kThreadsPerWarp);

  int ld_row_idx = local_rank % kHadamardDimension;
  int ld_col_idx = local_rank / kHadamardDimension + warp_id * 2;
  int swizzle_idx = swizzle_128B_atom_32B(ld_row_idx, ld_col_idx);

  uint32_t temp_amax_reg;
  uint32_t temp_amax_t_reg;

  if (kReturnIdentityAmax) {
    ldmatrix_x4_m8n8_shared_b16<false>(a_frag[0], a_frag[1], a_frag[2], a_frag[3],
                                       reinterpret_cast<uint4*>(in_sh_ptr) + swizzle_idx);

    mma_m16_n16_k16_b16_b16_b16_noacc<kReturnIdentityAmax>(
        a_frag[0], a_frag[1], a_frag[2], a_frag[3], b_frag_i[0], b_frag_i[1], b_frag_i[2],
        b_frag_i[3], c_frag[0], c_frag[1], c_frag[2], c_frag[3], temp_amax_reg);
    asm volatile("max.xorsign.abs.bf16x2 %0, %1, %2;\n\t"
                 : "=r"(local_amax_reg)
                 : "r"(local_amax_reg), "r"(temp_amax_reg));
  }

  if (kReturnTransposedAmax) {
    // TODO(Frank): This is not efficient, since we could directly load the
    // matrix in transposed layout.
    if (!kReturnIdentityAmax) {
      ldmatrix_x4_m8n8_shared_b16<false>(a_frag[0], a_frag[1], a_frag[2], a_frag[3],
                                         reinterpret_cast<uint4*>(in_sh_ptr) + swizzle_idx);
    }

    matrix_transpose_m8_n8_b16_inplace(a_frag[0]);
    matrix_transpose_m8_n8_b16_inplace(a_frag[1]);
    matrix_transpose_m8_n8_b16_inplace(a_frag[2]);
    matrix_transpose_m8_n8_b16_inplace(a_frag[3]);

    mma_m16_n16_k16_b16_b16_b16_noacc<kReturnTransposedAmax>(
        a_frag[0], a_frag[2], a_frag[1], a_frag[3], b_frag_t[0], b_frag_t[1], b_frag_t[2],
        b_frag_t[3], c_frag[0], c_frag[1], c_frag[2], c_frag[3], temp_amax_t_reg);
    asm volatile("max.xorsign.abs.bf16x2 %0, %1, %2;\n\t"
                 : "=r"(local_amax_t_reg)
                 : "r"(local_amax_t_reg), "r"(temp_amax_t_reg));
  }

  if (kReturnPreRhtAmax) {
    if (!kReturnIdentityAmax && !kReturnTransposedAmax) {
      ldmatrix_x4_m8n8_shared_b16<false>(a_frag[0], a_frag[1], a_frag[2], a_frag[3],
                                         reinterpret_cast<uint4*>(in_sh_ptr) + swizzle_idx);
    }

    asm volatile("max.xorsign.abs.bf16x2 %0, %1, %2;\n\t"
                 : "=r"(a_frag[0])
                 : "r"(a_frag[0]), "r"(a_frag[1]));
    asm volatile("max.xorsign.abs.bf16x2 %0, %1, %2;\n\t"
                 : "=r"(a_frag[2])
                 : "r"(a_frag[2]), "r"(a_frag[3]));
    asm volatile("max.xorsign.abs.bf16x2 %0, %1, %2;\n\t"
                 : "=r"(a_frag[0])
                 : "r"(a_frag[0]), "r"(a_frag[2]));
    asm volatile("max.xorsign.abs.bf16x2 %0, %1, %2;\n\t"
                 : "=r"(local_pre_rht_amax_reg)
                 : "r"(a_frag[0]), "r"(local_pre_rht_amax_reg));
  }
}

template <int kN>
__device__ __host__ constexpr int NextPowerOf2() {
  static_assert(kN > 0, "kN must be > 0");
  // Round up to the next power of 2 by counting leading zeros.
  return 1 << (32 - __builtin_clz(kN - 1));
}

template <int kNumWarps, bool kReturnPreRhtAmax, bool kReturnIdentityAmax,
          bool kReturnTransposedAmax>
__device__ __forceinline__ void ReduceMax(const float pre_rht_amax, const float identity_amax,
                                          const float transpose_amax, float* staging_for_pre_rht,
                                          float* staging_for_identity, float* staging_for_transpose,
                                          float* output_pre_rht_amax_ptr,
                                          float* output_identity_amax_ptr,
                                          float* output_transpose_amax_ptr, const int warpid) {
  // intra-warp reduction
  constexpr int kWarpSize = 32;
  int local_rank = threadIdx.x % 32;
  float warp_pre_rht_amax = kReturnPreRhtAmax ? warp_reduce_max<kWarpSize>(pre_rht_amax) : 0.0f;
  float warp_identity_amax = kReturnIdentityAmax ? warp_reduce_max<kWarpSize>(identity_amax) : 0.0f;
  float warp_transpose_amax =
      kReturnTransposedAmax ? warp_reduce_max<kWarpSize>(transpose_amax) : 0.0f;

  // inter-warp reduction
  if (threadIdx.x % 32 == 0) {
    if (kReturnPreRhtAmax) {
      staging_for_pre_rht[warpid] = warp_pre_rht_amax;
    }
    if (kReturnIdentityAmax) {
      staging_for_identity[warpid] = warp_identity_amax;
    }
    if (kReturnTransposedAmax) {
      staging_for_transpose[warpid] = warp_transpose_amax;
    }
  }
  __syncthreads();
  constexpr int kNumWarpsPow2 = NextPowerOf2<kNumWarps>();
  if (warpid == 0) {
    if (kReturnIdentityAmax) {
      float identity_accum = local_rank < kNumWarps ? staging_for_identity[local_rank] : 0.0f;
      identity_accum = warp_reduce_max<kNumWarpsPow2>(identity_accum);
      if (local_rank == 0) {
        atomicMaxFloat(output_identity_amax_ptr, identity_accum);
      }
    }
  }
  if (warpid == 1) {
    if (kReturnTransposedAmax) {
      float transpose_accum = local_rank < kNumWarps ? staging_for_transpose[local_rank] : 0.0f;
      transpose_accum = warp_reduce_max<kNumWarpsPow2>(transpose_accum);
      if (local_rank == 0) {
        atomicMaxFloat(output_transpose_amax_ptr, transpose_accum);
      }
    }
  }
  if (warpid == 2) {
    if (kReturnPreRhtAmax) {
      float pre_rht_accum = local_rank < kNumWarps ? staging_for_pre_rht[local_rank] : 0.0f;
      pre_rht_accum = warp_reduce_max<kNumWarpsPow2>(pre_rht_accum);
      if (local_rank == 0) {
        atomicMaxFloat(output_pre_rht_amax_ptr, pre_rht_accum);
      }
    }
  }
}

__launch_bounds__(1) __global__ void ZeroAmaxKernel(float* __restrict__ output_pre_rht_amax_ptr,
                                                    float* __restrict__ output_identity_amax_ptr,
                                                    float* __restrict__ output_transpose_amax_ptr) {
  if (output_pre_rht_amax_ptr != nullptr) {
    *output_pre_rht_amax_ptr = 0;
  }
  if (output_identity_amax_ptr != nullptr) {
    *output_identity_amax_ptr = 0;
  }
  if (output_transpose_amax_ptr != nullptr) {
    *output_transpose_amax_ptr = 0;
  }
}

template <typename IType, int kHadamardDimension, int CHUNK_DIM_Y, int CHUNK_DIM_X, int BUFF_DIM_Y,
          int BUFF_DIM_X, int THREADS_PER_CHUNK, int THREADS_PER_Y, bool kReturnPreRhtAmax,
          bool kReturnIdentityAmax, bool kReturnTransposedAmax>
__global__ void HadamardAmaxTmaKernel(const __grid_constant__ CUtensorMap tensor_map_input,
                                      float* __restrict__ output_pre_rht_amax_ptr,
                                      float* __restrict__ output_identity_amax_ptr,
                                      float* __restrict__ output_transpose_amax_ptr,
                                      uint16_t random_sign_mask, uint16_t random_sign_mask_t,
                                      uint64_t num_rows, uint64_t row_length) {
#if (defined __CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)

  static_assert(CHUNK_DIM_Y >= BUFF_DIM_Y && CHUNK_DIM_Y % BUFF_DIM_Y == 0);
  static_assert(CHUNK_DIM_X >= BUFF_DIM_X && CHUNK_DIM_X % BUFF_DIM_X == 0);

  constexpr size_t STAGES_Y = CHUNK_DIM_Y / BUFF_DIM_Y;
  constexpr size_t STAGES_X = CHUNK_DIM_X / BUFF_DIM_X;

  constexpr int kNumWarps = (THREADS_PER_CHUNK * THREADS_PER_Y) / kThreadsPerWarp;

  const int input_block_offset_Y = blockIdx.y * CHUNK_DIM_Y;
  const int input_block_offset_X = blockIdx.x * CHUNK_DIM_X;

  extern __shared__ __align__(128) char dynamic_shmem[];
  uintptr_t base_shmem_ptr = reinterpret_cast<uintptr_t>(dynamic_shmem);
  // Manually align dynamic SHMEM per TMA requirements using padding
  // __align__(128) Does not guarantee the pointer to be aligned!
  uint8_t* dshmem = reinterpret_cast<uint8_t*>((base_shmem_ptr + 127) & ~127ULL);

  // The destination shared memory buffer of a bulk tensor operation should be 16-byte aligned
  constexpr size_t in_buff_size = BUFF_DIM_X * BUFF_DIM_Y * sizeof(IType);
  IType* in_sh_0 = reinterpret_cast<IType*>(dshmem);
  dshmem += in_buff_size;
  IType* in_sh_1 = reinterpret_cast<IType*>(dshmem);
  dshmem += in_buff_size;

  IType* in_shs[2] = {in_sh_0, in_sh_1};

  constexpr int shmem_buff_size = BUFF_DIM_X * BUFF_DIM_Y * sizeof(IType);

  const bool is_master_thread = (threadIdx.x == 0 && threadIdx.y == 0);

  // Initialize shared memory barrier with the number of threads participating in the barrier.
#pragma nv_diag_suppress static_var_with_dynamic_init
  uint64_t* mbar = reinterpret_cast<uint64_t*>(dshmem);
  dshmem += sizeof(uint64_t) * (STAGES_X * STAGES_Y);

  float* max_staging_identity = reinterpret_cast<float*>(dshmem);
  dshmem += sizeof(float) * kNumWarps;
  float* max_staging_transpose = reinterpret_cast<float*>(dshmem);
  dshmem += sizeof(float) * kNumWarps;
  float* max_staging_pre_rht = reinterpret_cast<float*>(dshmem);
  dshmem += sizeof(float) * kNumWarps;

  initialize_barriers<STAGES_X * STAGES_Y, THREADS_PER_CHUNK * THREADS_PER_Y>(mbar,
                                                                              is_master_thread);

  copy_2d_to_shared(in_shs[0], reinterpret_cast<const void*>(&tensor_map_input),
                    input_block_offset_X, input_block_offset_Y, shmem_buff_size, &mbar[0],
                    is_master_thread);

  uint32_t had_frag_i[4];
  uint32_t had_frag_t[4];
  get_hadamard_matrix_fragment<kReturnIdentityAmax, kReturnTransposedAmax, false, false>(
      had_frag_i, random_sign_mask, had_frag_t, random_sign_mask_t);

  float local_pre_rht_amax = 0.0;
  float local_amax = 0.0;
  float local_amax_t = 0.0;
  uint32_t local_pre_rht_amax_reg = *reinterpret_cast<uint32_t*>(&local_pre_rht_amax);
  uint32_t local_amax_reg = *reinterpret_cast<uint32_t*>(&local_amax);
  uint32_t local_amax_t_reg = *reinterpret_cast<uint32_t*>(&local_amax_t);

  for (int stage_y = 0; stage_y < STAGES_Y; ++stage_y) {
    for (int stage_x = 0; stage_x < STAGES_X; ++stage_x) {
      int stage = STAGES_X * stage_y + stage_x;

      const int next_stage = stage + 1;
      const int next_stage_x = stage_x + 1 == STAGES_X ? 0 : stage_x + 1;
      const int next_stage_y = stage_x + 1 == STAGES_X ? stage_y + 1 : stage_y;

      if (next_stage < STAGES_X * STAGES_Y) {
        const int input_global_offset_Y = input_block_offset_Y + next_stage_y * BUFF_DIM_Y;
        const int input_global_offset_X = input_block_offset_X + next_stage_x * BUFF_DIM_X;

        copy_2d_to_shared(in_shs[next_stage % 2],  // ping-pong
                          reinterpret_cast<const void*>(&tensor_map_input), input_global_offset_X,
                          input_global_offset_Y, shmem_buff_size, &mbar[next_stage],
                          is_master_thread);
      }

      ptx::fence_proxy_async_shared_cta();

      // Wait for the data to have arrived
      ptx::mbarrier_wait_parity(&mbar[stage], 0);

      const size_t compute_stage_x_num =
          BUFF_DIM_X / (kHadamardDimension * (THREADS_PER_CHUNK / kThreadsPerWarp));
      const size_t compute_stage_y_num = BUFF_DIM_Y / (kHadamardDimension * THREADS_PER_Y);

      const size_t in_row_stride = BUFF_DIM_X;

      IType* in_sh_ptr = in_shs[stage % 2];

#pragma unroll
      for (size_t compute_stage_y = 0; compute_stage_y < compute_stage_y_num; compute_stage_y++) {
        const int row_idx_offset = (compute_stage_y * kHadamardDimension * THREADS_PER_Y +
                                    threadIdx.y * kHadamardDimension);
        const int in_row_offset = row_idx_offset * in_row_stride;

#pragma unroll
        for (size_t compute_stage_x = 0; compute_stage_x < compute_stage_x_num; compute_stage_x++) {
          ComputeKernel<IType, kHadamardDimension, BUFF_DIM_Y, BUFF_DIM_X, kReturnPreRhtAmax,
                        kReturnIdentityAmax, kReturnTransposedAmax>(
              had_frag_i, had_frag_t,
              in_sh_ptr + in_row_offset +
                  (compute_stage_x * kHadamardDimension * (THREADS_PER_CHUNK / kThreadsPerWarp)),
              local_pre_rht_amax_reg, local_amax_reg, local_amax_t_reg);
        }

        // Ensure all threads have finished their computation before new data over-writes the shared
        // memory.
        __syncthreads();
      }
    }
  }

  const int warpid = (threadIdx.x + threadIdx.y * blockDim.x) / kThreadsPerWarp;

  if constexpr (kReturnPreRhtAmax) {
    unpack_max_of_packed_bf16(local_pre_rht_amax_reg, local_pre_rht_amax);
  }
  if constexpr (kReturnIdentityAmax) {
    unpack_max_of_packed_bf16(local_amax_reg, local_amax);
  }
  if constexpr (kReturnTransposedAmax) {
    unpack_max_of_packed_bf16(local_amax_t_reg, local_amax_t);
  }

  ReduceMax<kNumWarps, kReturnPreRhtAmax, kReturnIdentityAmax, kReturnTransposedAmax>(
      local_pre_rht_amax, local_amax, local_amax_t, max_staging_pre_rht, max_staging_identity,
      max_staging_transpose, output_pre_rht_amax_ptr, output_identity_amax_ptr,
      output_transpose_amax_ptr, warpid);

  destroy_barriers<STAGES_X * STAGES_Y>(mbar, is_master_thread);
#else
  NVTE_DEVICE_ERROR("Kernel is only supported on SM 10.0+.");
#endif  // #if (defined __CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
}

template <typename T, int kHadamardDimension, bool kComputeIdentity, bool kComputeTransposed,
          bool kReturnIdentity, bool kReturnTransposed, bool kUpdateIdentityAmax,
          bool kUpdateTransposeAmax, bool kOutputTrueTransposed>
__global__ void HadamardTransformKernel(const T* __restrict__ input, T* __restrict__ output,
                                        T* __restrict__ output_t, uint16_t random_sign_mask,
                                        uint16_t random_sign_mask_t, uint64_t num_input_rows,
                                        uint64_t num_input_cols, float* __restrict__ amax,
                                        float* __restrict__ amax_t, bool inverse_hadamard) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
  static_assert(kHadamardDimension == 16, "Currently only hadamard dimension 16 is supported.");

  // The whole threadblock will share the same smem.
  extern __shared__ __align__(16) T smem[];

  // Each 32 threads process a 16x16 matrix. There is a (y, z) grid of 16x16.
  // If y = 4, z = 4, then each threadblock is processing a 4x4 grid of 16x16 matrices.
  int32_t tid = threadIdx.x;
  int32_t warp_id = threadIdx.y * blockDim.z + threadIdx.z;
  int32_t local_bx = threadIdx.y;
  int32_t local_by = threadIdx.z;

  // Define the register fragments
  uint32_t a_frag[4];    // A matrix fragment
  uint32_t b_frag_i[4];  // Transposed Hadamard matrix fragment, used for A @ B(col major)
  uint32_t b_frag_t[4];  // Hadamard matrix fragment, used for A.T @ B.T(col major)
  uint32_t c_frag[4];    // Result fragment

  // row and col for each thread. 32 threads will work together in 128 chunk to
  // load the data from global memory to shared memory.
  uint32_t row = tid / (kHadamardDimension * sizeof(T) / sizeof(uint4));
  uint32_t col = tid % (kHadamardDimension * sizeof(T) / sizeof(uint4));

  uint32_t smem_index = tid;

  uint32_t input_start_col = (blockIdx.x * blockDim.y + local_bx) * kHadamardDimension;
  uint32_t input_start_row = (blockIdx.y * blockDim.z + local_by) * kHadamardDimension;

  bool load = (input_start_col < num_input_cols) && (input_start_row < num_input_rows);
  if (!load) {
    // Out of bound, we are returning early. No thread divergence since the whole warp
    // will return early.
    return;
  }

  uint64_t global_offset = input_start_col + input_start_row * num_input_cols;
  uint64_t global_offset_t =
      kOutputTrueTransposed ? (input_start_row + input_start_col * num_input_rows) : global_offset;

  T* base_smem = smem + kHadamardDimension * kHadamardDimension * warp_id;

  uint32_t* smem_b32 = reinterpret_cast<uint32_t*>(base_smem);
  uint4* smem_b128 = reinterpret_cast<uint4*>(base_smem);

  // Asynchronously load the data from global memory to shared memory.
  const uint4* input_b128 = reinterpret_cast<const uint4*>(input + global_offset);
  // Each 16x16 chunk is divided into 4 8x8 matrices, we are trying to load each
  // 8x8 chunks consecutively into the smem, so we could leverage ldmatrix m8n8x4
  // to load the data in the tensor core swizzled format.
  __pipeline_memcpy_async(&smem_b128[smem_index],
                          &input_b128[row * num_input_cols / (sizeof(uint4) / sizeof(T)) + col],
                          sizeof(uint4));
  __pipeline_commit();  // Commit the memcpy. Wait when we are in the computation.

  if (inverse_hadamard) {
    get_hadamard_matrix_fragment<kComputeIdentity, kComputeTransposed,
                                 /*kInverseHadamard=*/true,
                                 /*kInverseHadamardTransposed=*/true>(b_frag_i, random_sign_mask,
                                                                      b_frag_t, random_sign_mask_t);
  } else {
    get_hadamard_matrix_fragment<kComputeIdentity, kComputeTransposed,
                                 /*kInverseHadamard=*/false,
                                 /*kInverseHadamardTransposed=*/false>(
        b_frag_i, random_sign_mask, b_frag_t, random_sign_mask_t);
  }

  float local_amax = 0.0;
  float local_amax_t = 0.0;
  uint32_t local_amax_reg = *reinterpret_cast<uint32_t*>(&local_amax);
  uint32_t local_amax_t_reg = *reinterpret_cast<uint32_t*>(&local_amax_t);
  __pipeline_wait_prior(0);

  __syncwarp();  // ensure all lanes finished their cp.async before reading smem

  // Load the A to a_frag.
  if constexpr (kComputeIdentity) {
    load_matrix_16x16_from_shared<false>(a_frag[0], a_frag[1], a_frag[2], a_frag[3], smem_b32,
                                         kHadamardDimension);

    // 16x16 @ 16x16 leveraging all threads in the warp.
    mma_m16_n16_k16_b16_b16_b16_noacc<kUpdateIdentityAmax>(
        a_frag[0], a_frag[1], a_frag[2], a_frag[3], b_frag_i[0], b_frag_i[1], b_frag_i[2],
        b_frag_i[3], c_frag[0], c_frag[1], c_frag[2], c_frag[3], local_amax_reg);

    // Store the result to the shared memory in non-transposed order.
    if constexpr (kReturnIdentity) {
      uint4* output_b128 = reinterpret_cast<uint4*>(output + global_offset);
      store_matrix_16x16_to_global<false>(c_frag[0], c_frag[1], c_frag[2], c_frag[3], output_b128,
                                          num_input_cols);
    }
  }

  if constexpr (kComputeTransposed) {
    if (kComputeIdentity) {
      matrix_transpose_m8_n8_b16_inplace(a_frag[0]);
      matrix_transpose_m8_n8_b16_inplace(a_frag[1]);
      matrix_transpose_m8_n8_b16_inplace(a_frag[2]);
      matrix_transpose_m8_n8_b16_inplace(a_frag[3]);
    } else {
      load_matrix_16x16_from_shared<true>(a_frag[0],
                                          a_frag[2],  // NOTE: intentional index swapping
                                          a_frag[1],  // NOTE: intentional index swapping
                                          a_frag[3], smem_b32, kHadamardDimension);
    }

    mma_m16_n16_k16_b16_b16_b16_noacc<kUpdateTransposeAmax>(
        a_frag[0],
        // 2,1 is used if we are using movmatrix instruction.
        // Thus loading the matrix in 2,1 order will just be normal.
        // This is to be compatible with the movmatrix instruction.
        a_frag[2],  // NOTE: intentional index swapping for transpose purpose.
        a_frag[1],  // NOTE: intentional index swapping for transpose purpose.
        a_frag[3], b_frag_t[0], b_frag_t[1], b_frag_t[2], b_frag_t[3], c_frag[0], c_frag[1],
        c_frag[2], c_frag[3], local_amax_t_reg);

    // Store the result to the shared memory in non-transposed order.
    if constexpr (kReturnTransposed) {
      uint4* output_t_b128 = reinterpret_cast<uint4*>(output_t + global_offset_t);
      store_matrix_16x16_to_global<!kOutputTrueTransposed>(
          c_frag[0], c_frag[1], c_frag[2], c_frag[3], output_t_b128,
          kOutputTrueTransposed ? num_input_rows : num_input_cols);
    }
  }

  if constexpr (kUpdateIdentityAmax) {
    unpack_max_of_packed_bf16(local_amax_reg, local_amax);
    local_amax = warp_reduce_max<kThreadsPerWarp>(local_amax);
    // broadcast the amax to all threads in a warp from the lane 0
    constexpr int lane_zero = 0;
    local_amax = __shfl_sync(0xFFFFFFFF, local_amax, lane_zero);
    // atomic CAS to output memory.
    if (tid % kThreadsPerWarp == 0) {
      atomicMaxFloat(amax, local_amax);
    }
  }
  if constexpr (kUpdateTransposeAmax) {
    unpack_max_of_packed_bf16(local_amax_t_reg, local_amax_t);
    local_amax_t = warp_reduce_max<kThreadsPerWarp>(local_amax_t);
    // broadcast the amax to all threads in a warp from the lane 0
    constexpr int lane_zero = 0;
    local_amax_t = __shfl_sync(0xFFFFFFFF, local_amax_t, lane_zero);
    // atomic CAS to output memory.
    if (tid % kThreadsPerWarp == 0) {
      atomicMaxFloat(amax_t, local_amax_t);
    }
  }
#else
  NVTE_DEVICE_ERROR("Kernel is only supported on SM 9.0+.");
#endif  // defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
}

}  // namespace

void hadamard_transform(const Tensor& input_, Tensor& output_, uint16_t random_sign_mask,
                        uint16_t random_sign_mask_t, hipStream_t stream) {
  NVTE_API_CALL(hadamard_transform);

  // Check tensors
  // NOTE (frsun): This is non-intuitive, we are writing the result of
  // transposed RHT to the output of rowwise.
  NVTE_CHECK(input_.scaling_mode == NVTE_DELAYED_TENSOR_SCALING,
             "Input tensor must be BF16 tensor, but scaling mode is ",
             to_string(input_.scaling_mode), ".");
  NVTE_CHECK(input_.dtype() == transformer_engine::DType::kBFloat16,
             "Input tensor must be BF16 tensor, but dtype is ", to_string(input_.dtype()), ".");
  NVTE_CHECK(input_.dim() >= 2, "Input must be a 2D tensor.");
  NVTE_CHECK(output_.scaling_mode == NVTE_DELAYED_TENSOR_SCALING,
             "Output tensor must be simple tensor, but scaling mode is ",
             to_string(output_.scaling_mode), ".");
  const SimpleTensor& input = input_.data;
  SimpleTensor output;
  SimpleTensor& output_t = output_.data;

  // Check requested outputs
  const bool return_identity = output.dptr != nullptr;
  const bool return_transposed = output_t.dptr != nullptr;
  if (!return_identity && !return_transposed) {  // Nothing to do/ill-defined behavior.
    return;
  }

  checkCuDriverContext(stream);

  const size_t ndim = input.shape.size();
  const size_t row_length = input.shape[ndim - 1];
  size_t num_rows = 1;
  for (size_t i = 0; i < ndim - 1; ++i) {
    num_rows *= input.shape[i];
  }

  using IType = bf16;

  constexpr int kHadamardDimension = 16;
  NVTE_CHECK(row_length % kHadamardDimension == 0,
             "row_length must be divisible by hadamard_dimension.");
  NVTE_CHECK(num_rows % kHadamardDimension == 0,
             "num_rows must be divisible by hadamard_dimension");

  constexpr uint64_t kThreadBlockX = 4;
  // Configure 4 is used for Hopper, 8 is used for Blackwell for extra memory bandwidth.
  constexpr uint64_t kThreadBlockY = 4;

  uint64_t kNumWarpsPerSM = kThreadBlockX * kThreadBlockY;

  // The shared memory number of bytes required for **the whole threadblock**.
  size_t shmem_bytes = kHadamardDimension * kHadamardDimension * sizeof(IType) * kNumWarpsPerSM;

  dim3 block(kThreadsPerWarp, kThreadBlockX, kThreadBlockY);

  dim3 grid(DIVUP(row_length / kHadamardDimension, kThreadBlockX),
            DIVUP(num_rows / kHadamardDimension, kThreadBlockY));

  TRANSFORMER_ENGINE_SWITCH_CONDITION(
      return_transposed, kReturnTransposed,

      TRANSFORMER_ENGINE_SWITCH_CONDITION(
          return_identity, kReturnIdentity,

          auto kernel =
              HadamardTransformKernel<IType, kHadamardDimension, kReturnIdentity, kReturnTransposed,
                                      kReturnIdentity, kReturnTransposed, false, false, true>;

          hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shmem_bytes);

          kernel<<<grid, block, shmem_bytes, stream>>>(
              reinterpret_cast<const IType*>(input.dptr), reinterpret_cast<IType*>(output.dptr),
              reinterpret_cast<IType*>(output_t.dptr), random_sign_mask, random_sign_mask_t,
              num_rows, row_length, nullptr, nullptr, false);););

  NVTE_CHECK_CUDA(hipGetLastError());
}

// Kernel that will apply the 16x16 hadamard transform the input and input.T, and then
// get the absolute max value of the result.
void hadamard_transform_amax(const Tensor& input_, Tensor& output_, uint16_t random_sign_mask,
                             uint16_t random_sign_mask_t, hipStream_t stream) {
  NVTE_API_CALL(hadamard_transform_amax);
#if CUDA_VERSION >= 12080

  // Check input tensor
  NVTE_CHECK(input_.scaling_mode == NVTE_DELAYED_TENSOR_SCALING,
             "Input tensor must be BF16 tensor, but scaling mode is ",
             to_string(input_.scaling_mode), ".");
  NVTE_CHECK(input_.dtype() == transformer_engine::DType::kBFloat16,
             "Input tensor must be BF16 tensor, but dtype is ", to_string(input_.dtype()), ".");
  NVTE_CHECK(input_.dim() >= 2, "Input must be a 2D tensor.");
  const SimpleTensor& input = input_.data;

  // Check amax tensors
  SimpleTensor& output_pre_rht_amax = output_.amax;
  SimpleTensor output_identity_amax;
  SimpleTensor& output_transpose_amax = output_.columnwise_amax;

  // Check requested outputs
  const bool return_pre_rht_amax = output_pre_rht_amax.dptr != nullptr;
  const bool return_identity_amax = output_identity_amax.dptr != nullptr;
  const bool return_transposed_amax = output_transpose_amax.dptr != nullptr;
  if (!return_identity_amax && !return_transposed_amax &&
      !return_pre_rht_amax) {  // Nothing to do/ill-defined behavior.
    return;
  }

  // Zero out amaxes if needed
  ZeroAmaxKernel<<<1, 1, 0, stream>>>(reinterpret_cast<float*>(output_pre_rht_amax.dptr),
                                      reinterpret_cast<float*>(output_identity_amax.dptr),
                                      reinterpret_cast<float*>(output_transpose_amax.dptr));
  NVTE_CHECK_CUDA(hipGetLastError());

  checkCuDriverContext(stream);

  using IType = bf16;

  const size_t ndim = input.shape.size();
  const size_t row_length = input.shape[ndim - 1];
  size_t num_rows = 1;
  for (size_t i = 0; i < ndim - 1; ++i) {
    num_rows *= input.shape[i];
  }

  constexpr int kHadamardDimension = 16;
  NVTE_CHECK(row_length % kHadamardDimension == 0,
             "row_length must be divisible by hadamard_dimension.");
  NVTE_CHECK(num_rows % kHadamardDimension == 0,
             "num_rows must be divisible by hadamard_dimension");

  constexpr uint64_t kChunkBlockXSmall = 128;
  constexpr uint64_t kChunkBlockYSmall = 128;
  constexpr uint64_t kBuffDimX = 64;
  constexpr uint64_t kBuffDimY = 64;

  alignas(64) CUtensorMap tensor_map_input{};

  create_2D_tensor_map(
      /*tensorMap=*/tensor_map_input,
      /*tensor=*/input,
      /*globalY=*/num_rows,
      /*globalX=*/row_length,
      /*shmemY=*/kBuffDimY,
      /*shmemX=*/kBuffDimX,
      /*stride_elems=*/row_length,
      /*offset_elems=*/0,
      /*type_num_bits=*/sizeof(IType) * 8,
      /*swizzle=*/CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_128B_ATOM_32B);

  constexpr uint64_t kThreadBlockX = 4;
  constexpr uint64_t kThreadBlockY = 1;
  constexpr uint64_t kNumWarps = kThreadBlockX * kThreadBlockY;

  dim3 block(kThreadBlockX * kThreadsPerWarp, kThreadBlockY);

  dim3 grid(DIVUP(row_length, kChunkBlockXSmall), DIVUP(num_rows, kChunkBlockYSmall));

  TRANSFORMER_ENGINE_SWITCH_CONDITION(
      return_transposed_amax, kReturnTransposedAmax,

      TRANSFORMER_ENGINE_SWITCH_CONDITION(
          return_identity_amax, kReturnIdentityAmax,

          TRANSFORMER_ENGINE_SWITCH_CONDITION(
              return_pre_rht_amax, kReturnPreRhtAmax,

              // *2 for ping-pong
              size_t in_sh_size = kBuffDimX * kBuffDimY * 2 * sizeof(IType);
              size_t mbar_size = sizeof(uint64_t) * (kChunkBlockXSmall / kBuffDimX) *
                                 (kChunkBlockYSmall / kBuffDimY);
              size_t shmem_bytes = in_sh_size + mbar_size + kNumWarps * sizeof(float) * 3;
              // Add padding in case shmem ptr is not aligned to 128 bytes.
              shmem_bytes = (shmem_bytes + 128);

              auto kernel = HadamardAmaxTmaKernel<
                  IType, kHadamardDimension, kChunkBlockYSmall, kChunkBlockXSmall, kBuffDimY,
                  kBuffDimX, kThreadBlockX * kThreadsPerWarp, kThreadBlockY, kReturnPreRhtAmax,
                  kReturnIdentityAmax, kReturnTransposedAmax>;
              hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize,
                                   shmem_bytes);

              kernel<<<grid, block, shmem_bytes, stream>>>(
                  tensor_map_input, reinterpret_cast<float*>(output_pre_rht_amax.dptr),
                  reinterpret_cast<float*>(output_identity_amax.dptr),
                  reinterpret_cast<float*>(output_transpose_amax.dptr), random_sign_mask,
                  random_sign_mask_t, num_rows, row_length);)));

  NVTE_CHECK_CUDA(hipGetLastError());
#else
  NVTE_ERROR("Hadamard transform requires CUDA 12.8+, but compile-time CUDA version is ",
             CUDA_VERSION);
#endif  // CUDA_VERSION >= 12080
}

}  // namespace transformer_engine

void nvte_hadamard_transform(const NVTETensor input, NVTETensor output, int random_sign_mask,
                             int random_sign_mask_t, hipStream_t stream) {
  NVTE_API_CALL(nvte_hadamard_transform);
  using namespace transformer_engine;
  hadamard_transform(*convertNVTETensorCheck(input), *convertNVTETensorCheck(output),
                     static_cast<uint16_t>(random_sign_mask),
                     static_cast<uint16_t>(random_sign_mask_t), stream);
}

void nvte_hadamard_transform_amax(const NVTETensor input, NVTETensor output, int random_sign_mask,
                                  int random_sign_mask_t, hipStream_t stream) {
  NVTE_API_CALL(nvte_hadamard_transform_amax);
  using namespace transformer_engine;
  hadamard_transform_amax(*convertNVTETensorCheck(input), *convertNVTETensorCheck(output),
                          static_cast<uint16_t>(random_sign_mask),
                          static_cast<uint16_t>(random_sign_mask_t), stream);
}
