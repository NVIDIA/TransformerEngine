#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "../utils.cuh"
#include "norms.h"

namespace transformer_engine {

__global__ void reciprocalKernel(float* value_inv, const float* value) {
  reciprocal(value_inv, *value);
}
void ComputeScaleInv(Tensor* z) {
  NVTE_CHECK(z->amax.dptr != nullptr, "FP8 output must have amax tensor.");
  NVTE_CHECK(z->amax.dtype == DType::kFloat32);
  NVTE_CHECK(z->amax.shape == std::vector<size_t>{1});
  NVTE_CHECK(z->scale_inv.dptr == nullptr, "FP8 output scale_inv should be empty.");
  NVTE_CHECK(z->scale_inv.dtype == DType::kFloat32);
  NVTE_CHECK(z->scale_inv.shape == std::vector<size_t>{1});
  reciprocalKernel<<<1, 1>>>(reinterpret_cast<float*>(z->scale_inv.dptr),
                             reinterpret_cast<float*>(z->amax.dptr));
}
void ComputeScaleInv(void* scale_inv, void* scale) {
  NVTE_CHECK(scale != nullptr, "amax should be allocated.");
  NVTE_CHECK(scale_inv != nullptr, "scale_inv should be allocated.");
  reciprocalKernel<<<1, 1>>>(reinterpret_cast<float*>(scale_inv), reinterpret_cast<float*>(scale));
}

}  // namespace transformer_engine
