#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "rmsnorm.h"
#include "rmsnorm_bwd_kernels.cuh"
#include "rmsnorm_kernel_traits.h"

using namespace transformer_engine::rmsnorm;

template <typename weight_t, typename input_t, typename output_t, typename compute_t,
          typename index_t, int HIDDEN_SIZE, int CTAS_PER_ROW, int WARPS_M, int WARPS_N,
          int BYTES_PER_LDG_MAIN, int BYTES_PER_LDG_FINAL>
void launch_tuned_(LaunchParams<BwdParams> &launch_params, const bool configure_params) {  // NOLINT(*)
    using Kernel_traits =
        rmsnorm::Kernel_traits<weight_t, input_t, output_t, compute_t, index_t, HIDDEN_SIZE,
                               CTAS_PER_ROW, WARPS_M, WARPS_N, BYTES_PER_LDG_MAIN>;
    auto kernel = &rmsnorm_bwd_tuned_kernel<Kernel_traits>;

    if (configure_params) {
        int ctas_per_sm;
        hipError_t status_ = hipOccupancyMaxActiveBlocksPerMultiprocessor(
            &ctas_per_sm, kernel, Kernel_traits::THREADS_PER_CTA, Kernel_traits::SMEM_BYTES);
        launch_params.params.ctas_per_row = CTAS_PER_ROW;
        launch_params.params.ctas_per_col =
            launch_params.multiprocessorCount * ctas_per_sm / launch_params.params.ctas_per_row;
        launch_params.barrier_size = 0;
        launch_params.workspace_bytes = 0;
        if (Kernel_traits::CTAS_PER_ROW > 1) {
            launch_params.barrier_size = 2 * launch_params.params.ctas_per_col;
            launch_params.workspace_bytes = launch_params.params.ctas_per_col *
                                            Kernel_traits::WARPS_M * Kernel_traits::CTAS_PER_ROW *
                                            sizeof(typename Kernel_traits::reduce_t) * 2;
        }
        return;
    }

    if (Kernel_traits::SMEM_BYTES >= 48 * 1024) {
        NVTE_CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize,
                                             Kernel_traits::SMEM_BYTES));
    }
    auto stream = launch_params.stream;
    auto ctas_per_col = launch_params.params.ctas_per_col;
    auto ctas_per_row = launch_params.params.ctas_per_row;

    if (ctas_per_row == 1) {
        kernel<<<ctas_per_col, Kernel_traits::THREADS_PER_CTA, Kernel_traits::SMEM_BYTES, stream>>>(
            launch_params.params);
    } else {
        dim3 grid(ctas_per_row * ctas_per_col);
        dim3 block(Kernel_traits::THREADS_PER_CTA);
        void *params_ = reinterpret_cast<void *>(&launch_params.params);
        hipLaunchCooperativeKernel(reinterpret_cast<void *>(kernel), grid, block,
                                    reinterpret_cast<void **>(&params_), Kernel_traits::SMEM_BYTES,
                                    stream);
    }

    using Kernel_traits_f =
        Kernel_traits_finalize<HIDDEN_SIZE, weight_t, input_t, output_t, compute_t, index_t,
                               32 * 32,  // THREADS_PER_CTA
                               BYTES_PER_LDG_FINAL>;

    auto kernel_f = &rmsnorm::rmsnorm_bwd_finalize_tuned_kernel<Kernel_traits_f>;
    kernel_f<<<Kernel_traits_f::CTAS, Kernel_traits_f::THREADS_PER_CTA, 0, stream>>>(
        launch_params.params);
}

template <typename weight_t, typename input_t, typename output_t, typename compute_t,
          typename index_t, int HIDDEN_SIZE, int WARPS_M, int WARPS_N, int BYTES_PER_LDG_MAIN,
          int BYTES_PER_LDG_FINAL>
void launch_general_(LaunchParams<BwdParams> &launch_params, const bool configure_params) {  // NOLINT(*)
    auto ceil_div = [](int x, int y) -> int { return (x + y - 1) / y; };

    // Instantiate kernel
    using Kernel_traits = Kernel_traits<weight_t, input_t, output_t, compute_t, index_t,
                                        HIDDEN_SIZE, 1, WARPS_M, WARPS_N, BYTES_PER_LDG_MAIN>;
    auto kernel = &rmsnorm_bwd_general_kernel<Kernel_traits>;

    // Configure kernel params
    const int rows = launch_params.params.rows;
    const int cols = launch_params.params.cols;
    int ctas_per_col = launch_params.params.ctas_per_col;
    int ctas_per_row = launch_params.params.ctas_per_row;
    if (configure_params) {
        int ctas_per_sm;
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&ctas_per_sm, kernel,
                                                      Kernel_traits::THREADS_PER_CTA, 0);
        const int max_ctas = launch_params.multiprocessorCount * ctas_per_sm;
        ctas_per_row = ceil_div(cols, HIDDEN_SIZE);
        ctas_per_col = std::min(ceil_div(rows, WARPS_M), max_ctas / ctas_per_row);
        launch_params.params.ctas_per_row = ctas_per_row;
        launch_params.params.ctas_per_col = ctas_per_col;

        launch_params.barrier_size = 0;
        launch_params.workspace_bytes = 0;
        if (launch_params.params.ctas_per_row > 1) {
            launch_params.barrier_size = 2 * ctas_per_col;
            launch_params.workspace_bytes = (ctas_per_col * WARPS_M * ctas_per_row *
                                             sizeof(typename Kernel_traits::reduce_t) * 2);
        }
        return;
    }

    // Launch kernel
    auto stream = launch_params.stream;
    dim3 grid(ctas_per_row * ctas_per_col);
    dim3 block(Kernel_traits::THREADS_PER_CTA);
    if (ctas_per_row == 1) {
        kernel<<<grid, block, 0, stream>>>(launch_params.params);
    } else {
        void *params_ = reinterpret_cast<void *>(&launch_params.params);
        hipLaunchCooperativeKernel(reinterpret_cast<void *>(kernel), grid, block,
                                    reinterpret_cast<void **>(&params_), 0, stream);
    }

    // Launch finalization kernel
    constexpr uint32_t WARPS_M_FINAL = 4;
    constexpr uint32_t WARPS_N_FINAL = 1;
    constexpr uint32_t ELTS_N_PER_CTA_FINAL =
        (Kernel_traits::THREADS_PER_WARP * WARPS_N_FINAL * BYTES_PER_LDG_FINAL / sizeof(compute_t));
    auto kernel_final =
        &rmsnorm_bwd_finalize_general_kernel<weight_t, compute_t, WARPS_M_FINAL, WARPS_N_FINAL,
                                             BYTES_PER_LDG_FINAL, Kernel_traits::THREADS_PER_WARP>;
    dim3 block_final(Kernel_traits::THREADS_PER_WARP * WARPS_N_FINAL, WARPS_M_FINAL);
    dim3 grid_final(ceil_div(cols, ELTS_N_PER_CTA_FINAL), 1);
    kernel_final<<<grid_final, block_final, 0, stream>>>(launch_params.params);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

#define REGISTER_BWD_TUNED_LAUNCHER(                                                               \
    HIDDEN_SIZE, WTYPE, ITYPE, OTYPE, CTYPE, CTAS_PER_ROW, WARPS_M, WARPS_N, BYTES_PER_LDG,        \
                                                                BYTES_PER_LDG_FINALIZE)            \
    void rmsnorm_bwd_tuned_##HIDDEN_SIZE##_##WTYPE##_##ITYPE##_##OTYPE##_##CTYPE(                  \
            LaunchParams<BwdParams>                                                                \
            &launch_params,                                                                        \
            const bool configure_params) {                                                         \
        launch_tuned_<WTYPE,                                                                       \
                ITYPE,                                                                             \
                OTYPE,                                                                             \
                CTYPE,                                                                             \
                uint32_t,                                                                          \
                HIDDEN_SIZE,                                                                       \
                CTAS_PER_ROW,                                                                      \
                WARPS_M,                                                                           \
                WARPS_N,                                                                           \
                BYTES_PER_LDG,                                                                     \
                BYTES_PER_LDG_FINALIZE>(launch_params, configure_params);                          \
    }                                                                                              \
    static BwdTunedRegistrar<WTYPE, ITYPE, OTYPE, CTYPE, HIDDEN_SIZE>                              \
                reg_tuned_##HIDDEN_SIZE##_##WTYPE##_##ITYPE##_##OTYPE##_##CTYPE(                   \
                rmsnorm_bwd_tuned_##HIDDEN_SIZE##_##WTYPE##_##ITYPE##_##OTYPE##_##CTYPE)

#define REGISTER_BWD_GENERAL_LAUNCHER(                                                             \
    HIDDEN_SIZE, WTYPE, ITYPE, OTYPE, CTYPE, WARPS_M, WARPS_N, BYTES_PER_LDG,                      \
                                                                BYTES_PER_LDG_FINALIZE)            \
    void rmsnorm_bwd_general_##HIDDEN_SIZE##_##WTYPE##_##ITYPE##_##OTYPE##_##CTYPE(                \
            LaunchParams<BwdParams>                                                                \
            &launch_params,                                                                        \
            const bool configure_params) {                                                         \
        launch_general_<WTYPE,                                                                     \
                ITYPE,                                                                             \
                OTYPE,                                                                             \
                CTYPE,                                                                             \
                uint32_t,                                                                          \
                HIDDEN_SIZE,                                                                       \
                WARPS_M,                                                                           \
                WARPS_N,                                                                           \
                BYTES_PER_LDG,                                                                     \
                BYTES_PER_LDG_FINALIZE>(launch_params, configure_params);                          \
    }                                                                                              \
    static BwdGeneralRegistrar<WTYPE, ITYPE, OTYPE, CTYPE, HIDDEN_SIZE>                            \
                reg_general_##HIDDEN_SIZE##_##WTYPE##_##ITYPE##_##OTYPE##_##CTYPE(                 \
                rmsnorm_bwd_general_##HIDDEN_SIZE##_##WTYPE##_##ITYPE##_##OTYPE##_##CTYPE)

////////////////////////////////////////////////////////////////////////////////////////////////////

// Create rmsnorm tuned launch function and register. Macro signature:
//  HIDDEN_SIZE, WTYPE, ITYPE, OTYPE, CTYPE, CTAS_PER_ROW, ...
//                             WARPS_M, WARPS_N, BYTES_PER_LDG, BYTES_PER_LDG_FINAL

REGISTER_BWD_TUNED_LAUNCHER(512, fp32, fp32, fp32, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_TUNED_LAUNCHER(512, fp16, fp16, fp16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_TUNED_LAUNCHER(512, bf16, bf16, bf16, fp32, 1, 4, 1, 16, 4);

REGISTER_BWD_TUNED_LAUNCHER(768, fp32, fp32, fp32, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_TUNED_LAUNCHER(768, fp16, fp16, fp16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_TUNED_LAUNCHER(768, bf16, bf16, bf16, fp32, 1, 4, 1, 16, 4);

REGISTER_BWD_TUNED_LAUNCHER(1024, fp32, fp32, fp32, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_TUNED_LAUNCHER(1024, fp16, fp16, fp16, fp32, 1, 4, 1, 16, 4);
REGISTER_BWD_TUNED_LAUNCHER(1024, bf16, bf16, bf16, fp32, 1, 4, 1, 16, 4);

REGISTER_BWD_TUNED_LAUNCHER(2048, fp32, fp32, fp32, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_TUNED_LAUNCHER(2048, fp16, fp16, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_TUNED_LAUNCHER(2048, bf16, bf16, bf16, fp32, 1, 1, 4, 16, 4);

REGISTER_BWD_TUNED_LAUNCHER(4096, fp32, fp32, fp32, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_TUNED_LAUNCHER(4096, fp16, fp16, fp16, fp32, 1, 1, 4, 16, 4);
REGISTER_BWD_TUNED_LAUNCHER(4096, bf16, bf16, bf16, fp32, 1, 1, 4, 16, 4);

// Create rmsnorm general launch function and register. Macro signature:
//  HIDDEN_SIZE, WTYPE, ITYPE, OTYPE, CTYPE, ...
//                             WARPS_M, WARPS_N, BYTES_PER_LDG, BYTES_PER_LDG_FINAL

REGISTER_BWD_GENERAL_LAUNCHER(128, fp32, fp32, fp32, fp32, 4, 1, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(128, fp16, fp16, fp16, fp32, 4, 1, 8, 4);
REGISTER_BWD_GENERAL_LAUNCHER(128, fp16, fp32, fp16, fp32, 4, 1, 8, 4);
REGISTER_BWD_GENERAL_LAUNCHER(128, bf16, bf16, bf16, fp32, 4, 1, 8, 4);
REGISTER_BWD_GENERAL_LAUNCHER(128, bf16, fp32, bf16, fp32, 4, 1, 8, 4);

REGISTER_BWD_GENERAL_LAUNCHER(512, fp32, fp32, fp32, fp32, 4, 1, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(512, fp16, fp16, fp16, fp32, 4, 1, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(512, fp16, fp32, fp16, fp32, 4, 1, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(512, bf16, bf16, bf16, fp32, 4, 1, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(512, bf16, fp32, bf16, fp32, 4, 1, 16, 4);

REGISTER_BWD_GENERAL_LAUNCHER(1024, fp32, fp32, fp32, fp32, 4, 1, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(1024, fp16, fp16, fp16, fp32, 4, 1, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(1024, fp16, fp32, fp16, fp32, 4, 1, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(1024, bf16, bf16, bf16, fp32, 4, 1, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(1024, bf16, fp32, bf16, fp32, 4, 1, 16, 4);

REGISTER_BWD_GENERAL_LAUNCHER(2048, fp32, fp32, fp32, fp32, 1, 4, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(2048, fp16, fp16, fp16, fp32, 1, 4, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(2048, fp16, fp32, fp16, fp32, 1, 4, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(2048, bf16, bf16, bf16, fp32, 1, 4, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(2048, bf16, fp32, bf16, fp32, 1, 4, 16, 4);

REGISTER_BWD_GENERAL_LAUNCHER(4096, fp32, fp32, fp32, fp32, 1, 4, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(4096, fp16, fp16, fp16, fp32, 1, 4, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(4096, fp16, fp32, fp16, fp32, 1, 4, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(4096, bf16, bf16, bf16, fp32, 1, 4, 16, 4);
REGISTER_BWD_GENERAL_LAUNCHER(4096, bf16, fp32, bf16, fp32, 1, 4, 16, 4);
