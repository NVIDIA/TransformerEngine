/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <hip/hip_runtime.h>
#include <transformer_engine/cast.h>

#include <cfloat>
#include <limits>
#include <string>

#include "../common.h"
#include "../transpose/cast_transpose.h"
#include "../util/vectorized_pointwise.h"
#include "../utils.cuh"
#include "cast_kernels.cuh"
#include "dequantize_kernels.cuh"
#include "math.h"
#include "ptx.cuh"
#include "transformer_engine/activation.h"
#include "transformer_engine/transpose.h"

void nvte_quantize(const NVTETensor input, NVTETensor output, hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = false;
  constexpr bool IS_DACT = false;
  constexpr bool IS_ACT = false;
  constexpr NVTETensor dbias = nullptr;
  constexpr NVTETensor workspace = nullptr;
  constexpr const NVTETensor grad = nullptr;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, nullptr>(input, grad, nullptr, output,
                                                                     dbias, workspace, stream);
}

void nvte_quantize_noop(const NVTETensor input, NVTETensor output, NVTETensor noop,
                        hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize_noop);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = false;
  constexpr bool IS_DACT = false;
  constexpr bool IS_ACT = false;
  constexpr NVTETensor dbias = nullptr;
  constexpr NVTETensor workspace = nullptr;
  constexpr const NVTETensor grad = nullptr;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, nullptr>(input, grad, noop, output,
                                                                     dbias, workspace, stream);
}

void nvte_quantize_dbias(const NVTETensor input, NVTETensor output, NVTETensor dbias,
                         NVTETensor workspace, hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize_dbias);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = true;
  constexpr bool IS_DACT = false;
  constexpr bool IS_ACT = false;
  constexpr const NVTETensor activation_input = nullptr;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, nullptr>(
      activation_input, input, nullptr, output, dbias, workspace, stream);
}

void nvte_quantize_dbias_dgelu(const NVTETensor input, const NVTETensor activation_input,
                               NVTETensor output, NVTETensor dbias, NVTETensor workspace,
                               hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize_dbias_dgelu);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = true;
  constexpr bool IS_DACT = true;
  constexpr bool IS_ACT = false;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, dgelu<fp32, fp32>>(
      activation_input, input, nullptr, output, dbias, workspace, stream);
}

void nvte_quantize_dbias_dsilu(const NVTETensor input, const NVTETensor activation_input,
                               NVTETensor output, NVTETensor dbias, NVTETensor workspace,
                               hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize_dbias_dsilu);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = true;
  constexpr bool IS_DACT = true;
  constexpr bool IS_ACT = false;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, dsilu<fp32, fp32>>(
      activation_input, input, nullptr, output, dbias, workspace, stream);
}

void nvte_quantize_dbias_drelu(const NVTETensor input, const NVTETensor activation_input,
                               NVTETensor output, NVTETensor dbias, NVTETensor workspace,
                               hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize_dbias_drelu);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = true;
  constexpr bool IS_DACT = true;
  constexpr bool IS_ACT = false;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, drelu<fp32, fp32>>(
      activation_input, input, nullptr, output, dbias, workspace, stream);
}

void nvte_quantize_dbias_dqgelu(const NVTETensor input, const NVTETensor activation_input,
                                NVTETensor output, NVTETensor dbias, NVTETensor workspace,
                                hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize_dbias_dqgelu);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = true;
  constexpr bool IS_DACT = true;
  constexpr bool IS_ACT = false;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, dqgelu<fp32, fp32>>(
      activation_input, input, nullptr, output, dbias, workspace, stream);
}

void nvte_quantize_dbias_dsrelu(const NVTETensor input, const NVTETensor activation_input,
                                NVTETensor output, NVTETensor dbias, NVTETensor workspace,
                                hipStream_t stream) {
  NVTE_API_CALL(nvte_quantize_dbias_dsrelu);
  using namespace transformer_engine;

  constexpr bool IS_DBIAS = true;
  constexpr bool IS_DACT = true;
  constexpr bool IS_ACT = false;

  detail::quantize_helper<IS_DBIAS, IS_DACT, IS_ACT, Empty, dsrelu<fp32, fp32>>(
      activation_input, input, nullptr, output, dbias, workspace, stream);
}

void nvte_dequantize(const NVTETensor input, NVTETensor output, hipStream_t stream) {
  NVTE_API_CALL(nvte_dequantize);
  using namespace transformer_engine;
  detail::dequantize_helper(*reinterpret_cast<const Tensor *>(input),
                            reinterpret_cast<Tensor *>(output), stream);
}
