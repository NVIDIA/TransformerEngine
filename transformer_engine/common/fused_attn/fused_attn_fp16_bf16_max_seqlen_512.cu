#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "fused_attn_fp16_bf16_max_seqlen_512.h"

#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <cudnn_frontend.h>
#include <map>
#include <vector>

#include "../common.h"
#include "utils.h"

#if (CUDNN_VERSION >= 8901)
#define Q_ID 1
#define K_ID 2
#define V_ID 3
#define O_ID 4
#define S_ID 5
#define B_ID 6
#define D_CONST_ID 7
#define S_CONST_ID 8
#define Q_SEQLEN_ID 9
#define K_SEQLEN_ID 10
#define dQ_ID 11
#define dK_ID 12
#define dV_ID 13
#define dO_ID 14
#define MASK_VAL_ID 15
#define dS_ID 16
#define dBias_ID 17

#define VIRTUAL_ID 20

namespace transformer_engine {
namespace fused_attn {

static void createScale(int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d,
                        NVTE_QKV_Layout layout, hipdnnDataType_t tensorType,
                        // NOLINTNEXTLINE(runtime/references)
                        std::vector<cudnn_frontend::Operation> &ops) {
    // scale
    int64_t scale_dim[4] = {1, 1, 1, 1};
    int64_t scale_stride[4] = {1, 1, 1, 1};

    int64_t k_dim[4] = {b, h, d, s_kv};
    int64_t k_stride[4];
    generateMatrixStrides(b, h, s_q, s_kv, d, k_stride, layout,
                          NVTE_QKV_Matrix::NVTE_K_Matrix_Transpose);

    auto scaleTensor =
        tensor_create(tensorType, S_CONST_ID, scale_dim, scale_stride, false, true);  // is by value
    auto kTensor = tensor_create(tensorType, K_ID, k_dim, k_stride, false, false);
    auto afterScaleKTensor =
        tensor_create(tensorType, VIRTUAL_ID, k_dim, k_stride, true, false);  // is virtual

    // Define the scale descriptor
    auto scaleDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);

    // Create a Scale Node.
    auto scale_op = binary_pw_op_create(kTensor, scaleTensor, afterScaleKTensor, scaleDesc);

    ops.push_back(std::move(scale_op));
}

static cudnn_frontend::Tensor createBMM1(int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d,
                                         NVTE_QKV_Layout layout, hipdnnDataType_t tensorType,
                                         bool zero_s,
                                         // NOLINTNEXTLINE(runtime/references)
                                         std::vector<cudnn_frontend::Operation> &ops) {
    // Creates the necessary tensor descriptors
    int64_t q_dim[4] = {b, h, s_q, d};
    int64_t q_stride[4];
    generateMatrixStrides(b, h, s_q, s_kv, d, q_stride, layout, NVTE_QKV_Matrix::NVTE_Q_Matrix);

    int64_t k_dim[4] = {b, h, d, s_kv};
    int64_t k_stride[4];
    generateMatrixStrides(b, h, s_q, s_kv, d, k_stride, layout,
                          NVTE_QKV_Matrix::NVTE_K_Matrix_Transpose);

    int64_t p_dim[4] = {b, h, s_q, s_kv};
    int64_t p_stride[4];
    generateMatrixStrides(b, h, s_q, s_kv, d, p_stride, layout, NVTE_QKV_Matrix::NVTE_S_Matrix);

    int64_t seqlen_dim[4] = {b, 1, 1, 1};
    int64_t seqlen_stride[4] = {1, 1, 1, 1};

    auto qTensor = tensor_create(tensorType, Q_ID, q_dim, q_stride, false, false);
    auto afterScaleKTensor =
        tensor_create(tensorType, VIRTUAL_ID, k_dim, k_stride, true, false);  // is virtual
    // first GEMM output
    auto pTensor = tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 1, p_dim, p_stride, true,
                                 false);  // is virtual

    auto seqlenQTensor =
        tensor_create(HIPDNN_DATA_INT32, Q_SEQLEN_ID, seqlen_dim, seqlen_stride, false, false);
    auto seqlenKTensor =
        tensor_create(HIPDNN_DATA_INT32, K_SEQLEN_ID, seqlen_dim, seqlen_stride, false, false);

    // Define the matmul 1 desc
    // set padding value optionally to 0 for writing zeros to S tensor (if not set, old behaviour)
    auto matmul_1_Desc =
        cudnn_frontend::MatMulDescBuilder().setComputeType(HIPDNN_DATA_FLOAT).build();

    if (zero_s) {
        matmul_1_Desc = cudnn_frontend::MatMulDescBuilder()
                            .setComputeType(HIPDNN_DATA_FLOAT)
                            .setPaddingValue(0.0f)
                            .build();
    }

    // Create a matmul 1 Node
    auto matmul_op1 = cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                          .setaMatDesc(qTensor)
                          .setbMatDesc(afterScaleKTensor)
                          .setcMatDesc(pTensor)
                          .setmOverrideDesc(seqlenQTensor)
                          .setnOverrideDesc(seqlenKTensor)
                          .setmatmulDesc(matmul_1_Desc)
                          .build();

    ops.push_back(std::move(matmul_op1));

    return pTensor;
}

static cudnn_frontend::Tensor createBias(int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d,
                                         NVTE_QKV_Layout layout, hipdnnDataType_t tensorType,
                                         // NOLINTNEXTLINE(runtime/references)
                                         std::vector<cudnn_frontend::Operation> &ops,
                                         cudnn_frontend::Tensor const &prevBlockOutputTensor) {
    NVTE_CHECK(ops.size() != 0, "Bias op constructed incorrectly as the first one.");

    int64_t b_dim[4] = {1, h, s_q, s_kv};
    int64_t b_stride[4] = {h * s_q * s_kv, s_q * s_kv, s_kv, 1};

    int64_t afterBias_dim[4] = {b, h, s_q, s_kv};
    int64_t afterBias_stride[4];
    generateMatrixStrides(b, h, s_q, s_kv, d, afterBias_stride, layout,
                          NVTE_QKV_Matrix::NVTE_S_Matrix);

    // bias
    auto bTensor = tensor_create(tensorType, B_ID, b_dim, b_stride, false, false);
    // output
    auto afterBiasTensor = tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 50, afterBias_dim,
                                         afterBias_stride, true, false);  // is virtual

    // Define the bias descriptor
    auto biasDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_ADD);

    // Create a Bias Node.
    auto bias_op = binary_pw_op_create(prevBlockOutputTensor, bTensor, afterBiasTensor, biasDesc);

    ops.push_back(std::move(bias_op));

    return afterBiasTensor;
}

static cudnn_frontend::Tensor createMask(int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d,
                                         NVTE_QKV_Layout layout, NVTE_Mask_Type mask_type,
                                         hipdnnDataType_t tensorType,
                                         // NOLINTNEXTLINE(runtime/references)
                                         std::vector<cudnn_frontend::Operation> &ops,
                                         cudnn_frontend::Tensor const &prevBlockOutputTensor,
                                         bool is_bprop) {
    NVTE_CHECK(ops.size() != 0, "Padding mask constructed incorrectly as the first one.");

    // subtraction output
    int64_t afterBMM1_dim[4] = {b, h, s_q, s_kv};
    int64_t afterBMM1_stride[4] = {h * s_q * s_kv, s_q * s_kv, s_kv, 1};

    int64_t seqlen_dim[4] = {b, 1, 1, 1};
    int64_t seqlen_stride[4] = {1, 1, 1, 1};

    int64_t maskVal_dim[4] = {1, 1, 1, 1};
    int64_t maskVal_stride[4] = {1, 1, 1, 1};

    // mask value to put in the masked pixels
    auto maskValTensor = tensor_create(HIPDNN_DATA_FLOAT, MASK_VAL_ID, maskVal_dim, maskVal_stride,
                                       false, true);  // is by value

    auto seqlenQTensor =
        tensor_create(HIPDNN_DATA_INT32, Q_SEQLEN_ID, seqlen_dim, seqlen_stride, false, false);
    auto seqlenKTensor =
        tensor_create(HIPDNN_DATA_INT32, K_SEQLEN_ID, seqlen_dim, seqlen_stride, false, false);
    // gen index row output
    auto rowIndexTensor = tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 100, afterBMM1_dim,
                                        afterBMM1_stride, true, false);  // is virtual
    // gen index column output
    auto columnIndexTensor = tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 101, afterBMM1_dim,
                                           afterBMM1_stride, true, false);  // is virtual
    // less than row output
    auto lessThanRowTensor =
        tensor_create(CUDNN_DATA_BOOLEAN, VIRTUAL_ID + 102, afterBMM1_dim, afterBMM1_stride, true,
                      false);  // is virtual
                               // less than column output
    auto lessThanColTensor = tensor_create(CUDNN_DATA_BOOLEAN, VIRTUAL_ID + 103, afterBMM1_dim,
                                           afterBMM1_stride, true, false);  // is virtual
    // padding mask (lessthanRow && lessthanCol)
    auto paddingMaskTensor = tensor_create(CUDNN_DATA_BOOLEAN, VIRTUAL_ID + 104, afterBMM1_dim,
                                           afterBMM1_stride, true, false);  // is virtual
    // row >= col check for causal mask
    auto rowGreaterColTensor = tensor_create(CUDNN_DATA_BOOLEAN, VIRTUAL_ID + 105, afterBMM1_dim,
                                             afterBMM1_stride, true, false);  // is virtual
    // create causal mask (padding && row >= col)
    auto causalMaskTensor = tensor_create(CUDNN_DATA_BOOLEAN, VIRTUAL_ID + 106, afterBMM1_dim,
                                          afterBMM1_stride, true, false);  // is virtual

    // output after masking
    int64_t maskOutputTensor_id = VIRTUAL_ID + 107;
    int64_t maskOutputTensor_virtual = true;
    hipdnnDataType_t maskOutputTensor_dataType = HIPDNN_DATA_FLOAT;
    auto maskOutputTensor_reorderType =
        cudnn_frontend::cudnnBackendTensorReordering_t::CUDNN_TENSOR_REORDERING_NONE;

    if (is_bprop) {
        maskOutputTensor_id = dS_ID;
        maskOutputTensor_virtual = false;
        maskOutputTensor_dataType = tensorType;
        maskOutputTensor_reorderType =
            cudnn_frontend::cudnnBackendTensorReordering_t::CUDNN_TENSOR_REORDERING_F16x16;
    }

    auto maskOutputTensor =
        cudnn_frontend::TensorBuilder()
            .setDim(4, afterBMM1_dim)
            .setStride(4, afterBMM1_stride)
            .setAlignment(16)  // 16B alignment is needed to run a tensor core engine
            .setByValue(false)
            .setDataType(maskOutputTensor_dataType)
            .setVirtual(maskOutputTensor_virtual)
            .setId(maskOutputTensor_id)
            .setReorderType(maskOutputTensor_reorderType)
            .build();

    // Define the gen index for row descriptor
    auto genIndexRowDesc = cudnn_frontend::PointWiseDescBuilder()
                               .setMode(CUDNN_POINTWISE_GEN_INDEX)
                               .setAxis(2)
                               .setComputeType(HIPDNN_DATA_FLOAT)
                               .build();

    // Create a gen index Node.
    auto genIndexRow_op =
        unary_pw_op_create(prevBlockOutputTensor, rowIndexTensor, genIndexRowDesc);

    // Define the gen index for row descriptor
    auto genIndexColumnDesc = cudnn_frontend::PointWiseDescBuilder()
                                  .setMode(CUDNN_POINTWISE_GEN_INDEX)
                                  .setAxis(3)
                                  .setComputeType(HIPDNN_DATA_FLOAT)
                                  .build();

    // Create a gen index Node.
    auto genIndexColumn_op =
        unary_pw_op_create(prevBlockOutputTensor, columnIndexTensor, genIndexColumnDesc);

    // Define the less than comparison for row descriptor
    auto lessThanRowDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_CMP_LT);

    // Create a less than comparison for row Node.
    auto lessThanRow_op =
        binary_pw_op_create(rowIndexTensor, seqlenQTensor, lessThanRowTensor, lessThanRowDesc);

    // Define the less than comparison for column descriptor
    auto lessThanColDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_CMP_LT);

    // Create a less than comparison for col Node.
    auto lessThanCol_op =
        binary_pw_op_create(columnIndexTensor, seqlenKTensor, lessThanColTensor, lessThanColDesc);

    // Define the less than comparison for column descriptor
    auto paddingMaskAndDesc = pw_desc_create(CUDNN_DATA_BOOLEAN, CUDNN_POINTWISE_LOGICAL_AND);

    // Create a and node for combining lessThanRow and lessThanCol
    auto paddingMaskAnd_op = binary_pw_op_create(lessThanRowTensor, lessThanColTensor,
                                                 paddingMaskTensor, paddingMaskAndDesc);

    // Define the greater than equal to comparison descriptor
    auto rowGreaterColDesc = pw_desc_create(CUDNN_DATA_BOOLEAN, CUDNN_POINTWISE_CMP_GE);

    // Create a greater than equal to Node.
    auto rowGreaterCol_op = binary_pw_op_create(rowIndexTensor, columnIndexTensor,
                                                rowGreaterColTensor, rowGreaterColDesc);

    // Define the and to create causal mask descriptor
    auto causalMaskAndDesc = pw_desc_create(CUDNN_DATA_BOOLEAN, CUDNN_POINTWISE_LOGICAL_AND);

    // Create a causal Mask Node.
    auto causalMaskAnd_op = binary_pw_op_create(paddingMaskTensor, rowGreaterColTensor,
                                                causalMaskTensor, causalMaskAndDesc);

    /////////////////// Apply the mask //////////////////////////

    auto maskTensor = (mask_type == NVTE_Mask_Type::NVTE_CAUSAL_MASK)
                          ? std::move(causalMaskTensor)
                          : std::move(paddingMaskTensor);

    // Define the binary select to perform masking descriptor
    auto maskDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_BINARY_SELECT);

    // Create a binary select Node.
    auto mask_op = ternary_pw_op_create(prevBlockOutputTensor, maskValTensor, maskTensor,
                                        maskOutputTensor, maskDesc);

    ops.push_back(std::move(genIndexRow_op));
    ops.push_back(std::move(genIndexColumn_op));
    ops.push_back(std::move(lessThanRow_op));
    ops.push_back(std::move(lessThanCol_op));
    ops.push_back(std::move(paddingMaskAnd_op));
    if (mask_type == NVTE_Mask_Type::NVTE_CAUSAL_MASK) {
        ops.push_back(std::move(rowGreaterCol_op));
        ops.push_back(std::move(causalMaskAnd_op));
    }
    ops.push_back(std::move(mask_op));

    return maskOutputTensor;
}

static cudnn_frontend::Tensor createSoftmaxForward(
    int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d, NVTE_QKV_Layout layout,
    bool enable_dropout, bool softmax_output_virtual, hipdnnDataType_t tensorType,
    // NOLINTNEXTLINE(runtime/references)
    std::vector<cudnn_frontend::Operation> &ops,
    cudnn_frontend::Tensor const &prevBlockOutputTensor) {
    int64_t afterBMM1_dim[4] = {b, h, s_q, s_kv};
    int64_t afterBMM1_stride[4] = {h * s_q * s_kv, s_q * s_kv, s_kv, 1};

    int64_t afterReduction_dim[4] = {b, h, s_q, 1};
    int64_t afterReduction_stride[4] = {h * s_q, s_q, 1, 1};

    hipdnnDataType_t softmaxOutputType =
        (enable_dropout || softmax_output_virtual) ? HIPDNN_DATA_FLOAT : tensorType;
    uint64_t softmaxOutputName = softmax_output_virtual ? VIRTUAL_ID + 154 : S_ID;

    // max (x)
    auto afterMaxReductionTensor =
        tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 150, afterReduction_dim, afterReduction_stride,
                      true, false);  // is virtual
    // x - max(x)
    auto afterSubtractionTensor = tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 151, afterBMM1_dim,
                                                afterBMM1_stride, true, false);  // is virtual
    // e^(x - max(x))
    auto afterExponentTensor = tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 152, afterBMM1_dim,
                                             afterBMM1_stride, true, false);  // is virtual;
    // sum (e^(x - max(x)))
    auto afterAddReductionTensor =
        tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 153, afterReduction_dim, afterReduction_stride,
                      true, false);  // is virtual
    // divide (e/ sum(e))

    auto reorder_type =
        cudnn_frontend::cudnnBackendTensorReordering_t::CUDNN_TENSOR_REORDERING_F16x16;

    auto afterDivisionTensor =
        cudnn_frontend::TensorBuilder()
            .setDim(4, afterBMM1_dim)
            .setStride(4, afterBMM1_stride)
            .setId(softmaxOutputName)
            .setAlignment(16)  // 16B alignment is needed to run a tensor core engine
            .setDataType(softmaxOutputType)
            .setVirtual(softmax_output_virtual)
            .setByValue(false)
            .setReorderType(reorder_type)
            .build();

    // Define the reduction descriptor
    auto reductionMaxDesc = cudnn_frontend::ReductionDescBuilder()
                                .setComputeType(HIPDNN_DATA_FLOAT)
                                .setReductionOp(HIPDNN_REDUCE_TENSOR_MAX)
                                .build();

    // Create a reduction max Node.
    auto reductionMax_op =
        cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_REDUCTION_DESCRIPTOR)
            .setxDesc(prevBlockOutputTensor)
            .setyDesc(afterMaxReductionTensor)
            .setreductionDesc(reductionMaxDesc)
            .build();

    // Define the subtract descriptor
    auto subtractDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_SUB);

    // Create a subtract Node.
    auto subtract_op = binary_pw_op_create(prevBlockOutputTensor, afterMaxReductionTensor,
                                           afterSubtractionTensor, subtractDesc);

    // Define the exponent descriptor
    auto exponentDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_EXP);

    // Create a exponent Node.
    auto exponent_op =
        unary_pw_op_create(afterSubtractionTensor, afterExponentTensor, exponentDesc);

    // Define the reduction descriptor
    auto reductionAddDesc = cudnn_frontend::ReductionDescBuilder()
                                .setComputeType(HIPDNN_DATA_FLOAT)
                                .setReductionOp(HIPDNN_REDUCE_TENSOR_ADD)
                                .build();

    // Create a reduction add Node.
    auto reductionAdd_op =
        cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_REDUCTION_DESCRIPTOR)
            .setxDesc(afterExponentTensor)
            .setyDesc(afterAddReductionTensor)
            .setreductionDesc(reductionAddDesc)
            .build();

    // Define the division descriptor
    auto divisionDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_DIV);

    // Create a subtract Node.
    auto division_op = binary_pw_op_create(afterExponentTensor, afterAddReductionTensor,
                                           afterDivisionTensor, divisionDesc);

    ops.push_back(std::move(reductionMax_op));
    ops.push_back(std::move(subtract_op));
    ops.push_back(std::move(exponent_op));
    ops.push_back(std::move(reductionAdd_op));
    ops.push_back(std::move(division_op));

    return afterDivisionTensor;
}

static cudnn_frontend::Tensor createDropout(int64_t b, int64_t h, int64_t s_q, int64_t s_kv,
                                            int64_t d, int64_t seed, double probability,
                                            hipdnnDataType_t tensorType,
                                            // NOLINTNEXTLINE(runtime/references)
                                            std::vector<cudnn_frontend::Operation> &ops,
                                            cudnn_frontend::Tensor const &prevBlockOutputTensor) {
    NVTE_CHECK(ops.size() != 0, "Dropout DAG constructed incorrectly as the first one");

    int64_t afterBMM1_dim[4] = {b, h, s_q, s_kv};
    int64_t afterBMM1_stride[4] = {h * s_q * s_kv, s_q * s_kv, s_kv, 1};

    int64_t scale_dim[4] = {1, 1, 1, 1};
    int64_t scale_stride[4] = {1, 1, 1, 1};

    // mask for the dropout
    auto dropoutMaskTensor = tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 200, afterBMM1_dim,
                                           afterBMM1_stride, true, false);  // is virtual

    auto reorder_type =
        cudnn_frontend::cudnnBackendTensorReordering_t::CUDNN_TENSOR_REORDERING_F16x16;

    // after dropout tensor
    auto afterDropoutTensor =
        cudnn_frontend::TensorBuilder()
            .setDim(4, afterBMM1_dim)
            .setStride(4, afterBMM1_stride)
            .setId(S_ID)
            .setAlignment(16)  // 16B alignment is needed to run a tensor core engine
            .setDataType(tensorType)
            .setVirtual(false)
            .setByValue(false)
            .setReorderType(reorder_type)
            .build();
    // scale after dropout
    auto scaleDropoutTensor = tensor_create(tensorType, D_CONST_ID, scale_dim, scale_stride, false,
                                            true);  // is by value
    // after Scale
    auto afterScaleTensor = tensor_create(tensorType, VIRTUAL_ID + 201, afterBMM1_dim,
                                          afterBMM1_stride, true, false);  // is virtual

    // Define the reduction descriptor
    auto rngDesc = cudnn_frontend::RngDescBuilder()
                       .setRngDistribution(CUDNN_RNG_DISTRIBUTION_BERNOULLI)
                       .setBernoulliDistProbability(1.0 - probability)
                       .build();

    // Create a rng Node.
    auto rng_op = cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_RNG_DESCRIPTOR)
                      .setyDesc(dropoutMaskTensor)
                      .setSeed(seed)
                      .setRngDesc(rngDesc)
                      .build();

    // Define the multiply mask descriptor
    auto maskMulDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);

    // Create a multiply mask Node.
    auto maskMul_op = binary_pw_op_create(prevBlockOutputTensor, dropoutMaskTensor,
                                          afterDropoutTensor, maskMulDesc);

    // Define the multiply scale descriptor
    auto scaleMulDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);

    // Create a multiply mask Node.
    auto scaleMul_op =
        binary_pw_op_create(afterDropoutTensor, scaleDropoutTensor, afterScaleTensor, scaleMulDesc);

    ops.push_back(std::move(rng_op));
    ops.push_back(std::move(maskMul_op));
    ops.push_back(std::move(scaleMul_op));

    return afterScaleTensor;
}

static void createBMM2(int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d,
                       NVTE_QKV_Layout layout, hipdnnDataType_t tensorType,
                       // NOLINTNEXTLINE(runtime/references)
                       std::vector<cudnn_frontend::Operation> &ops,
                       cudnn_frontend::Tensor const &prevBlockOutputTensor) {
    NVTE_CHECK(ops.size() != 0, "BMM2 op constructed incorrectly as the first one");

    int64_t seqlen_dim[4] = {b, 1, 1, 1};
    int64_t seqlen_stride[4] = {1, 1, 1, 1};

    int64_t v_dim[4] = {b, h, s_kv, d};
    int64_t v_stride[4];
    generateMatrixStrides(b, h, s_q, s_kv, d, v_stride, layout, NVTE_QKV_Matrix::NVTE_V_Matrix);

    int64_t o_dim[4] = {b, h, s_q, d};
    int64_t o_stride[4];
    generateMatrixStrides(b, h, s_q, s_kv, d, o_stride, layout, NVTE_QKV_Matrix::NVTE_O_Matrix);

    auto seqlenQTensor =
        tensor_create(HIPDNN_DATA_INT32, Q_SEQLEN_ID, seqlen_dim, seqlen_stride, false, false);
    auto seqlenKTensor =
        tensor_create(HIPDNN_DATA_INT32, K_SEQLEN_ID, seqlen_dim, seqlen_stride, false, false);
    auto vTensor = tensor_create(tensorType, V_ID, v_dim, v_stride, false, false);
    // second GEMM output
    auto oTensor = tensor_create(tensorType, O_ID, o_dim, o_stride, false, false);

    // Define the matmul 2 desc
    // set padding value optionally to 0 for writing zeros to O tensor (if not set, old behaviour)
    auto matmul_2_Desc = cudnn_frontend::MatMulDescBuilder()
                             .setComputeType(HIPDNN_DATA_FLOAT)
                             .setPaddingValue(0.0f)
                             .build();

    // Create a matmul 2 Node
    auto matmul_op2 = cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                          .setaMatDesc(prevBlockOutputTensor)
                          .setbMatDesc(vTensor)
                          .setcMatDesc(oTensor)
                          .setmOverrideDesc(seqlenQTensor)
                          .setkOverrideDesc(seqlenKTensor)
                          .setmatmulDesc(matmul_2_Desc)
                          .build();

    ops.push_back(std::move(matmul_op2));
}

static cudnn_frontend::Tensor createSoftmaxBackward(int64_t b, int64_t h, int64_t s_q, int64_t s_kv,
                                                    int64_t d, NVTE_QKV_Layout layout,
                                                    hipdnnDataType_t tensorType,
                                                    // NOLINTNEXTLINE(runtime/references)
                                                    std::vector<cudnn_frontend::Operation> &ops,
                                                    cudnn_frontend::Tensor const &yTensor,
                                                    cudnn_frontend::Tensor const &dyTensor) {
    NVTE_CHECK(ops.size() != 0, "Softmax backward constructed incorrectly as the first one");

    int64_t p_dim[4] = {b, h, s_q, s_kv};
    int64_t p_stride[4];
    generateMatrixStrides(b, h, s_q, s_kv, d, p_stride, layout, NVTE_QKV_Matrix::NVTE_S_Matrix);

    int64_t p_reduction_dim[4] = {b, h, s_q, 1};
    int64_t p_reduction_stride[4];

    p_reduction_stride[3] = 1;
    p_reduction_stride[2] = 1;
    p_reduction_stride[1] = s_q;
    p_reduction_stride[0] = s_q * h;

    int64_t const_dim[4] = {1, 1, 1, 1};
    int64_t const_stride[4] = {1, 1, 1, 1};

    // creating all tensors
    auto softmaxScaleTensor =
        tensor_create(HIPDNN_DATA_FLOAT, S_CONST_ID, const_dim, const_stride, false, true);
    auto dyMulYTensor =
        tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 250, p_dim, p_stride, true, false);
    auto dxAfterReductionTensor = tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 251, p_reduction_dim,
                                                p_reduction_stride, true, false);
    auto dxAfterSubtractionTensor =
        tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 252, p_dim, p_stride, true, false);
    auto dxUnscaleTensor =
        tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 253, p_dim, p_stride, true, false);
    auto dxTensor = tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 254, p_dim, p_stride, true, false);

    // creating all ops
    // mul (y * dy)
    auto mul_1_desc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);
    auto mul_1_op = binary_pw_op_create(yTensor, dyTensor, dyMulYTensor, mul_1_desc);

    // reduction add sum (y * dy)
    auto reductionAddDesc = cudnn_frontend::ReductionDescBuilder()
                                .setComputeType(HIPDNN_DATA_FLOAT)
                                .setReductionOp(HIPDNN_REDUCE_TENSOR_ADD)
                                .build();

    auto reductionAdd_op =
        cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_REDUCTION_DESCRIPTOR)
            .setxDesc(dyMulYTensor)
            .setyDesc(dxAfterReductionTensor)
            .setreductionDesc(reductionAddDesc)
            .build();

    // subtraction (dy - sum(y * dy))
    auto sub_0_desc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_SUB);
    auto sub_0_op =
        binary_pw_op_create(dyTensor, dxAfterReductionTensor, dxAfterSubtractionTensor, sub_0_desc);

    // mul (y * (dy - sum(y * dy)))
    auto mul_2_desc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);
    auto mul_2_op =
        binary_pw_op_create(yTensor, dxAfterSubtractionTensor, dxUnscaleTensor, mul_2_desc);

    // mul (scale * dx)
    auto mul_3_desc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);
    auto mul_3_op = binary_pw_op_create(dxUnscaleTensor, softmaxScaleTensor, dxTensor, mul_3_desc);

    ops.push_back(std::move(mul_1_op));
    ops.push_back(std::move(reductionAdd_op));
    ops.push_back(std::move(sub_0_op));
    ops.push_back(std::move(mul_2_op));
    ops.push_back(std::move(mul_3_op));

    return dxTensor;
}

void fused_attn_max_512_fwd_impl(int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d,
                                 bool is_training, float scaling_factor, float dropout_probability,
                                 NVTE_QKV_Layout layout, NVTE_Bias_Type bias_type,
                                 NVTE_Mask_Type mask_type, void *devPtrQ, void *devPtrK,
                                 void *devPtrV, void *devPtrS, void *devPtrO, void *devPtrBias,
                                 void *devCuSeqlenQ, void *devCuSeqlenK, void *workspace,
                                 size_t *workspace_size, hipdnnDataType_t tensorType,
                                 hipStream_t stream, hipdnnHandle_t handle) {
    try {
        constexpr int64_t seed = 0;  // TODO(rewang): replace this with device seed/offset
        NVTE_CHECK_CUDNN(hipdnnSetStream(handle, stream));

        FADescriptor descriptor{b,           h,
                                s_q,         s_kv,
                                d,           scaling_factor,
                                is_training, dropout_probability,
                                layout,      bias_type,
                                mask_type,   tensorType};

        using CacheType = std::map<FADescriptor, cudnn_frontend::ExecutionPlan>;
        static thread_local CacheType fmha_fprop_cache;

        bool enable_dropout = (dropout_probability != 0.0f);

        NVTE_CHECK(!enable_dropout,
                   "dropout probability > 0 in fused_attn_max_512 has not been implemented.");

        // Get plan from cache if cache is available, otherwise create one
        auto get_plan = [&](CacheType &cache, const FADescriptor &descriptor) {
            // if hit, return
            auto it = cache.find(descriptor);
            if (it != cache.end()) {
                auto plan = it->second;
                return plan;
            }

            // otherwise, build the op_graph and the plan. Then update cache
            std::vector<cudnn_frontend::Operation const *> all_ops;
            std::vector<cudnn_frontend::Operation> ops;

            createScale(b, h, s_q, s_kv, d, layout, tensorType, ops);

            // if bias, we need to memset the S buffer to correctly computate dbias
            auto zero_s = (bias_type != NVTE_Bias_Type::NVTE_NO_BIAS) ||
                          (mask_type == NVTE_Mask_Type::NVTE_CAUSAL_MASK);
            auto bmm1_output = createBMM1(b, h, s_q, s_kv, d, layout, tensorType, zero_s, ops);

            NVTE_CHECK(bias_type != NVTE_Bias_Type::NVTE_PRE_SCALE_BIAS,
                       "NVTE_Bias_Type::NVTE_PRE_SCALE_BIAS has not been implemented.");

            if (bias_type == NVTE_Bias_Type::NVTE_POST_SCALE_BIAS) {
                createBias(b, h, s_q, s_kv, d, layout, tensorType, ops, bmm1_output);
            }

            auto mask_output = createMask(b, h, s_q, s_kv, d, layout, mask_type, tensorType, ops,
                                          bmm1_output, false);

            NVTE_CHECK(dropout_probability != 1.0f, "Dropout probability cannot be 1.0.");

            // TODO(rewang): check whether devPtrS can be removed
            bool softmax_output_virtual = enable_dropout;  // || devPtrS == nullptr;
            auto softmax_output =
                createSoftmaxForward(b, h, s_q, s_kv, d, layout, enable_dropout,
                                     softmax_output_virtual, tensorType, ops, mask_output);

            if (dropout_probability != 0.0f) {
                auto dropout_output = createDropout(b, h, s_q, s_kv, d, seed, dropout_probability,
                                                    tensorType, ops, softmax_output);
                createBMM2(b, h, s_q, s_kv, d, layout, tensorType, ops, dropout_output);
            } else {
                createBMM2(b, h, s_q, s_kv, d, layout, tensorType, ops, softmax_output);
            }

            for (unsigned int i = 0; i < ops.size(); i++) {
                all_ops.push_back(&ops[i]);
            }

            // Create an Operation Graph
            auto opGraph = cudnn_frontend::OperationGraphBuilder()
                               .setHandle(handle)
                               .setOperationGraph(all_ops.size(), all_ops.data())
                               .build();

            cudnn_frontend::EngineConfigList filtered_configs;
            auto statuses = cudnn_frontend::get_heuristics_list<1>(
                {"heuristics_instant"}, opGraph, allowAllConfig, filtered_configs, true);

            if (filtered_configs.size() == 0) {
                cudnn_frontend::set_error_and_throw_exception(
                    nullptr, HIPDNN_STATUS_NOT_SUPPORTED,
                    "run_mha_fprop: No config returned by the heuristics");
            }
            auto plan = cudnn_frontend::ExecutionPlanBuilder()
                            .setHandle(handle)
                            .setEngineConfig(filtered_configs[0], opGraph.getTag())
                            .build();
            cache.insert({descriptor, plan});
            return plan;
        };

        auto plan = get_plan(fmha_fprop_cache, descriptor);

        auto plan_workspace_size = plan.getWorkspaceSize();

        // Exit to request upper level API to allocate memory if needed
        if (workspace == nullptr) {
            size_t actual_seqlen_workspace_size = 2 * b * sizeof(int32_t);
            *workspace_size = plan_workspace_size + actual_seqlen_workspace_size;
            return;
        }

        // Prepare actual seqlen
        constexpr size_t nthreads_per_block = 128;
        const size_t grid = (b + nthreads_per_block - 1) / nthreads_per_block;
        void *devActualSeqlenQ = static_cast<int8_t *>(workspace) + plan_workspace_size;
        void *devActualSeqlenK = static_cast<int8_t *>(devActualSeqlenQ) + b * sizeof(int32_t);
        cu_seqlens_to_actual_seqlens<<<grid, nthreads_per_block, 0, stream>>>(
            b, static_cast<const int32_t *>(devCuSeqlenQ),
            static_cast<const int32_t *>(devCuSeqlenK), static_cast<int32_t *>(devActualSeqlenQ),
            static_cast<int32_t *>(devActualSeqlenK));

        // change this if you have access to float_min
        float negInfinity = -1.0E+10;
        float scale_dropout = 1 / (1 - dropout_probability);

        std::set<std::pair<uint64_t, void *>> data_ptrs;
        // add all the data pointers to be used in the variant pack
        data_ptrs.insert(std::pair<uint64_t, void *>(Q_ID, devPtrQ));
        data_ptrs.insert(std::pair<uint64_t, void *>(K_ID, devPtrK));
        data_ptrs.insert(std::pair<uint64_t, void *>(V_ID, devPtrV));
        data_ptrs.insert(std::pair<uint64_t, void *>(Q_SEQLEN_ID, devActualSeqlenQ));
        data_ptrs.insert(std::pair<uint64_t, void *>(K_SEQLEN_ID, devActualSeqlenK));
        data_ptrs.insert(std::pair<uint64_t, void *>(MASK_VAL_ID, &negInfinity));

        if (tensorType == HIPDNN_DATA_FLOAT) {
            data_ptrs.insert(std::pair<uint64_t, void *>(S_CONST_ID, &scaling_factor));
        } else if (tensorType == HIPDNN_DATA_HALF) {
            __half cast_scaling_factor{scaling_factor};
            data_ptrs.insert(std::pair<uint64_t, void *>(S_CONST_ID, &cast_scaling_factor));
        } else if (tensorType == CUDNN_DATA_BFLOAT16) {
            __hip_bfloat16 cast_scaling_factor{scaling_factor};
            data_ptrs.insert(std::pair<uint64_t, void *>(S_CONST_ID, &cast_scaling_factor));
        } else {
            std::cerr << "Not supported tensorType." << std::endl;
        }

        data_ptrs.insert(std::pair<uint64_t, void *>(O_ID, devPtrO));

        if (bias_type != NVTE_Bias_Type::NVTE_NO_BIAS) {
            data_ptrs.insert(std::pair<uint64_t, void *>(B_ID, devPtrBias));
        }

        if (devPtrS != nullptr) {
            data_ptrs.insert(std::pair<uint64_t, void *>(S_ID, devPtrS));
        }

        if (enable_dropout) {
            data_ptrs.insert(std::pair<uint64_t, void *>(D_CONST_ID, &scale_dropout));
        }

        auto variantPack = cudnn_frontend::VariantPackBuilder()
                               .setWorkspacePointer(workspace)
                               .setDataPointers(data_ptrs)
                               .build();

        NVTE_CHECK_CUDNN(
            cudnnBackendExecute(handle, plan.get_raw_desc(), variantPack.get_raw_desc()));
    } catch (cudnn_frontend::cudnnException &e) {
        NVTE_ERROR(e.what());
    }
}

void fused_attn_max_512_bwd_impl(int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d,
                                 float scaling_factor, float dropout_probability,
                                 NVTE_QKV_Layout layout, NVTE_Mask_Type mask_type,
                                 NVTE_Bias_Type bias_type, void *devPtrQ, void *devPtrK,
                                 void *devPtrV, void *devPtrS, void *devPtrdQ, void *devPtrdK,
                                 void *devPtrdV, void *devPtrdO, void *devPtrdS, void *devPtrdBias,
                                 void *devCuSeqlenQ, void *devCuSeqlenK, void *workspace,
                                 size_t *workspace_size, hipdnnDataType_t tensorType,
                                 hipStream_t stream, hipdnnHandle_t handle) {
    try {
        // Create cudnn handle
        NVTE_CHECK_CUDNN(hipdnnSetStream(handle, stream));

        FADescriptor descriptor{
            b,      h,         s_q,       s_kv,      d, scaling_factor, true, dropout_probability,
            layout, bias_type, mask_type, tensorType};

        using CacheType = std::map<FADescriptor, cudnn_frontend::ExecutionPlan>;
        static thread_local CacheType fmha_bprop_cache;

        auto get_plan = [&](CacheType &cache, const FADescriptor &descriptor) {
            auto it = cache.find(descriptor);
            if (it != cache.end()) {
                return it->second;
            }

            std::vector<cudnn_frontend::Operation const *> all_ops;
            std::vector<cudnn_frontend::Operation> ops;

            // Creates the necessary tensor descriptors
            int64_t q_dim[4] = {b, h, s_q, d};
            int64_t q_stride[4];
            generateMatrixStrides(b, h, s_q, s_kv, d, q_stride, layout,
                                  NVTE_QKV_Matrix::NVTE_Q_Matrix);

            int64_t k_dim[4] = {b, h, s_kv, d};
            int64_t k_stride[4];
            generateMatrixStrides(
                b, h, s_q, s_kv, d, k_stride, layout,
                NVTE_QKV_Matrix::NVTE_K_Matrix);  // type is correct as K is not transposed

            int64_t v_dim[4] = {b, h, d, s_kv};
            int64_t v_stride[4];
            generateMatrixStrides(
                b, h, s_q, s_kv, d, v_stride, layout,
                NVTE_QKV_Matrix::NVTE_V_Matrix_Transpose);  // type is correct as V is transposed

            int64_t p_dim[4] = {b, h, s_q, s_kv};
            int64_t p_stride[4];
            generateMatrixStrides(b, h, s_q, s_kv, d, p_stride, layout,
                                  NVTE_QKV_Matrix::NVTE_S_Matrix);

            int64_t p_transpose_dim[4] = {b, h, s_kv, s_q};
            int64_t p_transpose_stride[4];
            p_transpose_stride[0] = p_stride[0];
            p_transpose_stride[1] = p_stride[1];
            p_transpose_stride[2] = p_stride[3];
            p_transpose_stride[3] = p_stride[2];

            int64_t o_dim[4] = {b, h, s_q, d};
            int64_t o_stride[4];
            generateMatrixStrides(b, h, s_q, s_kv, d, o_stride, layout,
                                  NVTE_QKV_Matrix::NVTE_O_Matrix);

            int64_t seqlen_dim[4] = {b, 1, 1, 1};
            int64_t seqlen_stride[4] = {1, 1, 1, 1};

            int64_t scale_dim[4] = {1, 1, 1, 1};
            int64_t scale_stride[4] = {1, 1, 1, 1};

            // inputs to fprop
            auto qTensor = tensor_create(tensorType, Q_ID, q_dim, q_stride, false, false);
            auto kTensor = tensor_create(tensorType, K_ID, k_dim, k_stride, false, false);
            auto vTensor = tensor_create(tensorType, V_ID, v_dim, v_stride, false, false);
            auto seqlenQTensor = tensor_create(HIPDNN_DATA_INT32, Q_SEQLEN_ID, seqlen_dim,
                                               seqlen_stride, false, false);
            auto seqlenKTensor = tensor_create(HIPDNN_DATA_INT32, K_SEQLEN_ID, seqlen_dim,
                                               seqlen_stride, false, false);

            // gradient of the output
            auto doTensor = tensor_create(tensorType, dO_ID, o_dim, o_stride, false, false);

            auto reorder_type =
                cudnn_frontend::cudnnBackendTensorReordering_t::CUDNN_TENSOR_REORDERING_F16x16;

            // activation from fprop
            auto pTensor =
                cudnn_frontend::TensorBuilder()
                    .setDim(4, p_dim)
                    .setStride(4, p_stride)
                    .setId(S_ID)
                    .setAlignment(16)  // 16B alignment is needed to run a tensor core engine
                    .setDataType(tensorType)
                    .setVirtual(false)
                    .setByValue(false)
                    .setReorderType(reorder_type)
                    .build();

            // outputs from bprop
            auto dqTensor = tensor_create(tensorType, dQ_ID, q_dim, q_stride, false, false);
            auto dkTensor = tensor_create(tensorType, dK_ID, k_dim, k_stride, false, false);
            auto dvTensor = tensor_create(tensorType, dV_ID, k_dim, k_stride, false,
                                          false);  // not transposed therefore k_dim and k_stride

            ////////////////////////////////////////////////////////
            // start creating the ops and the intermediate tensors
            auto pReshapeTensor = tensor_create(tensorType, VIRTUAL_ID + 300, p_transpose_dim,
                                                p_transpose_stride, true, false);

            // reshape to perform transpose and make pReshape
            auto reshape_op =
                cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_RESHAPE_DESCRIPTOR)
                    .setxDesc(pTensor)
                    .setyDesc(pReshapeTensor)
                    .build();

            ops.push_back(std::move(reshape_op));

            // scale dropout
            auto dropoutScaleTensor = tensor_create(HIPDNN_DATA_FLOAT, D_CONST_ID, scale_dim,
                                                    scale_stride, false, true);  // is by value
            auto pAfterScaleTensor = tensor_create(tensorType, VIRTUAL_ID + 301, p_transpose_dim,
                                                   p_transpose_stride, true, false);

            auto scaleMulDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);
            auto scaleMul_op = binary_pw_op_create(pReshapeTensor, dropoutScaleTensor,
                                                   pAfterScaleTensor, scaleMulDesc);
            ops.push_back(std::move(scaleMul_op));

            // perform absolute operation to remove the mask bit
            auto pTransposeAfterAbsTensor = tensor_create(
                tensorType, VIRTUAL_ID + 302, p_transpose_dim, p_transpose_stride, true, false);

            auto absDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_ABS);
            auto abs_op = unary_pw_op_create(pAfterScaleTensor, pTransposeAfterAbsTensor, absDesc);
            ops.push_back(std::move(abs_op));

            // matmul to calculate dvTensor
            // set padding value optionally to 0 for writing zeros to dV tensor (if not set, old
            // behaviour)
            auto matmul_0_Desc = cudnn_frontend::MatMulDescBuilder()
                                     .setComputeType(HIPDNN_DATA_FLOAT)
                                     .setPaddingValue(0.0f)
                                     .build();

            auto matmul_op0 =
                cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                    .setaMatDesc(pTransposeAfterAbsTensor)
                    .setbMatDesc(doTensor)
                    .setcMatDesc(dvTensor)
                    .setmOverrideDesc(seqlenKTensor)
                    .setkOverrideDesc(seqlenQTensor)
                    .setmatmulDesc(matmul_0_Desc)
                    .build();

            ops.push_back(std::move(matmul_op0));

            // matmul to calculate dpTensor
            auto dpTensor =
                tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 303, p_dim, p_stride, true, false);

            auto matmul_1_Desc =
                cudnn_frontend::MatMulDescBuilder().setComputeType(HIPDNN_DATA_FLOAT).build();

            auto matmul_op1 =
                cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                    .setaMatDesc(doTensor)
                    .setbMatDesc(vTensor)
                    .setcMatDesc(dpTensor)
                    .setmOverrideDesc(seqlenQTensor)
                    .setnOverrideDesc(seqlenKTensor)
                    .setmatmulDesc(matmul_1_Desc)
                    .build();

            ops.push_back(std::move(matmul_op1));

            // mask the values which were dropped in dropout
            auto pAbsTensor =
                tensor_create(tensorType, VIRTUAL_ID + 304, p_dim, p_stride, true, false);

            auto p_absDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_ABS);
            auto p_abs_op = unary_pw_op_create(pTensor, pAbsTensor, p_absDesc);
            ops.push_back(std::move(p_abs_op));

            // create the dropout mask
            auto zeroTensor = tensor_create(HIPDNN_DATA_FLOAT, MASK_VAL_ID, scale_dim, scale_stride,
                                            false, true);  // is by value
            auto dropoutMaskTensor =
                tensor_create(CUDNN_DATA_BOOLEAN, VIRTUAL_ID + 305, p_dim, p_stride, true, false);

            auto greater_than_0_desc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_CMP_GT);
            auto greater_than_0_op =
                binary_pw_op_create(pTensor, zeroTensor, dropoutMaskTensor, greater_than_0_desc);
            ops.push_back(std::move(greater_than_0_op));

            // scale for the dropout
            auto dpAfterScaleTensor =
                tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 306, p_dim, p_stride, true, false);

            auto mul_0_desc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);
            auto mul_0_op =
                binary_pw_op_create(dpTensor, dropoutScaleTensor, dpAfterScaleTensor, mul_0_desc);
            ops.push_back(std::move(mul_0_op));

            // drop the values based on the dropout mask
            auto dpAfterDropoutTensor =
                tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 307, p_dim, p_stride, true, false);

            auto selection_0_desc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_BINARY_SELECT);
            auto selection_0_op =
                ternary_pw_op_create(dpAfterScaleTensor, zeroTensor, dropoutMaskTensor,
                                     dpAfterDropoutTensor, selection_0_desc);
            ops.push_back(std::move(selection_0_op));

            // softmax backward
            auto dsTensor = createSoftmaxBackward(b, h, s_q, s_kv, d, layout, tensorType, ops,
                                                  pAbsTensor, dpAfterDropoutTensor);

            // mask
            auto dsAfterMaskTensor =
                createMask(b, h, s_q, s_kv, d, layout, mask_type, tensorType, ops, dsTensor, true);

            // dbias tensor
            int64_t dbias_dim[4] = {1, h, s_q, s_kv};
            int64_t dbias_stride[4] = {h * s_q * s_kv, s_q * s_kv, s_kv, 1};
            auto dBiasTensor =
                tensor_create(tensorType, dBias_ID, dbias_dim, dbias_stride, false, false);

            if (bias_type == NVTE_Bias_Type::NVTE_POST_SCALE_BIAS) {
                auto softmaxScaleTensor = tensor_create(HIPDNN_DATA_FLOAT, S_CONST_ID, scale_dim,
                                                        scale_stride, false, true);
                auto softmaxScaleReciprocalTensor = tensor_create(
                    HIPDNN_DATA_FLOAT, VIRTUAL_ID + 401, scale_dim, scale_stride, true, false);
                auto dbiasBeforeScaleTensor = tensor_create(HIPDNN_DATA_FLOAT, VIRTUAL_ID + 402,
                                                            dbias_dim, dbias_stride, true, false);

                // Define the reduction descriptor
                auto reductionAddDesc = cudnn_frontend::ReductionDescBuilder()
                                            .setComputeType(HIPDNN_DATA_FLOAT)
                                            .setReductionOp(HIPDNN_REDUCE_TENSOR_ADD)
                                            .build();

                // Create a reduction add node to compute the dbias
                auto reductionAdd_op =
                    cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_REDUCTION_DESCRIPTOR)
                        .setxDesc(dsAfterMaskTensor)
                        .setyDesc(dbiasBeforeScaleTensor)
                        .setreductionDesc(reductionAddDesc)
                        .build();
                ops.push_back(std::move(reductionAdd_op));

                // take the reciprocal of the scale
                auto reciprocal_scale_desc =
                    pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_RECIPROCAL);
                auto reciprocal_scale_op = unary_pw_op_create(
                    softmaxScaleTensor, softmaxScaleReciprocalTensor, reciprocal_scale_desc);
                ops.push_back(std::move(reciprocal_scale_op));

                // apply the scale
                auto dBias_scale_desc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);
                auto dBias_scale_op =
                    binary_pw_op_create(dbiasBeforeScaleTensor, softmaxScaleReciprocalTensor,
                                        dBiasTensor, dBias_scale_desc);
                ops.push_back(std::move(dBias_scale_op));
            }

            // matmul to calculate dqTensor
            // set padding value optionally to 0 for writing zeros to dqTensor (if not set, old
            // behaviour)
            auto matmul_2_Desc = cudnn_frontend::MatMulDescBuilder()
                                     .setComputeType(HIPDNN_DATA_FLOAT)
                                     .setPaddingValue(0.0f)
                                     .build();

            auto matmul_op2 =
                cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                    .setaMatDesc(dsAfterMaskTensor)
                    .setbMatDesc(kTensor)
                    .setcMatDesc(dqTensor)
                    .setmOverrideDesc(seqlenQTensor)
                    .setkOverrideDesc(seqlenKTensor)
                    .setmatmulDesc(matmul_2_Desc)
                    .build();

            ops.push_back(std::move(matmul_op2));

            // reshape for transpose of ds
            auto dsAfterMaskReshapeTensor = tensor_create(
                tensorType, VIRTUAL_ID + 308, p_transpose_dim, p_transpose_stride, true, false);

            auto reshape_2_op =
                cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_RESHAPE_DESCRIPTOR)
                    .setxDesc(dsAfterMaskTensor)
                    .setyDesc(dsAfterMaskReshapeTensor)
                    .build();

            ops.push_back(std::move(reshape_2_op));

            // matmul to calculate dkTensor
            // set padding value optionally to 0 for writing zeros to dktensor (if not set, old
            // behaviour)
            auto matmul_3_Desc = cudnn_frontend::MatMulDescBuilder()
                                     .setComputeType(HIPDNN_DATA_FLOAT)
                                     .setPaddingValue(0.0f)
                                     .build();

            auto matmul_op3 =
                cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                    .setaMatDesc(dsAfterMaskReshapeTensor)
                    .setbMatDesc(qTensor)
                    .setcMatDesc(dkTensor)
                    .setmOverrideDesc(seqlenKTensor)
                    .setkOverrideDesc(seqlenQTensor)
                    .setmatmulDesc(matmul_3_Desc)
                    .build();

            ops.push_back(std::move(matmul_op3));

            /////////////////////////////////////////////////////////////////

            for (unsigned int i = 0; i < ops.size(); i++) {
                all_ops.push_back(&ops[i]);
            }

            // Create an Operation Graph
            auto opGraph = cudnn_frontend::OperationGraphBuilder()
                               .setHandle(handle)
                               .setOperationGraph(all_ops.size(), all_ops.data())
                               .build();

            cudnn_frontend::EngineConfigList filtered_configs;
            auto statuses = cudnn_frontend::get_heuristics_list<1>(
                {"heuristics_instant"}, opGraph, allowAllConfig, filtered_configs, true);

            if (filtered_configs.size() == 0) {
                cudnn_frontend::set_error_and_throw_exception(
                    nullptr, HIPDNN_STATUS_NOT_SUPPORTED,
                    "run_mha_bprop: No config returned by the heuristics");
            }

            auto plan = cudnn_frontend::ExecutionPlanBuilder()
                            .setHandle(handle)
                            .setEngineConfig(filtered_configs[0], opGraph.getTag())
                            .build();
            cache.insert({descriptor, plan});
            return plan;
        };

        auto plan = get_plan(fmha_bprop_cache, descriptor);

        auto plan_workspace_size = plan.getWorkspaceSize();

        // Exit to request upper level API to allocate memory if needed
        if (workspace == nullptr) {
            size_t actual_seqlen_workspace_size = 2 * b * sizeof(int32_t);
            *workspace_size = plan_workspace_size + actual_seqlen_workspace_size;
            return;
        }

        constexpr size_t nthreads_per_block = 128;
        const size_t grid = (b + nthreads_per_block - 1) / nthreads_per_block;
        void *devActualSeqlenQ = static_cast<int8_t *>(workspace) + plan_workspace_size;
        void *devActualSeqlenK = static_cast<int8_t *>(devActualSeqlenQ) + b * sizeof(int32_t);
        cu_seqlens_to_actual_seqlens<<<grid, nthreads_per_block, 0, stream>>>(
            b, static_cast<const int32_t *>(devCuSeqlenQ),
            static_cast<const int32_t *>(devCuSeqlenK), static_cast<int32_t *>(devActualSeqlenQ),
            static_cast<int32_t *>(devActualSeqlenK));

        std::set<std::pair<uint64_t, void *>> data_ptrs;
        // add all the data pointers to be used in the variant pack
        data_ptrs.insert(std::pair<uint64_t, void *>(dQ_ID, devPtrdQ));
        data_ptrs.insert(std::pair<uint64_t, void *>(dK_ID, devPtrdK));
        data_ptrs.insert(std::pair<uint64_t, void *>(dV_ID, devPtrdV));

        data_ptrs.insert(std::pair<uint64_t, void *>(Q_ID, devPtrQ));
        data_ptrs.insert(std::pair<uint64_t, void *>(K_ID, devPtrK));
        data_ptrs.insert(std::pair<uint64_t, void *>(V_ID, devPtrV));
        data_ptrs.insert(std::pair<uint64_t, void *>(S_ID, devPtrS));
        data_ptrs.insert(std::pair<uint64_t, void *>(dO_ID, devPtrdO));
        data_ptrs.insert(std::pair<uint64_t, void *>(dS_ID, devPtrdS));
        data_ptrs.insert(std::pair<uint64_t, void *>(Q_SEQLEN_ID, devActualSeqlenQ));
        data_ptrs.insert(std::pair<uint64_t, void *>(K_SEQLEN_ID, devActualSeqlenK));

        if (bias_type != NVTE_Bias_Type::NVTE_NO_BIAS) {
            data_ptrs.insert(std::pair<uint64_t, void *>(dBias_ID, devPtrdBias));
        }

        NVTE_CHECK(dropout_probability == 0.f,
                   "dropout probability > 0 in fused_attn_max_512 has not been implemented.");

        float zeroVal = 0.0f;
        float dropoutScale = 1.0f / (1.0f - dropout_probability);

        data_ptrs.insert(std::pair<uint64_t, void *>(D_CONST_ID, &dropoutScale));
        data_ptrs.insert(std::pair<uint64_t, void *>(S_CONST_ID, &scaling_factor));
        data_ptrs.insert(std::pair<uint64_t, void *>(MASK_VAL_ID, &zeroVal));

        auto variantPack = cudnn_frontend::VariantPackBuilder()
                               .setWorkspacePointer(workspace)
                               .setDataPointers(data_ptrs)
                               .build();

        NVTE_CHECK_CUDNN(
            cudnnBackendExecute(handle, plan.get_raw_desc(), variantPack.get_raw_desc()));
    } catch (cudnn_frontend::cudnnException &e) {
        NVTE_ERROR(e.what());
    }
}

}  // namespace fused_attn

using namespace transformer_engine::fused_attn;
void fused_attn_max_512_fwd_qkvpacked(
    size_t batch, size_t max_seqlen, size_t num_head, size_t head_dim, bool is_training,
    float attn_scale, float p_dropout, NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type,
    NVTE_Mask_Type mask_type, const Tensor *input_QKV, const Tensor *input_Bias, Tensor *output_O,
    NVTETensorPack *Aux_Output_Tensors, const Tensor *cu_seqlens, const Tensor *rng_state,
    Tensor *workspace, hipStream_t stream, hipdnnHandle_t handle) {
    using namespace transformer_engine;

    // Only is_training is verified
    NVTE_CHECK(is_training, "is_training=False is not implemented in fused_attn_max_512.");
    NVTE_CHECK(qkv_layout == NVTE_QKV_Layout::NVTE_QKV_INTERLEAVED,
               "qkv_layout must be NVTE_QKV_Layout::NVTE_QKV_INTERLEAVED.");

    // QKV shape is [b, s, 3, h, d]
    void *devPtrQKV = input_QKV->data.dptr;
    const auto stride = num_head * head_dim;

    void *devPtrQ = static_cast<void *>(devPtrQKV);
    void *devPtrK = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + stride);
    void *devPtrV = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + 2 * stride);

    void *devPtrBias = static_cast<void *>(input_Bias->data.dptr);

    void *devPtrO = output_O->data.dptr;

    void *devPtrS = nullptr;

    if (Aux_Output_Tensors->size == 0) {
        Aux_Output_Tensors->size = 1;
        Tensor *output_S = reinterpret_cast<Tensor *>(Aux_Output_Tensors->tensors[0]);
        output_S->data.dptr = nullptr;
        output_S->data.shape = {batch, num_head, max_seqlen, max_seqlen};
        output_S->data.dtype = input_QKV->data.dtype;
    } else if (Aux_Output_Tensors->size == 1) {
        Tensor *output_S = reinterpret_cast<Tensor *>(Aux_Output_Tensors->tensors[0]);
        devPtrS = output_S->data.dptr;
    }

    void *devCuSeqlen = cu_seqlens->data.dptr;

    // TODO(rewang): dropout seed
    // void* devPtrDropoutSeed = reinterpret_cast<void *>(
    //                 reinterpret_cast<uint64_t*>(rng_state->data.dptr));
    // void* devPtrDropoutOffset = reinterpret_cast<void *>(
    //                 reinterpret_cast<uint64_t*>(rng_state->data.dptr) + 1);

    const DType QKV_type = input_QKV->data.dtype;
    size_t workspace_size = 0;

    // TODO(rewang): replace CPU seed
    fused_attn_max_512_fwd_impl(batch, num_head, max_seqlen, max_seqlen, head_dim, is_training,
                                attn_scale, p_dropout, qkv_layout, bias_type, mask_type, devPtrQ,
                                devPtrK, devPtrV, devPtrS, devPtrO, devPtrBias, devCuSeqlen,
                                devCuSeqlen, workspace->data.dptr, &workspace_size,
                                get_cudnn_dtype(QKV_type), stream, handle);

    if (workspace_size > 0) {
        if (workspace->data.dptr == nullptr) {
            workspace->data.shape = {workspace_size};
            workspace->data.dtype = DType::kByte;
            return;
        }
    } else if (workspace_size == 0) {
        workspace->data.shape = {1};
        workspace->data.dtype = DType::kByte;
        return;
    }
}

void fused_attn_max_512_fwd_kvpacked(size_t batch, size_t q_max_seqlen, size_t kv_max_seqlen,
                                     size_t num_head, size_t head_dim, bool is_training,
                                     float attn_scale, float p_dropout, NVTE_QKV_Layout qkv_layout,
                                     NVTE_Bias_Type bias_type, NVTE_Mask_Type mask_type,
                                     const Tensor *input_Q, const Tensor *input_KV,
                                     const Tensor *input_Bias, Tensor *output_O,
                                     NVTETensorPack *Aux_Output_Tensors, const Tensor *q_cu_seqlens,
                                     const Tensor *kv_cu_seqlens, const Tensor *rng_state,
                                     Tensor *workspace, hipStream_t stream, hipdnnHandle_t handle) {
    using namespace transformer_engine;

    // Only is_training is verified
    NVTE_CHECK(is_training, "is_training=False is not implemented in fused_attn_max_512.");
    NVTE_CHECK(qkv_layout == NVTE_QKV_Layout::NVTE_KV_INTERLEAVED,
               "qkv_layout must be NVTE_QKV_Layout::NVTE_KV_INTERLEAVED.");
    NVTE_CHECK(bias_type == NVTE_Bias_Type::NVTE_NO_BIAS ||
                   bias_type == NVTE_Bias_Type::NVTE_POST_SCALE_BIAS,
               "NVTE_PRE_SCALE_BIAS is not implemented in fused_attn_max_512.");

    // Q shape is [b, s, h, d]
    void *devPtrQ = input_Q->data.dptr;

    // KV shape is [b, s, 2, h, d]
    const auto stride = num_head * head_dim;
    void *devPtrK = input_KV->data.dptr;
    void *devPtrV = static_cast<void *>(static_cast<int8_t *>(devPtrK) + stride);

    void *devPtrBias = input_Bias->data.dptr;

    void *devPtrO = output_O->data.dptr;

    void *devPtrS = nullptr;

    const DType q_type = input_Q->data.dtype;
    const DType kv_type = input_KV->data.dtype;
    NVTE_CHECK(q_type == kv_type, "data type of Q must be equal to data type of KV.");

    if (Aux_Output_Tensors->size == 0) {
        Aux_Output_Tensors->size = 1;
        Tensor *output_S = reinterpret_cast<Tensor *>(Aux_Output_Tensors->tensors[0]);
        output_S->data.dptr = nullptr;
        output_S->data.shape = {batch, num_head, q_max_seqlen, kv_max_seqlen};
        output_S->data.dtype = q_type;
    } else if (Aux_Output_Tensors->size == 1) {
        Tensor *output_S = reinterpret_cast<Tensor *>(Aux_Output_Tensors->tensors[0]);
        devPtrS = output_S->data.dptr;
    }

    void *devQCuSeqlen = q_cu_seqlens->data.dptr;
    void *devKVCuSeqlen = kv_cu_seqlens->data.dptr;

    // TODO(rewang): dropout seed
    // void* devPtrDropoutSeed = reinterpret_cast<void *>(
    //                 reinterpret_cast<uint64_t*>(rng_state->data.dptr));
    // void* devPtrDropoutOffset = reinterpret_cast<void *>(
    //                 reinterpret_cast<uint64_t*>(rng_state->data.dptr) + 1);

    size_t workspace_size = 0;

    // TODO(rewang): replace CPU seed
    fused_attn_max_512_fwd_impl(batch, num_head, q_max_seqlen, kv_max_seqlen, head_dim, is_training,
                                attn_scale, p_dropout, qkv_layout, bias_type, mask_type, devPtrQ,
                                devPtrK, devPtrV, devPtrS, devPtrO, devPtrBias, devQCuSeqlen,
                                devKVCuSeqlen, workspace->data.dptr, &workspace_size,
                                get_cudnn_dtype(q_type), stream, handle);

    if (workspace_size > 0) {
        if (workspace->data.dptr == nullptr) {
            workspace->data.shape = {workspace_size};
            workspace->data.dtype = DType::kByte;
            return;
        }
    } else if (workspace_size == 0) {
        workspace->data.shape = {1};
        workspace->data.dtype = DType::kByte;
        return;
    }
}

void fused_attn_max_512_bwd_qkvpacked(size_t batch, size_t max_seqlen, size_t num_head,
                                      size_t head_dim, float attn_scale, float p_dropout,
                                      NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type,
                                      NVTE_Mask_Type mask_type, const Tensor *input_QKV,
                                      const Tensor *input_dO, const NVTETensorPack *Aux_CTX_Tensors,
                                      Tensor *output_dQKV, Tensor *output_dBias,
                                      const Tensor *cu_seqlens, Tensor *workspace,
                                      hipStream_t stream, hipdnnHandle_t handle) {
    using namespace transformer_engine;

    NVTE_CHECK(qkv_layout == NVTE_QKV_Layout::NVTE_QKV_INTERLEAVED,
               "qkv_layout must be NVTE_QKV_INTERLEAVED.");

    // QKV shape is [b, s, 3, h, d]
    void *devPtrQKV = input_QKV->data.dptr;

    auto stride = num_head * head_dim;
    void *devPtrQ = devPtrQKV;
    void *devPtrK = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + stride);
    void *devPtrV = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + 2 * stride);

    void *devPtrdO = input_dO->data.dptr;

    // dQKV shape is [b, s, 3, h, d]
    void *devPtrdQKV = output_dQKV->data.dptr;
    void *devPtrdQ = devPtrdQKV;
    void *devPtrdK = static_cast<void *>(static_cast<int8_t *>(devPtrdQKV) + stride);
    void *devPtrdV = static_cast<void *>(static_cast<int8_t *>(devPtrdQKV) + 2 * stride);

    void *devPtrdBias = output_dBias->data.dptr;

    NVTE_CHECK(Aux_CTX_Tensors->size == 1);
    void *devPtrS = nullptr;
    if (Aux_CTX_Tensors->size == 1) {
        Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
        devPtrS = output_S->data.dptr;
    }
    // devPtrdS reuses the memory of devPtrS
    void *devPtrdS = devPtrS;

    void *devPtrCuSeqlens = cu_seqlens->data.dptr;

    const auto qkv_type = input_QKV->data.dtype;
    size_t workspace_size = 0;

    fused_attn_max_512_bwd_impl(batch, num_head, max_seqlen, max_seqlen, head_dim, attn_scale,
                                p_dropout, qkv_layout, mask_type, bias_type, devPtrQ, devPtrK,
                                devPtrV, devPtrS, devPtrdQ, devPtrdK, devPtrdV, devPtrdO, devPtrdS,
                                devPtrdBias, devPtrCuSeqlens, devPtrCuSeqlens, workspace->data.dptr,
                                &workspace_size, get_cudnn_dtype(qkv_type), stream, handle);

    if (workspace_size > 0) {
        if (workspace->data.dptr == nullptr) {
            workspace->data.shape = {workspace_size};
            workspace->data.dtype = DType::kByte;
            return;
        }
    } else if (workspace_size == 0) {
        workspace->data.shape = {1};
        workspace->data.dtype = DType::kByte;
        return;
    }
}

void fused_attn_max_512_bwd_kvpacked(size_t batch, size_t q_max_seqlen, size_t kv_max_seqlen,
                                     size_t num_head, size_t head_dim, float attn_scale,
                                     float p_dropout, NVTE_QKV_Layout qkv_layout,
                                     NVTE_Bias_Type bias_type, NVTE_Mask_Type mask_type,
                                     const Tensor *input_Q, const Tensor *input_KV,
                                     const Tensor *input_dO, const NVTETensorPack *Aux_CTX_Tensors,
                                     Tensor *output_dQ, Tensor *output_dKV, Tensor *output_dBias,
                                     const Tensor *q_cu_seqlens, const Tensor *kv_cu_seqlens,
                                     Tensor *workspace, hipStream_t stream, hipdnnHandle_t handle) {
    using namespace transformer_engine;

    NVTE_CHECK(qkv_layout == NVTE_QKV_Layout::NVTE_KV_INTERLEAVED,
               "qkv_layout must be NVTE_KV_INTERLEAVED.");

    // Q shape is [b, s, h, d]
    // KV shape is [b, s, 2, h, d]
    auto stride = num_head * head_dim;
    void *devPtrQ = input_Q->data.dptr;
    void *devPtrK = input_KV->data.dptr;
    void *devPtrV = static_cast<void *>(static_cast<int8_t *>(devPtrK) + stride);

    void *devPtrdO = input_dO->data.dptr;

    // dQ shape is [b, s, h, d]
    // dKV shape is [b, s, 2, h, d]
    void *devPtrdQ = output_dQ->data.dptr;
    void *devPtrdK = output_dKV->data.dptr;
    void *devPtrdV = static_cast<void *>(static_cast<int8_t *>(devPtrdK) + stride);

    void *devPtrdBias = output_dBias->data.dptr;

    NVTE_CHECK(Aux_CTX_Tensors->size == 1);
    void *devPtrS = nullptr;
    if (Aux_CTX_Tensors->size == 1) {
        Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
        devPtrS = output_S->data.dptr;
    }
    // devPtrdS reuses the memory of devPtrS
    void *devPtrdS = devPtrS;

    void *devPtrQCuSeqlens = q_cu_seqlens->data.dptr;
    void *devPtrKVCuSeqlens = kv_cu_seqlens->data.dptr;

    const auto q_type = input_Q->data.dtype;
    const auto kv_type = input_KV->data.dtype;
    NVTE_CHECK(q_type == kv_type, "data type of Q must be equal to data type of KV.");
    size_t workspace_size = 0;

    fused_attn_max_512_bwd_impl(
        batch, num_head, q_max_seqlen, kv_max_seqlen, head_dim, attn_scale, p_dropout, qkv_layout,
        mask_type, bias_type, devPtrQ, devPtrK, devPtrV, devPtrS, devPtrdQ, devPtrdK, devPtrdV,
        devPtrdO, devPtrdS, devPtrdBias, devPtrQCuSeqlens, devPtrKVCuSeqlens, workspace->data.dptr,
        &workspace_size, get_cudnn_dtype(q_type), stream, handle);

    if (workspace_size > 0) {
        if (workspace->data.dptr == nullptr) {
            workspace->data.shape = {workspace_size};
            workspace->data.dtype = DType::kByte;
            return;
        }
    } else if (workspace_size == 0) {
        workspace->data.shape = {1};
        workspace->data.dtype = DType::kByte;
        return;
    }
}
}  // namespace transformer_engine
#endif  // CUDNN_VERSION >= 8901
