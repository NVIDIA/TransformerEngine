#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "fused_attn_f16_arbitrary_seqlen.h"

#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <cudnn_frontend.h>
#include <map>
#include <vector>

#include "../common.h"
#include "utils.h"

#if (CUDNN_VERSION >= 8900)
#define Q_ID 1
#define K_ID 2
#define V_ID 3
#define O_ID 4
#define S_ID 5
#define B_ID 6
#define D_CONST_ID 7
#define S_CONST_ID 8
#define Q_SEQLEN_ID 9
#define K_SEQLEN_ID 10
#define dQ_ID 11
#define dK_ID 12
#define dV_ID 13
#define dO_ID 14
#define MASK_VAL_ID 15
#define dS_ID 16
#define D_SEED_ID 17
#define D_OFFSET_ID 18
#define S_STATS_ID 19
#define S_SUM_ID 20
#define SCALE_PROB 21
#define K_TRANSPOSE_ID 22
#define dQ_ACCUM_ID 23

#define VIRTUAL_ID 30

namespace transformer_engine {
namespace fused_attn {

static cudnn_frontend::Tensor
createScale(int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d,
            NVTE_QKV_Layout layout, hipdnnDataType_t tensorType,
            const cudnn_frontend::Tensor& sTensor,
            std::vector<cudnn_frontend::Operation>* ops) {
    // scale
    int64_t scale_dim[4] = {1, 1, 1, 1};
    int64_t scale_stride[4] = {1, 1, 1, 1};

    int64_t s_dim[4] =  {b, h, s_q, s_kv};
    int64_t s_stride[4];
    generateMatrixStrides(b, h, s_q, s_kv, d, s_stride, layout, NVTE_QKV_Matrix::NVTE_S_Matrix);

    auto scaleTensor = tensor_create(
                       tensorType, S_CONST_ID, scale_dim,
                       scale_stride, false, true);  // is by value
    auto sScaleTensor = tensor_create(
                        tensorType, VIRTUAL_ID + 2000, s_dim,
                        s_stride, true, false);  // is virtual

    // Define the scale descriptor
    auto scaleDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);

    // Create a scale node
    auto scale_op = binary_pw_op_create(sTensor, scaleTensor, sScaleTensor, scaleDesc);

    ops->push_back(std::move(scale_op));
    return sScaleTensor;
}

static cudnn_frontend::Tensor
createQKBMM(int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d,
           NVTE_QKV_Layout layout, hipdnnDataType_t tensorType,
           std::vector<cudnn_frontend::Operation>* ops) {
    // Creates the necessary tensor descriptors
    int64_t q_dim[4] = {b, h, s_q, d};
    int64_t q_stride[4];
    generateMatrixStrides(b, h, s_q, s_kv, d, q_stride, layout, NVTE_QKV_Matrix::NVTE_Q_Matrix);

    int64_t k_dim[4] =  {b, h, d, s_kv};
    int64_t k_stride[4];
    generateMatrixStrides(
            b, h, s_q, s_kv, d, k_stride, layout, NVTE_QKV_Matrix::NVTE_K_Matrix_Transpose);

    int64_t s_dim[4] = {b, h, s_q, s_kv};
    int64_t s_stride[4];
    generateMatrixStrides(b, h, s_q, s_kv, d, s_stride, layout, NVTE_QKV_Matrix::NVTE_S_Matrix);

    auto qTensor = tensor_create(tensorType, Q_ID, q_dim, q_stride, false, false);
    auto kTransposeTensor = tensor_create(
                            tensorType, K_ID, k_dim, k_stride, false, false);  // is virtual
    // first GEMM output
    auto sTensor = tensor_create(
                   HIPDNN_DATA_FLOAT, VIRTUAL_ID + 1, s_dim, s_stride, true, false);  // is virtual

    // Define the matmul 1 desc
    auto matmul_1_Desc = cudnn_frontend::MatMulDescBuilder()
                            .setComputeType(HIPDNN_DATA_FLOAT)
                            .build();

    // Create a matmul 1 node
    auto matmul_op1 = cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                            .setaMatDesc(qTensor)
                            .setbMatDesc(kTransposeTensor)
                            .setcMatDesc(sTensor)
                            .setmatmulDesc(matmul_1_Desc)
                            .build();

    ops->push_back(std::move(matmul_op1));

    return sTensor;
}

static cudnn_frontend::Tensor
createCausalMask(int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d,
           NVTE_QKV_Layout layout, hipdnnDataType_t tensorType,
           std::vector<cudnn_frontend::Operation>* ops,
           const cudnn_frontend::Tensor& prevBlockOutputTensor) {
    CUDNN_FRONTEND_UNUSED(d);
    CUDNN_FRONTEND_UNUSED(layout);
    CUDNN_FRONTEND_UNUSED(tensorType);

    NVTE_CHECK(ops->size() != 0, "Padding Mask constructed incorrectly as the first one");

    // subtraction output
    int64_t afterBMM1_dim[4] = {b, h, s_q, s_kv};
    int64_t afterBMM1_stride[4] = {h * s_q * s_kv, s_q * s_kv, s_kv, 1};

    int64_t maskVal_dim[4] =  {1, 1, 1, 1};
    int64_t maskVal_stride[4] = {1, 1, 1, 1};

    // mask value to put in the masked pixels
    auto maskValTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, MASK_VAL_ID, maskVal_dim,
                            maskVal_stride, false, true);  // is by value
    // gen index row output
    auto rowIndexTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 100, afterBMM1_dim,
                            afterBMM1_stride, true, false);  // is virtual
    // gen index column output
    auto columnIndexTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 101, afterBMM1_dim,
                            afterBMM1_stride, true, false);  // is virtual
    // create causal mask (row >= col)
    auto causalMaskTensor = tensor_create(
                            CUDNN_DATA_BOOLEAN, VIRTUAL_ID + 106, afterBMM1_dim,
                            afterBMM1_stride, true, false);  // is virtual

    // output after masking
    auto maskOutputTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 107, afterBMM1_dim,
                            afterBMM1_stride, true, false);  // is virtual

    // Define the gen index for row descriptor
    auto genIndexRowDesc = cudnn_frontend::PointWiseDescBuilder()
                            .setMode(CUDNN_POINTWISE_GEN_INDEX)
                            .setAxis(2)
                            .setComputeType(HIPDNN_DATA_FLOAT)
                            .build();

    // Create a gen index node
    auto genIndexRow_op = unary_pw_op_create(
                            prevBlockOutputTensor, rowIndexTensor, genIndexRowDesc);

    // Define the gen index for row descriptor
    auto genIndexColumnDesc = cudnn_frontend::PointWiseDescBuilder()
                            .setMode(CUDNN_POINTWISE_GEN_INDEX)
                            .setAxis(3)
                            .setComputeType(HIPDNN_DATA_FLOAT)
                            .build();

    // Create a gen index node
    auto genIndexColumn_op = unary_pw_op_create(
                            prevBlockOutputTensor, columnIndexTensor, genIndexColumnDesc);

    // Define the greater than equal to comparison descriptor
    auto rowGreaterColDesc = pw_desc_create(CUDNN_DATA_BOOLEAN, CUDNN_POINTWISE_CMP_GE);

    // Create a greater than equal to node
    auto rowGreaterCol_op = binary_pw_op_create(
                            rowIndexTensor, columnIndexTensor, causalMaskTensor, rowGreaterColDesc);

    // Define the binary select to perform masking descriptor
    auto maskDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_BINARY_SELECT);

    // Create a binary select node
    auto mask_op = ternary_pw_op_create(
                            prevBlockOutputTensor, maskValTensor,
                            causalMaskTensor, maskOutputTensor, maskDesc);

    ops->push_back(std::move(genIndexRow_op));
    ops->push_back(std::move(genIndexColumn_op));
    ops->push_back(std::move(rowGreaterCol_op));
    ops->push_back(std::move(mask_op));

    return maskOutputTensor;
}

static cudnn_frontend::Tensor
createSoftmaxForward(int64_t b, int64_t h, int64_t s_q, int64_t s_kv, bool isTraining,
                     std::vector<cudnn_frontend::Operation>* ops,
                     const cudnn_frontend::Tensor& sAfterMaskTensor) {
    int64_t afterBMM1_dim[4] = {b, h, s_q, s_kv};
    int64_t afterBMM1_stride[4] = {h * s_q * s_kv, s_q * s_kv, s_kv, 1};

    int64_t afterReduction_dim[4] = {b, h, s_q, 1};
    int64_t afterReduction_stride[4] = {h * s_q, s_q, 1, 1};

    // max (x)
    auto afterMaxReductionTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 150, afterReduction_dim,
                            afterReduction_stride, true, false);  // is virtual

    // x - max(x)
    auto afterSubtractionTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 151, afterBMM1_dim,
                            afterBMM1_stride, true, false);  // is virtual

    // e^(x - max(x))
    auto afterExponentTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 152, afterBMM1_dim,
                            afterBMM1_stride, true, false);  // is virtual;

    // sum (e^(x - max(x)))
    auto afterAddReductionTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 153, afterReduction_dim,
                            afterReduction_stride, true, false);  // is virtual

    // log (sum (e^(x - max(x))))
    auto afterLogLTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 154, afterReduction_dim,
                            afterReduction_stride, true, false);

    // M + log (sum (e^(x - max(x))))
    auto softmaxStatsTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, S_STATS_ID, afterReduction_dim,
                            afterReduction_stride, !isTraining, false);
                            // not virtual if training is true, virtual if training is false

    // divide (e/ sum(e))
    auto afterSoftmaxTensor = cudnn_frontend::TensorBuilder()
            .setDim(4, afterBMM1_dim)
            .setStride(4, afterBMM1_stride)
            .setId(VIRTUAL_ID + 156)
            .setAlignment(16)  // 16B alignment is needed to run a tensor core engine
            .setDataType(HIPDNN_DATA_FLOAT)
            .setVirtual(true)
            .setByValue(false)
            .setReorderType(
                cudnn_frontend::cudnnBackendTensorReordering_t::CUDNN_TENSOR_REORDERING_F16x16)
            .build();

    // Define the reduction descriptor
    auto reductionMaxDesc = cudnn_frontend::ReductionDescBuilder()
                                .setComputeType(HIPDNN_DATA_FLOAT)
                                .setReductionOp(HIPDNN_REDUCE_TENSOR_MAX)
                                .build();

    // Create a reduction max node
    auto reductionMax_op = cudnn_frontend::OperationBuilder(
                                CUDNN_BACKEND_OPERATION_REDUCTION_DESCRIPTOR)
                                .setxDesc(sAfterMaskTensor)
                                .setyDesc(afterMaxReductionTensor)
                                .setreductionDesc(reductionMaxDesc)
                                .build();

    // Define the subtract descriptor
    auto subtractDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_SUB);

    // Create a subtract node
    auto subtract_op = binary_pw_op_create(
                                sAfterMaskTensor, afterMaxReductionTensor,
                                afterSubtractionTensor, subtractDesc);

    // Define the exponent descriptor
    auto exponentDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_EXP);

    // Create a exponent node
    auto exponent_op = unary_pw_op_create(
                                afterSubtractionTensor, afterExponentTensor, exponentDesc);

    // Define the reduction descriptor
    auto reductionAddDesc = cudnn_frontend::ReductionDescBuilder()
                                .setComputeType(HIPDNN_DATA_FLOAT)
                                .setReductionOp(HIPDNN_REDUCE_TENSOR_ADD)
                                .build();

    // Create a reduction add node
    auto reductionAdd_op = cudnn_frontend::OperationBuilder(
                                CUDNN_BACKEND_OPERATION_REDUCTION_DESCRIPTOR)
                                .setxDesc(afterExponentTensor)
                                .setyDesc(afterAddReductionTensor)
                                .setreductionDesc(reductionAddDesc)
                                .build();

    // Create log descriptor
    auto logDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_LOG);

    // Create log node
    auto log_op = unary_pw_op_create(afterAddReductionTensor, afterLogLTensor, logDesc);

    // Create add descriptor
    auto addDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_ADD);

    // Create add node
    auto add_op = binary_pw_op_create(
                                afterMaxReductionTensor, afterLogLTensor,
                                softmaxStatsTensor, addDesc);

    // Define the division descriptor
    auto divisionDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_DIV);

    // Create a subtract node
    auto division_op = binary_pw_op_create(
                                afterExponentTensor, afterAddReductionTensor,
                                afterSoftmaxTensor, divisionDesc);

    ops->push_back(std::move(reductionMax_op));
    ops->push_back(std::move(subtract_op));
    ops->push_back(std::move(exponent_op));
    ops->push_back(std::move(reductionAdd_op));
    ops->push_back(std::move(log_op));
    ops->push_back(std::move(add_op));
    ops->push_back(std::move(division_op));

    return afterSoftmaxTensor;
}

static cudnn_frontend::Tensor
createDropoutForward(int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d,
              double probability, hipdnnDataType_t tensorType,
              std::vector<cudnn_frontend::Operation>* ops,
              const cudnn_frontend::Tensor& afterSoftmaxTensor) {
    CUDNN_FRONTEND_UNUSED(d);

    NVTE_CHECK(ops->size() != 0, "Dropout DAG constructed incorrectly as the first one");

    int64_t afterBMM1_dim[4] = {b, h, s_q, s_kv};
    int64_t afterBMM1_stride[4] = {h * s_q * s_kv, s_q * s_kv, s_kv, 1};

    int64_t scale_dim[4] = {1, 1, 1, 1};
    int64_t scale_stride[4] = {1, 1, 1, 1};

    auto dropoutSeed = tensor_create(
                            CUDNN_DATA_INT64, D_SEED_ID, scale_dim,
                            scale_stride, false, false);  // not virtual
    auto dropoutOffset = tensor_create(
                            CUDNN_DATA_INT64, D_OFFSET_ID, scale_dim,
                            scale_stride, false, false);  // not virtual

    // mask for the dropout
    auto dropoutMaskTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 200, afterBMM1_dim,
                            afterBMM1_stride, true, false);  // is virtual
    // after dropout tensor
    auto afterDropoutTensor = cudnn_frontend::TensorBuilder()
            .setDim(4, afterBMM1_dim)
            .setStride(4, afterBMM1_stride)
            .setId(VIRTUAL_ID + 201)
            .setAlignment(16)  // 16B alignment is needed to run a tensor core engine
            .setDataType(tensorType)
            .setVirtual(true)
            .setByValue(false)
            .setReorderType(
                cudnn_frontend::cudnnBackendTensorReordering_t::CUDNN_TENSOR_REORDERING_F16x16)
            .build();
    // scale after dropout
    auto scaleDropoutTensor = tensor_create(
                            tensorType, D_CONST_ID, scale_dim,
                            scale_stride, false, true);  // is by value
    // after Scale
    auto afterScaleTensor = tensor_create(
                            tensorType, VIRTUAL_ID + 202, afterBMM1_dim,
                            afterBMM1_stride, true, false);  // is virtual

    // Define the reduction descriptor
    auto rngDesc = cudnn_frontend::RngDescBuilder()
                            .setRngDistribution(CUDNN_RNG_DISTRIBUTION_BERNOULLI)
                            .setBernoulliDistProbability(1.0 - probability)
                            .build();

    // Create a rng node
    auto rng_op = cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_RNG_DESCRIPTOR)
                            .setyDesc(dropoutMaskTensor)
                            .setSeedDesc(dropoutSeed)
                            .setOffsetDesc(dropoutOffset)
                            .setRngDesc(rngDesc)
                            .build();

    // Define the multiply mask descriptor
    auto maskMulDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);

    // Create a multiply mask node
    auto maskMul_op = binary_pw_op_create(
                            afterSoftmaxTensor, dropoutMaskTensor,
                            afterDropoutTensor, maskMulDesc);

    // Define the multiply scale descriptor
    auto scaleMulDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);

    // Create a multiply scale node
    auto scaleMul_op = binary_pw_op_create(
                            afterDropoutTensor, scaleDropoutTensor,
                            afterScaleTensor, scaleMulDesc);

    ops->push_back(std::move(rng_op));
    ops->push_back(std::move(maskMul_op));
    ops->push_back(std::move(scaleMul_op));

    return afterScaleTensor;
}

static cudnn_frontend::Tensor
createDropoutBackward(int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d,
              double probability, hipdnnDataType_t tensorType,
              std::vector<cudnn_frontend::Operation>* ops,
              const cudnn_frontend::Tensor& afterSoftmaxTensor,
              const cudnn_frontend::Tensor& dropoutMaskTensor) {
    CUDNN_FRONTEND_UNUSED(d);

    NVTE_CHECK(ops->size() != 0, "Dropout DAG constructed incorrectly as the first one");

    int64_t afterBMM1_dim[4] = {b, h, s_q, s_kv};
    int64_t afterBMM1_stride[4] = {h * s_q * s_kv, s_q * s_kv, s_kv, 1};

    int64_t scale_dim[4] = {1, 1, 1, 1};
    int64_t scale_stride[4] = {1, 1, 1, 1};

    auto dropoutSeed = tensor_create(
                            CUDNN_DATA_INT64, D_SEED_ID, scale_dim,
                            scale_stride, false, false);  // not virtual
    auto dropoutOffset = tensor_create(
                            CUDNN_DATA_INT64, D_OFFSET_ID, scale_dim,
                            scale_stride, false, false);  // not virtual

    // after dropout tensor
    auto afterDropoutTensor = cudnn_frontend::TensorBuilder()
            .setDim(4, afterBMM1_dim)
            .setStride(4, afterBMM1_stride)
            .setId(VIRTUAL_ID + 201)
            .setAlignment(16)  // 16B alignment is needed to run a tensor core engine
            .setDataType(tensorType)
            .setVirtual(true)
            .setByValue(false)
            .setReorderType(
                cudnn_frontend::cudnnBackendTensorReordering_t::CUDNN_TENSOR_REORDERING_F16x16)
            .build();
    // scale after dropout
    auto scaleDropoutTensor = tensor_create(
                            tensorType, D_CONST_ID, scale_dim,
                            scale_stride, false, true);  // is by value
    // after Scale
    auto afterScaleTensor = tensor_create(
                            tensorType, VIRTUAL_ID + 202, afterBMM1_dim,
                            afterBMM1_stride, true, false);  // is virtual

    // Define the reduction descriptor
    auto rngDesc = cudnn_frontend::RngDescBuilder()
                            .setRngDistribution(CUDNN_RNG_DISTRIBUTION_BERNOULLI)
                            .setBernoulliDistProbability(1.0 - probability)
                            .build();

    // Create a rng node
    auto rng_op = cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_RNG_DESCRIPTOR)
                            .setyDesc(dropoutMaskTensor)
                            .setSeedDesc(dropoutSeed)
                            .setOffsetDesc(dropoutOffset)
                            .setRngDesc(rngDesc)
                            .build();

    // Define the multiply mask descriptor
    auto maskMulDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);

    // Create a multiply mask node
    auto maskMul_op = binary_pw_op_create(
                            afterSoftmaxTensor, dropoutMaskTensor,
                            afterDropoutTensor, maskMulDesc);

    // Define the multiply scale descriptor
    auto scaleMulDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);

    // Create a multiply scale node
    auto scaleMul_op = binary_pw_op_create(
                            afterDropoutTensor, scaleDropoutTensor,
                            afterScaleTensor, scaleMulDesc);

    ops->push_back(std::move(rng_op));
    ops->push_back(std::move(maskMul_op));
    ops->push_back(std::move(scaleMul_op));

    return afterScaleTensor;
}

static void
createSVBMM(int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d,
           NVTE_QKV_Layout layout, hipdnnDataType_t tensorType,
           std::vector<cudnn_frontend::Operation>* ops,
           cudnn_frontend::Tensor const &afterScaleDropoutTensor) {
    NVTE_CHECK(ops->size() != 0, "BMM2 op constructed incorrectly as the first one");

    int64_t v_dim[4] =  {b, h, s_kv, d};
    int64_t v_stride[4];
    generateMatrixStrides(b, h, s_q, s_kv, d, v_stride, layout, NVTE_QKV_Matrix::NVTE_V_Matrix);

    int64_t o_dim[4] =  {b, h, s_q, d};
    int64_t o_stride[4];
    generateMatrixStrides(b, h, s_q, s_kv, d, o_stride, layout, NVTE_QKV_Matrix::NVTE_O_Matrix);

    auto vTensor = tensor_create(tensorType, V_ID, v_dim, v_stride, false, false);
    // second GEMM output
    auto oTensor = tensor_create(tensorType, O_ID, o_dim, o_stride, false, false);

    // Define the matmul 2 desc
    auto matmul_2_Desc = cudnn_frontend::MatMulDescBuilder()
                            .setComputeType(HIPDNN_DATA_FLOAT)
                            .build();

    // Create a matmul 2 node
    auto matmul_op2 = cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                            .setaMatDesc(afterScaleDropoutTensor)
                            .setbMatDesc(vTensor)
                            .setcMatDesc(oTensor)
                            .setmatmulDesc(matmul_2_Desc)
                            .build();

    ops->push_back(std::move(matmul_op2));
}

void fused_attn_arbitrary_seqlen_fwd_impl(
                                int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d,
                                bool is_training, float scaling_factor, float dropout_probability,
                                NVTE_QKV_Layout layout,
                                void *devPtrQ, void *devPtrK, void *devPtrV,
                                void *devPtrSoftmaxStats, void *devPtrO,
                                void* devPtrDropoutSeed, void* devPtrDropoutOffset,
                                hipdnnDataType_t tensorType,
                                void *workspace, size_t *workspace_size,
                                hipStream_t stream, hipdnnHandle_t handle) {
    try {
        NVTE_CHECK_CUDNN(hipdnnSetStream(handle, stream));

        if (!is_training) {
          dropout_probability == 0.0f;
        }

        FADescriptor descriptor{b,           h,
                                s_q,         s_kv,
                                d,           scaling_factor,
                                is_training, dropout_probability,
                                layout,      NVTE_Bias_Type::NVTE_NO_BIAS,
                                NVTE_Mask_Type::NVTE_CAUSAL_MASK,   tensorType};

        using CacheType = std::map<FADescriptor, cudnn_frontend::ExecutionPlan>;
        static thread_local CacheType fmha_fprop_cache;

        // Get plan from cache if cache is available, otherwise create one
        auto get_plan = [&](CacheType &cache, const FADescriptor &descriptor) {
            // if hit, return
            auto it = cache.find(descriptor);
            if (it != cache.end()) {
                auto plan = it->second;
                return plan;
            }

            // otherwise, build the op_graph and the plan. Then update cache
            std::vector<cudnn_frontend::Operation const*> all_ops;
            std::vector<cudnn_frontend::Operation> ops;

            // Q * K^T
            auto sTensor = createQKBMM(b, h, s_q, s_kv, d, layout, tensorType, &ops);

            // Q * K^T * bmmScale
            auto sScaleTensor = createScale(
                                b, h, s_q, s_kv, d, layout, HIPDNN_DATA_FLOAT, sTensor, &ops);

            // Causual mask
            auto sAfterMaskTensor = createCausalMask(
                                b, h, s_q, s_kv, d, layout, tensorType, &ops, sScaleTensor);

            NVTE_CHECK(dropout_probability != 1.0f,
                                "Dropout probability cannot be 1.0");

            auto softmax_output = createSoftmaxForward(
                                b, h, s_q, s_kv, is_training, &ops, sAfterMaskTensor);

            // Dropout(softmax)
            auto dropout_output = createDropoutForward(
                                b, h, s_q, s_kv, d,
                                dropout_probability, tensorType, &ops, softmax_output);
            createSVBMM(b, h, s_q, s_kv, d, layout, tensorType, &ops, dropout_output);

            for (unsigned int i = 0; i < ops.size(); i++) {
                all_ops.push_back(&ops[i]);
            }

            // Create an Operation Graph
            auto opGraph = cudnn_frontend::OperationGraphBuilder()
                                .setHandle(handle)
                                .setOperationGraph(all_ops.size(), all_ops.data())
                                .build();

            cudnn_frontend::EngineConfigList filtered_configs;
            auto statuses = cudnn_frontend::get_heuristics_list<1>(
                                {"heuristics_instant"}, opGraph, allowAllConfig,
                                filtered_configs, true);

            if (filtered_configs.size() == 0) {
                cudnn_frontend::set_error_and_throw_exception(
                        nullptr,
                        HIPDNN_STATUS_NOT_SUPPORTED,
                        "run_mha_fprop: No config returned by the heuristics");
            }

            auto plan = cudnn_frontend::ExecutionPlanBuilder()
                                .setHandle(handle)
                                .setEngineConfig(filtered_configs[0], opGraph.getTag())
                                .build();

            cache.insert({descriptor, plan});
            return plan;
        };

        auto plan = get_plan(fmha_fprop_cache, descriptor);

        auto plan_workspace_size = plan.getWorkspaceSize();

        // Exit to request upper level API to allocate memory if needed
        if (workspace == nullptr) {
            *workspace_size = plan_workspace_size;
            return;
        }

        std::set<std::pair<uint64_t, void*>> data_ptrs;
        // Add all the data pointers to be used in the variant pack
        float negInfinity = -1.0E+10f;
        float scale_dropout = 1.0f/(1.0f - dropout_probability);

        data_ptrs.insert(std::pair<uint64_t, void*>(Q_ID, devPtrQ));
        data_ptrs.insert(std::pair<uint64_t, void*>(K_ID, devPtrK));
        data_ptrs.insert(std::pair<uint64_t, void*>(V_ID, devPtrV));
        data_ptrs.insert(std::pair<uint64_t, void*>(MASK_VAL_ID, &negInfinity));
        data_ptrs.insert(std::pair<uint64_t, void*>(S_CONST_ID, &scaling_factor));
        data_ptrs.insert(std::pair<uint64_t, void*>(O_ID, devPtrO));
        data_ptrs.insert(std::pair<uint64_t, void*>(D_SEED_ID, devPtrDropoutSeed));
        data_ptrs.insert(std::pair<uint64_t, void*>(D_OFFSET_ID, devPtrDropoutOffset));
        data_ptrs.insert(std::pair<uint64_t, void*>(D_CONST_ID, &scale_dropout));

        // If training mode, we write out softmax stats
        if (is_training) {
            data_ptrs.insert(std::pair<uint64_t, void*>(S_STATS_ID, devPtrSoftmaxStats));
        }

        auto variantPack = cudnn_frontend::VariantPackBuilder()
                               .setWorkspacePointer(workspace)
                               .setDataPointers(data_ptrs)
                               .build();

        NVTE_CHECK_CUDNN(
            cudnnBackendExecute(handle, plan.get_raw_desc(), variantPack.get_raw_desc()));
    } catch (cudnn_frontend::cudnnException &e) {
        NVTE_ERROR(e.what());
    }
}

void fused_attn_arbitrary_seqlen_bwd_impl(
                            int64_t b, int64_t h, int64_t s_q, int64_t s_kv, int64_t d,
                            float scaling_factor, float dropout_probability, NVTE_QKV_Layout layout,
                            void* devPtrQ, void* devPtrKTranspose, void* devPtrVTranspose,
                            void* devPtrO, void* devPtrSoftmaxStats,
                            void* devPtrdQ, void* devPtrdK, void* devPtrdV, void* devPtrdO,
                            void* devPtrDropoutSeed, void* devPtrDropoutOffset,
                            hipdnnDataType_t tensorType, void *workspace, size_t *workspace_size,
                            hipStream_t stream, hipdnnHandle_t handle) {
    try {
        NVTE_CHECK_CUDNN(hipdnnSetStream(handle, stream));

        FADescriptor descriptor{b,           h,
                                s_q,         s_kv,
                                d,           scaling_factor,
                                true,        dropout_probability,
                                layout,      NVTE_Bias_Type::NVTE_NO_BIAS,
                                NVTE_Mask_Type::NVTE_CAUSAL_MASK,   tensorType};

        using CacheType = std::map<FADescriptor, cudnn_frontend::ExecutionPlan>;
        static thread_local CacheType fmha_bprop_cache;

        auto get_plan = [&](CacheType &cache, const FADescriptor &descriptor) {
            auto it = cache.find(descriptor);
            if (it != cache.end()) {
                return it->second;
            }

            std::vector<cudnn_frontend::Operation const*> all_ops;
            std::vector<cudnn_frontend::Operation> ops;

            // Creates the necessary tensor descriptors
            int64_t q_dim[4] = {b, h, s_q, d};
            int64_t q_stride[4];
            generateMatrixStrides(
                            b, h, s_q, s_kv, d, q_stride,
                            layout, NVTE_QKV_Matrix::NVTE_Q_Matrix);

            int64_t k_transpose_dim[4] =  {b, h, d, s_kv};
            int64_t k_transpose_stride[4];
            generateMatrixStrides(
                            b, h, s_q, s_kv, d, k_transpose_stride,
                            layout, NVTE_QKV_Matrix::NVTE_K_Matrix_Transpose);

            int64_t v_transpose_dim[4] =  {b, h, d, s_kv};
            int64_t v_transpose_stride[4];
            generateMatrixStrides(
                            b, h, s_q, s_kv, d, v_transpose_stride,
                            layout, NVTE_QKV_Matrix::NVTE_V_Matrix_Transpose);

            int64_t p_dim[4] = {b, h, s_q, s_kv};
            int64_t p_stride[4];
            generateMatrixStrides(
                            b, h, s_q, s_kv, d, p_stride,
                            layout, NVTE_QKV_Matrix::NVTE_S_Matrix);

            int64_t p_transpose_dim[4] = {b, h, s_kv, s_q};
            int64_t p_transpose_stride[4];
            p_transpose_stride[0] = p_stride[0];
            p_transpose_stride[1] = p_stride[1];
            p_transpose_stride[2] = p_stride[3];
            p_transpose_stride[3] = p_stride[2];

            int64_t o_dim[4] =  {b, h, s_q, d};
            int64_t o_stride[4];
            generateMatrixStrides(
                            b, h, s_q, s_kv, d, o_stride,
                            layout, NVTE_QKV_Matrix::NVTE_O_Matrix);

            int64_t scale_dim[4] = {1, 1, 1, 1};
            int64_t scale_stride[4] = {1, 1, 1, 1};

            /*******************************************************************************
             *                          Dot product dO * O                                */ 

            // output and gradient of the output
            auto oTensor = tensor_create(tensorType, O_ID, o_dim, o_stride, false, false);
            auto dOTensor = tensor_create(tensorType, dO_ID, o_dim, o_stride, false, false);

            auto dotProductTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID, o_dim,
                            o_stride, true, false);  // is virtual

            // Create pointwise mul
            auto multiplyDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);

            // do * O
            auto dotProductOp = binary_pw_op_create(
                            dOTensor, oTensor, dotProductTensor, multiplyDesc);
            ops.push_back(std::move(dotProductOp));

            /*******************************************************************************
             *                         Reduction(dO * O)                                  */

            int64_t reduction_dim[4] = {b, h, s_q, 1};
            int64_t reduction_stride[4] = {h * s_q, s_q, 1, 1};

            // reduction(dO * O)
            auto afterReductionTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 1, reduction_dim,
                            reduction_stride, true, false);  // is virtual
            auto reductionMaxDesc = cudnn_frontend::ReductionDescBuilder()
                            .setComputeType(HIPDNN_DATA_FLOAT)
                            .setReductionOp(HIPDNN_REDUCE_TENSOR_MAX)
                            .build();

            // Create a reduction max node
            auto reductionMax_op = cudnn_frontend::OperationBuilder(
                            CUDNN_BACKEND_OPERATION_REDUCTION_DESCRIPTOR)
                            .setxDesc(dotProductTensor)
                            .setyDesc(afterReductionTensor)
                            .setreductionDesc(reductionMaxDesc)
                            .build();
            ops.push_back(std::move(reductionMax_op));


            /*******************************************************************************
             *                        reduction(dO * O) * scale prob -> softmaxSum         */

            auto softmaxSumTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, S_SUM_ID, reduction_dim,
                            reduction_stride, false, false);  // not virtual
            auto scaleProbTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, SCALE_PROB, scale_dim,
                            scale_stride, false, true);  // not virtual
            auto softmaxSumOp = binary_pw_op_create(
                            afterReductionTensor, scaleProbTensor,
                            softmaxSumTensor, multiplyDesc);
            ops.push_back(std::move(softmaxSumOp));

            /*******************************************************************************
             *                        Q @ K.T -> P                                        */

            // Inputs from fprop
            auto qTensor = tensor_create(tensorType, Q_ID, q_dim, q_stride, false, false);
            auto kTransposeTensor = tensor_create(
                            tensorType, K_ID, k_transpose_dim,
                            k_transpose_stride, false, false);
            auto pTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 2, p_dim,
                            p_stride, true, false);  // is virtual

            // matmul to calculate dvTensor
            auto matmul_0_Desc = cudnn_frontend::MatMulDescBuilder()
                            .setComputeType(HIPDNN_DATA_FLOAT)
                            .build();

            auto matmul_op0 = cudnn_frontend::OperationBuilder(
                            CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                            .setaMatDesc(qTensor)
                            .setbMatDesc(kTransposeTensor)
                            .setcMatDesc(pTensor)
                            .setmatmulDesc(matmul_0_Desc)
                            .build();

            ops.push_back(std::move(matmul_op0));

            /*******************************************************************************
             *                        P * bmmScale -> pAfterScale                         */

            auto bmmScaleTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, S_CONST_ID, scale_dim,
                            scale_stride, false, true);  // not virtual and by value
            auto pAfterScaleTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 2000, p_dim,
                            p_stride, true, false);  // virtual
            auto scaleOp = binary_pw_op_create(
                            pTensor, bmmScaleTensor, pAfterScaleTensor, multiplyDesc);
            ops.push_back(std::move(scaleOp));

            /*******************************************************************************
             *                          Causal masking -> pAfterMaskTensor                */

            auto pAfterMaskTensor = createCausalMask(
                            b, h, s_q, s_kv, d, layout, tensorType, &ops, pAfterScaleTensor);

            /*******************************************************************************
             *                          pAfterMaskTensor - softmaxStats -> pAfterSubtract */

            auto pAfterSubtractTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 3, p_dim,
                            p_stride, true, false);  // is virtual
            auto softmaxStatsTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, S_STATS_ID, reduction_dim,
                            reduction_stride, false, false);  // not virtual
            auto subtractDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_SUB);
            auto subtract_op = binary_pw_op_create(
                            pAfterMaskTensor, softmaxStatsTensor,
                            pAfterSubtractTensor, subtractDesc);
            ops.push_back(std::move(subtract_op));

            /*******************************************************************************
             *                          e^(pAfterSubtract) -> pAfterSoftmax               */

            auto pAfterSoftmaxTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 4, p_dim,
                            p_stride, true, false);  // is virtual
            auto expDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_EXP);
            auto exp_op = unary_pw_op_create(
                            pAfterSubtractTensor, pAfterSoftmaxTensor, expDesc);
            ops.push_back(std::move(exp_op));

            /*******************************************************************************
             *                          Dropout -> afterScaleDropout                      */

            auto dropoutMaskTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 5, p_dim,
                            p_stride, true, false);  // is virtual
            auto afterScaleDropoutTensor = createDropoutBackward(
                            b, h, s_q, s_kv, d, dropout_probability, tensorType,
                            &ops, pAfterSoftmaxTensor, dropoutMaskTensor);

            /*******************************************************************************
             *                          afterScaleDropout -> sTransposeTensor             */

            auto sTransposeTensor = tensor_create(
                            tensorType, VIRTUAL_ID + 6, p_transpose_dim,
                            p_transpose_stride, true, false);  // is virtual
            auto reshape_op = cudnn_frontend::OperationBuilder(
                            CUDNN_BACKEND_OPERATION_RESHAPE_DESCRIPTOR)
                            .setxDesc(afterScaleDropoutTensor)
                            .setyDesc(sTransposeTensor)
                            .build();
            ops.push_back(std::move(reshape_op));

            // Outputs of bprop
            int64_t dqkv_dim[4] = {b, h, s_kv, d};
            int64_t dqkv_stride[4];
            generateMatrixStrides(
                            b, h, s_q, s_kv, d, dqkv_stride,
                            layout, NVTE_QKV_Matrix::NVTE_Q_Matrix);

            // Outputs of backprop
            auto dQTensor = tensor_create(tensorType, dQ_ID, dqkv_dim, dqkv_stride, false, false);
            auto dKTensor = tensor_create(tensorType, dK_ID, dqkv_dim, dqkv_stride, false, false);
            auto dVTensor = tensor_create(tensorType, dV_ID, dqkv_dim, dqkv_stride, false, false);
                            // not virtual

            /*******************************************************************************
             *                          sTransposeTensor @ dO -> dV                       */

            auto matmul_1_Desc = cudnn_frontend::MatMulDescBuilder()
                            .setComputeType(HIPDNN_DATA_FLOAT)
                            .build();

            auto matmul_op1 = cudnn_frontend::OperationBuilder(
                            CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                            .setaMatDesc(sTransposeTensor)
                            .setbMatDesc(dOTensor)
                            .setcMatDesc(dVTensor)
                            .setmatmulDesc(matmul_1_Desc)
                            .build();

            ops.push_back(std::move(matmul_op1));

            /*******************************************************************************
             *                          dO @ V.T -> dS                                    */

            auto vTransposeTensor = tensor_create(
                            tensorType, V_ID, v_transpose_dim,
                            v_transpose_stride, false, false);
            auto dSTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 7, p_dim,
                            p_stride, true, false);  // is virtual

            auto matmul_2_Desc = cudnn_frontend::MatMulDescBuilder()
                            .setComputeType(HIPDNN_DATA_FLOAT)
                            .build();

            auto matmul_op2 = cudnn_frontend::OperationBuilder(
                            CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                            .setaMatDesc(dOTensor)
                            .setbMatDesc(vTransposeTensor)
                            .setcMatDesc(dSTensor)
                            .setmatmulDesc(matmul_2_Desc)
                            .build();

            ops.push_back(std::move(matmul_op2));

            /*******************************************************************************
             *                          dS * dropoutMask -> dSAfterDropout                */

            auto dSAfterDropoutTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 8, p_dim,
                            p_stride, true, false);  // is virtual
            auto multiply_op = binary_pw_op_create(
                            dSTensor, dropoutMaskTensor,
                            dSAfterDropoutTensor, multiplyDesc);
            ops.push_back(std::move(multiply_op));

            /*******************************************************************************
             *                          dSAfterDropout - softmaxSum -> dsAfterSubtract    */

            auto dsAfterSubtractTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 9, p_dim,
                            p_stride, true, false);  // is virtual
            auto subtract_op2 = binary_pw_op_create(
                            dSAfterDropoutTensor, softmaxSumTensor,
                            dsAfterSubtractTensor, subtractDesc);
            ops.push_back(std::move(subtract_op2));

            /*******************************************************************************
             *                          dsAfterSubtract * afterSoftmax -> dP              */

            auto dPTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 10, p_dim,
                            p_stride, true, false);  // is virtual
            auto multiply_op2 = binary_pw_op_create(
                            dsAfterSubtractTensor, pAfterSoftmaxTensor,
                            dPTensor, multiplyDesc);
            ops.push_back(std::move(multiply_op2));

            /*******************************************************************************
             *                          dP * scaleDropout -> dPAfterDropoutScale          */

            auto dPAfterDropoutScaleTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 11, p_dim,
                            p_stride, true, false);  // is virtual
            auto scaleDropoutTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, D_CONST_ID, scale_dim,
                            scale_stride, false, true);  // is by value
            auto multiply_op3 = binary_pw_op_create(
                            dPTensor, scaleDropoutTensor,
                            dPAfterDropoutScaleTensor, multiplyDesc);
            ops.push_back(std::move(multiply_op3));

            /*******************************************************************************
             *                          dPAfterDropoutScale * bmmScale -> dPScaledTensor  */

            auto dPScaledTensor = tensor_create(
                            HIPDNN_DATA_FLOAT, VIRTUAL_ID + 12, p_dim,
                            p_stride, true, false);  // is virtual
            auto multiply_op4 = binary_pw_op_create(
                            dPAfterDropoutScaleTensor, bmmScaleTensor,
                            dPScaledTensor, multiplyDesc);
            ops.push_back(std::move(multiply_op4));

            /*******************************************************************************
             *                          K.T -> K                                          */

            int64_t kDim[4] = {b, h, s_kv, d};
            int64_t kStride[4];
            generateMatrixStrides(
                            b, h, s_q, s_kv, d, kStride,
                            layout, NVTE_QKV_Matrix::NVTE_K_Matrix);
            auto kTensor = tensor_create(
                            tensorType, VIRTUAL_ID + 13, kDim,
                            kStride, true, false);  // is virtual
            auto reshape_op2 = cudnn_frontend::OperationBuilder(
                            CUDNN_BACKEND_OPERATION_RESHAPE_DESCRIPTOR)
                            .setxDesc(kTransposeTensor)
                            .setyDesc(kTensor)
                            .build();
            ops.push_back(std::move(reshape_op2));

            /*******************************************************************************
             *                          dP @ K -> dqAccumTensor                           */

            auto dqAccumTensor = cudnn_frontend::TensorBuilder()
                .setDim(4, dqkv_dim)
                .setStride(4, dqkv_stride)
                .setId(dQ_ACCUM_ID)
                .setAlignment(16)  // 16B alignment is needed to run a tensor core engine
                .setDataType(HIPDNN_DATA_FLOAT)
                .setVirtual(false)
                .setByValue(false)
                .setReorderType(
                cudnn_frontend::cudnnBackendTensorReordering_t::CUDNN_TENSOR_REORDERING_F16x16)
                .build();

            auto matmul_3_Desc = cudnn_frontend::MatMulDescBuilder()
                                .setComputeType(HIPDNN_DATA_FLOAT)
                                .build();
            auto matmul_op3 = cudnn_frontend::OperationBuilder(
                                CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                                .setaMatDesc(dPTensor)
                                .setbMatDesc(kTensor)
                                .setcMatDesc(dqAccumTensor)
                                .setmatmulDesc(matmul_3_Desc)
                                .build();

            ops.push_back(std::move(matmul_op3));

            /*******************************************************************************
             *                          dP.T @ Q -> dK                                    */

            auto dPTransposeTensor = tensor_create(
                                HIPDNN_DATA_FLOAT, VIRTUAL_ID + 14, p_transpose_dim,
                                p_transpose_stride, true, false);  // is virtual
            auto reshape_op3 = cudnn_frontend::OperationBuilder(
                                CUDNN_BACKEND_OPERATION_RESHAPE_DESCRIPTOR)
                                .setxDesc(dPTensor)
                                .setyDesc(dPTransposeTensor)
                                .build();
            ops.push_back(std::move(reshape_op3));

            auto matmul_4_Desc = cudnn_frontend::MatMulDescBuilder()
                                .setComputeType(HIPDNN_DATA_FLOAT)
                                .build();
            auto matmul_op4 = cudnn_frontend::OperationBuilder(
                                CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                                .setaMatDesc(dPTransposeTensor)
                                .setbMatDesc(qTensor)
                                .setcMatDesc(dKTensor)
                                .setmatmulDesc(matmul_4_Desc)
                                .build();

            ops.push_back(std::move(matmul_op4));

            /*******************************************************************************
             *                          dqAccumTensor @ identity -> dqTensor              */

            auto identityDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_IDENTITY);
            auto identity_op = unary_pw_op_create(dqAccumTensor, dQTensor, identityDesc);
            ops.push_back(std::move(identity_op));

            for (unsigned int i = 0; i < ops.size(); i++) {
                all_ops.push_back(&ops[i]);
            }

            // Create an Operation Graph
            auto opGraph = cudnn_frontend::OperationGraphBuilder()
                               .setHandle(handle)
                               .setOperationGraph(all_ops.size(), all_ops.data())
                               .build();

            cudnn_frontend::EngineConfigList filtered_configs;
            auto statuses = cudnn_frontend::get_heuristics_list<1>(
                {"heuristics_instant"}, opGraph, allowAllConfig, filtered_configs, true);

            if (filtered_configs.size() == 0) {
                cudnn_frontend::set_error_and_throw_exception(
                    nullptr, HIPDNN_STATUS_NOT_SUPPORTED,
                    "run_mha_bprop: No config returned by the heuristics");
            }

            auto plan = cudnn_frontend::ExecutionPlanBuilder()
                            .setHandle(handle)
                            .setEngineConfig(filtered_configs[0], opGraph.getTag())
                            .build();

            cache.insert({descriptor, plan});
            return plan;
        };

        auto plan = get_plan(fmha_bprop_cache, descriptor);

        auto plan_workspace_size = plan.getWorkspaceSize();

        // Exit to request upper level API to allocate memory if needed
        size_t softmaxSum_workspace_size = b * h * s_q * sizeof(float);
        size_t dqAccum_workspace_size = b * s_q * h * d * sizeof(float);
        if (workspace == nullptr) {
            *workspace_size = plan_workspace_size + softmaxSum_workspace_size
                              + dqAccum_workspace_size;
            return;
        }

        void *devPtrSoftmaxSum = static_cast<int8_t *>(workspace) + plan_workspace_size;
        void *devPtrdQAccumulator = static_cast<int8_t *>(devPtrSoftmaxSum)
                                    + softmaxSum_workspace_size;
        NVTE_CHECK_CUDA(hipMemset(devPtrdQAccumulator, 0, dqAccum_workspace_size));

        std::set<std::pair<uint64_t, void *>> data_ptrs;
        // add all the data pointers to be used in the variant pack
        float negInfinity = -1.0E+10f;
        float scale_dropout = 1.0f/(1.0f - dropout_probability);
        data_ptrs.insert(std::pair<uint64_t, void*>(dQ_ID, devPtrdQ));
        data_ptrs.insert(std::pair<uint64_t, void*>(dQ_ACCUM_ID, devPtrdQAccumulator));
        data_ptrs.insert(std::pair<uint64_t, void*>(dK_ID, devPtrdK));
        data_ptrs.insert(std::pair<uint64_t, void*>(dV_ID, devPtrdV));

        data_ptrs.insert(std::pair<uint64_t, void*>(Q_ID, devPtrQ));
        data_ptrs.insert(std::pair<uint64_t, void*>(K_ID, devPtrKTranspose));
        data_ptrs.insert(std::pair<uint64_t, void*>(V_ID, devPtrVTranspose));
        data_ptrs.insert(std::pair<uint64_t, void*>(O_ID, devPtrO));
        data_ptrs.insert(std::pair<uint64_t, void*>(dO_ID, devPtrdO));
        data_ptrs.insert(std::pair<uint64_t, void*>(S_STATS_ID, devPtrSoftmaxStats));
        data_ptrs.insert(std::pair<uint64_t, void*>(S_SUM_ID, devPtrSoftmaxSum));
        data_ptrs.insert(std::pair<uint64_t, void*>(D_SEED_ID, devPtrDropoutSeed));
        data_ptrs.insert(std::pair<uint64_t, void*>(D_OFFSET_ID, devPtrDropoutOffset));
        data_ptrs.insert(std::pair<uint64_t, void*>(MASK_VAL_ID, &negInfinity));

        float scaleProb = 1.0f - dropout_probability;
        data_ptrs.insert(std::pair<uint64_t, void*>(D_CONST_ID, &scale_dropout));
        data_ptrs.insert(std::pair<uint64_t, void*>(S_CONST_ID, &scaling_factor));
        data_ptrs.insert(std::pair<uint64_t, void*>(SCALE_PROB, &scaleProb));

        auto variantPack = cudnn_frontend::VariantPackBuilder()
                               .setWorkspacePointer(workspace)
                               .setDataPointers(data_ptrs)
                               .build();

        NVTE_CHECK_CUDNN(
            cudnnBackendExecute(handle, plan.get_raw_desc(), variantPack.get_raw_desc()));
    } catch (cudnn_frontend::cudnnException &e) {
        NVTE_ERROR(e.what());
    }
}

}  // namespace fused_attn

using namespace transformer_engine::fused_attn;
void fused_attn_arbitrary_seqlen_fwd_qkvpacked(
    size_t batch, size_t max_seqlen, size_t num_head, size_t head_dim, bool is_training,
    float attn_scale, float p_dropout, NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type,
    NVTE_Mask_Type mask_type, const Tensor *input_QKV, const Tensor *input_Bias, Tensor *output_O,
    NVTETensorPack *Aux_CTX_Tensors, const Tensor *cu_seqlens, const Tensor *rng_state,
    Tensor *workspace, hipStream_t stream, hipdnnHandle_t handle) {
    using namespace transformer_engine;

    NVTE_CHECK(qkv_layout == NVTE_QKV_Layout::NVTE_QKV_INTERLEAVED,
               "qkv_layout must be NVTE_QKV_Layout::NVTE_QKV_INTERLEAVED.");

    // QKV shape is [b, s, 3, h, d]
    void *devPtrQKV = input_QKV->data.dptr;
    const auto stride = num_head * head_dim;

    void *devPtrQ = static_cast<void *>(devPtrQKV);
    void *devPtrK = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + stride);
    void *devPtrV = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + 2 * stride);

    void *devPtrO = output_O->data.dptr;

    void *devPtrS = nullptr;

    if (Aux_CTX_Tensors->size == 0) {
        Aux_CTX_Tensors->size = 2;
        Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
        output_S->data.dptr = nullptr;
        output_S->data.shape = {batch, num_head, max_seqlen, 1};
        output_S->data.dtype = DType::kFloat32;
        Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
        output_rng_state->data.dptr = nullptr;
        output_rng_state->data.shape = {2};
        output_rng_state->data.dtype = DType::kInt64;
    } else if (Aux_CTX_Tensors->size == 2) {
        Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
        devPtrS = output_S->data.dptr;
        Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
        output_rng_state->data.dptr = rng_state->data.dptr;
    }

    void* devPtrDropoutSeed = rng_state->data.dptr;
    void* devPtrDropoutOffset = reinterpret_cast<void *>(
                    reinterpret_cast<uint64_t*>(rng_state->data.dptr) + 1);

    const DType QKV_type = input_QKV->data.dtype;
    size_t workspace_size = 0;

    fused_attn_arbitrary_seqlen_fwd_impl(batch, num_head, max_seqlen, max_seqlen, head_dim,
                                is_training, attn_scale, p_dropout, qkv_layout,
                                devPtrQ, devPtrK, devPtrV, devPtrS, devPtrO,
                                devPtrDropoutSeed, devPtrDropoutOffset,
                                get_cudnn_dtype(QKV_type),
                                workspace->data.dptr, &workspace_size, stream, handle);

    if (workspace_size > 0) {
        if (workspace->data.dptr == nullptr) {
            workspace->data.shape = {workspace_size};
            workspace->data.dtype = DType::kByte;
            return;
        }
    } else if (workspace_size == 0) {
        workspace->data.shape = {1};
        workspace->data.dtype = DType::kByte;
        return;
    }
}

void fused_attn_arbitrary_seqlen_bwd_qkvpacked(size_t batch, size_t max_seqlen, size_t num_head,
                                  size_t head_dim, float attn_scale, float p_dropout,
                                  NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type,
                                  NVTE_Mask_Type mask_type,
                                  const Tensor *input_QKV, const Tensor *input_O,
                                  const Tensor *input_dO, Tensor *output_S,
                                  Tensor *output_dQKV, Tensor *output_dBias,
                                  const Tensor *cu_seqlens, const Tensor *rng_state,
                                  Tensor *workspace, hipStream_t stream, hipdnnHandle_t handle) {
    using namespace transformer_engine;

    NVTE_CHECK(qkv_layout == NVTE_QKV_Layout::NVTE_QKV_INTERLEAVED,
               "qkv_layout must be NVTE_QKV_INTERLEAVED.");

    // QKV shape is [b, s, 3, h, d]
    void *devPtrQKV = input_QKV->data.dptr;

    auto stride = num_head * head_dim;
    void *devPtrQ = devPtrQKV;
    void *devPtrK = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + stride);
    void *devPtrV = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + 2 * stride);

    void* devPtrO = input_O->data.dptr;
    void *devPtrdO = input_dO->data.dptr;

    // dQKV shape is [b, s, 3, h, d]
    void *devPtrdQKV = output_dQKV->data.dptr;
    void *devPtrdQ = devPtrdQKV;
    void *devPtrdK = static_cast<void *>(static_cast<int8_t *>(devPtrdQKV) + stride);
    void *devPtrdV = static_cast<void *>(static_cast<int8_t *>(devPtrdQKV) + 2 * stride);

    void *devPtrSoftmaxStats = nullptr;
    devPtrSoftmaxStats = output_S->data.dptr;

    void* devPtrDropoutSeed = rng_state->data.dptr;
    void* devPtrDropoutOffset = reinterpret_cast<void *>(
                    reinterpret_cast<uint64_t*>(rng_state->data.dptr) + 1);

    const auto qkv_type = input_QKV->data.dtype;
    size_t workspace_size = 0;

    fused_attn_arbitrary_seqlen_bwd_impl(batch, num_head, max_seqlen, max_seqlen, head_dim,
                                attn_scale, p_dropout, qkv_layout,
                                devPtrQ, devPtrK, devPtrV, devPtrO, devPtrSoftmaxStats,
                                devPtrdQ, devPtrdK, devPtrdV, devPtrdO,
                                devPtrDropoutSeed, devPtrDropoutOffset,
                                get_cudnn_dtype(qkv_type),
                                workspace->data.dptr, &workspace_size, stream, handle);

    if (workspace_size > 0) {
        if (workspace->data.dptr == nullptr) {
            workspace->data.shape = {workspace_size};
            workspace->data.dtype = DType::kByte;
            return;
        }
    } else if (workspace_size == 0) {
        workspace->data.shape = {1};
        workspace->data.dtype = DType::kByte;
        return;
    }
}
}  // namespace transformer_engine
#endif  // CUDNN_VERSION >= 8900
