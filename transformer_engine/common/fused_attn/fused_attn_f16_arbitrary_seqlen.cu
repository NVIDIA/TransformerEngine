/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "fused_attn_f16_arbitrary_seqlen.h"

#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <cudnn_frontend.h>
#include <cudnn_frontend_utils.h>
#include <map>
#include <vector>

#include "../common.h"
#include "utils.h"
#include "../util/hip/hip_runtime.h"
#include "../util/system.h"

#if (CUDNN_VERSION >= 8900)
#define Q_ID 1
#define K_ID 2
#define V_ID 3
#define O_ID 4
#define S_ID 5
#define B_ID 6
#define D_CONST_ID 7
#define S_CONST_ID 8
#define Q_SEQLEN_ID 9
#define K_SEQLEN_ID 10
#define dQ_ID 11
#define dK_ID 12
#define dV_ID 13
#define dO_ID 14
#define MASK_VAL_ID 15
#define dS_ID 16
#define D_SEED_ID 17
#define D_OFFSET_ID 18
#define S_STATS_ID 19
#define S_SUM_ID 20
#define SCALE_PROB 21
#define K_TRANSPOSE_ID 22
#define dQ_ACCUM_ID 23

#define VIRTUAL_ID 30

namespace transformer_engine {
namespace fused_attn {
void fused_attn_arbitrary_seqlen_fwd_impl(
                int64_t b, int64_t h, int64_t hg, int64_t s_q, int64_t s_kv, int64_t d,
                int64_t bias_b, int64_t bias_h,
                bool is_training, float scaling_factor, float dropout_probability,
                NVTE_QKV_Layout layout,
                NVTE_Bias_Type bias_type, NVTE_Mask_Type mask_type,
                void *devPtrQ, void *devPtrK, void *devPtrV, void *devPtrBias,
                void *devPtrSoftmaxStats, void *devPtrO,
                void* devPtrDropoutSeed, void* devPtrDropoutOffset,
                void* devPtrCuSeqlensQ, void* devPtrCuSeqlensKV,
                cudnn_frontend::DataType_t tensorType,
                void *workspace, size_t *workspace_size,
                hipStream_t stream, hipdnnHandle_t handle) {
    bool is_bias = (bias_type == NVTE_Bias_Type::NVTE_POST_SCALE_BIAS);
    bool is_alibi = (bias_type == NVTE_Bias_Type::NVTE_ALIBI);
    bool is_causal = ((mask_type == NVTE_Mask_Type::NVTE_CAUSAL_MASK)
        || (mask_type == NVTE_Mask_Type::NVTE_PADDING_CAUSAL_MASK));
    bool is_padding = ((mask_type == NVTE_Mask_Type::NVTE_PADDING_MASK)
        || (mask_type == NVTE_Mask_Type::NVTE_PADDING_CAUSAL_MASK));
    bool is_dropout = (is_training && dropout_probability != 0.0f);

    try {
        FADescriptor_v1 descriptor{b,                   h,
                                   hg,                  s_q,
                                   s_kv,                d,
                                   bias_b,              bias_h,
                                   scaling_factor,      is_training,
                                   dropout_probability, layout,
                                   bias_type,           mask_type,
                                   tensorType};

        namespace fe = cudnn_frontend;
        using graph_and_tensors = std::tuple<std::shared_ptr<fe::graph::Graph>,
              std::shared_ptr<fe::graph::Tensor_attributes>,  // Q
              std::shared_ptr<fe::graph::Tensor_attributes>,  // K
              std::shared_ptr<fe::graph::Tensor_attributes>,  // V
              std::shared_ptr<fe::graph::Tensor_attributes>,  // attn_scale
              std::shared_ptr<fe::graph::Tensor_attributes>,  // O
              std::shared_ptr<fe::graph::Tensor_attributes>,  // Stats
              std::shared_ptr<fe::graph::Tensor_attributes>,  // bias
              std::shared_ptr<fe::graph::Tensor_attributes>,  // seq_q
              std::shared_ptr<fe::graph::Tensor_attributes>,  // seq_kv
              std::shared_ptr<fe::graph::Tensor_attributes>,  // dropout_seed
              std::shared_ptr<fe::graph::Tensor_attributes> >;  // dropout_offset

        using CacheType = std::map<FADescriptor_v1, graph_and_tensors>;
        static thread_local CacheType sdpa_f16_fprop_cache;

        // Get plan from cache if cache is available, otherwise create one
        auto get_graph = [&](CacheType &cache, const FADescriptor_v1 &descriptor)
            -> graph_and_tensors {
            // if hit, return
            auto it = cache.find(descriptor);
            if (it != cache.end()) {
                auto graph = it->second;
                return graph;
            }

            // otherwise, build the op_graph and the plan. Then update cache
            auto mha_graph = std::make_shared<fe::graph::Graph>();
            mha_graph->set_io_data_type(tensorType)
                    .set_intermediate_data_type(fe::DataType_t::FLOAT)
                    .set_compute_data_type(fe::DataType_t::FLOAT);

            std::shared_ptr<fe::graph::Tensor_attributes> Q, K, V, attn_scale;
            std::shared_ptr<fe::graph::Tensor_attributes> bias, seq_q, seq_kv;
            std::shared_ptr<fe::graph::Tensor_attributes> dropout_seed, dropout_offset;

            std::vector<int64_t> q_stride(4);
            std::vector<int64_t> k_stride(4);
            std::vector<int64_t> v_stride(4);
            generateMatrixStrides(b, h, s_q, s_kv, d, q_stride.data(),
                    layout, NVTE_QKV_Matrix::NVTE_Q_Matrix);
            generateMatrixStrides(b, hg, s_q, s_kv, d, k_stride.data(),
                    layout, NVTE_QKV_Matrix::NVTE_K_Matrix);
            generateMatrixStrides(b, hg, s_q, s_kv, d, v_stride.data(),
                    layout, NVTE_QKV_Matrix::NVTE_V_Matrix);
            Q = mha_graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("Q")
                            .set_dim({b, h, s_q, d})
                            .set_stride(q_stride));
            K = mha_graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("K")
                            .set_dim({b, hg, s_kv, d})
                            .set_stride(k_stride));
            V = mha_graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("V")
                            .set_dim({b, hg, s_kv, d})
                            .set_stride(v_stride));

            attn_scale = mha_graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("attn_scale")
                            .set_dim({1, 1, 1, 1})
                            .set_stride({1, 1, 1, 1})
                            .set_is_pass_by_value(true)
                            .set_data_type(fe::DataType_t::FLOAT));

            fe::graph::SDPA_attributes sdpa_options;
            sdpa_options = fe::graph::SDPA_attributes()
                            .set_name("flash_attention")
                            .set_is_inference(!is_training)
                            .set_causal_mask(is_causal)
                            .set_attn_scale(attn_scale);

            sdpa_options.set_alibi_mask(is_alibi);

            if (is_bias) {
                bias = mha_graph->tensor(fe::graph::Tensor_attributes()
                                .set_name("bias")
                                .set_dim({bias_b, bias_h, s_q, s_kv})
                                .set_stride({bias_h * s_q * s_kv, s_q * s_kv, s_kv, 1}));
                sdpa_options.set_bias(bias);
            }

            if (is_padding) {
                seq_q  = mha_graph->tensor(fe::graph::Tensor_attributes()
                                .set_name("seq_q")
                                .set_dim({b, 1, 1, 1})
                                .set_stride({1, 1, 1, 1})
                                .set_data_type(fe::DataType_t::INT32));
                seq_kv = mha_graph->tensor(fe::graph::Tensor_attributes()
                                .set_name("seq_kv")
                                .set_dim({b, 1, 1, 1})
                                .set_stride({1, 1, 1, 1})
                                .set_data_type(fe::DataType_t::INT32));
                sdpa_options.set_padding_mask(is_padding)
                                .set_seq_len_q(seq_q)
                                .set_seq_len_kv(seq_kv);
            }

            if (is_dropout) {
                dropout_seed = mha_graph->tensor(fe::graph::Tensor_attributes()
                                .set_name("Seed")
                                .set_dim({1, 1, 1, 1})
                                .set_stride({1, 1, 1, 1})
                                .set_data_type(fe::DataType_t::INT64));
                dropout_offset = mha_graph->tensor(fe::graph::Tensor_attributes()
                                .set_name("Offset")
                                .set_dim({1, 1, 1, 1})
                                .set_stride({1, 1, 1, 1})
                                .set_data_type(fe::DataType_t::INT64));
                sdpa_options.set_dropout(
                                dropout_probability, dropout_seed, dropout_offset);
            }

            auto [O, Stats] = mha_graph->sdpa(Q, K, V, sdpa_options);

            std::vector<int64_t> o_stride(4);
            generateMatrixStrides(b, h, s_q, s_kv, d, o_stride.data(),
                    layout, NVTE_QKV_Matrix::NVTE_O_Matrix);
            O->set_output(true).set_dim({b, h, s_q, d}).set_stride(o_stride);

            if (is_training) {
                Stats->set_output(true).set_data_type(fe::DataType_t::FLOAT)
                        .set_dim({b, h, s_q, 1})
                        .set_stride({h * s_q, s_q, 1, 1});
            }

            std::tuple<std::shared_ptr<fe::graph::Tensor_attributes>,  // Q
                    std::shared_ptr<fe::graph::Tensor_attributes>,  // K
                    std::shared_ptr<fe::graph::Tensor_attributes>,  // V
                    std::shared_ptr<fe::graph::Tensor_attributes>,  // attn_scale
                    std::shared_ptr<fe::graph::Tensor_attributes> >  // O
            key_tensors_tuple = std::make_tuple(Q, K, V, attn_scale, O);
            auto Stats_tuple = is_training ? std::make_tuple(Stats) : std::make_tuple(nullptr);
            auto bias_tuple = is_bias ? std::make_tuple(bias) : std::make_tuple(nullptr);
            auto padding_tuple = is_padding ?
                std::make_tuple(seq_q, seq_kv) : std::make_tuple(nullptr, nullptr);
            auto dropout_tuple = is_dropout ?
                std::make_tuple(dropout_seed, dropout_offset) : std::make_tuple(nullptr, nullptr);
            auto return_empty_tuple = std::tuple_cat(
                std::make_tuple(nullptr), key_tensors_tuple,
                Stats_tuple, bias_tuple, padding_tuple, dropout_tuple);

            NVTE_CHECK_CUDNN_FE(mha_graph->validate());
            NVTE_CHECK_CUDNN_FE(mha_graph->build_operation_graph(handle));
            NVTE_CHECK_CUDNN_FE(mha_graph->create_execution_plans({fe::HeurMode_t::A}));
            NVTE_CHECK_CUDNN_FE(mha_graph->check_support(handle));
            NVTE_CHECK_CUDNN_FE(mha_graph->build_plans(handle));

            auto return_tuple = std::tuple_cat(
                std::make_tuple(mha_graph), key_tensors_tuple,
                Stats_tuple, bias_tuple, padding_tuple, dropout_tuple);
            cache.insert({descriptor, return_tuple});

            return return_tuple;
        };

        auto [mha_graph, Q, K, V, attn_scale, O, Stats,
            bias, seq_q, seq_kv, dropout_seed, dropout_offset] = get_graph(
                sdpa_f16_fprop_cache, descriptor);

        auto plan_workspace_size = mha_graph->get_workspace_size();

        // Exit to request upper level API to allocate memory if needed
        size_t actual_seqlen_workspace_size = 2 * b * sizeof(int32_t);
        if (workspace == nullptr) {
            *workspace_size = plan_workspace_size + actual_seqlen_workspace_size;
            return;
        }

        // cuDNN stream check needs to be moved here to support dummy kernel calls with
        // null streams for sizing the cuDNN workspace.
        NVTE_CHECK_CUDNN(hipdnnSetStream(handle, stream));

        // Build variant pack
        std::unordered_map<std::shared_ptr<fe::graph::Tensor_attributes>, void*> variant_pack = {
            {Q, devPtrQ},
            {K, devPtrK},
            {V, devPtrV},
            {attn_scale, &scaling_factor},
            {O, devPtrO}};

        if (is_training) {
            variant_pack[Stats] = devPtrSoftmaxStats;
        }

        if (is_bias) {
            variant_pack[bias] = devPtrBias;
        }

        if (is_padding) {
            constexpr size_t nthreads_per_block = 128;
            const size_t grid = (b + nthreads_per_block - 1) / nthreads_per_block;
            void *devActualSeqlenQ = static_cast<int8_t *>(workspace) + plan_workspace_size;
            void *devActualSeqlenKV = static_cast<int8_t *>(devActualSeqlenQ) + b * sizeof(int32_t);
            cu_seqlens_to_actual_seqlens<<<grid, nthreads_per_block, 0, stream>>>(
                b, static_cast<const int32_t *>(devPtrCuSeqlensQ),
                static_cast<const int32_t *>(devPtrCuSeqlensKV),
                static_cast<int32_t *>(devActualSeqlenQ),
                static_cast<int32_t *>(devActualSeqlenKV));
            variant_pack[seq_q]  = devActualSeqlenQ;
            variant_pack[seq_kv] = devActualSeqlenKV;
        }

        if (is_dropout) {
            variant_pack[dropout_seed] = devPtrDropoutSeed;
            variant_pack[dropout_offset] = devPtrDropoutOffset;
        }

        NVTE_CHECK_CUDNN_FE(mha_graph->execute(handle, variant_pack, workspace));
    } catch (cudnn_frontend::cudnnException &e) {
        NVTE_ERROR(e.what());
    }
}

void fused_attn_arbitrary_seqlen_bwd_impl(
                int64_t b, int64_t h, int64_t hg, int64_t s_q, int64_t s_kv, int64_t d,
                int64_t bias_b, int64_t bias_h,
                float scaling_factor, float dropout_probability, NVTE_QKV_Layout layout,
                NVTE_Bias_Type bias_type, NVTE_Mask_Type mask_type,
                void* devPtrQ, void* devPtrKTranspose, void* devPtrVTranspose,
                void* devPtrO, void* devPtrSoftmaxStats, void* devPtrBias,
                void* devPtrdQ, void* devPtrdK, void* devPtrdV, void* devPtrdO, void* devPtrdBias,
                void* devPtrDropoutSeed, void* devPtrDropoutOffset,
                void* devPtrCuSeqlensQ, void* devPtrCuSeqlensKV,
                cudnn_frontend::DataType_t tensorType, void *workspace, size_t *workspace_size,
                hipStream_t stream, hipdnnHandle_t handle) {
    bool is_bias = (bias_type == NVTE_Bias_Type::NVTE_POST_SCALE_BIAS);
    bool is_alibi = (bias_type == NVTE_Bias_Type::NVTE_ALIBI);
    bool is_causal = ((mask_type == NVTE_Mask_Type::NVTE_CAUSAL_MASK)
        || (mask_type == NVTE_Mask_Type::NVTE_PADDING_CAUSAL_MASK));
    bool is_padding = ((mask_type == NVTE_Mask_Type::NVTE_PADDING_MASK)
        || (mask_type == NVTE_Mask_Type::NVTE_PADDING_CAUSAL_MASK));
    bool is_dropout = (dropout_probability != 0.0f);

    try {
        FADescriptor_v1 descriptor{b,                   h,
                                   hg,                  s_q,
                                   s_kv,                d,
                                   bias_b,              bias_h,
                                   scaling_factor,      true,
                                   dropout_probability, layout,
                                   bias_type,           mask_type,
                                   tensorType};

        namespace fe = cudnn_frontend;
        using graph_and_tensors = std::tuple<std::shared_ptr<fe::graph::Graph>,
              std::shared_ptr<fe::graph::Tensor_attributes>,  // q
              std::shared_ptr<fe::graph::Tensor_attributes>,  // k
              std::shared_ptr<fe::graph::Tensor_attributes>,  // v
              std::shared_ptr<fe::graph::Tensor_attributes>,  // o
              std::shared_ptr<fe::graph::Tensor_attributes>,  // dO
              std::shared_ptr<fe::graph::Tensor_attributes>,  // stats
              std::shared_ptr<fe::graph::Tensor_attributes>,  // attn_scale
              std::shared_ptr<fe::graph::Tensor_attributes>,  // dQ
              std::shared_ptr<fe::graph::Tensor_attributes>,  // dK
              std::shared_ptr<fe::graph::Tensor_attributes>,  // dV
              std::shared_ptr<fe::graph::Tensor_attributes>,  // bias
              std::shared_ptr<fe::graph::Tensor_attributes>,  // dBias
              std::shared_ptr<fe::graph::Tensor_attributes>,  // seq_q
              std::shared_ptr<fe::graph::Tensor_attributes>,  // seq_kv
              std::shared_ptr<fe::graph::Tensor_attributes>,  // dropout_seed
              std::shared_ptr<fe::graph::Tensor_attributes> >;  // dropout_offset

        using CacheType = std::map<FADescriptor_v1, graph_and_tensors>;
        static thread_local CacheType sdpa_f16_bprop_cache;

        // Get plan from cache if cache is available, otherwise create one
        auto get_graph = [&](CacheType &cache, const FADescriptor_v1 &descriptor)
            -> graph_and_tensors {
            // if hit, return
            auto it = cache.find(descriptor);
            if (it != cache.end()) {
                auto graph = it->second;
                return graph;
            }

            // otherwise, build the op_graph and the plan. Then update cache
            auto mha_graph = std::make_shared<fe::graph::Graph>();
            mha_graph->set_io_data_type(tensorType)
                    .set_intermediate_data_type(fe::DataType_t::FLOAT)
                    .set_compute_data_type(fe::DataType_t::FLOAT);

            std::shared_ptr<fe::graph::Tensor_attributes> q, k, v, o, dO, stats, attn_scale;
            std::shared_ptr<fe::graph::Tensor_attributes> bias, dBias, seq_q, seq_kv;
            std::shared_ptr<fe::graph::Tensor_attributes> dropout_seed, dropout_offset;

            std::vector<int64_t> q_stride(4);
            std::vector<int64_t> k_stride(4);
            std::vector<int64_t> v_stride(4);
            std::vector<int64_t> o_stride(4);
            generateMatrixStrides(b, h, s_q, s_kv, d, q_stride.data(),
                    layout, NVTE_QKV_Matrix::NVTE_Q_Matrix);
            generateMatrixStrides(b, hg, s_q, s_kv, d, k_stride.data(),
                    layout, NVTE_QKV_Matrix::NVTE_K_Matrix);
            generateMatrixStrides(b, hg, s_q, s_kv, d, v_stride.data(),
                    layout, NVTE_QKV_Matrix::NVTE_V_Matrix);
            generateMatrixStrides(b, h, s_q, s_kv, d, o_stride.data(),
                    layout, NVTE_QKV_Matrix::NVTE_O_Matrix);
            q = mha_graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("Q")
                            .set_dim({b, h, s_q, d})
                            .set_stride(q_stride));
            k = mha_graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("K")
                            .set_dim({b, hg, s_kv, d})
                            .set_stride(k_stride));
            v = mha_graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("V")
                            .set_dim({b, hg, s_kv, d})
                            .set_stride(v_stride));
            o = mha_graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("O")
                            .set_dim({b, h, s_q, d})
                            .set_stride(o_stride));
            dO = mha_graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("dO")
                            .set_dim({b, h, s_q, d})
                            .set_stride(o_stride));
            stats = mha_graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("stats")
                            .set_dim({b, h, s_q, 1})
                            .set_stride({h * s_q, s_q, 1, 1})
                            .set_data_type(fe::DataType_t::FLOAT));

            attn_scale = mha_graph->tensor(fe::graph::Tensor_attributes()
                            .set_name("attn_scale")
                            .set_dim({1, 1, 1, 1})
                            .set_stride({1, 1, 1, 1})
                            .set_is_pass_by_value(true)
                            .set_data_type(fe::DataType_t::FLOAT));

            fe::graph::SDPA_backward_attributes sdpa_backward_options;
            sdpa_backward_options = fe::graph::SDPA_backward_attributes()
                            .set_name("flash_attention_backward")
                            .set_causal_mask(is_causal)
                            .set_attn_scale(attn_scale);

            sdpa_backward_options.set_alibi_mask(is_alibi);

            if (is_bias) {
                bias = mha_graph->tensor(fe::graph::Tensor_attributes()
                                .set_name("bias")
                                .set_dim({bias_b, bias_h, s_q, s_kv})
                                .set_stride({bias_h * s_q * s_kv, s_q * s_kv, s_kv, 1}));
                dBias = mha_graph->tensor(fe::graph::Tensor_attributes()
                                .set_name("dBias")
                                .set_dim({bias_b, bias_h, s_q, s_kv})
                                .set_stride({bias_h * s_q * s_kv, s_q * s_kv, s_kv, 1}));
                sdpa_backward_options.set_bias(bias);
                // shapes [1, 1, s, s], [b, 1, s, s], [b, h, s, s]
                // are not supported for dbias calculation but they are
                // supported for forward bias calculation
                if ((bias_b == 1) && (bias_h == h)) {
                  sdpa_backward_options.set_dbias(dBias);
                }
            }

            if (is_padding) {
                seq_q  = mha_graph->tensor(fe::graph::Tensor_attributes()
                                .set_name("seq_q")
                                .set_dim({b, 1, 1, 1})
                                .set_stride({1, 1, 1, 1})
                                .set_data_type(fe::DataType_t::INT32));
                seq_kv = mha_graph->tensor(fe::graph::Tensor_attributes()
                                .set_name("seq_kv")
                                .set_dim({b, 1, 1, 1})
                                .set_stride({1, 1, 1, 1})
                                .set_data_type(fe::DataType_t::INT32));
                sdpa_backward_options.set_padding_mask(is_padding)
                                .set_seq_len_q(seq_q)
                                .set_seq_len_kv(seq_kv);
            }

            if (is_dropout) {
                dropout_seed = mha_graph->tensor(fe::graph::Tensor_attributes()
                                .set_name("Seed")
                                .set_dim({1, 1, 1, 1})
                                .set_stride({1, 1, 1, 1})
                                .set_data_type(fe::DataType_t::INT64));
                dropout_offset = mha_graph->tensor(fe::graph::Tensor_attributes()
                                .set_name("Offset")
                                .set_dim({1, 1, 1, 1})
                                .set_stride({1, 1, 1, 1})
                                .set_data_type(fe::DataType_t::INT64));
                sdpa_backward_options.set_dropout(
                                dropout_probability, dropout_seed, dropout_offset);
            }

            auto [dQ, dK, dV] = mha_graph->sdpa_backward(
                q, k, v, o, dO, stats, sdpa_backward_options);

            dQ->set_output(true)
                    .set_dim({b, h, s_q, d})
                    .set_stride(q_stride);
            dK->set_output(true)
                    .set_dim({b, hg, s_kv, d})
                    .set_stride(k_stride);
            dV->set_output(true)
                    .set_dim({b, hg, s_kv, d})
                    .set_stride(v_stride);

            std::tuple<std::shared_ptr<fe::graph::Tensor_attributes>,  // q
                    std::shared_ptr<fe::graph::Tensor_attributes>,  // k
                    std::shared_ptr<fe::graph::Tensor_attributes>,  // v
                    std::shared_ptr<fe::graph::Tensor_attributes>,  // o
                    std::shared_ptr<fe::graph::Tensor_attributes>,  // dO
                    std::shared_ptr<fe::graph::Tensor_attributes>,  // stats
                    std::shared_ptr<fe::graph::Tensor_attributes>,  // attn_scale
                    std::shared_ptr<fe::graph::Tensor_attributes>,  // dQ
                    std::shared_ptr<fe::graph::Tensor_attributes>,  // dK
                    std::shared_ptr<fe::graph::Tensor_attributes> >  // dV
            key_tensors_tuple = std::make_tuple(q, k, v, o, dO, stats, attn_scale, dQ, dK, dV);
            auto bias_tuple = is_bias ?
                std::make_tuple(bias, dBias) : std::make_tuple(nullptr, nullptr);
            auto padding_tuple = is_padding ?
                std::make_tuple(seq_q, seq_kv) : std::make_tuple(nullptr, nullptr);
            auto dropout_tuple = is_dropout ?
                std::make_tuple(dropout_seed, dropout_offset) : std::make_tuple(nullptr, nullptr);
            auto return_empty_tuple = std::tuple_cat(
                std::make_tuple(nullptr), key_tensors_tuple,
                bias_tuple, padding_tuple, dropout_tuple);

            NVTE_CHECK_CUDNN_FE(mha_graph->validate());
            NVTE_CHECK_CUDNN_FE(mha_graph->build_operation_graph(handle));
            NVTE_CHECK_CUDNN_FE(mha_graph->create_execution_plans({fe::HeurMode_t::A}));
            NVTE_CHECK_CUDNN_FE(mha_graph->check_support(handle));
            NVTE_CHECK_CUDNN_FE(mha_graph->build_plans(handle));

            auto return_tuple = std::tuple_cat(
                std::make_tuple(mha_graph), key_tensors_tuple,
                bias_tuple, padding_tuple, dropout_tuple);
            cache.insert({descriptor, return_tuple});

            return return_tuple;
        };

        auto [mha_graph, q, k, v, o, dO, stats, attn_scale, dQ, dK, dV,
            bias, dBias, seq_q, seq_kv, dropout_seed, dropout_offset] = get_graph(
                sdpa_f16_bprop_cache, descriptor);

        auto plan_workspace_size = mha_graph->get_workspace_size();

        // Exit to request upper level API to allocate memory if needed
        size_t actual_seqlen_workspace_size = 2 * b * sizeof(int32_t);
        if (workspace == nullptr) {
            *workspace_size = plan_workspace_size + actual_seqlen_workspace_size;
            return;
        }

        // cuDNN stream check needs to be moved here to support dummy kernel calls with
        // null streams for sizing the cuDNN workspace.
        NVTE_CHECK_CUDNN(hipdnnSetStream(handle, stream));

        // build variant pack
        std::unordered_map<std::shared_ptr<fe::graph::Tensor_attributes>, void*> variant_pack = {
            {q, devPtrQ},
            {k, devPtrKTranspose},
            {v, devPtrVTranspose},
            {o, devPtrO},
            {dO, devPtrdO},
            {stats, devPtrSoftmaxStats},
            {attn_scale, &scaling_factor},
            {dQ, devPtrdQ},
            {dK, devPtrdK},
            {dV, devPtrdV},
        };

        if (is_bias) {
            variant_pack[bias] = devPtrBias;
            if ((bias_b == 1) && (bias_h == h)) {
              variant_pack[dBias] = devPtrdBias;
            } else {
              variant_pack[dBias] = nullptr;
            }
        }

        if (is_padding) {
            constexpr size_t nthreads_per_block = 128;
            const size_t grid = (b + nthreads_per_block - 1) / nthreads_per_block;
            void *devActualSeqlenQ = static_cast<int8_t *>(workspace) + plan_workspace_size;
            void *devActualSeqlenKV = static_cast<int8_t *>(devActualSeqlenQ) + b * sizeof(int32_t);
            cu_seqlens_to_actual_seqlens<<<grid, nthreads_per_block, 0, stream>>>(
                b, static_cast<const int32_t *>(devPtrCuSeqlensQ),
                static_cast<const int32_t *>(devPtrCuSeqlensKV),
                static_cast<int32_t *>(devActualSeqlenQ),
                static_cast<int32_t *>(devActualSeqlenKV));
            variant_pack[seq_q]  = devActualSeqlenQ;
            variant_pack[seq_kv] = devActualSeqlenKV;
        }

        if (is_dropout) {
            variant_pack[dropout_seed] = devPtrDropoutSeed;
            variant_pack[dropout_offset] = devPtrDropoutOffset;
        }

        NVTE_CHECK_CUDNN_FE(mha_graph->execute(handle, variant_pack, workspace));
    } catch (cudnn_frontend::cudnnException &e) {
        NVTE_ERROR(e.what());
    }
}
}  // namespace fused_attn

using namespace transformer_engine::fused_attn;
void fused_attn_arbitrary_seqlen_fwd_qkvpacked(
    size_t batch, size_t num_attn_heads, size_t max_seqlen, size_t head_dim, bool is_training,
    float attn_scale, float p_dropout, NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type,
    NVTE_Mask_Type mask_type, const Tensor *input_QKV, const Tensor *input_Bias, Tensor *output_O,
    NVTETensorPack *Aux_CTX_Tensors, const Tensor *cu_seqlens, const Tensor *rng_state,
    Tensor *workspace, hipStream_t stream, hipdnnHandle_t handle) {
    using namespace transformer_engine;

    const auto QKV_type = input_QKV->data.dtype;
    void *devPtrQKV = input_QKV->data.dptr;
    NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(qkv_layout);
    size_t stride = 0;
    if (layout_group == NVTE_QKV_Layout_Group::NVTE_3HD) {
        stride = typeToSize(QKV_type) * num_attn_heads * head_dim;
    } else if (layout_group == NVTE_QKV_Layout_Group::NVTE_H3D) {
        stride = typeToSize(QKV_type) * head_dim;
    }
    void *devPtrQ = static_cast<void *>(devPtrQKV);
    void *devPtrK = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + stride);
    void *devPtrV = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + 2 * stride);

    void *devPtrBias = nullptr;
    size_t bias_b = 0;
    size_t bias_h = 0;
    if ((bias_type != NVTE_Bias_Type::NVTE_NO_BIAS) && (bias_type != NVTE_Bias_Type::NVTE_ALIBI)) {
        devPtrBias = input_Bias->data.dptr;
        bias_b = input_Bias->data.shape[0];
        bias_h = input_Bias->data.shape[1];
    }
    void *devPtrO = output_O->data.dptr;
    void *devPtrS = nullptr;
    void *devPtrCuSeqlens = cu_seqlens->data.dptr;

    if (Aux_CTX_Tensors->size == 0) {
        if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI)) {
            Aux_CTX_Tensors->size = 3;
            Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
            output_S->data.dptr = nullptr;
            output_S->data.shape = {batch, num_attn_heads, max_seqlen, 1};
            output_S->data.dtype = DType::kFloat32;
            Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
            output_rng_state->data.dptr = nullptr;
            output_rng_state->data.shape = {2};
            output_rng_state->data.dtype = DType::kInt64;
            Tensor *output_bias = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[2]);
            output_bias->data.dptr = nullptr;
            output_bias->data.shape = {bias_b, bias_h, max_seqlen, max_seqlen};
            output_bias->data.dtype = QKV_type;
        } else {
            Aux_CTX_Tensors->size = 2;
            Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
            output_S->data.dptr = nullptr;
            output_S->data.shape = {batch, num_attn_heads, max_seqlen, 1};
            output_S->data.dtype = DType::kFloat32;
            Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
            output_rng_state->data.dptr = nullptr;
            output_rng_state->data.shape = {2};
            output_rng_state->data.dtype = DType::kInt64;
        }
    } else if (Aux_CTX_Tensors->size == 2) {
        Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
        devPtrS = output_S->data.dptr;
        Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
        output_rng_state->data.dptr = rng_state->data.dptr;
    } else if (Aux_CTX_Tensors->size == 3) {
        Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
        devPtrS = output_S->data.dptr;
        Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
        output_rng_state->data.dptr = rng_state->data.dptr;
        Tensor *output_bias = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[2]);
        output_bias->data.dptr = devPtrBias;
    } else {
        NVTE_ERROR("Unexpected Aux_CTX_Tensors->size.");
    }

    void* devPtrDropoutSeed = rng_state->data.dptr;
    void* devPtrDropoutOffset = reinterpret_cast<void *>(
                    reinterpret_cast<uint64_t*>(rng_state->data.dptr) + 1);

    size_t workspace_size = 0;

    fused_attn_arbitrary_seqlen_fwd_impl(batch, num_attn_heads, num_attn_heads,
                                max_seqlen, max_seqlen, head_dim, bias_b, bias_h,
                                is_training, attn_scale, p_dropout, qkv_layout,
                                bias_type, mask_type,
                                devPtrQ, devPtrK, devPtrV, devPtrBias, devPtrS, devPtrO,
                                devPtrDropoutSeed, devPtrDropoutOffset,
                                devPtrCuSeqlens, devPtrCuSeqlens,
                                get_cudnn_fe_dtype(QKV_type),
                                workspace->data.dptr, &workspace_size,
                                stream, handle);

    if (workspace_size > 0) {
        if (workspace->data.dptr == nullptr) {
            workspace->data.shape = {workspace_size};
            workspace->data.dtype = DType::kByte;
            return;
        }
    } else if (workspace_size == 0) {
        workspace->data.shape = {1};
        workspace->data.dtype = DType::kByte;
        return;
    } else {
        NVTE_ERROR("Unexpected workspace_size.");
    }
}

void fused_attn_arbitrary_seqlen_bwd_qkvpacked(size_t batch, size_t num_attn_heads,
                                  size_t max_seqlen, size_t head_dim, float attn_scale,
                                  float p_dropout, NVTE_QKV_Layout qkv_layout,
                                  NVTE_Bias_Type bias_type, NVTE_Mask_Type mask_type,
                                  const Tensor *input_QKV, const Tensor *input_O,
                                  const Tensor *input_dO, const Tensor *input_Bias,
                                  Tensor *output_S,
                                  Tensor *output_dQKV, Tensor *output_dBias,
                                  const Tensor *cu_seqlens, const Tensor *rng_state,
                                  Tensor *workspace, hipStream_t stream, hipdnnHandle_t handle) {
    using namespace transformer_engine;

    const auto QKV_type = input_QKV->data.dtype;
    void *devPtrQKV = input_QKV->data.dptr;

    NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(qkv_layout);
    size_t stride = 0;
    if (layout_group == NVTE_QKV_Layout_Group::NVTE_3HD) {
        stride = typeToSize(QKV_type) * num_attn_heads * head_dim;
    } else if (layout_group == NVTE_QKV_Layout_Group::NVTE_H3D) {
        stride = typeToSize(QKV_type) * head_dim;
    }
    void *devPtrQ = devPtrQKV;
    void *devPtrK = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + stride);
    void *devPtrV = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + 2 * stride);

    void* devPtrO = input_O->data.dptr;
    void *devPtrdO = input_dO->data.dptr;
    void *devPtrBias = nullptr;
    void *devPtrdBias = nullptr;
    size_t bias_b = 0;
    size_t bias_h = 0;
    if ((bias_type != NVTE_Bias_Type::NVTE_NO_BIAS) && (bias_type != NVTE_Bias_Type::NVTE_ALIBI)) {
        devPtrBias = input_Bias->data.dptr;
        devPtrdBias = output_dBias->data.dptr;
        bias_b = output_dBias->data.shape[0];
        bias_h = output_dBias->data.shape[1];
    }

    void *devPtrdQKV = output_dQKV->data.dptr;
    void *devPtrdQ = devPtrdQKV;
    void *devPtrdK = static_cast<void *>(static_cast<int8_t *>(devPtrdQKV) + stride);
    void *devPtrdV = static_cast<void *>(static_cast<int8_t *>(devPtrdQKV) + 2 * stride);

    void *devPtrSoftmaxStats = nullptr;
    devPtrSoftmaxStats = output_S->data.dptr;

    void *devPtrCuSeqlens = cu_seqlens->data.dptr;

    void* devPtrDropoutSeed = rng_state->data.dptr;
    void* devPtrDropoutOffset = reinterpret_cast<void *>(
                    reinterpret_cast<uint64_t*>(rng_state->data.dptr) + 1);

    size_t workspace_size = 0;

    fused_attn_arbitrary_seqlen_bwd_impl(batch, num_attn_heads, num_attn_heads,
                                max_seqlen, max_seqlen, head_dim, bias_b, bias_h,
                                attn_scale, p_dropout, qkv_layout,
                                bias_type, mask_type,
                                devPtrQ, devPtrK, devPtrV, devPtrO, devPtrSoftmaxStats, devPtrBias,
                                devPtrdQ, devPtrdK, devPtrdV, devPtrdO, devPtrdBias,
                                devPtrDropoutSeed, devPtrDropoutOffset,
                                devPtrCuSeqlens, devPtrCuSeqlens,
                                get_cudnn_fe_dtype(QKV_type), workspace->data.dptr,
                                &workspace_size, stream, handle);

    if (workspace_size > 0) {
        if (workspace->data.dptr == nullptr) {
            workspace->data.shape = {workspace_size};
            workspace->data.dtype = DType::kByte;
            return;
        }
    } else if (workspace_size == 0) {
        workspace->data.shape = {1};
        workspace->data.dtype = DType::kByte;
        return;
    } else {
        NVTE_ERROR("Unexpected workspace_size.");
    }
}
void fused_attn_arbitrary_seqlen_fwd_kvpacked(
    size_t batch, size_t num_attn_heads, size_t num_gqa_groups,
    size_t max_seqlen_q, size_t max_seqlen_kv, size_t head_dim, bool is_training,
    float attn_scale, float p_dropout, NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type,
    NVTE_Mask_Type mask_type, const Tensor *input_Q, const Tensor *input_KV,
    const Tensor *input_Bias, Tensor *output_O,
    NVTETensorPack *Aux_CTX_Tensors, const Tensor *cu_seqlens_q, const Tensor *cu_seqlens_kv,
    const Tensor *rng_state, Tensor *workspace, hipStream_t stream, hipdnnHandle_t handle) {
    using namespace transformer_engine;

    const auto QKV_type = input_Q->data.dtype;
    void *devPtrQ = input_Q->data.dptr;
    void *devPtrKV = input_KV->data.dptr;
    NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(qkv_layout);
    size_t stride = 0;
    if (layout_group == NVTE_QKV_Layout_Group::NVTE_HD_2HD) {
        stride = typeToSize(QKV_type) * num_gqa_groups * head_dim;
    } else if (layout_group == NVTE_QKV_Layout_Group::NVTE_HD_H2D) {
        stride = typeToSize(QKV_type) * head_dim;
    }
    void *devPtrK = devPtrKV;
    void *devPtrV = static_cast<void *>(static_cast<int8_t *>(devPtrKV) + stride);

    void *devPtrBias = nullptr;
    size_t bias_b = 0;
    size_t bias_h = 0;
    if ((bias_type != NVTE_Bias_Type::NVTE_NO_BIAS) && (bias_type != NVTE_Bias_Type::NVTE_ALIBI)) {
        devPtrBias = input_Bias->data.dptr;
        bias_b = input_Bias->data.shape[0];
        bias_h = input_Bias->data.shape[1];
    }
    void *devPtrO = output_O->data.dptr;
    void *devPtrS = nullptr;

    void *devPtrCuSeqlensQ = cu_seqlens_q->data.dptr;
    void *devPtrCuSeqlensKV = cu_seqlens_kv->data.dptr;

    if (Aux_CTX_Tensors->size == 0) {
        if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI)) {
            Aux_CTX_Tensors->size = 3;
            Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
            output_S->data.dptr = nullptr;
            output_S->data.shape = {batch, num_attn_heads, max_seqlen_q, 1};
            output_S->data.dtype = DType::kFloat32;
            Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
            output_rng_state->data.dptr = nullptr;
            output_rng_state->data.shape = {2};
            output_rng_state->data.dtype = DType::kInt64;
            Tensor *output_bias = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[2]);
            output_bias->data.dptr = nullptr;
            output_bias->data.shape = {bias_b, bias_h, max_seqlen_q, max_seqlen_kv};
            output_bias->data.dtype = QKV_type;
        } else {
            Aux_CTX_Tensors->size = 2;
            Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
            output_S->data.dptr = nullptr;
            output_S->data.shape = {batch, num_attn_heads, max_seqlen_q, 1};
            output_S->data.dtype = DType::kFloat32;
            Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
            output_rng_state->data.dptr = nullptr;
            output_rng_state->data.shape = {2};
            output_rng_state->data.dtype = DType::kInt64;
        }
    } else if (Aux_CTX_Tensors->size == 2) {
        Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
        devPtrS = output_S->data.dptr;
        Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
        output_rng_state->data.dptr = rng_state->data.dptr;
    } else if (Aux_CTX_Tensors->size == 3) {
        Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
        devPtrS = output_S->data.dptr;
        Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
        output_rng_state->data.dptr = rng_state->data.dptr;
        Tensor *output_bias = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[2]);
        output_bias->data.dptr = devPtrBias;
    } else {
        NVTE_ERROR("Unexpected Aux_CTX_Tensors->size.");
    }

    void* devPtrDropoutSeed = rng_state->data.dptr;
    void* devPtrDropoutOffset = reinterpret_cast<void *>(
                    reinterpret_cast<uint64_t*>(rng_state->data.dptr) + 1);

    size_t workspace_size = 0;

    fused_attn_arbitrary_seqlen_fwd_impl(batch, num_attn_heads, num_gqa_groups,
                                max_seqlen_q, max_seqlen_kv, head_dim, bias_b, bias_h,
                                is_training, attn_scale, p_dropout, qkv_layout,
                                bias_type, mask_type,
                                devPtrQ, devPtrK, devPtrV, devPtrBias, devPtrS, devPtrO,
                                devPtrDropoutSeed, devPtrDropoutOffset,
                                devPtrCuSeqlensQ, devPtrCuSeqlensKV,
                                get_cudnn_fe_dtype(QKV_type),
                                workspace->data.dptr, &workspace_size,
                                stream, handle);

    if (workspace_size > 0) {
        if (workspace->data.dptr == nullptr) {
            workspace->data.shape = {workspace_size};
            workspace->data.dtype = DType::kByte;
            return;
        }
    } else if (workspace_size == 0) {
        workspace->data.shape = {1};
        workspace->data.dtype = DType::kByte;
        return;
    } else {
        NVTE_ERROR("Unexpected workspace_size.");
    }
}

void fused_attn_arbitrary_seqlen_bwd_kvpacked(
                                  size_t batch, size_t num_attn_heads, size_t num_gqa_groups,
                                  size_t max_seqlen_q, size_t max_seqlen_kv, size_t head_dim,
                                  float attn_scale, float p_dropout, NVTE_QKV_Layout qkv_layout,
                                  NVTE_Bias_Type bias_type, NVTE_Mask_Type mask_type,
                                  const Tensor *input_Q, const Tensor *input_KV,
                                  const Tensor *input_O, const Tensor *input_dO,
                                  const Tensor *input_Bias, Tensor *output_S,
                                  Tensor *output_dQ, Tensor *output_dKV,
                                  Tensor *output_dBias, const Tensor *cu_seqlens_q,
                                  const Tensor *cu_seqlens_kv,
                                  const Tensor *rng_state, Tensor *workspace,
                                  hipStream_t stream, hipdnnHandle_t handle) {
    using namespace transformer_engine;

    const auto QKV_type = input_Q->data.dtype;
    void *devPtrQ = input_Q->data.dptr;
    void *devPtrKV = input_KV->data.dptr;
    NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(qkv_layout);
    size_t stride = 0;
    if (layout_group == NVTE_QKV_Layout_Group::NVTE_HD_2HD) {
        stride = typeToSize(QKV_type) * num_gqa_groups * head_dim;
    } else if (layout_group == NVTE_QKV_Layout_Group::NVTE_HD_H2D) {
        stride = typeToSize(QKV_type) * head_dim;
    }
    void *devPtrK = devPtrKV;
    void *devPtrV = static_cast<void *>(static_cast<int8_t *>(devPtrKV) + stride);

    void* devPtrO = input_O->data.dptr;
    void *devPtrdO = input_dO->data.dptr;
    void *devPtrBias = nullptr;
    void *devPtrdBias = nullptr;
    size_t bias_b = 0;
    size_t bias_h = 0;
    if ((bias_type != NVTE_Bias_Type::NVTE_NO_BIAS) && (bias_type != NVTE_Bias_Type::NVTE_ALIBI)) {
        devPtrBias = input_Bias->data.dptr;
        devPtrdBias = output_dBias->data.dptr;
        bias_b = output_dBias->data.shape[0];
        bias_h = output_dBias->data.shape[1];
    }

    void *devPtrdQ = output_dQ->data.dptr;
    void *devPtrdKV = output_dKV->data.dptr;
    void *devPtrdK = devPtrdKV;
    void *devPtrdV = static_cast<void *>(static_cast<int8_t *>(devPtrdKV) + stride);

    void *devPtrSoftmaxStats = nullptr;
    devPtrSoftmaxStats = output_S->data.dptr;

    void *devPtrCuSeqlensQ = cu_seqlens_q->data.dptr;
    void *devPtrCuSeqlensKV = cu_seqlens_kv->data.dptr;

    void* devPtrDropoutSeed = rng_state->data.dptr;
    void* devPtrDropoutOffset = reinterpret_cast<void *>(
                    reinterpret_cast<uint64_t*>(rng_state->data.dptr) + 1);

    size_t workspace_size = 0;

    fused_attn_arbitrary_seqlen_bwd_impl(batch, num_attn_heads, num_gqa_groups,
                                max_seqlen_q, max_seqlen_kv, head_dim, bias_b, bias_h,
                                attn_scale, p_dropout, qkv_layout,
                                bias_type, mask_type,
                                devPtrQ, devPtrK, devPtrV, devPtrO, devPtrSoftmaxStats, devPtrBias,
                                devPtrdQ, devPtrdK, devPtrdV, devPtrdO, devPtrdBias,
                                devPtrDropoutSeed, devPtrDropoutOffset,
                                devPtrCuSeqlensQ, devPtrCuSeqlensKV,
                                get_cudnn_fe_dtype(QKV_type), workspace->data.dptr,
                                &workspace_size, stream, handle);

    if (workspace_size > 0) {
        if (workspace->data.dptr == nullptr) {
            workspace->data.shape = {workspace_size};
            workspace->data.dtype = DType::kByte;
            return;
        }
    } else if (workspace_size == 0) {
        workspace->data.shape = {1};
        workspace->data.dtype = DType::kByte;
        return;
    } else {
        NVTE_ERROR("Unexpected workspace_size.");
    }
}

void fused_attn_arbitrary_seqlen_fwd(
    size_t batch, size_t num_attn_heads, size_t num_gqa_groups,
    size_t max_seqlen_q, size_t max_seqlen_kv, size_t head_dim, bool is_training,
    float attn_scale, float p_dropout, NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type,
    NVTE_Mask_Type mask_type, const Tensor *input_Q, const Tensor *input_K,
    const Tensor *input_V, const Tensor *input_Bias, Tensor *output_O,
    NVTETensorPack *Aux_CTX_Tensors, const Tensor *cu_seqlens_q, const Tensor *cu_seqlens_kv,
    const Tensor *rng_state,
    Tensor *workspace, hipStream_t stream, hipdnnHandle_t handle) {
    using namespace transformer_engine;

    const auto QKV_type = input_Q->data.dtype;
    void *devPtrQ = input_Q->data.dptr;
    void *devPtrK = input_K->data.dptr;
    void *devPtrV = input_V->data.dptr;
    void *devPtrO = output_O->data.dptr;
    void *devPtrS = nullptr;
    void *devPtrBias = nullptr;
    size_t bias_b = 0;
    size_t bias_h = 0;
    if ((bias_type != NVTE_Bias_Type::NVTE_NO_BIAS) && (bias_type != NVTE_Bias_Type::NVTE_ALIBI)) {
        devPtrBias = input_Bias->data.dptr;
        bias_b = input_Bias->data.shape[0];
        bias_h = input_Bias->data.shape[1];
    }

    void *devPtrCuSeqlensQ = cu_seqlens_q->data.dptr;
    void *devPtrCuSeqlensKV = cu_seqlens_kv->data.dptr;

    if (Aux_CTX_Tensors->size == 0) {
        if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI)) {
            Aux_CTX_Tensors->size = 3;
            Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
            output_S->data.dptr = nullptr;
            output_S->data.shape = {batch, num_attn_heads, max_seqlen_q, 1};
            output_S->data.dtype = DType::kFloat32;
            Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
            output_rng_state->data.dptr = nullptr;
            output_rng_state->data.shape = {2};
            output_rng_state->data.dtype = DType::kInt64;
            Tensor *output_bias = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[2]);
            output_bias->data.dptr = nullptr;
            output_bias->data.shape = {bias_b, bias_h, max_seqlen_q, max_seqlen_kv};
            output_bias->data.dtype = QKV_type;
        } else {
            Aux_CTX_Tensors->size = 2;
            Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
            output_S->data.dptr = nullptr;
            output_S->data.shape = {batch, num_attn_heads, max_seqlen_q, 1};
            output_S->data.dtype = DType::kFloat32;
            Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
            output_rng_state->data.dptr = nullptr;
            output_rng_state->data.shape = {2};
            output_rng_state->data.dtype = DType::kInt64;
        }
    } else if (Aux_CTX_Tensors->size == 2) {
        Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
        devPtrS = output_S->data.dptr;
        Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
        output_rng_state->data.dptr = rng_state->data.dptr;
    } else if (Aux_CTX_Tensors->size == 3) {
        Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
        devPtrS = output_S->data.dptr;
        Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
        output_rng_state->data.dptr = rng_state->data.dptr;
        Tensor *output_bias = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[2]);
        output_bias->data.dptr = devPtrBias;
    } else {
        NVTE_ERROR("Unexpected Aux_CTX_Tensors->size.");
    }

    void* devPtrDropoutSeed = rng_state->data.dptr;
    void* devPtrDropoutOffset = reinterpret_cast<void *>(
                    reinterpret_cast<uint64_t*>(rng_state->data.dptr) + 1);

    size_t workspace_size = 0;

    fused_attn_arbitrary_seqlen_fwd_impl(batch, num_attn_heads, num_gqa_groups,
                                max_seqlen_q, max_seqlen_kv, head_dim, bias_b, bias_h,
                                is_training, attn_scale, p_dropout, qkv_layout,
                                bias_type, mask_type,
                                devPtrQ, devPtrK, devPtrV, devPtrBias, devPtrS, devPtrO,
                                devPtrDropoutSeed, devPtrDropoutOffset,
                                devPtrCuSeqlensQ, devPtrCuSeqlensKV,
                                get_cudnn_fe_dtype(QKV_type),
                                workspace->data.dptr, &workspace_size,
                                stream, handle);

    if (workspace_size > 0) {
        if (workspace->data.dptr == nullptr) {
            workspace->data.shape = {workspace_size};
            workspace->data.dtype = DType::kByte;
            return;
        }
    } else if (workspace_size == 0) {
        workspace->data.shape = {1};
        workspace->data.dtype = DType::kByte;
        return;
    } else {
        NVTE_ERROR("Unexpected workspace_size.");
    }
}

void fused_attn_arbitrary_seqlen_bwd(size_t batch, size_t num_attn_heads, size_t num_gqa_groups,
                                  size_t max_seqlen_q, size_t max_seqlen_kv, size_t head_dim,
                                  float attn_scale, float p_dropout, NVTE_QKV_Layout qkv_layout,
                                  NVTE_Bias_Type bias_type, NVTE_Mask_Type mask_type,
                                  const Tensor *input_Q, const Tensor *input_K,
                                  const Tensor *input_V, const Tensor *input_O,
                                  const Tensor *input_dO, const Tensor *input_Bias,
                                  Tensor *output_S,
                                  Tensor *output_dQ, Tensor *output_dK, Tensor *output_dV,
                                  Tensor *output_dBias, const Tensor *cu_seqlens_q,
                                  const Tensor *cu_seqlens_kv,
                                  const Tensor *rng_state, Tensor *workspace,
                                  hipStream_t stream, hipdnnHandle_t handle) {
    using namespace transformer_engine;

    const auto QKV_type = input_Q->data.dtype;
    void *devPtrQ = input_Q->data.dptr;
    void *devPtrK = input_K->data.dptr;
    void *devPtrV = input_V->data.dptr;
    void* devPtrO = input_O->data.dptr;
    void *devPtrdO = input_dO->data.dptr;
    void *devPtrBias = nullptr;
    void *devPtrdBias = nullptr;
    size_t bias_b = 0;
    size_t bias_h = 0;
    if ((bias_type != NVTE_Bias_Type::NVTE_NO_BIAS) && (bias_type != NVTE_Bias_Type::NVTE_ALIBI)) {
        devPtrBias = input_Bias->data.dptr;
        devPtrdBias = output_dBias->data.dptr;
        bias_b = output_dBias->data.shape[0];
        bias_h = output_dBias->data.shape[1];
    }

    void *devPtrdQ = output_dQ->data.dptr;
    void *devPtrdK = output_dK->data.dptr;
    void *devPtrdV = output_dV->data.dptr;
    void *devPtrSoftmaxStats = nullptr;
    devPtrSoftmaxStats = output_S->data.dptr;

    void *devPtrCuSeqlensQ = cu_seqlens_q->data.dptr;
    void *devPtrCuSeqlensKV = cu_seqlens_kv->data.dptr;

    void* devPtrDropoutSeed = rng_state->data.dptr;
    void* devPtrDropoutOffset = reinterpret_cast<void *>(
                    reinterpret_cast<uint64_t*>(rng_state->data.dptr) + 1);

    size_t workspace_size = 0;

    fused_attn_arbitrary_seqlen_bwd_impl(batch, num_attn_heads, num_gqa_groups,
                                max_seqlen_q, max_seqlen_kv, head_dim, bias_b, bias_h,
                                attn_scale, p_dropout, qkv_layout, bias_type, mask_type,
                                devPtrQ, devPtrK, devPtrV, devPtrO, devPtrSoftmaxStats, devPtrBias,
                                devPtrdQ, devPtrdK, devPtrdV, devPtrdO, devPtrdBias,
                                devPtrDropoutSeed, devPtrDropoutOffset,
                                devPtrCuSeqlensQ, devPtrCuSeqlensKV,
                                get_cudnn_fe_dtype(QKV_type), workspace->data.dptr,
                                &workspace_size, stream, handle);

    if (workspace_size > 0) {
        if (workspace->data.dptr == nullptr) {
            workspace->data.shape = {workspace_size};
            workspace->data.dtype = DType::kByte;
            return;
        }
    } else if (workspace_size == 0) {
        workspace->data.shape = {1};
        workspace->data.dtype = DType::kByte;
        return;
    } else {
        NVTE_ERROR("Unexpected workspace_size.");
    }
}
}  // namespace transformer_engine
#endif  // CUDNN_VERSION >= 8900
