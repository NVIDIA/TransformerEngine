/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <cudnn_frontend.h>
#include <cudnn_frontend_utils.h>

#include <map>
#include <vector>

#include "../common.h"
#include "../util/hip/hip_runtime.h"
#include "../util/system.h"
#include "fused_attn_f16_arbitrary_seqlen.h"
#include "utils.h"

#if (CUDNN_VERSION >= 8900)
#define Q_ID 1
#define K_ID 2
#define V_ID 3
#define O_ID 4
#define S_ID 5
#define B_ID 6
#define D_CONST_ID 7
#define S_CONST_ID 8
#define Q_SEQLEN_ID 9
#define K_SEQLEN_ID 10
#define dQ_ID 11
#define dK_ID 12
#define dV_ID 13
#define dO_ID 14
#define MASK_VAL_ID 15
#define dS_ID 16
#define D_SEED_ID 17
#define D_OFFSET_ID 18
#define S_STATS_ID 19
#define S_SUM_ID 20
#define SCALE_PROB 21
#define K_TRANSPOSE_ID 22
#define dQ_ACCUM_ID 23

#define VIRTUAL_ID 30

namespace transformer_engine {
namespace fused_attn {
void fused_attn_arbitrary_seqlen_fwd_impl(
    int64_t b, int64_t h, int64_t hg, int64_t s_q, int64_t s_kv, int64_t d, int64_t bias_b,
    int64_t bias_h, bool is_training, float scaling_factor, float dropout_probability,
    NVTE_QKV_Layout layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type mask_type,
    int64_t window_size_left, int64_t window_size_right, void *devPtrQ, void *devPtrK,
    void *devPtrV, void *devPtrBias, void *devPtrSoftmaxStats, void *devPtrO,
    void *devPtrDropoutSeed, void *devPtrDropoutOffset, void *devPtrCuSeqlensQ,
    void *devPtrCuSeqlensKV, void *devPtrSeqOffsetsQ, void *devPtrSeqOffsetsKV,
    cudnn_frontend::DataType_t tensorType, void *workspace, size_t *workspace_size,
    hipStream_t stream, hipdnnHandle_t handle) {
  using namespace transformer_engine;
  bool is_bias = (bias_type == NVTE_Bias_Type::NVTE_POST_SCALE_BIAS);
  bool is_alibi = (bias_type == NVTE_Bias_Type::NVTE_ALIBI);
  bool is_causal = ((mask_type == NVTE_Mask_Type::NVTE_CAUSAL_MASK) ||
                    (mask_type == NVTE_Mask_Type::NVTE_PADDING_CAUSAL_MASK));
  bool is_bottom_right = ((mask_type == NVTE_Mask_Type::NVTE_CAUSAL_BOTTOM_RIGHT_MASK) ||
                          (mask_type == NVTE_Mask_Type::NVTE_PADDING_CAUSAL_BOTTOM_RIGHT_MASK));
  if (is_bottom_right && s_q == s_kv) {
    is_causal = true;
    is_bottom_right = false;
  }
  bool is_padding = ((mask_type == NVTE_Mask_Type::NVTE_PADDING_MASK) ||
                     (mask_type == NVTE_Mask_Type::NVTE_PADDING_CAUSAL_MASK));
  bool is_dropout = (is_training && dropout_probability != 0.0f);
  bool is_ragged = (nvte_get_qkv_format(layout) == NVTE_QKV_Format::NVTE_THD);
  if (is_ragged) {
    NVTE_CHECK(is_padding, "Ragged QKV input requires padding or padding_causal mask!");
  }
  if (window_size_left == -1) {
    window_size_left = s_q;
  }
  auto cudnn_runtime_version = hipdnnGetVersion();

  try {
    FADescriptor_v1 descriptor{b,
                               h,
                               hg,
                               s_q,
                               s_kv,
                               d,
                               bias_b,
                               bias_h,
                               scaling_factor,
                               is_training,
                               dropout_probability,
                               layout,
                               bias_type,
                               mask_type,
                               window_size_left,
                               window_size_right,
                               true,
                               tensorType,
                               tensorType};

    namespace fe = cudnn_frontend;
    using graph_and_tensors =
        std::tuple<std::shared_ptr<fe::graph::Graph>,
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // Q
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // K
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // V
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // attn_scale
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // O
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // Stats
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // bias
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // seq_q
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // seq_kv
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // offset_q
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // offset_k
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // offset_v
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // offset_o
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // dropout_seed
                   std::shared_ptr<fe::graph::Tensor_attributes>>;  // dropout_offset

    using CacheType = std::map<FADescriptor_v1, graph_and_tensors>;
    static thread_local CacheType sdpa_f16_fprop_cache;

    // Get plan from cache if cache is available, otherwise create one
    auto get_graph = [&](CacheType &cache, const FADescriptor_v1 &descriptor) -> graph_and_tensors {
      // if hit, return
      auto it = cache.find(descriptor);
      if (it != cache.end()) {
        auto graph = it->second;
        return graph;
      }

      // otherwise, build the op_graph and the plan. Then update cache
      auto mha_graph = std::make_shared<fe::graph::Graph>();
      mha_graph->set_io_data_type(tensorType)
          .set_intermediate_data_type(fe::DataType_t::FLOAT)
          .set_compute_data_type(fe::DataType_t::FLOAT);

      std::shared_ptr<fe::graph::Tensor_attributes> Q, K, V, attn_scale;
      std::shared_ptr<fe::graph::Tensor_attributes> bias, seq_q, seq_kv;
      std::shared_ptr<fe::graph::Tensor_attributes> offset_q, offset_k, offset_v, offset_o;
      std::shared_ptr<fe::graph::Tensor_attributes> dropout_seed, dropout_offset;

      offset_q = mha_graph->tensor(fe::graph::Tensor_attributes()
                                       .set_name("offset_q")
                                       .set_dim({b + 1, 1, 1, 1})
                                       .set_stride({1, 1, 1, 1})
                                       .set_data_type(fe::DataType_t::INT32));
      offset_k = mha_graph->tensor(fe::graph::Tensor_attributes()
                                       .set_name("offset_k")
                                       .set_dim({b + 1, 1, 1, 1})
                                       .set_stride({1, 1, 1, 1})
                                       .set_data_type(fe::DataType_t::INT32));
      offset_v = mha_graph->tensor(fe::graph::Tensor_attributes()
                                       .set_name("offset_v")
                                       .set_dim({b + 1, 1, 1, 1})
                                       .set_stride({1, 1, 1, 1})
                                       .set_data_type(fe::DataType_t::INT32));
      offset_o = mha_graph->tensor(fe::graph::Tensor_attributes()
                                       .set_name("offset_o")
                                       .set_dim({b + 1, 1, 1, 1})
                                       .set_stride({1, 1, 1, 1})
                                       .set_data_type(fe::DataType_t::INT32));

      std::vector<int64_t> q_stride(4);
      std::vector<int64_t> k_stride(4);
      std::vector<int64_t> v_stride(4);
      generateMatrixStrides(b, h, s_q, s_kv, d, q_stride.data(), layout,
                            NVTE_QKV_Matrix::NVTE_Q_Matrix);
      generateMatrixStrides(b, hg, s_q, s_kv, d, k_stride.data(), layout,
                            NVTE_QKV_Matrix::NVTE_K_Matrix);
      generateMatrixStrides(b, hg, s_q, s_kv, d, v_stride.data(), layout,
                            NVTE_QKV_Matrix::NVTE_V_Matrix);

      if (is_ragged) {
        Q = mha_graph->tensor(fe::graph::Tensor_attributes()
                                  .set_name("Q")
                                  .set_dim({b, h, s_q, d})
                                  .set_stride(q_stride)
                                  .set_ragged_offset(offset_q));
        K = mha_graph->tensor(fe::graph::Tensor_attributes()
                                  .set_name("K")
                                  .set_dim({b, hg, s_kv, d})
                                  .set_stride(k_stride)
                                  .set_ragged_offset(offset_k));
        V = mha_graph->tensor(fe::graph::Tensor_attributes()
                                  .set_name("V")
                                  .set_dim({b, hg, s_kv, d})
                                  .set_stride(v_stride)
                                  .set_ragged_offset(offset_v));
      } else {
        Q = mha_graph->tensor(fe::graph::Tensor_attributes()
                                  .set_name("Q")
                                  .set_dim({b, h, s_q, d})
                                  .set_stride(q_stride));
        K = mha_graph->tensor(fe::graph::Tensor_attributes()
                                  .set_name("K")
                                  .set_dim({b, hg, s_kv, d})
                                  .set_stride(k_stride));
        V = mha_graph->tensor(fe::graph::Tensor_attributes()
                                  .set_name("V")
                                  .set_dim({b, hg, s_kv, d})
                                  .set_stride(v_stride));
      }

      attn_scale = mha_graph->tensor(fe::graph::Tensor_attributes()
                                         .set_name("attn_scale")
                                         .set_dim({1, 1, 1, 1})
                                         .set_stride({1, 1, 1, 1})
                                         .set_is_pass_by_value(true)
                                         .set_data_type(fe::DataType_t::FLOAT));

      fe::graph::SDPA_attributes sdpa_options;
      sdpa_options = fe::graph::SDPA_attributes()
                         .set_name("flash_attention")
                         .set_is_inference(false)
                         .set_causal_mask(is_causal)
                         .set_causal_mask_bottom_right(is_bottom_right)
                         .set_attn_scale(attn_scale);

      if (cudnn_runtime_version >= 90200 && window_size_left != s_q) {
        sdpa_options.set_sliding_window_length(window_size_left);
      }

      sdpa_options.set_alibi_mask(is_alibi);

      if (is_bias) {
        bias = mha_graph->tensor(fe::graph::Tensor_attributes()
                                     .set_name("bias")
                                     .set_dim({bias_b, bias_h, s_q, s_kv})
                                     .set_stride({bias_h * s_q * s_kv, s_q * s_kv, s_kv, 1}));
        sdpa_options.set_bias(bias);
      }

      if (is_padding) {
        seq_q = mha_graph->tensor(fe::graph::Tensor_attributes()
                                      .set_name("seq_q")
                                      .set_dim({b, 1, 1, 1})
                                      .set_stride({1, 1, 1, 1})
                                      .set_data_type(fe::DataType_t::INT32));
        seq_kv = mha_graph->tensor(fe::graph::Tensor_attributes()
                                       .set_name("seq_kv")
                                       .set_dim({b, 1, 1, 1})
                                       .set_stride({1, 1, 1, 1})
                                       .set_data_type(fe::DataType_t::INT32));
        sdpa_options.set_padding_mask(is_padding).set_seq_len_q(seq_q).set_seq_len_kv(seq_kv);
      }

      if (is_dropout) {
        dropout_seed = mha_graph->tensor(fe::graph::Tensor_attributes()
                                             .set_name("Seed")
                                             .set_dim({1, 1, 1, 1})
                                             .set_stride({1, 1, 1, 1})
                                             .set_data_type(fe::DataType_t::INT64));
        dropout_offset = mha_graph->tensor(fe::graph::Tensor_attributes()
                                               .set_name("Offset")
                                               .set_dim({1, 1, 1, 1})
                                               .set_stride({1, 1, 1, 1})
                                               .set_data_type(fe::DataType_t::INT64));
        sdpa_options.set_dropout(dropout_probability, dropout_seed, dropout_offset);
      }

      auto [O, Stats] = mha_graph->sdpa(Q, K, V, sdpa_options);

      std::vector<int64_t> o_stride(4);
      generateMatrixStrides(b, h, s_q, s_kv, d, o_stride.data(), layout,
                            NVTE_QKV_Matrix::NVTE_O_Matrix);
      if (is_ragged) {
        O->set_output(true)
            .set_dim({b, h, s_q, d})
            .set_stride(o_stride)
            .set_ragged_offset(offset_o);
      } else {
        O->set_output(true).set_dim({b, h, s_q, d}).set_stride(o_stride);
      }

      Stats->set_output(true)
          .set_data_type(fe::DataType_t::FLOAT)
          .set_dim({b, h, s_q, 1})
          .set_stride({h * s_q, s_q, 1, 1});

      std::tuple<std::shared_ptr<fe::graph::Tensor_attributes>,  // Q
                 std::shared_ptr<fe::graph::Tensor_attributes>,  // K
                 std::shared_ptr<fe::graph::Tensor_attributes>,  // V
                 std::shared_ptr<fe::graph::Tensor_attributes>,  // attn_scale
                 std::shared_ptr<fe::graph::Tensor_attributes>>  // O
          key_tensors_tuple = std::make_tuple(Q, K, V, attn_scale, O);
      auto Stats_tuple = std::make_tuple(Stats);
      auto bias_tuple = is_bias ? std::make_tuple(bias) : std::make_tuple(nullptr);
      auto padding_tuple =
          is_padding ? std::make_tuple(seq_q, seq_kv) : std::make_tuple(nullptr, nullptr);
      auto offset_tuple = is_ragged ? std::make_tuple(offset_q, offset_k, offset_v, offset_o)
                                    : std::make_tuple(nullptr, nullptr, nullptr, nullptr);
      auto dropout_tuple = is_dropout ? std::make_tuple(dropout_seed, dropout_offset)
                                      : std::make_tuple(nullptr, nullptr);

      NVTE_CHECK_CUDNN_FE(mha_graph->validate());
      NVTE_CHECK_CUDNN_FE(mha_graph->build_operation_graph(handle));
      NVTE_CHECK_CUDNN_FE(mha_graph->create_execution_plans({fe::HeurMode_t::A}));
      NVTE_CHECK_CUDNN_FE(mha_graph->check_support(handle));
      NVTE_CHECK_CUDNN_FE(mha_graph->build_plans(handle));

      auto return_tuple = std::tuple_cat(std::make_tuple(mha_graph), key_tensors_tuple, Stats_tuple,
                                         bias_tuple, padding_tuple, offset_tuple, dropout_tuple);
      cache.insert({descriptor, return_tuple});

      return return_tuple;
    };

    auto [mha_graph, Q, K, V, attn_scale, O, Stats, bias, seq_q, seq_kv, offset_q, offset_k,
          offset_v, offset_o, dropout_seed, dropout_offset] =
        get_graph(sdpa_f16_fprop_cache, descriptor);

    auto plan_workspace_size = mha_graph->get_workspace_size();
    // Exit to request upper level API to allocate memory if needed
    size_t actual_seqlen_workspace_size = 2 * b * sizeof(int32_t);
    size_t seqlen_offsets_workspace_size = 4 * (b + 1) * sizeof(int32_t);
    if (workspace == nullptr) {
      *workspace_size =
          plan_workspace_size + actual_seqlen_workspace_size + seqlen_offsets_workspace_size;
      return;
    }

    // cuDNN stream check needs to be moved here to support dummy kernel calls with
    // null streams for sizing the cuDNN workspace.
    NVTE_CHECK_CUDNN(hipdnnSetStream(handle, stream));

    // Build variant pack
    std::unordered_map<std::shared_ptr<fe::graph::Tensor_attributes>, void *> variant_pack = {
        {Q, devPtrQ}, {K, devPtrK},
        {V, devPtrV}, {attn_scale, &scaling_factor},
        {O, devPtrO}, {Stats, devPtrSoftmaxStats}};

    if (is_bias) {
      variant_pack[bias] = devPtrBias;
    }

    if (is_padding) {
      constexpr size_t nthreads_per_block = 128;
      const size_t grid = (b + nthreads_per_block - 1) / nthreads_per_block;
      void *devActualSeqlenQ = static_cast<int8_t *>(workspace) + plan_workspace_size;
      void *devActualSeqlenKV = static_cast<int8_t *>(devActualSeqlenQ) + b * sizeof(int32_t);
      cu_seqlens_to_actual_seqlens<<<grid, nthreads_per_block, 0, stream>>>(
          b, static_cast<const int32_t *>(devPtrCuSeqlensQ),
          static_cast<const int32_t *>(devPtrCuSeqlensKV), static_cast<int32_t *>(devActualSeqlenQ),
          static_cast<int32_t *>(devActualSeqlenKV));
      variant_pack[seq_q] = devActualSeqlenQ;
      variant_pack[seq_kv] = devActualSeqlenKV;
    }

    if (is_ragged) {
      constexpr size_t nthreads_per_block = 128;
      const size_t grid = (b + nthreads_per_block) / nthreads_per_block;
      void *devOffsetsQ =
          static_cast<int8_t *>(workspace) + plan_workspace_size + actual_seqlen_workspace_size;
      void *devOffsetsK = static_cast<int8_t *>(devOffsetsQ) + (b + 1) * sizeof(int32_t);
      void *devOffsetsV = static_cast<int8_t *>(devOffsetsK) + (b + 1) * sizeof(int32_t);
      void *devOffsetsO = static_cast<int8_t *>(devOffsetsV) + (b + 1) * sizeof(int32_t);
      NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(layout);
      cu_seqlens_padded_to_offsets<<<grid, nthreads_per_block, 0, stream>>>(
          layout_group, b, h, hg, d, static_cast<int32_t *>(devPtrSeqOffsetsQ),
          static_cast<int32_t *>(devPtrSeqOffsetsKV), static_cast<int32_t *>(devOffsetsQ),
          static_cast<int32_t *>(devOffsetsK), static_cast<int32_t *>(devOffsetsV),
          static_cast<int32_t *>(devOffsetsO));
      variant_pack[offset_q] = devOffsetsQ;
      variant_pack[offset_k] = devOffsetsK;
      variant_pack[offset_v] = devOffsetsV;
      variant_pack[offset_o] = devOffsetsO;
    }

    if (is_dropout) {
      variant_pack[dropout_seed] = devPtrDropoutSeed;
      variant_pack[dropout_offset] = devPtrDropoutOffset;
    }
    NVTE_CHECK_CUDNN_FE(mha_graph->execute(handle, variant_pack, workspace));
  } catch (cudnn_frontend::cudnnException &e) {
    NVTE_ERROR(e.what());
  }
}

void fused_attn_arbitrary_seqlen_bwd_impl(
    int64_t b, int64_t h, int64_t hg, int64_t s_q, int64_t s_kv, int64_t d, int64_t bias_b,
    int64_t bias_h, float scaling_factor, float dropout_probability, NVTE_QKV_Layout layout,
    NVTE_Bias_Type bias_type, NVTE_Mask_Type mask_type, int64_t window_size_left,
    int64_t window_size_right, bool deterministic, void *devPtrQ, void *devPtrKTranspose,
    void *devPtrVTranspose, void *devPtrO, void *devPtrSoftmaxStats, void *devPtrBias,
    void *devPtrdQ, void *devPtrdK, void *devPtrdV, void *devPtrdO, void *devPtrdBias,
    void *devPtrDropoutSeed, void *devPtrDropoutOffset, void *devPtrCuSeqlensQ,
    void *devPtrCuSeqlensKV, void *devPtrSeqOffsetsQ, void *devPtrSeqOffsetsKV,
    cudnn_frontend::DataType_t tensorType, void *workspace, size_t *workspace_size,
    hipStream_t stream, hipdnnHandle_t handle) {
  using namespace transformer_engine;
  bool is_bias = (bias_type == NVTE_Bias_Type::NVTE_POST_SCALE_BIAS);
  bool is_alibi = (bias_type == NVTE_Bias_Type::NVTE_ALIBI);
  bool is_causal = ((mask_type == NVTE_Mask_Type::NVTE_CAUSAL_MASK) ||
                    (mask_type == NVTE_Mask_Type::NVTE_PADDING_CAUSAL_MASK));
  bool is_bottom_right = ((mask_type == NVTE_Mask_Type::NVTE_CAUSAL_BOTTOM_RIGHT_MASK) ||
                          (mask_type == NVTE_Mask_Type::NVTE_PADDING_CAUSAL_BOTTOM_RIGHT_MASK));
  if (is_bottom_right && s_q == s_kv) {
    is_causal = true;
    is_bottom_right = false;
  }
  bool is_padding = ((mask_type == NVTE_Mask_Type::NVTE_PADDING_MASK) ||
                     (mask_type == NVTE_Mask_Type::NVTE_PADDING_CAUSAL_MASK));
  bool is_dropout = (dropout_probability != 0.0f);
  bool is_ragged = (nvte_get_qkv_format(layout) == NVTE_QKV_Format::NVTE_THD);
  if (window_size_left == -1) {
    window_size_left = s_q;
  }
  auto cudnn_runtime_version = hipdnnGetVersion();
  const int device_id = cuda::current_device();
  const int sm_arch_ = cuda::sm_arch(device_id);

  try {
    FADescriptor_v1 descriptor{b,
                               h,
                               hg,
                               s_q,
                               s_kv,
                               d,
                               bias_b,
                               bias_h,
                               scaling_factor,
                               true,
                               dropout_probability,
                               layout,
                               bias_type,
                               mask_type,
                               window_size_left,
                               window_size_right,
                               deterministic,
                               tensorType,
                               tensorType};

    namespace fe = cudnn_frontend;
    using graph_and_tensors =
        std::tuple<std::shared_ptr<fe::graph::Graph>,
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // q
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // k
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // v
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // o
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // dO
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // stats
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // attn_scale
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // dQ
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // dK
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // dV
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // bias
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // dBias
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // seq_q
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // seq_kv
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // offset_q
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // offset_k
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // offset_v
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // offset_o
                   std::shared_ptr<fe::graph::Tensor_attributes>,   // dropout_seed
                   std::shared_ptr<fe::graph::Tensor_attributes>>;  // dropout_offset

    using CacheType = std::map<FADescriptor_v1, graph_and_tensors>;
    static thread_local CacheType sdpa_f16_bprop_cache;

    // Get plan from cache if cache is available, otherwise create one
    auto get_graph = [&](CacheType &cache, const FADescriptor_v1 &descriptor) -> graph_and_tensors {
      // if hit, return
      auto it = cache.find(descriptor);
      if (it != cache.end()) {
        auto graph = it->second;
        return graph;
      }

      // otherwise, build the op_graph and the plan. Then update cache
      auto mha_graph = std::make_shared<fe::graph::Graph>();
      mha_graph->set_io_data_type(tensorType)
          .set_intermediate_data_type(fe::DataType_t::FLOAT)
          .set_compute_data_type(fe::DataType_t::FLOAT);

      std::shared_ptr<fe::graph::Tensor_attributes> q, k, v, o, dO, stats, attn_scale;
      std::shared_ptr<fe::graph::Tensor_attributes> bias, dBias, seq_q, seq_kv;
      std::shared_ptr<fe::graph::Tensor_attributes> offset_q, offset_k, offset_v, offset_o;
      std::shared_ptr<fe::graph::Tensor_attributes> dropout_seed, dropout_offset;

      offset_q = mha_graph->tensor(fe::graph::Tensor_attributes()
                                       .set_name("offset_q")
                                       .set_dim({b + 1, 1, 1, 1})
                                       .set_stride({1, 1, 1, 1})
                                       .set_data_type(fe::DataType_t::INT32));
      offset_k = mha_graph->tensor(fe::graph::Tensor_attributes()
                                       .set_name("offset_k")
                                       .set_dim({b + 1, 1, 1, 1})
                                       .set_stride({1, 1, 1, 1})
                                       .set_data_type(fe::DataType_t::INT32));
      offset_v = mha_graph->tensor(fe::graph::Tensor_attributes()
                                       .set_name("offset_v")
                                       .set_dim({b + 1, 1, 1, 1})
                                       .set_stride({1, 1, 1, 1})
                                       .set_data_type(fe::DataType_t::INT32));
      offset_o = mha_graph->tensor(fe::graph::Tensor_attributes()
                                       .set_name("offset_o")
                                       .set_dim({b + 1, 1, 1, 1})
                                       .set_stride({1, 1, 1, 1})
                                       .set_data_type(fe::DataType_t::INT32));
      std::vector<int64_t> q_stride(4);
      std::vector<int64_t> k_stride(4);
      std::vector<int64_t> v_stride(4);
      std::vector<int64_t> o_stride(4);
      generateMatrixStrides(b, h, s_q, s_kv, d, q_stride.data(), layout,
                            NVTE_QKV_Matrix::NVTE_Q_Matrix);
      generateMatrixStrides(b, hg, s_q, s_kv, d, k_stride.data(), layout,
                            NVTE_QKV_Matrix::NVTE_K_Matrix);
      generateMatrixStrides(b, hg, s_q, s_kv, d, v_stride.data(), layout,
                            NVTE_QKV_Matrix::NVTE_V_Matrix);
      generateMatrixStrides(b, h, s_q, s_kv, d, o_stride.data(), layout,
                            NVTE_QKV_Matrix::NVTE_O_Matrix);

      if (is_ragged) {
        q = mha_graph->tensor(fe::graph::Tensor_attributes()
                                  .set_name("Q")
                                  .set_dim({b, h, s_q, d})
                                  .set_stride(q_stride)
                                  .set_ragged_offset(offset_q));
        k = mha_graph->tensor(fe::graph::Tensor_attributes()
                                  .set_name("K")
                                  .set_dim({b, hg, s_kv, d})
                                  .set_stride(k_stride)
                                  .set_ragged_offset(offset_k));
        v = mha_graph->tensor(fe::graph::Tensor_attributes()
                                  .set_name("V")
                                  .set_dim({b, hg, s_kv, d})
                                  .set_stride(v_stride)
                                  .set_ragged_offset(offset_v));
        o = mha_graph->tensor(fe::graph::Tensor_attributes()
                                  .set_name("O")
                                  .set_dim({b, h, s_q, d})
                                  .set_stride(o_stride)
                                  .set_ragged_offset(offset_o));
        dO = mha_graph->tensor(fe::graph::Tensor_attributes()
                                   .set_name("dO")
                                   .set_dim({b, h, s_q, d})
                                   .set_stride(o_stride)
                                   .set_ragged_offset(offset_o));
      } else {
        q = mha_graph->tensor(fe::graph::Tensor_attributes()
                                  .set_name("Q")
                                  .set_dim({b, h, s_q, d})
                                  .set_stride(q_stride));
        k = mha_graph->tensor(fe::graph::Tensor_attributes()
                                  .set_name("K")
                                  .set_dim({b, hg, s_kv, d})
                                  .set_stride(k_stride));
        v = mha_graph->tensor(fe::graph::Tensor_attributes()
                                  .set_name("V")
                                  .set_dim({b, hg, s_kv, d})
                                  .set_stride(v_stride));
        o = mha_graph->tensor(fe::graph::Tensor_attributes()
                                  .set_name("O")
                                  .set_dim({b, h, s_q, d})
                                  .set_stride(o_stride));
        dO = mha_graph->tensor(fe::graph::Tensor_attributes()
                                   .set_name("dO")
                                   .set_dim({b, h, s_q, d})
                                   .set_stride(o_stride));
      }
      stats = mha_graph->tensor(fe::graph::Tensor_attributes()
                                    .set_name("stats")
                                    .set_dim({b, h, s_q, 1})
                                    .set_stride({h * s_q, s_q, 1, 1})
                                    .set_data_type(fe::DataType_t::FLOAT));

      attn_scale = mha_graph->tensor(fe::graph::Tensor_attributes()
                                         .set_name("attn_scale")
                                         .set_dim({1, 1, 1, 1})
                                         .set_stride({1, 1, 1, 1})
                                         .set_is_pass_by_value(true)
                                         .set_data_type(fe::DataType_t::FLOAT));

      fe::graph::SDPA_backward_attributes sdpa_backward_options;
      sdpa_backward_options = fe::graph::SDPA_backward_attributes()
                                  .set_name("flash_attention_backward")
                                  .set_causal_mask(is_causal)
                                  .set_causal_mask_bottom_right(is_bottom_right)
                                  .set_attn_scale(attn_scale);

      if (cudnn_runtime_version >= 90200 && window_size_left != s_q) {
        sdpa_backward_options.set_sliding_window_length(window_size_left);
      }

      if (cudnn_runtime_version >= 90000 && sm_arch_ >= 90) {
        sdpa_backward_options.set_deterministic_algorithm(deterministic);
      }

      sdpa_backward_options.set_alibi_mask(is_alibi);

      if (is_bias) {
        bias = mha_graph->tensor(fe::graph::Tensor_attributes()
                                     .set_name("bias")
                                     .set_dim({bias_b, bias_h, s_q, s_kv})
                                     .set_stride({bias_h * s_q * s_kv, s_q * s_kv, s_kv, 1}));
        dBias = mha_graph->tensor(fe::graph::Tensor_attributes()
                                      .set_name("dBias")
                                      .set_dim({bias_b, bias_h, s_q, s_kv})
                                      .set_stride({bias_h * s_q * s_kv, s_q * s_kv, s_kv, 1}));
        sdpa_backward_options.set_bias(bias);
        // shapes [1, 1, s, s], [b, 1, s, s], [b, h, s, s]
        // are not supported for dbias calculation but they are
        // supported for forward bias calculation
        if ((bias_b == 1) && (bias_h == h)) {
          sdpa_backward_options.set_dbias(dBias);
        }
      }

      if (is_padding) {
        seq_q = mha_graph->tensor(fe::graph::Tensor_attributes()
                                      .set_name("seq_q")
                                      .set_dim({b, 1, 1, 1})
                                      .set_stride({1, 1, 1, 1})
                                      .set_data_type(fe::DataType_t::INT32));
        seq_kv = mha_graph->tensor(fe::graph::Tensor_attributes()
                                       .set_name("seq_kv")
                                       .set_dim({b, 1, 1, 1})
                                       .set_stride({1, 1, 1, 1})
                                       .set_data_type(fe::DataType_t::INT32));
        sdpa_backward_options.set_padding_mask(is_padding)
            .set_seq_len_q(seq_q)
            .set_seq_len_kv(seq_kv);
      }

      if (is_dropout) {
        dropout_seed = mha_graph->tensor(fe::graph::Tensor_attributes()
                                             .set_name("Seed")
                                             .set_dim({1, 1, 1, 1})
                                             .set_stride({1, 1, 1, 1})
                                             .set_data_type(fe::DataType_t::INT64));
        dropout_offset = mha_graph->tensor(fe::graph::Tensor_attributes()
                                               .set_name("Offset")
                                               .set_dim({1, 1, 1, 1})
                                               .set_stride({1, 1, 1, 1})
                                               .set_data_type(fe::DataType_t::INT64));
        sdpa_backward_options.set_dropout(dropout_probability, dropout_seed, dropout_offset);
      }

      auto [dQ, dK, dV] = mha_graph->sdpa_backward(q, k, v, o, dO, stats, sdpa_backward_options);

      if (is_ragged) {
        dQ->set_output(true)
            .set_dim({b, h, s_q, d})
            .set_stride(q_stride)
            .set_ragged_offset(offset_q);
        dK->set_output(true)
            .set_dim({b, hg, s_kv, d})
            .set_stride(k_stride)
            .set_ragged_offset(offset_k);
        dV->set_output(true)
            .set_dim({b, hg, s_kv, d})
            .set_stride(v_stride)
            .set_ragged_offset(offset_v);
      } else {
        dQ->set_output(true).set_dim({b, h, s_q, d}).set_stride(q_stride);
        dK->set_output(true).set_dim({b, hg, s_kv, d}).set_stride(k_stride);
        dV->set_output(true).set_dim({b, hg, s_kv, d}).set_stride(v_stride);
      }

      std::tuple<std::shared_ptr<fe::graph::Tensor_attributes>,  // q
                 std::shared_ptr<fe::graph::Tensor_attributes>,  // k
                 std::shared_ptr<fe::graph::Tensor_attributes>,  // v
                 std::shared_ptr<fe::graph::Tensor_attributes>,  // o
                 std::shared_ptr<fe::graph::Tensor_attributes>,  // dO
                 std::shared_ptr<fe::graph::Tensor_attributes>,  // stats
                 std::shared_ptr<fe::graph::Tensor_attributes>,  // attn_scale
                 std::shared_ptr<fe::graph::Tensor_attributes>,  // dQ
                 std::shared_ptr<fe::graph::Tensor_attributes>,  // dK
                 std::shared_ptr<fe::graph::Tensor_attributes>>  // dV
          key_tensors_tuple = std::make_tuple(q, k, v, o, dO, stats, attn_scale, dQ, dK, dV);
      auto bias_tuple = is_bias ? std::make_tuple(bias, dBias) : std::make_tuple(nullptr, nullptr);
      auto padding_tuple =
          is_padding ? std::make_tuple(seq_q, seq_kv) : std::make_tuple(nullptr, nullptr);
      auto offset_tuple = is_ragged ? std::make_tuple(offset_q, offset_k, offset_v, offset_o)
                                    : std::make_tuple(nullptr, nullptr, nullptr, nullptr);
      auto dropout_tuple = is_dropout ? std::make_tuple(dropout_seed, dropout_offset)
                                      : std::make_tuple(nullptr, nullptr);

      NVTE_CHECK_CUDNN_FE(mha_graph->validate());
      NVTE_CHECK_CUDNN_FE(mha_graph->build_operation_graph(handle));
      NVTE_CHECK_CUDNN_FE(mha_graph->create_execution_plans({fe::HeurMode_t::A}));
      NVTE_CHECK_CUDNN_FE(mha_graph->check_support(handle));
      NVTE_CHECK_CUDNN_FE(mha_graph->build_plans(handle));

      auto return_tuple = std::tuple_cat(std::make_tuple(mha_graph), key_tensors_tuple, bias_tuple,
                                         padding_tuple, offset_tuple, dropout_tuple);
      cache.insert({descriptor, return_tuple});

      return return_tuple;
    };

    auto [mha_graph, q, k, v, o, dO, stats, attn_scale, dQ, dK, dV, bias, dBias, seq_q, seq_kv,
          offset_q, offset_k, offset_v, offset_o, dropout_seed, dropout_offset] =
        get_graph(sdpa_f16_bprop_cache, descriptor);

    auto plan_workspace_size = mha_graph->get_workspace_size();

    // Exit to request upper level API to allocate memory if needed
    size_t actual_seqlen_workspace_size = 2 * b * sizeof(int32_t);
    size_t seqlen_offsets_workspace_size = 4 * (b + 1) * sizeof(int32_t);
    if (workspace == nullptr) {
      *workspace_size =
          plan_workspace_size + actual_seqlen_workspace_size + seqlen_offsets_workspace_size;
      return;
    }

    // cuDNN stream check needs to be moved here to support dummy kernel calls with
    // null streams for sizing the cuDNN workspace.
    NVTE_CHECK_CUDNN(hipdnnSetStream(handle, stream));

    // build variant pack
    std::unordered_map<std::shared_ptr<fe::graph::Tensor_attributes>, void *> variant_pack = {
        {q, devPtrQ},
        {k, devPtrKTranspose},
        {v, devPtrVTranspose},
        {o, devPtrO},
        {dO, devPtrdO},
        {stats, devPtrSoftmaxStats},
        {attn_scale, &scaling_factor},
        {dQ, devPtrdQ},
        {dK, devPtrdK},
        {dV, devPtrdV},
    };

    if (is_bias) {
      variant_pack[bias] = devPtrBias;
      if ((bias_b == 1) && (bias_h == h)) {
        variant_pack[dBias] = devPtrdBias;
      } else {
        variant_pack[dBias] = nullptr;
      }
    }

    if (is_padding) {
      constexpr size_t nthreads_per_block = 128;
      const size_t grid = (b + nthreads_per_block - 1) / nthreads_per_block;
      void *devActualSeqlenQ = static_cast<int8_t *>(workspace) + plan_workspace_size;
      void *devActualSeqlenKV = static_cast<int8_t *>(devActualSeqlenQ) + b * sizeof(int32_t);
      cu_seqlens_to_actual_seqlens<<<grid, nthreads_per_block, 0, stream>>>(
          b, static_cast<const int32_t *>(devPtrCuSeqlensQ),
          static_cast<const int32_t *>(devPtrCuSeqlensKV), static_cast<int32_t *>(devActualSeqlenQ),
          static_cast<int32_t *>(devActualSeqlenKV));
      variant_pack[seq_q] = devActualSeqlenQ;
      variant_pack[seq_kv] = devActualSeqlenKV;
    }

    if (is_ragged) {
      constexpr size_t nthreads_per_block = 128;
      const size_t grid = (b + nthreads_per_block) / nthreads_per_block;
      void *devOffsetsQ =
          static_cast<int8_t *>(workspace) + plan_workspace_size + actual_seqlen_workspace_size;
      void *devOffsetsK = static_cast<int8_t *>(devOffsetsQ) + (b + 1) * sizeof(int32_t);
      void *devOffsetsV = static_cast<int8_t *>(devOffsetsK) + (b + 1) * sizeof(int32_t);
      void *devOffsetsO = static_cast<int8_t *>(devOffsetsV) + (b + 1) * sizeof(int32_t);
      NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(layout);
      cu_seqlens_padded_to_offsets<<<grid, nthreads_per_block, 0, stream>>>(
          layout_group, b, h, hg, d, static_cast<int32_t *>(devPtrSeqOffsetsQ),
          static_cast<int32_t *>(devPtrSeqOffsetsKV), static_cast<int32_t *>(devOffsetsQ),
          static_cast<int32_t *>(devOffsetsK), static_cast<int32_t *>(devOffsetsV),
          static_cast<int32_t *>(devOffsetsO));
      variant_pack[offset_q] = devOffsetsQ;
      variant_pack[offset_k] = devOffsetsK;
      variant_pack[offset_v] = devOffsetsV;
      variant_pack[offset_o] = devOffsetsO;
    }

    if (is_dropout) {
      variant_pack[dropout_seed] = devPtrDropoutSeed;
      variant_pack[dropout_offset] = devPtrDropoutOffset;
    }

    NVTE_CHECK_CUDNN_FE(mha_graph->execute(handle, variant_pack, workspace));
  } catch (cudnn_frontend::cudnnException &e) {
    NVTE_ERROR(e.what());
  }
}
}  // namespace fused_attn

using namespace transformer_engine::fused_attn;
void fused_attn_arbitrary_seqlen_fwd_qkvpacked(
    size_t batch, size_t num_attn_heads, size_t max_seqlen, size_t head_dim, bool is_training,
    float attn_scale, float p_dropout, NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type,
    NVTE_Mask_Type mask_type, int64_t window_size_left, int64_t window_size_right,
    const Tensor *input_QKV, const Tensor *input_Bias, Tensor *output_O,
    NVTETensorPack *Aux_CTX_Tensors, const Tensor *cu_seqlens, const Tensor *cu_seqlens_padded,
    const Tensor *rng_state, Tensor *workspace, hipStream_t stream, hipdnnHandle_t handle) {
  using namespace transformer_engine;

  const auto QKV_type = input_QKV->data.dtype;
  void *devPtrQKV = input_QKV->data.dptr;
  NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(qkv_layout);
  size_t stride = 0;
  if (layout_group == NVTE_QKV_Layout_Group::NVTE_3HD) {
    stride = typeToSize(QKV_type) * num_attn_heads * head_dim;
  } else if (layout_group == NVTE_QKV_Layout_Group::NVTE_H3D) {
    stride = typeToSize(QKV_type) * head_dim;
  }
  void *devPtrQ = static_cast<void *>(devPtrQKV);
  void *devPtrK = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + stride);
  void *devPtrV = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + 2 * stride);

  void *devPtrBias = nullptr;
  size_t bias_b = 0;
  size_t bias_h = 0;
  if ((bias_type != NVTE_Bias_Type::NVTE_NO_BIAS) && (bias_type != NVTE_Bias_Type::NVTE_ALIBI)) {
    devPtrBias = input_Bias->data.dptr;
    bias_b = input_Bias->data.shape[0];
    bias_h = input_Bias->data.shape[1];
  }
  void *devPtrO = output_O->data.dptr;
  void *devPtrS = nullptr;
  void *devPtrCuSeqlens = cu_seqlens->data.dptr;
  void *devPtrSeqOffsets = cu_seqlens_padded->data.dptr;

  if (Aux_CTX_Tensors->size == 0) {
    if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI)) {
      Aux_CTX_Tensors->size = 3;
      Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
      output_S->data.dptr = nullptr;
      output_S->data.shape = {batch, num_attn_heads, max_seqlen, 1};
      output_S->data.dtype = DType::kFloat32;
      Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
      output_rng_state->data.dptr = nullptr;
      output_rng_state->data.shape = {2};
      output_rng_state->data.dtype = DType::kInt64;
      Tensor *output_bias = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[2]);
      output_bias->data.dptr = nullptr;
      output_bias->data.shape = {bias_b, bias_h, max_seqlen, max_seqlen};
      output_bias->data.dtype = QKV_type;
    } else {
      Aux_CTX_Tensors->size = 2;
      Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
      output_S->data.dptr = nullptr;
      output_S->data.shape = {batch, num_attn_heads, max_seqlen, 1};
      output_S->data.dtype = DType::kFloat32;
      Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
      output_rng_state->data.dptr = nullptr;
      output_rng_state->data.shape = {2};
      output_rng_state->data.dtype = DType::kInt64;
    }
  } else if (Aux_CTX_Tensors->size == 2) {
    Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
    devPtrS = output_S->data.dptr;
    Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
    output_rng_state->data.dptr = rng_state->data.dptr;
  } else if (Aux_CTX_Tensors->size == 3) {
    Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
    devPtrS = output_S->data.dptr;
    Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
    output_rng_state->data.dptr = rng_state->data.dptr;
    Tensor *output_bias = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[2]);
    output_bias->data.dptr = devPtrBias;
  } else {
    NVTE_ERROR("Unexpected Aux_CTX_Tensors->size.");
  }

  void *devPtrDropoutSeed = rng_state->data.dptr;
  void *devPtrDropoutOffset =
      reinterpret_cast<void *>(reinterpret_cast<uint64_t *>(rng_state->data.dptr) + 1);

  size_t workspace_size = 0;

  fused_attn_arbitrary_seqlen_fwd_impl(
      batch, num_attn_heads, num_attn_heads, max_seqlen, max_seqlen, head_dim, bias_b, bias_h,
      is_training, attn_scale, p_dropout, qkv_layout, bias_type, mask_type, window_size_left,
      window_size_right, devPtrQ, devPtrK, devPtrV, devPtrBias, devPtrS, devPtrO, devPtrDropoutSeed,
      devPtrDropoutOffset, devPtrCuSeqlens, devPtrCuSeqlens, devPtrSeqOffsets, devPtrSeqOffsets,
      get_cudnn_fe_dtype(QKV_type), workspace->data.dptr, &workspace_size, stream, handle);

  if (workspace_size > 0) {
    if (workspace->data.dptr == nullptr) {
      workspace->data.shape = {workspace_size};
      workspace->data.dtype = DType::kByte;
      return;
    }
  } else if (workspace_size == 0) {
    workspace->data.shape = {1};
    workspace->data.dtype = DType::kByte;
    return;
  } else {
    NVTE_ERROR("Unexpected workspace_size.");
  }
}

void fused_attn_arbitrary_seqlen_bwd_qkvpacked(
    size_t batch, size_t num_attn_heads, size_t max_seqlen, size_t head_dim, float attn_scale,
    float p_dropout, NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type mask_type,
    int64_t window_size_left, int64_t window_size_right, bool deterministic,
    const Tensor *input_QKV, const Tensor *input_O, const Tensor *input_dO,
    const Tensor *input_Bias, Tensor *output_S, Tensor *output_dQKV, Tensor *output_dBias,
    const Tensor *cu_seqlens, const Tensor *cu_seqlens_padded, const Tensor *rng_state,
    Tensor *workspace, hipStream_t stream, hipdnnHandle_t handle) {
  using namespace transformer_engine;

  const auto QKV_type = input_QKV->data.dtype;
  void *devPtrQKV = input_QKV->data.dptr;

  NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(qkv_layout);
  size_t stride = 0;
  if (layout_group == NVTE_QKV_Layout_Group::NVTE_3HD) {
    stride = typeToSize(QKV_type) * num_attn_heads * head_dim;
  } else if (layout_group == NVTE_QKV_Layout_Group::NVTE_H3D) {
    stride = typeToSize(QKV_type) * head_dim;
  }
  void *devPtrQ = devPtrQKV;
  void *devPtrK = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + stride);
  void *devPtrV = static_cast<void *>(static_cast<int8_t *>(devPtrQKV) + 2 * stride);

  void *devPtrO = input_O->data.dptr;
  void *devPtrdO = input_dO->data.dptr;
  void *devPtrBias = nullptr;
  void *devPtrdBias = nullptr;
  size_t bias_b = 0;
  size_t bias_h = 0;
  if ((bias_type != NVTE_Bias_Type::NVTE_NO_BIAS) && (bias_type != NVTE_Bias_Type::NVTE_ALIBI)) {
    devPtrBias = input_Bias->data.dptr;
    devPtrdBias = output_dBias->data.dptr;
    bias_b = output_dBias->data.shape[0];
    bias_h = output_dBias->data.shape[1];
  }

  void *devPtrdQKV = output_dQKV->data.dptr;
  void *devPtrdQ = devPtrdQKV;
  void *devPtrdK = static_cast<void *>(static_cast<int8_t *>(devPtrdQKV) + stride);
  void *devPtrdV = static_cast<void *>(static_cast<int8_t *>(devPtrdQKV) + 2 * stride);

  void *devPtrSoftmaxStats = nullptr;
  devPtrSoftmaxStats = output_S->data.dptr;

  void *devPtrCuSeqlens = cu_seqlens->data.dptr;
  void *devPtrSeqOffsets = cu_seqlens_padded->data.dptr;

  void *devPtrDropoutSeed = rng_state->data.dptr;
  void *devPtrDropoutOffset =
      reinterpret_cast<void *>(reinterpret_cast<uint64_t *>(rng_state->data.dptr) + 1);

  size_t workspace_size = 0;

  fused_attn_arbitrary_seqlen_bwd_impl(
      batch, num_attn_heads, num_attn_heads, max_seqlen, max_seqlen, head_dim, bias_b, bias_h,
      attn_scale, p_dropout, qkv_layout, bias_type, mask_type, window_size_left, window_size_right,
      deterministic, devPtrQ, devPtrK, devPtrV, devPtrO, devPtrSoftmaxStats, devPtrBias, devPtrdQ,
      devPtrdK, devPtrdV, devPtrdO, devPtrdBias, devPtrDropoutSeed, devPtrDropoutOffset,
      devPtrCuSeqlens, devPtrCuSeqlens, devPtrSeqOffsets, devPtrSeqOffsets,
      get_cudnn_fe_dtype(QKV_type), workspace->data.dptr, &workspace_size, stream, handle);

  if (workspace_size > 0) {
    if (workspace->data.dptr == nullptr) {
      workspace->data.shape = {workspace_size};
      workspace->data.dtype = DType::kByte;
      return;
    }
  } else if (workspace_size == 0) {
    workspace->data.shape = {1};
    workspace->data.dtype = DType::kByte;
    return;
  } else {
    NVTE_ERROR("Unexpected workspace_size.");
  }
}
void fused_attn_arbitrary_seqlen_fwd_kvpacked(
    size_t batch, size_t num_attn_heads, size_t num_gqa_groups, size_t max_seqlen_q,
    size_t max_seqlen_kv, size_t head_dim, bool is_training, float attn_scale, float p_dropout,
    NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type mask_type,
    int64_t window_size_left, int64_t window_size_right, const Tensor *input_Q,
    const Tensor *input_KV, const Tensor *input_Bias, Tensor *output_O,
    NVTETensorPack *Aux_CTX_Tensors, const Tensor *cu_seqlens_q, const Tensor *cu_seqlens_kv,
    const Tensor *cu_seqlens_q_padded, const Tensor *cu_seqlens_kv_padded, const Tensor *rng_state,
    Tensor *workspace, hipStream_t stream, hipdnnHandle_t handle) {
  using namespace transformer_engine;

  const auto QKV_type = input_Q->data.dtype;
  void *devPtrQ = input_Q->data.dptr;
  void *devPtrKV = input_KV->data.dptr;
  NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(qkv_layout);
  size_t stride = 0;
  if (layout_group == NVTE_QKV_Layout_Group::NVTE_HD_2HD) {
    stride = typeToSize(QKV_type) * num_gqa_groups * head_dim;
  } else if (layout_group == NVTE_QKV_Layout_Group::NVTE_HD_H2D) {
    stride = typeToSize(QKV_type) * head_dim;
  }
  void *devPtrK = devPtrKV;
  void *devPtrV = static_cast<void *>(static_cast<int8_t *>(devPtrKV) + stride);

  void *devPtrBias = nullptr;
  size_t bias_b = 0;
  size_t bias_h = 0;
  if ((bias_type != NVTE_Bias_Type::NVTE_NO_BIAS) && (bias_type != NVTE_Bias_Type::NVTE_ALIBI)) {
    devPtrBias = input_Bias->data.dptr;
    bias_b = input_Bias->data.shape[0];
    bias_h = input_Bias->data.shape[1];
  }
  void *devPtrO = output_O->data.dptr;
  void *devPtrS = nullptr;

  void *devPtrCuSeqlensQ = cu_seqlens_q->data.dptr;
  void *devPtrCuSeqlensKV = cu_seqlens_kv->data.dptr;
  void *devPtrSeqOffsetsQ = cu_seqlens_q_padded->data.dptr;
  void *devPtrSeqOffsetsKV = cu_seqlens_kv_padded->data.dptr;

  if (Aux_CTX_Tensors->size == 0) {
    if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI)) {
      Aux_CTX_Tensors->size = 3;
      Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
      output_S->data.dptr = nullptr;
      output_S->data.shape = {batch, num_attn_heads, max_seqlen_q, 1};
      output_S->data.dtype = DType::kFloat32;
      Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
      output_rng_state->data.dptr = nullptr;
      output_rng_state->data.shape = {2};
      output_rng_state->data.dtype = DType::kInt64;
      Tensor *output_bias = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[2]);
      output_bias->data.dptr = nullptr;
      output_bias->data.shape = {bias_b, bias_h, max_seqlen_q, max_seqlen_kv};
      output_bias->data.dtype = QKV_type;
    } else {
      Aux_CTX_Tensors->size = 2;
      Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
      output_S->data.dptr = nullptr;
      output_S->data.shape = {batch, num_attn_heads, max_seqlen_q, 1};
      output_S->data.dtype = DType::kFloat32;
      Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
      output_rng_state->data.dptr = nullptr;
      output_rng_state->data.shape = {2};
      output_rng_state->data.dtype = DType::kInt64;
    }
  } else if (Aux_CTX_Tensors->size == 2) {
    Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
    devPtrS = output_S->data.dptr;
    Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
    output_rng_state->data.dptr = rng_state->data.dptr;
  } else if (Aux_CTX_Tensors->size == 3) {
    Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
    devPtrS = output_S->data.dptr;
    Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
    output_rng_state->data.dptr = rng_state->data.dptr;
    Tensor *output_bias = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[2]);
    output_bias->data.dptr = devPtrBias;
  } else {
    NVTE_ERROR("Unexpected Aux_CTX_Tensors->size.");
  }

  void *devPtrDropoutSeed = rng_state->data.dptr;
  void *devPtrDropoutOffset =
      reinterpret_cast<void *>(reinterpret_cast<uint64_t *>(rng_state->data.dptr) + 1);

  size_t workspace_size = 0;

  fused_attn_arbitrary_seqlen_fwd_impl(
      batch, num_attn_heads, num_gqa_groups, max_seqlen_q, max_seqlen_kv, head_dim, bias_b, bias_h,
      is_training, attn_scale, p_dropout, qkv_layout, bias_type, mask_type, window_size_left,
      window_size_right, devPtrQ, devPtrK, devPtrV, devPtrBias, devPtrS, devPtrO, devPtrDropoutSeed,
      devPtrDropoutOffset, devPtrCuSeqlensQ, devPtrCuSeqlensKV, devPtrSeqOffsetsQ,
      devPtrSeqOffsetsKV, get_cudnn_fe_dtype(QKV_type), workspace->data.dptr, &workspace_size,
      stream, handle);

  if (workspace_size > 0) {
    if (workspace->data.dptr == nullptr) {
      workspace->data.shape = {workspace_size};
      workspace->data.dtype = DType::kByte;
      return;
    }
  } else if (workspace_size == 0) {
    workspace->data.shape = {1};
    workspace->data.dtype = DType::kByte;
    return;
  } else {
    NVTE_ERROR("Unexpected workspace_size.");
  }
}

void fused_attn_arbitrary_seqlen_bwd_kvpacked(
    size_t batch, size_t num_attn_heads, size_t num_gqa_groups, size_t max_seqlen_q,
    size_t max_seqlen_kv, size_t head_dim, float attn_scale, float p_dropout,
    NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type mask_type,
    int64_t window_size_left, int64_t window_size_right, bool deterministic, const Tensor *input_Q,
    const Tensor *input_KV, const Tensor *input_O, const Tensor *input_dO, const Tensor *input_Bias,
    Tensor *output_S, Tensor *output_dQ, Tensor *output_dKV, Tensor *output_dBias,
    const Tensor *cu_seqlens_q, const Tensor *cu_seqlens_kv, const Tensor *cu_seqlens_q_padded,
    const Tensor *cu_seqlens_kv_padded, const Tensor *rng_state, Tensor *workspace,
    hipStream_t stream, hipdnnHandle_t handle) {
  using namespace transformer_engine;

  const auto QKV_type = input_Q->data.dtype;
  void *devPtrQ = input_Q->data.dptr;
  void *devPtrKV = input_KV->data.dptr;
  NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(qkv_layout);
  size_t stride = 0;
  if (layout_group == NVTE_QKV_Layout_Group::NVTE_HD_2HD) {
    stride = typeToSize(QKV_type) * num_gqa_groups * head_dim;
  } else if (layout_group == NVTE_QKV_Layout_Group::NVTE_HD_H2D) {
    stride = typeToSize(QKV_type) * head_dim;
  }
  void *devPtrK = devPtrKV;
  void *devPtrV = static_cast<void *>(static_cast<int8_t *>(devPtrKV) + stride);

  void *devPtrO = input_O->data.dptr;
  void *devPtrdO = input_dO->data.dptr;
  void *devPtrBias = nullptr;
  void *devPtrdBias = nullptr;
  size_t bias_b = 0;
  size_t bias_h = 0;
  if ((bias_type != NVTE_Bias_Type::NVTE_NO_BIAS) && (bias_type != NVTE_Bias_Type::NVTE_ALIBI)) {
    devPtrBias = input_Bias->data.dptr;
    devPtrdBias = output_dBias->data.dptr;
    bias_b = output_dBias->data.shape[0];
    bias_h = output_dBias->data.shape[1];
  }

  void *devPtrdQ = output_dQ->data.dptr;
  void *devPtrdKV = output_dKV->data.dptr;
  void *devPtrdK = devPtrdKV;
  void *devPtrdV = static_cast<void *>(static_cast<int8_t *>(devPtrdKV) + stride);

  void *devPtrSoftmaxStats = nullptr;
  devPtrSoftmaxStats = output_S->data.dptr;

  void *devPtrCuSeqlensQ = cu_seqlens_q->data.dptr;
  void *devPtrCuSeqlensKV = cu_seqlens_kv->data.dptr;
  void *devPtrSeqOffsetsQ = cu_seqlens_q_padded->data.dptr;
  void *devPtrSeqOffsetsKV = cu_seqlens_kv_padded->data.dptr;

  void *devPtrDropoutSeed = rng_state->data.dptr;
  void *devPtrDropoutOffset =
      reinterpret_cast<void *>(reinterpret_cast<uint64_t *>(rng_state->data.dptr) + 1);

  size_t workspace_size = 0;

  fused_attn_arbitrary_seqlen_bwd_impl(
      batch, num_attn_heads, num_gqa_groups, max_seqlen_q, max_seqlen_kv, head_dim, bias_b, bias_h,
      attn_scale, p_dropout, qkv_layout, bias_type, mask_type, window_size_left, window_size_right,
      deterministic, devPtrQ, devPtrK, devPtrV, devPtrO, devPtrSoftmaxStats, devPtrBias, devPtrdQ,
      devPtrdK, devPtrdV, devPtrdO, devPtrdBias, devPtrDropoutSeed, devPtrDropoutOffset,
      devPtrCuSeqlensQ, devPtrCuSeqlensKV, devPtrSeqOffsetsQ, devPtrSeqOffsetsKV,
      get_cudnn_fe_dtype(QKV_type), workspace->data.dptr, &workspace_size, stream, handle);

  if (workspace_size > 0) {
    if (workspace->data.dptr == nullptr) {
      workspace->data.shape = {workspace_size};
      workspace->data.dtype = DType::kByte;
      return;
    }
  } else if (workspace_size == 0) {
    workspace->data.shape = {1};
    workspace->data.dtype = DType::kByte;
    return;
  } else {
    NVTE_ERROR("Unexpected workspace_size.");
  }
}

void fused_attn_arbitrary_seqlen_fwd(
    size_t batch, size_t num_attn_heads, size_t num_gqa_groups, size_t max_seqlen_q,
    size_t max_seqlen_kv, size_t head_dim, bool is_training, float attn_scale, float p_dropout,
    NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type mask_type,
    int64_t window_size_left, int64_t window_size_right, const Tensor *input_Q,
    const Tensor *input_K, const Tensor *input_V, const Tensor *input_Bias, Tensor *output_O,
    NVTETensorPack *Aux_CTX_Tensors, const Tensor *cu_seqlens_q, const Tensor *cu_seqlens_kv,
    const Tensor *cu_seqlens_q_padded, const Tensor *cu_seqlens_kv_padded, const Tensor *rng_state,
    Tensor *workspace, hipStream_t stream, hipdnnHandle_t handle) {
  using namespace transformer_engine;

  const auto QKV_type = input_Q->data.dtype;
  void *devPtrQ = input_Q->data.dptr;
  void *devPtrK = input_K->data.dptr;
  void *devPtrV = input_V->data.dptr;
  void *devPtrO = output_O->data.dptr;
  void *devPtrS = nullptr;
  void *devPtrBias = nullptr;
  size_t bias_b = 0;
  size_t bias_h = 0;
  if ((bias_type != NVTE_Bias_Type::NVTE_NO_BIAS) && (bias_type != NVTE_Bias_Type::NVTE_ALIBI)) {
    devPtrBias = input_Bias->data.dptr;
    bias_b = input_Bias->data.shape[0];
    bias_h = input_Bias->data.shape[1];
  }

  void *devPtrCuSeqlensQ = cu_seqlens_q->data.dptr;
  void *devPtrCuSeqlensKV = cu_seqlens_kv->data.dptr;
  void *devPtrSeqOffsetsQ = cu_seqlens_q_padded->data.dptr;
  void *devPtrSeqOffsetsKV = cu_seqlens_kv_padded->data.dptr;

  if (Aux_CTX_Tensors->size == 0) {
    if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI)) {
      Aux_CTX_Tensors->size = 3;
      Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
      output_S->data.dptr = nullptr;
      output_S->data.shape = {batch, num_attn_heads, max_seqlen_q, 1};
      output_S->data.dtype = DType::kFloat32;
      Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
      output_rng_state->data.dptr = nullptr;
      output_rng_state->data.shape = {2};
      output_rng_state->data.dtype = DType::kInt64;
      Tensor *output_bias = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[2]);
      output_bias->data.dptr = nullptr;
      output_bias->data.shape = {bias_b, bias_h, max_seqlen_q, max_seqlen_kv};
      output_bias->data.dtype = QKV_type;
    } else {
      Aux_CTX_Tensors->size = 2;
      Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
      output_S->data.dptr = nullptr;
      output_S->data.shape = {batch, num_attn_heads, max_seqlen_q, 1};
      output_S->data.dtype = DType::kFloat32;
      Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
      output_rng_state->data.dptr = nullptr;
      output_rng_state->data.shape = {2};
      output_rng_state->data.dtype = DType::kInt64;
    }
  } else if (Aux_CTX_Tensors->size == 2) {
    Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
    devPtrS = output_S->data.dptr;
    Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
    output_rng_state->data.dptr = rng_state->data.dptr;
  } else if (Aux_CTX_Tensors->size == 3) {
    Tensor *output_S = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[0]);
    devPtrS = output_S->data.dptr;
    Tensor *output_rng_state = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[1]);
    output_rng_state->data.dptr = rng_state->data.dptr;
    Tensor *output_bias = reinterpret_cast<Tensor *>(Aux_CTX_Tensors->tensors[2]);
    output_bias->data.dptr = devPtrBias;
  } else {
    NVTE_ERROR("Unexpected Aux_CTX_Tensors->size.");
  }

  void *devPtrDropoutSeed = rng_state->data.dptr;
  void *devPtrDropoutOffset =
      reinterpret_cast<void *>(reinterpret_cast<uint64_t *>(rng_state->data.dptr) + 1);

  size_t workspace_size = 0;

  fused_attn_arbitrary_seqlen_fwd_impl(
      batch, num_attn_heads, num_gqa_groups, max_seqlen_q, max_seqlen_kv, head_dim, bias_b, bias_h,
      is_training, attn_scale, p_dropout, qkv_layout, bias_type, mask_type, window_size_left,
      window_size_right, devPtrQ, devPtrK, devPtrV, devPtrBias, devPtrS, devPtrO, devPtrDropoutSeed,
      devPtrDropoutOffset, devPtrCuSeqlensQ, devPtrCuSeqlensKV, devPtrSeqOffsetsQ,
      devPtrSeqOffsetsKV, get_cudnn_fe_dtype(QKV_type), workspace->data.dptr, &workspace_size,
      stream, handle);

  if (workspace_size > 0) {
    if (workspace->data.dptr == nullptr) {
      workspace->data.shape = {workspace_size};
      workspace->data.dtype = DType::kByte;
      return;
    }
  } else if (workspace_size == 0) {
    workspace->data.shape = {1};
    workspace->data.dtype = DType::kByte;
    return;
  } else {
    NVTE_ERROR("Unexpected workspace_size.");
  }
}

void fused_attn_arbitrary_seqlen_bwd(
    size_t batch, size_t num_attn_heads, size_t num_gqa_groups, size_t max_seqlen_q,
    size_t max_seqlen_kv, size_t head_dim, float attn_scale, float p_dropout,
    NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type mask_type,
    int64_t window_size_left, int64_t window_size_right, bool deterministic, const Tensor *input_Q,
    const Tensor *input_K, const Tensor *input_V, const Tensor *input_O, const Tensor *input_dO,
    const Tensor *input_Bias, Tensor *output_S, Tensor *output_dQ, Tensor *output_dK,
    Tensor *output_dV, Tensor *output_dBias, const Tensor *cu_seqlens_q,
    const Tensor *cu_seqlens_kv, const Tensor *cu_seqlens_q_padded,
    const Tensor *cu_seqlens_kv_padded, const Tensor *rng_state, Tensor *workspace,
    hipStream_t stream, hipdnnHandle_t handle) {
  using namespace transformer_engine;
  const auto QKV_type = input_Q->data.dtype;
  void *devPtrQ = input_Q->data.dptr;
  void *devPtrK = input_K->data.dptr;
  void *devPtrV = input_V->data.dptr;
  void *devPtrO = input_O->data.dptr;
  void *devPtrdO = input_dO->data.dptr;
  void *devPtrBias = nullptr;
  void *devPtrdBias = nullptr;
  size_t bias_b = 0;
  size_t bias_h = 0;
  if ((bias_type != NVTE_Bias_Type::NVTE_NO_BIAS) && (bias_type != NVTE_Bias_Type::NVTE_ALIBI)) {
    devPtrBias = input_Bias->data.dptr;
    devPtrdBias = output_dBias->data.dptr;
    bias_b = output_dBias->data.shape[0];
    bias_h = output_dBias->data.shape[1];
  }

  void *devPtrdQ = output_dQ->data.dptr;
  void *devPtrdK = output_dK->data.dptr;
  void *devPtrdV = output_dV->data.dptr;
  void *devPtrSoftmaxStats = nullptr;
  devPtrSoftmaxStats = output_S->data.dptr;

  void *devPtrCuSeqlensQ = cu_seqlens_q->data.dptr;
  void *devPtrCuSeqlensKV = cu_seqlens_kv->data.dptr;
  void *devPtrSeqOffsetsQ = cu_seqlens_q_padded->data.dptr;
  void *devPtrSeqOffsetsKV = cu_seqlens_kv_padded->data.dptr;

  void *devPtrDropoutSeed = rng_state->data.dptr;
  void *devPtrDropoutOffset =
      reinterpret_cast<void *>(reinterpret_cast<uint64_t *>(rng_state->data.dptr) + 1);

  size_t workspace_size = 0;

  fused_attn_arbitrary_seqlen_bwd_impl(
      batch, num_attn_heads, num_gqa_groups, max_seqlen_q, max_seqlen_kv, head_dim, bias_b, bias_h,
      attn_scale, p_dropout, qkv_layout, bias_type, mask_type, window_size_left, window_size_right,
      deterministic, devPtrQ, devPtrK, devPtrV, devPtrO, devPtrSoftmaxStats, devPtrBias, devPtrdQ,
      devPtrdK, devPtrdV, devPtrdO, devPtrdBias, devPtrDropoutSeed, devPtrDropoutOffset,
      devPtrCuSeqlensQ, devPtrCuSeqlensKV, devPtrSeqOffsetsQ, devPtrSeqOffsetsKV,
      get_cudnn_fe_dtype(QKV_type), workspace->data.dptr, &workspace_size, stream, handle);

  if (workspace_size > 0) {
    if (workspace->data.dptr == nullptr) {
      workspace->data.shape = {workspace_size};
      workspace->data.dtype = DType::kByte;
      return;
    }
  } else if (workspace_size == 0) {
    workspace->data.shape = {1};
    workspace->data.dtype = DType::kByte;
    return;
  } else {
    NVTE_ERROR("Unexpected workspace_size.");
  }
}
}  // namespace transformer_engine
#endif  // CUDNN_VERSION >= 8900
