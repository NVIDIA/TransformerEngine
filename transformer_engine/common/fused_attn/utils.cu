#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "transformer_engine/fused_attn.h"
#include "../common.h"
#include "utils.h"

namespace transformer_engine {
namespace fused_attn {

using namespace transformer_engine;

// get matrix strides based on matrix type
void generateMatrixStrides(
            int64_t b, int64_t h,
            int64_t s_q, int64_t s_kv,
            int64_t d, int64_t* strideA,
            NVTE_QKV_Layout layout, NVTE_QKV_Matrix matrix) {
    constexpr int batch_dim_idx   = 0;
    constexpr int head_dim_idx    = 1;
    constexpr int seqlen_dim_idx  = 2;
    constexpr int hidden_dim_idx  = 3;

    constexpr int seqlen_transpose_dim_idx = 3;
    constexpr int hidden_transpose_dim_idx = 2;

    constexpr int seqlen_q_dim_idx = 2;
    constexpr int seqlen_kv_dim_idx = 3;

    switch (matrix) {
        case NVTE_QKV_Matrix::NVTE_Q_Matrix:
            if (layout == NVTE_QKV_Layout::NVTE_QKV_INTERLEAVED) {
                strideA[hidden_dim_idx] = 1;
                strideA[seqlen_dim_idx] = 3 * h * d;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_q * 3 * h * d;
            } else {
                strideA[hidden_dim_idx] = 1;
                strideA[seqlen_dim_idx] = h * d;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_q * h * d;
            }
            break;
        case NVTE_QKV_Matrix::NVTE_K_Matrix:
            if (layout == NVTE_QKV_Layout::NVTE_QKV_INTERLEAVED) {
                strideA[seqlen_dim_idx] = 3 * h * d;
                strideA[hidden_dim_idx] = 1;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * 3 * h * d;
            } else if (layout == NVTE_QKV_Layout::NVTE_KV_INTERLEAVED) {
                strideA[seqlen_transpose_dim_idx] = 2 * h * d;
                strideA[hidden_transpose_dim_idx] = 1;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * 2 * h * d;
            } else {
                strideA[seqlen_transpose_dim_idx] = h * d;
                strideA[hidden_transpose_dim_idx] = 1;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * h * d;
            }
            break;
        case NVTE_QKV_Matrix::NVTE_K_Matrix_Transpose:
            if (layout == NVTE_QKV_Layout::NVTE_QKV_INTERLEAVED) {
                strideA[seqlen_transpose_dim_idx] = 3 * h * d;
                strideA[hidden_transpose_dim_idx] = 1;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * 3 * h * d;
            } else if (layout == NVTE_QKV_Layout::NVTE_KV_INTERLEAVED) {
                strideA[seqlen_transpose_dim_idx] = 2 * h * d;
                strideA[hidden_transpose_dim_idx] = 1;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * 2 * h * d;
            } else {
                strideA[seqlen_transpose_dim_idx] = h * d;
                strideA[hidden_transpose_dim_idx] = 1;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * h * d;
            }
            break;
        case NVTE_QKV_Matrix::NVTE_V_Matrix:
            if (layout == NVTE_QKV_Layout::NVTE_QKV_INTERLEAVED) {
                strideA[hidden_dim_idx] = 1;
                strideA[seqlen_dim_idx] = 3 * h * d;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * 3 * h * d;
            } else if (layout == NVTE_QKV_Layout::NVTE_KV_INTERLEAVED) {
                strideA[hidden_dim_idx] = 1;
                strideA[seqlen_dim_idx] = 2* h * d;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * 2 * h * d;
            } else {
                strideA[hidden_dim_idx] = 1;
                strideA[seqlen_dim_idx] = h * d;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * h * d;
            }
            break;
        case NVTE_QKV_Matrix::NVTE_V_Matrix_Transpose:
            if (layout == NVTE_QKV_Layout::NVTE_QKV_INTERLEAVED) {
                    strideA[hidden_transpose_dim_idx] = 1;
                    strideA[seqlen_transpose_dim_idx] = 3 * h * d;
                    strideA[head_dim_idx] = d;
                    strideA[batch_dim_idx] = s_kv * 3 * h * d;
                } else if (layout == NVTE_QKV_Layout::NVTE_KV_INTERLEAVED) {
                    strideA[hidden_transpose_dim_idx] = 1;
                    strideA[seqlen_transpose_dim_idx] = 2* h * d;
                    strideA[head_dim_idx] = d;
                    strideA[batch_dim_idx] = s_kv * 2 * h * d;
                } else {
                    strideA[hidden_transpose_dim_idx] = 1;
                    strideA[seqlen_transpose_dim_idx] = h * d;
                    strideA[head_dim_idx] = d;
                    strideA[batch_dim_idx] = s_kv * h * d;
                }
            break;
        case NVTE_QKV_Matrix::NVTE_S_Matrix:
            strideA[seqlen_kv_dim_idx] = 1;
            strideA[seqlen_q_dim_idx] = s_kv;
            strideA[head_dim_idx] = s_q * s_kv;
            strideA[batch_dim_idx] = h * s_q * s_kv;
            break;
        case NVTE_QKV_Matrix::NVTE_O_Matrix:
            strideA[seqlen_kv_dim_idx] = 1;
            strideA[seqlen_q_dim_idx] = h * d;
            strideA[head_dim_idx] = d;
            strideA[batch_dim_idx] = s_q * h * d;
            break;
    }
}

// convert cu_seqlens_q to qkv/o_ragged_offset and actual_seqlens_q
__global__ void cu_seqlens_to_offsets(size_t b, size_t h, size_t d,
                int32_t *cu_seqlens_q, int32_t *actual_seqlens_q,
                int32_t *qkv_ragged_offset, int32_t *o_ragged_offset) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < b) {
    actual_seqlens_q[tid] = cu_seqlens_q[tid + 1] - cu_seqlens_q[tid];
  }
  if (tid < b + 1) {
    qkv_ragged_offset[tid] = cu_seqlens_q[tid] * 3 * h * d;
    o_ragged_offset[tid] = cu_seqlens_q[tid] * h * d;
  }
}
}  // namespace fused_attn

// get cuDNN data type
hipdnnDataType_t get_cudnn_dtype(const transformer_engine::DType t) {
  using namespace transformer_engine;
  switch (t) {
    case DType::kFloat16:
      return HIPDNN_DATA_HALF;
    case DType::kFloat32:
      return HIPDNN_DATA_FLOAT;
    case DType::kBFloat16:
      return CUDNN_DATA_BFLOAT16;
    case DType::kFloat8E4M3:
      return CUDNN_DATA_FP8_E4M3;
    case DType::kFloat8E5M2:
      return CUDNN_DATA_FP8_E5M2;
    default:
      NVTE_ERROR("Invalid cuDNN data type. \n");
  }
}
}  // namespace transformer_engine
