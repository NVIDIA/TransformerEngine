#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "../cudnn_utils.h"
#include "thd_utils.h"

namespace transformer_engine {
namespace fused_attn {

__global__ void thd_partition_indices_kernel(int *output, int *cu_seqlens, int batch,
                                             int total_tokens, int world_size, int rank) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    int seqlen = cu_seqlens[i];
    // Currently we assume that each sequence length is divisible by (world_size*2) since we have
    // to distribute each sequence evenly to different GPUs.
    assert(seqlen % (world_size * 2) == 0);
    cu_seqlens_s[i] = seqlen / world_size;
  }
  __syncthreads();

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;

  for (int token_id = tid; token_id < total_tokens / world_size; token_id += num_threads) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
    int seq_len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];
    int index = token_id - cu_seqlens_s[seq_id];
    int offset = index < seq_len / 2 ? rank : (world_size - 1) * 2 - rank;
    index += cu_seqlens_s[seq_id] * world_size + seq_len / 2 * offset;
    output[token_id] = index;
  }
}

__global__ void thd_read_half_tensor_kernel(void *half, void *tensor, int *cu_seqlens, int batch,
                                            int hidden_size_in_bytes, int half_idx,
                                            int dim_size_of_token) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / 2;
  }
  __syncthreads();

  int warpid = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  int laneid = threadIdx.x % 32;
  int num_warps = (blockDim.x * gridDim.x) / 32;
  int num_total_tokens = cu_seqlens_s[batch];
  int num_float4s_per_token = hidden_size_in_bytes / sizeof(float4);

  size_t offset = static_cast<size_t>(dim_size_of_token) * hidden_size_in_bytes;
  half = reinterpret_cast<void *>(reinterpret_cast<char *>(half) + offset / 2 * blockIdx.y);
  tensor = reinterpret_cast<void *>(reinterpret_cast<char *>(tensor) + offset * blockIdx.y);

  for (int token_id = warpid; token_id < num_total_tokens; token_id += num_warps) {
    int seqid = binary_search(token_id, cu_seqlens_s, batch + 1);

    size_t offset_in_bytes = static_cast<size_t>(token_id) * hidden_size_in_bytes;
    float4 *cur_half_token =
        reinterpret_cast<float4 *>(reinterpret_cast<char *>(half) + offset_in_bytes);

    offset_in_bytes =
        (static_cast<size_t>(token_id) + cu_seqlens_s[seqid + half_idx]) * hidden_size_in_bytes;
    float4 *cur_token =
        reinterpret_cast<float4 *>(reinterpret_cast<char *>(tensor) + offset_in_bytes);

    for (int idx = laneid; idx < num_float4s_per_token; idx += 32) {
      cur_half_token[idx] = cur_token[idx];
    }
  }
}

}  // namespace fused_attn
}  // namespace transformer_engine
