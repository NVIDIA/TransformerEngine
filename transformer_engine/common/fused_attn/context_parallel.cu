#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

#include "../common.h"
#include "transformer_engine/fused_attn.h"

namespace transformer_engine {
namespace context_parallel {

struct LseCorrectionFunctor {
  __forceinline__ __device__ static void run(float *lse, float *half_lse, size_t idx,
                                             size_t half_idx) {
    float val = lse[idx];
    float val_per_step = half_lse[half_idx];
    float max_scale = max(val, val_per_step);
    float min_scale = min(val, val_per_step);
    lse[idx] = max_scale + log1pf(expf(min_scale - max_scale));
  }
};

struct ReadLseFunctor {
  __forceinline__ __device__ static void run(float *lse, float *half_lse, size_t idx,
                                             size_t half_idx) {
    half_lse[half_idx] = lse[idx];
  }
};

struct EmptyFunctor {
  __forceinline__ __device__ static void run(void *token, void *token_per_step, int idx) {}
};

struct CopyFunctor {
  __forceinline__ __device__ static void run(void *token, void *token_per_step, int idx) {
    reinterpret_cast<float4 *>(token)[idx] = reinterpret_cast<float4 *>(token_per_step)[idx];
  }
};

template <typename dtype>
struct AddFunctor {
  __forceinline__ __device__ static void run(dtype *token, dtype *token_per_step, int idx) {
    float4 d_ = reinterpret_cast<float4 *>(token)[idx];
    dtype *p_ = reinterpret_cast<dtype *>(&d_);

    float4 d = reinterpret_cast<float4 *>(token_per_step)[idx];
    dtype *p = reinterpret_cast<dtype *>(&d);

#pragma unroll
    for (int i = 0; i < sizeof(float4) / sizeof(dtype); i++) {
      p_[i] = p_[i] + p[i];
    }

    reinterpret_cast<float4 *>(token)[idx] = d_;
  }
};

/***************************************************************************************************
 * Support THD format for Context Parallel: Binary search an array for a target value
 **************************************************************************************************/

__forceinline__ __device__ int binary_search(int target, int *array, int len) {
  int left = 1, right = len - 1;
  while (left < right) {
    int mid = (left + right) / 2;
    if (array[mid] <= target) {
      left = mid + 1;
    } else {
      right = mid;
    }
  }
  return left - 1;
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Generate partitioned indices for input tokens
 **************************************************************************************************/
__global__ void thd_partition_indices_kernel(int *output, int *cu_seqlens, int batch,
                                             int total_tokens, int world_size, int rank) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    int seqlen = cu_seqlens[i];
    // Currently we assume that each sequence length is divisible by (world_size*2) since we have
    // to distribute each sequence evenly to different GPUs.
    assert(seqlen % (world_size * 2) == 0);
    cu_seqlens_s[i] = seqlen / world_size;
  }
  __syncthreads();

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;

  for (int token_id = tid; token_id < total_tokens / world_size; token_id += num_threads) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
    int seq_len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];
    int index = token_id - cu_seqlens_s[seq_id];
    int offset = index < seq_len / 2 ? rank : (world_size - 1) * 2 - rank;
    index += cu_seqlens_s[seq_id] * world_size + seq_len / 2 * offset;
    output[token_id] = index;
  }
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Read the half of a THD tensor
 **************************************************************************************************/

__global__ void thd_read_half_tensor_kernel(void *half, void *tensor, int *cu_seqlens, int batch,
                                            int hidden_size_in_bytes, int half_idx,
                                            int dim_size_of_token) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / 2;
  }
  __syncthreads();

  int warpid = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  int laneid = threadIdx.x % 32;
  int num_warps = (blockDim.x * gridDim.x) / 32;
  int num_total_tokens = cu_seqlens_s[batch];
  int num_float4s_per_token = hidden_size_in_bytes / sizeof(float4);

  size_t offset = static_cast<size_t>(dim_size_of_token) * hidden_size_in_bytes;
  half = reinterpret_cast<void *>(reinterpret_cast<char *>(half) + offset / 2 * blockIdx.y);
  tensor = reinterpret_cast<void *>(reinterpret_cast<char *>(tensor) + offset * blockIdx.y);

  for (int token_id = warpid; token_id < num_total_tokens; token_id += num_warps) {
    int seqid = binary_search(token_id, cu_seqlens_s, batch + 1);

    size_t offset_in_bytes = static_cast<size_t>(token_id) * hidden_size_in_bytes;
    float4 *cur_half_token =
        reinterpret_cast<float4 *>(reinterpret_cast<char *>(half) + offset_in_bytes);

    offset_in_bytes =
        (static_cast<size_t>(token_id) + cu_seqlens_s[seqid + half_idx]) * hidden_size_in_bytes;
    float4 *cur_token =
        reinterpret_cast<float4 *>(reinterpret_cast<char *>(tensor) + offset_in_bytes);

    for (int idx = laneid; idx < num_float4s_per_token; idx += 32) {
      cur_half_token[idx] = cur_token[idx];
    }
  }
}

/***************************************************************************************************
 * Support THD format for Context Parallel: softmax_lse related operations
 **************************************************************************************************/

template <bool lse_packed, typename Functor>
__global__ void thd_lse_kernel(float *lse, float *half_lse, int *cu_seqlens, int batch,
                               int num_heads, int lse_seqlen, int second_half_lse_seqlen) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / 2;
  }
  __syncthreads();

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;
  int num_total_tokens = cu_seqlens_s[batch];

  for (int token_id = tid; token_id < num_total_tokens; token_id += num_threads) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
    for (int head_id = blockIdx.y; head_id < num_heads; head_id += gridDim.y) {
      size_t idx, half_idx;
      if constexpr (lse_packed) {
        idx = head_id * lse_seqlen + token_id + cu_seqlens_s[seq_id + 1];
        half_idx = head_id * second_half_lse_seqlen + token_id;
      } else {
        size_t row = static_cast<size_t>(seq_id) * num_heads + head_id;
        int col = token_id - cu_seqlens_s[seq_id];
        int seq_len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];

        idx = row * lse_seqlen + col + seq_len;
        half_idx = row * second_half_lse_seqlen + col;
      }

      Functor::run(lse, half_lse, idx, half_idx);
    }
  }
}

/***************************************************************************************************
  * Support BSHD, SBHD, and THD formats for Context Parallel: Out correction in forward
  **************************************************************************************************/

// Stores pointers to output and lse tensors for batch kernel launch.
template <int n>
struct TensorList {
  void *addresses_out[n];
  void *addresses_lse[n];
  int num_tensors_this_launch;
};

// describe QKV output tensor format for simplified computation.
struct QKVIndexCalculatorBase {
  int batch_size_, seq_len_, num_heads_, dim_per_head_, num_total_tokens_;
  int *half_cu_seqlens_;

  __forceinline__ __device__ QKVIndexCalculatorBase(int batch_size, int seq_len, int num_heads,
                                                    int dim_per_head, int num_total_tokens,
                                                    int *half_cu_seqlens)
      : batch_size_(batch_size),
        seq_len_(seq_len),
        num_heads_(num_heads),
        dim_per_head_(dim_per_head),
        num_total_tokens_(num_total_tokens),
        half_cu_seqlens_(half_cu_seqlens) {}
};

template <NVTE_QKV_Format format>
struct QKVIndexCalculator;

template <>
struct QKVIndexCalculator<NVTE_QKV_Format::NVTE_SBHD> : QKVIndexCalculatorBase {
  __forceinline__ __device__ QKVIndexCalculator(int batch_size, int seq_len, int num_heads,
                                                int dim_per_head, int num_total_tokens,
                                                int *half_cu_seqlens)
      : QKVIndexCalculatorBase(batch_size, seq_len, num_heads, dim_per_head, num_total_tokens,
                               half_cu_seqlens) {}

  // We design the thread's lowest-level traversal to follow the sequence dimension (since LSE is in BHS or HT format). This ensures coalesced memory access when reading the LSE tensor. Consequently, the SBHD layout's computation logic for seq_id and token_id remains identical to BSHD - both patterns make the underlying threads expand along the sequence dimension. This design preserves computational correctness while optimizing memory performance.
  __forceinline__ __device__ int compute_seq_id(int flat_token_id) {
    int half_seq_len = seq_len_ / 2;
    return flat_token_id / half_seq_len;
  }

  // When blockIdx.z == 1, the computation processes the second half of the sequence. Therefore, we need to apply an address offset equal to half of the seq_len_ length.
  __forceinline__ __device__ int compute_token_id(int flat_token_id, int seq_id) {
    int half_seq_len = seq_len_ / 2;
    return flat_token_id % half_seq_len + blockIdx.z * half_seq_len;
  }

  __forceinline__ __device__ int compute_full_tensor_offset(int seq_id, int token_id, int head_id) {
    int offset = token_id * batch_size_ * num_heads_ + seq_id * num_heads_ + head_id;
    offset *= dim_per_head_;
    return offset;
  }

  __forceinline__ __device__ int compute_half_tensor_offset(int seq_id, int token_id, int head_id) {
    int half_seq_len = seq_len_ / 2;
    int half_token_id = token_id - half_seq_len;
    int offset = half_token_id * batch_size_ * num_heads_ + seq_id * num_heads_ + head_id;
    offset *= dim_per_head_;
    return offset;
  }
};

template <>
struct QKVIndexCalculator<NVTE_QKV_Format::NVTE_BSHD> : QKVIndexCalculatorBase {
  __forceinline__ __device__ QKVIndexCalculator(int batch_size, int seq_len, int num_heads,
                                                int dim_per_head, int num_total_tokens,
                                                int *half_cu_seqlens)
      : QKVIndexCalculatorBase(batch_size, seq_len, num_heads, dim_per_head, num_total_tokens,
                               half_cu_seqlens) {}

  __forceinline__ __device__ int compute_seq_id(int flat_token_id) {
    int half_seq_len = seq_len_ / 2;
    return flat_token_id / half_seq_len;
  }

  __forceinline__ __device__ int compute_token_id(int flat_token_id, int seq_id) {
    int half_seq_len = seq_len_ / 2;
    return flat_token_id % half_seq_len + blockIdx.z * half_seq_len;
  }

  __forceinline__ __device__ int compute_full_tensor_offset(int seq_id, int token_id, int head_id) {
    int offset = seq_id * seq_len_ * num_heads_ + token_id * num_heads_ + head_id;
    offset *= dim_per_head_;
    return offset;
  }

  __forceinline__ __device__ int compute_half_tensor_offset(int seq_id, int token_id, int head_id) {
    int half_seq_len = seq_len_ / 2;
    int half_token_id = token_id - half_seq_len;
    int offset = seq_id * half_seq_len * num_heads_ + half_token_id * num_heads_ + head_id;
    offset *= dim_per_head_;
    return offset;
  }
};

template <>
struct QKVIndexCalculator<NVTE_QKV_Format::NVTE_THD> : QKVIndexCalculatorBase {
  __forceinline__ __device__ QKVIndexCalculator(int batch_size, int seq_len, int num_heads,
                                                int dim_per_head, int num_total_tokens,
                                                int *half_cu_seqlens)
      : QKVIndexCalculatorBase(batch_size, seq_len, num_heads, dim_per_head, num_total_tokens,
                               half_cu_seqlens) {}

  __forceinline__ __device__ int compute_seq_id(int flat_token_id) {
    return binary_search(flat_token_id, half_cu_seqlens_, batch_size_ + 1);
  }

  __forceinline__ __device__ int compute_token_id(int flat_token_id, int seq_id) {
    bool is_padding = (flat_token_id >= half_cu_seqlens_[batch_size_]);
    int half_seq_len = is_padding ? (num_total_tokens_ / 2 - half_cu_seqlens_[batch_size_])
                                  : (half_cu_seqlens_[seq_id + 1] - half_cu_seqlens_[seq_id]);

    return flat_token_id - half_cu_seqlens_[seq_id] + blockIdx.z * half_seq_len;
  }

  __forceinline__ __device__ int compute_full_tensor_offset(int seq_id, int token_id, int head_id) {
    int flat_token_id = half_cu_seqlens_[seq_id] * 2 + token_id;
    int offset = flat_token_id * num_heads_ + head_id;
    offset *= dim_per_head_;
    return offset;
  }

  __forceinline__ __device__ int compute_half_tensor_offset(int seq_id, int token_id, int head_id) {
    int flat_token_id = half_cu_seqlens_[seq_id] * 2 + token_id;
    int half_flat_token_id = flat_token_id - half_cu_seqlens_[seq_id + 1];
    int offset = half_flat_token_id * num_heads_ + head_id;
    offset *= dim_per_head_;
    return offset;
  }
};

// describe lse tensor format for simplified computation.
struct LseIndexCalculatorBase {
  int batch_size_, seq_len_, num_heads_, num_total_tokens_;
  int *half_cu_seqlens_;

  __forceinline__ __device__ LseIndexCalculatorBase(int batch_size, int seq_len, int num_heads,
                                                    int num_total_tokens, int *half_cu_seqlens)
      : batch_size_(batch_size),
        seq_len_(seq_len),
        num_heads_(num_heads),
        num_total_tokens_(num_total_tokens),
        half_cu_seqlens_(half_cu_seqlens) {}
};

template <NVTE_QKV_Format out_format, bool softmax_lse_in_packed_format>
struct LseIndexCalculator : LseIndexCalculatorBase {
  /// When the pack format is not employed, the shape of lse is BHS
  __forceinline__ __device__ LseIndexCalculator(int batch_size, int seq_len, int num_heads,
                                                int num_total_tokens, int *half_cu_seqlens)
      : LseIndexCalculatorBase(batch_size, seq_len, num_heads, num_total_tokens, half_cu_seqlens) {}

  __forceinline__ __device__ int compute_full_tensor_offset(int seq_id, int token_id, int head_id) {
    int offset = seq_id * num_heads_ * seq_len_ + head_id * seq_len_ + token_id;
    return offset;
  }
  __forceinline__ __device__ int compute_half_tensor_offset(int seq_id, int token_id, int head_id) {
    int half_seq_len = seq_len_ / 2;
    int half_token_id;
    if constexpr (out_format == NVTE_QKV_Format::NVTE_THD) {
      half_token_id = token_id - (half_cu_seqlens_[seq_id + 1] - half_cu_seqlens_[seq_id]);
    } else {
      half_token_id = token_id - half_seq_len;
    }
    int offset = seq_id * num_heads_ * half_seq_len + head_id * half_seq_len + half_token_id;
    return offset;
  }
};

template <>
struct LseIndexCalculator<NVTE_QKV_Format::NVTE_THD, true> : LseIndexCalculatorBase {
  /// When the pack format is employed, the shape of lse is HT
  __forceinline__ __device__ LseIndexCalculator(int BatchSize, int SeqLen, int NumHeads,
                                                int NumTotalTokens, int *CuSeqlens)
      : LseIndexCalculatorBase(BatchSize, SeqLen, NumHeads, NumTotalTokens, CuSeqlens) {}

  __forceinline__ __device__ int compute_full_tensor_offset(int seq_id, int token_id, int head_id) {
    int flat_token_id = token_id + half_cu_seqlens_[seq_id] * 2;
    int offset = head_id * num_total_tokens_ + flat_token_id;
    return offset;
  }
  __forceinline__ __device__ int compute_half_tensor_offset(int seq_id, int token_id, int head_id) {
    int flat_token_id = token_id + half_cu_seqlens_[seq_id] * 2;
    int half_flat_token_id = flat_token_id - half_cu_seqlens_[seq_id + 1];
    int num_half_total_tokens = num_total_tokens_ / 2;
    int offset = head_id * num_half_total_tokens + half_flat_token_id;
    return offset;
  }
};

template <typename dtype, int tile_size, bool causal, NVTE_QKV_Format out_format,
          bool softmax_lse_in_packed_format, int max_tensors, int block>
__global__ void fused_out_correction_kernel(dtype *out, TensorList<max_tensors> tensors, float *lse,
                                            int *cu_seqlens, int batch, int num_heads,
                                            int dim_per_head, int lse_seqlen, int num_total_tokens,
                                            int cp_size, int rank, int start) {
  extern __shared__ int cu_seqlens_s[];

  constexpr int num_lse_per_block = block / tile_size;

  // Preload all lse elements needed by this block into shared memory, must first determine the memory addresses of LSE elements.
  __shared__ int lse_half_idx[num_lse_per_block];
  __shared__ int lse_full_idx[num_lse_per_block];

  // Cache lse values in shared memory
  __shared__ float lse_temp[num_lse_per_block];

  // Cache lse_per_step values in shared memory
  __shared__ float lse_temp_per_step[num_lse_per_block * max_tensors];

  int full_compute_tensor_end;
  int num_total_valid_tokens;  // Number of total tokens actually involved in the computation

  if constexpr (out_format == NVTE_QKV_Format::NVTE_THD) {
    for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
      cu_seqlens_s[i] = cu_seqlens[i] / 2;
    }
    __syncthreads();
    num_total_valid_tokens = cu_seqlens_s[batch] * 2;
  } else if constexpr (out_format == NVTE_QKV_Format::NVTE_SBHD ||
                       out_format == NVTE_QKV_Format::NVTE_BSHD) {
    num_total_valid_tokens = lse_seqlen * batch;
  }

  // Last tensor index for full tensor computation in this round.
  if constexpr (causal) {
    full_compute_tensor_end = min(start + tensors.num_tensors_this_launch, max(rank + 1, start));
  } else {
    full_compute_tensor_end = start + tensors.num_tensors_this_launch;
  }

  // It's necessary to handle out and lse differently because their formats maybe different.
  QKVIndexCalculator<out_format> out_calculator(batch, lse_seqlen, num_heads, dim_per_head,
                                                num_total_tokens, cu_seqlens_s);
  LseIndexCalculator<out_format, softmax_lse_in_packed_format> lse_calculator(
      batch, lse_seqlen, num_heads, num_total_tokens, cu_seqlens_s);

  int tile_id = (blockIdx.x * blockDim.x + threadIdx.x) / tile_size;
  int lane_id = threadIdx.x % tile_size;
  int num_tiles = (blockDim.x * gridDim.x) / tile_size;
  int num_loops_per_head = dim_per_head * sizeof(dtype) / sizeof(float4);

  for (int token_id = tile_id; token_id < num_total_tokens / 2; token_id += num_tiles) {
    int seq_id = out_calculator.compute_seq_id(token_id);
    int local_token_id = out_calculator.compute_token_id(token_id, seq_id);
    int head_id = blockIdx.y;

    size_t idx_out_full =
        out_calculator.compute_full_tensor_offset(seq_id, local_token_id, head_id);
    size_t idx_lse_full =
        lse_calculator.compute_full_tensor_offset(seq_id, local_token_id, head_id);

    size_t idx_out_half, idx_lse_half;

    // start and end define the range of tensors to compute.
    int end = full_compute_tensor_end;
    bool is_second_half = (blockIdx.z == 1);
    if (start + tensors.num_tensors_this_launch > full_compute_tensor_end && is_second_half) {
      // If the half part needs to be computed, end must be reassigned.
      end = start + tensors.num_tensors_this_launch;
      idx_out_half = out_calculator.compute_half_tensor_offset(seq_id, local_token_id, head_id);
      idx_lse_half = lse_calculator.compute_half_tensor_offset(seq_id, local_token_id, head_id);
    }

    if (lane_id == 0) {
      lse_half_idx[threadIdx.x / tile_size] = idx_lse_half;
      lse_full_idx[threadIdx.x / tile_size] = idx_lse_full;
    }

    dtype *cur_out = out + idx_out_full;
    if (token_id >= num_total_valid_tokens / 2) {
      // padding zeros
      for (int j = lane_id; j < num_loops_per_head; j += tile_size) {
        float4 data = {0.0f, 0.0f, 0.0f, 0.0f};
        reinterpret_cast<float4 *>(cur_out)[j] = data;
      }
      continue;
    }

    __syncthreads();

    /// load lse and lse_per_step into shared memory
    for (int i = threadIdx.x; i < num_lse_per_block; i += blockDim.x) {
      lse_temp[i] = lse[lse_full_idx[i]];
    }

    for (int i = threadIdx.x; i < (end - start) * num_lse_per_block; i += blockDim.x) {
      int tensor_id = start + threadIdx.x / num_lse_per_block;
      if (causal && is_second_half && tensor_id > rank) {
        lse_temp_per_step[i] = reinterpret_cast<float *>(
            tensors.addresses_lse[tensor_id])[lse_half_idx[i % num_lse_per_block]];
      } else {
        lse_temp_per_step[i] = reinterpret_cast<float *>(
            tensors.addresses_lse[tensor_id])[lse_full_idx[i % num_lse_per_block]];
      }
    }

    __syncthreads();

    for (int j = lane_id; j < num_loops_per_head; j += tile_size) {
      float4 data;
      if (start == 0) {
        data = {0.0f, 0.0f, 0.0f, 0.0f};
      } else {
        data = reinterpret_cast<float4 *>(cur_out)[j];
      }

      dtype *p = reinterpret_cast<dtype *>(&data);

      for (int i = start; i < end; i++) {
        size_t idx_out;
        size_t idx_lse;
        if (causal && is_second_half && i > rank) {
          idx_out = idx_out_half;
          idx_lse = idx_lse_half;
        } else {
          idx_out = idx_out_full;
          idx_lse = idx_lse_full;
        }
        dtype *cur_out_per_step = reinterpret_cast<dtype *>(tensors.addresses_out[i]) + idx_out;
        float4 data_per_step = reinterpret_cast<float4 *>(cur_out_per_step)[j];
        float lse_corrected_exp =
            exp(lse_temp_per_step[(i - start) * num_lse_per_block + threadIdx.x / tile_size] -
                lse_temp[threadIdx.x / tile_size]);
        dtype *p_per_step = reinterpret_cast<dtype *>(&data_per_step);
        for (int k = 0; k < sizeof(float4) / sizeof(dtype); k++) {
          p[k] += (p_per_step[k] == static_cast<dtype>(0)
                       ? 0
                       : static_cast<float>(p_per_step[k]) * lse_corrected_exp);
        }
      }
      reinterpret_cast<float4 *>(cur_out)[j] = data;
    }
  }
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Gradients correction in backward
 **************************************************************************************************/

template <typename dtype, typename Functor_0, typename Functor_1, int functor_idx, int group_size>
__global__ void thd_grad_correction_kernel(dtype *grad, dtype *grad_per_step, int *cu_seqlens,
                                           int batch, int hidden_size, int dim_size_of_token) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    if constexpr (functor_idx < 2) {
      cu_seqlens_s[i] = cu_seqlens[i] / 2;
    } else {
      cu_seqlens_s[i] = cu_seqlens[i];
    }
  }
  __syncthreads();

  int group_id = (blockIdx.x * blockDim.x + threadIdx.x) / group_size;
  int lane_id = threadIdx.x % group_size;
  int num_groups = (blockDim.x * gridDim.x) / group_size;
  int num_total_tokens = cu_seqlens_s[batch];
  int num_inner_loops = hidden_size * sizeof(dtype) / sizeof(float4);

  size_t offset = static_cast<size_t>(dim_size_of_token) * hidden_size;
  if constexpr (functor_idx < 2) {
    grad_per_step = grad_per_step + offset / 2 * blockIdx.y;
  } else {
    grad_per_step = grad_per_step + offset * blockIdx.y;
  }
  grad = grad + offset * blockIdx.y;

  for (int token_id = group_id; token_id < num_total_tokens; token_id += num_groups) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);

    int token_offset;
    bool is_first_half;
    if constexpr (functor_idx < 2) {
      token_offset = cu_seqlens_s[seq_id + functor_idx];
      is_first_half = (functor_idx == 0);
    } else {
      token_offset = 0;
      int len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];
      is_first_half = (token_id - cu_seqlens_s[seq_id]) < (len / 2);
    }

    dtype *token = &grad[(token_id + token_offset) * static_cast<size_t>(hidden_size)];
    dtype *token_per_step = &grad_per_step[token_id * static_cast<size_t>(hidden_size)];
    for (int idx = lane_id; idx < num_inner_loops; idx += group_size) {
      if (is_first_half) {
        Functor_0::run(token, token_per_step, idx);
      } else {
        Functor_1::run(token, token_per_step, idx);
      }
    }
  }
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Read the half of a THD tensor
 **************************************************************************************************/

void thd_read_half_tensor(const Tensor &tensor, const Tensor &cu_seqlens, Tensor &half,
                          int half_idx, hipStream_t stream) {
  using namespace transformer_engine;
  NVTE_CHECK(tensor.dim() == 3 || tensor.dim() == 4);
  NVTE_CHECK(cu_seqlens.dtype() == DType::kInt32);

  auto cu_seqlens_shape = cu_seqlens.shape();
  auto tensor_shape = tensor.shape();

  NVTE_CHECK(cu_seqlens.dim() == 1);
  NVTE_CHECK(cu_seqlens_shape[0] >= 2);

  // Shapes of q and dq are [t, h, d], so the dimension of "t" is 0
  // Shapes of kv and dkv are [2, t, h, d], so the dimension of "t" is 1
  int seq_dim = tensor.dim() == 3 ? 0 : 1;

  int batch = cu_seqlens_shape[0] - 1;
  int num_heads = tensor_shape[seq_dim + 1];
  int dim_per_head = tensor_shape[seq_dim + 2];
  int hidden_size_in_bytes = (num_heads * dim_per_head * typeToNumBits(tensor.dtype())) / 8;

  // For 128-bits load/store
  NVTE_CHECK(hidden_size_in_bytes % 16 == 0);

  // Launch Kernel
  constexpr unsigned int block = 256;
  unsigned int grid_x = (tensor_shape[seq_dim] / 2 * 32 + block - 1) / block;
  unsigned int grid_y = 1;
  for (int i = 0; i < seq_dim; i++) {
    grid_y *= tensor_shape[i];
  }
  dim3 grid = {grid_x, grid_y};
  thd_read_half_tensor_kernel<<<grid, block, sizeof(int) * (batch + 1), stream>>>(
      half.data.dptr, tensor.data.dptr, reinterpret_cast<int *>(cu_seqlens.data.dptr), batch,
      hidden_size_in_bytes, half_idx, tensor_shape[seq_dim]);
}

/***************************************************************************************************
 * Support THD format for Context Parallel: softmax_lse related operations
 **************************************************************************************************/

void thd_second_half_lse_correction(Tensor lse, const Tensor &lse_per_step,
                                    const Tensor &cu_seqlens, bool lse_packed,
                                    hipStream_t stream) {
  using namespace transformer_engine;
  NVTE_CHECK(lse.dtype() == DType::kFloat32);
  NVTE_CHECK(lse_per_step.dtype() == DType::kFloat32);
  NVTE_CHECK(cu_seqlens.dtype() == DType::kInt32);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  int batch, num_heads, lse_seqlen, second_half_lse_seqlen;
  auto cu_seqlens_shape = cu_seqlens.shape();
  auto lse_shape = lse.shape();
  auto lse_per_step_shape = lse_per_step.shape();

  if (lse_packed) {
    NVTE_CHECK(lse.dim() == 2);
    NVTE_CHECK(lse_per_step.dim() == 2);

    batch = cu_seqlens_shape[0] - 1;
    num_heads = lse_shape[0];
    lse_seqlen = lse_shape[1];
    second_half_lse_seqlen = lse_per_step_shape[1];

    NVTE_CHECK(lse_per_step_shape[0] == num_heads);
    NVTE_CHECK(second_half_lse_seqlen >= lse_seqlen / 2);
  } else {
    NVTE_CHECK(lse.dim() == 3);
    NVTE_CHECK(lse_per_step.dim() == 3);

    batch = lse_shape[0];
    num_heads = lse_shape[1];
    lse_seqlen = lse_shape[2];
    second_half_lse_seqlen = lse_per_step_shape[2];

    NVTE_CHECK(lse_per_step_shape[0] == batch);
    NVTE_CHECK(lse_per_step_shape[1] == num_heads);
    NVTE_CHECK(second_half_lse_seqlen == lse_seqlen / 2);
    NVTE_CHECK(cu_seqlens_shape[0] == batch + 1);
  }

  constexpr unsigned int block = 256;
  unsigned int grid_x = (lse_seqlen / 2 + block - 1) / block;
  unsigned int grid_y = num_heads;
  dim3 grid = {grid_x, grid_y};

  if (lse_packed) {
    thd_lse_kernel<true, LseCorrectionFunctor><<<grid, block, sizeof(int) * (batch + 1), stream>>>(
        reinterpret_cast<float *>(lse.data.dptr), reinterpret_cast<float *>(lse_per_step.data.dptr),
        reinterpret_cast<int *>(cu_seqlens.data.dptr), batch, num_heads, lse_seqlen,
        second_half_lse_seqlen);
  } else {
    thd_lse_kernel<false, LseCorrectionFunctor><<<grid, block, sizeof(int) * (batch + 1), stream>>>(
        reinterpret_cast<float *>(lse.data.dptr), reinterpret_cast<float *>(lse_per_step.data.dptr),
        reinterpret_cast<int *>(cu_seqlens.data.dptr), batch, num_heads, lse_seqlen,
        second_half_lse_seqlen);
  }
}

void thd_read_second_half_lse(const Tensor &lse, const Tensor &cu_seqlens, Tensor &half_lse,
                              bool lse_packed, int second_half_lse_seqlen, hipStream_t stream) {
  using namespace transformer_engine;
  NVTE_CHECK(lse.dtype() == DType::kFloat32);
  NVTE_CHECK(cu_seqlens.dtype() == DType::kInt32);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  int batch, num_heads, lse_seqlen;

  auto cu_seqlens_shape = cu_seqlens.shape();
  auto lse_shape = lse.shape();

  if (lse_packed) {
    NVTE_CHECK(lse.dim() == 2);

    batch = cu_seqlens_shape[0] - 1;
    num_heads = lse_shape[0];
    lse_seqlen = lse_shape[1];

    NVTE_CHECK(second_half_lse_seqlen >= lse_seqlen / 2);
  } else {
    NVTE_CHECK(lse.dim() == 3);

    batch = lse_shape[0];
    num_heads = lse_shape[1];
    lse_seqlen = lse_shape[2];

    NVTE_CHECK(cu_seqlens_shape[0] == batch + 1);
    NVTE_CHECK(second_half_lse_seqlen == lse_seqlen / 2);
  }

  constexpr unsigned int block = 256;
  unsigned int grid_x = (lse_seqlen / 2 + block - 1) / block;
  unsigned int grid_y = num_heads;
  dim3 grid = {grid_x, grid_y};

  if (lse_packed) {
    thd_lse_kernel<true, ReadLseFunctor><<<grid, block, sizeof(int) * (batch + 1), stream>>>(
        reinterpret_cast<float *>(lse.data.dptr), reinterpret_cast<float *>(half_lse.data.dptr),
        reinterpret_cast<int *>(cu_seqlens.data.dptr), batch, num_heads, lse_seqlen,
        second_half_lse_seqlen);
  } else {
    thd_lse_kernel<false, ReadLseFunctor><<<grid, block, sizeof(int) * (batch + 1), stream>>>(
        reinterpret_cast<float *>(lse.data.dptr), reinterpret_cast<float *>(half_lse.data.dptr),
        reinterpret_cast<int *>(cu_seqlens.data.dptr), batch, num_heads, lse_seqlen,
        second_half_lse_seqlen);
  }
}

/***************************************************************************************************
  * Support BSHD, SBHD, and THD formats for Context Parallel: Fused out correction in forward
  **************************************************************************************************/

#define DISPATCH_SBHD_BSHD_AND_THD(TYPE, LEVEL, NAME, ...)                   \
  switch (TYPE) {                                                            \
    case NVTE_QKV_Format::NVTE_SBHD: {                                       \
      constexpr NVTE_QKV_Format LEVEL = NVTE_QKV_Format::NVTE_SBHD;          \
      __VA_ARGS__;                                                           \
      break;                                                                 \
    }                                                                        \
    case NVTE_QKV_Format::NVTE_BSHD: {                                       \
      constexpr NVTE_QKV_Format LEVEL = NVTE_QKV_Format::NVTE_BSHD;          \
      __VA_ARGS__;                                                           \
      break;                                                                 \
    }                                                                        \
    case NVTE_QKV_Format::NVTE_THD: {                                        \
      constexpr NVTE_QKV_Format LEVEL = NVTE_QKV_Format::NVTE_THD;           \
      __VA_ARGS__;                                                           \
      break;                                                                 \
    }                                                                        \
    default:                                                                 \
      NVTE_ERROR("only implemented for NVTE_THD, NVTE_BSHD and NVTE_SBHD "); \
  }

// #define DISPATCH_FLOAT_HALF_AND_BFLOAT(TYPE, LEVEL, NAME, ...)        \
//   switch (TYPE) {                                                     \
//     case DType::kFloat32: {                                           \
//       using scalar_t_##LEVEL = float;                                 \
//       __VA_ARGS__;                                                    \
//       break;                                                          \
//     }                                                                 \
//     case DType::kFloat16: {                                           \
//       using scalar_t_##LEVEL = half;                                  \
//       __VA_ARGS__;                                                    \
//       break;                                                          \
//     }                                                                 \
//     case DType::kBFloat16: {                                          \
//       \ using scalar_t_##LEVEL = DType::kBFloat16;                    \
//       __VA_ARGS__;                                                    \
//       break;                                                          \
//     }                                                                 \
//     default:                                                          \
//       AT_ERROR(#NAME, " not implemented for '", toString(TYPE), "'"); \
//   }

template <typename dtype, bool causal>
void fused_out_correction_helper(Tensor &out, const NVTETensorPack *out_per_step, const Tensor &lse,
                                 const NVTETensorPack *lse_per_step, const Tensor &cu_seqlens,
                                 NVTE_QKV_Format qkv_format, int cp_size, int rank,
                                 bool softmax_lse_in_packed_format, hipStream_t stream) {
  int lse_seqlen;
  int batch;
  int num_heads;
  int dim_per_head;
  int total_tokens;
  int cu_seqlens_size = 0;
  int *cu_seqlens_ptr = nullptr;

  auto out_shape = out.shape();

  if (qkv_format == NVTE_QKV_Format::NVTE_SBHD) {
    lse_seqlen = out_shape[0];
    batch = out_shape[1];
    num_heads = out_shape[2];
    dim_per_head = out_shape[3];
    total_tokens = lse_seqlen * batch;
  } else if (qkv_format == NVTE_QKV_Format::NVTE_BSHD) {
    lse_seqlen = out_shape[1];
    batch = out_shape[0];
    num_heads = out_shape[2];
    dim_per_head = out_shape[3];
    total_tokens = lse_seqlen * batch;
  } else if (qkv_format == NVTE_QKV_Format::NVTE_THD) {
    total_tokens = out_shape[0];
    num_heads = out_shape[1];
    dim_per_head = out_shape[2];
    auto cu_seqlens_shape = cu_seqlens.shape();
    batch = cu_seqlens_shape[0] - 1;
    cu_seqlens_size = sizeof(int) * (batch + 1);
    cu_seqlens_ptr = reinterpret_cast<int *>(cu_seqlens.data.dptr);
    if (softmax_lse_in_packed_format) {
      lse_seqlen = total_tokens;
    } else {
      auto lse_shape = lse.shape();
      lse_seqlen = lse_shape[2];
    }
  }
  constexpr int tile = 8;
  constexpr int block = 256;
  unsigned int grid_x;

  grid_x = (static_cast<size_t>(total_tokens) * tile / 2 + block - 1) / block;
  dim3 grid = {grid_x, (unsigned int)num_heads, 2};

  constexpr int max_tensors = 64;
  TensorList<max_tensors> tensors;

  for (int i = 0; i < cp_size; i += max_tensors) {
    int num_tensors = std::min(max_tensors, cp_size - i);
    tensors.num_tensors_this_launch = num_tensors;
    for (int j = 0; j < num_tensors; j++) {
      Tensor *out_temp = convertNVTETensorCheck(out_per_step->tensors[i + j]);
      tensors.addresses_out[j] = reinterpret_cast<dtype *>(out_temp->data.dptr);
      Tensor *lse_temp = convertNVTETensorCheck(lse_per_step->tensors[i + j]);
      tensors.addresses_lse[j] = reinterpret_cast<float *>(lse_temp->data.dptr);
    }

    NVTE_CHECK(!(softmax_lse_in_packed_format == true && qkv_format != NVTE_QKV_Format::NVTE_THD),
               "Packed lse only supports THD format.");

    DISPATCH_SBHD_BSHD_AND_THD(
        qkv_format, qkv_format_type, "fused_out_correction",
        TRANSFORMER_ENGINE_SWITCH_CONDITION(
            softmax_lse_in_packed_format, bool_softmax_lse_in_packed_format,
            fused_out_correction_kernel<dtype, tile, causal, qkv_format_type,
                                        bool_softmax_lse_in_packed_format, max_tensors, block>
            <<<grid, block, cu_seqlens_size, stream>>>(
                reinterpret_cast<dtype *>(out.data.dptr), tensors,
                reinterpret_cast<float *>(lse.data.dptr), cu_seqlens_ptr, batch, num_heads,
                dim_per_head, lse_seqlen, total_tokens, cp_size, rank, i););)
  }
}

// fused out correction after qkv calculation
void fused_out_correction(Tensor &out, const NVTETensorPack *out_per_step, const Tensor &lse,
                          const NVTETensorPack *lse_per_step, Tensor &cu_seqlens,
                          NVTE_QKV_Format qkv_format, int cp_size, int rank, bool causal,
                          bool softmax_lse_in_packed_format, hipStream_t stream) {
  // in-place optimization: use out_per_step[0] as the final output to avoid extra allocation

  TRANSFORMER_ENGINE_SWITCH_CONDITION(
      causal, bool_causal,
      TRANSFORMER_ENGINE_TYPE_SWITCH_NON_FP8ONLY(
          out.dtype(), dtype,
          fused_out_correction_helper<dtype, bool_causal>(out, out_per_step, lse, lse_per_step,
                                                          cu_seqlens, qkv_format, cp_size, rank,
                                                          softmax_lse_in_packed_format, stream););)
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Gradients correction in backward
 **************************************************************************************************/

template <typename dtype, typename Functor_0, typename Functor_1, int functor_idx>
static void thd_grad_correction_helper(Tensor grad, const Tensor &grad_per_step,
                                       const Tensor &cu_seqlens, hipStream_t stream) {
  using namespace transformer_engine;
  NVTE_CHECK(grad.dim() == 3 || grad.dim() == 4);
  NVTE_CHECK(cu_seqlens.dtype() == DType::kInt32);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  auto grad_shape = grad.shape();
  auto cu_seqlens_shape = cu_seqlens.shape();
  auto grad_per_step_shape = grad_per_step.shape();

  // Shape of dq is [t, h, d], so the dimension of "t" is 0
  // Shape of dkv is [2, t, h, d], so the dimension of "t" is 1
  int seq_dim = grad.dim() == 3 ? 0 : 1;

  int total_tokens = grad_shape[seq_dim];
  int num_heads = grad_shape[seq_dim + 1];
  int dim_per_head = grad_shape[seq_dim + 2];
  int batch = cu_seqlens_shape[0] - 1;

  if constexpr (functor_idx < 2) {
    NVTE_CHECK(grad_per_step_shape[seq_dim] == total_tokens / 2);
  } else {
    NVTE_CHECK(grad_per_step_shape[seq_dim] == total_tokens);
  }
  NVTE_CHECK(grad_per_step_shape[seq_dim + 1] == num_heads);
  NVTE_CHECK(grad_per_step_shape[seq_dim + 2] == dim_per_head);

  size_t hidden_size = num_heads * dim_per_head;
  NVTE_CHECK(((hidden_size * typeToNumBits(grad.dtype())) / 8) % 16 == 0);

  constexpr unsigned int block = 256;
  unsigned int grid_x;
  if constexpr (functor_idx < 2) {
    grid_x = (total_tokens / 2 * 32 + block - 1) / block;
  } else {
    grid_x = (total_tokens * 32 + block - 1) / block;
  }
  unsigned int grid_y = 1;
  for (int i = 0; i < seq_dim; i++) {
    grid_y *= grad_shape[i];
  }
  dim3 grid = {grid_x, grid_y};

  thd_grad_correction_kernel<dtype, Functor_0, Functor_1, functor_idx, 32>
      <<<grid, block, sizeof(int) * (batch + 1), stream>>>(
          reinterpret_cast<dtype *>(grad.data.dptr),
          reinterpret_cast<dtype *>(grad_per_step.data.dptr),
          reinterpret_cast<int *>(cu_seqlens.data.dptr), batch, hidden_size, total_tokens);
}

template <typename dtype>
static void thd_grad_dispatcher(Tensor grad, const Tensor &grad_per_step, const Tensor &cu_seqlens,
                                const std::string &first_half, const std::string &second_half,
                                hipStream_t stream) {
  using namespace transformer_engine;
  if (first_half == "add" && second_half == "none") {
    thd_grad_correction_helper<dtype, AddFunctor<dtype>, EmptyFunctor, 0>(grad, grad_per_step,
                                                                          cu_seqlens, stream);
  } else if (first_half == "copy" && second_half == "none") {
    thd_grad_correction_helper<dtype, CopyFunctor, EmptyFunctor, 0>(grad, grad_per_step, cu_seqlens,
                                                                    stream);
  } else if (first_half == "none" && second_half == "add") {
    thd_grad_correction_helper<dtype, EmptyFunctor, AddFunctor<dtype>, 1>(grad, grad_per_step,
                                                                          cu_seqlens, stream);
  } else if (first_half == "none" && second_half == "copy") {
    thd_grad_correction_helper<dtype, EmptyFunctor, CopyFunctor, 1>(grad, grad_per_step, cu_seqlens,
                                                                    stream);
  } else if (first_half == "add" && second_half == "copy") {
    thd_grad_correction_helper<dtype, AddFunctor<dtype>, CopyFunctor, 2>(grad, grad_per_step,
                                                                         cu_seqlens, stream);
  } else if (first_half == "copy" && second_half == "add") {
    thd_grad_correction_helper<dtype, CopyFunctor, AddFunctor<dtype>, 2>(grad, grad_per_step,
                                                                         cu_seqlens, stream);
  } else {
    NVTE_ERROR("Unsupported Functor of first half and second_half\n");
  }
}

void thd_grad_correction(Tensor grad, const Tensor &grad_per_step, const Tensor &cu_seqlens,
                         const std::string &first_half, const std::string &second_half,
                         hipStream_t stream) {
  using namespace transformer_engine;
  TRANSFORMER_ENGINE_TYPE_SWITCH_NON_FP8ONLY(
      grad.dtype(), dtype,
      thd_grad_dispatcher<dtype>(grad, grad_per_step, cu_seqlens, first_half, second_half,
                                 stream););
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Generate partitioned indices for input tokens
 **************************************************************************************************/

void thd_get_partitioned_indices(const Tensor &cu_seqlens, Tensor output, int total_tokens,
                                 int world_size, int rank, hipStream_t stream) {
  using namespace transformer_engine;
  NVTE_CHECK(cu_seqlens.dtype() == DType::kInt32);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  auto cu_seqlens_shape = cu_seqlens.shape();
  auto output_shape = output.shape();

  NVTE_CHECK(cu_seqlens_shape[0] >= 2);
  NVTE_CHECK(rank >= 0 && rank < world_size);
  NVTE_CHECK(world_size > 0);
  NVTE_CHECK(total_tokens > 0 && total_tokens % (world_size * 2) == 0);

  int batch = cu_seqlens_shape[0] - 1;

  constexpr unsigned int block = 256;
  unsigned int grid = (output_shape[0] + block - 1) / block;
  thd_partition_indices_kernel<<<grid, block, sizeof(int) * (batch + 1), stream>>>(
      reinterpret_cast<int *>(output.data.dptr), reinterpret_cast<int *>(cu_seqlens.data.dptr),
      batch, total_tokens, world_size, rank);
}

}  // namespace context_parallel
}  // namespace transformer_engine

void nvte_cp_thd_read_half_tensor(const NVTETensor &tensor, const NVTETensor &cu_seqlens,
                                  NVTETensor half, int half_idx, hipStream_t stream) {
  NVTE_API_CALL(nvte_thd_read_half_tensor);
  using namespace transformer_engine;

  context_parallel::thd_read_half_tensor(*convertNVTETensorCheck(tensor),
                                         *convertNVTETensorCheck(cu_seqlens),
                                         *convertNVTETensorCheck(half), half_idx, stream);
}

void nvte_cp_thd_second_half_lse_correction(NVTETensor lse, const NVTETensor &lse_per_step,
                                            const NVTETensor &cu_seqlens, int lse_packed,
                                            hipStream_t stream) {
  NVTE_API_CALL(nvte_thd_second_half_lse_correction);
  using namespace transformer_engine;

  context_parallel::thd_second_half_lse_correction(
      *convertNVTETensorCheck(lse), *convertNVTETensorCheck(lse_per_step),
      *convertNVTETensorCheck(cu_seqlens), lse_packed, stream);
}

void nvte_cp_thd_read_second_half_lse(const NVTETensor &lse, const NVTETensor &cu_seqlens,
                                      NVTETensor half_lse, int lse_packed,
                                      int second_half_lse_seqlen, hipStream_t stream) {
  NVTE_API_CALL(nvte_thd_read_second_half_lse);
  using namespace transformer_engine;

  context_parallel::thd_read_second_half_lse(
      *convertNVTETensorCheck(lse), *convertNVTETensorCheck(cu_seqlens),
      *convertNVTETensorCheck(half_lse), lse_packed, second_half_lse_seqlen, stream);
}

void nvte_cp_fused_out_correction(NVTETensor out, const NVTETensorPack *out_per_step,
                                  const NVTETensor &lse, const NVTETensorPack *lse_per_step,
                                  const NVTETensor &cu_seqlens, NVTE_QKV_Format qkv_format,
                                  int cp_size, int rank, bool causal,
                                  bool softmax_lse_in_packed_format, hipStream_t stream) {
  NVTE_API_CALL(nvte_fused_out_correction);
  using namespace transformer_engine;

  context_parallel::fused_out_correction(*convertNVTETensorCheck(out), out_per_step,
                                         *convertNVTETensorCheck(lse), lse_per_step,
                                         *convertNVTETensorCheck(cu_seqlens), qkv_format, cp_size,
                                         rank, causal, softmax_lse_in_packed_format, stream);
}

void nvte_cp_thd_grad_correction(NVTETensor grad, const NVTETensor &grad_per_step,
                                 const NVTETensor &cu_seqlens, const char *first_half,
                                 const char *second_half, hipStream_t stream) {
  NVTE_API_CALL(nvte_thd_grad_correction);
  using namespace transformer_engine;

  std::string first_half_str(first_half);
  std::string second_half_str(second_half);

  context_parallel::thd_grad_correction(
      *convertNVTETensorCheck(grad), *convertNVTETensorCheck(grad_per_step),
      *convertNVTETensorCheck(cu_seqlens), first_half_str, second_half_str, stream);
}

void nvte_cp_thd_get_partitioned_indices(const NVTETensor &cu_seqlens, NVTETensor output,
                                         int total_tokens, int world_size, int rank,
                                         hipStream_t stream) {
  NVTE_API_CALL(nvte_thd_get_partitioned_indices);
  using namespace transformer_engine;

  context_parallel::thd_get_partitioned_indices(*convertNVTETensorCheck(cu_seqlens),
                                                *convertNVTETensorCheck(output), total_tokens,
                                                world_size, rank, stream);
}
