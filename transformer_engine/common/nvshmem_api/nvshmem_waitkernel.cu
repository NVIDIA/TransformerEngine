#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <nvshmem.h>

#include <cstdio>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <sstream>
#include <string>

#include "../util/logging.h"
#include "nvshmem_waitkernel.h"

__global__ void __launch_bounds__(1)
    wait_until_on_stream_and_reset(uint64_t* wait_flag, uint64_t wait_value,
                                   uint64_t signal_reset) {
  nvshmem_uint64_wait_until(wait_flag, NVSHMEM_CMP_EQ, wait_value);
  *wait_flag = signal_reset;
}
void nvshmem_wait_on_stream(uint64_t* sig_addr, WaitKind wait_kind, hipStream_t stream) {
  uint64_t wait_value = 1;
  uint64_t signal_reset = 0;
  hipStream_t cur_stream = stream;

  NVTE_CHECK(wait_kind >= WaitKind::KERNEL_WAIT && wait_kind <= WaitKind::STREAM_WAIT,
             "Invalid wait kind: ", static_cast<int>(wait_kind));

  switch (wait_kind) {
    case WaitKind::KERNEL_WAIT:
      wait_until_on_stream_and_reset<<<1, 1, 0, cur_stream>>>(sig_addr, wait_value, signal_reset);
      break;
    case WaitKind::NVSHMEM_WAIT:
      nvshmemx_uint64_wait_until_on_stream(sig_addr, NVSHMEM_CMP_EQ, wait_value, cur_stream);
      hipStreamWriteValue64((hipStream_t)cur_stream, (hipDeviceptr_t)sig_addr, (cuuint64_t)signal_reset,
                           CU_STREAM_WRITE_VALUE_DEFAULT);
      break;
    case WaitKind::STREAM_WAIT:
      hipStreamWaitValue64((hipStream_t)cur_stream, (hipDeviceptr_t)sig_addr, (cuuint64_t)wait_value,
                          hipStreamWaitValueGte);
      hipStreamWriteValue64((hipStream_t)cur_stream, (hipDeviceptr_t)sig_addr, (cuuint64_t)signal_reset,
                           CU_STREAM_WRITE_VALUE_DEFAULT);
      break;
  }
}
