#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <assert.h>
#include <hip/hip_fp8.h>
#include <transformer_engine/multi_tensor.h>
#include <transformer_engine/transformer_engine.h>

#include "../utils.cuh"
#include "multi_tensor_apply.cuh"

namespace transformer_engine {
namespace multi_tensor_adam {

#define BLOCK_SIZE 512
#define ILP 4
#define THREADS_PER_WARP 32

typedef enum {
  ADAM_MODE_0 = 0,  // L2 regularization mode
  ADAM_MODE_1 = 1   // Decoupled weight decay mode(AdamW)
} adamMode_t;

using MATH_T = float;
using fp8e4m3 = __hip_fp8_e4m3_fnuz;
using fp8e5m2 = __hip_fp8_e5m2_fnuz;

template <typename T>
struct is_fp8 : std::false_type {};

template <>
struct is_fp8<fp8e4m3> : std::true_type {};

template <>
struct is_fp8<fp8e5m2> : std::true_type {};

template <bool is_fp8>
struct FP8Data {
  float scale;
  float *amax_ptr;
  float *scale_inv_ptr;
  float max;
  int warp_id;
};

template <>
struct FP8Data<false> {};

template <typename PARAM_T, typename GRAD_T, typename FULL_T, typename index_t>
struct AdamFunctorMaster {
  static constexpr bool is_fp8_type = is_fp8<PARAM_T>::value;

  __device__ __forceinline__ void operator()(index_t chunk_size, volatile int *noop_gmem,
                                             TensorListMetadata<5, is_fp8_type> &tl,  // NOLINT(*)
                                             const float beta1, const float beta2,
                                             const float beta1_correction,
                                             const float beta2_correction, const float epsilon,
                                             const float lr, adamMode_t mode, const float decay) {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    FP8Data<is_fp8_type> fp8_data;

    index_t tensor_loc = tl.block_to_tensor[blockIdx.x];

    // potentially use to pass in list of scalar
    // int tensor_num = tl.start_tensor_this_launch + tensor_loc;

    index_t chunk_idx = tl.block_to_chunk[blockIdx.x];
    index_t n = tl.sizes[tensor_loc];

    GRAD_T *g = reinterpret_cast<GRAD_T *>(tl.addresses[0][tensor_loc]);
    g += chunk_idx * chunk_size;

    PARAM_T *p = reinterpret_cast<PARAM_T *>(tl.addresses[1][tensor_loc]);
    p += chunk_idx * chunk_size;

    FULL_T *m = reinterpret_cast<FULL_T *>(tl.addresses[2][tensor_loc]);
    m += chunk_idx * chunk_size;

    FULL_T *v = reinterpret_cast<FULL_T *>(tl.addresses[3][tensor_loc]);
    v += chunk_idx * chunk_size;

    FULL_T *p_master = reinterpret_cast<FULL_T *>(tl.addresses[4][tensor_loc]);
    p_master += chunk_idx * chunk_size;

    n -= chunk_idx * chunk_size;

    if constexpr (is_fp8_type) {
      float *scale_ptr = reinterpret_cast<float *>(tl.fp8_meta_addresses[0][tensor_loc]);
      fp8_data.scale = scale_ptr != nullptr ? *scale_ptr : 1;
      fp8_data.amax_ptr = reinterpret_cast<float *>(tl.fp8_meta_addresses[1][tensor_loc]);
      fp8_data.scale_inv_ptr = reinterpret_cast<float *>(tl.fp8_meta_addresses[2][tensor_loc]);
      fp8_data.warp_id = threadIdx.x / THREADS_PER_WARP;
      fp8_data.max = 0;
    }

    // see note in multi_tensor_scale_kernel.cu
    for (index_t i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x * ILP) {
      MATH_T r_g[ILP];
      MATH_T r_p[ILP];
      MATH_T r_m[ILP];
      MATH_T r_v[ILP];
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          r_g[ii] = static_cast<MATH_T>(g[i]);
          r_p[ii] = static_cast<MATH_T>(p_master[i]);
          r_m[ii] = static_cast<MATH_T>(m[i]);
          r_v[ii] = static_cast<MATH_T>(v[i]);
        } else {
          r_g[ii] = MATH_T(0);
          r_p[ii] = MATH_T(0);
          r_m[ii] = MATH_T(0);
          r_v[ii] = MATH_T(0);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        if (mode == ADAM_MODE_0) {  // L2
          r_g[ii] = r_g[ii] + (decay * r_p[ii]);
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = next_m_unbiased / denom;
          r_p[ii] = r_p[ii] - (lr * update);
        } else {  // weight decay
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = (next_m_unbiased / denom) + (decay * r_p[ii]);
          r_p[ii] = r_p[ii] - (lr * update);
        }
      }

#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          p_master[i] = static_cast<FULL_T>(r_p[ii]);
          m[i] = static_cast<FULL_T>(r_m[ii]);
          v[i] = static_cast<FULL_T>(r_v[ii]);
          if constexpr (is_fp8_type) {
            __builtin_assume(fp8_data.max >= 0);
            fp8_data.max = fmaxf(fabsf(r_p[ii]), fp8_data.max);
            p[i] = static_cast<PARAM_T>(r_p[ii] * fp8_data.scale);
          } else {
            p[i] = static_cast<PARAM_T>(r_p[ii]);
          }
        }
      }
    }

    if constexpr (is_fp8_type) {
      fp8_data.max = transformer_engine::reduce_max<BLOCK_SIZE / THREADS_PER_WARP>(
          fp8_data.max, fp8_data.warp_id);
      if (threadIdx.x == 0) {
        if (fp8_data.amax_ptr != nullptr) {
          transformer_engine::atomicMaxFloat(fp8_data.amax_ptr, fp8_data.max);
        }
        if (fp8_data.scale_inv_ptr != nullptr) {
          *fp8_data.scale_inv_ptr = __frcp_rn(fp8_data.scale);
        }
      }
    }
  }
};

template <typename GRAD_T, typename FULL_T, typename index_t>
struct AdamFunctorMasterParamRemainder {
  __device__ __forceinline__ void operator()(index_t chunk_size, volatile int *noop_gmem,
                                             TensorListMetadata<5> &tl,  // NOLINT(*)
                                             const float beta1, const float beta2,
                                             const float beta1_correction,
                                             const float beta2_correction, const float epsilon,
                                             const float lr, adamMode_t mode, const float decay) {
    index_t tensor_loc = tl.block_to_tensor[blockIdx.x];

    index_t chunk_idx = tl.block_to_chunk[blockIdx.x];
    index_t n = tl.sizes[tensor_loc];

    GRAD_T *g = reinterpret_cast<GRAD_T *>(tl.addresses[0][tensor_loc]);
    g += chunk_idx * chunk_size;

    int16_t *p = reinterpret_cast<int16_t *>(tl.addresses[1][tensor_loc]);
    p += chunk_idx * chunk_size;

    FULL_T *m = reinterpret_cast<FULL_T *>(tl.addresses[2][tensor_loc]);
    m += chunk_idx * chunk_size;

    FULL_T *v = reinterpret_cast<FULL_T *>(tl.addresses[3][tensor_loc]);
    v += chunk_idx * chunk_size;

    int16_t *p_remainder = reinterpret_cast<int16_t *>(tl.addresses[4][tensor_loc]);
    p_remainder += chunk_idx * chunk_size;

    n -= chunk_idx * chunk_size;

    // see note in multi_tensor_scale_kernel.cu
    for (index_t i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x * ILP) {
      union fp32_or_int162 {
        float fp32;
        int16_t int16[2];
      };
      fp32_or_int162 local_master_param[ILP];
      int16_t local_p[ILP];
      int16_t local_p_rem[ILP];
      MATH_T r_g[ILP];
      MATH_T r_m[ILP];
      MATH_T r_v[ILP];
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          r_g[ii] = static_cast<MATH_T>(g[i]);
          r_m[ii] = static_cast<MATH_T>(m[i]);
          r_v[ii] = static_cast<MATH_T>(v[i]);

          local_p[ii] = static_cast<int16_t>(p[i]);
          local_p_rem[ii] = static_cast<int16_t>(p_remainder[i]);
        } else {
          r_g[ii] = MATH_T(0);
          r_m[ii] = MATH_T(0);
          r_v[ii] = MATH_T(0);

          local_p[ii] = int16_t(0);
          local_p_rem[ii] = int16_t(0);
        }
      }
// Reconstruct FP32 params
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        if (local_p_rem[ii] < 0) local_p[ii]--;  // Undo rounding
        local_master_param[ii].int16[1] = local_p[ii];
        local_master_param[ii].int16[0] = local_p_rem[ii];
      }

      MATH_T *r_p = reinterpret_cast<MATH_T *>(local_master_param);

#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        if (mode == ADAM_MODE_0) {  // L2
          r_g[ii] = r_g[ii] + (decay * r_p[ii]);
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = next_m_unbiased / denom;
          r_p[ii] = r_p[ii] - (lr * update);
        } else {  // weight decay
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = (next_m_unbiased / denom) + (decay * r_p[ii]);
          r_p[ii] = r_p[ii] - (lr * update);
        }
      }

// Split into BF16 params (rounded-to-nearest) and remainders
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        local_p[ii] = local_master_param[ii].int16[1];
        local_p_rem[ii] = local_master_param[ii].int16[0];
        if (local_p_rem[ii] < 0) local_p[ii]++;  // Round up
      }

#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          p_remainder[i] = static_cast<int16_t>(local_p_rem[ii]);
          p[i] = static_cast<int16_t>(local_p[ii]);

          m[i] = static_cast<FULL_T>(r_m[ii]);
          v[i] = static_cast<FULL_T>(r_v[ii]);
        }
      }
    }
  }
};

template <typename PARAM_T, typename GRAD_T, typename FULL_T, typename index_t>
struct AdamFunctor {
  __device__ __forceinline__ void operator()(index_t chunk_size, volatile int *noop_gmem,
                                             TensorListMetadata<4> &tl,  // NOLINT(*)
                                             const float beta1, const float beta2,
                                             const float beta1_correction,
                                             const float beta2_correction, const float epsilon,
                                             const float lr, adamMode_t mode, const float decay) {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    index_t tensor_loc = tl.block_to_tensor[blockIdx.x];

    // potentially use to pass in list of scalar
    // int tensor_num = tl.start_tensor_this_launch + tensor_loc;

    index_t chunk_idx = tl.block_to_chunk[blockIdx.x];
    index_t n = tl.sizes[tensor_loc];

    GRAD_T *g = reinterpret_cast<GRAD_T *>(tl.addresses[0][tensor_loc]);
    g += chunk_idx * chunk_size;

    PARAM_T *p = reinterpret_cast<PARAM_T *>(tl.addresses[1][tensor_loc]);
    p += chunk_idx * chunk_size;

    FULL_T *m = reinterpret_cast<FULL_T *>(tl.addresses[2][tensor_loc]);
    m += chunk_idx * chunk_size;

    FULL_T *v = reinterpret_cast<FULL_T *>(tl.addresses[3][tensor_loc]);
    v += chunk_idx * chunk_size;

    n -= chunk_idx * chunk_size;

    // see note in multi_tensor_scale_kernel.cu
    for (index_t i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x * ILP) {
      MATH_T r_g[ILP];
      MATH_T r_p[ILP];
      MATH_T r_m[ILP];
      MATH_T r_v[ILP];
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          r_g[ii] = static_cast<MATH_T>(g[i]);
          r_p[ii] = static_cast<MATH_T>(p[i]);
          r_m[ii] = static_cast<MATH_T>(m[i]);
          r_v[ii] = static_cast<MATH_T>(v[i]);
        } else {
          r_g[ii] = MATH_T(0);
          r_p[ii] = MATH_T(0);
          r_m[ii] = MATH_T(0);
          r_v[ii] = MATH_T(0);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        if (mode == ADAM_MODE_0) {  // L2
          r_g[ii] = r_g[ii] + (decay * r_p[ii]);
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = next_m_unbiased / denom;
          r_p[ii] = r_p[ii] - (lr * update);
        } else {  // weight decay
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = (next_m_unbiased / denom) + (decay * r_p[ii]);
          r_p[ii] = r_p[ii] - (lr * update);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          p[i] = static_cast<PARAM_T>(r_p[ii]);
          m[i] = static_cast<FULL_T>(r_m[ii]);
          v[i] = static_cast<FULL_T>(r_v[ii]);
        }
      }
    }
  }
};

template <typename T, typename FULL_T>
struct AdamCapturableFunctor {
  __device__ __forceinline__ void operator()(int chunk_size, volatile int *noop_gmem,
                                             TensorListMetadata<4> &tl,  // NOLINT(*)
                                             const float beta1, const float beta2, const int *step,
                                             const int bias_correction, const float epsilon,
                                             const float *lr, adamMode_t mode, const float decay,
                                             const float *inv_scale) {
    if (*noop_gmem == 1) return;

    float beta1_correction = 1.0f, beta2_correction = 1.0f;
    if (bias_correction == 1) {
      beta1_correction = 1 - pow(beta1, *step);
      beta2_correction = 1 - pow(beta2, *step);
    }

    int tensor_loc = tl.block_to_tensor[blockIdx.x];

    // potentially use to pass in list of scalar
    // int tensor_num = tl.start_tensor_this_launch + tensor_loc;

    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    T *g = reinterpret_cast<T *>(tl.addresses[0][tensor_loc]);
    g += chunk_idx * chunk_size;

    T *p = reinterpret_cast<T *>(tl.addresses[1][tensor_loc]);
    p += chunk_idx * chunk_size;

    FULL_T *m = reinterpret_cast<FULL_T *>(tl.addresses[2][tensor_loc]);
    m += chunk_idx * chunk_size;

    FULL_T *v = reinterpret_cast<FULL_T *>(tl.addresses[3][tensor_loc]);
    v += chunk_idx * chunk_size;

    n -= chunk_idx * chunk_size;

    // see note in multi_tensor_scale_kernel.cu
    for (int i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x * ILP) {
      MATH_T r_g[ILP];
      MATH_T r_p[ILP];
      MATH_T r_m[ILP];
      MATH_T r_v[ILP];
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          r_g[ii] = static_cast<MATH_T>(g[i]) * (*inv_scale);
          g[i] = static_cast<T>(r_g[ii]);
          r_p[ii] = static_cast<MATH_T>(p[i]);
          r_m[ii] = static_cast<MATH_T>(m[i]);
          r_v[ii] = static_cast<MATH_T>(v[i]);
        } else {
          r_g[ii] = MATH_T(0);
          r_p[ii] = MATH_T(0);
          r_m[ii] = MATH_T(0);
          r_v[ii] = MATH_T(0);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        if (mode == ADAM_MODE_0) {  // L2
          r_g[ii] = r_g[ii] + (decay * r_p[ii]);
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = next_m_unbiased / denom;
          r_p[ii] = r_p[ii] - (*lr * update);
        } else {  // weight decay
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = (next_m_unbiased / denom) + (decay * r_p[ii]);
          r_p[ii] = r_p[ii] - (*lr * update);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          p[i] = static_cast<T>(r_p[ii]);
          m[i] = static_cast<T>(r_m[ii]);
          v[i] = static_cast<T>(r_v[ii]);
        }
      }
    }
  }
};

template <typename T, typename FULL_T>
struct AdamCapturableMasterFunctor {
  __device__ __forceinline__ void operator()(int chunk_size, volatile int *noop_gmem,
                                             TensorListMetadata<5> &tl,  // NOLINT(*)
                                             const float beta1, const float beta2, const int *step,
                                             const int bias_correction, const float epsilon,
                                             const float *lr, adamMode_t mode, const float decay,
                                             const float *inv_scale) {
    if (*noop_gmem == 1) return;

    float beta1_correction = 1.0f, beta2_correction = 1.0f;
    if (bias_correction == 1) {
      beta1_correction = 1 - pow(beta1, *step);
      beta2_correction = 1 - pow(beta2, *step);
    }

    int tensor_loc = tl.block_to_tensor[blockIdx.x];

    // potentially use to pass in list of scalar
    // int tensor_num = tl.start_tensor_this_launch + tensor_loc;

    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    T *g = reinterpret_cast<T *>(tl.addresses[0][tensor_loc]);
    g += chunk_idx * chunk_size;

    T *p = reinterpret_cast<T *>(tl.addresses[1][tensor_loc]);
    p += chunk_idx * chunk_size;

    FULL_T *m = reinterpret_cast<FULL_T *>(tl.addresses[2][tensor_loc]);
    m += chunk_idx * chunk_size;

    FULL_T *v = reinterpret_cast<FULL_T *>(tl.addresses[3][tensor_loc]);
    v += chunk_idx * chunk_size;

    FULL_T *p_master = reinterpret_cast<FULL_T *>(tl.addresses[4][tensor_loc]);
    p_master += chunk_idx * chunk_size;

    n -= chunk_idx * chunk_size;

    // see note in multi_tensor_scale_kernel.cu
    for (int i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x * ILP) {
      MATH_T r_g[ILP];
      MATH_T r_p[ILP];
      MATH_T r_m[ILP];
      MATH_T r_v[ILP];
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          r_g[ii] = static_cast<MATH_T>(g[i]) * (*inv_scale);
          g[i] = static_cast<T>(r_g[ii]);
          r_p[ii] = static_cast<MATH_T>(p_master[i]);
          r_m[ii] = static_cast<MATH_T>(m[i]);
          r_v[ii] = static_cast<MATH_T>(v[i]);
        } else {
          r_g[ii] = MATH_T(0);
          r_p[ii] = MATH_T(0);
          r_m[ii] = MATH_T(0);
          r_v[ii] = MATH_T(0);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        if (mode == ADAM_MODE_0) {  // L2
          r_g[ii] = r_g[ii] + (decay * r_p[ii]);
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = next_m_unbiased / denom;
          r_p[ii] = r_p[ii] - (*lr * update);
        } else {  // weight decay
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = (next_m_unbiased / denom) + (decay * r_p[ii]);
          r_p[ii] = r_p[ii] - (*lr * update);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          p[i] = static_cast<T>(r_p[ii]);
          p_master[i] = static_cast<FULL_T>(r_p[ii]);
          m[i] = static_cast<FULL_T>(r_m[ii]);
          v[i] = static_cast<FULL_T>(r_v[ii]);
        }
      }
    }
  }
};

void multi_tensor_adam_cuda(int chunk_size, Tensor noop_flag,
                            std::vector<std::vector<Tensor *>> tensor_lists, const float lr,
                            const float beta1, const float beta2, const float epsilon,
                            const int step, const int mode, const int bias_correction,
                            const float weight_decay, const int device_id, hipStream_t stream) {
  const size_t num_tensor_lists = tensor_lists.size();
  const size_t num_tensors_per_list = tensor_lists[0].size();

  // Handle bias correction mode
  float bias_correction1 = 1.0f, bias_correction2 = 1.0f;
  if (bias_correction == 1) {
    bias_correction1 = 1 - std::pow(beta1, step);
    bias_correction2 = 1 - std::pow(beta2, step);
  }

  size_t max_size = 0;
  bool requires_64bit_indexing = false;
  for (size_t i = 0; i < num_tensor_lists; i++) {
    for (size_t j = 0; j < num_tensors_per_list; j++) {
      if (tensor_lists[i][j]->numel() > max_size) {
        max_size = tensor_lists[i][j]->numel();
        if (max_size >= INT_MAX) {
          requires_64bit_indexing = true;
          break;
        }
      }
    }
    if (requires_64bit_indexing) {
      break;
    }
  }

  const auto g_in_type_te = tensor_lists[0][0]->dtype();
  const auto p_in_type_te = tensor_lists[1][0]->dtype();

  // case 4:  g, p, m, v
  // case 5:  g, p, m, v, p_master
  NVTE_CHECK(num_tensor_lists == 4 || num_tensor_lists == 5, "tensor list must contain 4 or 5");

  if (requires_64bit_indexing) {
    if (num_tensor_lists == 4) {
      // Assume single type across p,g,m1,m2 now
      TRANSFORMER_ENGINE_TYPE_SWITCH_NON_FP8ONLY(
          p_in_type_te, p_in_type,
          TRANSFORMER_ENGINE_TYPE_SWITCH_NON_FP8ONLY(
              g_in_type_te, g_in_type,
              multi_tensor_apply<4>((int64_t)BLOCK_SIZE, (int64_t)chunk_size, noop_flag,
                                    tensor_lists,
                                    AdamFunctor<p_in_type, g_in_type, float, int64_t>(), device_id,
                                    stream, beta1, beta2, bias_correction1, bias_correction2,
                                    epsilon, lr, (adamMode_t)mode, weight_decay);));
    } else {
      // g, p, m, v, p_master
      TRANSFORMER_ENGINE_TYPE_SWITCH_NON_FP8ONLY(
          p_in_type_te, p_in_type,
          TRANSFORMER_ENGINE_TYPE_SWITCH_NON_FP8ONLY(
              g_in_type_te, g_in_type,
              multi_tensor_apply<5>(
                  (int64_t)BLOCK_SIZE, (int64_t)chunk_size, noop_flag, tensor_lists,
                  AdamFunctorMaster<p_in_type, g_in_type, float, int64_t>(), device_id, stream,
                  beta1, beta2, bias_correction1, bias_correction2, epsilon, lr, (adamMode_t)mode,
                  weight_decay);));
    }
  } else {
    if (num_tensor_lists == 4) {
      // Assume single type across p,g,m1,m2 now
      TRANSFORMER_ENGINE_TYPE_SWITCH_NON_FP8ONLY(
          p_in_type_te, p_in_type,
          TRANSFORMER_ENGINE_TYPE_SWITCH_NON_FP8ONLY(
              g_in_type_te, g_in_type,
              multi_tensor_apply<4>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                                    AdamFunctor<p_in_type, g_in_type, float, int32_t>(), device_id,
                                    stream, beta1, beta2, bias_correction1, bias_correction2,
                                    epsilon, lr, (adamMode_t)mode, weight_decay);));
    } else {
      TRANSFORMER_ENGINE_TYPE_SWITCH_NON_FP8ONLY(
          p_in_type_te, p_in_type,
          TRANSFORMER_ENGINE_TYPE_SWITCH_NON_FP8ONLY(
              g_in_type_te, g_in_type,
              multi_tensor_apply<5>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                                    AdamFunctorMaster<p_in_type, g_in_type, float, int32_t>(),
                                    device_id, stream, beta1, beta2, bias_correction1,
                                    bias_correction2, epsilon, lr, (adamMode_t)mode,
                                    weight_decay);));
    }
  }
  NVTE_CHECK_CUDA(hipGetLastError());
}

void multi_tensor_adam_param_remainder_cuda(int chunk_size, Tensor noop_flag,
                                            std::vector<std::vector<Tensor *>> tensor_lists,
                                            const float lr, const float beta1, const float beta2,
                                            const float epsilon, const int step, const int mode,
                                            const int bias_correction, const float weight_decay,
                                            const int device_id, hipStream_t stream) {
  const size_t num_tensor_lists = tensor_lists.size();

  // Handle bias correction mode
  float bias_correction1 = 1.0f, bias_correction2 = 1.0f;
  if (bias_correction == 1) {
    bias_correction1 = 1 - std::pow(beta1, step);
    bias_correction2 = 1 - std::pow(beta2, step);
  }

  const auto g_in_type_te = tensor_lists[0][0]->dtype();
  const auto p_in_type_te = tensor_lists[1][0]->dtype();

  // case 5:  g, p, m, v, p_master
  NVTE_CHECK(num_tensor_lists == 5, "tensor list must contain 5");
  NVTE_CHECK(p_in_type_te == DType::kBFloat16,
             "Adam with BF16 param remainders requires BF16 params");

  // g, p, m, v, p_master

  TRANSFORMER_ENGINE_TYPE_SWITCH_NON_FP8ONLY(
      g_in_type_te, g_in_type,
      multi_tensor_apply<5>((int64_t)BLOCK_SIZE, (int64_t)chunk_size, noop_flag, tensor_lists,
                            AdamFunctorMasterParamRemainder<g_in_type, float, int64_t>(), device_id,
                            stream, beta1, beta2, bias_correction1, bias_correction2, epsilon, lr,
                            (adamMode_t)mode, weight_decay););

  NVTE_CHECK_CUDA(hipGetLastError());
}

void multi_tensor_adam_fp8_cuda(int chunk_size, Tensor noop_flag,
                                std::vector<std::vector<Tensor *>> tensor_lists, const float lr,
                                const float beta1, const float beta2, const float epsilon,
                                const int step, const int mode, const int bias_correction,
                                const float weight_decay, const DType fp8_dtype,
                                const int device_id, hipStream_t stream) {
  const size_t num_tensor_lists = tensor_lists.size();
  const size_t num_tensors_per_list = tensor_lists[0].size();

  // Handle bias correction mode
  float bias_correction1 = 1.0f, bias_correction2 = 1.0f;
  if (bias_correction == 1) {
    bias_correction1 = 1 - std::pow(beta1, step);
    bias_correction2 = 1 - std::pow(beta2, step);
  }

  size_t max_size = 0;
  bool requires_64bit_indexing = false;
  for (size_t i = 0; i < num_tensor_lists; i++) {
    for (size_t j = 0; j < num_tensors_per_list; j++) {
      if (tensor_lists[i][j]->numel() > max_size) {
        max_size = tensor_lists[i][j]->numel();
        if (max_size >= INT_MAX) {
          requires_64bit_indexing = true;
          break;
        }
      }
    }
    if (requires_64bit_indexing) {
      break;
    }
  }

  const auto g_in_type_te = tensor_lists[0][0]->dtype();

  // case 8:  g, p_fp8, m, v, p_master, scale, amax, scale_inv
  NVTE_CHECK(num_tensor_lists == 8, "tensor list must contain 8 tensors");

  if (requires_64bit_indexing) {
    TRANSFORMER_ENGINE_TYPE_SWITCH_FP8ONLY(
        fp8_dtype, FP8_T,
        TRANSFORMER_ENGINE_TYPE_SWITCH_NON_FP8ONLY(
            g_in_type_te, g_in_type,
            multi_tensor_apply<5, true>(
                (int64_t)BLOCK_SIZE, (int64_t)chunk_size, noop_flag, tensor_lists,
                AdamFunctorMaster<FP8_T, g_in_type, float, int64_t>(), device_id, stream, beta1,
                beta2, bias_correction1, bias_correction2, epsilon, lr, (adamMode_t)mode,
                weight_decay);));
  } else {
    TRANSFORMER_ENGINE_TYPE_SWITCH_FP8ONLY(
        fp8_dtype, FP8_T,
        TRANSFORMER_ENGINE_TYPE_SWITCH_NON_FP8ONLY(
            g_in_type_te, g_in_type,
            multi_tensor_apply<5, true>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                                        AdamFunctorMaster<FP8_T, g_in_type, float, int32_t>(),
                                        device_id, stream, beta1, beta2, bias_correction1,
                                        bias_correction2, epsilon, lr, (adamMode_t)mode,
                                        weight_decay);));
  }
  NVTE_CHECK_CUDA(hipGetLastError());
}

void multi_tensor_adam_capturable_cuda(int chunk_size, Tensor noop_flag,
                                       std::vector<std::vector<Tensor *>> tensor_lists, Tensor lr,
                                       const float beta1, const float beta2, const float epsilon,
                                       Tensor step, const int mode, const int bias_correction,
                                       const float weight_decay, Tensor inv_scale,
                                       const int device_id, hipStream_t stream) {
  TRANSFORMER_ENGINE_TYPE_SWITCH_NON_FP8ONLY(
      tensor_lists[0][0]->dtype(), dtype,
      multi_tensor_apply<4>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                            AdamCapturableFunctor<dtype, float>(), device_id, stream, beta1, beta2,
                            reinterpret_cast<int *>(step.data.dptr), bias_correction, epsilon,
                            reinterpret_cast<float *>(lr.data.dptr), (adamMode_t)mode, weight_decay,
                            reinterpret_cast<float *>(inv_scale.data.dptr));)

  NVTE_CHECK_CUDA(hipGetLastError());
}

void multi_tensor_adam_capturable_master_cuda(int chunk_size, Tensor noop_flag,
                                              std::vector<std::vector<Tensor *>> tensor_lists,
                                              Tensor lr, const float beta1, const float beta2,
                                              const float epsilon, Tensor step, const int mode,
                                              const int bias_correction, const float weight_decay,
                                              Tensor inv_scale, const int device_id,
                                              hipStream_t stream) {
  TRANSFORMER_ENGINE_TYPE_SWITCH_NON_FP8ONLY(
      tensor_lists[0][0]->dtype(), dtype,
      multi_tensor_apply<5>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                            AdamCapturableMasterFunctor<dtype, float>(), device_id, stream, beta1,
                            beta2, reinterpret_cast<int *>(step.data.dptr), bias_correction,
                            epsilon, reinterpret_cast<float *>(lr.data.dptr), (adamMode_t)mode,
                            weight_decay, reinterpret_cast<float *>(inv_scale.data.dptr));)

  NVTE_CHECK_CUDA(hipGetLastError());
}

}  // namespace multi_tensor_adam
}  // namespace transformer_engine

void nvte_multi_tensor_adam_cuda(int chunk_size, NVTETensor noop_flag, NVTETensor **tensor_lists,
                                 const size_t num_tensor_lists, const size_t num_tensors_per_list,
                                 const float lr, const float beta1, const float beta2,
                                 const float epsilon, const int step, const int mode,
                                 const int bias_correction, const float weight_decay,
                                 const int device_id, hipStream_t stream) {
  NVTE_API_CALL(nvte_multi_tensor_adam_cuda);
  using namespace transformer_engine;

  multi_tensor_adam::multi_tensor_adam_cuda(
      chunk_size, *reinterpret_cast<Tensor *>(noop_flag),
      convert_tensor_array(tensor_lists, num_tensor_lists, num_tensors_per_list), lr, beta1, beta2,
      epsilon, step, mode, bias_correction, weight_decay, device_id, stream);
}

void nvte_multi_tensor_adam_param_remainder_cuda(
    int chunk_size, NVTETensor noop_flag, NVTETensor **tensor_lists, const size_t num_tensor_lists,
    const size_t num_tensors_per_list, const float lr, const float beta1, const float beta2,
    const float epsilon, const int step, const int mode, const int bias_correction,
    const float weight_decay, const int device_id, hipStream_t stream) {
  NVTE_API_CALL(nvte_multi_tensor_adam_param_remainder_cuda);
  using namespace transformer_engine;

  multi_tensor_adam::multi_tensor_adam_param_remainder_cuda(
      chunk_size, *reinterpret_cast<Tensor *>(noop_flag),
      convert_tensor_array(tensor_lists, num_tensor_lists, num_tensors_per_list), lr, beta1, beta2,
      epsilon, step, mode, bias_correction, weight_decay, device_id, stream);
}

void nvte_multi_tensor_adam_fp8_cuda(int chunk_size, NVTETensor noop_flag,
                                     NVTETensor **tensor_lists, const size_t num_tensor_lists,
                                     const size_t num_tensors_per_list, const float lr,
                                     const float beta1, const float beta2, const float epsilon,
                                     const int step, const int mode, const int bias_correction,
                                     const float weight_decay, const NVTEDType fp8_dtype,
                                     const int device_id, hipStream_t stream) {
  NVTE_API_CALL(nvte_multi_tensor_adam_fp8_cuda);
  using namespace transformer_engine;

  multi_tensor_adam::multi_tensor_adam_fp8_cuda(
      chunk_size, *reinterpret_cast<Tensor *>(noop_flag),
      convert_tensor_array(tensor_lists, num_tensor_lists, num_tensors_per_list), lr, beta1, beta2,
      epsilon, step, mode, bias_correction, weight_decay, static_cast<DType>(fp8_dtype), device_id,
      stream);
}

void nvte_multi_tensor_adam_capturable_cuda(
    int chunk_size, NVTETensor noop_flag, NVTETensor **tensor_lists, const size_t num_tensor_lists,
    const size_t num_tensors_per_list, NVTETensor lr, const float beta1, const float beta2,
    const float epsilon, NVTETensor step, const int mode, const int bias_correction,
    const float weight_decay, NVTETensor inv_scale, const int device_id, hipStream_t stream) {
  NVTE_API_CALL(nvte_multi_tensor_adam_capturable_cuda);
  using namespace transformer_engine;

  multi_tensor_adam::multi_tensor_adam_capturable_cuda(
      chunk_size, *reinterpret_cast<Tensor *>(noop_flag),
      convert_tensor_array(tensor_lists, num_tensor_lists, num_tensors_per_list),
      *reinterpret_cast<Tensor *>(lr), beta1, beta2, epsilon, *reinterpret_cast<Tensor *>(step),
      mode, bias_correction, weight_decay, *reinterpret_cast<Tensor *>(inv_scale), device_id,
      stream);
}

void nvte_multi_tensor_adam_capturable_master_cuda(
    int chunk_size, NVTETensor noop_flag, NVTETensor **tensor_lists, const size_t num_tensor_lists,
    const size_t num_tensors_per_list, NVTETensor lr, const float beta1, const float beta2,
    const float epsilon, NVTETensor step, const int mode, const int bias_correction,
    const float weight_decay, NVTETensor inv_scale, const int device_id, hipStream_t stream) {
  NVTE_API_CALL(nvte_multi_tensor_adam_capturable_master_cuda);
  using namespace transformer_engine;

  multi_tensor_adam::multi_tensor_adam_capturable_master_cuda(
      chunk_size, *reinterpret_cast<Tensor *>(noop_flag),
      convert_tensor_array(tensor_lists, num_tensor_lists, num_tensors_per_list),
      *reinterpret_cast<Tensor *>(lr), beta1, beta2, epsilon, *reinterpret_cast<Tensor *>(step),
      mode, bias_correction, weight_decay, *reinterpret_cast<Tensor *>(inv_scale), device_id,
      stream);
}
