#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <transformer_engine/recipe.h>

#include <cmath>
#include <string>
#include <limits>

#include "../common.h"
#include "../util/logging.h"
#include "../util/hip/hip_runtime.h"

namespace transformer_engine {
namespace delayed_scaling_recipe {

namespace {

// amax value to use for updating scaling factor
enum class AmaxComputeAlgo { INVALID, MOST_RECENT, MAX };

const char* dtype_name(DType dtype) {
  TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(dtype, Type,
    return TypeInfo<Type>::name;
  );  // NOLINT(*)
  return "";
}

// Maximum representable value of an FP8 dtype
inline float fp8_dtype_max(DType dtype) {
  switch (dtype) {
  case DType::kFloat8E4M3: return 448;
  case DType::kFloat8E5M2: return 57344;
  default:
    NVTE_ERROR("Expected FP8 dtype, but got ", dtype_name(dtype));
  }
  return 0;
}

// struct for amax parameters
struct AmaxParam {
  int num_scale = 0;
  float* amax_history = nullptr;
  float* scale = nullptr;
  float* scale_inv = nullptr;
};

// dummy struct for kernel_bulk's other params
struct OtherParams {
  float* a;
  size_t b;
  AmaxComputeAlgo c;
  float d;
};

#if CUDART_VERSION >= 12010
constexpr size_t max_constant_memory_per_kernel = 32768;
constexpr size_t AMAX_PARAMS_LIMIT = (
  max_constant_memory_per_kernel - sizeof(OtherParams)) / sizeof(AmaxParam);
#else
constexpr size_t max_constant_memory_per_kernel = 4096;
constexpr size_t AMAX_PARAMS_LIMIT = (
  max_constant_memory_per_kernel - sizeof(OtherParams)) / sizeof(AmaxParam);
#endif

struct AmaxParams {
  AmaxParam param[AMAX_PARAMS_LIMIT];
};

namespace amax_and_scale_update_impl {

// CUDA block size
constexpr size_t bsize = 256;

/* CUDA kernel to update amax history and FP8 scaling factors
 *
 * Block dims: bsize x 1 x 1
 *
 * Grid dims: num_scales x 1 x 1
 */
__global__ void __launch_bounds__(bsize)
kernel(const float* amax_history_ptr,
       const float* scale_ptr,
       const float* scale_inv_ptr,
       const unsigned char* scale_inv_mask_ptr,
       float* updated_amax_history_ptr,
       float* updated_scale_ptr,
       float* updated_scale_inv_ptr,
       size_t amax_history_length,
       size_t amax_history_stride,
       AmaxComputeAlgo amax_compute_algo,
       float scaled_max) {
  const size_t tid = threadIdx.x;
  const size_t bid = blockIdx.x;

  // Update amax
  float amax = 0;
  {
    // Roll amax history
    const auto* amax_history = amax_history_ptr + bid;
    auto* updated_amax_history = updated_amax_history_ptr + bid;
    const auto last_amax = amax_history[0];
    const auto& length = amax_history_length;
    const auto& stride = amax_history_stride;
    for (size_t off = 0; off < length; off += bsize) {
      const size_t i = off + tid;
      float a = 0;
      if (i < length) {
        a = (i < length - 1) ? amax_history[(i+1)*stride] : last_amax;
        amax = fmaxf(amax, a);
      }
      __syncthreads();  // In case roll is in-place
      if (i < length) {
        updated_amax_history[i*stride] = (i > 0) ? a : 0;
      }
    }

    // Compute amax to use for scaling factor
    switch (amax_compute_algo) {
    case AmaxComputeAlgo::MOST_RECENT:
      amax = last_amax;
      break;
    case AmaxComputeAlgo::MAX:
      {
        __shared__ float shared_amax[bsize];
        shared_amax[tid] = amax;
        __syncthreads();
#pragma unroll
        for (size_t off = bsize / 2; off > 0; off /= 2) {
          if (tid < off) {
            shared_amax[tid] = fmaxf(shared_amax[tid], shared_amax[tid + off]);
          }
          __syncthreads();
        }
        amax = shared_amax[tid];
      }
      break;
    default:
      amax = 0;
    }
  }

  // Update scale and scale inverse
  if (tid == 0) {
    // Update scale
    float scale;
    if (isfinite(amax) && amax > 0) {
      scale = scaled_max / amax;
    } else {
      scale = scale_ptr[bid];
    }
    // When the amax is too tiny that the scale becoming infinite in FP32,
    // we set the scale to the max value of FP32. In this case, the tensor’s
    // amax won't get mapped to the FP8 max representable, but rather
    // something below that, but this is the best thing we can do.
    if (isinf(scale)) {
        scale = std::numeric_limits<float>::max();
    }
    updated_scale_ptr[bid] = scale;

    // Update scale inverse
    float scale_inv;
    if (scale_inv_mask_ptr == nullptr || scale_inv_mask_ptr[bid]) {
      scale_inv = 1 / scale;
    } else {
      scale_inv = scale_inv_ptr[bid];
    }
    updated_scale_inv_ptr[bid] = scale_inv;
  }
}

/* CUDA kernel to bulk-update amax history and FP8 scaling factors
 *
 * Block dims: bsize x 1 x 1
 *
 * Grid dims: num_tensors x 1 x 1
 */
__global__ void __launch_bounds__(bsize)
kernel_bulk(
       float* amax_reduction_buffer,
       AmaxParams p,
       size_t amax_history_length,
       AmaxComputeAlgo amax_compute_algo,
       float scaled_max) {
  const size_t bid = blockIdx.x;
  const size_t tid = threadIdx.x;
  const int num_scale = p.param[bid].num_scale;

  int offset_in_buffer = 0;
  for (int j = 0; j < bid; j++) {
    offset_in_buffer += p.param[j].num_scale;
  }

  for (int count = 0; count < num_scale; count++) {
    // Update amax
    float amax = 0;
    {
      // Roll amax history
      const auto& length = amax_history_length;
      const auto& stride = p.param[bid].num_scale;
      auto* amax_history = p.param[bid].amax_history+count;
      const auto last_amax = ((amax_reduction_buffer != nullptr)
            && (amax_reduction_buffer[offset_in_buffer+count] != 0.0f)) ?
            amax_reduction_buffer[offset_in_buffer+count] : amax_history[0];
      if (last_amax != 0.0f) {
        for (size_t off = 0; off < length; off += bsize) {
          const size_t i = off + tid;
          float a = 0;
          if (i < length) {
            a = (i < length - 1) ? amax_history[(i+1)*stride] : last_amax;
            amax = fmaxf(amax, a);
          }
          __syncthreads();  // Inplace roll
          if (i < length) {
            amax_history[i*stride] = (i > 0) ? a : 0;
          }
        }
      }

      // Compute amax to use for scaling factor
      switch (amax_compute_algo) {
      case AmaxComputeAlgo::MOST_RECENT:
        amax = last_amax;
        break;
      case AmaxComputeAlgo::MAX:
        {
          __shared__ float shared_amax[bsize];
          shared_amax[tid] = amax;
          __syncthreads();
#pragma unroll
          for (size_t off = bsize / 2; off > 0; off /= 2) {
            if (tid < off) {
              shared_amax[tid] = fmaxf(shared_amax[tid], shared_amax[tid + off]);
            }
            __syncthreads();
          }
          amax = shared_amax[tid];
        }
        break;
      default:
        amax = 0;
      }
    }

    // Update scale and scale inverse
    if (tid == 0) {
      // Computing the scaling factor requires consideration of the following scenarios:
      // 1. amax == 0:
      //    No action is possible, set scale to the previous scale (or 1).
      // 2. 0 < amax < tiny_amax
      //    The amax is too tiny that the scale becomes infinite in FP32.
      //    Set scale = FP32_max
      // 3. tiny_amax <= amax < FP32_max:
      //    Set scale = FP8_max (or scaled_max) / amax
      // 4. When amax == inf or amax == nan:
      //    No action is possible, set scale to the previous scale (or 1).

      float scale;
      if (isfinite(amax) && amax > 0) {
        scale = scaled_max / amax;
      } else {
        scale = p.param[bid].scale[count];
      }
      // When the amax is too tiny that the scale becoming infinite in FP32,
      // we set the scale to the max value of FP32. In this case, the tensor’s
      // amax won't get mapped to the FP8 max representable, but rather
      // something below that, but this is the best thing we can do.
      if (isinf(scale)) {
          scale = std::numeric_limits<float>::max();
      }
      p.param[bid].scale[count] = scale;
      p.param[bid].scale_inv[count] = 1 / scale;
    }
  }
}

}  // namespace amax_and_scale_update_impl

}  // namespace


void amax_and_scale_update(const Tensor &amax_history,
                           const Tensor &scale,
                           const Tensor &scale_inv,
                           const Tensor &scale_inv_mask,
                           Tensor *updated_amax_history_,
                           Tensor *updated_scale_,
                           Tensor *updated_scale_inv_,
                           const std::string &amax_compute_algo,
                           DType fp8_dtype,
                           float margin,
                           hipStream_t stream) {
  auto& updated_amax_history = *updated_amax_history_;
  auto& updated_scale = *updated_scale_;
  auto& updated_scale_inv = *updated_scale_inv_;

  // Number of elements in tensor
  auto numel = [] (const Tensor &tensor) -> size_t {
    size_t acc = 1;
    for (const auto& dim : tensor.data.shape) {
      acc *= dim;
    }
    return acc;
  };

  // Check tensors
  NVTE_CHECK(amax_history.data.shape.size() == 2,
             "Found ", amax_history.data.shape.size(), " dims");
  const size_t amax_history_length = amax_history.data.shape[0];
  const size_t num_scales = amax_history.data.shape[1];
  NVTE_CHECK(amax_history.data.dtype == DType::kFloat32,
             "Found ", dtype_name(amax_history.data.dtype), ".");
  NVTE_CHECK(numel(scale) == num_scales,
             "Expected ", num_scales, " elements, ",
             "but found ", numel(scale), ".");
  NVTE_CHECK(scale.data.dtype == DType::kFloat32,
             "Found ", dtype_name(scale.data.dtype), ".");
  if (scale_inv_mask.data.dptr != nullptr) {
    NVTE_CHECK(numel(scale_inv) == num_scales,
               "Expected ", num_scales, " elements, ",
               "but found ", numel(scale_inv), ".");
    NVTE_CHECK(scale_inv.data.dtype == DType::kFloat32);
    NVTE_CHECK(numel(scale_inv_mask) == num_scales,
               "Expected ", num_scales, " elements, ",
               "but found ", numel(scale_inv_mask), ".");
    NVTE_CHECK(scale_inv_mask.data.dtype == DType::kByte,
               "Found ", dtype_name(scale_inv_mask.data.dtype), ".");
  }
  NVTE_CHECK(updated_amax_history.data.shape.size() == 2,
             "Found ", updated_amax_history.data.shape.size(), " dims.");
  NVTE_CHECK(updated_amax_history.data.shape[0] == amax_history_length,
             "Expected ", amax_history_length, ", ",
             "but found ", updated_amax_history.data.shape[0]);
  NVTE_CHECK(updated_amax_history.data.shape[1] == num_scales,
             "Expected ", num_scales, ", ",
             "but found ", updated_amax_history.data.shape[1]);
  NVTE_CHECK(updated_amax_history.data.dtype == DType::kFloat32,
             "Got ", dtype_name(updated_amax_history.data.dtype), ".");
  NVTE_CHECK(numel(updated_scale) == num_scales,
             "Expected ", num_scales, " elements, ",
             "but found ", numel(updated_scale), ".");
  NVTE_CHECK(updated_scale.data.dtype == DType::kFloat32,
             "Got ", dtype_name(updated_scale.data.dtype), ".");
  NVTE_CHECK(numel(updated_scale_inv) == num_scales,
             "Expected ", num_scales, " elements, ",
             "but found ", numel(updated_scale_inv), ".");
  NVTE_CHECK(updated_scale_inv.data.dtype == DType::kFloat32,
             "Got ", dtype_name(updated_scale_inv.data.dtype), ".");

  // amax value to use for updating scaling factor
  AmaxComputeAlgo amax_compute_algo_ = AmaxComputeAlgo::INVALID;
  if (amax_compute_algo == "max") {
    amax_compute_algo_ = AmaxComputeAlgo::MAX;
  } else if (amax_compute_algo == "most_recent") {
    amax_compute_algo_ = AmaxComputeAlgo::MOST_RECENT;
  } else {
    NVTE_ERROR("Unsupported amax compute algorithm (", amax_compute_algo, ")");
  }

  // Expected maximum value after scale is applied
  const float scaled_max = fp8_dtype_max(fp8_dtype) * std::pow(2.f, -margin);

  // Launch CUDA kernel
  constexpr size_t block_size = amax_and_scale_update_impl::bsize;
  const size_t grid_size = num_scales;
  amax_and_scale_update_impl::kernel
    <<<grid_size, block_size, 0, stream>>>(
      static_cast<const float*>(amax_history.data.dptr),
      static_cast<const float*>(scale.data.dptr),
      static_cast<const float*>(scale_inv.data.dptr),
      static_cast<const unsigned char*>(scale_inv_mask.data.dptr),
      static_cast<float*>(updated_amax_history.data.dptr),
      static_cast<float*>(updated_scale.data.dptr),
      static_cast<float*>(updated_scale_inv.data.dptr),
      amax_history_length,
      num_scales,
      amax_compute_algo_,
      scaled_max);
  NVTE_CHECK_CUDA(hipGetLastError());
}


void amax_and_scale_update_after_reduction(const Tensor &amax_reduction_buffer,
                                           std::vector<Tensor*> amax_histories,
                                           std::vector<Tensor*> scales,
                                           std::vector<Tensor*> scale_invs,
                                           const std::string &amax_compute_algo,
                                           DType fp8_dtype,
                                           float margin,
                                           hipStream_t stream) {
  using namespace transformer_engine;

  // amax value to use for updating scaling factor
  AmaxComputeAlgo amax_compute_algo_ = AmaxComputeAlgo::INVALID;
  if (amax_compute_algo == "max") {
    amax_compute_algo_ = AmaxComputeAlgo::MAX;
  } else if (amax_compute_algo == "most_recent") {
    amax_compute_algo_ = AmaxComputeAlgo::MOST_RECENT;
  } else {
    NVTE_ERROR("Unsupported amax compute algorithm (", amax_compute_algo, ")");
  }

  // Expected maximum value after scale is applied
  const float scaled_max = fp8_dtype_max(fp8_dtype) * std::pow(2.f, -margin);

  // Number of elements in tensor
  auto numel = [] (const Tensor *tensor) -> size_t {
    size_t acc = 1;
    for (const auto& dim : tensor->data.shape) {
      acc *= dim;
    }
    return acc;
  };

  // Number of tensors in the bulk
  const size_t num_tensors = amax_histories.size();
  size_t num_remaining_tensors = num_tensors;
  const int num_kernels = (num_tensors+AMAX_PARAMS_LIMIT-1)/AMAX_PARAMS_LIMIT;
  size_t amax_history_length = 0;
  if (num_tensors > 0) {
    amax_history_length = amax_histories[0]->data.shape[0];
  }

  // amax parameters
  float* amax_buffer = static_cast<float*>(amax_reduction_buffer.data.dptr);
  AmaxParams p;
  for (int iter = 0; iter < num_kernels; iter++) {
    size_t kernel_num_scales = 0;
    size_t kernel_num_tensors = (iter == (num_kernels - 1))
          ? num_remaining_tensors: AMAX_PARAMS_LIMIT;
    for (size_t pi = 0; pi < kernel_num_tensors; pi++) {
      size_t i = iter * AMAX_PARAMS_LIMIT + pi;

      // Check tensors
      int num_scale = amax_histories[i]->data.shape[1];
      NVTE_CHECK(amax_histories[i]->data.dtype == DType::kFloat32,
                 "Found ", dtype_name(amax_histories[i]->data.dtype), ".");
      NVTE_CHECK(amax_histories[i]->data.shape.size() == 2,
                 "Found ", amax_histories[i]->data.shape.size(), " dims");
      NVTE_CHECK(numel(amax_histories[i]) == amax_history_length * num_scale,
                 "Expected ", amax_history_length * num_scale, " elements, ",
                 "but found ", numel(amax_histories[i]), ".");
      NVTE_CHECK(scales[i]->data.dtype == DType::kFloat32,
                 "Found ", dtype_name(scales[i]->data.dtype), ".");
      NVTE_CHECK(scales[i]->data.shape.size() == 1,
                 "Found ", scales[i]->data.shape.size(), " dims");
      NVTE_CHECK(numel(scales[i]) == num_scale,
                 "Expected ", num_scale, " elements, ",
                 "Found ", numel(scales[i]), ".");

      // amax parameters
      kernel_num_scales += num_scale;
      p.param[pi].num_scale = num_scale;
      p.param[pi].amax_history = static_cast<float*>(amax_histories[i]->data.dptr);
      p.param[pi].scale = static_cast<float*>(scales[i]->data.dptr);
      p.param[pi].scale_inv = static_cast<float*>(scale_invs[i]->data.dptr);
    }

    // Launch CUDA kernel
    size_t grid_size = kernel_num_tensors;
    const size_t block_size = amax_and_scale_update_impl::bsize;
    amax_and_scale_update_impl::kernel_bulk
      <<<grid_size, block_size, 0, stream>>>(
        amax_buffer,
        p,
        amax_history_length,
        amax_compute_algo_,
        scaled_max);
    NVTE_CHECK_CUDA(hipGetLastError());

    // shift amax buffer pointer
    if (amax_buffer != nullptr) {
      amax_buffer += kernel_num_scales;
    }
    num_remaining_tensors -= AMAX_PARAMS_LIMIT;
  }
}

}  // namespace delayed_scaling_recipe
}  // namespace transformer_engine


void nvte_delayed_scaling_recipe_amax_and_scale_update(const NVTETensor amax_history,
                                                       const NVTETensor scale,
                                                       const NVTETensor scale_inv,
                                                       const NVTETensor scale_inv_mask,
                                                       NVTETensor updated_amax_history,
                                                       NVTETensor updated_scale,
                                                       NVTETensor updated_scale_inv,
                                                       const char *amax_compute_algo,
                                                       NVTEDType fp8_dtype,
                                                       float margin,
                                                       hipStream_t stream) {
  NVTE_API_CALL(nvte_delayed_scaling_recipe_amax_and_scale_update);
  using namespace transformer_engine;
  delayed_scaling_recipe::amax_and_scale_update(
    *reinterpret_cast<const Tensor*>(amax_history),
    *reinterpret_cast<const Tensor*>(scale),
    *reinterpret_cast<const Tensor*>(scale_inv),
    *reinterpret_cast<const Tensor*>(scale_inv_mask),
    reinterpret_cast<Tensor*>(updated_amax_history),
    reinterpret_cast<Tensor*>(updated_scale),
    reinterpret_cast<Tensor*>(updated_scale_inv),
    amax_compute_algo,
    static_cast<DType>(fp8_dtype),
    margin,
    stream);
}


void nvte_delayed_scaling_recipe_amax_and_scale_update_after_reduction(
                           const NVTETensor amax_reduction_buffer,
                           std::vector<NVTETensor> amax_histories,
                           std::vector<NVTETensor> scales,
                           std::vector<NVTETensor> scale_invs,
                           const char *amax_compute_algo,
                           NVTEDType fp8_dtype,
                           float margin,
                           hipStream_t stream) {
  NVTE_API_CALL(nvte_delayed_scaling_recipe_amax_and_scale_update_after_reduction);
  using namespace transformer_engine;
  size_t num_tensors = amax_histories.size();
  std::vector<Tensor*> t_amax_histories, t_scales, t_scale_invs;
  for (size_t i = 0; i < num_tensors; i++) {
    t_amax_histories.push_back(reinterpret_cast<Tensor*>(amax_histories[i]));
    t_scales.push_back(reinterpret_cast<Tensor*>(scales[i]));
    t_scale_invs.push_back(reinterpret_cast<Tensor*>(scale_invs[i]));
  }
  delayed_scaling_recipe::amax_and_scale_update_after_reduction(
    *reinterpret_cast<const Tensor*>(amax_reduction_buffer),
    t_amax_histories,
    t_scales,
    t_scale_invs,
    amax_compute_algo,
    static_cast<DType>(fp8_dtype),
    margin,
    stream);
}
