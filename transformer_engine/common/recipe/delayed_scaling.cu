#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <transformer_engine/recipe.h>

#include <cmath>
#include <string>

#include "../common.h"
#include "../util/logging.h"
#include "../util/hip/hip_runtime.h"

namespace transformer_engine {
namespace delayed_scaling_recipe {

namespace {

// amax value to use for updating scaling factor
enum class AmaxComputeAlgo { INVALID, MOST_RECENT, MAX };

const char* dtype_name(DType dtype) {
  TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(dtype, Type,
    return TypeInfo<Type>::name;
  );  // NOLINT(*)
  return "";
}

// Maximum representable value of an FP8 dtype
inline float fp8_dtype_max(DType dtype) {
  switch (dtype) {
  case DType::kFloat8E4M3: return 448;
  case DType::kFloat8E5M2: return 57344;
  default:
    NVTE_ERROR("Expected FP8 dtype, but got ", dtype_name(dtype));
  }
  return 0;
}

// struct for amax parameters
struct AmaxParam {
  int num_scale = 0;
  float* amax_history = nullptr;
  float* scale = nullptr;
  float* scale_inv = nullptr;
};

// dummy struct for kernel_bulk's other params
struct OtherParams {
  float* a;
  size_t b;
  AmaxComputeAlgo c;
  float d;
};

#if CUDART_VERSION >= 12010
constexpr size_t max_constant_memory_per_kernel = 32768;
constexpr size_t AMAX_PARAMS_LIMIT = (
  max_constant_memory_per_kernel - sizeof(OtherParams)) / sizeof(AmaxParam);
#else
constexpr size_t max_constant_memory_per_kernel = 4096;
constexpr size_t AMAX_PARAMS_LIMIT = (
  max_constant_memory_per_kernel - sizeof(OtherParams)) / sizeof(AmaxParam);
#endif

struct AmaxParams {
  AmaxParam param[AMAX_PARAMS_LIMIT];
};

namespace amax_and_scale_update_impl {

// CUDA block size
constexpr size_t bsize = 256;

/* CUDA kernel to bulk-update amax history and FP8 scaling factors
 *
 * Block dims: bsize x 1 x 1
 *
 * Grid dims: num_tensors x 1 x 1
 */
__global__ void __launch_bounds__(bsize)
kernel_bulk(
       float* amax_reduction_buffer,
       AmaxParams p,
       size_t amax_history_length,
       AmaxComputeAlgo amax_compute_algo,
       float scaled_max) {
  const size_t bid = blockIdx.x;
  const size_t tid = threadIdx.x;
  const int num_scale = p.param[bid].num_scale;

  int offset_in_buffer = 0;
  for (int j = 0; j < bid; j++) {
    offset_in_buffer += p.param[j].num_scale;
  }

  for (int count = 0; count < num_scale; count++) {
    // Update amax
    float amax = 0;
    {
      // Roll amax history
      const auto& length = amax_history_length;
      const auto& stride = p.param[bid].num_scale;
      auto* amax_history = p.param[bid].amax_history+count;
      const auto last_amax = ((amax_reduction_buffer != nullptr)
            && (amax_reduction_buffer[offset_in_buffer+count] != 0.0f)) ?
            amax_reduction_buffer[offset_in_buffer+count] : amax_history[0];
      for (size_t off = 0; off < length; off += bsize) {
        const size_t i = off + tid;
        float a = 0;
        if (i < length) {
          a = (i < length - 1) ? amax_history[(i+1)*stride] : last_amax;
          amax = fmaxf(amax, a);
        }
        __syncthreads();  // Inplace roll
        if (i < length) {
          amax_history[i*stride] = (i > 0) ? a : 0;
        }
      }

      // Compute amax to use for scaling factor
      switch (amax_compute_algo) {
      case AmaxComputeAlgo::MOST_RECENT:
        amax = last_amax;
        break;
      case AmaxComputeAlgo::MAX:
        {
          __shared__ float shared_amax[bsize];
          shared_amax[tid] = amax;
          __syncthreads();
#pragma unroll
          for (size_t off = bsize / 2; off > 0; off /= 2) {
            if (tid < off) {
              shared_amax[tid] = fmaxf(shared_amax[tid], shared_amax[tid + off]);
            }
            __syncthreads();
          }
          amax = shared_amax[tid];
        }
        break;
      default:
        amax = 0;
      }
    }

    // Update scale and scale inverse
    if (tid == 0) {
      float scale;
      if (isfinite(amax) && amax > 0) {
        scale = scaled_max / amax;
      } else {
        scale = p.param[bid].scale[count];
      }
      p.param[bid].scale[count] = scale;
      p.param[bid].scale_inv[count] = 1 / scale;
    }
  }
}

}  // namespace amax_and_scale_update_impl

}  // namespace


void amax_and_scale_update_after_reduction(const Tensor &amax_reduction_buffer,
                                           std::vector<Tensor*> amax_histories,
                                           std::vector<Tensor*> scales,
                                           std::vector<Tensor*> scale_invs,
                                           const std::string &amax_compute_algo,
                                           DType fp8_dtype,
                                           float margin,
                                           hipStream_t stream) {
  using namespace transformer_engine;

  // amax value to use for updating scaling factor
  AmaxComputeAlgo amax_compute_algo_ = AmaxComputeAlgo::INVALID;
  if (amax_compute_algo == "max") {
    amax_compute_algo_ = AmaxComputeAlgo::MAX;
  } else if (amax_compute_algo == "most_recent") {
    amax_compute_algo_ = AmaxComputeAlgo::MOST_RECENT;
  } else {
    NVTE_ERROR("Unsupported amax compute algorithm (", amax_compute_algo, ")");
  }

  // Expected maximum value after scale is applied
  const float scaled_max = fp8_dtype_max(fp8_dtype) * std::pow(2.f, -margin);

  // Number of elements in tensor
  auto numel = [] (const Tensor *tensor) -> size_t {
    size_t acc = 1;
    for (const auto& dim : tensor->data.shape) {
      acc *= dim;
    }
    return acc;
  };

  // Number of tensors in the bulk
  const size_t num_tensors = amax_histories.size();
  const int num_kernels = (num_tensors+AMAX_PARAMS_LIMIT-1)/AMAX_PARAMS_LIMIT;
  size_t amax_history_length = 0;
  if (num_tensors > 0) {
    amax_history_length = amax_histories[0]->data.shape[0];
  }

  // amax parameters
  float* amax_buffer = static_cast<float*>(amax_reduction_buffer.data.dptr);
  AmaxParams p;
  for (int iter = 0; iter < num_kernels; iter++) {
    size_t kernel_num_scales = 0;
    size_t kernel_num_tensors = (iter == (num_kernels -1))
          ? num_tensors % AMAX_PARAMS_LIMIT: AMAX_PARAMS_LIMIT;
    for (size_t pi = 0; pi < kernel_num_tensors; pi++) {
      size_t i = iter * AMAX_PARAMS_LIMIT + pi;

      // Check tensors
      int num_scale = amax_histories[i]->data.shape[1];
      NVTE_CHECK(amax_histories[i]->data.dtype == DType::kFloat32,
                 "Found ", dtype_name(amax_histories[i]->data.dtype), ".");
      NVTE_CHECK(amax_histories[i]->data.shape.size() == 2,
                 "Found ", amax_histories[i]->data.shape.size(), " dims");
      NVTE_CHECK(numel(amax_histories[i]) == amax_history_length * num_scale,
                 "Expected ", amax_history_length * num_scale, " elements, ",
                 "but found ", numel(amax_histories[i]), ".");
      NVTE_CHECK(scales[i]->data.dtype == DType::kFloat32,
                 "Found ", dtype_name(scales[i]->data.dtype), ".");
      NVTE_CHECK(scales[i]->data.shape.size() == 1,
                 "Found ", scales[i]->data.shape.size(), " dims");
      NVTE_CHECK(numel(scales[i]) == num_scale,
                 "Expected ", num_scale, " elements, ",
                 "Found ", numel(scales[i]), ".");

      // amax parameters
      kernel_num_scales += num_scale;
      p.param[pi].num_scale = num_scale;
      p.param[pi].amax_history = static_cast<float*>(amax_histories[i]->data.dptr);
      p.param[pi].scale = static_cast<float*>(scales[i]->data.dptr);
      p.param[pi].scale_inv = static_cast<float*>(scale_invs[i]->data.dptr);
    }

    // Launch CUDA kernel
    size_t grid_size = kernel_num_tensors;
    const size_t block_size = amax_and_scale_update_impl::bsize;
    amax_and_scale_update_impl::kernel_bulk
      <<<grid_size, block_size, 0, stream>>>(
        amax_buffer,
        p,
        amax_history_length,
        amax_compute_algo_,
        scaled_max);
    NVTE_CHECK_CUDA(hipGetLastError());

    // shift amax buffer pointer
    if (amax_buffer != nullptr) {
      amax_buffer += kernel_num_scales;
    }
  }
}


}  // namespace delayed_scaling_recipe
}  // namespace transformer_engine


void nvte_delayed_scaling_recipe_amax_and_scale_update_after_reduction(
                           const NVTETensor amax_reduction_buffer,
                           std::vector<NVTETensor> amax_histories,
                           std::vector<NVTETensor> scales,
                           std::vector<NVTETensor> scale_invs,
                           const char *amax_compute_algo,
                           NVTEDType fp8_dtype,
                           float margin,
                           hipStream_t stream) {
  NVTE_API_CALL(nvte_delayed_scaling_recipe_amax_and_scale_update_after_reduction);
  using namespace transformer_engine;
  size_t num_tensors = amax_histories.size();
  std::vector<Tensor*> t_amax_histories, t_scales, t_scale_invs;
  for (size_t i = 0; i < num_tensors; i++) {
    t_amax_histories.push_back(reinterpret_cast<Tensor*>(amax_histories[i]));
    t_scales.push_back(reinterpret_cast<Tensor*>(scales[i]));
    t_scale_invs.push_back(reinterpret_cast<Tensor*>(scale_invs[i]));
  }
  delayed_scaling_recipe::amax_and_scale_update_after_reduction(
    *reinterpret_cast<const Tensor*>(amax_reduction_buffer),
    t_amax_histories,
    t_scales,
    t_scale_invs,
    amax_compute_algo,
    static_cast<DType>(fp8_dtype),
    margin,
    stream);
}
