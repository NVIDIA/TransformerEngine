#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <transformer_engine/activation.h>
#include <hip/hip_runtime.h>
#include <cfloat>
#include <iostream>
#include "../utils.cuh"
#include "../common.h"
#include <cstdlib>
#include <../util/vectorized_pointwise.h>

namespace transformer_engine {

namespace detail {

struct GELUParam {};

__device__ inline fp32 gelu(fp32 value, const GELUParam &) {
  return value * (0.5F + 0.5F * tanhf(value * (0.79788456F + 0.03567741F * value * value)));
}

}

void gelu_cast(const Tensor &input,
               Tensor *output,
               hipStream_t stream) {
  CheckInputTensor(input, "gelu_input");
  CheckOutputTensor(*output, "gelu_output");
  NVTE_CHECK(input.data.shape.size() == 2, "Input must have 2 dimensions.");
  NVTE_CHECK(output->data.shape.size() == 2, "Output must have 2 dimensions.");
  NVTE_CHECK(input.data.shape == output->data.shape, "Input and output shapes must match.");
  const size_t tot_elts = input.data.shape[1] * input.data.shape[0];

  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(input.data.dtype, IType,
    TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(output->data.dtype, OType,
      constexpr int nvec = 32 / sizeof(IType);
      VectorizedUnaryKernelLauncher<nvec, detail::GELUParam, detail::gelu>(
        reinterpret_cast<const IType*>(input.data.dptr),
        reinterpret_cast<OType*>(output->data.dptr),
        reinterpret_cast<const fp32*>(output->scale.dptr),
        reinterpret_cast<fp32*>(output->scale_inv.dptr),
        reinterpret_cast<fp32*>(output->amax.dptr),
        tot_elts,
        {},
        stream);
    );  // NOLINT(*)
  );  // NOLINT(*)
}

}  // namespace transformer_engine

void nvte_gelu(const NVTETensor input,
               NVTETensor output,
               hipStream_t stream) {
  using namespace transformer_engine;
  gelu_cast(*reinterpret_cast<const Tensor*>(input),
            reinterpret_cast<Tensor*>(output),
            stream);
}
