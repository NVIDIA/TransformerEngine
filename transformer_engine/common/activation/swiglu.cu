/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "../util/math.h"
#include "./activation_template.h"

void nvte_silu(const NVTETensor input, NVTETensor output, hipStream_t stream) {
  NVTE_API_CALL(nvte_silu);
  using namespace transformer_engine;
  act_fn<fp32, Empty, silu<fp32, fp32>>(input, output, stream);
}

void nvte_dsilu(const NVTETensor grad, const NVTETensor input, NVTETensor output,
                hipStream_t stream) {
  NVTE_API_CALL(nvte_dsilu);
  using namespace transformer_engine;
  dact_fn<fp32, Empty, dsilu<fp32, fp32>>(grad, input, output, stream);
}

void nvte_swiglu(const NVTETensor input, NVTETensor output, hipStream_t stream) {
  NVTE_API_CALL(nvte_swiglu);
  using namespace transformer_engine;
  Empty e = {};
  gated_act_fn<fp32, Empty, silu<fp32, fp32>>(input, output, e, stream);
}

void nvte_dswiglu(const NVTETensor grad, const NVTETensor input, NVTETensor output,
                  hipStream_t stream) {
  NVTE_API_CALL(nvte_dswiglu);
  using namespace transformer_engine;
  Empty e = {};
  dgated_act_fn<fp32, Empty, silu<fp32, fp32>, dsilu<fp32, fp32>>(grad, input, output, e, stream);
}

void nvte_clamped_swiglu(const NVTETensor input, NVTETensor output, float limit, float alpha,
                         hipStream_t stream) {
  NVTE_API_CALL(nvte_clamped_swiglu);
  using namespace transformer_engine;
  ClampedSwiGLUParam param = {limit, alpha};
  gated_act_fn<fp32, ClampedSwiGLUParam, oss_silu<fp32, fp32>>(input, output, param, stream);
}

void nvte_clamped_dswiglu(const NVTETensor grad, const NVTETensor input, NVTETensor output,
                          float limit, float alpha, hipStream_t stream) {
  NVTE_API_CALL(nvte_clamped_dswiglu);
  using namespace transformer_engine;
  ClampedSwiGLUParam param = {limit, alpha};
  dgated_act_fn<fp32, ClampedSwiGLUParam, oss_silu<fp32, fp32>, oss_dsilu<fp32, fp32>>(
      grad, input, output, param, stream);
}
