/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <transformer_engine/activation.h>
#include <hip/hip_runtime.h>
#include "../util/vectorized_pointwise.h"
#include "../util/math.h"
#include "../common.h"


namespace transformer_engine {

void swiglu(const Tensor &input,
            Tensor *output,
            hipStream_t stream) {
  CheckInputTensor(input, "geglu_input");
  CheckOutputTensor(*output, "geglu_output");
  NVTE_CHECK(input.data.shape.size() == 2, "Input must have 2 dimensions.");
  NVTE_CHECK(output->data.shape.size() == 2, "Output must have 2 dimensions.");
  NVTE_CHECK(input.data.shape[0] == output->data.shape[0],
             "Input shape[0] must be equal to output shape[0].");
  NVTE_CHECK(input.data.shape[1] == output->data.shape[1] * 2,
             "Input shape[1] must be 2x larger than output shape[1].");

  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(input.data.dtype, IType,
    TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(output->data.dtype, OType,
      constexpr int nvec = 32 / sizeof(IType);
      GatedActivationKernelLauncher<nvec, fp32, Empty, swish<fp32, fp32>>(
        reinterpret_cast<const IType*>(input.data.dptr),
        reinterpret_cast<OType*>(output->data.dptr),
        reinterpret_cast<const fp32*>(output->scale.dptr),
        reinterpret_cast<fp32*>(output->amax.dptr),
        output->data.shape[0],
        output->data.shape[1],
        {},
        stream);
    );  // NOLINT(*)
  );  // NOLINT(*)
}

void dswiglu(const Tensor &grad,
             const Tensor &input,
             Tensor *output,
             hipStream_t stream) {
  CheckInputTensor(grad, "dswiglu_grad");
  CheckInputTensor(input, "dswiglu_input");
  CheckOutputTensor(*output, "dswiglu_output");
  NVTE_CHECK(grad.data.shape.size() == 2, "Grad must have 2 dimensions.");
  NVTE_CHECK(input.data.shape.size() == 2, "Input must have 2 dimensions.");
  NVTE_CHECK(output->data.shape.size() == 2, "Output must have 2 dimensions.");
  NVTE_CHECK(output->data.shape[0] == grad.data.shape[0],
             "Output shape[0] must be equal to grad shape[0].");
  NVTE_CHECK(output->data.shape[1] == grad.data.shape[1] * 2,
             "Output shape[1] must be 2x larger than grad shape[1].");
  NVTE_CHECK(input.data.shape == output->data.shape,
             "Input and output shapes must match.");

  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(input.data.dtype, IType,
    TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(output->data.dtype, OType,
      constexpr int nvec = 32 / sizeof(IType);
      DGatedActivationKernelLauncher<nvec, fp32, Empty, swish<fp32, fp32>, dswish<fp32, fp32>>(
        reinterpret_cast<const IType*>(grad.data.dptr),
        reinterpret_cast<const IType*>(input.data.dptr),
        reinterpret_cast<OType*>(output->data.dptr),
        grad.data.shape[0],
        grad.data.shape[1],
        {},
        stream);
    );  // NOLINT(*)
  );  // NOLINT(*)
}

}  // namespace transformer_engine

void nvte_swiglu(const NVTETensor input,
                 NVTETensor output,
                 hipStream_t stream) {
  NVTE_API_CALL(nvte_swiglu);
  using namespace transformer_engine;
  swiglu(*reinterpret_cast<const Tensor*>(input),
         reinterpret_cast<Tensor*>(output),
         stream);
}

void nvte_dswiglu(const NVTETensor grad,
                  const NVTETensor input,
                  NVTETensor output,
                  hipStream_t stream) {
  NVTE_API_CALL(nvte_dswiglu);
  using namespace transformer_engine;
  dswiglu(*reinterpret_cast<const Tensor*>(grad),
          *reinterpret_cast<const Tensor*>(input),
          reinterpret_cast<Tensor*>(output),
          stream);
}
