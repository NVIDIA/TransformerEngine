#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <assert.h>
#include <stdint.h>

#include <cfloat>
#include <limits>
#include <array>
#include <functional>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <transformer_engine/softmax.h>
#include "../common.h"
#include "../utils.cuh"
#include "../util/logging.h"


namespace transformer_engine {

template<typename Datatype, int ELEMENTS_PER_LDG>
__device__ __inline__ void copy_vector(Datatype *dst, const Datatype *src);

template<>
__device__ __inline__ void copy_vector<bf16, 1>(bf16 *dst, const bf16 *src) {
    *dst = *src;
}

template<>
__device__ __inline__ void copy_vector<bf16, 4>(bf16 *dst, const bf16 *src) {
    *((uint64_t*) dst) = *((uint64_t*) src);    // NOLINT(*)
}

template<>
__device__ __inline__ void copy_vector<fp16, 1>(fp16 *dst, const fp16 *src) {
    *dst = *src;
}

template<>
__device__ __inline__ void copy_vector<fp16, 4>(fp16 *dst, const fp16 *src) {
    *((uint64_t*) dst) = *((uint64_t*) src);    // NOLINT(*)
}

template<>
__device__ __inline__ void copy_vector<uint8_t, 1>(uint8_t *dst, const uint8_t *src) {
    *dst = *src;
}

template<>
__device__ __inline__ void copy_vector<uint8_t, 4>(uint8_t *dst, const uint8_t *src) {
    *((uint32_t*) dst) = *((uint32_t*) src);      // NOLINT(*)
}

template <typename Datatype, int ELEMENTS_PER_LDG>
__device__ __inline__ void copy_zero_vector(Datatype *dst);

template <>
__device__ __inline__ void copy_zero_vector<bf16, 1>(bf16 *dst) {
    *dst = 0.0f;
}

template <>
__device__ __inline__ void copy_zero_vector<bf16, 4>(bf16 *dst) {
    *((float2*) dst) = make_float2(0.0f, 0.0f);  // NOLINT(*)
}

template <>
__device__ __inline__ void copy_zero_vector<fp16, 1>(fp16 *dst) {
    *dst = 0.0f;
}

template <>
__device__ __inline__ void copy_zero_vector<fp16, 4>(fp16 *dst) {
    *((float2*) dst) = make_float2(0.0f, 0.0f);  // NOLINT(*)
}


template<typename T>
struct Add {
    __device__ __forceinline__ T operator()(T a, T b) const {
        return a + b;
    }
};

template<typename T>
struct Max {
    __device__ __forceinline__ T operator()(T a, T b) const {
        return a < b ? b : a;
    }
};

template<typename T>
__device__ __forceinline__ T WARP_SHFL_XOR_NATIVE(T value, int laneMask, int width = warpSize,
                                                  unsigned int mask = 0xffffffff) {
#if CUDA_VERSION >= 9000
    return __shfl_xor_sync(mask, value, laneMask, width);
#else
    return __shfl_xor(value, laneMask, width);
#endif
}

template<typename acc_t, int WARP_ROWS, int WARP_SIZE, template<typename> class ReduceOp>
__device__ __forceinline__ void warp_reduce(acc_t* sum) {
    ReduceOp<acc_t> r;
    #pragma unroll
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        #pragma unroll
        for (int i = 0;  i < WARP_ROWS;  ++i) {
            acc_t b = WARP_SHFL_XOR_NATIVE(sum[i], offset, WARP_SIZE);
            sum[i] = r(sum[i], b);
        }
    }
}

/*
 * Extended softmax (from native aten pytorch) with the following additional features
 * 1) input scaling
 * 2) implicit causal masking
 * 
 * works for all cases:
 *  k > q
 *  k < q
 *  k = q
 * 
 * where:
 * microbatches = batches * attn_heads * query_seq_len
 * rows = query_seq_len
 * cols = key_seq_len
 */
template <typename input_t, typename output_t, typename acc_t, int log2_elements>
__global__ void scaled_aligned_causal_masked_softmax_warp_forward(
    output_t *dst,
    const input_t *src,
    const acc_t scale,
    const int microbatches,
    const int rows,
    const int cols
) {
    // 1) WARP_WIDTH must match the value of warp_size
    // 2) WARP_ROWS must match the value of rows_per_warp
    // of the dispatch_scaled_aligned_causal_masked_softmax_forward method.
    constexpr int next_power_of_two = 1 << log2_elements;
    constexpr int WARP_WIDTH = (next_power_of_two < THREADS_PER_WARP) ? next_power_of_two
                                                                      : THREADS_PER_WARP;
    constexpr int WARP_ITERATIONS = next_power_of_two / WARP_WIDTH;
    constexpr int WARP_ROWS = (next_power_of_two <= 128) ? 2 : 1;
    constexpr int ELEMENTS_PER_LDG_STG = (WARP_ITERATIONS < 4) ? 1 : 4;

    const int global_row_idx = (blockIdx.x * blockDim.y + threadIdx.y) * WARP_ROWS;
    const int col = threadIdx.x * ELEMENTS_PER_LDG_STG;

    const size_t thread_offset = global_row_idx * cols + col;

    src += thread_offset;
    dst += thread_offset;

    // load data from global memory into registers WITH scaling
    acc_t elements[WARP_ROWS][WARP_ITERATIONS];
    input_t temp_data[ELEMENTS_PER_LDG_STG];

    #pragma unroll
    for (int w = 0; w < WARP_ROWS; ++w) {
        const int microbatch = global_row_idx + w;
        const int i = microbatch % rows;                    // local row index of attention matrix
        const int masked_elements = i + cols - rows + 1;

        if (microbatch >= microbatches) {
            break;
        }

        #pragma unroll
        for (int it = 0;  it < WARP_ITERATIONS;  it += ELEMENTS_PER_LDG_STG) {
            const int j = col + it * WARP_WIDTH;
            const int itr_idx = w * cols + it * WARP_WIDTH;

            if (j < masked_elements) {
                copy_vector<input_t, ELEMENTS_PER_LDG_STG>(temp_data, src + itr_idx);
                #pragma unroll
                for (int element = 0; element < ELEMENTS_PER_LDG_STG; ++element) {
                    if (j + element < masked_elements) {
                        elements[w][it + element] = (acc_t)temp_data[element] * scale;
                    } else {
                        elements[w][it + element] = (acc_t)( -10'000 );
                    }
                }
            } else {
                #pragma unroll
                for (int element = 0; element < ELEMENTS_PER_LDG_STG; ++element) {
                    elements[w][it + element] = (acc_t)( -10'000 );
                }
            }
        }
    }

    // compute max_value
    acc_t max_value[WARP_ROWS];
    #pragma unroll
    for (int w = 0;  w < WARP_ROWS;  ++w) {
        max_value[w] = elements[w][0];
        #pragma unroll
        for (int it = 1;  it < WARP_ITERATIONS;  ++it) {
            max_value[w] =
                (max_value[w] > elements[w][it]) ? max_value[w] : elements[w][it];
        }
    }
    warp_reduce<acc_t, WARP_ROWS, WARP_WIDTH, Max>(max_value);

    acc_t sum[WARP_ROWS] { 0.0f };
    #pragma unroll
    for (int w = 0;  w < WARP_ROWS;  ++w) {
        #pragma unroll
        for (int it = 0;  it < WARP_ITERATIONS;  ++it) {
            elements[w][it] = expf((elements[w][it] - max_value[w]));
            sum[w] += elements[w][it];
        }
    }
    warp_reduce<acc_t, WARP_ROWS, WARP_WIDTH, Add>(sum);

    output_t out[ELEMENTS_PER_LDG_STG] { 0.0f };
    // store result
    #pragma unroll
    for (int w = 0;  w < WARP_ROWS;  ++w) {
        const int microbatch = global_row_idx + w;
        const int i = microbatch % rows;
        const int masked_elements = i + cols - rows + 1;

        // out of Attention matrix bounds (rows)
        if (microbatch >= microbatches) {
            break;
        }

        #pragma unroll
        for (int it = 0;  it < WARP_ITERATIONS;  it += ELEMENTS_PER_LDG_STG) {
            const int j = col + it * WARP_WIDTH;              // index of the first column
            const int itr_idx = w * cols + it * WARP_WIDTH;

            if (j < masked_elements) {
                #pragma unroll
                for (int element = 0; element < ELEMENTS_PER_LDG_STG; ++element) {
                    if (j + element < masked_elements) {
                        out[element] = elements[w][it + element] / sum[w];
                    } else {
                        out[element] = (output_t)( 0.0f );
                    }
                }
                copy_vector<output_t, ELEMENTS_PER_LDG_STG>(dst + itr_idx, out);
            } else if (j < cols) {
                copy_zero_vector<output_t, ELEMENTS_PER_LDG_STG>(dst + itr_idx);
            } else {
                break;
            }
        }
    }
}


template <typename input_t, typename output_t, typename acc_t, int log2_elements>
__global__ void scaled_aligned_causal_masked_softmax_warp_backward(
    output_t *gradInput,
    const input_t *grad,
    const input_t *softmax_output,
    const acc_t scale,
    const int microbatches,
    const int rows,
    const int cols
) {
    // 1) WARP_WIDTH must match the value of warp_size
    // 2) WARP_ROWS must match the value of rows_per_warp
    // of the dispatch_scaled_aligned_causal_masked_softmax_forward method.
    constexpr int next_power_of_two = 1 << log2_elements;
    constexpr int WARP_WIDTH = (next_power_of_two < THREADS_PER_WARP) ? next_power_of_two
                                                                      : THREADS_PER_WARP;
    constexpr int WARP_ITERATIONS = next_power_of_two / WARP_WIDTH;
    constexpr int WARP_ROWS = (next_power_of_two <= 128) ? 2 : 1;
    constexpr int ELEMENTS_PER_LDG_STG = (WARP_ITERATIONS < 4) ? 1 : 4;

    const int global_row_idx = (blockIdx.x * blockDim.y + threadIdx.y) * WARP_ROWS;
    const int col = threadIdx.x * ELEMENTS_PER_LDG_STG;

    const size_t thread_offset = global_row_idx * cols + col;

    grad += thread_offset;
    softmax_output += thread_offset;
    gradInput += thread_offset;

    // load data from global memory into registers
    acc_t grad_reg[WARP_ROWS][WARP_ITERATIONS] { 0.0f };
    acc_t softmax_output_reg[WARP_ROWS][WARP_ITERATIONS] { 0.0f };
    input_t temp_grad[ELEMENTS_PER_LDG_STG];
    input_t temp_output[ELEMENTS_PER_LDG_STG];

    #pragma unroll
    for (int w = 0; w < WARP_ROWS; ++w) {
        const int microbatch = global_row_idx + w;
        const int i = microbatch % rows;                    // local row index of attention matrix
        const int masked_elements = i + cols - rows + 1;

        if (microbatch >= microbatches) {
            break;
        }

        #pragma unroll
        for (int it = 0;  it < WARP_ITERATIONS;  it += ELEMENTS_PER_LDG_STG) {
            const int j = col + it * WARP_WIDTH;                // index of the first column
            const int itr_idx = w * cols + it * WARP_WIDTH;

            if (j < masked_elements) {
                copy_vector<input_t, ELEMENTS_PER_LDG_STG>(temp_grad, grad + itr_idx);
                copy_vector<input_t, ELEMENTS_PER_LDG_STG>(temp_output, softmax_output + itr_idx);
                #pragma unroll
                for (int element = 0; element < ELEMENTS_PER_LDG_STG; ++element) {
                    if (j + element < masked_elements) {
                        softmax_output_reg[w][it + element] = (acc_t)temp_output[element];
                        grad_reg[w][it + element] =
                            (acc_t)temp_grad[element] * softmax_output_reg[w][it + element];
                    }
                }
            }
        }
    }

    acc_t sum[WARP_ROWS];
    #pragma unroll
    for (int w = 0; w < WARP_ROWS; ++w) {
        sum[w] = grad_reg[w][0];
        #pragma unroll
        for (int it = 1;  it < WARP_ITERATIONS;  ++it) {
            sum[w] += grad_reg[w][it];
        }
    }

    warp_reduce<acc_t, WARP_ROWS, WARP_WIDTH, Add>(sum);

    // store result
    #pragma unroll
    for (int w = 0;  w < WARP_ROWS;  ++w) {
        const int microbatch = global_row_idx + w;
        if (microbatch >= microbatches) {
            break;
        }

        #pragma unroll
        for (int it = 0;  it < WARP_ITERATIONS;  it += ELEMENTS_PER_LDG_STG) {
            const int j = col + it * WARP_WIDTH;              // index of the first column
            const int itr_idx = w * cols + it * WARP_WIDTH;

            if (j < cols) {
                output_t out[ELEMENTS_PER_LDG_STG];
                #pragma unroll
                for (int element = 0; element < ELEMENTS_PER_LDG_STG; ++element) {
                    out[element] = (output_t)(scale * (grad_reg[w][it + element] -
                                                    softmax_output_reg[w][it + element] * sum[w]));
                }
                copy_vector<output_t, ELEMENTS_PER_LDG_STG>(gradInput + itr_idx, out);
            }
        }
    }
}

template<typename input_t, typename output_t, typename acc_t, int log2_elements>
void call_kernel_scaled_aligned_causal_masked_softmax_forward(
    dim3 grid_size,
    dim3 block_size,
    const int shmem_size,
    hipStream_t stream,
    output_t *dst,
    const input_t *src,
    const acc_t scale,
    const int microbatches,
    const int query_seq_len,
    const int key_seq_len
) {
    scaled_aligned_causal_masked_softmax_warp_forward<input_t, output_t, acc_t, log2_elements>
        <<<grid_size, block_size, shmem_size, stream>>>(
            dst, src, scale, microbatches, query_seq_len, key_seq_len);
}

template<typename input_t, typename output_t, typename acc_t, int log2_elements>
void call_kernel_scaled_aligned_causal_masked_softmax_backward(
    dim3 grid_size,
    dim3 block_size,
    const int shmem_size,
    hipStream_t stream,
    output_t *gradInput,
    const input_t *grad,
    const input_t *output,
    const acc_t scale,
    const int microbatches,
    const int query_seq_len,
    const int key_seq_len
) {
    scaled_aligned_causal_masked_softmax_warp_backward<input_t, output_t, acc_t, log2_elements>
        <<<grid_size, block_size, 0, stream>>>(
            gradInput, grad, output, scale, microbatches, query_seq_len, key_seq_len);
}

template<typename input_t, typename output_t, typename acc_t>
struct FunctionWrapper {
    using ForwardType = std::function<
        void(
            dim3 grid_size,
            dim3 block_size,
            const int shmem_size,
            hipStream_t stream,
            output_t *dst,
            const input_t *src,
            const acc_t scale,
            const int microbatches,
            const int query_seq_len,
            const int key_seq_len
        )
    >;
    using BackwardType = std::function<
        void(
            dim3 grid_size,
            dim3 block_size,
            const int shmem_size,
            hipStream_t stream,
            output_t *gradInput,
            const input_t *grad,
            const input_t *output,
            const acc_t scale,
            const int microbatches,
            const int query_seq_len,
            const int key_seq_len
        )
    >;
};


constexpr int MIN_SUPPORTED_POWER = 4;
constexpr int MAX_SUPPORTED_POWER = 14;
constexpr int MIN_POWER = MIN_SUPPORTED_POWER - 1;
constexpr int MAX_POWER = MAX_SUPPORTED_POWER + 1;

// Recursively instantiate the function for the limit of "log2_elements",
// i.e. "MAX_POWER" defined above.
template <typename input_t, typename output_t, typename acc_t, int log2_elements>
struct CompileTimeLoopForward {
    using ForwardFuncType = typename FunctionWrapper<input_t, output_t, acc_t>::ForwardType;
    static void populate(std::array<ForwardFuncType, MAX_POWER>* arr) {
        CompileTimeLoopForward<input_t, output_t, acc_t, log2_elements - 1>::populate(arr);
        (*arr)[log2_elements] = &call_kernel_scaled_aligned_causal_masked_softmax_forward<
                              output_t, input_t, acc_t, log2_elements>;
    }
};

template <typename input_t, typename output_t, typename acc_t>
struct CompileTimeLoopForward<input_t, output_t, acc_t, MIN_POWER> {
    using ForwardFuncType = typename FunctionWrapper<input_t, output_t, acc_t>::ForwardType;
    static void populate(std::array<ForwardFuncType, MAX_POWER>* arr) {
        (*arr)[MIN_POWER] = nullptr;
    }
};

template <typename input_t, typename output_t, typename acc_t, int log2_elements>
struct CompileTimeLoopBackward {
    using BackwardFuncType = typename FunctionWrapper<input_t, output_t, acc_t>::BackwardType;
    static void populate(std::array<BackwardFuncType, MAX_POWER>* arr) {
        CompileTimeLoopBackward<input_t, output_t, acc_t, log2_elements - 1>::populate(arr);
        (*arr)[log2_elements] = &call_kernel_scaled_aligned_causal_masked_softmax_backward<
                              output_t, input_t, acc_t, log2_elements>;
    }
};

template <typename input_t, typename output_t, typename acc_t>
struct CompileTimeLoopBackward<input_t, output_t, acc_t, MIN_POWER> {
    using BackwardFuncType = typename FunctionWrapper<input_t, output_t, acc_t>::BackwardType;
    static void populate(std::array<BackwardFuncType, MAX_POWER>* arr) {
        (*arr)[MIN_POWER] = nullptr;
    }
};

template<typename input_t, typename output_t, typename acc_t>
void dispatch_scaled_aligned_causal_masked_softmax_forward(
    output_t *dst,
    const input_t *src,
    const input_t scale,
    int query_seq_len,
    int key_seq_len,
    int batches,
    int attn_heads,
    hipStream_t stream
) {
    NVTE_CHECK(key_seq_len >= 0 && key_seq_len <= 16384, "Unsupported shape.");

    if (key_seq_len == 0) {
        return;
    }
    int log2_elements = log2_ceil(key_seq_len);
    const int next_power_of_two = 1 << log2_elements;

    // This value must match the WARP_WIDTH constexpr
    // value computed inside scaled_aligned_causal_masked_softmax_warp_forward.
    int warp_width = (next_power_of_two < THREADS_PER_WARP) ? next_power_of_two
                                                            : THREADS_PER_WARP;

    // This value must match the WARP_ROWS constexpr
    // value computed inside scaled_aligned_causal_masked_softmax_warp_forward.
    int microbatches_per_warp = (next_power_of_two <= 128) ? 2 : 1;

    // use 128 threads per block to maximimize gpu utilization
    constexpr int threads_per_block = 128;

    int warps_per_block = threads_per_block / warp_width;
    int microbatches_per_block = warps_per_block * microbatches_per_warp;
    int microbatches = batches * attn_heads * query_seq_len;
    int blocks = DIVUP(microbatches, microbatches_per_block);

    dim3 block_size(warp_width, warps_per_block);
    dim3 grid_size(blocks);

    // create an array of pointers to functions
    using ForwardFuncType = typename FunctionWrapper<input_t, output_t, acc_t>::ForwardType;
    static std::array<ForwardFuncType, MAX_POWER> forwardFunctionArray;
    static bool is_initialized = false;
    if (!is_initialized) {
        CompileTimeLoopForward<input_t, output_t, acc_t, MAX_SUPPORTED_POWER>::populate(
            &forwardFunctionArray);
        is_initialized = true;
    }
    // Call the corresponding kernel
    forwardFunctionArray[log2_elements](grid_size, block_size, 0, stream, dst, src, scale,
                                        microbatches, query_seq_len, key_seq_len);
}

template<typename input_t, typename output_t, typename acc_t>
void dispatch_scaled_aligned_causal_masked_softmax_backward(
    output_t *grad_input,
    const input_t *grad,
    const input_t *output,
    const acc_t scale,
    int query_seq_len,
    int key_seq_len,
    int batches,
    int attn_heads,
    hipStream_t stream
) {
    NVTE_CHECK(key_seq_len >= 0 && key_seq_len <= 16384, "Unsupported shape.");

    if (key_seq_len == 0) {
        return;
    }
    int log2_elements = log2_ceil(key_seq_len);
    const int next_power_of_two = 1 << log2_elements;

    // This value must match the WARP_WIDTH constexpr
    // value computed inside scaled_aligned_causal_masked_softmax_warp_forward.
    int warp_width = (next_power_of_two < THREADS_PER_WARP) ? next_power_of_two : THREADS_PER_WARP;

    // This value must match the WARP_ROWS constexpr
    // value computed inside scaled_aligned_causal_masked_softmax_warp_forward.
    int microbatches_per_warp = (next_power_of_two <= 128) ? 2 : 1;

    // use 128 threads per block to maximimize gpu utilization
    constexpr int threads_per_block = 128;

    int warps_per_block = threads_per_block / warp_width;
    int microbatches_per_block = warps_per_block * microbatches_per_warp;
    int microbatches = batches * attn_heads * query_seq_len;
    int blocks = DIVUP(microbatches, microbatches_per_block);

    dim3 block_size(warp_width, warps_per_block);
    dim3 grid_size(blocks);

    // create an array of pointers to functions
    using BackwardFuncType = typename FunctionWrapper<input_t, output_t, acc_t>::BackwardType;
    static std::array<BackwardFuncType, MAX_POWER> backwardFunctionArray;
    static bool is_initialized = false;
    if (!is_initialized) {
        CompileTimeLoopBackward<input_t, output_t, acc_t, MAX_SUPPORTED_POWER>::populate(
            &backwardFunctionArray);
        is_initialized = true;
    }
    // Call the corresponding kernel
    backwardFunctionArray[log2_elements](grid_size, block_size, 0, stream, grad_input, grad,
                                         output, scale, microbatches, query_seq_len, key_seq_len);
}


void scaled_aligned_causal_masked_softmax_forward(
    const Tensor &input,
    Tensor *softmax_results,
    float scale_factor,
    hipStream_t stream) {

    const int batches = input.data.shape[0];
    const int attn_heads = input.data.shape[1];
    const int query_seq_len = input.data.shape[2];
    const int key_seq_len = input.data.shape[3];

    TRANSFORMER_ENGINE_TYPE_SWITCH_16BIT(input.data.dtype, softmax_type,
        dispatch_scaled_aligned_causal_masked_softmax_forward<softmax_type, softmax_type, float>(
            reinterpret_cast<softmax_type*>(softmax_results->data.dptr),
            reinterpret_cast<const softmax_type*>(input.data.dptr),
            scale_factor,
            query_seq_len,
            key_seq_len,
            batches,
            attn_heads,
            stream););
}

void scaled_aligned_causal_masked_softmax_backward(
    Tensor output_grads,
    const Tensor incoming_grads,
    const Tensor softmax_results,
    float scale_factor,
    hipStream_t stream) {

    // output grads is a 4d tensor with dimensions [batches, attn_heads, seq_len, seq_len]
    const int batches = output_grads.data.shape[0];
    const int attn_heads = output_grads.data.shape[1];
    const int query_seq_len = output_grads.data.shape[2];
    const int key_seq_len = output_grads.data.shape[3];

    // Softmax Grad
    TRANSFORMER_ENGINE_TYPE_SWITCH_16BIT(output_grads.data.dtype, softmax_type,
        dispatch_scaled_aligned_causal_masked_softmax_backward<softmax_type, softmax_type, float>(
            reinterpret_cast<softmax_type*>(output_grads.data.dptr),
            reinterpret_cast<softmax_type const*>(incoming_grads.data.dptr),
            reinterpret_cast<softmax_type const*>(softmax_results.data.dptr),
            scale_factor,
            query_seq_len,
            key_seq_len,
            batches,
            attn_heads,
            stream););
}
}  // end namespace transformer_engine


void nvte_scaled_aligned_causal_masked_softmax_forward(
    const NVTETensor input,
    NVTETensor softmax_results,
    float scale_factor,
    hipStream_t stream
) {
    NVTE_API_CALL(nvte_scaled_aligned_causal_masked_softmax_forward);
    using namespace transformer_engine;
    scaled_aligned_causal_masked_softmax_forward(
        *reinterpret_cast<const Tensor*>(input),
        reinterpret_cast<Tensor*>(softmax_results),
        scale_factor,
        stream);
}


void nvte_scaled_aligned_causal_masked_softmax_backward(
    const NVTETensor incoming_grads,
    const NVTETensor softmax_results,
    NVTETensor output_grads,
    float scale_factor,
    hipStream_t stream
) {
    NVTE_API_CALL(nvte_scaled_aligned_causal_masked_softmax_backward);
    using namespace transformer_engine;
    scaled_aligned_causal_masked_softmax_backward(
        *reinterpret_cast<Tensor*>(output_grads),
        *reinterpret_cast<const Tensor*>(incoming_grads),
        *reinterpret_cast<const Tensor*>(softmax_results),
        scale_factor,
        stream);
}
