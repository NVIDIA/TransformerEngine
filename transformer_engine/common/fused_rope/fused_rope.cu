#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights
 *reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_runtime.h>
#include <transformer_engine/fused_rope.h>

#include "../common.h"
#include "../util/logging.h"
#include "../utils.cuh"

namespace transformer_engine {

template <typename scalar_t>
__global__ void fused_rope_forward_kernel(
    const scalar_t *src, const float *freqs, scalar_t *dst, const int h,
    const int d, const int d2, const int stride_s, const int stride_b,
    const int stride_h, const int stride_d, const int o_stride_s,
    const int o_stride_b, const int o_stride_h, const int o_stride_d) {
  int s_id = blockIdx.x, b_id = blockIdx.y;
  int offset_block = s_id * stride_s + b_id * stride_b;
  int offset_block_dst = s_id * o_stride_s + b_id * o_stride_b;
#pragma unroll
  for (int d_id = threadIdx.x; d_id < d2; d_id += blockDim.x) {
    float v_cos, v_sin;
    sincosf(freqs[s_id * d2 + d_id], &v_sin, &v_cos);
#pragma unroll
    for (int h_id = threadIdx.y; h_id < h; h_id += blockDim.y) {
      int offset_src = offset_block + h_id * stride_h + d_id * stride_d;
      int offset_dst = offset_block_dst + h_id * o_stride_h + d_id * o_stride_d;
      scalar_t v_src = src[offset_src];
      scalar_t v_src_rotate = (d_id + d2 / 2 < d2)
                                  ? -src[offset_src + (d2 / 2) * stride_d]
                                  : src[offset_src + (d2 / 2 - d2) * stride_d];
      dst[offset_dst] =
          v_src * (scalar_t)v_cos + v_src_rotate * (scalar_t)v_sin;
    }
  }

  // copy the rest
  if (d > d2) {
#pragma unroll
    for (int h_id = threadIdx.y; h_id < h; h_id += blockDim.y) {
      int offset_head = offset_block + h_id * stride_h;
      int offset_head_dst = offset_block_dst + h_id * o_stride_h;
#pragma unroll
      for (int d_id = d2 + threadIdx.x; d_id < d; d_id += blockDim.x) {
        dst[offset_head_dst + d_id * o_stride_d] =
            src[offset_head + d_id * stride_d];
      }
    }
  }
}

template <typename scalar_t>
__global__ void fused_rope_backward_kernel(
    const scalar_t *src, const float *freqs, scalar_t *dst, const int h,
    const int d, const int d2, const int stride_s, const int stride_b,
    const int stride_h, const int stride_d, const int o_stride_s,
    const int o_stride_b, const int o_stride_h, const int o_stride_d) {
  int s_id = blockIdx.x, b_id = blockIdx.y;
  int offset_block = s_id * stride_s + b_id * stride_b;
  int offset_block_dst = s_id * o_stride_s + b_id * o_stride_b;
#pragma unroll
  for (int d_id = threadIdx.x; d_id < d2; d_id += blockDim.x) {
    scalar_t v_cos = cosf(freqs[s_id * d2 + d_id]);
    scalar_t v_sin = (d_id + d2 / 2 < d2)
                         ? sinf(freqs[s_id * d2 + d_id + d2 / 2])
                         : -sinf(freqs[s_id * d2 + d_id + d2 / 2 - d2]);
#pragma unroll
    for (int h_id = threadIdx.y; h_id < h; h_id += blockDim.y) {
      int offset_src = offset_block + h_id * stride_h + d_id * stride_d;
      int offset_dst = offset_block_dst + h_id * o_stride_h + d_id * o_stride_d;
      scalar_t v_src = src[offset_src];
      scalar_t v_src_rotate = (d_id + d2 / 2 < d2)
                                  ? src[offset_src + (d2 / 2) * stride_d]
                                  : src[offset_src + (d2 / 2 - d2) * stride_d];
      dst[offset_dst] = v_src * v_cos + v_src_rotate * v_sin;
    }
  }

  // handle the tail
  if (d > d2) {
#pragma unroll
    for (int h_id = threadIdx.y; h_id < h; h_id += blockDim.y) {
      int offset_head = offset_block + h_id * stride_h;
      int offset_head_dst = offset_block_dst + h_id * o_stride_h;
#pragma unroll
      for (int d_id = d2 + threadIdx.x; d_id < d; d_id += blockDim.x) {
        dst[offset_head_dst + d_id * o_stride_d] =
            src[offset_head + d_id * stride_d];
      }
    }
  }
}

template <typename scalar_t>
void fused_rope_forward_launcher(const scalar_t *input, const float *freqs,
                                 scalar_t *output, const int s, const int b,
                                 const int h, const int d, const int d2,
                                 const int stride_s, const int stride_b,
                                 const int stride_h, const int stride_d,
                                 const int o_stride_s, const int o_stride_b,
                                 const int o_stride_h, const int o_stride_d,
                                 hipStream_t stream) {
  int warps_per_block = h < 16 ? 4 : 8;
  dim3 blocks(s, b);
  dim3 threads(THREADS_PER_WARP, warps_per_block);

  fused_rope_forward_kernel<<<blocks, threads, 0, stream>>>(
      input, freqs, output, h, d, d2, stride_s, stride_b, stride_h, stride_d,
      o_stride_s, o_stride_b, o_stride_h, o_stride_d);
  NVTE_CHECK_CUDA(hipGetLastError());
}

template <typename scalar_t>
void fused_rope_backward_launcher(const scalar_t *incoming_grads,
                                  const float *freqs, scalar_t *output_grads,
                                  const int s, const int b, const int h,
                                  const int d, const int d2, const int stride_s,
                                  const int stride_b, const int stride_h,
                                  const int stride_d, const int o_stride_s,
                                  const int o_stride_b, const int o_stride_h,
                                  const int o_stride_d, hipStream_t stream) {
  int warps_per_block = h < 16 ? 4 : 8;
  dim3 blocks(s, b);
  dim3 threads(THREADS_PER_WARP, warps_per_block);

  fused_rope_backward_kernel<<<blocks, threads, 0, stream>>>(
      incoming_grads, freqs, output_grads, h, d, d2, stride_s, stride_b,
      stride_h, stride_d, o_stride_s, o_stride_b, o_stride_h, o_stride_d);
  NVTE_CHECK_CUDA(hipGetLastError());
}

void fused_rope_forward(const Tensor &input, const Tensor &freqs,
                        Tensor *output, const int s, const int b, const int h,
                        const int d, const int d2, const int stride_s,
                        const int stride_b, const int stride_h,
                        const int stride_d, const int o_stride_s,
                        const int o_stride_b, const int o_stride_h,
                        const int o_stride_d, hipStream_t stream) {
  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(
      input.data.dtype, scalar_t,
      fused_rope_forward_launcher(
          reinterpret_cast<const scalar_t *>(input.data.dptr),
          reinterpret_cast<const float *>(freqs.data.dptr),
          reinterpret_cast<scalar_t *>(output->data.dptr), s, b, h, d, d2,
          stride_s, stride_b, stride_h, stride_d, o_stride_s, o_stride_b,
          o_stride_h, o_stride_d, stream););
}

void fused_rope_backward(const Tensor &incoming_grads, const Tensor &freqs,
                         Tensor *output_grads, const int s, const int b,
                         const int h, const int d, const int d2,
                         const int stride_s, const int stride_b,
                         const int stride_h, const int stride_d,
                         const int o_stride_s, const int o_stride_b,
                         const int o_stride_h, const int o_stride_d,
                         hipStream_t stream) {
  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(
      incoming_grads.data.dtype, scalar_t,
      fused_rope_backward_launcher(
          reinterpret_cast<const scalar_t *>(incoming_grads.data.dptr),
          reinterpret_cast<const float *>(freqs.data.dptr),
          reinterpret_cast<scalar_t *>(output_grads->data.dptr), s, b, h, d, d2,
          stride_s, stride_b, stride_h, stride_d, o_stride_s, o_stride_b,
          o_stride_h, o_stride_d, stream););
}

}  // end namespace transformer_engine

void nvte_fused_rope_forward(const NVTETensor input, const NVTETensor freqs,
                             NVTETensor output, const int s, const int b,
                             const int h, const int d, const int d2,
                             const int stride_s, const int stride_b,
                             const int stride_h, const int stride_d,
                             const int o_stride_s, const int o_stride_b,
                             const int o_stride_h, const int o_stride_d,
                             hipStream_t stream) {
  NVTE_API_CALL(nvte_fused_rope_forward);
  using namespace transformer_engine;
  fused_rope_forward(*reinterpret_cast<const Tensor *>(input),
                     *reinterpret_cast<const Tensor *>(freqs),
                     reinterpret_cast<Tensor *>(output), s, b, h, d, d2,
                     stride_s, stride_b, stride_h, stride_d, o_stride_s,
                     o_stride_b, o_stride_h, o_stride_d, stream);
}

void nvte_fused_rope_backward(const NVTETensor incoming_grads,
                              const NVTETensor freqs, NVTETensor output_grads,
                              const int s, const int b, const int h,
                              const int d, const int d2, const int stride_s,
                              const int stride_b, const int stride_h,
                              const int stride_d, const int o_stride_s,
                              const int o_stride_b, const int o_stride_h,
                              const int o_stride_d, hipStream_t stream) {
  NVTE_API_CALL(nvte_fused_rope_backward);
  using namespace transformer_engine;
  fused_rope_backward(*reinterpret_cast<const Tensor *>(incoming_grads),
                      *reinterpret_cast<const Tensor *>(freqs),
                      reinterpret_cast<Tensor *>(output_grads), s, b, h, d, d2,
                      stride_s, stride_b, stride_h, stride_d, o_stride_s,
                      o_stride_b, o_stride_h, o_stride_d, stream);
}
