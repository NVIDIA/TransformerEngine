#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights
 *reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_runtime.h>
#include <transformer_engine/fused_rope.h>

#include "../common.h"
#include "../util/logging.h"
#include "../utils.cuh"

namespace transformer_engine {

template <typename scalar_t>
__global__ void fused_rope_forward_kernel(
    const scalar_t *src, const scalar_t *cos, const scalar_t *sin,
    scalar_t *dst, const int s, const int b, const int h, const int d,
    const int d2, const int stride_s, const int stride_b, const int stride_h,
    const int stride_d, const int o_stride_s, const int o_stride_b,
    const int o_stride_h, const int o_stride_d) {
  int s_id = blockIdx.x, b_id = blockIdx.y;
  int offset_block = s_id * stride_s + b_id * stride_b;
  int offset_block_dst = s_id * o_stride_s + b_id * o_stride_b;
#pragma unroll
  for (int d_id = threadIdx.x; d_id < d2; d_id += blockDim.x) {
    scalar_t v_cos = cos[s_id * d2 + d_id];
    scalar_t v_sin = sin[s_id * d2 + d_id];
#pragma unroll
    for (int h_id = threadIdx.y; h_id < h; h_id += blockDim.y) {
      int offset_src = offset_block + h_id * stride_h + d_id * stride_d;
      int offset_dst = offset_block_dst + h_id * o_stride_h + d_id * o_stride_d;
      scalar_t v_src = src[offset_src];
      scalar_t v_src_rotate = (d_id + d2 / 2 < d2)
                                  ? -src[offset_src + (d2 / 2) * stride_d]
                                  : src[offset_src + (d2 / 2 - d2) * stride_d];
      dst[offset_dst] = v_src * v_cos + v_src_rotate * v_sin;
    }
  }

  // copy the rest
  if (d > d2) {
#pragma unroll
    for (int h_id = threadIdx.y; h_id < h; h_id += blockDim.y) {
      int offset_head = offset_block + h_id * stride_h;
      int offset_head_dst = offset_block_dst + h_id * o_stride_h;
#pragma unroll
      for (int d_id = d2 + threadIdx.x; d_id < d; d_id += blockDim.x) {
        dst[offset_head_dst + d_id * o_stride_d] =
            src[offset_head + d_id * stride_d];
      }
    }
  }
}

template <typename scalar_t>
__global__ void fused_rope_backward_kernel(
    const scalar_t *src, const scalar_t *cos, const scalar_t *sin,
    scalar_t *dst, const int s, const int b, const int h, const int d,
    const int d2, const int stride_s, const int stride_b, const int stride_h,
    const int stride_d, const int o_stride_s, const int o_stride_b,
    const int o_stride_h, const int o_stride_d) {
  int s_id = blockIdx.x, b_id = blockIdx.y;
  int offset_block = s_id * stride_s + b_id * stride_b;
  int offset_block_dst = s_id * o_stride_s + b_id * o_stride_b;
#pragma unroll
  for (int d_id = threadIdx.x; d_id < d2; d_id += blockDim.x) {
    scalar_t v_cos = cos[s_id * d2 + d_id];
    scalar_t v_sin = (d_id + d2 / 2 < d2)
                         ? sin[s_id * d2 + d_id + d2 / 2]
                         : -sin[s_id * d2 + d_id + d2 / 2 - d2];
#pragma unroll
    for (int h_id = threadIdx.y; h_id < h; h_id += blockDim.y) {
      int offset_src = offset_block + h_id * stride_h + d_id * stride_d;
      int offset_dst = offset_block_dst + h_id * o_stride_h + d_id * o_stride_d;
      scalar_t v_src = src[offset_src];
      scalar_t v_src_rotate = (d_id + d2 / 2 < d2)
                                  ? src[offset_src + (d2 / 2) * stride_d]
                                  : src[offset_src + (d2 / 2 - d2) * stride_d];
      dst[offset_dst] = v_src * v_cos + v_src_rotate * v_sin;
    }
  }

  // handle the tail
  if (d > d2) {
#pragma unroll
    for (int h_id = threadIdx.y; h_id < h; h_id += blockDim.y) {
      int offset_head = offset_block + h_id * stride_h;
      int offset_head_dst = offset_block_dst + h_id * o_stride_h;
#pragma unroll
      for (int d_id = d2 + threadIdx.x; d_id < d; d_id += blockDim.x) {
        dst[offset_head_dst + d_id * o_stride_d] =
            src[offset_head + d_id * stride_d];
      }
    }
  }
}

template <typename scalar_t>
void fused_rope_forward_launcher(const scalar_t *input, const scalar_t *cos,
                                 const scalar_t *sin, scalar_t *output,
                                 const int s, const int b, const int h,
                                 const int d, const int d2, const int stride_s,
                                 const int stride_b, const int stride_h,
                                 const int stride_d, const int o_stride_s,
                                 const int o_stride_b, const int o_stride_h,
                                 const int o_stride_d, hipStream_t stream) {
  int warps_per_block = h < 16 ? 4 : 8;
  dim3 blocks(s, b);
  dim3 threads(THREADS_PER_WARP, warps_per_block);

  fused_rope_forward_kernel<<<blocks, threads, 0, stream>>>(
      input, cos, sin, output, s, b, h, d, d2, stride_s, stride_b, stride_h,
      stride_d, o_stride_s, o_stride_b, o_stride_h, o_stride_d);
  NVTE_CHECK_CUDA(hipGetLastError());
}

template <typename scalar_t>
void fused_rope_backward_launcher(
    const scalar_t *incoming_grads, const scalar_t *cos, const scalar_t *sin,
    scalar_t *output_grads, const int s, const int b, const int h, const int d,
    const int d2, const int stride_s, const int stride_b, const int stride_h,
    const int stride_d, const int o_stride_s, const int o_stride_b,
    const int o_stride_h, const int o_stride_d, hipStream_t stream) {
  int warps_per_block = h < 16 ? 4 : 8;
  dim3 blocks(s, b);
  dim3 threads(THREADS_PER_WARP, warps_per_block);

  fused_rope_backward_kernel<<<blocks, threads, 0, stream>>>(
      incoming_grads, cos, sin, output_grads, s, b, h, d, d2, stride_s,
      stride_b, stride_h, stride_d, o_stride_s, o_stride_b, o_stride_h,
      o_stride_d);
  NVTE_CHECK_CUDA(hipGetLastError());
}

void fused_rope_forward(const Tensor &input, const Tensor &cos,
                        const Tensor &sin, Tensor *output, const int s,
                        const int b, const int h, const int d, const int d2,
                        const int stride_s, const int stride_b,
                        const int stride_h, const int stride_d,
                        const int o_stride_s, const int o_stride_b,
                        const int o_stride_h, const int o_stride_d,
                        hipStream_t stream) {
  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(
      input.data.dtype, scalar_t,
      fused_rope_forward_launcher(
          reinterpret_cast<const scalar_t *>(input.data.dptr),
          reinterpret_cast<const scalar_t *>(cos.data.dptr),
          reinterpret_cast<const scalar_t *>(sin.data.dptr),
          reinterpret_cast<scalar_t *>(output->data.dptr), s, b, h, d, d2,
          stride_s, stride_b, stride_h, stride_d, o_stride_s, o_stride_b,
          o_stride_h, o_stride_d, stream););
}

void fused_rope_backward(const Tensor &incoming_grads, const Tensor &cos,
                         const Tensor &sin, Tensor *output_grads, const int s,
                         const int b, const int h, const int d, const int d2,
                         const int stride_s, const int stride_b,
                         const int stride_h, const int stride_d,
                         const int o_stride_s, const int o_stride_b,
                         const int o_stride_h, const int o_stride_d,
                         hipStream_t stream) {
  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(
      incoming_grads.data.dtype, scalar_t,
      fused_rope_backward_launcher(
          reinterpret_cast<const scalar_t *>(incoming_grads.data.dptr),
          reinterpret_cast<const scalar_t *>(cos.data.dptr),
          reinterpret_cast<const scalar_t *>(sin.data.dptr),
          reinterpret_cast<scalar_t *>(output_grads->data.dptr), s, b, h, d, d2,
          stride_s, stride_b, stride_h, stride_d, o_stride_s, o_stride_b,
          o_stride_h, o_stride_d, stream););
}

}  // end namespace transformer_engine

void nvte_fused_rope_forward(const NVTETensor input, const NVTETensor cos,
                             const NVTETensor sin, NVTETensor output,
                             const int s, const int b, const int h, const int d,
                             const int d2, const int stride_s,
                             const int stride_b, const int stride_h,
                             const int stride_d, const int o_stride_s,
                             const int o_stride_b, const int o_stride_h,
                             const int o_stride_d, hipStream_t stream) {
  NVTE_API_CALL(nvte_fused_rope_forward);
  using namespace transformer_engine;
  fused_rope_forward(*reinterpret_cast<const Tensor *>(input),
                     *reinterpret_cast<const Tensor *>(cos),
                     *reinterpret_cast<const Tensor *>(sin),
                     reinterpret_cast<Tensor *>(output), s, b, h, d, d2,
                     stride_s, stride_b, stride_h, stride_d, o_stride_s,
                     o_stride_b, o_stride_h, o_stride_d, stream);
}

void nvte_fused_rope_backward(
    const NVTETensor incoming_grads, const NVTETensor cos, const NVTETensor sin,
    NVTETensor output_grads, const int s, const int b, const int h, const int d,
    const int d2, const int stride_s, const int stride_b, const int stride_h,
    const int stride_d, const int o_stride_s, const int o_stride_b,
    const int o_stride_h, const int o_stride_d, hipStream_t stream) {
  NVTE_API_CALL(nvte_fused_rope_backward);
  using namespace transformer_engine;
  fused_rope_backward(*reinterpret_cast<const Tensor *>(incoming_grads),
                      *reinterpret_cast<const Tensor *>(cos),
                      *reinterpret_cast<const Tensor *>(sin),
                      reinterpret_cast<Tensor *>(output_grads), s, b, h, d, d2,
                      stride_s, stride_b, stride_h, stride_d, o_stride_s,
                      o_stride_b, o_stride_h, o_stride_d, stream);
}
