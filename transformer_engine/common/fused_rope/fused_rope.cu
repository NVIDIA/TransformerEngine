#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_runtime.h>
#include <transformer_engine/fused_rope.h>

#include "../common.h"
#include "../util/logging.h"
#include "../utils.cuh"

namespace transformer_engine {

template <typename scalar_t>
__device__ void fused_rope_block_forward(
    const scalar_t *src, const float *freqs, scalar_t *dst,
    const int offset_block, const int offset_block_dst, const int h,
    const int d, const int d2, const int stride_h, const int stride_d,
    const int o_stride_h, const int o_stride_d) {
  int s_id = blockIdx.x;
#pragma unroll
  for (int d_id = threadIdx.x; d_id < d2; d_id += blockDim.x) {
    float v_cos, v_sin;
    sincosf(freqs[s_id * d2 + d_id], &v_sin, &v_cos);
#pragma unroll
    for (int h_id = threadIdx.y; h_id < h; h_id += blockDim.y) {
      int offset_src = offset_block + h_id * stride_h + d_id * stride_d;
      int offset_dst = offset_block_dst + h_id * o_stride_h + d_id * o_stride_d;
      float v_src = src[offset_src];
      float v_src_rotate = (d_id + d2 / 2 < d2)
                                  ? -static_cast<float>(src[offset_src + (d2 / 2) * stride_d])
                                  : static_cast<float>(src[offset_src + (d2 / 2 - d2) * stride_d]);
      dst[offset_dst] =
          v_src * v_cos + v_src_rotate * v_sin;
    }
  }

  // copy the rest
  if (d > d2) {
#pragma unroll
    for (int h_id = threadIdx.y; h_id < h; h_id += blockDim.y) {
      int offset_head = offset_block + h_id * stride_h;
      int offset_head_dst = offset_block_dst + h_id * o_stride_h;
#pragma unroll
      for (int d_id = d2 + threadIdx.x; d_id < d; d_id += blockDim.x) {
        dst[offset_head_dst + d_id * o_stride_d] =
            src[offset_head + d_id * stride_d];
      }
    }
  }
}

template <typename scalar_t>
__device__ void fused_rope_block_backward(
    const scalar_t *src, const float *freqs, scalar_t *dst,
    const int offset_block, const int offset_block_dst, const int h,
    const int d, const int d2, const int stride_h, const int stride_d,
    const int o_stride_h, const int o_stride_d) {
  int s_id = blockIdx.x;
#pragma unroll
  for (int d_id = threadIdx.x; d_id < d2; d_id += blockDim.x) {
    float v_cos = cosf(freqs[s_id * d2 + d_id]);
    float v_sin = (d_id + d2 / 2 < d2)
                         ? sinf(freqs[s_id * d2 + d_id + d2 / 2])
                         : -sinf(freqs[s_id * d2 + d_id + d2 / 2 - d2]);
#pragma unroll
    for (int h_id = threadIdx.y; h_id < h; h_id += blockDim.y) {
      int offset_src = offset_block + h_id * stride_h + d_id * stride_d;
      int offset_dst = offset_block_dst + h_id * o_stride_h + d_id * o_stride_d;
      float v_src = src[offset_src];
      float v_src_rotate = (d_id + d2 / 2 < d2)
                                  ? src[offset_src + (d2 / 2) * stride_d]
                                  : src[offset_src + (d2 / 2 - d2) * stride_d];
      dst[offset_dst] = v_src * v_cos + v_src_rotate * v_sin;
    }
  }

  // handle the tail
  if (d > d2) {
#pragma unroll
    for (int h_id = threadIdx.y; h_id < h; h_id += blockDim.y) {
      int offset_head = offset_block + h_id * stride_h;
      int offset_head_dst = offset_block_dst + h_id * o_stride_h;
#pragma unroll
      for (int d_id = d2 + threadIdx.x; d_id < d; d_id += blockDim.x) {
        dst[offset_head_dst + d_id * o_stride_d] =
            src[offset_head + d_id * stride_d];
      }
    }
  }
}

template <typename scalar_t>
__global__ void fused_rope_forward_kernel(
    const scalar_t *src, const float *freqs, scalar_t *dst, const int h,
    const int d, const int d2, const int stride_s, const int stride_b,
    const int stride_h, const int stride_d, const int o_stride_s,
    const int o_stride_b, const int o_stride_h, const int o_stride_d) {
  int s_id = blockIdx.x, b_id = blockIdx.y;
  int offset_block = s_id * stride_s + b_id * stride_b;
  int offset_block_dst = s_id * o_stride_s + b_id * o_stride_b;
  fused_rope_block_forward(src, freqs, dst, offset_block, offset_block_dst, h,
                           d, d2, stride_h, stride_d, o_stride_h, o_stride_d);
}

template <typename scalar_t>
__global__ void fused_rope_backward_kernel(
    const scalar_t *src, const float *freqs, scalar_t *dst, const int h,
    const int d, const int d2, const int stride_s, const int stride_b,
    const int stride_h, const int stride_d, const int o_stride_s,
    const int o_stride_b, const int o_stride_h, const int o_stride_d) {
  int s_id = blockIdx.x, b_id = blockIdx.y;
  int offset_block = s_id * stride_s + b_id * stride_b;
  int offset_block_dst = s_id * o_stride_s + b_id * o_stride_b;
  fused_rope_block_backward(src, freqs, dst, offset_block, offset_block_dst, h,
                            d, d2, stride_h, stride_d, o_stride_h, o_stride_d);
}

template <typename scalar_t>
__global__ void fused_rope_thd_forward_kernel(
    const scalar_t *src, const int *cu_seqlens, const float *freqs,
    scalar_t *dst, const int h, const int d, const int d2, const int stride_t,
    const int stride_h, const int stride_d, const int o_stride_t,
    const int o_stride_h, const int o_stride_d) {
  int s_id = blockIdx.x, b_id = blockIdx.y;
  int t_id = s_id + cu_seqlens[b_id];
  if (t_id >= cu_seqlens[b_id + 1]) return;
  int offset_block = t_id * stride_t;
  int offset_block_dst = t_id * o_stride_t;
  fused_rope_block_forward(src, freqs, dst, offset_block, offset_block_dst, h,
                           d, d2, stride_h, stride_d, o_stride_h, o_stride_d);
}

template <typename scalar_t>
__global__ void fused_rope_thd_backward_kernel(
    const scalar_t *src, const int *cu_seqlens, const float *freqs,
    scalar_t *dst, const int h, const int d, const int d2, const int stride_t,
    const int stride_h, const int stride_d, const int o_stride_t,
    const int o_stride_h, const int o_stride_d) {
  int s_id = blockIdx.x, b_id = blockIdx.y;
  int t_id = s_id + cu_seqlens[b_id];
  if (t_id >= cu_seqlens[b_id + 1]) return;
  int offset_block = t_id * stride_t;
  int offset_block_dst = t_id * o_stride_t;
  fused_rope_block_backward(src, freqs, dst, offset_block, offset_block_dst, h,
                            d, d2, stride_h, stride_d, o_stride_h, o_stride_d);
}

template <typename scalar_t>
void fused_rope_forward_launcher(const scalar_t *input, const float *freqs,
                                 scalar_t *output, const int s, const int b,
                                 const int h, const int d, const int d2,
                                 const int stride_s, const int stride_b,
                                 const int stride_h, const int stride_d,
                                 const int o_stride_s, const int o_stride_b,
                                 const int o_stride_h, const int o_stride_d,
                                 hipStream_t stream) {
  int warps_per_block = h < 16 ? 4 : 8;
  dim3 blocks(s, b);
  dim3 threads(THREADS_PER_WARP, warps_per_block);

  fused_rope_forward_kernel<<<blocks, threads, 0, stream>>>(
      input, freqs, output, h, d, d2, stride_s, stride_b, stride_h, stride_d,
      o_stride_s, o_stride_b, o_stride_h, o_stride_d);
  NVTE_CHECK_CUDA(hipGetLastError());
}

template <typename scalar_t>
void fused_rope_backward_launcher(const scalar_t *output_grads,
                                  const float *freqs, scalar_t *input_grads,
                                  const int s, const int b, const int h,
                                  const int d, const int d2, const int stride_s,
                                  const int stride_b, const int stride_h,
                                  const int stride_d, const int o_stride_s,
                                  const int o_stride_b, const int o_stride_h,
                                  const int o_stride_d, hipStream_t stream) {
  int warps_per_block = h < 16 ? 4 : 8;
  dim3 blocks(s, b);
  dim3 threads(THREADS_PER_WARP, warps_per_block);

  fused_rope_backward_kernel<<<blocks, threads, 0, stream>>>(
      output_grads, freqs, input_grads, h, d, d2, stride_s, stride_b, stride_h,
      stride_d, o_stride_s, o_stride_b, o_stride_h, o_stride_d);
  NVTE_CHECK_CUDA(hipGetLastError());
}

template <typename scalar_t>
void fused_rope_thd_forward_launcher(
    const scalar_t *input, const int *cu_seqlens, const float *freqs,
    scalar_t *output, const int max_s, const int b, const int h, const int d,
    const int d2, const int stride_t, const int stride_h, const int stride_d,
    const int o_stride_t, const int o_stride_h, const int o_stride_d,
    hipStream_t stream) {
  int warps_per_block = h < 16 ? 4 : 8;
  dim3 blocks(max_s, b);
  dim3 threads(THREADS_PER_WARP, warps_per_block);

  fused_rope_thd_forward_kernel<<<blocks, threads, 0, stream>>>(
      input, cu_seqlens, freqs, output, h, d, d2, stride_t, stride_h, stride_d,
      o_stride_t, o_stride_h, o_stride_d);
  NVTE_CHECK_CUDA(hipGetLastError());
}

template <typename scalar_t>
void fused_rope_thd_backward_launcher(
    const scalar_t *output_grads, const int *cu_seqlens, const float *freqs,
    scalar_t *input_grads, const int max_s, const int b, const int h,
    const int d, const int d2, const int stride_t, const int stride_h,
    const int stride_d, const int o_stride_t, const int o_stride_h,
    const int o_stride_d, hipStream_t stream) {
  int warps_per_block = h < 16 ? 4 : 8;
  dim3 blocks(max_s, b);
  dim3 threads(THREADS_PER_WARP, warps_per_block);

  fused_rope_thd_backward_kernel<<<blocks, threads, 0, stream>>>(
      output_grads, cu_seqlens, freqs, input_grads, h, d, d2, stride_t,
      stride_h, stride_d, o_stride_t, o_stride_h, o_stride_d);
  NVTE_CHECK_CUDA(hipGetLastError());
}

void fused_rope_forward(const Tensor &input, const Tensor &freqs,
                        Tensor *output, const int s, const int b, const int h,
                        const int d, const int d2, const int stride_s,
                        const int stride_b, const int stride_h,
                        const int stride_d, const int o_stride_s,
                        const int o_stride_b, const int o_stride_h,
                        const int o_stride_d, hipStream_t stream) {
  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(
      input.data.dtype, scalar_t,
      fused_rope_forward_launcher(
          reinterpret_cast<const scalar_t *>(input.data.dptr),
          reinterpret_cast<const float *>(freqs.data.dptr),
          reinterpret_cast<scalar_t *>(output->data.dptr), s, b, h, d, d2,
          stride_s, stride_b, stride_h, stride_d, o_stride_s, o_stride_b,
          o_stride_h, o_stride_d, stream););
}

void fused_rope_backward(const Tensor &output_grads, const Tensor &freqs,
                         Tensor *input_grads, const int s, const int b,
                         const int h, const int d, const int d2,
                         const int stride_s, const int stride_b,
                         const int stride_h, const int stride_d,
                         const int o_stride_s, const int o_stride_b,
                         const int o_stride_h, const int o_stride_d,
                         hipStream_t stream) {
  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(
      output_grads.data.dtype, scalar_t,
      fused_rope_backward_launcher(
          reinterpret_cast<const scalar_t *>(output_grads.data.dptr),
          reinterpret_cast<const float *>(freqs.data.dptr),
          reinterpret_cast<scalar_t *>(input_grads->data.dptr), s, b, h, d, d2,
          stride_s, stride_b, stride_h, stride_d, o_stride_s, o_stride_b,
          o_stride_h, o_stride_d, stream););
}

void fused_rope_thd_forward(const Tensor &input, const Tensor &cu_seqlens,
                            const Tensor &freqs, Tensor *output,
                            const int max_s, const int b, const int h,
                            const int d, const int d2, const int stride_t,
                            const int stride_h, const int stride_d,
                            const int o_stride_t, const int o_stride_h,
                            const int o_stride_d, hipStream_t stream) {
  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(
      input.data.dtype, scalar_t,
      fused_rope_thd_forward_launcher(
          reinterpret_cast<const scalar_t *>(input.data.dptr),
          reinterpret_cast<const int *>(cu_seqlens.data.dptr),
          reinterpret_cast<const float *>(freqs.data.dptr),
          reinterpret_cast<scalar_t *>(output->data.dptr), max_s, b, h, d, d2,
          stride_t, stride_h, stride_d, o_stride_t, o_stride_h, o_stride_d,
          stream););
}

void fused_rope_thd_backward(const Tensor &output_grads,
                             const Tensor &cu_seqlens, const Tensor &freqs,
                             Tensor *input_grads, const int max_s, const int b,
                             const int h, const int d, const int d2,
                             const int stride_t, const int stride_h,
                             const int stride_d, const int o_stride_t,
                             const int o_stride_h, const int o_stride_d,
                             hipStream_t stream) {
  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(
      output_grads.data.dtype, scalar_t,
      fused_rope_thd_backward_launcher(
          reinterpret_cast<const scalar_t *>(output_grads.data.dptr),
          reinterpret_cast<const int *>(cu_seqlens.data.dptr),
          reinterpret_cast<const float *>(freqs.data.dptr),
          reinterpret_cast<scalar_t *>(input_grads->data.dptr), max_s, b, h, d,
          d2, stride_t, stride_h, stride_d, o_stride_t, o_stride_h, o_stride_d,
          stream););
}

}  // end namespace transformer_engine

void nvte_fused_rope_forward(const NVTETensor input, const NVTETensor freqs,
                             NVTETensor output, const int s, const int b,
                             const int h, const int d, const int d2,
                             const int stride_s, const int stride_b,
                             const int stride_h, const int stride_d,
                             const int o_stride_s, const int o_stride_b,
                             const int o_stride_h, const int o_stride_d,
                             hipStream_t stream) {
  NVTE_API_CALL(nvte_fused_rope_forward);
  using namespace transformer_engine;
  fused_rope_forward(*reinterpret_cast<const Tensor *>(input),
                     *reinterpret_cast<const Tensor *>(freqs),
                     reinterpret_cast<Tensor *>(output), s, b, h, d, d2,
                     stride_s, stride_b, stride_h, stride_d, o_stride_s,
                     o_stride_b, o_stride_h, o_stride_d, stream);
}

void nvte_fused_rope_backward(const NVTETensor output_grads,
                              const NVTETensor freqs, NVTETensor input_grads,
                              const int s, const int b, const int h,
                              const int d, const int d2, const int stride_s,
                              const int stride_b, const int stride_h,
                              const int stride_d, const int o_stride_s,
                              const int o_stride_b, const int o_stride_h,
                              const int o_stride_d, hipStream_t stream) {
  NVTE_API_CALL(nvte_fused_rope_backward);
  using namespace transformer_engine;
  fused_rope_backward(*reinterpret_cast<const Tensor *>(output_grads),
                      *reinterpret_cast<const Tensor *>(freqs),
                      reinterpret_cast<Tensor *>(input_grads), s, b, h, d, d2,
                      stride_s, stride_b, stride_h, stride_d, o_stride_s,
                      o_stride_b, o_stride_h, o_stride_d, stream);
}

void nvte_fused_rope_thd_forward(const NVTETensor input,
                                 const NVTETensor cu_seqlens,
                                 const NVTETensor freqs, NVTETensor output,
                                 const int max_s, const int b, const int h,
                                 const int d, const int d2, const int stride_t,
                                 const int stride_h, const int stride_d,
                                 const int o_stride_t, const int o_stride_h,
                                 const int o_stride_d, hipStream_t stream) {
  NVTE_API_CALL(nvte_fused_rope_thd_forward);
  using namespace transformer_engine;
  fused_rope_thd_forward(*reinterpret_cast<const Tensor *>(input),
                         *reinterpret_cast<const Tensor *>(cu_seqlens),
                         *reinterpret_cast<const Tensor *>(freqs),
                         reinterpret_cast<Tensor *>(output), max_s, b, h, d, d2,
                         stride_t, stride_h, stride_d, o_stride_t, o_stride_h,
                         o_stride_d, stream);
}

void nvte_fused_rope_thd_backward(
    const NVTETensor output_grads, const NVTETensor cu_seqlens,
    const NVTETensor freqs, NVTETensor input_grads, const int max_s,
    const int b, const int h, const int d, const int d2, const int stride_t,
    const int stride_h, const int stride_d, const int o_stride_t,
    const int o_stride_h, const int o_stride_d, hipStream_t stream) {
  NVTE_API_CALL(nvte_fused_rope_thd_backward);
  using namespace transformer_engine;
  fused_rope_thd_backward(*reinterpret_cast<const Tensor *>(output_grads),
                          *reinterpret_cast<const Tensor *>(cu_seqlens),
                          *reinterpret_cast<const Tensor *>(freqs),
                          reinterpret_cast<Tensor *>(input_grads), max_s, b, h,
                          d, d2, stride_t, stride_h, stride_d, o_stride_t,
                          o_stride_h, o_stride_d, stream);
}
