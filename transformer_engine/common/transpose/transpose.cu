#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <transformer_engine/transpose.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cfloat>
#include "../common.h"
#include "../utils.cuh"
#include "../util/string.h"
#include "../util/rtc.h"

namespace transformer_engine {

namespace {

// String with RTC kernel implementation
#include "code_string_transpose_rtc_transpose_cu.h"

// Hard-coded kernel parameters
constexpr int warps_per_tile = 4;
constexpr int block_size = THREADS_PER_WARP * warps_per_tile;

}  // namespace

template <int load_size, int store_size, typename Type>
__global__ void
__launch_bounds__(block_size)
transpose_general_kernel(const Type * __restrict__ const input,
                         Type * __restrict__ const output,
                         const int row_length,
                         const int num_rows) {
  // Vectorized load/store sizes
  constexpr int nvec_in = load_size / sizeof(Type);
  constexpr int nvec_out = store_size / sizeof(Type);
  using IVec = Vec<Type, nvec_in>;
  using OVec = Vec<Type, nvec_out>;

  // Thread indices
  // Note: Block is interpreted as a warp_size x num_warps grid
  constexpr int bdimx = THREADS_PER_WARP;
  constexpr int bdimy = warps_per_tile;
  const int tid = threadIdx.x;
  const int tidx = tid % bdimx;
  const int tidy = tid / bdimx;
  const int bid = blockIdx.x;

  // Input tensors are divided into tiles
  // Note: Each tile is a warp_size x warp_size grid of nvec_out x nvec_in subtiles
  constexpr int tile_dim_m = THREADS_PER_WARP * nvec_out;
  constexpr int tile_dim_n = THREADS_PER_WARP * nvec_in;

  // Position of tile within tensor
  const int num_tiles_m = (num_rows + tile_dim_m - 1) / tile_dim_m;
  const int tile_id_m = bid % num_tiles_m;
  const int tile_id_n = bid / num_tiles_m;
  const int tile_row = tile_id_m * tile_dim_m;
  const int tile_col = tile_id_n * tile_dim_n;

  // Number of nvec_out x nvec_in subtiles for each thread to
  // load/store
  constexpr int num_iterations = THREADS_PER_WARP / warps_per_tile;

  // Load input and store to registers
  // Note: Each thread loads num_iterations subtiles and transposes in
  // registers.
  OVec local_output[nvec_in][num_iterations];
  #pragma unroll
  for (int iter = 0; iter < num_iterations; ++iter) {
    const int i1 = tidy + iter * bdimy;
    const int j1 = tidx;
    #pragma unroll
    for (int i2 = 0; i2 < nvec_out; ++i2) {
      const int row = tile_row + i1 * nvec_out + i2;
      const int col = tile_col + j1 * nvec_in;
      IVec local_input;
      local_input.clear();
      if (row < num_rows) {
        #pragma unroll
        for (int j2 = 0; j2 < nvec_in; ++j2) {
          if (col + j2 < row_length) {
            local_input.data.elt[j2] = input[row * row_length + col + j2];
          }
        }
      }
      #pragma unroll
      for (int j2 = 0; j2 < nvec_in; ++j2) {
        local_output[j2][iter].data.elt[i2] = local_input.data.elt[j2];
      }
    }
  }

  // Copy transposed output from registers to global memory
  __shared__ OVec shared_output[THREADS_PER_WARP][THREADS_PER_WARP+1];
  #pragma unroll
  for (int j2 = 0; j2 < nvec_in; ++j2) {
    #pragma unroll
    for (int iter = 0; iter < num_iterations; ++iter) {
      const int i1 = tidy + iter * bdimy;
      const int j1 = tidx;
      shared_output[j1][i1] = local_output[j2][iter];
    }
    __syncthreads();
    #pragma unroll
    for (int iter = 0; iter < num_iterations; ++iter) {
      const int i1 = tidx;
      const int j1 = tidy + iter * bdimy;
      const int row = tile_row + i1 * nvec_out;
      const int col = tile_col + j1 * nvec_in + j2;
      if (col < row_length) {
        #pragma unroll
        for (int i2 = 0; i2 < nvec_out; ++i2) {
          if (row + i2 < num_rows) {
            output[col * num_rows + row + i2] = shared_output[j1][i1].data.elt[i2];
          }
        }
      }
    }
    __syncthreads();
  }
}

void transpose(const Tensor &input,
               Tensor *output_,
               hipStream_t stream) {
  Tensor &output = *output_;
  NVTE_CHECK(input.data.shape.size() == 2, "Input must have 2 dimensions.");
  NVTE_CHECK(output.data.shape.size() == 2, "Output must have 2 dimensions.");
  const int row_length = input.data.shape[1];
  const int num_rows = input.data.shape[0];

  NVTE_CHECK(output.data.shape[0] == row_length, "Wrong dimension of output.");
  NVTE_CHECK(output.data.shape[1] == num_rows, "Wrong dimension of output.");

  NVTE_CHECK(input.data.dptr != nullptr, "Input is not allocated.");
  NVTE_CHECK(output.data.dptr != nullptr, "Output is not allocated.");
  NVTE_CHECK(input.data.dtype == output.data.dtype,
             "Input and output type must match.");

  TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(input.data.dtype, Type,
    constexpr const char *type_name = TypeInfo<Type>::name;
    constexpr int type_size = sizeof(Type);

    // Choose between runtime-compiled or statically-compiled kernel
    const bool aligned = (row_length % THREADS_PER_WARP == 0
                          && num_rows % THREADS_PER_WARP == 0);
    if (aligned && rtc::is_enabled()) {  // Runtime-compiled tuned kernel
      // Determine kernel config
      int load_size = 8;
      int store_size = 8;
      auto is_tile_aligned = [&](int load_size_, int store_size_) -> bool {
        return (row_length % (load_size / type_size * THREADS_PER_WARP) == 0
                && num_rows % (store_size / type_size * THREADS_PER_WARP) == 0);
      };
      auto num_blocks = [&](int load_size_, int store_size_) -> int {
        const int row_tile_size = load_size_ / type_size * THREADS_PER_WARP;
        const int col_tile_size = store_size_ / type_size * THREADS_PER_WARP;
        return (row_length / row_tile_size) * (num_rows / col_tile_size);
      };
      do {
        const int sm_count = cuda::sm_count();

        // Try maximizing SM occupancy without sacrificing cache
        // efficiency
        // Note: 32 threads/warp access 128B L1 cache line, so 4B
        // loads/stores achieve full cache efficiency
        if constexpr (type_size > 4) break;
        if (is_tile_aligned(load_size, store_size)
            && num_blocks(load_size, store_size) >= 4*sm_count) {
          break;
        }
        load_size = 4; store_size = 8;
        if (is_tile_aligned(load_size, store_size)
            && num_blocks(load_size, store_size) >= 4*sm_count) {
          break;
        }
        load_size = 4; store_size = 4;
        if (is_tile_aligned(load_size, store_size)
            && num_blocks(load_size, store_size) >= sm_count) {
          break;
        }

        // Simple performance model to balance SM occupancy and cache
        // efficiency
        auto cost = [&](int load_size_, int store_size_) -> double {
          int active_sms = std::min(sm_count, num_blocks(load_size_, store_size_));
          // Amortize memory accesses over 128B L1 cache line
          double load_cost = 1.0 / std::min(128u, load_size_ * THREADS_PER_WARP);
          double store_cost = 1.0 / std::min(128u, store_size_ * THREADS_PER_WARP);
          return (load_cost + store_cost) / active_sms;
        };
        if constexpr (type_size > 2) break;
        if (is_tile_aligned(load_size, store_size)
            && cost(2, 4) >= cost(load_size, store_size)) {
          break;
        }
        load_size = 2; store_size = 4;
        if (is_tile_aligned(load_size, store_size)
            && cost(2, 2) >= cost(load_size, store_size)) {
          break;
        }
        load_size = 2; store_size = 2;
        if constexpr (type_size > 1) break;
        if (is_tile_aligned(load_size, store_size)
            && cost(1, 2) >= cost(load_size, store_size)) {
          break;
        }
        load_size = 1; store_size = 2;
        if (is_tile_aligned(load_size, store_size)
            && cost(1, 1) >= cost(load_size, store_size)) {
          break;
        }
        load_size = 1; store_size = 1;
      } while (false);
      NVTE_CHECK(is_tile_aligned(load_size, store_size),
                 "memory accesses are not properly aligned");

      // Compile NVRTC kernel if needed and launch
      auto& rtc_manager = rtc::KernelManager::instance();
      const std::string kernel_label = concat_strings("transpose"
                                                      ",type=", type_name,
                                                      ",load_size=", load_size,
                                                      ",store_size", store_size);
      if (!rtc_manager.is_compiled(kernel_label)) {
        std::string code = code_string_transpose_rtc_transpose_cu;
        code = regex_replace(code, "__TYPE__", type_name);
        code = regex_replace(code, "__LOAD_SIZE__", load_size);
        code = regex_replace(code, "__STORE_SIZE__", store_size);
        code = regex_replace(code, "__WARPS_PER_TILE__", warps_per_tile);
        code = regex_replace(code, "__BLOCK_SIZE__", block_size);
        rtc_manager.compile(kernel_label,
                            "transpose_optimized_kernel",
                            code,
                            "transformer_engine/common/transpose/rtc/transpose.cu");
      }
      rtc_manager.launch(kernel_label,
                         num_blocks(load_size, store_size), block_size, 0, stream,
                         static_cast<const Type *>(input.data.dptr),
                         static_cast<Type*>(output.data.dptr),
                         row_length, num_rows);
    } else {  // Statically-compiled general kernel
      constexpr int load_size = 4;
      constexpr int store_size = 4;
      constexpr int row_tile_size = load_size / type_size * THREADS_PER_WARP;
      constexpr int col_tile_size = store_size / type_size * THREADS_PER_WARP;
      const int num_blocks = (DIVUP(row_length, row_tile_size)
                              * DIVUP(num_rows, col_tile_size));
      transpose_general_kernel<load_size, store_size, Type><<<num_blocks, block_size, 0, stream>>>(
        static_cast<const Type *>(input.data.dptr),
        static_cast<Type *>(output.data.dptr),
        row_length, num_rows);
    }
  );  // NOLINT(*)
}

}  // namespace transformer_engine

void nvte_transpose(const NVTETensor input,
                    NVTETensor output,
                    hipStream_t stream) {
  NVTE_API_CALL(nvte_transpose);
  using namespace transformer_engine;
  transpose(*reinterpret_cast<const Tensor*>(input),
            reinterpret_cast<Tensor*>(output),
            stream);
}
