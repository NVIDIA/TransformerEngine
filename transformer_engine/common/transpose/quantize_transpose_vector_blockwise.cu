#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cfloat>
#include <cuda/barrier>
#include <utility>

#include "common/common.h"
#include "common/recipe/recipe_common.cuh"
#include "common/transpose/cast_transpose.h"
#include "common/utils.cuh"

namespace transformer_engine {
namespace {

using transformer_engine::detail::FP8BlockwiseColumnwiseOption;
using transformer_engine::detail::FP8BlockwiseRowwiseOption;

// clang-format off
/*

Step 1: Load input to shared memory
* shard memory: 128x128 elements with type=InputType (below graph doesn't consider padding)
* 8 warps
* Loop 8 times
* What each thread does in each loop:
    * 8 elements are read from the input at a time
    * 2 elements are written to the shared memory at a time, for a total of 4 times
+-------------------------------+-------------------------------+-------------------------------+-------------------------------+
|  T0   |  T1   |  T2   |  T3   |  T4   |  T5   |  T6   |  T7   |  T8   |  T9   |  T10  |  T11  |  T12  |  T13  |  T14  |  T15  |
|  T16  |  T17  |  T18  |  T19  |  T20  |  T21  |  T22  |  T23  |  T24  |  T25  |  T26  |  T27  |  T28  |  T29  |  T30  |  T31  |
+-------------------------------+-------------------------------+-------------------------------+-------------------------------+
|                                                             Warp 1                                                            |
|                                                                                                                               |
+-------------------------------+-------------------------------+-------------------------------+-------------------------------+
|                                                              ...                                                              |
|                                                              ...                                                              |
|                                                              ...                                                              |
+-------------------------------+-------------------------------+-------------------------------+-------------------------------+
|                                                             Warp 7                                                            |
|                                                                                                                               |
+-------------------------------+-------------------------------+-------------------------------+-------------------------------+
|                                                              ...                                                              |
|                                                              ...                                                              |
|                                                              ...                                                              |
|                                                              ...                                                              |
|                                                          Loop 8 times                                                         |
|                                                              ...                                                              |
|                                                              ...                                                              |
|                                                              ...                                                              |
|                                                              ...                                                              |
+-------------------------------+-------------------------------+-------------------------------+-------------------------------+

Step 2: Cast and store to output_c
* shard memory: 128x128 elements with type=InputType (below graph doesn't consider padding)
* 8 warps
* Loop 4 times
* What each thread does in each loop:
    * 2 elements are read from the shared memory at a time, for a total of 8 times
    * Every 8 consecutive threads do reduction and calculate the amax of each row
    * 16 elements are quantized and write to output_c at a time
+-------------------------------+-------------------------------+-------------------------------+-------------------------------+
|      T0       |      T1       |      T2       |      T3       |      T4       |      T5       |      T6       |      T7       |
|      T8       |      T9       |      T10      |      T11      |      T12      |      T13      |      T14      |      T15      |
|      T16      |      T17      |      T18      |      T19      |      T20      |      T21      |      T22      |      T23      |
|      T24      |      T25      |      T26      |      T27      |      T28      |      T29      |      T30      |      T31      |
+-------------------------------+-------------------------------+-------------------------------+-------------------------------+
|                                                                                                                               |
|                                                             Warp 1                                                            |
|                                                                                                                               |
|                                                                                                                               |
+-------------------------------+-------------------------------+-------------------------------+-------------------------------+
|                                                              ...                                                              |
|                                                              ...                                                              |
|                                                              ...                                                              |
+-------------------------------+-------------------------------+-------------------------------+-------------------------------+
|                                                                                                                               |
|                                                             Warp 7                                                            |
|                                                                                                                               |
|                                                                                                                               |
+-------------------------------+-------------------------------+-------------------------------+-------------------------------+
|                                                              ...                                                              |
|                                                              ...                                                              |
|                                                              ...                                                              |
|                                                              ...                                                              |
|                                                          Loop 4 times                                                         |
|                                                              ...                                                              |
|                                                              ...                                                              |
|                                                              ...                                                              |
|                                                              ...                                                              |
+-------------------------------+-------------------------------+-------------------------------+-------------------------------+

Step 3 (if columnwise transpose is True, GEMM_READY): Transpose, cast and store to output_t
* shard memory: 128x128 elements with type=InputType (below graph doesn't consider padding)
* 8 warps
* Loop 2 times
* What each thread does in each loop:
    * 2 elements (in a row) are read from the shared memory at a time, for a total of 16 times
    * Every 8 consecutive threads do reduction and calculate the amax of each column
    * 16 elements are quantized and write to output_t at a time, for a total of 2 times
+------8 elements-------+------8 elements-------+-----40 elements-------+------8 elements-------+------8 elements-------+------8 elements-------+-----40 elements-------+------8 elements-------+
| T0  | T8  | T16 | T24 |                       |                       |                       | T0  | T8  | T16 | T24 |                       |                       |                       |
| T1  | T9  | T17 | T25 |                       |                       |                       | T1  | T9  | T17 | T25 |                       |                       |                       |
| T2  | T10 | T18 | T26 |                       |                       |                       | T2  | T10 | T18 | T26 |                       |                       |                       |
| T3  | T11 | T19 | T27 |        Warp 1         |         ...           |        Warp 7         | T3  | T11 | T19 | T27 |        Warp 1         |         ...           |        Warp 7         |
| T4  | T12 | T20 | T28 |                       |                       |                       | T4  | T12 | T20 | T28 |                       |                       |                       |
| T5  | T13 | T21 | T29 |                       |                       |                       | T5  | T13 | T21 | T29 |                       |                       |                       |
| T6  | T14 | T22 | T30 |                       |                       |                       | T6  | T14 | T22 | T30 |                       |                       |                       |
| T7  | T15 | T23 | T31 |                       |                       |                       | T7  | T15 | T23 | T31 |                       |                       |                       |
+-----------------------+-----------------------+-----------------------+-----------------------+-----------------------+-----------------------+-----------------------+-----------------------+

Step 3 (if columnwise transpose is False, COMPACT format): Skip Transpose, cast and store to output_t
* shard memory: 128x128 elements with type=InputType (below graph doesn't consider padding)
* 8 warps
* Loop 1 times
* What each thread does in each loop:
    * 16 elements (in a row) are read from the shared memory, for a total of 4 rows,
    * it needs 8 reads in smem to get 16 elements in a row, thread tile shape is 16x4
    * Every 32 consecutive threads in a warp do reduction and calculate the amax of each column,
    * so each thread will do warp shuffle 16 times to get the amax of each column
    * 16 elements are quantized and write to output_t at a time, for a total of 4 times
+------16 elements-------+------16 elements-------+-----80 elements-----+------16 elements------+
|           T0          |                       |                       |                       |
|           T1          |                       |                       |                       |
|           T2          |                       |                       |                       |
|           T3          |                       |                       |                       |
|           T4          |                       |                       |                       |
|           T5          |                       |                       |                       |
|           T6          |                       |                       |                       |
|           T7          |                       |                       |                       |
|           ...         |                       |                       |                       |
|           T31         |                       |                       |                       |
+-----------------------+-----------------------+-----------------------+-----------------------+

*/
// clang-format on

constexpr size_t kThreadsPerWarp = 32;

// Hyperparameters for performance tuning
constexpr int kTileDim = 128;  // Fixed to 128 beacause we are using 1x128 and 128x1 quantization
constexpr int kNVecIn = 8;     // The number of elements each LDG touches
constexpr int kNVecOut = 16;   // The number of elements each STG touches
constexpr int kNVecSMem = 2;   // The number of elements each LDS/STS touches
constexpr int kThreadsPerBlock = 256;  // Thread block size, 8 warps in total

// Auto-calculated constants, do not modify directly)
static_assert(kNVecIn % kNVecSMem == 0, "kNVecIn must be divisible by kNVecSMem");
static_assert(kNVecOut % kNVecSMem == 0, "kNVecOut must be divisible by kNVecSMem");
constexpr int kSMemRow = kTileDim;
constexpr int kSMemCol = (kTileDim / kNVecSMem) + 1;
constexpr int kSMemSize = kSMemRow * kSMemCol * kNVecSMem;
constexpr int kNumThreadsLoad = kTileDim / kNVecIn;
constexpr int kNumThreadsStore = kTileDim / kNVecOut;
static_assert(kNumThreadsLoad <= kThreadsPerWarp, "kNumThreadsLoad must be <= kThreadsPerWarp");
static_assert(kNumThreadsStore <= kThreadsPerWarp, "kNumThreadsStore must be <= kThreadsPerWarp");
constexpr int kNumWarps = kThreadsPerBlock / kThreadsPerWarp;

template <bool kAligned, typename CType, typename IType, typename OType>
__global__ void __launch_bounds__(kThreadsPerBlock) block_scaled_1d_cast_transpose_kernel(
    const IType* const input, OType* const output_c, OType* const output_t,
    CType* const tile_scales_inv_c, CType* const tile_scales_inv_t, const size_t row_length,
    const size_t num_rows, const size_t scale_stride_x, const size_t scale_stride_y,
    const size_t scale_t_stride_x, const size_t scale_t_stride_y, const float epsilon,
    FP8BlockwiseRowwiseOption rowwise_option, FP8BlockwiseColumnwiseOption columnwise_option,
    const bool pow_2_scaling) {
  bool return_rowwise = rowwise_option != FP8BlockwiseRowwiseOption::NONE;
  bool return_columnwise_gemm_ready =
      columnwise_option == FP8BlockwiseColumnwiseOption::COLUMNWISE_GEMM_READY;
  bool return_columnwise_compact =
      columnwise_option == FP8BlockwiseColumnwiseOption::COLUMNWISE_COMPACT;

  using SMemVec = Vec<IType, kNVecSMem>;
  using OVec = Vec<OType, kNVecOut>;
  union IVec {
    Vec<IType, kNVecIn> input_type;
    Vec<SMemVec, kNVecIn / kNVecSMem> smem_type;
  };

  extern __shared__ char smem_base[];
  SMemVec* smem = reinterpret_cast<SMemVec*>(&smem_base[0]);

  // Step 1: Load input to shared memory
  {
    constexpr int r_stride = kThreadsPerBlock / kNumThreadsLoad;  // stride in rows of shared memory
    constexpr int num_iterations = kTileDim / r_stride;
    const int c_s =
        (threadIdx.x % kNumThreadsLoad) * (kNVecIn / kNVecSMem);  // Column in shared memory
    int r_s = threadIdx.x / kNumThreadsLoad;                      // Row in shared memory
    const size_t c_g =
        static_cast<size_t>(blockIdx.x) * kTileDim + c_s * kNVecSMem;    // Column in global memory
    size_t r_g = static_cast<size_t>(blockIdx.y) * kTileDim + r_s;       // Row in global memory
    const size_t stride_g = static_cast<size_t>(r_stride) * row_length;  // Stride in global memory
    const size_t num_ele = c_g < row_length ? min(static_cast<size_t>(kNVecIn), row_length - c_g)
                                            : 0;            // For not aligned case
    const IType* input_g = &input[r_g * row_length + c_g];  // Input address in global memory
#pragma unroll
    for (int iter = 0; iter < num_iterations; ++iter) {
      IVec input_vec;
      // Step 1.1: Load from global memory (input) to registers
      if constexpr (kAligned) {
        input_vec.input_type.load_from(input_g);
      } else {
        if (r_g < num_rows) {
          input_vec.input_type.load_from_elts(input_g, 0, num_ele);
        } else {
          input_vec.input_type.clear();
        }
      }
      // Step 1.2: Write to shared memory
#pragma unroll
      for (int i = 0; i < kNVecIn / kNVecSMem; ++i) {
        int c = c_s + i;
        int r = r_s;
        smem[r * kSMemCol + c] = input_vec.smem_type.data.elt[i];
      }
      // Step 1.3: Update input address, row index of shared memory, (and row index of global memory for not aligned case)
      input_g += stride_g;
      r_s += r_stride;
      if constexpr (!kAligned) {
        r_g += r_stride;
      }
    }
  }

  __syncthreads();

  // Step 2: Cast and store to output_c
  if (return_rowwise) {
    constexpr int r_stride =
        kThreadsPerBlock / kNumThreadsStore;  // stride in rows of shared memory
    constexpr int num_iterations = kTileDim / r_stride;
    const int c_s =
        (threadIdx.x % kNumThreadsStore) * (kNVecOut / kNVecSMem);  // Column in shared memory
    int r_s = threadIdx.x / kNumThreadsStore;                       // Row in shared memory
    const size_t c_g =
        static_cast<size_t>(blockIdx.x) * kTileDim + c_s * kNVecSMem;    // Column in global memory
    size_t r_g = static_cast<size_t>(blockIdx.y) * kTileDim + r_s;       // Row in global memory
    const size_t stride_g = static_cast<size_t>(r_stride) * row_length;  // Stride in global memory
    const size_t num_ele = c_g < row_length ? min(static_cast<size_t>(kNVecOut), row_length - c_g)
                                            : 0;          // For not aligned case
    OType* output_g = &output_c[r_g * row_length + c_g];  // Output address in global memory
    // Each kNumThreadsStore threads form a warp process one row, we need to find the lane id of
    // the first thread to do the reduction.
    const unsigned src_lane = (threadIdx.x % kThreadsPerWarp) / kNumThreadsStore * kNumThreadsStore;
    // This mask represents which threads should do the reduction together.
    const unsigned mask = ((1 << kNumThreadsStore) - 1) << src_lane;
    const bool is_src_lane = (threadIdx.x % kNumThreadsStore) == 0;
#pragma unroll
    for (int iter = 0; iter < num_iterations; ++iter) {
      SMemVec smem_vec[kNVecOut / kNVecSMem];
      // Step 2.1: Load from shared memory to registers
#pragma unroll
      for (int i = 0; i < kNVecOut / kNVecSMem; ++i) {
        int c = c_s + i;
        int r = r_s;
        smem_vec[i] = smem[r * kSMemCol + c];
      }
      // Step 2.2: Compute local amax
      CType amax = 0;
#pragma unroll
      for (int i = 0; i < kNVecOut / kNVecSMem; ++i) {
#pragma unroll
        for (int j = 0; j < kNVecSMem; ++j) {
          __builtin_assume(amax >= 0);
          amax = fmaxf(amax, fabsf(smem_vec[i].data.elt[j]));
        }
      }
      // Step 2.3: Reduce amax
#pragma unroll
      for (int delta = kNumThreadsStore / 2; delta > 0; delta /= 2) {
        const float other_amax = __shfl_down_sync(mask, amax, delta);
        __builtin_assume(amax >= 0);
        __builtin_assume(other_amax >= 0);
        amax = fmaxf(amax, other_amax);
      }
      amax = __shfl_sync(mask, amax, src_lane);
      CType scale;
      // Step 2.4: Compute scale
      scale = compute_scale_from_types<IType, OType>(amax, epsilon, pow_2_scaling);
      // Step 2.5: Write scale_inv
      bool write_scale_inv = is_src_lane;
      if constexpr (!kAligned) {
        write_scale_inv &= (r_g < num_rows);
      }
      if (write_scale_inv) {
        CType scale_inv = 1.0 / scale;
        size_t row_idx = static_cast<size_t>(blockIdx.y) * kTileDim + r_s;
        size_t col_idx = static_cast<size_t>(blockIdx.x);
        tile_scales_inv_c[row_idx * scale_stride_y + col_idx * scale_stride_x] = scale_inv;
      }
      // Step 2.6: Quantize
      OVec output_vec;
#pragma unroll
      for (int i = 0; i < kNVecOut / kNVecSMem; ++i) {
#pragma unroll
        for (int j = 0; j < kNVecSMem; ++j) {
          output_vec.data.elt[i * kNVecSMem + j] =
              static_cast<OType>(static_cast<CType>(smem_vec[i].data.elt[j]) * scale);
        }
      }
      // Step 2.7: Store output_c
      if constexpr (kAligned) {
        output_vec.store_to(output_g);
      } else {
        if (r_g < num_rows) {
          output_vec.store_to_elts(output_g, 0, num_ele);
        }
      }
      // Step 2.8: Update output address, row index of shared memory (and row index of global memory for not aligned case)
      output_g += stride_g;
      r_s += r_stride;
      if constexpr (!kAligned) {
        r_g += r_stride;
      }
    }
  }

  // Step 3 (return_columnwise_gemm_ready): Transpose, cast and store to output_t
  if (return_columnwise_gemm_ready) {
    constexpr int c_stride =
        kThreadsPerBlock / kNumThreadsStore;  // Stride in columns of shared memory
    constexpr int num_iterations = kTileDim / (c_stride * kNVecSMem);
    const int r_s = (threadIdx.x % kNumThreadsStore) * kNVecOut;  // Row in shared memory
    int c_s = threadIdx.x / kNumThreadsStore;                     // Column in shared memory
    size_t r_g =
        static_cast<size_t>(blockIdx.x) * kTileDim + c_s * kNVecSMem;     // Row in global memory
    const size_t c_g = static_cast<size_t>(blockIdx.y) * kTileDim + r_s;  // Column in global memory
    const size_t stride_g =
        static_cast<size_t>(c_stride) * kNVecSMem * num_rows;  // Stride in global memory
    const size_t num_ele = c_g < num_rows ? min(static_cast<size_t>(kNVecOut), num_rows - c_g)
                                          : 0;          // For not aligned case
    OType* output_g = &output_t[r_g * num_rows + c_g];  // Output address in global memory
    // Each kNumThreadsStore threads form a warp process one row, we need to find the lane id of
    // the first thread to do the reduction.
    const unsigned src_lane = (threadIdx.x % kThreadsPerWarp) / kNumThreadsStore * kNumThreadsStore;
    // This mask represents which threads should do the reduction together.
    const unsigned mask = ((1 << kNumThreadsStore) - 1) << src_lane;
    const bool is_src_lane = (threadIdx.x % kNumThreadsStore) == 0;
#pragma unroll
    for (int iter = 0; iter < num_iterations; ++iter) {
      SMemVec smem_vec[kNVecOut];
      // Step 3.1: Load from shared memory to registers
#pragma unroll
      for (int i = 0; i < kNVecOut; ++i) {
        int r = r_s + i;
        int c = c_s;
        smem_vec[i] = smem[r * kSMemCol + c];
      }
#pragma unroll
      for (int smem_idx = 0; smem_idx < kNVecSMem; ++smem_idx) {
        // Step 3.2: Compute local amax
        CType amax = 0;
#pragma unroll
        for (int i = 0; i < kNVecOut; ++i) {
          amax = fmaxf(amax, fabsf(smem_vec[i].data.elt[smem_idx]));
        }
        // Step 3.3: Reduce amax
#pragma unroll
        for (int delta = kNumThreadsStore / 2; delta > 0; delta /= 2) {
          const float other_amax = __shfl_down_sync(mask, amax, delta);
          __builtin_assume(amax >= 0);
          __builtin_assume(other_amax >= 0);
          amax = fmaxf(amax, other_amax);
        }
        amax = __shfl_sync(mask, amax, src_lane);
        // Step 3.4: Compute scale
        CType scale;
        scale = compute_scale_from_types<IType, OType>(amax, epsilon, pow_2_scaling);
        // Step 3.5: Write scale_inv_t
        bool write_scale_inv = is_src_lane;
        if constexpr (!kAligned) {
          write_scale_inv &= (r_g + smem_idx < row_length);
        }
        if (write_scale_inv) {
          CType scale_inv = 1.0 / scale;
          size_t row_idx = static_cast<size_t>(blockIdx.x) * kTileDim + c_s * kNVecSMem + smem_idx;
          size_t col_idx = static_cast<size_t>(blockIdx.y);
          tile_scales_inv_t[row_idx * scale_t_stride_y + col_idx * scale_t_stride_x] = scale_inv;
        }
        // Step 3.6: Quantize
        OVec output_vec;
#pragma unroll
        for (int i = 0; i < kNVecOut; ++i) {
          output_vec.data.elt[i] =
              static_cast<OType>(static_cast<CType>(smem_vec[i].data.elt[smem_idx]) * scale);
        }
        // Step 3.7: Store output_t
        if constexpr (kAligned) {
          output_vec.store_to(output_g + smem_idx * num_rows);
        } else {
          if (r_g + smem_idx < row_length) {
            output_vec.store_to_elts(output_g + smem_idx * num_rows, 0, num_ele);
          }
        }
      }
      // Step 3.8: Update output address, column index of shared memory (and row index of global memory for not aligned case)
      output_g += stride_g;
      c_s += c_stride;
      if constexpr (!kAligned) {
        r_g += c_stride * kNVecSMem;
      }
    }
  }

  // Step 4 (return_columnwise_compact): cast in 128x1 style and store to output, skip transpose
  if (return_columnwise_compact) {
    // thread tile should be 4x16, 16 means 8 smem reads
    constexpr int kThreadTileRow = kTileDim / kThreadsPerWarp;
    constexpr int kThreadTileCol = kNVecOut;
    using RegVec = Vec<IType, kThreadTileCol>;
    using RegScaleVec = Vec<CType, kThreadTileCol>;
    constexpr int num_smem_reads = kNVecOut / kNVecSMem;
    // c_stride will not be used here because we only have one iteration
    // constexpr int c_stride = kThreadTileCol * kNumWarps / kNVecSMem;
    constexpr int num_iterations =
        kTileDim / (kNumWarps * kThreadTileCol);  // should be only one iteration
    static_assert(num_iterations == 1,
                  "num_iterations should be 1 for columnwise non-transpose case");
    const int thr_idx_in_warp = threadIdx.x % kThreadsPerWarp;
    const int warp_idx = threadIdx.x / kThreadsPerWarp;
    const int r_s = thr_idx_in_warp * kThreadTileRow;               // Row in shared memory
    int c_s = warp_idx * num_smem_reads;                            // Column in shared memory
    size_t r_g = static_cast<size_t>(blockIdx.y) * kTileDim + r_s;  // Row in global memory
    const size_t c_g =
        static_cast<size_t>(blockIdx.x) * kTileDim + c_s * kNVecSMem;  // Column in global memory
    const size_t num_ele = c_g < row_length
                               ? min(static_cast<size_t>(kThreadTileCol), row_length - c_g)
                               : 0;  // For not aligned case
#pragma unroll
    for (int iter = 0; iter < num_iterations; ++iter) {
      RegVec reg_vec[kThreadTileRow];
      RegScaleVec thr_scale;

      // Step 3.1: Load from shared memory to registers
#pragma unroll
      for (int i = 0; i < kThreadTileRow; ++i) {
        int r = r_s + i;
#pragma unroll
        for (int j = 0; j < num_smem_reads; ++j) {
          int c = c_s + j;
          SMemVec smem_vec = smem[r * kSMemCol + c];
          // copy smem_vec to reg vec with its elements
#pragma unroll
          for (int k = 0; k < kNVecSMem; ++k) {
            reg_vec[i].data.elt[j * kNVecSMem + k] = smem_vec.data.elt[k];
          }
        }
      }
#pragma unroll
      for (int reg_idx = 0; reg_idx < kThreadTileCol; ++reg_idx) {
        // Step 3.2: Compute local amax
        CType amax = 0;
#pragma unroll
        for (int i = 0; i < kThreadTileRow; ++i) {
          amax = fmaxf(amax, fabsf(reg_vec[i].data.elt[reg_idx]));
        }
        // Step 3.3: Reduce amax
        const bool is_src_lane = thr_idx_in_warp == 0;
        amax = warp_reduce_max<kThreadsPerWarp>(amax);
        constexpr int lane_zero = 0;
        amax = __shfl_sync(0xFFFFFFFF, amax, lane_zero);
        // Step 3.4: Compute scale
        CType scale;
        scale = compute_scale_from_types<IType, OType>(amax, epsilon, pow_2_scaling);
        thr_scale.data.elt[reg_idx] = scale;
        // Step 3.5: Write scale_inv_t
        bool write_scale_inv = is_src_lane;
        if constexpr (!kAligned) {
          write_scale_inv &= (c_g + reg_idx < row_length);
        }
        if (write_scale_inv) {
          CType scale_inv = 1.0 / scale;
          size_t row_idx = static_cast<size_t>(blockIdx.y);
          size_t col_idx = static_cast<size_t>(blockIdx.x) * kTileDim + c_s * kNVecSMem + reg_idx;
          tile_scales_inv_t[row_idx * scale_t_stride_y + col_idx * scale_t_stride_x] = scale_inv;
        }
      }
      // Step 3.6: Quantize
      for (int row_idx = 0; row_idx < kThreadTileRow; ++row_idx) {
        OType* output_g =
            &output_t[(r_g + row_idx) * row_length + c_g];  // Output address in global memory
        OVec output_vec;
#pragma unroll
        for (int i = 0; i < kThreadTileCol; ++i) {
          output_vec.data.elt[i] = static_cast<OType>(
              static_cast<CType>(reg_vec[row_idx].data.elt[i]) * thr_scale.data.elt[i]);
        }
        // Step 3.7: Store output_t
        if constexpr (kAligned) {
          output_vec.store_to(output_g);
        } else {
          if (r_g + row_idx < num_rows) {
            output_vec.store_to_elts(output_g, 0, num_ele);
          }
        }
      }
      // Step 3.8: Update output address, column index of shared memory
      // this section shouldn't matter since we only have one iteration
    }
  }
}

}  // namespace
}  // namespace transformer_engine

namespace transformer_engine::detail {

void quantize_transpose_vector_blockwise(const SimpleTensor& input, SimpleTensor& scale_inv,
                                         SimpleTensor& scale_inv_t, SimpleTensor& output,
                                         SimpleTensor& output_t, const float epsilon,
                                         FP8BlockwiseRowwiseOption rowwise_option,
                                         FP8BlockwiseColumnwiseOption columnwise_option,
                                         const bool pow2_scale, hipStream_t stream) {
  NVTE_API_CALL(quantize_transpose_vector_blockwise);

  const size_t row_length = input.shape.size() > 0 ? input.shape.at(input.shape.size() - 1) : 1u;
  size_t num_elements = row_length;
  size_t num_rows = 1;
  for (size_t i = 0; (i < input.shape.size() - 1) && (input.shape.size() > 0); ++i) {
    num_rows *= input.shape.at(i);
    num_elements *= input.shape.at(i);
  }

  // Early return if the input tensor is empty
  if (num_elements == 0) {
    return;
  }

  // Options for scale layout of cuBLAS GEMM kernel.
  size_t scale_stride_x = 0;
  size_t scale_stride_y = 0;
  size_t scale_t_stride_x = 0;
  size_t scale_t_stride_y = 0;

  if (rowwise_option != FP8BlockwiseRowwiseOption::NONE) {
    NVTE_CHECK(rowwise_option == FP8BlockwiseRowwiseOption::ROWWISE_GEMM_READY ||
                   rowwise_option == FP8BlockwiseRowwiseOption::ROWWISE_COMPACT,
               "Unexpected rowwise enum value");
    NVTE_CHECK(input.shape == output.shape, "Input and output must have the same shape.");
    NVTE_CHECK(scale_inv.shape.size() == 2, "Scale dimension must be 2.");
    size_t scale_k = scale_inv.shape[1];
    bool rowwise_compact = rowwise_option == FP8BlockwiseRowwiseOption::ROWWISE_COMPACT;
    scale_stride_x = rowwise_compact ? 1 : scale_k;
    scale_stride_y = rowwise_compact ? scale_k : 1;
  }

  if (columnwise_option != FP8BlockwiseColumnwiseOption::NONE) {
    NVTE_CHECK(output_t.shape.size() == input.shape.size(),
               "output_t must have same number of dimensions as input.");

    if (output_t.shape.size() > 0) {
      if (columnwise_option == FP8BlockwiseColumnwiseOption::COLUMNWISE_GEMM_READY) {
        NVTE_CHECK(output_t.shape[0] == row_length, "Wrong dimension 0 of output_t.");
        for (size_t i = 1; i < output_t.shape.size(); ++i) {
          NVTE_CHECK(output_t.shape.at(i) == input.shape.at(i - 1), "Wrong dimension in output_t");
        }
      } else {
        NVTE_CHECK(columnwise_option == FP8BlockwiseColumnwiseOption::COLUMNWISE_COMPACT,
                   "Unexpected columnwise option enum value");
        NVTE_CHECK(output_t.shape[0] == input.shape[0], "Wrong dimension 0 of output_t.");
        NVTE_CHECK(
            input.shape == output_t.shape,
            "Input and output_t must have the same shape for columnwise non-transpose case.");
      }
    }

    NVTE_CHECK(output.dtype == output_t.dtype, "output and output_t need to have the same dtype.");
    NVTE_CHECK(scale_inv_t.shape.size() == 2, "Scale_t dimension must be 2.");
    bool columnwise_compact = columnwise_option == FP8BlockwiseColumnwiseOption::COLUMNWISE_COMPACT;
    size_t scale_t_k = scale_inv_t.shape[1];
    scale_t_stride_x = columnwise_compact ? 1 : scale_t_k;
    scale_t_stride_y = columnwise_compact ? scale_t_k : 1;
  }

  const size_t num_blocks_x = DIVUP(row_length, (size_t)kTileDim);
  const size_t num_blocks_y = DIVUP(num_rows, (size_t)kTileDim);

  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(
      input.dtype, InputType,

      TRANSFORMER_ENGINE_TYPE_SWITCH_FP8ONLY(
          output.dtype, OutputType,

          dim3 grid(num_blocks_x, num_blocks_y, 1);

          const bool full_tile = row_length % kTileDim == 0 && num_rows % kTileDim == 0;

          TRANSFORMER_ENGINE_SWITCH_CONDITION(
              full_tile, kAligned,

              size_t smem_bytes = kSMemSize * sizeof(InputType);
              // shared memory must be requested up
              if (smem_bytes >= 48 * 1024) {
                hipError_t err = hipFuncSetAttribute(reinterpret_cast<const void*>(
                    &block_scaled_1d_cast_transpose_kernel<kAligned), float, InputType, OutputType>,
                    hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
                NVTE_CHECK(err == hipSuccess, "Failed to set dynamic shared memory size.");
              } block_scaled_1d_cast_transpose_kernel<kAligned, float, InputType, OutputType>
              <<<grid, kThreadsPerBlock, smem_bytes, stream>>>(
                  reinterpret_cast<const InputType*>(input.dptr),
                  reinterpret_cast<OutputType*>(output.dptr),
                  reinterpret_cast<OutputType*>(output_t.dptr),
                  reinterpret_cast<float*>(scale_inv.dptr),
                  reinterpret_cast<float*>(scale_inv_t.dptr), row_length, num_rows, scale_stride_x,
                  scale_stride_y, scale_t_stride_x, scale_t_stride_y, epsilon, rowwise_option,
                  columnwise_option, pow2_scale);)  // kAligned
          )                                         // OutputType
      )                                             // InputType
  NVTE_CHECK_CUDA(hipGetLastError());
}

}  // namespace transformer_engine::detail
