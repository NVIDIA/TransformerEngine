#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <transformer_engine/transpose.h>
#include <hip/hip_runtime.h>
#include <cfloat>
#include <iostream>
#include <type_traits>
#include "../utils.cuh"
#include "../common.h"

namespace transformer_engine {

template <int nvec_in, int nvec_out,
          typename IVec, typename OVec, typename CVec, typename CType>
inline __device__ void transpose_regs_partial_dbias(const IVec (&in)[nvec_out],
                                                    OVec (&out_trans)[nvec_in],
                                                    CVec &out_dbias,  // NOLINT(*)
                                                    const CType scale_inv,
                                                    const int dbias_shfl_src_lane) {
  using T = typename OVec::type;
  using OVecC = Vec<T, nvec_in>;

  CVec step_dbias; step_dbias.clear();

#pragma unroll
  for (unsigned int i = 0; i < nvec_out; ++i) {
#pragma unroll
    for (unsigned int j = 0; j < nvec_in; ++j) {
      const CType tmp = static_cast<CType>(in[i].data.elt[j]) * scale_inv;
      const T elt_o = in[i].data.elt[j];

      /* dbias: thread tile local accumulation */
      step_dbias.data.elt[j] += tmp;

      out_trans[j].data.elt[i] = elt_o;  // thread tile transpose
    }
  }

#pragma unroll
  for (unsigned int j = 0; j < nvec_in; ++j) {
    CType elt = step_dbias.data.elt[j];
    elt = __shfl_sync(0xffffffff, elt, dbias_shfl_src_lane);  // shuffle data in warp
    out_dbias.data.elt[j] += elt;
  }
}

// STUFF TO TUNE
constexpr unsigned int n_warps_per_tile = 4;
constexpr int desired_load_size = 8;
constexpr int desired_store_size = 8;

constexpr unsigned int max_threads_per_block = 256;
static_assert(n_warps_per_tile * THREADS_PER_WARP <= max_threads_per_block);
constexpr unsigned int cast_transpose_num_threads = n_warps_per_tile * THREADS_PER_WARP;

namespace {

template <typename IType, typename OType, typename CType>
struct TDBiasParam {
    using InputType = IType;
    using OutputType = OType;
    using ComputeType = CType;
    const IType *input;
    OType *output_t;
    const CType *scale_inv;
    CType *workspace;
};

}  // namespace

template <int nvec_in, int nvec_out, typename Param>
__global__ void
__launch_bounds__(cast_transpose_num_threads)
transpose_dbias_kernel(const Param param,
                            const size_t row_length,
                            const size_t num_rows,
                            const size_t num_tiles) {
  using IType = typename Param::InputType;
  using OType = typename Param::OutputType;
  using CType = typename Param::ComputeType;
  using IVec = Vec<IType, nvec_in>;
  using OVec = Vec<OType, nvec_out>;
  using CVec = Vec<CType, nvec_in>;

  extern __shared__ char scratch[];

  const int warp_id = threadIdx.x / THREADS_PER_WARP;
  const unsigned int my_id_in_warp = threadIdx.x % THREADS_PER_WARP;
  const size_t num_tiles_x = row_length / (nvec_in * THREADS_PER_WARP);
  // const size_t num_tiles_y = num_rows / (nvec * THREADS_PER_WARP);
  const size_t tile_id = blockIdx.x * blockDim.x / (THREADS_PER_WARP * n_warps_per_tile) +
                         warp_id / n_warps_per_tile;
  if (tile_id >= num_tiles) return;
  const size_t tile_id_x = tile_id % num_tiles_x;
  const size_t tile_id_y = tile_id / num_tiles_x;

  const IType * const my_input_tile = param.input + (tile_id_x * nvec_in +
                                                     tile_id_y * row_length * nvec_out) *
                                                    THREADS_PER_WARP;
  OType * const my_output_t_tile = param.output_t + (tile_id_y * nvec_out +
                                                     tile_id_x * num_rows * nvec_in) *
                                                    THREADS_PER_WARP;
  CType * const my_partial_dbias_tile = param.workspace +
                                        (tile_id_x * (nvec_in * THREADS_PER_WARP) +
                                         tile_id_y * row_length);

  OVec * const my_scratch = reinterpret_cast<OVec *>(scratch) +
                            (my_id_in_warp + warp_id / n_warps_per_tile * THREADS_PER_WARP) *
                            (THREADS_PER_WARP + 1);

  CVec * const my_dbias_scratch = reinterpret_cast<CVec *>(scratch);

  IVec in[2][nvec_out];
  const unsigned int warp_id_in_tile = warp_id % n_warps_per_tile;
  constexpr unsigned int n_iterations = THREADS_PER_WARP / n_warps_per_tile;
  OVec out_space[n_iterations][nvec_in];
  CVec partial_dbias;

  const size_t stride = row_length / nvec_in;
  const size_t output_stride = num_rows / nvec_out;
  size_t current_stride = warp_id_in_tile * n_iterations * nvec_out * stride;
  unsigned int my_place = (my_id_in_warp + THREADS_PER_WARP -
                           warp_id_in_tile * n_iterations) %
                          THREADS_PER_WARP;
  const CType scale_inv = param.scale_inv != nullptr ? *param.scale_inv : 1;

  partial_dbias.clear();

#pragma unroll
  for (unsigned int i = 0; i < nvec_out; ++i) {
    in[0][i].load_from(my_input_tile, current_stride + my_place + stride * i);
  }
#pragma unroll
  for (unsigned int i = 0; i < n_iterations; ++i) {
    const unsigned int my_place_in = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
    const unsigned int current_in = (i + 1) % 2;
    if (i < n_iterations - 1) {
#pragma unroll
      for (unsigned int j = 0; j < nvec_out; ++j) {
        in[current_in][j].load_from(my_input_tile,
                                    current_stride + my_place_in + stride * (nvec_out + j));
      }
    }
    OVec out_trans[nvec_in];  // NOLINT(*)
    transpose_regs_partial_dbias(
                    in[current_in ^ 1],
                    out_trans,
                    partial_dbias,
                    scale_inv,
                    (my_id_in_warp + i + warp_id_in_tile * n_iterations) % THREADS_PER_WARP);

#pragma unroll
    for (unsigned int j = 0; j < nvec_in; ++j) {
      out_space[i][j].data.vec = out_trans[j].data.vec;
    }
    my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
    current_stride += nvec_out * stride;
  }

  for (unsigned int i = 0; i < nvec_in; ++i) {
#pragma unroll
    for (unsigned int j = 0; j < n_iterations; ++j) {
      my_scratch[(my_id_in_warp + THREADS_PER_WARP -
                  j - warp_id_in_tile * n_iterations) % THREADS_PER_WARP] = out_space[j][i];
    }
    __syncthreads();
    my_place = (my_id_in_warp + THREADS_PER_WARP - warp_id_in_tile * n_iterations) %
               THREADS_PER_WARP;
    current_stride = i * output_stride +
                     warp_id_in_tile * n_iterations * output_stride * nvec_in;
    for (unsigned int j = 0; j < n_iterations; ++j) {
      my_scratch[j + warp_id_in_tile * n_iterations].store_to(my_output_t_tile,
                                                              current_stride + my_place);
      my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
      current_stride += output_stride * nvec_in;
    }
    __syncthreads();
  }

  my_dbias_scratch[threadIdx.x] = partial_dbias;
  __syncthreads();
  // TODO(ptredak): check if the regular reduction is better
  if (warp_id_in_tile == 0) {
#pragma unroll
    for (unsigned int i = 1; i < n_warps_per_tile; ++i) {
      CVec tmp = my_dbias_scratch[threadIdx.x + i * THREADS_PER_WARP];
#pragma unroll
      for (unsigned int j = 0; j < nvec_in; ++j) {
        partial_dbias.data.elt[j] += tmp.data.elt[j];
      }
    }

    partial_dbias.store_to(my_partial_dbias_tile, my_id_in_warp);
  }
}

template <int nvec_in, int nvec_out, typename Param>
__global__ void
__launch_bounds__(cast_transpose_num_threads)
transpose_dbias_kernel_notaligned(const Param param,
                                       const size_t row_length,
                                       const size_t num_rows,
                                       const size_t num_tiles) {
  using IType = typename Param::InputType;
  using OType = typename Param::OutputType;
  using CType = typename Param::ComputeType;
  using IVec = Vec<IType, nvec_in>;
  using OVec = Vec<OType, nvec_out>;
  using CVec = Vec<CType, nvec_in>;

  extern __shared__ char scratch[];

  const int warp_id = threadIdx.x / THREADS_PER_WARP;
  const unsigned int my_id_in_warp = threadIdx.x % THREADS_PER_WARP;
  const size_t num_tiles_x = (row_length + nvec_in * THREADS_PER_WARP - 1) /
                             (nvec_in * THREADS_PER_WARP);
  const size_t tile_id = blockIdx.x * blockDim.x / (THREADS_PER_WARP * n_warps_per_tile) +
                         warp_id / n_warps_per_tile;
  if (tile_id >= num_tiles) return;
  const size_t tile_id_x = tile_id % num_tiles_x;
  const size_t tile_id_y = tile_id / num_tiles_x;

  const IType * const my_input_tile = param.input + (tile_id_x * nvec_in +
                                                     tile_id_y * row_length * nvec_out) *
                                                    THREADS_PER_WARP;
  OType * const my_output_t_tile = param.output_t + (tile_id_y * nvec_out +
                                                     tile_id_x * num_rows * nvec_in) *
                                                    THREADS_PER_WARP;
  CType * const my_partial_dbias_tile = param.workspace +
                                        (tile_id_x * (nvec_in * THREADS_PER_WARP) +
                                         tile_id_y * row_length);

  const size_t stride = row_length / nvec_in;
  const size_t output_stride = num_rows / nvec_out;
  const size_t row_length_rest = stride - tile_id_x * THREADS_PER_WARP;
  const size_t row_height_rest = output_stride - tile_id_y * THREADS_PER_WARP;
  const unsigned int tile_length = row_length_rest > THREADS_PER_WARP ? THREADS_PER_WARP
                                                                      : row_length_rest;
  const unsigned int tile_height = row_height_rest > THREADS_PER_WARP ? THREADS_PER_WARP
                                                                      : row_height_rest;

  OVec * const my_scratch = reinterpret_cast<OVec *>(scratch) +
                            (my_id_in_warp + warp_id / n_warps_per_tile * THREADS_PER_WARP) *
                            (THREADS_PER_WARP + 1);

  CVec * const my_dbias_scratch = reinterpret_cast<CVec *>(scratch);

  IVec in[2][nvec_out];
  const unsigned int warp_id_in_tile = warp_id % n_warps_per_tile;
  constexpr unsigned int n_iterations = THREADS_PER_WARP / n_warps_per_tile;
  OVec out_space[n_iterations][nvec_in];
  CVec partial_dbias;

  size_t current_stride = warp_id_in_tile * n_iterations * nvec_out * stride;
  unsigned int my_place = (my_id_in_warp + THREADS_PER_WARP -
                           warp_id_in_tile * n_iterations) %
                          THREADS_PER_WARP;
  const CType scale_inv = param.scale_inv != nullptr ? *param.scale_inv : 1;

  partial_dbias.clear();

  {
    const bool valid_load = my_place < tile_length &&
                            warp_id_in_tile * n_iterations < tile_height;
#pragma unroll
    for (unsigned int i = 0; i < nvec_out; ++i) {
      if (valid_load) {
        in[0][i].load_from(my_input_tile, current_stride + my_place + stride * i);
      } else {
        in[0][i].clear();
      }
    }
  }
#pragma unroll
  for (unsigned int i = 0; i < n_iterations; ++i) {
    const unsigned int my_place_in = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
    const unsigned int current_in = (i + 1) % 2;
    if (i < n_iterations - 1) {
      const bool valid_load = my_place_in < tile_length &&
                              warp_id_in_tile * n_iterations + i + 1 < tile_height;
#pragma unroll
      for (unsigned int j = 0; j < nvec_out; ++j) {
        if (valid_load) {
          in[current_in][j].load_from(my_input_tile,
                                      current_stride + my_place_in + stride * (nvec_out + j));
        } else {
          in[current_in][j].clear();
        }
      }
    }
    OVec out_trans[nvec_in];  // NOLINT(*)
    const bool valid_store = my_place < tile_length &&
                             warp_id_in_tile * n_iterations + i < tile_height;
    transpose_regs_partial_dbias(
                    in[current_in ^ 1],
                    out_trans,
                    partial_dbias,
                    scale_inv,
                    (my_id_in_warp + i + warp_id_in_tile * n_iterations) % THREADS_PER_WARP);

#pragma unroll
    for (unsigned int j = 0; j < nvec_in; ++j) {
      out_space[i][j].data.vec = out_trans[j].data.vec;
    }
    my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
    current_stride += nvec_out * stride;
  }

  for (unsigned int i = 0; i < nvec_in; ++i) {
#pragma unroll
    for (unsigned int j = 0; j < n_iterations; ++j) {
      my_scratch[(my_id_in_warp + THREADS_PER_WARP -
                  j - warp_id_in_tile * n_iterations) % THREADS_PER_WARP] = out_space[j][i];
    }
    __syncthreads();
    my_place = (my_id_in_warp + THREADS_PER_WARP - warp_id_in_tile * n_iterations) %
               THREADS_PER_WARP;
    current_stride = i * output_stride +
                     warp_id_in_tile * n_iterations * output_stride * nvec_in;
    for (unsigned int j = 0; warp_id_in_tile * n_iterations + j < tile_length; ++j) {
      const bool valid_store = my_place < tile_height;
      if (valid_store) {
        my_scratch[j + warp_id_in_tile * n_iterations].store_to(my_output_t_tile,
                                                                current_stride + my_place);
      }
      my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
      current_stride += output_stride * nvec_in;
    }
    __syncthreads();
  }

  my_dbias_scratch[threadIdx.x] = partial_dbias;
  __syncthreads();
  // TODO(ptredak): check if the regular reduction is better
  if (warp_id_in_tile == 0) {
#pragma unroll
    for (unsigned int i = 1; i < n_warps_per_tile; ++i) {
      CVec tmp = my_dbias_scratch[threadIdx.x + i * THREADS_PER_WARP];
#pragma unroll
      for (unsigned int j = 0; j < nvec_in; ++j) {
        partial_dbias.data.elt[j] += tmp.data.elt[j];
      }
    }

    if (my_id_in_warp < tile_length) {
      partial_dbias.store_to(my_partial_dbias_tile, my_id_in_warp);
    }
  }
}

constexpr size_t reduce_dbias_num_threads = 256;

template<int nvec, typename ComputeType, typename OutputType>
__global__ void
__launch_bounds__(reduce_dbias_num_threads)
reduce_dbias_kernel(OutputType*  const dbias_output,
                    const ComputeType* const dbias_partial,
                    const int row_length,
                    const int num_rows) {
  using ComputeVec = Vec<ComputeType, nvec>;
  using OutputVec  = Vec<OutputType,  nvec>;

  const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_id * nvec >= row_length) return;

  const ComputeType* const thread_in_base  = dbias_partial + thread_id * nvec;
  OutputType*  const thread_out_base = dbias_output  + thread_id * nvec;

  const int stride_in_vec = row_length / nvec;

  ComputeVec ldg_vec;
  ComputeVec acc_vec; acc_vec.clear();
  for (int i = 0; i < num_rows; ++i) {
    ldg_vec.load_from(thread_in_base, i * stride_in_vec);
#pragma unroll
    for (int e = 0; e < nvec; ++e) {
      acc_vec.data.elt[e] += ldg_vec.data.elt[e];
    }
  }

  OutputVec  stg_vec;
#pragma unroll
  for (int e = 0; e < nvec; ++e) {
    stg_vec.data.elt[e] = OutputType(acc_vec.data.elt[e]);
  }
  stg_vec.store_to(thread_out_base, 0);
}

void populate_transpose_dbias_workspace_config(const Tensor &input, /*cast*/
                                                    Tensor* workspace,
                                                    const int nvec_out) {
  const size_t row_length = input.data.shape[1];
  const size_t num_rows   = input.data.shape[0];

  const size_t tile_size_y = (nvec_out * THREADS_PER_WARP);
  NVTE_CHECK(num_rows % nvec_out == 0, "Unsupported shape.");

  const size_t num_rows_partial_dbias = DIVUP(num_rows, tile_size_y);

  workspace->data.shape = {num_rows_partial_dbias, row_length};
  workspace->data.dtype = DType::kFloat32;
}

template <typename BiasType>
void reduce_dbias(const Tensor &workspace, Tensor *dbias,
                  const size_t row_length, const size_t num_rows, const int nvec_out,
                  hipStream_t stream) {
  constexpr int reduce_dbias_store_bytes  = 8;  // stg.64
  constexpr int reduce_dbias_nvec         = reduce_dbias_store_bytes / sizeof(BiasType);

  NVTE_CHECK(row_length % reduce_dbias_nvec == 0, "Unsupported shape.");

  const size_t reduce_dbias_row_length = row_length;
  const size_t reduce_dbias_num_rows   = DIVUP(num_rows,
                                               static_cast<size_t>(nvec_out *
                                                                   THREADS_PER_WARP));
  const size_t reduce_dbias_num_blocks = DIVUP(row_length,
                                               reduce_dbias_num_threads * reduce_dbias_nvec);

  reduce_dbias_kernel<reduce_dbias_nvec, fp32, BiasType>
    <<<reduce_dbias_num_blocks,
    reduce_dbias_num_threads,
    0,
    stream>>>(
        reinterpret_cast<BiasType *>(dbias->data.dptr),
        reinterpret_cast<const fp32 *>(workspace.data.dptr),
        reduce_dbias_row_length,
        reduce_dbias_num_rows);
}

void fp8_transpose_dbias(const Tensor &input,
                          Tensor *transposed_output,
                          Tensor *dbias,
                          Tensor *workspace,
                          hipStream_t stream) {
  CheckInputTensor(input, "fp8_transpose_dbias_input");
  CheckOutputTensor(*transposed_output, "transposed_output");
  CheckOutputTensor(*dbias, "dbias");

  NVTE_CHECK(input.data.shape.size() == 2, "Input must have 2 dimensions.");
  NVTE_CHECK(transposed_output->data.shape.size() == 2, "T output must have 2 dimensions.");
  const size_t row_length = input.data.shape[1];
  const size_t num_rows = input.data.shape[0];

  NVTE_CHECK(transposed_output->data.shape[0] == row_length, "Wrong dimension of T output.");
  NVTE_CHECK(transposed_output->data.shape[1] == num_rows, "Wrong dimension of T output.");

  NVTE_CHECK(transposed_output->data.dtype == input.data.dtype,
                           "T output must have the same type as input.");
  NVTE_CHECK(dbias->data.shape == std::vector<size_t>{ row_length }, "Wrong shape of DBias.");

  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(dbias->data.dtype, BiasType,
    TRANSFORMER_ENGINE_TYPE_SWITCH_FP8ONLY(input.data.dtype, Type,
      constexpr int type_size = sizeof(Type);
      constexpr int nvec_in = desired_load_size / type_size;
      constexpr int nvec_out = desired_store_size / type_size;

      if (workspace->data.dptr == nullptr) {
        populate_transpose_dbias_workspace_config(input, workspace, nvec_out);
        return;
      }

      NVTE_CHECK(row_length % nvec_in  == 0, "Unsupported shape.");
      NVTE_CHECK(num_rows   % nvec_out == 0, "Unsupported shape.");
      const size_t n_tiles = DIVUP(row_length, static_cast<size_t>(nvec_in * THREADS_PER_WARP)) *
                             DIVUP(num_rows, static_cast<size_t>(nvec_out * THREADS_PER_WARP));
      const size_t n_warps_per_block = cast_transpose_num_threads / THREADS_PER_WARP;
      const size_t n_blocks = DIVUP(n_tiles * n_warps_per_tile, n_warps_per_block);

      const bool full_tile = row_length % (nvec_in * THREADS_PER_WARP) == 0 &&
                             num_rows % (nvec_out * THREADS_PER_WARP) == 0;

      using ComputeType = fp32;
      constexpr size_t shared_size_transpose = cast_transpose_num_threads / n_warps_per_tile *
                                               (THREADS_PER_WARP + 1) *
                                               sizeof(Vec<Type, nvec_out>);
      constexpr size_t shared_size_dbias = cast_transpose_num_threads *
                                           sizeof(Vec<ComputeType, nvec_in>);
      static_assert(shared_size_transpose >= shared_size_dbias);
      using Param = TDBiasParam<Type, Type, ComputeType>;
      Param param;
      param.input     = reinterpret_cast<const Type *>(input.data.dptr);
      param.output_t  = reinterpret_cast<Type *>(transposed_output->data.dptr);
      param.scale_inv = reinterpret_cast<const ComputeType *>(transposed_output->scale_inv.dptr);
      param.workspace = reinterpret_cast<ComputeType *>(workspace->data.dptr);

      if (full_tile) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(transpose_dbias_kernel<nvec_in), nvec_out, Param>,
                             hipFuncAttributePreferredSharedMemoryCarveout,
                             100);
        transpose_dbias_kernel<nvec_in, nvec_out, Param>
          <<<n_blocks,
             cast_transpose_num_threads,
             shared_size_transpose,
             stream>>>(param, row_length, num_rows, n_tiles);
      } else {
        hipFuncSetAttribute(reinterpret_cast<const void*>(transpose_dbias_kernel_notaligned<nvec_in), nvec_out, Param>,
                             hipFuncAttributePreferredSharedMemoryCarveout,
                             100);
        transpose_dbias_kernel_notaligned<nvec_in, nvec_out, Param>
          <<<n_blocks,
             cast_transpose_num_threads,
             shared_size_transpose,
             stream>>>(param, row_length, num_rows, n_tiles);
      }

      reduce_dbias<BiasType>(*workspace, dbias, row_length, num_rows, nvec_out, stream);
    );  // NOLINT(*)
  );  // NOLINT(*)
}


}  // namespace transformer_engine

void nvte_fp8_transpose_dbias(const NVTETensor input,
                               NVTETensor transposed_output,
                               NVTETensor dbias,
                               NVTETensor workspace,
                               hipStream_t stream) {
  NVTE_API_CALL(nvte_fp8_transpose_dbias);
  using namespace transformer_engine;
  fp8_transpose_dbias(*reinterpret_cast<const Tensor*>(input),
                       reinterpret_cast<Tensor*>(transposed_output),
                       reinterpret_cast<Tensor*>(dbias),
                       reinterpret_cast<Tensor*>(workspace),
                       stream);
}
