#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hip/hip_runtime.h>
#include <transformer_engine/cast_transpose_noop.h>
#include <transformer_engine/transpose.h>

#include <algorithm>

#include "../common.h"
#include "../util/rtc.h"
#include "../util/string.h"
#include "../utils.cuh"

namespace transformer_engine {

namespace {

// String with RTC kernel implementation
#include "string_code_transpose_rtc_cast_transpose_cu.h"

// Hard-coded kernel parameters
using CType = float;
constexpr size_t warps_per_tile = 4;
constexpr size_t block_size = THREADS_PER_WARP * warps_per_tile;

/* Performance heuristics for optimized kernel parameters */
struct KernelConfig {
  /** Vector load size */
  size_t load_size = 0;
  /** Vector store size to transposed output */
  size_t store_size = 0;

  /* Whether config is valid */
  bool valid = false;
  /* Number of CUDA blocks */
  size_t num_blocks = 0;

  /* Number of active SMs */
  size_t active_sm_count = 0;
  /* Elements per L1 cache load */
  size_t elements_per_load = 0;
  /* Elements per L1 cache store to cast output*/
  size_t elements_per_store_c = 0;
  /* Elements per L1 cache store to transposed output */
  size_t elements_per_store_t = 0;

  KernelConfig(size_t row_length, size_t num_rows, size_t itype_size, size_t otype_size,
               size_t load_size_, size_t store_size_)
      : load_size{load_size_}, store_size{store_size_} {
    // Check that tiles are correctly aligned
    constexpr size_t cache_line_size = 128;
    if (load_size % itype_size != 0 || store_size % otype_size != 0 ||
        cache_line_size % itype_size != 0 || cache_line_size % otype_size != 0) {
      return;
    }
    const size_t row_tile_elements = load_size * THREADS_PER_WARP / itype_size;
    const size_t col_tile_elements = store_size * THREADS_PER_WARP / otype_size;
    valid = (row_length % row_tile_elements == 0 && num_rows % col_tile_elements == 0);
    if (!valid) {
      return;
    }

    // Number of CUDA blocks
    num_blocks = (row_length / row_tile_elements) * (num_rows / col_tile_elements);

    // Parameters for performance model
    constexpr size_t warps_per_sm = 16;  // Rough estimate for saturated SMs
    active_sm_count = std::min(DIVUP(num_blocks * warps_per_tile, warps_per_sm),
                               static_cast<size_t>(cuda::sm_count()));
    elements_per_load = (std::min(cache_line_size, row_tile_elements * itype_size) / itype_size);
    elements_per_store_c = (std::min(cache_line_size, row_tile_elements * otype_size) / otype_size);
    elements_per_store_t = (std::min(cache_line_size, col_tile_elements * otype_size) / otype_size);
  }

  /* Compare by estimated cost */
  bool operator<(const KernelConfig &other) const {
    if (this->valid && other.valid) {
      // cost ~ (1/elements_per_load
      //         + 1/elements_per_store_c
      //         + 1/elements_per_store_t) / active_sms
      // Note: Integer arithmetic ensures stable ordering
      const auto &l1 = this->elements_per_load;
      const auto &sc1 = this->elements_per_store_c;
      const auto &st1 = this->elements_per_store_t;
      const auto &p1 = this->active_sm_count;
      const auto &l2 = other.elements_per_load;
      const auto &sc2 = other.elements_per_store_c;
      const auto &st2 = other.elements_per_store_t;
      const auto &p2 = other.active_sm_count;
      const auto scale = l1 * sc1 * st1 * p1 * l2 * sc2 * st2 * p2;
      const auto cost1 = (scale / l1 + scale / sc1 + scale / st1) / p1;
      const auto cost2 = (scale / l2 + scale / sc2 + scale / st2) / p2;
      return cost1 < cost2;
    } else {
      return this->valid && !other.valid;
    }
  }
};

template <size_t load_size, size_t store_size, typename IType, typename OType>
__global__ void __launch_bounds__(block_size)
    cast_transpose_general_kernel(const IType *__restrict__ const input,
                                  const CType *__restrict__ const noop,
                                  OType *__restrict__ const output_c,
                                  OType *__restrict__ const output_t,
                                  const CType *__restrict__ const scale_ptr,
                                  CType *__restrict__ const amax_ptr, const size_t row_length,
                                  const size_t num_rows) {
  if (noop != nullptr && noop[0] == 1.0f) return;

  // Vectorized load/store sizes
  constexpr size_t nvec_in = load_size / sizeof(IType);
  constexpr size_t nvec_out = store_size / sizeof(OType);
  using IVec = Vec<IType, nvec_in>;
  using OVecT = Vec<OType, nvec_out>;

  // Thread indices
  // Note: Block is interpreted as a warp_size x num_warps grid
  constexpr size_t bdimx = THREADS_PER_WARP;
  constexpr size_t bdimy = warps_per_tile;
  const size_t tid = threadIdx.x;
  const size_t tidx = tid % bdimx;
  const size_t tidy = tid / bdimx;
  const size_t bid = blockIdx.x;

  // Input tensors are divided into tiles
  // Note: Each tile is a warp_size x warp_size grid of nvec_out x nvec_in subtiles
  constexpr size_t tile_dim_m = THREADS_PER_WARP * nvec_out;
  constexpr size_t tile_dim_n = THREADS_PER_WARP * nvec_in;

  // Position of tile within tensor
  const size_t num_tiles_m = (num_rows + tile_dim_m - 1) / tile_dim_m;
  const size_t tile_id_m = bid % num_tiles_m;
  const size_t tile_id_n = bid / num_tiles_m;
  const size_t tile_row = tile_id_m * tile_dim_m;
  const size_t tile_col = tile_id_n * tile_dim_n;

  // Number of nvec_out x nvec_in subtiles for each thread to
  // load/store
  constexpr size_t num_iterations = THREADS_PER_WARP / warps_per_tile;

  // FP8 factors
  const CType scale = scale_ptr == nullptr ? 1 : *scale_ptr;
  CType amax = 0;

  // Load input and store to registers
  // Note: Each thread loads num_iterations subtiles, computes amax,
  // casts type, and transposes in registers.
  OVecT local_output_t[nvec_in][num_iterations];
#pragma unroll
  for (size_t iter = 0; iter < num_iterations; ++iter) {
    const size_t i1 = tidy + iter * bdimy;
    const size_t j1 = tidx;
#pragma unroll
    for (size_t i2 = 0; i2 < nvec_out; ++i2) {
      const size_t row = tile_row + i1 * nvec_out + i2;
      const size_t col = tile_col + j1 * nvec_in;
      if (row < num_rows) {
#pragma unroll
        for (size_t j2 = 0; j2 < nvec_in; ++j2) {
          if (col + j2 < row_length) {
            const CType in = input[row * row_length + col + j2];
            const OType out = OType(in * scale);
            __builtin_assume(amax >= 0);
            amax = fmaxf(fabsf(in), amax);
            output_c[row * row_length + col + j2] = out;
            local_output_t[j2][iter].data.elt[i2] = out;
          }
        }
      }
    }
  }

  // Copy transposed output from registers to global memory
  __shared__ OVecT shared_output_t[THREADS_PER_WARP][THREADS_PER_WARP + 1];
#pragma unroll
  for (size_t j2 = 0; j2 < nvec_in; ++j2) {
#pragma unroll
    for (size_t iter = 0; iter < num_iterations; ++iter) {
      const size_t i1 = tidy + iter * bdimy;
      const size_t j1 = tidx;
      shared_output_t[j1][i1] = local_output_t[j2][iter];
    }
    __syncthreads();
#pragma unroll
    for (size_t iter = 0; iter < num_iterations; ++iter) {
      const size_t i1 = tidx;
      const size_t j1 = tidy + iter * bdimy;
      const size_t row = tile_row + i1 * nvec_out;
      const size_t col = tile_col + j1 * nvec_in + j2;
      if (col < row_length) {
#pragma unroll
        for (size_t i2 = 0; i2 < nvec_out; ++i2) {
          if (row + i2 < num_rows) {
            output_t[col * num_rows + row + i2] = shared_output_t[j1][i1].data.elt[i2];
          }
        }
      }
    }
    __syncthreads();
  }

  // Reduce amax over block
  if (amax_ptr != nullptr) {
    amax = reduce_max<warps_per_tile>(amax, tidy);
    if (threadIdx.x == 0) {
      atomicMaxFloat(amax_ptr, amax);
    }
  }
}

}  // namespace

void cast_transpose(const Tensor &input, const Tensor &noop, Tensor *cast_output_,
                    Tensor *transposed_output_, hipStream_t stream) {
  Tensor &cast_output = *cast_output_;
  Tensor &transposed_output = *transposed_output_;

  // Check no-op flag
  if (noop.data.dptr != nullptr) {
    size_t numel = 1;
    for (const auto &dim : noop.data.shape) {
      numel *= dim;
    }
    NVTE_CHECK(numel == 1, "Expected 1 element, but found ", numel, ".");
    NVTE_CHECK(noop.data.dtype == DType::kFloat32);
    NVTE_CHECK(noop.data.dptr != nullptr);
  }

  // Check tensor dims
  CheckInputTensor(input, "cast_transpose_input");
  CheckOutputTensor(cast_output, "cast_output");
  CheckOutputTensor(transposed_output, "transposed_output");
  NVTE_CHECK(input.data.shape.size() == 2, "Input must have 2 dimensions.");
  NVTE_CHECK(cast_output.data.shape.size() == 2, "Cast output must have 2 dimensions.");
  NVTE_CHECK(transposed_output.data.shape.size() == 2, "Transposed output must have 2 dimensions.");
  const size_t row_length = input.data.shape[1];
  const size_t num_rows = input.data.shape[0];
  NVTE_CHECK(cast_output.data.shape[0] == num_rows, "Wrong dimension of cast output.");
  NVTE_CHECK(cast_output.data.shape[1] == row_length, "Wrong dimension of cast output.");
  NVTE_CHECK(transposed_output.data.shape[0] == row_length,
             "Wrong dimension of transposed output.");
  NVTE_CHECK(transposed_output.data.shape[1] == num_rows, "Wrong dimension of transposed output.");

  // Check tensor pointers
  NVTE_CHECK(input.data.dptr != nullptr, "Input is not allocated.");
  NVTE_CHECK(cast_output.data.dptr != nullptr, "Cast output is not allocated.");
  NVTE_CHECK(transposed_output.data.dptr != nullptr, "Transposed output is not allocated.");
  NVTE_CHECK(cast_output.data.dtype == transposed_output.data.dtype,
             "Cast and transposed output types must match.");
  NVTE_CHECK(cast_output.amax.dptr == transposed_output.amax.dptr,
             "Cast and transposed outputs need to share amax tensor.");
  NVTE_CHECK(cast_output.scale.dptr == transposed_output.scale.dptr,
             "Cast and transposed outputs need to share scale tensor.");

  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(
      input.data.dtype, InputType,
      TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(
          cast_output.data.dtype, OutputType,
          constexpr const char *itype_name = TypeInfo<InputType>::name;
          constexpr const char *otype_name = TypeInfo<OutputType>::name;
          constexpr size_t itype_size = sizeof(InputType);
          constexpr size_t otype_size = sizeof(OutputType);

          // Choose between runtime-compiled or statically-compiled kernel
          const bool aligned =
              (row_length % THREADS_PER_WARP == 0 && num_rows % THREADS_PER_WARP == 0);
          if (aligned && rtc::is_enabled()) {  // Runtime-compiled tuned kernel
            // Pick kernel config
            std::vector<KernelConfig> kernel_configs;
            kernel_configs.reserve(16);
            auto add_config = [&](size_t load_size, size_t store_size) {
              kernel_configs.emplace_back(row_length, num_rows, itype_size, otype_size, load_size,
                                          store_size);
            };
            add_config(8, 8);
            add_config(4, 8);
            add_config(8, 4);
            add_config(4, 4);
            add_config(2, 8);
            add_config(8, 2);
            add_config(2, 4);
            add_config(4, 2);
            add_config(2, 2);
            add_config(1, 8);
            add_config(8, 1);
            add_config(1, 4);
            add_config(4, 1);
            add_config(1, 2);
            add_config(2, 1);
            add_config(1, 1);
            const auto &kernel_config =
                *std::min_element(kernel_configs.begin(), kernel_configs.end());
            NVTE_CHECK(kernel_config.valid, "invalid kernel config");
            const size_t load_size = kernel_config.load_size;
            const size_t store_size = kernel_config.store_size;
            const size_t num_blocks = kernel_config.num_blocks;

            // Compile NVRTC kernel if needed and launch
            auto &rtc_manager = rtc::KernelManager::instance();
            const std::string kernel_label = concat_strings(
                "cast_transpose"
                ",itype=",
                itype_name, ",otype=", otype_name, ",load_size=", load_size,
                ",store_size=", store_size);
            if (!rtc_manager.is_compiled(kernel_label)) {
              std::string code = string_code_transpose_rtc_cast_transpose_cu;
              code = regex_replace(code, "__ITYPE__", itype_name);
              code = regex_replace(code, "__OTYPE__", otype_name);
              code = regex_replace(code, "__LOAD_SIZE__", load_size);
              code = regex_replace(code, "__STORE_SIZE__", store_size);
              code = regex_replace(code, "__WARPS_PER_TILE__", warps_per_tile);
              code = regex_replace(code, "__BLOCK_SIZE__", block_size);
              rtc_manager.compile(kernel_label, "cast_transpose_optimized_kernel", code,
                                  "transformer_engine/common/transpose/rtc/cast_transpose.cu");
            }
            rtc_manager.launch(kernel_label, num_blocks, block_size, 0, stream,
                               static_cast<const InputType *>(input.data.dptr),
                               reinterpret_cast<const CType *>(noop.data.dptr),
                               static_cast<OutputType *>(cast_output.data.dptr),
                               static_cast<OutputType *>(transposed_output.data.dptr),
                               static_cast<const CType *>(cast_output.scale.dptr),
                               static_cast<CType *>(cast_output.amax.dptr), row_length, num_rows);
          } else {  // Statically-compiled general kernel
            constexpr size_t load_size = 4;
            constexpr size_t store_size = 4;
            constexpr size_t row_tile_size = load_size / itype_size * THREADS_PER_WARP;
            constexpr size_t col_tile_size = store_size / otype_size * THREADS_PER_WARP;
            const int num_blocks =
                (DIVUP(row_length, row_tile_size) * DIVUP(num_rows, col_tile_size));
            cast_transpose_general_kernel<load_size, store_size, InputType, OutputType>
                <<<num_blocks, block_size, 0, stream>>>(
                    static_cast<const InputType *>(input.data.dptr),
                    reinterpret_cast<const CType *>(noop.data.dptr),
                    static_cast<OutputType *>(cast_output.data.dptr),
                    static_cast<OutputType *>(transposed_output.data.dptr),
                    static_cast<const CType *>(cast_output.scale.dptr),
                    static_cast<CType *>(cast_output.amax.dptr), row_length, num_rows);
          });  // NOLINT(*)
  );           // NOLINT(*)
}

}  // namespace transformer_engine

void nvte_cast_transpose(const NVTETensor input, NVTETensor cast_output,
                         NVTETensor transposed_output, hipStream_t stream) {
  NVTE_API_CALL(nvte_cast_transpose);
  using namespace transformer_engine;
  auto noop = Tensor();
  cast_transpose(*reinterpret_cast<const Tensor *>(input), noop,
                 reinterpret_cast<Tensor *>(cast_output),
                 reinterpret_cast<Tensor *>(transposed_output), stream);
}

void nvte_cast_transpose_with_noop(const NVTETensor input, const NVTETensor noop,
                                   NVTETensor cast_output, NVTETensor transposed_output,
                                   hipStream_t stream) {
  NVTE_API_CALL(nvte_cast_transpose_with_noop);
  using namespace transformer_engine;
  cast_transpose(*reinterpret_cast<const Tensor *>(input), *reinterpret_cast<const Tensor *>(noop),
                 reinterpret_cast<Tensor *>(cast_output),
                 reinterpret_cast<Tensor *>(transposed_output), stream);
}
