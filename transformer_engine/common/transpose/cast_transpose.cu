#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <transformer_engine/transpose.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cfloat>
#include "../utils.cuh"
#include "../common.h"

namespace transformer_engine {

template <bool full_tile, int nvec_in, int nvec_out, typename IVec, typename OVec, typename CType>
inline __device__ void cast_and_transpose_regs(const IVec (&in)[nvec_out],
                                               OVec (&out_trans)[nvec_in],
                                               typename OVec::type *output_cast_tile,
                                               const size_t current_place,
                                               const size_t stride,
                                               CType &max,  // NOLINT(*)
                                               const CType scale,
                                               const bool valid_store) {
    using T = typename OVec::type;
    using OVecC = Vec<T, nvec_in>;
#pragma unroll
    for (unsigned int i = 0; i < nvec_out; ++i) {
        OVecC out_cast;
#pragma unroll
        for (unsigned int j = 0; j < nvec_in; ++j) {
            const CType tmp = static_cast<CType>(in[i].data.elt[j]);
            const T elt_o = T(scale * tmp);

            out_cast.data.elt[j]     = elt_o;
            out_trans[j].data.elt[i] = elt_o;  // thread tile transpose

            __builtin_assume(max >= 0);
            max = fmaxf(fabsf(tmp), max);
        }
        if (full_tile || valid_store) {
          out_cast.store_to(output_cast_tile, current_place + stride * i);
        }
    }
}


// STUFF TO TUNE
constexpr unsigned int n_warps_per_tile = 4;

constexpr unsigned int max_threads_per_block = 256;
static_assert(n_warps_per_tile * THREADS_PER_WARP <= max_threads_per_block);
constexpr unsigned int cast_transpose_num_threads = n_warps_per_tile * THREADS_PER_WARP;

template <int nvec_in, int nvec_out, typename CType, typename IType, typename OType>
__global__ void
__launch_bounds__(cast_transpose_num_threads)
cast_transpose_kernel(const IType * const input,
                      OType * const output_c,
                      OType * const output_t,
                      const CType * const scale_ptr,
                      CType * const amax,
                      const size_t row_length,
                      const size_t num_rows,
                      const size_t num_tiles) {
  using IVec = Vec<IType, nvec_in>;
  using OVec = Vec<OType, nvec_out>;

  extern __shared__ char scratch[];

  const int warp_id = threadIdx.x / THREADS_PER_WARP;
  const int my_id_in_warp = threadIdx.x % THREADS_PER_WARP;
  const size_t num_tiles_x = row_length / (nvec_in * THREADS_PER_WARP);
  const size_t tile_id = blockIdx.x * blockDim.x / (THREADS_PER_WARP * n_warps_per_tile) +
                         warp_id / n_warps_per_tile;
  if (tile_id >= num_tiles) return;
  const size_t tile_id_x = tile_id % num_tiles_x;
  const size_t tile_id_y = tile_id / num_tiles_x;

  const IType * const my_input_tile = input + (tile_id_x * nvec_in +
                                               tile_id_y * row_length * nvec_out) *
                                              THREADS_PER_WARP;
  OType * const my_output_c_tile = output_c + (tile_id_x * nvec_in +
                                               tile_id_y * row_length * nvec_out) *
                                              THREADS_PER_WARP;
  OType * const my_output_t_tile = output_t + (tile_id_y * nvec_out +
                                               tile_id_x * num_rows * nvec_in) *
                                              THREADS_PER_WARP;
  OVec * const my_scratch = reinterpret_cast<OVec*>(scratch) +
                            (my_id_in_warp + warp_id / n_warps_per_tile * THREADS_PER_WARP) *
                            (THREADS_PER_WARP + 1);

  IVec in[2][nvec_out];
  const unsigned int warp_id_in_tile = warp_id % n_warps_per_tile;
  constexpr unsigned int n_iterations = THREADS_PER_WARP / n_warps_per_tile;
  OVec out_space[n_iterations][nvec_in];

  const size_t stride = row_length / nvec_in;
  const size_t output_stride = num_rows / nvec_out;
  size_t current_stride = warp_id_in_tile * n_iterations * nvec_out * stride;
  unsigned int my_place = (my_id_in_warp + THREADS_PER_WARP -
                           warp_id_in_tile * n_iterations) %
                         THREADS_PER_WARP;
  CType max = 0;
  const CType scale = scale_ptr != nullptr ? *scale_ptr : 1;
#pragma unroll
  for (unsigned int i = 0; i < nvec_out; ++i) {
    in[0][i].load_from(my_input_tile, current_stride + my_place + stride * i);
  }
#pragma unroll
  for (unsigned int i = 0; i < n_iterations; ++i) {
    const size_t current_place = current_stride + my_place;
    const unsigned int my_place_in = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
    const unsigned int current_in = (i + 1) % 2;
    if (i < n_iterations - 1) {
#pragma unroll
      for (unsigned int j = 0; j < nvec_out; ++j) {
        in[current_in][j].load_from(my_input_tile,
                                    current_stride + my_place_in + stride * (nvec_out + j));
      }
    }
    OVec out_trans[nvec_in];  // NOLINT(*)
    cast_and_transpose_regs<true>(in[current_in ^ 1], out_trans, my_output_c_tile,
                                  current_place, stride, max, scale, true);
#pragma unroll
    for (unsigned int j = 0; j < nvec_in; ++j) {
      out_space[i][j].data.vec = out_trans[j].data.vec;
    }
    my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
    current_stride += nvec_out * stride;
  }

  for (unsigned int i = 0; i < nvec_in; ++i) {
#pragma unroll
    for (unsigned int j = 0; j < n_iterations; ++j) {
      my_scratch[(my_id_in_warp + THREADS_PER_WARP -
                  j - warp_id_in_tile * n_iterations) % THREADS_PER_WARP] = out_space[j][i];
    }
    __syncthreads();
    my_place = (my_id_in_warp + THREADS_PER_WARP - warp_id_in_tile * n_iterations) %
               THREADS_PER_WARP;
    current_stride = i * output_stride +
                     warp_id_in_tile * n_iterations * output_stride * nvec_in;
    for (unsigned int j = 0; j < n_iterations; ++j) {
      my_scratch[j + warp_id_in_tile * n_iterations].store_to(my_output_t_tile,
                                                              current_stride + my_place);
      my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
      current_stride += output_stride * nvec_in;
    }
    __syncthreads();
  }

  /* warp tile amax reduce*/
  max = reduce_max<cast_transpose_num_threads / THREADS_PER_WARP>(max, warp_id);

  if (threadIdx.x == 0) {
    static_assert(std::is_same<CType, float>::value);
    if (amax != nullptr) atomicMaxFloat(amax, max);
  }
}

template <int nvec_in, int nvec_out, typename CType, typename IType, typename OType>
__global__ void
__launch_bounds__(cast_transpose_num_threads)
cast_transpose_kernel_notaligned(const IType * const input,
                                 OType * const output_c,
                                 OType * const output_t,
                                 const CType * const scale_ptr,
                                 CType * const amax,
                                 const size_t row_length,
                                 const size_t num_rows,
                                 const size_t num_tiles) {
  using IVec = Vec<IType, nvec_in>;
  using OVec = Vec<OType, nvec_out>;

  extern __shared__ char scratch[];

  const int warp_id = threadIdx.x / THREADS_PER_WARP;
  const int my_id_in_warp = threadIdx.x % THREADS_PER_WARP;
  const size_t num_tiles_x = (row_length + nvec_in * THREADS_PER_WARP - 1) /
                             (nvec_in * THREADS_PER_WARP);
  const size_t tile_id = blockIdx.x * blockDim.x / (THREADS_PER_WARP * n_warps_per_tile) +
                         warp_id / n_warps_per_tile;
  if (tile_id >= num_tiles) return;
  const size_t tile_id_x = tile_id % num_tiles_x;
  const size_t tile_id_y = tile_id / num_tiles_x;

  const IType * const my_input_tile = input + (tile_id_x * nvec_in +
                                               tile_id_y * row_length * nvec_out) *
                                              THREADS_PER_WARP;
  OType * const my_output_c_tile = output_c + (tile_id_x * nvec_in +
                                               tile_id_y * row_length * nvec_out) *
                                              THREADS_PER_WARP;
  OType * const my_output_t_tile = output_t + (tile_id_y * nvec_out +
                                               tile_id_x * num_rows * nvec_in) *
                                              THREADS_PER_WARP;
  const size_t stride = row_length / nvec_in;
  const size_t output_stride = num_rows / nvec_out;
  const size_t row_length_rest = stride - tile_id_x * THREADS_PER_WARP;
  const size_t row_height_rest = output_stride - tile_id_y * THREADS_PER_WARP;
  const unsigned int tile_length = row_length_rest > THREADS_PER_WARP ? THREADS_PER_WARP
                                                                      : row_length_rest;
  const unsigned int tile_height = row_height_rest > THREADS_PER_WARP ? THREADS_PER_WARP
                                                                      : row_height_rest;

  OVec * const my_scratch = reinterpret_cast<OVec*>(scratch) +
                            (my_id_in_warp + warp_id / n_warps_per_tile * THREADS_PER_WARP) *
                            (THREADS_PER_WARP + 1);

  IVec in[2][nvec_out];
  const unsigned int warp_id_in_tile = warp_id % n_warps_per_tile;
  constexpr unsigned int n_iterations = THREADS_PER_WARP / n_warps_per_tile;
  OVec out_space[n_iterations][nvec_in];

  size_t current_stride = warp_id_in_tile * n_iterations * nvec_out * stride;
  unsigned int my_place = (my_id_in_warp + THREADS_PER_WARP -
                           warp_id_in_tile * n_iterations) %
                          THREADS_PER_WARP;
  CType max = 0;
  const CType scale = scale_ptr != nullptr ? *scale_ptr : 1;
  {
    const bool valid_load = my_place < tile_length &&
                            warp_id_in_tile * n_iterations < tile_height;
#pragma unroll
    for (unsigned int i = 0; i < nvec_out; ++i) {
      if (valid_load) {
        in[0][i].load_from(my_input_tile, current_stride + my_place + stride * i);
      } else {
        in[0][i].clear();
      }
    }
  }
#pragma unroll
  for (unsigned int i = 0; i < n_iterations; ++i) {
    const size_t current_place = current_stride + my_place;
    const unsigned int my_place_in = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
    const unsigned int current_in = (i + 1) % 2;
    if (i < n_iterations - 1) {
      const bool valid_load = my_place_in < tile_length &&
                              warp_id_in_tile * n_iterations + i + 1 < tile_height;
#pragma unroll
        for (unsigned int j = 0; j < nvec_out; ++j) {
          if (valid_load) {
            in[current_in][j].load_from(my_input_tile,
                                        current_stride + my_place_in + stride * (nvec_out + j));
          } else {
            in[current_in][j].clear();
          }
        }
    }
    OVec out_trans[nvec_in];  // NOLINT(*)
    const bool valid_store = my_place < tile_length &&
                             warp_id_in_tile * n_iterations + i < tile_height;
    cast_and_transpose_regs<false>(in[current_in ^ 1], out_trans, my_output_c_tile,
                                   current_place, stride, max, scale, valid_store);
#pragma unroll
    for (unsigned int j = 0; j < nvec_in; ++j) {
      out_space[i][j].data.vec = out_trans[j].data.vec;
    }
    my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
    current_stride += nvec_out * stride;
  }

  for (unsigned int i = 0; i < nvec_in; ++i) {
#pragma unroll
    for (unsigned int j = 0; j < n_iterations; ++j) {
        my_scratch[(my_id_in_warp + THREADS_PER_WARP -
                    j - warp_id_in_tile * n_iterations) % THREADS_PER_WARP] = out_space[j][i];
    }
    __syncthreads();
    my_place = (my_id_in_warp + THREADS_PER_WARP - warp_id_in_tile * n_iterations) %
               THREADS_PER_WARP;
    current_stride = i * output_stride +
                     warp_id_in_tile * n_iterations * output_stride * nvec_in;
    for (unsigned int j = 0; warp_id_in_tile * n_iterations + j < tile_length; ++j) {
      const bool valid_store = my_place < tile_height;
      if (valid_store) {
        my_scratch[j + warp_id_in_tile * n_iterations].store_to(my_output_t_tile,
                                                                current_stride + my_place);
      }
      my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
      current_stride += output_stride * nvec_in;
    }
    __syncthreads();
  }

  /* warp tile amax reduce*/
  max = reduce_max<cast_transpose_num_threads / THREADS_PER_WARP>(max, warp_id);

  if (threadIdx.x == 0) {
    static_assert(std::is_same<CType, float>::value);
    if (amax != nullptr) atomicMaxFloat(amax, max);
  }
}

void cast_transpose(const Tensor &input,
                    Tensor *cast_output,
                    Tensor *transposed_output,
                    hipStream_t stream) {
  CheckInputTensor(input, "cast_transpose_input");
  CheckOutputTensor(*cast_output, "cast_output");
  CheckOutputTensor(*transposed_output, "transposed_output");

  NVTE_CHECK(input.data.shape.size() == 2, "Input must have 2 dimensions.");
  NVTE_CHECK(cast_output->data.shape.size() == 2, "C output must have 2 dimensions.");
  NVTE_CHECK(transposed_output->data.shape.size() == 2, "T output must have 2 dimensions.");
  NVTE_CHECK(input.data.shape == cast_output->data.shape,
             "Input and C output must have the same shape.");
  const size_t row_length = input.data.shape[1];
  const size_t num_rows = input.data.shape[0];

  NVTE_CHECK(transposed_output->data.shape[0] == row_length, "Wrong dimension of T output.");
  NVTE_CHECK(transposed_output->data.shape[1] == num_rows, "Wrong dimension of T output.");

  NVTE_CHECK(cast_output->data.dtype == transposed_output->data.dtype,
             "C and T outputs need to have the same type.");
  NVTE_CHECK(cast_output->amax.dptr == transposed_output->amax.dptr,
             "C and T outputs need to share amax tensor.");
  NVTE_CHECK(cast_output->scale.dptr == transposed_output->scale.dptr,
             "C and T outputs need to share scale tensor.");

// Launch specific cast-transpose kernel
#define LAUNCH_KERNEL(kernel, nvec_in, nvec_out, n_tiles, n_blocks, InputType, OutputType) \
  do {                                                                  \
    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel<nvec_in), nvec_out, fp32, InputType, OutputType>, \
                         hipFuncAttributePreferredSharedMemoryCarveout, \
                         100);                                          \
    kernel<nvec_in, nvec_out, fp32, InputType, OutputType>              \
      <<<n_blocks,                                                      \
         cast_transpose_num_threads,                                    \
         cast_transpose_num_threads / n_warps_per_tile *                \
         (THREADS_PER_WARP + 1) * sizeof(Vec<OutputType, nvec_out>),    \
         stream>>>(                                                     \
          reinterpret_cast<const InputType *>(input.data.dptr),         \
          reinterpret_cast<OutputType *>(cast_output->data.dptr),       \
          reinterpret_cast<OutputType *>(transposed_output->data.dptr), \
          reinterpret_cast<const fp32 *>(cast_output->scale.dptr),      \
          reinterpret_cast<fp32 *>(cast_output->amax.dptr),             \
          row_length, num_rows, n_tiles);                               \
  } while (false)

// Launch cast-transpose kernel for given vector sizes
#define LAUNCH_KERNEL_VEC_SIZES(load_size, store_size, InputType, OutputType) \
  do {                                                                  \
    constexpr int nvec_in = load_size / sizeof(InputType);              \
    constexpr int nvec_out = store_size / sizeof(OutputType);           \
                                                                        \
    NVTE_CHECK(row_length % nvec_in  == 0, "Unsupported shape.");       \
    NVTE_CHECK(num_rows   % nvec_out == 0, "Unsupported shape.");       \
                                                                        \
    const size_t n_tiles = get_n_tiles(load_size, store_size);          \
    const size_t n_blocks = get_n_blocks(n_tiles);                      \
                                                                        \
    const bool full_tile = row_length % (nvec_in * THREADS_PER_WARP) == 0 && \
                           num_rows % (nvec_out * THREADS_PER_WARP) == 0; \
                                                                        \
    if (full_tile) {                                                    \
      LAUNCH_KERNEL(cast_transpose_kernel,                              \
                    nvec_in, nvec_out, n_tiles, n_blocks,               \
                    InputType, OutputType);                             \
    } else {                                                            \
      LAUNCH_KERNEL(cast_transpose_kernel_notaligned,                   \
                    nvec_in, nvec_out, n_tiles, n_blocks,               \
                    InputType, OutputType);                             \
    }                                                                   \
  } while (false)

  TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(input.data.dtype, InputType,
    TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(cast_output->data.dtype, OutputType,

      // Estimate number of SMs
      // Note: H100 has 132 SMs, A100 has 108 SMs.
      // Note: Directly querying number of SMs with hipGetDeviceProperties is
      // slow (>1 ms). Consider querying once and caching.
      const int n_sms = 128;

      // Helper functions to get kernel configuration
      auto get_n_tiles = [=] (size_t load_size, size_t store_size) -> int {
        constexpr size_t threads_per_warp = static_cast<size_t>(THREADS_PER_WARP);
        size_t nvec_in = load_size / sizeof(InputType);
        size_t nvec_out = store_size / sizeof(OutputType);
        size_t n_tiles = DIVUP(row_length, nvec_in * threads_per_warp) *
                         DIVUP(num_rows, nvec_out * threads_per_warp);
        return n_tiles;
      };
      auto get_n_blocks = [=] (size_t n_tiles) -> int {
        size_t n_warps_per_block = cast_transpose_num_threads / THREADS_PER_WARP;
        size_t n_blocks = DIVUP(n_tiles * n_warps_per_tile, n_warps_per_block);
        return n_blocks;
      };

      // Estimate optimal vector sizes and run
      // Note: Consider reducing to 2B or 1B loads/stores for
      // sufficiently small matrices. Need to consider whether reduced
      // cache efficiency is worth increased SM utilization. Also need
      // to keep in mind whether datatype can fit.
      const size_t estimated_n_tiles = get_n_tiles(8, 8);
      const size_t estimated_n_blocks = get_n_blocks(estimated_n_tiles);
      if (estimated_n_blocks >= n_sms) {
        LAUNCH_KERNEL_VEC_SIZES(8, 8, InputType, OutputType);
      } else {
        LAUNCH_KERNEL_VEC_SIZES(4, 4, InputType, OutputType);
      }

    );  // NOLINT(*)
  );  // NOLINT(*)

#undef LAUNCH_KERNEL
#undef LAUNCH_KERNEL_VEC_SIZES
}

}  // namespace transformer_engine

void nvte_cast_transpose(const NVTETensor input,
                         NVTETensor cast_output,
                         NVTETensor transposed_output,
                         hipStream_t stream) {
  NVTE_API_CALL(nvte_cast_transpose);
  using namespace transformer_engine;
  cast_transpose(*reinterpret_cast<const Tensor*>(input),
                 reinterpret_cast<Tensor*>(cast_output),
                 reinterpret_cast<Tensor*>(transposed_output),
                 stream);
}
