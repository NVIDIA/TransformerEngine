#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "utils.cuh"

using namespace transformer_engine;

namespace {

// Parameters
using CType = float;
using IType = __ITYPE__;
using OType = __OTYPE__;
constexpr size_t load_size = __LOAD_SIZE__;
constexpr size_t store_size = __STORE_SIZE__;
constexpr size_t warps_per_tile = __WARPS_PER_TILE__;
constexpr size_t block_size = __BLOCK_SIZE__;

}  // namespace

__global__ void
__launch_bounds__(block_size)
cast_transpose_optimized_kernel(const IType * __restrict__ const input,
                                const CType * __restrict__ const noop,
                                OType * __restrict__  const output_c,
                                OType * __restrict__  const output_t,
                                const CType * __restrict__ const scale_ptr,
                                CType * __restrict__ const amax_ptr,
                                const size_t row_length,
                                const size_t num_rows) {
  if (noop != nullptr && noop[0] == 1.0f) return;

  // Vectorized load/store sizes
  constexpr size_t nvec_in = load_size / sizeof(IType);
  constexpr size_t nvec_out = store_size / sizeof(OType);
  using IVec = Vec<IType, nvec_in>;
  using OVecC = Vec<OType, nvec_in>;
  using OVecT = Vec<OType, nvec_out>;

  // Thread indices
  // Note: Block is interpreted as a warp_size x num_warps grid
  constexpr size_t bdimx = THREADS_PER_WARP;
  constexpr size_t bdimy = warps_per_tile;
  const size_t tid = threadIdx.x;
  const size_t tidx = tid % bdimx;
  const size_t tidy = tid / bdimx;
  const size_t bid = blockIdx.x;

  // Input tensors are divided into tiles
  // Note: Each tile is a warp_size x warp_size grid of nvec_out x nvec_in subtiles
  constexpr size_t tile_dim_m = THREADS_PER_WARP * nvec_out;
  constexpr size_t tile_dim_n = THREADS_PER_WARP * nvec_in;

  // Position of tile within tensor
  const size_t num_tiles_m = num_rows / tile_dim_m;
  const size_t tile_id_m = bid % num_tiles_m;
  const size_t tile_id_n = bid / num_tiles_m;
  const size_t tile_row = tile_id_m * tile_dim_m;
  const size_t tile_col = tile_id_n * tile_dim_n;

  // Number of nvec_out x nvec_in subtiles for each thread to
  // load/store
  constexpr size_t num_iterations = THREADS_PER_WARP / warps_per_tile;

  // FP8 factors
  const CType scale = scale_ptr == nullptr ? 1 : *scale_ptr;
  CType amax = 0;

  // Load input to registers and transpose
  // Note: Each thread loads num_iterations subtiles, computes amax,
  // casts type, and transposes in registers.
  OVecT local_output_t[nvec_in][num_iterations];
  #pragma unroll
  for (size_t iter = 0; iter < num_iterations; ++iter) {
    const size_t i1 = tidy + iter * bdimy;
    const size_t j1 = tidx;
    #pragma unroll
    for (size_t i2 = 0; i2 < nvec_out; ++i2) {
      const size_t row = tile_row + i1 * nvec_out + i2;
      const size_t col = tile_col + j1 * nvec_in;
      IVec local_input;
      OVecC local_output_c;
      local_input.load_from(&input[row * row_length + col]);
      #pragma unroll
      for (size_t j2 = 0; j2 < nvec_in; ++j2) {
        const CType in = static_cast<CType>(local_input.data.elt[j2]);
        const OType out = OType(in * scale);
        __builtin_assume(amax >= 0);
        amax = fmaxf(fabsf(in), amax);
        local_output_c.data.elt[j2] = out;
        local_output_t[j2][iter].data.elt[i2] = out;
      }
      local_output_c.store_to(&output_c[row * row_length + col]);
    }
  }

  // Copy from registers to shared memory to global memory
  __shared__ OVecT shared_output_t[THREADS_PER_WARP][THREADS_PER_WARP+1];
  #pragma unroll
  for (size_t j2 = 0; j2 < nvec_in; ++j2) {
    #pragma unroll
    for (size_t iter = 0; iter < num_iterations; ++iter) {
      const size_t i1 = tidy + iter * bdimy;
      const size_t j1 = tidx;
      shared_output_t[j1][i1] = local_output_t[j2][iter];
    }
    __syncthreads();
    #pragma unroll
    for (size_t iter = 0; iter < num_iterations; ++iter) {
      const size_t i1 = tidx;
      const size_t j1 = tidy + iter * bdimy;
      const size_t row = tile_row + i1 * nvec_out;
      const size_t col = tile_col + j1 * nvec_in + j2;
      shared_output_t[j1][i1].store_to(&output_t[col * num_rows + row]);
    }
    __syncthreads();
  }

  // Reduce amax over block
  if (amax_ptr != nullptr) {
    amax = reduce_max<warps_per_tile>(amax, tidy);
    if (threadIdx.x == 0) {
      atomicMaxFloat(amax_ptr, amax);
    }
  }
}
