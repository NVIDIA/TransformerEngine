#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <transformer_engine/transpose.h>
#include <hip/hip_runtime.h>
#include <cfloat>
#include <iostream>
#include <type_traits>
#include "../utils.cuh"
#include "../common.h"
#include "../util/math.h"

namespace transformer_engine {

// STUFF TO TUNE
constexpr unsigned int n_warps_per_tile = 4;
constexpr unsigned int max_threads_per_block = 256;
static_assert(n_warps_per_tile * THREADS_PER_WARP <= max_threads_per_block);
constexpr unsigned int cast_transpose_num_threads = n_warps_per_tile * THREADS_PER_WARP;

template <bool full_tile, int nvec_in, int nvec_out, typename IVec, typename OVec, typename CType>
inline __device__ void cast_and_transpose_regs(const IVec (&in)[nvec_out],
                                               OVec (&out_trans)[nvec_in],
                                               typename OVec::type *output_cast_tile,
                                               const size_t current_place,
                                               const size_t stride,
                                               CType &max,  // NOLINT(*)
                                               const CType scale,
                                               const bool valid_store) {
    using T = typename OVec::type;
    using OVecC = Vec<T, nvec_in>;
    #pragma unroll
    for (unsigned int i = 0; i < nvec_out; ++i) {
        OVecC out_cast;
        #pragma unroll
        for (unsigned int j = 0; j < nvec_in; ++j) {
            const CType tmp = static_cast<CType>(in[i].data.elt[j]);
            const T elt_o = T(scale * tmp);

            out_cast.data.elt[j]     = elt_o;
            out_trans[j].data.elt[i] = elt_o;  // thread tile transpose

            __builtin_assume(max >= 0);
            max = fmaxf(fabsf(tmp), max);
        }
        if (full_tile || valid_store) {
            out_cast.store_to(output_cast_tile, current_place + stride * i);
        }
    }
}

template <int nvec_in, int nvec_out,
          typename CType, typename IType, typename OType, typename ParamOP,
          CType (*OP1)(CType, const ParamOP&), 
          CType (*OP2)(CType, const ParamOP&)>
__global__ void
__launch_bounds__(cast_transpose_num_threads)
dgated_act_cast_transpose_kernel(const IType * const input,
                                 const IType * const act_input,
                                 OType * const output_c,
                                 OType * const output_t,
                                 const CType * const scale_ptr,
                                 CType * const amax,
                                 CType * const scale_inv,
                                 const size_t row_length,
                                 const size_t num_rows,
                                 const size_t num_tiles) {
    using IVec = Vec<IType, nvec_in>;
    using OVec = Vec<OType, nvec_out>;
    using CVec = Vec<CType, nvec_in>;

    extern __shared__ char scratch[];

    const int warp_id = threadIdx.x / THREADS_PER_WARP;
    const int my_id_in_warp = threadIdx.x % THREADS_PER_WARP;
    const size_t num_tiles_x = row_length / (nvec_in * THREADS_PER_WARP);
    const size_t tile_id = blockIdx.x * blockDim.x / (THREADS_PER_WARP * n_warps_per_tile) +
                            warp_id / n_warps_per_tile;
    if (tile_id >= num_tiles) {
        return;
    }
  
    const size_t tile_id_x = tile_id % num_tiles_x;
    const size_t tile_id_y = tile_id / num_tiles_x;

    const IType * const my_input_tile = input + (tile_id_x * nvec_in +
                                                tile_id_y * row_length * nvec_out) *
                                                THREADS_PER_WARP;
    const IType * const my_act_input_tile = act_input + (tile_id_x * nvec_in +
                                                tile_id_y * row_length * 2 * nvec_out) *
                                                THREADS_PER_WARP;
    const IType * const my_gate_input_tile = act_input + (tile_id_x * nvec_in +
                                                tile_id_y * row_length * 2 * nvec_out) *
                                                THREADS_PER_WARP + row_length;
    OType * const my_output_c_tile_0 = output_c + (tile_id_x * nvec_in +
                                                tile_id_y * row_length * 2 * nvec_out) *
                                                THREADS_PER_WARP;
    OType * const my_output_c_tile_1 = output_c + (tile_id_x * nvec_in +
                                                tile_id_y * row_length * 2 * nvec_out) *
                                                THREADS_PER_WARP + row_length;
    OType * const my_output_t_tile_0 = output_t + (tile_id_y * nvec_out +
                                                tile_id_x * num_rows * nvec_in) *
                                                THREADS_PER_WARP;
    OType * const my_output_t_tile_1 = output_t + (tile_id_y * nvec_out +
                                                tile_id_x * num_rows * nvec_in) *
                                                THREADS_PER_WARP + row_length * num_rows;
    OVec * const my_scratch = reinterpret_cast<OVec*>(scratch) +
                                (my_id_in_warp + warp_id / n_warps_per_tile * THREADS_PER_WARP) *
                                (THREADS_PER_WARP + 1);

    IVec in[2][nvec_out];
    IVec act_in[2][nvec_out];
    IVec gate_in[2][nvec_out];
    const unsigned int warp_id_in_tile = warp_id % n_warps_per_tile;
    constexpr unsigned int n_iterations = THREADS_PER_WARP / n_warps_per_tile;
    OVec out_space_0[n_iterations][nvec_in];
    OVec out_space_1[n_iterations][nvec_in];

    const size_t stride = row_length / nvec_in;
    const size_t output_stride = num_rows / nvec_out;
    size_t current_stride = warp_id_in_tile * n_iterations * nvec_out * stride;
    unsigned int my_place = (my_id_in_warp + THREADS_PER_WARP -
                            warp_id_in_tile * n_iterations) %
                            THREADS_PER_WARP;
    const size_t stride2 = 2 * row_length / nvec_in;
    size_t current_stride2 = warp_id_in_tile * n_iterations * nvec_out * stride2;
    CType max = 0;
    const CType scale = scale_ptr != nullptr ? *scale_ptr : 1;
    #pragma unroll
    for (unsigned int i = 0; i < nvec_out; ++i) {
        in[0][i].load_from(my_input_tile, current_stride + my_place + stride * i);
        act_in[0][i].load_from(my_act_input_tile, current_stride2 + my_place + stride2 * i);
        gate_in[0][i].load_from(my_gate_input_tile, current_stride2 + my_place + stride2 * i);
    }
    #pragma unroll
    for (unsigned int i = 0; i < n_iterations; ++i) {
        const size_t current_place = current_stride2 + my_place;
        const unsigned int my_place_in = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
        const unsigned int current_in = (i + 1) % 2;
        if (i < n_iterations - 1) {
            #pragma unroll
            for (unsigned int j = 0; j < nvec_out; ++j) {
                in[current_in][j].load_from(my_input_tile,
                                            current_stride + my_place_in + stride * (nvec_out + j));
                act_in[current_in][j].load_from(my_act_input_tile,
                                            current_stride2 + my_place_in + stride2 * (nvec_out + j));
                gate_in[current_in][j].load_from(my_gate_input_tile,
                                            current_stride2 + my_place_in + stride2 * (nvec_out + j));
            }
        }
        CVec after_dact[nvec_out];  // NOLINT(*)
        CVec after_dgate[nvec_out];  // NOLINT(*)
        #pragma unroll
        for (unsigned int j = 0; j < nvec_out; ++j) {
            #pragma unroll
            for (unsigned int k = 0; k < nvec_in; ++k) {
                after_dact[j].data.elt[k] = OP1(act_in[current_in ^ 1][j].data.elt[k], {}) *
                                            CType(in[current_in ^ 1][j].data.elt[k]) *
                                            CType(gate_in[current_in ^ 1][j].data.elt[k]);
                after_dgate[j].data.elt[k] = CType(in[current_in ^ 1][j].data.elt[k]) *
                                            OP2(act_in[current_in ^ 1][j].data.elt[k], {});
            }
        }
        OVec out_trans_0[nvec_in];  // NOLINT(*)
        cast_and_transpose_regs<true>(after_dact, out_trans_0, my_output_c_tile_0,
                                    current_place, stride2, max, scale, true);
        OVec out_trans_1[nvec_in];  // NOLINT(*)
        cast_and_transpose_regs<true>(after_dgate, out_trans_1, my_output_c_tile_1,
                                    current_place, stride2, max, scale, true);
        #pragma unroll
        for (unsigned int j = 0; j < nvec_in; ++j) {
            out_space_0[i][j].data.vec = out_trans_0[j].data.vec;
            out_space_1[i][j].data.vec = out_trans_1[j].data.vec;
        }
        my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
        current_stride += nvec_out * stride;
        current_stride2 += nvec_out * stride2;
    }

    for (unsigned int i = 0; i < nvec_in; ++i) {
        #pragma unroll
        for (unsigned int j = 0; j < n_iterations; ++j) {
            my_scratch[(my_id_in_warp + THREADS_PER_WARP -
                    j - warp_id_in_tile * n_iterations) % THREADS_PER_WARP] = out_space_0[j][i];
        }
        __syncthreads();
        my_place = (my_id_in_warp + THREADS_PER_WARP - warp_id_in_tile * n_iterations) %
                THREADS_PER_WARP;
        current_stride = i * output_stride +
                        warp_id_in_tile * n_iterations * output_stride * nvec_in;
        for (unsigned int j = 0; j < n_iterations; ++j) {
            my_scratch[j + warp_id_in_tile * n_iterations].store_to(my_output_t_tile_0,
                                                                    current_stride + my_place);
            my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
            current_stride += output_stride * nvec_in;
        }
        __syncthreads();
        #pragma unroll
        for (unsigned int j = 0; j < n_iterations; ++j) {
            my_scratch[(my_id_in_warp + THREADS_PER_WARP -
                    j - warp_id_in_tile * n_iterations) % THREADS_PER_WARP] = out_space_1[j][i];
        }
        __syncthreads();
        my_place = (my_id_in_warp + THREADS_PER_WARP - warp_id_in_tile * n_iterations) %
                THREADS_PER_WARP;
        current_stride = i * output_stride +
                        warp_id_in_tile * n_iterations * output_stride * nvec_in;
        for (unsigned int j = 0; j < n_iterations; ++j) {
            my_scratch[j + warp_id_in_tile * n_iterations].store_to(my_output_t_tile_1,
                                                                    current_stride + my_place);
            my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
            current_stride += output_stride * nvec_in;
        }
        __syncthreads();
    }

    /* warp tile amax reduce*/
    max = reduce_max<cast_transpose_num_threads / THREADS_PER_WARP>(max, warp_id);

    if (threadIdx.x == 0) {
        static_assert(std::is_same<CType, float>::value);
        if (amax != nullptr) {
            atomicMaxFloat(amax, max);
        }
        if (scale_inv != nullptr) {
            reciprocal<float>(scale_inv, scale);
        }
    }
}

template <int nvec_in, int nvec_out,
         typename CType, typename IType, typename OType,
         typename ParamOP,
         CType (*OP1)(CType, const ParamOP&),
         CType (*OP2)(CType, const ParamOP&)>
__global__ void
__launch_bounds__(cast_transpose_num_threads)
dgated_act_cast_transpose_kernel_notaligned(const IType * const input,
                                        const IType * const act_input,
                                        OType * const output_c,
                                        OType * const output_t,
                                        const CType * const scale_ptr,
                                        CType * const amax,
                                        CType * const scale_inv,
                                        const size_t row_length,
                                        const size_t num_rows,
                                        const size_t num_tiles) {
    using IVec = Vec<IType, nvec_in>;
    using OVec = Vec<OType, nvec_out>;
    using CVec = Vec<CType, nvec_in>;

    extern __shared__ char scratch[];

    const int warp_id = threadIdx.x / THREADS_PER_WARP;
    const int my_id_in_warp = threadIdx.x % THREADS_PER_WARP;
    const size_t num_tiles_x = (row_length + nvec_in * THREADS_PER_WARP - 1) /
                                (nvec_in * THREADS_PER_WARP);
    const size_t tile_id = blockIdx.x * blockDim.x / (THREADS_PER_WARP * n_warps_per_tile) +
                            warp_id / n_warps_per_tile;
    if (tile_id >= num_tiles) return;
    const size_t tile_id_x = tile_id % num_tiles_x;
    const size_t tile_id_y = tile_id / num_tiles_x;

    const IType * const my_input_tile = input + (tile_id_x * nvec_in +
                                                tile_id_y * row_length * nvec_out) *
                                                THREADS_PER_WARP;
    const IType * const my_act_input_tile = act_input + (tile_id_x * nvec_in +
                                                tile_id_y * row_length * 2 * nvec_out) *
                                                THREADS_PER_WARP;
    const IType * const my_gate_input_tile = act_input + (tile_id_x * nvec_in +
                                                tile_id_y * row_length * 2 * nvec_out) *
                                                THREADS_PER_WARP + row_length;
    OType * const my_output_c_tile_0 = output_c + (tile_id_x * nvec_in +
                                                tile_id_y * row_length * 2 * nvec_out) *
                                                THREADS_PER_WARP;
    OType * const my_output_c_tile_1 = output_c + (tile_id_x * nvec_in +
                                                tile_id_y * row_length * 2 * nvec_out) *
                                                THREADS_PER_WARP + row_length;
    OType * const my_output_t_tile_0 = output_t + (tile_id_y * nvec_out +
                                                tile_id_x * num_rows * nvec_in) *
                                                THREADS_PER_WARP;
    OType * const my_output_t_tile_1 = output_t + (tile_id_y * nvec_out +
                                                tile_id_x * num_rows * nvec_in) *
                                                THREADS_PER_WARP + row_length * num_rows;
    const size_t stride = row_length / nvec_in;
    const size_t stride2 = 2 * row_length / nvec_in;
    const size_t output_stride = num_rows / nvec_out;
    const size_t row_length_rest = stride - tile_id_x * THREADS_PER_WARP;
    const size_t row_height_rest = output_stride - tile_id_y * THREADS_PER_WARP;
    const unsigned int tile_length = row_length_rest > THREADS_PER_WARP ? THREADS_PER_WARP
                                                                        : row_length_rest;
    const unsigned int tile_height = row_height_rest > THREADS_PER_WARP ? THREADS_PER_WARP
                                                                        : row_height_rest;

    OVec * const my_scratch = reinterpret_cast<OVec*>(scratch) +
                                (my_id_in_warp + warp_id / n_warps_per_tile * THREADS_PER_WARP) *
                                (THREADS_PER_WARP + 1);

    IVec in[2][nvec_out];
    IVec act_in[2][nvec_out];
    IVec gate_in[2][nvec_out];
    const unsigned int warp_id_in_tile = warp_id % n_warps_per_tile;
    constexpr unsigned int n_iterations = THREADS_PER_WARP / n_warps_per_tile;
    OVec out_space_0[n_iterations][nvec_in];
    OVec out_space_1[n_iterations][nvec_in];

    size_t current_stride = warp_id_in_tile * n_iterations * nvec_out * stride;
    size_t current_stride2 = warp_id_in_tile * n_iterations * nvec_out * stride2;
    unsigned int my_place = (my_id_in_warp + THREADS_PER_WARP -
                            warp_id_in_tile * n_iterations) %
                            THREADS_PER_WARP;
    CType max = 0;
    const CType scale = scale_ptr != nullptr ? *scale_ptr : 1;
    {
        const bool valid_load = my_place < tile_length &&
                                warp_id_in_tile * n_iterations < tile_height;
    #pragma unroll
        for (unsigned int i = 0; i < nvec_out; ++i) {
            if (valid_load) {
                in[0][i].load_from(my_input_tile, current_stride + my_place + stride * i);
                act_in[0][i].load_from(my_act_input_tile, current_stride2 + my_place + stride2 * i);
                gate_in[0][i].load_from(my_gate_input_tile, current_stride2 + my_place + stride2 * i);
            } else {
                in[0][i].clear();
                act_in[0][i].clear();
                gate_in[0][i].clear();
            }
        }
    }
    #pragma unroll
    for (unsigned int i = 0; i < n_iterations; ++i) {
        const size_t current_place = current_stride2 + my_place;
        const unsigned int my_place_in = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
        const unsigned int current_in = (i + 1) % 2;
        if (i < n_iterations - 1) {
            {
                const bool valid_load = my_place_in < tile_length &&
                                        warp_id_in_tile * n_iterations + i + 1 < tile_height;
                #pragma unroll
                for (unsigned int j = 0; j < nvec_out; ++j) {
                    if (valid_load) {
                        in[current_in][j].load_from(my_input_tile,
                                                    current_stride + my_place_in + stride * (nvec_out + j));
                        act_in[current_in][j].load_from(my_act_input_tile,
                                                    current_stride2 + my_place_in + stride2 * (nvec_out + j));
                        gate_in[current_in][j].load_from(my_gate_input_tile,
                                                    current_stride2 + my_place_in + stride2 * (nvec_out + j));
                    } else {
                        in[current_in][j].clear();
                        act_in[current_in][j].clear();
                        gate_in[current_in][j].clear();
                    }
                }
            }
        }
        CVec after_dact[nvec_out];  // NOLINT(*)
        CVec after_dgate[nvec_out];  // NOLINT(*)
        #pragma unroll
        for (unsigned int j = 0; j < nvec_out; ++j) {
            #pragma unroll
            for (unsigned int k = 0; k < nvec_in; ++k) {
                after_dact[j].data.elt[k] = OP1(act_in[current_in ^ 1][j].data.elt[k], {}) *
                                            CType(in[current_in ^ 1][j].data.elt[k]) *
                                            CType(gate_in[current_in ^ 1][j].data.elt[k]);
                after_dgate[j].data.elt[k] = CType(in[current_in ^ 1][j].data.elt[k]) *
                                            OP2(act_in[current_in ^ 1][j].data.elt[k], {});
            }
        }
        OVec out_trans_0[nvec_in];  // NOLINT(*)
        OVec out_trans_1[nvec_in];  // NOLINT(*)
        const bool valid_store = my_place < tile_length &&
                                warp_id_in_tile * n_iterations + i < tile_height;
        cast_and_transpose_regs<false>(after_dact, out_trans_0, my_output_c_tile_0,
                                    current_place, stride2, max, scale, valid_store);
        cast_and_transpose_regs<false>(after_dgate, out_trans_1, my_output_c_tile_1,
                                    current_place, stride2, max, scale, valid_store);
        #pragma unroll
        for (unsigned int j = 0; j < nvec_in; ++j) {
            out_space_0[i][j].data.vec = out_trans_0[j].data.vec;
            out_space_1[i][j].data.vec = out_trans_1[j].data.vec;
        }
        my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
        current_stride += nvec_out * stride;
        current_stride2 += nvec_out * stride2;
    }

    for (unsigned int i = 0; i < nvec_in; ++i) {
        #pragma unroll
        for (unsigned int j = 0; j < n_iterations; ++j) {
            my_scratch[(my_id_in_warp + THREADS_PER_WARP -
                        j - warp_id_in_tile * n_iterations) % THREADS_PER_WARP] = out_space_0[j][i];
        }
        __syncthreads();
        my_place = (my_id_in_warp + THREADS_PER_WARP - warp_id_in_tile * n_iterations) %
                THREADS_PER_WARP;
        current_stride = i * output_stride +
                        warp_id_in_tile * n_iterations * output_stride * nvec_in;
        for (unsigned int j = 0; warp_id_in_tile * n_iterations + j < tile_length; ++j) {
            const bool valid_store = my_place < tile_height;
            if (valid_store) {
                my_scratch[j + warp_id_in_tile * n_iterations].store_to(my_output_t_tile_0,
                                                                        current_stride + my_place);
            }
            my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
            current_stride += output_stride * nvec_in;
        }
        __syncthreads();
        #pragma unroll
        for (unsigned int j = 0; j < n_iterations; ++j) {
            my_scratch[(my_id_in_warp + THREADS_PER_WARP -
                        j - warp_id_in_tile * n_iterations) % THREADS_PER_WARP] = out_space_1[j][i];
        }
        __syncthreads();
        my_place = (my_id_in_warp + THREADS_PER_WARP - warp_id_in_tile * n_iterations) %
                THREADS_PER_WARP;
        current_stride = i * output_stride +
                        warp_id_in_tile * n_iterations * output_stride * nvec_in;
        for (unsigned int j = 0; warp_id_in_tile * n_iterations + j < tile_length; ++j) {
            const bool valid_store = my_place < tile_height;
            if (valid_store) {
                my_scratch[j + warp_id_in_tile * n_iterations].store_to(my_output_t_tile_1,
                                                                        current_stride + my_place);
            }
            my_place = (my_place + THREADS_PER_WARP - 1) % THREADS_PER_WARP;
            current_stride += output_stride * nvec_in;
        }
        __syncthreads();
    }

    /* warp tile amax reduce*/
    max = reduce_max<cast_transpose_num_threads / THREADS_PER_WARP>(max, warp_id);

    if (threadIdx.x == 0) {
        static_assert(std::is_same<CType, float>::value);
        if (amax != nullptr)  {
            atomicMaxFloat(amax, max);
        }
        if (scale_inv != nullptr) {
            reciprocal<float>(scale_inv, scale);
        }
    }
}

template <typename ComputeType, typename ParamOP,
         ComputeType (*OP1)(ComputeType, const ParamOP&),
         ComputeType (*OP2)(ComputeType, const ParamOP&)>
void dgated_act_cast_transpose(const Tensor &input,
                           const Tensor &gated_act_input,
                           Tensor *cast_output,
                           Tensor *transposed_output,
                           hipStream_t stream) {
    CheckInputTensor(input, "dgated_act_cast_transpose_input");
    CheckInputTensor(gated_act_input, "dgated_act_cast_transpose_gated_act_input");
    CheckOutputTensor(*cast_output, "dgated_act_cast_transpose_cast_output");
    CheckOutputTensor(*transposed_output, "dgated_act_cast_transpose_transposed_output");

    NVTE_CHECK(input.data.shape.size() == 2, "Input must have 2 dimensions.");
    NVTE_CHECK(gated_act_input.data.shape.size() == 2, "Input must have 2 dimensions.");
    NVTE_CHECK(cast_output->data.shape.size() == 2, "C output must have 2 dimensions.");
    NVTE_CHECK(transposed_output->data.shape.size() == 2, "T output must have 2 dimensions.");
    const size_t row_length = input.data.shape[1];
    const size_t num_rows = input.data.shape[0];

    NVTE_CHECK(gated_act_input.data.shape[0] == num_rows, "Wrong dimension of output.");
    NVTE_CHECK(gated_act_input.data.shape[1] == row_length * 2, "Wrong dimension of output.");
    NVTE_CHECK(cast_output->data.shape[0] == num_rows, "Wrong dimension of output.");
    NVTE_CHECK(cast_output->data.shape[1] == row_length * 2, "Wrong dimension of output.");
    NVTE_CHECK(transposed_output->data.shape[0] == row_length * 2, "Wrong dimension of T output.");
    NVTE_CHECK(transposed_output->data.shape[1] == num_rows, "Wrong dimension of T output.");

    NVTE_CHECK(input.data.dtype == gated_act_input.data.dtype, "Types of both inputs must match.");

    NVTE_CHECK(cast_output->data.dtype == transposed_output->data.dtype, "C and T outputs need to have the same type.");
    NVTE_CHECK(cast_output->amax.dptr == transposed_output->amax.dptr, "C and T outputs need to share amax tensor.");
    NVTE_CHECK(cast_output->scale.dptr == transposed_output->scale.dptr, "C and T outputs need to share scale tensor.");
    NVTE_CHECK(cast_output->scale_inv.dptr == transposed_output->scale_inv.dptr, "C and T outputs need to share scale inverse tensor.");

    TRANSFORMER_ENGINE_TYPE_SWITCH_INPUT(input.data.dtype, InputType,
        TRANSFORMER_ENGINE_TYPE_SWITCH_OUTPUT(cast_output->data.dtype, OutputType,
            using InputType2 = InputType;
            /* dact fusion kernel uses more registers */
            constexpr int desired_load_size_dact = 4;
            constexpr int desired_store_size_dact = 4;
            constexpr int itype_size = sizeof(InputType);
            constexpr int otype_size = sizeof(OutputType);
            constexpr int nvec_in = desired_load_size_dact / itype_size;
            constexpr int nvec_out = desired_store_size_dact / otype_size;

            NVTE_CHECK(row_length % nvec_in  == 0, "Unsupported shape.");
            NVTE_CHECK(num_rows   % nvec_out == 0, "Unsupported shape.");
            const size_t n_tiles = DIVUP(row_length, static_cast<size_t>(nvec_in * THREADS_PER_WARP)) *
                                    DIVUP(num_rows, static_cast<size_t>(nvec_out * THREADS_PER_WARP));
            const size_t n_warps_per_block = cast_transpose_num_threads / THREADS_PER_WARP;
            const size_t n_blocks = DIVUP(n_tiles * n_warps_per_tile, n_warps_per_block);

            const bool full_tile = row_length % (nvec_in * THREADS_PER_WARP) == 0 &&
                                    num_rows % (nvec_out * THREADS_PER_WARP) == 0;
            const size_t shmem_size = cast_transpose_num_threads / n_warps_per_tile * 
                                       (THREADS_PER_WARP + 1) * sizeof(Vec<OutputType, nvec_out>);
            if (full_tile) {
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    dgated_act_cast_transpose_kernel
                        <nvec_in), nvec_out, ComputeType, InputType, OutputType, Empty, OP1, OP2>,
                    hipFuncAttributePreferredSharedMemoryCarveout,
                    100);

                dgated_act_cast_transpose_kernel
                    <nvec_in, nvec_out, ComputeType, InputType, OutputType, Empty, OP1, OP2>
                    <<<n_blocks, cast_transpose_num_threads, shmem_size, stream>>>(
                        reinterpret_cast<const InputType *>(input.data.dptr),
                        reinterpret_cast<const InputType *>(gated_act_input.data.dptr),
                        reinterpret_cast<OutputType *>(cast_output->data.dptr),
                        reinterpret_cast<OutputType *>(transposed_output->data.dptr),
                        reinterpret_cast<const fp32 *>(cast_output->scale.dptr),
                        reinterpret_cast<fp32 *>(cast_output->amax.dptr),
                        reinterpret_cast<fp32 *>(cast_output->scale_inv.dptr),
                        row_length, num_rows, n_tiles);
            } else {
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    dgated_act_cast_transpose_kernel_notaligned
                        <nvec_in), nvec_out, ComputeType, InputType, OutputType, Empty, OP1, OP2>,
                    hipFuncAttributePreferredSharedMemoryCarveout,
                    100);
                dgated_act_cast_transpose_kernel_notaligned
                    <nvec_in, nvec_out, ComputeType, InputType, OutputType, Empty, OP1, OP2>
                    <<<n_blocks, cast_transpose_num_threads, shmem_size, stream>>>(
                        reinterpret_cast<const InputType *>(input.data.dptr),
                        reinterpret_cast<const InputType *>(gated_act_input.data.dptr),
                        reinterpret_cast<OutputType *>(cast_output->data.dptr),
                        reinterpret_cast<OutputType *>(transposed_output->data.dptr),
                        reinterpret_cast<const fp32 *>(cast_output->scale.dptr),
                        reinterpret_cast<fp32 *>(cast_output->amax.dptr),
                        reinterpret_cast<fp32 *>(cast_output->scale_inv.dptr),
                        row_length, num_rows, n_tiles);
            }
        ); // NOLINT(*)
    );  // NOLINT(*)
}

}  // namespace transformer_engine

void nvte_dgeglu_cast_transpose(const NVTETensor input,
                                const NVTETensor gated_act_input,
                                NVTETensor cast_output,
                                NVTETensor transposed_output,
                                hipStream_t stream) {
    NVTE_API_CALL(nvte_dgeglu_cast_transpose);
    using namespace transformer_engine;

    constexpr auto dActivation = &dgelu<fp32, fp32>;
    constexpr auto  Activation =  &gelu<fp32, fp32>;

    dgated_act_cast_transpose<fp32, Empty, dActivation, Activation>(
        *reinterpret_cast<const Tensor*>(input),
        *reinterpret_cast<const Tensor*>(gated_act_input),
        reinterpret_cast<Tensor*>(cast_output),
        reinterpret_cast<Tensor*>(transposed_output),
        stream);
}

void nvte_dswiglu_cast_transpose(const NVTETensor input,
                                 const NVTETensor swiglu_input,
                                 NVTETensor cast_output,
                                 NVTETensor transposed_output,
                                 hipStream_t stream) {
    NVTE_API_CALL(nvte_dswiglu_cast_transpose);
    using namespace transformer_engine;

    constexpr auto dActivation = &dsilu<fp32, fp32>;
    constexpr auto  Activation =  &silu<fp32, fp32>;

    dgated_act_cast_transpose<fp32, Empty, dActivation, Activation>(
        *reinterpret_cast<const Tensor*>(input),
        *reinterpret_cast<const Tensor*>(swiglu_input),
        reinterpret_cast<Tensor*>(cast_output),
        reinterpret_cast<Tensor*>(transposed_output),
        stream);
}

void nvte_dreglu_cast_transpose(const NVTETensor input,
                                const NVTETensor gated_act_input,
                                NVTETensor cast_output,
                                NVTETensor transposed_output,
                                hipStream_t stream) {
    NVTE_API_CALL(nvte_dreglu_cast_transpose);
    using namespace transformer_engine;

    constexpr auto dActivation = &drelu<fp32, fp32>;
    constexpr auto  Activation =  &relu<fp32, fp32>;

    dgated_act_cast_transpose<fp32, Empty, dActivation, Activation>(
        *reinterpret_cast<const Tensor*>(input),
        *reinterpret_cast<const Tensor*>(gated_act_input),
        reinterpret_cast<Tensor*>(cast_output),
        reinterpret_cast<Tensor*>(transposed_output),
        stream);
}

void nvte_dsreglu_cast_transpose(const NVTETensor input,
                                 const NVTETensor gated_act_input,
                                 NVTETensor cast_output,
                                 NVTETensor transposed_output,
                                 hipStream_t stream) {
    NVTE_API_CALL(nvte_dsreglu_cast_transpose);
    using namespace transformer_engine;

    constexpr auto dActivation = &dsrelu<fp32, fp32>;
    constexpr auto  Activation =  &srelu<fp32, fp32>;

    dgated_act_cast_transpose<fp32, Empty, dActivation, Activation>(
        *reinterpret_cast<const Tensor*>(input),
        *reinterpret_cast<const Tensor*>(gated_act_input),
        reinterpret_cast<Tensor*>(cast_output),
        reinterpret_cast<Tensor*>(transposed_output),
        stream);
}

void nvte_dqgeglu_cast_transpose(const NVTETensor input,
                                 const NVTETensor gated_act_input,
                                 NVTETensor cast_output,
                                 NVTETensor transposed_output,
                                 hipStream_t stream) {
    NVTE_API_CALL(nvte_dqgeglu_cast_transpose);
    using namespace transformer_engine;

    constexpr auto dActivation = &dqgelu<fp32, fp32>;
    constexpr auto  Activation =  &qgelu<fp32, fp32>;

    dgated_act_cast_transpose<fp32, Empty, dActivation, Activation>(
        *reinterpret_cast<const Tensor*>(input),
        *reinterpret_cast<const Tensor*>(gated_act_input),
        reinterpret_cast<Tensor*>(cast_output),
        reinterpret_cast<Tensor*>(transposed_output),
        stream);
}
