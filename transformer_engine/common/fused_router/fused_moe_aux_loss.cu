#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <transformer_engine/fused_router.h>

#include "../common.h"
#include "../util/logging.h"
#include "../utils.cuh"
#include "common/util/hip/hip_runtime.h"
#include "utils.h"

namespace transformer_engine {

// Using Double to hanld all the calculations
using CompType = double;

template <typename DataType, typename IndexType>
__global__ void fused_moe_aux_loss_forward_kernel(const DataType* probs,
                                                  const IndexType* tokens_per_expert,
                                                  int total_num_tokens, int num_experts,
                                                  int num_rows, int num_cols, int topk, float coeff,
                                                  DataType* aux_loss, float* Const_buf) {
#if __CUDA_ARCH__ >= 900
  // Using cooperative_groups to manage the cluster
  namespace cg = cooperative_groups;
  cg::cluster_group cluster = cg::this_cluster();
  int thread_id = cg::this_grid().thread_rank();
  int lane_id = thread_id % kThreadsPerWarp;
  int warp_id = thread_id / kThreadsPerWarp;
  int warp_num = blockDim.x * gridDim.x / kThreadsPerWarp;
  // Only 1 block in the cluster
  int block_id = cluster.block_rank();
  int block_num = cluster.dim_blocks().x;
  int cluster_id = blockIdx.x / block_num;
  if (cluster_id > 0) return;  // Only use the cluster 0

  extern __shared__ float shmem_aux_loss[];
  CompType* aggregated_probs_per_expert = reinterpret_cast<CompType*>(shmem_aux_loss);
  // Clear the shmem
  for (int i = threadIdx.x; i < num_cols; i += blockDim.x) {
    aggregated_probs_per_expert[i] = CompType(0);
  }
  __syncthreads();

  /**
     * Section: Reduce the probs to the aggregated_probs_per_expert
     * 1. reduce on the block
     * 2. reduce on the cluster
     */
  // Loop: for all positions in each row
  for (int i = lane_id; i < num_cols; i += kThreadsPerWarp) {
    CompType tmp = CompType(0);
    // Loop: for all rows that this warp is responsible for
    for (int j = warp_id; j < num_rows; j += warp_num) {
      tmp += CompType(probs[j * num_cols + i]);
    }
    atomicAdd(&aggregated_probs_per_expert[i], tmp);
  }
  cluster.sync();
  // The block 0 will reduce the results of all blocks
  if (block_id == 0) {
    for (int i = 1; i < block_num; i++) {
      // Map the shared memory of the block i to the current block
      CompType* dst_smem = reinterpret_cast<CompType*>(cluster.map_shared_rank(shmem_aux_loss, i));
      for (int j = threadIdx.x; j < num_cols; j += blockDim.x) {
        atomicAdd(&aggregated_probs_per_expert[j], dst_smem[j]);
      }
    }
  }
  cluster.sync();

  /**
     * Section: aggregated_probs_per_expert * tokens_per_expert
     * In-place update on shmem
     */
  if (block_id == 0) {
    for (int i = threadIdx.x; i < num_cols; i += blockDim.x) {
      aggregated_probs_per_expert[i] *= CompType(tokens_per_expert[i]);
    }
    __syncthreads();

    if (warp_id == 0) {
      /**
             * Section: Reduce to get the sum of aggregated_probs_per_expert
             */
      CompType intermediate_result =
          warp_reduce_on_shmem(aggregated_probs_per_expert, num_cols, ReduceFuncType::SUM, lane_id);
      __syncwarp();

      if (lane_id == 0) {
        /**
                    * Section: Compute the aux_loss
                    */
        float C_coeff = (num_experts * coeff) / topk / total_num_tokens / total_num_tokens;
        aux_loss[0] = static_cast<DataType>(static_cast<double>(intermediate_result) * C_coeff);
        Const_buf[0] = C_coeff;
      }
    }
  }
#else
  // Use Only 1 block/1024 threads to avoid the grid sync
  if (blockIdx.x > 0) return;
  int warp_num = blockDim.x / kThreadsPerWarp;
  int warp_id = threadIdx.x / kThreadsPerWarp;
  int lane_id = threadIdx.x % kThreadsPerWarp;
  extern __shared__ float shmem_aux_loss[];
  CompType* aggregated_probs_per_expert = reinterpret_cast<CompType*>(shmem_aux_loss);

  // Clear the shmem
  for (int i = threadIdx.x; i < num_cols; i += blockDim.x) {
    aggregated_probs_per_expert[i] = CompType(0);
  }
  __syncthreads();

  /**
     * Section: Reduce the probs to the aggregated_probs_per_expert
     */
  // Loop: for all positions in each row
  for (int i = lane_id; i < num_cols; i += kThreadsPerWarp) {
    CompType tmp = CompType(0);
    // Loop: for all rows that this warp is responsible for
    for (int j = warp_id; j < num_rows; j += warp_num) {
      tmp += CompType(probs[j * num_cols + i]);
    }
    atomicAdd(&aggregated_probs_per_expert[i], tmp);
  }
  __syncthreads();

  /**
     * Section: aggregated_probs_per_expert * tokens_per_expert
     * In-place update on shmem
     */
  for (int i = threadIdx.x; i < num_cols; i += blockDim.x) {
    aggregated_probs_per_expert[i] *= CompType(tokens_per_expert[i]);
  }
  __syncthreads();

  if (warp_id == 0) {
    /**
         * Section: Reduce to get the sum of aggregated_probs_per_expert
         */
    CompType intermediate_result =
        warp_reduce_on_shmem(aggregated_probs_per_expert, num_cols, ReduceFuncType::SUM, lane_id);
    __syncwarp();

    if (lane_id == 0) {
      /**
             * Section: Compute the aux_loss
             */
      float C_coeff = (num_experts * coeff) / topk / total_num_tokens / total_num_tokens;
      aux_loss[0] = static_cast<DataType>(static_cast<double>(intermediate_result) * C_coeff);
      Const_buf[0] = C_coeff;
    }
  }
#endif
}

template <typename DataType, typename IndexType>
void fused_moe_aux_loss_forward_kernel_launcher(const DataType* probs,
                                                const IndexType* tokens_per_expert,
                                                int total_num_tokens, int num_experts, int num_rows,
                                                int num_cols, int topk, float coeff,
                                                DataType* aux_loss, float* Const_buf,
                                                hipStream_t stream) {
  if (cuda::sm_arch(cuda::current_device()) >= 90) {
    hipLaunchConfig_t config = {0};
    int cluster_size = 8;
    config.gridDim = cluster_size;
    config.blockDim = 1024;
    config.dynamicSmemBytes = sizeof(CompType) * num_cols;
    config.stream = stream;

    // Update the max cluster size based on the device
    NVTE_CHECK_CUDA(cudaOccupancyMaxPotentialClusterSize(
        &cluster_size,
        reinterpret_cast<void*>(fused_moe_aux_loss_forward_kernel<DataType, IndexType>), &config));

    hipLaunchAttribute attribute[1];
    attribute[0].id = cudaLaunchAttributeClusterDimension;
    attribute[0].val.clusterDim.x = cluster_size;
    attribute[0].val.clusterDim.y = 1;
    attribute[0].val.clusterDim.z = 1;
    config.numAttrs = 1;
    config.attrs = attribute;

    NVTE_CHECK_CUDA(cudaLaunchKernelEx(
        &config, fused_moe_aux_loss_forward_kernel<DataType, IndexType>, probs, tokens_per_expert,
        total_num_tokens, num_experts, num_rows, num_cols, topk, coeff, aux_loss, Const_buf));
  } else {
    size_t smem_size = sizeof(CompType) * num_cols;
    fused_moe_aux_loss_forward_kernel<DataType, IndexType>
        <<<1, 1024, smem_size, stream>>>(probs, tokens_per_expert, total_num_tokens, num_experts,
                                         num_rows, num_cols, topk, coeff, aux_loss, Const_buf);
    NVTE_CHECK_CUDA(hipGetLastError());
  }
}

void fused_moe_aux_loss_forward(const Tensor& probs, const Tensor& tokens_per_expert,
                                int total_num_tokens, int num_experts, int num_rows, int num_cols,
                                int topk, float coeff, Tensor& aux_loss, Tensor& Const_buf,
                                hipStream_t stream) {
  TE_ROUTER_PROBS_TYPE_SWITCH_ALL(
      probs.data.dtype, DataType,
      TE_ROUTER_INDEX_TYPE_SWITCH_ALL(
          tokens_per_expert.data.dtype, IndexType,
          fused_moe_aux_loss_forward_kernel_launcher<DataType, IndexType>(
              reinterpret_cast<DataType*>(probs.data.dptr),
              reinterpret_cast<IndexType*>(tokens_per_expert.data.dptr), total_num_tokens,
              num_experts, num_rows, num_cols, topk, coeff,
              reinterpret_cast<DataType*>(aux_loss.data.dptr),
              reinterpret_cast<float*>(Const_buf.data.dptr), stream);););
}

template <typename DataType, typename IndexType>
__global__ void fused_moe_aux_loss_backward_kernel(const float* Const_buf,
                                                   const IndexType* tokens_per_expert, int num_rows,
                                                   int num_cols, DataType* grad_aux_loss,
                                                   DataType* grad_probs) {
  int global_warp_num = gridDim.x * blockDim.x / kThreadsPerWarp;
  int global_warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / kThreadsPerWarp;
  int lane_id = threadIdx.x % kThreadsPerWarp;

  // Loop: for all positions in each row
  for (int i = lane_id; i < num_cols; i += kThreadsPerWarp) {
    float C_coeff = Const_buf[0];
    double tokens_per_expert_i = static_cast<double>(tokens_per_expert[i]);
    double grad_aux_loss_value = static_cast<double>(grad_aux_loss[0]);
    // Loop: for all rows
    for (int j = global_warp_id; j < num_rows; j += global_warp_num) {
      grad_probs[j * num_cols + i] = C_coeff * tokens_per_expert_i * grad_aux_loss_value;
    }
  }
}

template <typename DataType, typename IndexType>
void fused_moe_aux_loss_backward_kernel_launcher(const float* Const_buf,
                                                 const IndexType* tokens_per_expert, int num_rows,
                                                 int num_cols, DataType* grad_aux_loss,
                                                 DataType* grad_probs, hipStream_t stream) {
  // Meta data for the kernel
  int block_size = 256;
  int grid_size = (num_rows + block_size - 1) / block_size;
  fused_moe_aux_loss_backward_kernel<DataType, IndexType><<<grid_size, block_size, 0, stream>>>(
      Const_buf, tokens_per_expert, num_rows, num_cols, grad_aux_loss, grad_probs);
  NVTE_CHECK_CUDA(hipGetLastError());
}

void fused_moe_aux_loss_backward(const Tensor& Const_buf, const Tensor& tokens_per_expert,
                                 int num_rows, int num_cols, Tensor& grad_aux_loss,
                                 Tensor& grad_probs, hipStream_t stream) {
  TE_ROUTER_PROBS_TYPE_SWITCH_ALL(
      grad_aux_loss.data.dtype, DataType,
      TE_ROUTER_INDEX_TYPE_SWITCH_ALL(
          tokens_per_expert.data.dtype, IndexType,
          fused_moe_aux_loss_backward_kernel_launcher<DataType, IndexType>(
              reinterpret_cast<float*>(Const_buf.data.dptr),
              reinterpret_cast<IndexType*>(tokens_per_expert.data.dptr), num_rows, num_cols,
              reinterpret_cast<DataType*>(grad_aux_loss.data.dptr),
              reinterpret_cast<DataType*>(grad_probs.data.dptr), stream);););
}

}  // namespace transformer_engine

void nvte_fused_moe_aux_loss_forward(const NVTETensor probs, const NVTETensor tokens_per_expert,
                                     int total_num_tokens, int num_experts, int num_rows,
                                     int num_cols, int topk, float coeff, NVTETensor aux_loss,
                                     NVTETensor Const_buf, hipStream_t stream) {
  NVTE_API_CALL(nvte_fused_moe_aux_loss_forward);
  using namespace transformer_engine;
  fused_moe_aux_loss_forward(
      *convertNVTETensorCheck(probs), *convertNVTETensorCheck(tokens_per_expert), total_num_tokens,
      num_experts, num_rows, num_cols, topk, coeff, *convertNVTETensorCheck(aux_loss),
      *convertNVTETensorCheck(Const_buf), stream);
}

void nvte_fused_moe_aux_loss_backward(const NVTETensor Const_buf,
                                      const NVTETensor tokens_per_expert, int num_rows,
                                      int num_cols, NVTETensor grad_aux_loss, NVTETensor grad_probs,
                                      hipStream_t stream) {
  NVTE_API_CALL(nvte_fused_moe_aux_loss_backward);
  using namespace transformer_engine;
  fused_moe_aux_loss_backward(*convertNVTETensorCheck(Const_buf),
                              *convertNVTETensorCheck(tokens_per_expert), num_rows, num_cols,
                              *convertNVTETensorCheck(grad_aux_loss),
                              *convertNVTETensorCheck(grad_probs), stream);
}
