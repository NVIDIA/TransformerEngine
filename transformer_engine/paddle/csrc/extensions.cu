/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "common.h"

namespace transformer_engine {
namespace paddle_ext {

size_t get_cublasLt_version() { return cublasLtGetVersion(); }

PYBIND11_MODULE(transformer_engine_paddle, m) {
    // Misc
    m.def("get_cublasLt_version", &get_cublasLt_version, "Get cublasLt version");
    m.def("get_fused_attn_backend", &get_fused_attn_backend, "Get Fused Attention backend");
    m.def("get_nvte_qkv_layout", &get_nvte_qkv_layout, "Get qkv layout enum by the string");
    // Data structures
    py::enum_<DType>(m, "DType", py::module_local())
        .value("kByte", DType::kByte)
        .value("kInt32", DType::kInt32)
        .value("kFloat32", DType::kFloat32)
        .value("kFloat16", DType::kFloat16)
        .value("kBFloat16", DType::kBFloat16)
        .value("kFloat8E4M3", DType::kFloat8E4M3)
        .value("kFloat8E5M2", DType::kFloat8E5M2);

    py::enum_<NVTE_Bias_Type>(m, "NVTE_Bias_Type")
        .value("NVTE_NO_BIAS", NVTE_Bias_Type::NVTE_NO_BIAS)
        .value("NVTE_PRE_SCALE_BIAS", NVTE_Bias_Type::NVTE_PRE_SCALE_BIAS)
        .value("NVTE_POST_SCALE_BIAS", NVTE_Bias_Type::NVTE_POST_SCALE_BIAS);

    py::enum_<NVTE_Mask_Type>(m, "NVTE_Mask_Type")
        .value("NVTE_NO_MASK", NVTE_Mask_Type::NVTE_NO_MASK)
        .value("NVTE_PADDING_MASK", NVTE_Mask_Type::NVTE_PADDING_MASK)
        .value("NVTE_CAUSAL_MASK", NVTE_Mask_Type::NVTE_CAUSAL_MASK);

    py::enum_<NVTE_QKV_Layout>(m, "NVTE_QKV_Layout")
        .value("NVTE_SB3HD", NVTE_QKV_Layout::NVTE_SB3HD)
        .value("NVTE_SBH3D", NVTE_QKV_Layout::NVTE_SBH3D)
        .value("NVTE_SBHD_SB2HD", NVTE_QKV_Layout::NVTE_SBHD_SB2HD)
        .value("NVTE_SBHD_SBH2D", NVTE_QKV_Layout::NVTE_SBHD_SBH2D)
        .value("NVTE_SBHD_SBHD_SBHD", NVTE_QKV_Layout::NVTE_SBHD_SBHD_SBHD)
        .value("NVTE_BS3HD", NVTE_QKV_Layout::NVTE_BS3HD)
        .value("NVTE_BSH3D", NVTE_QKV_Layout::NVTE_BSH3D)
        .value("NVTE_BSHD_BS2HD", NVTE_QKV_Layout::NVTE_BSHD_BS2HD)
        .value("NVTE_BSHD_BSH2D", NVTE_QKV_Layout::NVTE_BSHD_BSH2D)
        .value("NVTE_BSHD_BSHD_BSHD", NVTE_QKV_Layout::NVTE_BSHD_BSHD_BSHD)
        .value("NVTE_T3HD", NVTE_QKV_Layout::NVTE_T3HD)
        .value("NVTE_TH3D", NVTE_QKV_Layout::NVTE_TH3D)
        .value("NVTE_THD_T2HD", NVTE_QKV_Layout::NVTE_THD_T2HD)
        .value("NVTE_THD_TH2D", NVTE_QKV_Layout::NVTE_THD_TH2D)
        .value("NVTE_THD_THD_THD", NVTE_QKV_Layout::NVTE_THD_THD_THD);

    py::enum_<NVTE_Fused_Attn_Backend>(m, "NVTE_Fused_Attn_Backend", py::module_local())
        .value("NVTE_F16_max512_seqlen", NVTE_Fused_Attn_Backend::NVTE_F16_max512_seqlen)
        .value("NVTE_F16_arbitrary_seqlen", NVTE_Fused_Attn_Backend::NVTE_F16_arbitrary_seqlen)
        .value("NVTE_FP8", NVTE_Fused_Attn_Backend::NVTE_FP8)
        .value("NVTE_No_Backend", NVTE_Fused_Attn_Backend::NVTE_No_Backend);
}
}  // namespace paddle_ext
}  // namespace transformer_engine
