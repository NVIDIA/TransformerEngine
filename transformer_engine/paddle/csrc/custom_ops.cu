#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <vector>
#include "../common.h"
#include "common.h"

namespace transformer_engine {
namespace paddle_ext {

// MHA utils
// convert QKV layout to enum
NVTE_QKV_Layout get_nvte_qkv_layout(const std::string qkv_layout) {
    if (qkv_layout == "not_interleaved") {
        return NVTE_QKV_Layout::NVTE_NOT_INTERLEAVED;
    } else if (qkv_layout == "qkv_interleaved") {
        return NVTE_QKV_Layout::NVTE_QKV_INTERLEAVED;
    } else if (qkv_layout == "kv_interleaved") {
        return NVTE_QKV_Layout::NVTE_KV_INTERLEAVED;
    } else {
        NVTE_ERROR("Invalid QKV layout. \n");
    }
}

// convert bias type to enum
NVTE_Bias_Type get_nvte_bias_type(const std::string bias_type) {
    if (bias_type == "no_bias") {
        return NVTE_Bias_Type::NVTE_NO_BIAS;
    } else if (bias_type == "pre_scale_bias") {
        return NVTE_Bias_Type::NVTE_PRE_SCALE_BIAS;
    } else if (bias_type == "post_scale_bias") {
        return NVTE_Bias_Type::NVTE_POST_SCALE_BIAS;
    } else {
        NVTE_ERROR("Invalid bias type. \n");
    }
}

// convert attn mask type to enum
NVTE_Mask_Type get_nvte_mask_type(const std::string mask_type) {
    if (mask_type == "padding") {
        return NVTE_Mask_Type::NVTE_PADDING_MASK;
    } else if (mask_type == "causal") {
        return NVTE_Mask_Type::NVTE_CAUSAL_MASK;
    } else if (mask_type == "no_mask") {
        return NVTE_Mask_Type::NVTE_NO_MASK;
    } else {
        NVTE_ERROR("Invalid attention mask type. \n");
    }
}

std::vector<paddle::Tensor> cast_to_fp8(const paddle::Tensor &input, const paddle::Tensor &scale,
                                        paddle::Tensor &amax, paddle::Tensor &scale_inv,  // NOLINT
                                        int64_t index, int64_t otype) {
    auto shape = GetShapeArray(input);

    auto output = paddle::empty_like(input, Nvte2PaddleDType(Int2NvteDType(otype)));

    auto input_cu = MakeNvteTensor(input);
    auto output_cu = MakeNvteTensor(
        output.data(), shape, Int2NvteDType(otype), GetDataPtr<float>(amax, index),
        const_cast<void *>(GetDataPtr<float>(scale, index)), GetDataPtr<float>(scale_inv, index));

    nvte_fp8_quantize(input_cu.data(), output_cu.data(), input.stream());

    return {output};
}

std::vector<paddle::Tensor> cast_from_fp8(const paddle::Tensor &input,
                                          const paddle::Tensor &scale_inv, int64_t index,
                                          int64_t itype, int64_t otype) {
    auto shape = GetShapeArray(input);

    auto output = paddle::empty_like(input, Nvte2PaddleDType(Int2NvteDType(otype)));
    auto input_cu =
        MakeNvteTensor(const_cast<void *>(input.data()), shape, Int2NvteDType(itype), nullptr,
                       nullptr, const_cast<void *>(GetDataPtr<float>(scale_inv, index)));
    auto output_cu = MakeNvteTensor(output);

    nvte_fp8_dequantize(input_cu.data(), output_cu.data(), input.stream());

    return {output};
}

std::vector<paddle::Tensor> te_transpose(const paddle::Tensor &input, int64_t otype) {
    auto shape = GetShapeArray(input);
    NVTE_CHECK(shape.size() == 2, "Expect the input to have 2 dimensions.");
    size_t M = shape[0];
    size_t N = shape[1];

    auto output = paddle::empty({input.shape()[1], input.shape()[0]}, input.dtype(), input.place());

    auto input_cu = MakeNvteTensor(const_cast<void *>(input.data()), {M, N}, Int2NvteDType(otype));
    auto output_cu = MakeNvteTensor(output.data(), {N, M}, Int2NvteDType(otype));

    nvte_transpose(input_cu.data(), output_cu.data(), input.stream());

    return {output};
}

std::vector<paddle::Tensor> te_cast_transpose(const paddle::Tensor &input,
                                              const paddle::Tensor &scale,
                                              paddle::Tensor &amax,       // NOLINT
                                              paddle::Tensor &scale_inv,  // NOLINT
                                              int64_t index, int64_t otype) {
    auto shape = GetShapeArray(input);
    NVTE_CHECK(shape.size() == 2, "Expect the input to have 2 dimensions.");

    size_t M = shape[0];
    size_t N = shape[1];

    auto input_cast =
        paddle::empty_like(input, Nvte2PaddleDType(Int2NvteDType(otype)), input.place());
    auto input_transpose = paddle::empty({input.shape()[1], input.shape()[0]},
                                         Nvte2PaddleDType(Int2NvteDType(otype)), input.place());

    auto input_cu = MakeNvteTensor(input);
    void *amax_data = GetDataPtr<float>(amax, index);
    void *scale_data = const_cast<void *>(GetDataPtr<float>(scale, index));
    void *scale_inv_data = GetDataPtr<float>(scale_inv, index);
    auto output_cast_cu = MakeNvteTensor(input_cast.data(), {M, N}, Int2NvteDType(otype), amax_data,
                                         scale_data, scale_inv_data);
    auto output_transpose_cu = MakeNvteTensor(input_transpose.data(), {N, M}, Int2NvteDType(otype),
                                              amax_data, scale_data, scale_inv_data);

    nvte_cast_transpose(input_cu.data(), output_cast_cu.data(), output_transpose_cu.data(),
                        input.stream());

    return {input_cast, input_transpose};
}

std::vector<paddle::Tensor> te_cast_transpose_bgrad(const paddle::Tensor &grad_output,
                                                    const paddle::Tensor &scale,
                                                    paddle::Tensor &amax,       // NOLINT
                                                    paddle::Tensor &scale_inv,  // NOLINT
                                                    int64_t index, int64_t otype) {
    auto shape = GetShapeArray(grad_output);
    NVTE_CHECK(shape.size() == 2, "Expect the input to have 2 dimensions.");

    size_t M = shape[0];
    size_t N = shape[1];

    auto grad_bias =
        paddle::empty({grad_output.shape()[1]}, grad_output.dtype(), grad_output.place());
    auto grad_output_cast = paddle::empty_like(grad_output, Nvte2PaddleDType(Int2NvteDType(otype)),
                                               grad_output.place());
    auto grad_output_transpose =
        paddle::empty({grad_output.shape()[1], grad_output.shape()[0]},
                      Nvte2PaddleDType(Int2NvteDType(otype)), grad_output.place());

    auto input_cu = MakeNvteTensor(grad_output);
    void *amax_data = GetDataPtr<float>(amax, index);
    void *scale_data = const_cast<void *>(GetDataPtr<float>(scale, index));
    void *scale_inv_data = GetDataPtr<float>(scale_inv, index);
    auto output_cast_cu = MakeNvteTensor(grad_output_cast.data(), {M, N}, Int2NvteDType(otype),
                                         amax_data, scale_data, scale_inv_data);
    auto output_transpose_cu =
        MakeNvteTensor(grad_output_transpose.data(), {N, M}, Int2NvteDType(otype), amax_data,
                       scale_data, scale_inv_data);
    auto dbias_cu = MakeNvteTensor(grad_bias);
    transformer_engine::TensorWrapper workspace;

    nvte_cast_transpose_dbias(input_cu.data(), output_cast_cu.data(), output_transpose_cu.data(),
                              dbias_cu.data(), workspace.data(), grad_output.stream());

    // Fill workspace
    auto workspace_data = AllocateSpace(workspace.shape(), workspace.dtype(), grad_output.place());
    workspace = MakeNvteTensor(workspace_data.data(), workspace.shape(), workspace.dtype());

    nvte_cast_transpose_dbias(input_cu.data(), output_cast_cu.data(), output_transpose_cu.data(),
                              dbias_cu.data(), workspace.data(), grad_output.stream());

    return {grad_bias, grad_output_cast, grad_output_transpose};
}

void te_gemm(const paddle::Tensor &A, const paddle::optional<paddle::Tensor> &A_scale_inverse,
             const paddle::Tensor &B, const paddle::optional<paddle::Tensor> &B_scale_inverse,
             const paddle::optional<paddle::Tensor> &bias, paddle::Tensor &D,            // NOLINT
             paddle::optional<paddle::Tensor> &D_scale,                                  // NOLINT
             paddle::optional<paddle::Tensor> &D_amax,                                   // NOLINT
             paddle::optional<paddle::Tensor> &pre_gelu_out, paddle::Tensor &workspace,  // NOLINT
             int64_t A_index, int64_t B_index, int64_t D_index, int64_t A_type, int64_t B_type,
             int64_t D_type, int64_t bias_type, bool transa, bool transb, bool grad,
             int64_t workspace_size, bool accumulate, bool use_split_accumulator,
             int64_t math_sm_count) {
    auto te_A = MakeNvteTensor(
        const_cast<void *>(A.data()), GetShapeArray(A), Int2NvteDType(A_type), nullptr, nullptr,
        const_cast<void *>(GetOptionalDataPtr<float>(A_scale_inverse, A_index)));
    auto te_B = MakeNvteTensor(
        const_cast<void *>(B.data()), GetShapeArray(B), Int2NvteDType(B_type), nullptr, nullptr,
        const_cast<void *>(GetOptionalDataPtr<float>(B_scale_inverse, B_index)));
    auto te_D = MakeNvteTensor(D.data(), GetShapeArray(D), Int2NvteDType(D_type),
                               GetOptionalDataPtr<float>(D_amax, D_index),
                               GetOptionalDataPtr<float>(D_scale, D_index), nullptr);

    auto te_bias = MakeNvteTensor(const_cast<void *>(GetOptionalDataPtr(bias)), GetShapeArray(bias),
                                  Int2NvteDType(bias_type));

    DType gelu_dtype =
        pre_gelu_out ? Paddle2NvteDType(pre_gelu_out->dtype()) : Int2NvteDType(D_type);
    auto te_pre_gelu_out =
        MakeNvteTensor(GetOptionalDataPtr(pre_gelu_out), GetShapeArray(pre_gelu_out), gelu_dtype);
    auto te_workspace =
        MakeNvteTensor(workspace.data(), {static_cast<size_t>(workspace_size)}, DType::kByte);

    nvte_cublas_gemm(te_A.data(), te_B.data(), te_D.data(), te_bias.data(), te_pre_gelu_out.data(),
                     transa, transb, grad, te_workspace.data(), accumulate, use_split_accumulator,
                     math_sm_count, A.stream());
}

std::vector<paddle::Tensor> te_gelu_fp8(const paddle::Tensor &input, const paddle::Tensor &scale,
                                        paddle::Tensor &amax,       // NOLINT
                                        paddle::Tensor &scale_inv,  // NOLINT
                                        int64_t index, int64_t otype) {
    auto output = paddle::empty_like(input, Nvte2PaddleDType(DType::kByte), input.place());

    auto input_cu = MakeNvteTensor(input);
    auto output_cu = MakeNvteTensor(
        output.data(), GetShapeArray(input), Int2NvteDType(otype), GetDataPtr<float>(amax, index),
        const_cast<void *>(GetDataPtr<float>(scale, index)), GetDataPtr<float>(scale_inv, index));

    nvte_gelu(input_cu.data(), output_cu.data(), input.stream());

    return {output};
}

std::vector<paddle::Tensor> te_gelu(const paddle::Tensor &input, int64_t otype) {
    auto output = paddle::empty_like(input, Nvte2PaddleDType(Int2NvteDType(otype)), input.place());

    auto input_cu = MakeNvteTensor(input);
    auto output_cu = MakeNvteTensor(output.data(), GetShapeArray(input), Int2NvteDType(otype));

    nvte_gelu(input_cu.data(), output_cu.data(), input.stream());

    return {output};
}

std::vector<paddle::Tensor> te_cast_transpose_bgrad_dgelu(const paddle::Tensor &grad_output,
                                                          const paddle::Tensor &gelu_input,
                                                          const paddle::Tensor &scale,
                                                          paddle::Tensor &amax,       // NOLINT
                                                          paddle::Tensor &scale_inv,  // NOLINT
                                                          int64_t index, int64_t otype) {
    auto shape = GetShapeArray(grad_output);
    NVTE_CHECK(shape.size() == 2, "Expect the grad_output to have 2 dimensions.");

    size_t M = shape[0];
    size_t N = shape[1];

    // DType grad_output_type = GetTransformerEngineDType(grad_output.scalar_type());
    auto grad_bias =
        paddle::empty({grad_output.shape()[1]}, grad_output.dtype(), grad_output.place());

    auto dgelu =
        paddle::empty_like(grad_output, Nvte2PaddleDType(DType::kByte), grad_output.place());

    auto dgelu_transpose = paddle::empty({grad_output.shape()[1], grad_output.shape()[0]},
                                         Nvte2PaddleDType(DType::kByte), grad_output.place());

    void *amax_data = GetDataPtr<float>(amax, index);
    void *scale_data = const_cast<void *>(GetDataPtr<float>(scale, index));
    void *scale_inv_data = GetDataPtr<float>(scale_inv, index);

    TensorWrapper workspace;

    auto gelu_input_cu = MakeNvteTensor(gelu_input);
    auto input_cu = MakeNvteTensor(grad_output);
    auto cast_output_cu = MakeNvteTensor(dgelu.data(), {M, N}, Int2NvteDType(otype), amax_data,
                                         scale_data, scale_inv_data);
    auto transposed_output_cu = MakeNvteTensor(dgelu_transpose.data(), {N, M}, Int2NvteDType(otype),
                                               amax_data, scale_data, scale_inv_data);
    auto dbias_cu = MakeNvteTensor(grad_bias);

    nvte_cast_transpose_dbias_dgelu(input_cu.data(), gelu_input_cu.data(), cast_output_cu.data(),
                                    transposed_output_cu.data(), dbias_cu.data(), workspace.data(),
                                    grad_output.stream());

    // Fill workspace
    auto workspace_data = AllocateSpace(workspace.shape(), workspace.dtype(), grad_output.place());
    workspace = MakeNvteTensor(workspace_data.data(), workspace.shape(), workspace.dtype());

    nvte_cast_transpose_dbias_dgelu(input_cu.data(), gelu_input_cu.data(), cast_output_cu.data(),
                                    transposed_output_cu.data(), dbias_cu.data(), workspace.data(),
                                    grad_output.stream());

    return {dgelu, dgelu_transpose, grad_bias};
}

std::vector<paddle::Tensor> te_layernorm_fwd_fp8(const paddle::Tensor &input,
                                                 const paddle::Tensor &weight,
                                                 const paddle::Tensor &bias,
                                                 const paddle::Tensor &scale,
                                                 paddle::Tensor &amax,       // NOLINT
                                                 paddle::Tensor &scale_inv,  // NOLINT
                                                 float eps, int64_t index, int64_t otype,
                                                 int64_t sm_margin, bool zero_centered_gamma) {
    auto shape = GetShapeArray(input);
    NVTE_CHECK(shape.size() == 2, "Expect the grad_output to have 2 dimensions.");

    size_t N = shape[0];
    size_t H = shape[1];

    auto ln_out = paddle::empty_like(input, input.dtype(), input.place());
    auto mu = paddle::empty({static_cast<int64_t>(N)}, paddle::DataType::FLOAT32, input.place());
    auto rsigma =
        paddle::empty({static_cast<int64_t>(N)}, paddle::DataType::FLOAT32, input.place());
    auto input_cu = MakeNvteTensor(input);
    auto gamma_cu = MakeNvteTensor(weight);
    auto beta_cu = MakeNvteTensor(bias);
    auto z_cu = MakeNvteTensor(
        ln_out.data(), {N, H}, Int2NvteDType(otype), GetDataPtr<float>(amax, index),
        const_cast<void *>(GetDataPtr<float>(scale, index)), GetDataPtr<float>(scale_inv, index));
    auto mu_cu = MakeNvteTensor(mu);
    auto rsigma_cu = MakeNvteTensor(rsigma);
    TensorWrapper workspace, barrier;

    auto num_sm = cudaDevicePropertiesManager::Instance().GetMultiProcessorCount();

    // This call populates workspace and barrier tensors with the required config
    const auto func = zero_centered_gamma ? nvte_layernorm1p_fwd : nvte_layernorm_fwd;
    func(input_cu.data(), gamma_cu.data(), beta_cu.data(), eps, z_cu.data(), mu_cu.data(),
         rsigma_cu.data(), input.stream(), num_sm - sm_margin, workspace.data(), barrier.data());

    // Fill workspace and barrier
    auto workspace_data = AllocateSpace(workspace.shape(), workspace.dtype(), input.place());
    auto barrier_data = AllocateSpace(barrier.shape(), barrier.dtype(), input.place(), true);
    workspace = MakeNvteTensor(workspace_data.data(), workspace.shape(), workspace.dtype());
    barrier = MakeNvteTensor(barrier_data.data(), barrier.shape(), barrier.dtype());

    // Actual call to fwd kernel
    func(input_cu.data(), gamma_cu.data(), beta_cu.data(), eps, z_cu.data(), mu_cu.data(),
         rsigma_cu.data(), input.stream(), num_sm - sm_margin, workspace.data(), barrier.data());

    return {ln_out, mu, rsigma};
}

std::vector<paddle::Tensor> te_layernorm_fwd(const paddle::Tensor &input,
                                             const paddle::Tensor &weight,
                                             const paddle::Tensor &bias, float eps, int64_t otype,
                                             int64_t sm_margin, bool zero_centered_gamma) {
    auto shape = GetShapeArray(input);
    NVTE_CHECK(shape.size() == 2, "Expect the grad_output to have 2 dimensions.");

    size_t N = shape[0];
    size_t H = shape[1];

    auto ln_out = paddle::empty_like(input, input.dtype(), input.place());
    auto mu = paddle::empty({static_cast<int64_t>(N)}, paddle::DataType::FLOAT32, input.place());
    auto rsigma =
        paddle::empty({static_cast<int64_t>(N)}, paddle::DataType::FLOAT32, input.place());
    auto input_cu = MakeNvteTensor(input);
    auto gamma_cu = MakeNvteTensor(weight);
    auto beta_cu = MakeNvteTensor(bias);
    auto z_cu = MakeNvteTensor(ln_out.data(), {N, H}, Int2NvteDType(otype));
    auto mu_cu = MakeNvteTensor(mu);
    auto rsigma_cu = MakeNvteTensor(rsigma);
    TensorWrapper workspace, barrier;

    auto num_sm = cudaDevicePropertiesManager::Instance().GetMultiProcessorCount();

    // This call populates workspace and barrier tensors with the required config
    const auto func = zero_centered_gamma ? nvte_layernorm1p_fwd : nvte_layernorm_fwd;
    func(input_cu.data(), gamma_cu.data(), beta_cu.data(), eps, z_cu.data(), mu_cu.data(),
         rsigma_cu.data(), input.stream(), num_sm - sm_margin, workspace.data(), barrier.data());

    // Fill workspace and barrier
    auto workspace_data = AllocateSpace(workspace.shape(), workspace.dtype(), input.place());
    auto barrier_data = AllocateSpace(barrier.shape(), barrier.dtype(), input.place(), true);
    workspace = MakeNvteTensor(workspace_data.data(), workspace.shape(), workspace.dtype());
    barrier = MakeNvteTensor(barrier_data.data(), barrier.shape(), barrier.dtype());

    // Actual call to fwd kernel
    func(input_cu.data(), gamma_cu.data(), beta_cu.data(), eps, z_cu.data(), mu_cu.data(),
         rsigma_cu.data(), input.stream(), num_sm - sm_margin, workspace.data(), barrier.data());

    return {ln_out, mu, rsigma};
}

std::vector<paddle::Tensor> te_layernorm_bwd(const paddle::Tensor &dz, const paddle::Tensor &x,
                                             const paddle::Tensor &mu, const paddle::Tensor &rsigma,
                                             const paddle::Tensor &gamma, int64_t sm_margin,
                                             bool zero_centered_gamma) {
    auto dx = paddle::empty_like(x, x.dtype(), x.place());
    auto dgamma = paddle::empty_like(gamma, gamma.dtype(), gamma.place());
    auto dbeta = paddle::empty_like(gamma, gamma.dtype(), gamma.place());

    TensorWrapper workspace, barrier, dgamma_part, dbeta_part;

    auto dz_cu = MakeNvteTensor(dz);
    auto x_cu = MakeNvteTensor(x);
    auto mu_cu = MakeNvteTensor(mu);
    auto rsigma_cu = MakeNvteTensor(rsigma);
    auto gamma_cu = MakeNvteTensor(gamma);
    auto dx_cu = MakeNvteTensor(dx);
    auto dgamma_cu = MakeNvteTensor(dgamma);
    auto dbeta_cu = MakeNvteTensor(dbeta);

    auto num_sm = cudaDevicePropertiesManager::Instance().GetMultiProcessorCount();

    // This call populates tensors with the required config.
    const auto bwd_fun = zero_centered_gamma ? nvte_layernorm1p_bwd : nvte_layernorm_bwd;
    bwd_fun(dz_cu.data(), x_cu.data(), mu_cu.data(), rsigma_cu.data(), gamma_cu.data(),
            dx_cu.data(), dgamma_cu.data(), dbeta_cu.data(), dgamma_part.data(), dbeta_part.data(),
            dz.stream(), num_sm - sm_margin, workspace.data(), barrier.data());

    // Alloc space for Tensors.
    auto workspace_data = AllocateSpace(workspace.shape(), workspace.dtype(), x.place());
    auto barrier_data = AllocateSpace(barrier.shape(), barrier.dtype(), x.place(), true);
    auto dgamma_part_data = AllocateSpace(dgamma_part.shape(), dgamma_part.dtype(), x.place());
    auto dbeta_part_data = AllocateSpace(dbeta_part.shape(), dbeta_part.dtype(), x.place());
    workspace = MakeNvteTensor(workspace_data.data(), workspace.shape(), workspace.dtype());
    barrier = MakeNvteTensor(barrier_data.data(), barrier.shape(), barrier.dtype());
    dgamma_part = MakeNvteTensor(dgamma_part_data.data(), dgamma_part.shape(), dgamma_part.dtype());
    dbeta_part = MakeNvteTensor(dbeta_part_data.data(), dbeta_part.shape(), dbeta_part.dtype());

    // Actual call to bwd kernel.
    bwd_fun(dz_cu.data(), x_cu.data(), mu_cu.data(), rsigma_cu.data(), gamma_cu.data(),
            dx_cu.data(), dgamma_cu.data(), dbeta_cu.data(), dgamma_part.data(), dbeta_part.data(),
            dz.stream(), num_sm - sm_margin, workspace.data(), barrier.data());

    return {dx, dgamma, dbeta};
}

std::vector<paddle::Tensor> te_rmsnorm_fwd(const paddle::Tensor &input,
                                           const paddle::Tensor &weight, float eps, int64_t otype,
                                           int64_t sm_margin) {
    auto shape = GetShapeArray(input);
    NVTE_CHECK(shape.size() == 2, "Expect the grad_output to have 2 dimensions.");

    size_t N = shape[0];
    size_t H = shape[1];

    auto ln_out = paddle::empty_like(input, input.dtype(), input.place());
    auto rsigma =
        paddle::empty({static_cast<int64_t>(N)}, paddle::DataType::FLOAT32, input.place());
    auto input_cu = MakeNvteTensor(input);
    auto gamma_cu = MakeNvteTensor(weight);
    auto z_cu = MakeNvteTensor(ln_out.data(), {N, H}, Int2NvteDType(otype));
    auto rsigma_cu = MakeNvteTensor(rsigma);
    TensorWrapper workspace, barrier;

    auto num_sm = cudaDevicePropertiesManager::Instance().GetMultiProcessorCount();

    // This call populates workspace and barrier tensors with the required config

    nvte_rmsnorm_fwd(input_cu.data(), gamma_cu.data(), eps, z_cu.data(), rsigma_cu.data(),
                     input.stream(), num_sm - sm_margin, workspace.data(), barrier.data());

    // Fill workspace and barrier
    auto workspace_data = AllocateSpace(workspace.shape(), workspace.dtype(), input.place());
    auto barrier_data = AllocateSpace(barrier.shape(), barrier.dtype(), input.place(), true);
    workspace = MakeNvteTensor(workspace_data.data(), workspace.shape(), workspace.dtype());
    barrier = MakeNvteTensor(barrier_data.data(), barrier.shape(), barrier.dtype());

    // Actual call to fwd kernel
    nvte_rmsnorm_fwd(input_cu.data(), gamma_cu.data(), eps, z_cu.data(), rsigma_cu.data(),
                     input.stream(), num_sm - sm_margin, workspace.data(), barrier.data());

    return {ln_out, rsigma};
}

std::vector<paddle::Tensor> te_rmsnorm_fwd_fp8(const paddle::Tensor &input,
                                               const paddle::Tensor &weight,
                                               const paddle::Tensor &scale,
                                               paddle::Tensor &amax,       // NOLINT
                                               paddle::Tensor &scale_inv,  // NOLINT
                                               float eps, int64_t index, int64_t otype,
                                               int64_t sm_margin) {
    auto shape = GetShapeArray(input);
    NVTE_CHECK(shape.size() == 2, "Expect the grad_output to have 2 dimensions.");

    size_t N = shape[0];
    size_t H = shape[1];

    auto ln_out = paddle::empty_like(input, input.dtype(), input.place());
    auto rsigma =
        paddle::empty({static_cast<int64_t>(N)}, paddle::DataType::FLOAT32, input.place());
    auto input_cu = MakeNvteTensor(input);
    auto gamma_cu = MakeNvteTensor(weight);
    auto z_cu = MakeNvteTensor(
        ln_out.data(), {N, H}, Int2NvteDType(otype), GetDataPtr<float>(amax, index),
        const_cast<void *>(GetDataPtr<float>(scale, index)), GetDataPtr<float>(scale_inv, index));
    auto rsigma_cu = MakeNvteTensor(rsigma);
    TensorWrapper workspace, barrier;

    auto num_sm = cudaDevicePropertiesManager::Instance().GetMultiProcessorCount();

    // This call populates workspace and barrier tensors with the required config
    nvte_rmsnorm_fwd(input_cu.data(), gamma_cu.data(), eps, z_cu.data(), rsigma_cu.data(),
                     input.stream(), num_sm - sm_margin, workspace.data(), barrier.data());

    // Fill workspace and barrier
    auto workspace_data = AllocateSpace(workspace.shape(), workspace.dtype(), input.place());
    auto barrier_data = AllocateSpace(barrier.shape(), barrier.dtype(), input.place(), true);
    workspace = MakeNvteTensor(workspace_data.data(), workspace.shape(), workspace.dtype());
    barrier = MakeNvteTensor(barrier_data.data(), barrier.shape(), barrier.dtype());

    // Actual call to fwd kernel
    nvte_rmsnorm_fwd(input_cu.data(), gamma_cu.data(), eps, z_cu.data(), rsigma_cu.data(),
                     input.stream(), num_sm - sm_margin, workspace.data(), barrier.data());

    return {ln_out, rsigma};
}

std::vector<paddle::Tensor> te_rmsnorm_bwd(const paddle::Tensor &dz, const paddle::Tensor &x,
                                           const paddle::Tensor &rsigma,
                                           const paddle::Tensor &gamma, int64_t sm_margin) {
    auto dx = paddle::empty_like(x, x.dtype(), x.place());
    auto dgamma = paddle::empty_like(gamma, gamma.dtype(), gamma.place());

    TensorWrapper workspace, barrier, dgamma_part;

    auto dz_cu = MakeNvteTensor(dz);
    auto x_cu = MakeNvteTensor(x);
    auto rsigma_cu = MakeNvteTensor(rsigma);
    auto gamma_cu = MakeNvteTensor(gamma);
    auto dx_cu = MakeNvteTensor(dx);
    auto dgamma_cu = MakeNvteTensor(dgamma);

    auto num_sm = cudaDevicePropertiesManager::Instance().GetMultiProcessorCount();

    // This call populates tensors with the required config.
    nvte_rmsnorm_bwd(dz_cu.data(), x_cu.data(), rsigma_cu.data(), gamma_cu.data(), dx_cu.data(),
                     dgamma_cu.data(), dgamma_part.data(), dz.stream(), num_sm - sm_margin,
                     workspace.data(), barrier.data());

    // Alloc space for Tensors.
    auto workspace_data = AllocateSpace(workspace.shape(), workspace.dtype(), x.place());
    auto barrier_data = AllocateSpace(barrier.shape(), barrier.dtype(), x.place(), true);
    auto dgamma_part_data = AllocateSpace(dgamma_part.shape(), dgamma_part.dtype(), x.place());
    workspace = MakeNvteTensor(workspace_data.data(), workspace.shape(), workspace.dtype());
    barrier = MakeNvteTensor(barrier_data.data(), barrier.shape(), barrier.dtype());
    dgamma_part = MakeNvteTensor(dgamma_part_data.data(), dgamma_part.shape(), dgamma_part.dtype());

    // Actual call to bwd kernel.
    nvte_rmsnorm_bwd(dz_cu.data(), x_cu.data(), rsigma_cu.data(), gamma_cu.data(), dx_cu.data(),
                     dgamma_cu.data(), dgamma_part.data(), dz.stream(), num_sm - sm_margin,
                     workspace.data(), barrier.data());

    return {dx, dgamma};
}

void te_fused_attn_fwd_qkvpacked(const paddle::Tensor &QKV, const paddle::Tensor &cu_seqlens,
                                 const paddle::optional<paddle::Tensor> &Bias,
                                 paddle::Tensor &O,                              // NOLINT
                                 paddle::optional<paddle::Tensor> &softmax_aux,  // NOLINT
                                 paddle::Tensor &rng_state,                      // NOLINT
                                 int64_t b, int64_t h, int64_t d, int64_t total_seqs,
                                 int64_t max_seqlen, bool is_training, float attn_scale,
                                 float p_dropout, const std::string &qkv_layout,
                                 const std::string &bias_type, const std::string &attn_mask_type,
                                 const int64_t qkv_type) {
    if (is_training && !softmax_aux) {
        NVTE_ERROR("softmax_aux must be provided when training. \n");
    }

    auto qkv_dtype = Int2NvteDType(qkv_type);
    // construct NVTE tensors
    TensorWrapper te_QKV, te_S, te_O, te_Bias, te_cu_seqlens;
    if (qkv_dtype == DType::kBFloat16 || qkv_dtype == DType::kFloat16) {
        // BF16 or FP16
        te_QKV = MakeNvteTensor(QKV);
        te_S = MakeNvteTensor(nullptr, std::vector<size_t>{0}, DType::kFloat32);
        te_O = MakeNvteTensor(O);
    } else {  // TODO: support fp8
        NVTE_ERROR("Fused attention only supports BF16/FP16 data types. \n");
    }
    if ((bias_type != "no_bias") && Bias) {
        auto bias_shape = Bias->shape();
        std::vector<size_t> shape{bias_shape.begin(), bias_shape.end()};
        te_Bias = MakeNvteTensor(GetOptionalDataPtr(Bias), shape, DType::kFloat32);
    }
    te_cu_seqlens = MakeNvteTensor(cu_seqlens.data(), {static_cast<size_t>(b + 1)}, DType::kInt32);

    // convert strings to enums
    NVTE_QKV_Layout qkv_layout_enum = get_nvte_qkv_layout(qkv_layout);
    NVTE_Bias_Type bias_type_enum = get_nvte_bias_type(bias_type);
    NVTE_Mask_Type attn_mask_type_enum = get_nvte_mask_type(attn_mask_type);

    // extract random number generator seed and offset
    auto te_rng_state = MakeNvteTensor(rng_state);

    // create auxiliary output tensors
    NVTETensorPack nvte_aux_tensor_pack;
    nvte_tensor_pack_create(&nvte_aux_tensor_pack);

    // create workspace
    TensorWrapper workspace;

    // populate tensors with appropriate shapes and dtypes
    nvte_fused_attn_fwd_qkvpacked(
        te_QKV.data(), te_Bias.data(), te_S.data(), te_O.data(), &nvte_aux_tensor_pack,
        te_cu_seqlens.data(), te_rng_state.data(), max_seqlen, is_training, attn_scale, p_dropout,
        qkv_layout_enum, bias_type_enum, attn_mask_type_enum, workspace.data(), QKV.stream());

    // allocate memory for workspace and auxiliary output tensors
    auto workspace_data = AllocateSpace(workspace.shape(), workspace.dtype(), QKV.place());
    workspace = MakeNvteTensor(workspace_data.data(), workspace.shape(), workspace.dtype());

    auto *output_s =
        reinterpret_cast<transformer_engine::Tensor *>(nvte_aux_tensor_pack.tensors[0]);
    output_s->data.dptr = GetOptionalDataPtr(softmax_aux);

    // execute the kernel
    nvte_fused_attn_fwd_qkvpacked(
        te_QKV.data(), te_Bias.data(), te_S.data(), te_O.data(), &nvte_aux_tensor_pack,
        te_cu_seqlens.data(), te_rng_state.data(), max_seqlen, is_training, attn_scale, p_dropout,
        qkv_layout_enum, bias_type_enum, attn_mask_type_enum, workspace.data(), QKV.stream());

    // destroy tensor wrappers, but not allocated memory
    nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);
}

// fused attention BWD with packed QKV
void te_fused_attn_bwd_qkvpacked(const paddle::Tensor &QKV, const paddle::Tensor &cu_seqlens,
                                 const paddle::Tensor &O, const paddle::Tensor &dO,
                                 const paddle::Tensor &softmax_aux,
                                 paddle::Tensor &dQKV,                     // NOLINT
                                 paddle::optional<paddle::Tensor> &dBias,  // NOLINT
                                 paddle::Tensor &rng_state,                // NOLINT
                                 int64_t b, int64_t h, int64_t d, int64_t total_seqs,
                                 int64_t max_seqlen, float attn_scale, float p_dropout,
                                 const std::string &qkv_layout, const std::string &bias_type,
                                 const std::string &attn_mask_type, int64_t qkv_type) {
    TensorWrapper te_dBias;
    if (bias_type != "no_bias" && dBias) {
        auto bias_shape = dBias->shape();
        std::vector<size_t> shape{bias_shape.begin(), bias_shape.end()};
        te_dBias = MakeNvteTensor(GetOptionalDataPtr(dBias), shape, DType::kFloat32);
    }

    auto qkv_dtype = Int2NvteDType(qkv_type);
    // construct NVTE tensors
    TensorWrapper te_QKV, te_O, te_dO, te_S, te_dP, te_dQKV;
    if (qkv_dtype == DType::kBFloat16 || qkv_dtype == DType::kFloat16) {
        // BF16 or FP16
        te_QKV = MakeNvteTensor(QKV);
        te_O = MakeNvteTensor(O);
        te_dO = MakeNvteTensor(dO);
        te_S = MakeNvteTensor(nullptr, std::vector<size_t>(0), DType::kFloat32);
        te_dP = MakeNvteTensor(nullptr, std::vector<size_t>(0), DType::kFloat32);
        te_dQKV = MakeNvteTensor(dQKV);
    } else {
        NVTE_ERROR("Fused attention only supports BF16/FP16 data types. \n");
    }

    // convert strings to enums
    NVTE_QKV_Layout qkv_layout_enum = get_nvte_qkv_layout(qkv_layout);
    NVTE_Bias_Type bias_type_enum = get_nvte_bias_type(bias_type);
    NVTE_Mask_Type attn_mask_type_enum = get_nvte_mask_type(attn_mask_type);

    // convert auxiliary tensors from forward into NVTETensors
    NVTETensorPack nvte_aux_tensor_pack;
    nvte_tensor_pack_create(&nvte_aux_tensor_pack);

    nvte_aux_tensor_pack.size = 2;  // 1. softmax_aux  2. rng_state
    auto *output_s = reinterpret_cast<Tensor *>(nvte_aux_tensor_pack.tensors[0]);
    auto *fwd_rng_state = reinterpret_cast<Tensor *>(nvte_aux_tensor_pack.tensors[1]);
    output_s->data.shape =
        std::vector<size_t>({static_cast<size_t>(b), static_cast<size_t>(h),
                             static_cast<size_t>(max_seqlen), static_cast<size_t>(max_seqlen)});
    output_s->data.dptr = const_cast<void *>(softmax_aux.data());
    fwd_rng_state->data.shape = std::vector<size_t>({2});
    fwd_rng_state->data.dptr = const_cast<void *>(rng_state.data());

    // create cu_seqlens tensorwrappers
    TensorWrapper te_cu_seqlens;
    te_cu_seqlens = MakeNvteTensor(cu_seqlens.data(), {static_cast<size_t>(b + 1)}, DType::kInt32);

    // create workspace
    TensorWrapper workspace;

    // populate tensors with appropriate shapes and dtypes
    nvte_fused_attn_bwd_qkvpacked(
        te_QKV.data(), te_O.data(), te_dO.data(), te_S.data(), te_dP.data(), &nvte_aux_tensor_pack,
        te_dQKV.data(), te_dBias.data(), te_cu_seqlens.data(), max_seqlen, attn_scale, p_dropout,
        qkv_layout_enum, bias_type_enum, attn_mask_type_enum, workspace.data(), QKV.stream());

    // allocate memory for workspace
    auto workspace_data = AllocateSpace(workspace.shape(), workspace.dtype(), QKV.place());
    workspace = MakeNvteTensor(workspace_data.data(), workspace.shape(), workspace.dtype());

    // execute kernel
    nvte_fused_attn_bwd_qkvpacked(
        te_QKV.data(), te_O.data(), te_dO.data(), te_S.data(), te_dP.data(), &nvte_aux_tensor_pack,
        te_dQKV.data(), te_dBias.data(), te_cu_seqlens.data(), max_seqlen, attn_scale, p_dropout,
        qkv_layout_enum, bias_type_enum, attn_mask_type_enum, workspace.data(), QKV.stream());

    // destroy tensor wrappers
    nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);
}

void te_fused_attn_fwd_kvpacked(const paddle::Tensor &Q, const paddle::Tensor &KV,
                                const paddle::Tensor &cu_seqlens_q,
                                const paddle::Tensor &cu_seqlens_kv,
                                const paddle::optional<paddle::Tensor> &Bias,
                                paddle::Tensor &O,                              // NOLINT
                                paddle::optional<paddle::Tensor> &softmax_aux,  // NOLINT
                                paddle::Tensor &rng_state,                      // NOLINT
                                int64_t b, int64_t h, int64_t d, int64_t total_seqs_q,
                                int64_t total_seqs_kv, int64_t max_seqlen_q, int64_t max_seqlen_kv,
                                bool is_training, float attn_scale, float p_dropout,
                                const std::string &qkv_layout, const std::string &bias_type,
                                const std::string &attn_mask_type, const int64_t qkv_type) {
    if (is_training && !softmax_aux) {
        NVTE_ERROR("softmax_aux must be provided when training. \n");
    }

    auto qkv_dtype = Int2NvteDType(qkv_type);

    // construct NVTE tensors
    TensorWrapper te_Q, te_KV, te_S, te_O, te_Bias, te_cu_seqlens_q, te_cu_seqlens_kv;
    if (qkv_dtype == DType::kBFloat16 || qkv_dtype == DType::kFloat16) {
        // BF16 or FP16
        te_Q = MakeNvteTensor(
            Q.data(),
            {static_cast<size_t>(total_seqs_q), static_cast<size_t>(h), static_cast<size_t>(d)},
            qkv_dtype);
        te_KV = MakeNvteTensor(
            KV.data(),
            {static_cast<size_t>(total_seqs_kv), 2, static_cast<size_t>(h), static_cast<size_t>(d)},
            qkv_dtype);
        te_S = MakeNvteTensor(nullptr, std::vector<size_t>{0}, DType::kFloat32);
        te_O = MakeNvteTensor(
            O.data(),
            {static_cast<size_t>(total_seqs_q), static_cast<size_t>(h), static_cast<size_t>(d)},
            qkv_dtype);
    } else {
        NVTE_ERROR("Fused attention only supports BF16/FP16 data types. \n");
    }

    if ((bias_type != "no_bias") && Bias) {
        auto bias_shape = Bias->shape();
        std::vector<size_t> shape{bias_shape.begin(), bias_shape.end()};
        te_Bias = MakeNvteTensor(GetOptionalDataPtr(Bias), shape, DType::kFloat32);
    }

    te_cu_seqlens_q =
        MakeNvteTensor(cu_seqlens_q.data(), {static_cast<size_t>(b + 1)}, DType::kInt32);
    te_cu_seqlens_kv =
        MakeNvteTensor(cu_seqlens_kv.data(), {static_cast<size_t>(b + 1)}, DType::kInt32);

    // convert strings to enums
    NVTE_QKV_Layout qkv_layout_enum = get_nvte_qkv_layout(qkv_layout);
    NVTE_Bias_Type bias_type_enum = get_nvte_bias_type(bias_type);
    NVTE_Mask_Type attn_mask_type_enum = get_nvte_mask_type(attn_mask_type);

    auto te_rng_state = MakeNvteTensor(rng_state);

    // create auxiliary output tensors
    NVTETensorPack nvte_aux_tensor_pack;
    nvte_tensor_pack_create(&nvte_aux_tensor_pack);

    // create workspace
    TensorWrapper workspace;

    // populate tensors with appropriate shapes and dtypes
    nvte_fused_attn_fwd_kvpacked(te_Q.data(), te_KV.data(), te_Bias.data(), te_S.data(),
                                 te_O.data(), &nvte_aux_tensor_pack, te_cu_seqlens_q.data(),
                                 te_cu_seqlens_kv.data(), te_rng_state.data(), max_seqlen_q,
                                 max_seqlen_kv, is_training, attn_scale, p_dropout, qkv_layout_enum,
                                 bias_type_enum, attn_mask_type_enum, workspace.data(), Q.stream());

    // allocate memory for workspace and auxiliary output tensors
    auto workspace_data = AllocateSpace(workspace.shape(), workspace.dtype(), Q.place());
    workspace = MakeNvteTensor(workspace_data.data(), workspace.shape(), workspace.dtype());

    auto *output_s =
        reinterpret_cast<transformer_engine::Tensor *>(nvte_aux_tensor_pack.tensors[0]);
    output_s->data.dptr = GetOptionalDataPtr(softmax_aux);

    // execute the kernel
    nvte_fused_attn_fwd_kvpacked(te_Q.data(), te_KV.data(), te_Bias.data(), te_S.data(),
                                 te_O.data(), &nvte_aux_tensor_pack, te_cu_seqlens_q.data(),
                                 te_cu_seqlens_kv.data(), te_rng_state.data(), max_seqlen_q,
                                 max_seqlen_kv, is_training, attn_scale, p_dropout, qkv_layout_enum,
                                 bias_type_enum, attn_mask_type_enum, workspace.data(), Q.stream());

    // destroy tensor wrappers, but not allocated memory
    nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);
}

// fused attention BWD with packed KV
void te_fused_attn_bwd_kvpacked(const paddle::Tensor &Q, const paddle::Tensor &KV,
                                const paddle::Tensor &cu_seqlens_q,
                                const paddle::Tensor &cu_seqlens_kv, const paddle::Tensor &O,
                                const paddle::Tensor &dO, const paddle::Tensor &softmax_aux,
                                paddle::Tensor &dQ,                       // NOLINT
                                paddle::Tensor &dKV,                      // NOLINT
                                paddle::optional<paddle::Tensor> &dBias,  // NOLINT
                                paddle::Tensor &rng_state,                // NOLINT
                                int64_t b, int64_t h, int64_t d, int64_t total_seqs_q,
                                int64_t total_seqs_kv, int64_t max_seqlen_q, int64_t max_seqlen_kv,
                                float attn_scale, float p_dropout, const std::string &qkv_layout,
                                const std::string &bias_type, const std::string &attn_mask_type,
                                int64_t qkv_type) {
    TensorWrapper te_dBias;
    if (bias_type != "no_bias" && dBias) {
        auto bias_shape = dBias->shape();
        std::vector<size_t> shape{bias_shape.begin(), bias_shape.end()};
        te_dBias = MakeNvteTensor(GetOptionalDataPtr(dBias), shape, DType::kFloat32);
    }

    auto qkv_dtype = Int2NvteDType(qkv_type);
    // construct NVTE tensors
    TensorWrapper te_Q, te_KV, te_O, te_dO, te_S, te_dP, te_dQ, te_dKV;
    if (qkv_dtype == DType::kBFloat16 || qkv_dtype == DType::kFloat16) {
        // BF16 or FP16
        te_Q = MakeNvteTensor(Q);
        te_KV = MakeNvteTensor(KV);
        te_O = MakeNvteTensor(O);
        te_dO = MakeNvteTensor(dO);
        te_S = MakeNvteTensor(nullptr, std::vector<size_t>(0), DType::kFloat32);
        te_dP = MakeNvteTensor(nullptr, std::vector<size_t>(0), DType::kFloat32);
        te_dQ = MakeNvteTensor(dQ);
        te_dKV = MakeNvteTensor(dKV);
    } else {
        NVTE_ERROR("Fused attention only supports BF16/FP16 data types. \n");
    }

    // convert strings to enums
    NVTE_QKV_Layout qkv_layout_enum = get_nvte_qkv_layout(qkv_layout);
    NVTE_Bias_Type bias_type_enum = get_nvte_bias_type(bias_type);
    NVTE_Mask_Type attn_mask_type_enum = get_nvte_mask_type(attn_mask_type);

    // convert auxiliary tensors from forward into NVTETensors
    NVTETensorPack nvte_aux_tensor_pack;
    nvte_tensor_pack_create(&nvte_aux_tensor_pack);

    nvte_aux_tensor_pack.size = 2;
    auto *output_s = reinterpret_cast<Tensor *>(nvte_aux_tensor_pack.tensors[0]);
    auto *fwd_rng_state = reinterpret_cast<Tensor *>(nvte_aux_tensor_pack.tensors[1]);
    output_s->data.shape = std::vector<size_t>({static_cast<size_t>(b), static_cast<size_t>(h),
                                                static_cast<size_t>(max_seqlen_q),
                                                static_cast<size_t>(max_seqlen_kv)});
    output_s->data.dptr = const_cast<void *>(softmax_aux.data());
    fwd_rng_state->data.shape = std::vector<size_t>({2});
    fwd_rng_state->data.dptr = const_cast<void *>(rng_state.data());

    // create cu_seqlens tensorwrappers
    TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
    te_cu_seqlens_q =
        MakeNvteTensor(cu_seqlens_q.data(), {static_cast<size_t>(b + 1)}, DType::kInt32);
    te_cu_seqlens_kv =
        MakeNvteTensor(cu_seqlens_kv.data(), {static_cast<size_t>(b + 1)}, DType::kInt32);

    // create workspace
    TensorWrapper workspace;

    // populate tensors with appropriate shapes and dtypes
    nvte_fused_attn_bwd_kvpacked(
        te_Q.data(), te_KV.data(), te_O.data(), te_dO.data(), te_S.data(), te_dP.data(),
        &nvte_aux_tensor_pack, te_dQ.data(), te_dKV.data(), te_dBias.data(), te_cu_seqlens_q.data(),
        te_cu_seqlens_kv.data(), max_seqlen_q, max_seqlen_kv, attn_scale, p_dropout,
        qkv_layout_enum, bias_type_enum, attn_mask_type_enum, workspace.data(), Q.stream());

    // allocate memory for workspace
    auto workspace_data = AllocateSpace(workspace.shape(), workspace.dtype(), Q.place());
    workspace = MakeNvteTensor(workspace_data.data(), workspace.shape(), workspace.dtype());

    // execute kernel
    nvte_fused_attn_bwd_kvpacked(
        te_Q.data(), te_KV.data(), te_O.data(), te_dO.data(), te_S.data(), te_dP.data(),
        &nvte_aux_tensor_pack, te_dQ.data(), te_dKV.data(), te_dBias.data(), te_cu_seqlens_q.data(),
        te_cu_seqlens_kv.data(), max_seqlen_q, max_seqlen_kv, attn_scale, p_dropout,
        qkv_layout_enum, bias_type_enum, attn_mask_type_enum, workspace.data(), Q.stream());

    // destroy tensor wrappers
    nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);
}

std::vector<paddle::Tensor> te_scaled_softmax_forward(const paddle::Tensor &input,
                                                      float scale_factor) {
    NVTE_CHECK(input.shape().size() == 4, "expected 4D tensor");
    NVTE_CHECK((input.dtype() == paddle::DataType::FLOAT16) ||
                   (input.dtype() == paddle::DataType::BFLOAT16),
               "Only fp16 and bf16 are supported");

    const int batches = input.shape()[0];
    const int attn_heads = input.shape()[1];
    const int query_seq_len = input.shape()[2];
    const int key_seq_len = input.shape()[3];

    NVTE_CHECK(key_seq_len <= 4096);
    NVTE_CHECK(query_seq_len > 1);

    // Output
    auto softmax_results = paddle::empty_like(input, input.dtype(), input.place());

    auto input_cu = MakeNvteTensor(input);
    auto softmax_results_cu = MakeNvteTensor(softmax_results);

    nvte_scaled_softmax_forward(input_cu.data(), softmax_results_cu.data(), scale_factor,
                                input.stream());

    return {softmax_results};
}

void te_scaled_softmax_backward(paddle::Tensor &output_grads,  // NOLINT
                                const paddle::Tensor &softmax_results, float scale_factor) {
    NVTE_CHECK(output_grads.shape().size() == 4, "expected 4D tensor");
    NVTE_CHECK(softmax_results.shape().size() == 4, "expected 4D tensor");

    NVTE_CHECK((output_grads.dtype() == paddle::DataType::FLOAT16) ||
                   (output_grads.dtype() == paddle::DataType::BFLOAT16),
               "Only fp16 and bf16 are supported");
    NVTE_CHECK((softmax_results.dtype() == paddle::DataType::FLOAT16) ||
                   (softmax_results.dtype() == paddle::DataType::BFLOAT16),
               "Only fp16 and bf16 are supported");

    auto output_grads_cu = MakeNvteTensor(output_grads);
    auto softmax_results_cu = MakeNvteTensor(softmax_results);

    // Produce gradients in place.
    nvte_scaled_softmax_backward(output_grads_cu.data(), softmax_results_cu.data(),
                                 output_grads_cu.data(), scale_factor, softmax_results.stream());
}

std::vector<paddle::Tensor> te_scaled_masked_softmax_forward(const paddle::Tensor &input,
                                                             const paddle::Tensor &mask,
                                                             float scale_factor) {
    NVTE_CHECK(input.shape().size() == 4, "expected 4D tensor");
    NVTE_CHECK(mask.shape().size() == 4, "expected 4D tensor");
    NVTE_CHECK((input.dtype() == paddle::DataType::FLOAT16) ||
                   (input.dtype() == paddle::DataType::BFLOAT16),
               "Only fp16 and bf16 are supported");

    const int batches = input.shape()[0];
    const int pad_batches = mask.shape()[0];
    const int attn_heads = input.shape()[1];
    const int query_seq_len = input.shape()[2];
    const int key_seq_len = input.shape()[3];

    NVTE_CHECK(key_seq_len <= 4096);
    NVTE_CHECK(query_seq_len > 1);
    NVTE_CHECK(pad_batches == 1 || pad_batches == batches);
    NVTE_CHECK(mask.shape()[1] == 1);
    NVTE_CHECK(mask.shape()[2] == query_seq_len);
    NVTE_CHECK(mask.shape()[3] == key_seq_len);

    // Output
    auto softmax_results = paddle::empty_like(input, input.dtype(), input.place());

    auto input_cu = MakeNvteTensor(input);
    auto mask_cu = MakeNvteTensor(mask);
    auto softmax_results_cu = MakeNvteTensor(softmax_results);

    nvte_scaled_masked_softmax_forward(input_cu.data(), mask_cu.data(), softmax_results_cu.data(),
                                       scale_factor, input.stream());

    return {softmax_results};
}

void te_scaled_masked_softmax_backward(paddle::Tensor &output_grads,  // NOLINT
                                       const paddle::Tensor &softmax_results, float scale_factor) {
    NVTE_CHECK(output_grads.shape().size() == 4, "expected 4D tensor");
    NVTE_CHECK(softmax_results.shape().size() == 4, "expected 4D tensor");

    NVTE_CHECK((output_grads.dtype() == paddle::DataType::FLOAT16) ||
                   (output_grads.dtype() == paddle::DataType::BFLOAT16),
               "Only fp16 and bf16 are supported");
    NVTE_CHECK((softmax_results.dtype() == paddle::DataType::FLOAT16) ||
                   (softmax_results.dtype() == paddle::DataType::BFLOAT16),
               "Only fp16 and bf16 are supported");

    auto output_grads_cu = MakeNvteTensor(output_grads);
    auto softmax_results_cu = MakeNvteTensor(softmax_results);

    // Produce gradients in place.
    nvte_scaled_softmax_backward(output_grads_cu.data(), softmax_results_cu.data(),
                                 output_grads_cu.data(), scale_factor, softmax_results.stream());
}

std::vector<paddle::Tensor> te_scaled_upper_triang_masked_softmax_forward(
    const paddle::Tensor &input, float scale_factor) {
    NVTE_CHECK(input.shape().size() == 3, "expected 3D tensor");
    NVTE_CHECK((input.dtype() == paddle::DataType::FLOAT16) ||
                   (input.dtype() == paddle::DataType::BFLOAT16),
               "Only fp16 and bf16 are supported");

    const int attn_batches = input.shape()[0];
    const int seq_len = input.shape()[1];
    NVTE_CHECK(seq_len <= 2048);

    // Output
    auto softmax_results = paddle::empty_like(input, input.dtype(), input.place());

    auto input_cu = MakeNvteTensor(input);
    auto softmax_results_cu = MakeNvteTensor(softmax_results);

    nvte_scaled_upper_triang_masked_softmax_forward(input_cu.data(), softmax_results_cu.data(),
                                                    scale_factor, input.stream());

    return {softmax_results};
}

void te_scaled_upper_triang_masked_softmax_backward(paddle::Tensor &output_grads,  // NOLINT
                                                    const paddle::Tensor &softmax_results,
                                                    float scale_factor) {
    NVTE_CHECK(output_grads.shape().size() == 3, "expected 3D tensor");
    NVTE_CHECK(softmax_results.shape().size() == 3, "expected 3D tensor");

    NVTE_CHECK((output_grads.dtype() == paddle::DataType::FLOAT16) ||
                   (output_grads.dtype() == paddle::DataType::BFLOAT16),
               "Only fp16 and bf16 are supported");
    NVTE_CHECK((softmax_results.dtype() == paddle::DataType::FLOAT16) ||
                   (softmax_results.dtype() == paddle::DataType::BFLOAT16),
               "Only fp16 and bf16 are supported");
    NVTE_CHECK(output_grads.shape()[1] == output_grads.shape()[2]);

    auto output_grads_cu = MakeNvteTensor(output_grads);
    auto softmax_results_cu = MakeNvteTensor(softmax_results);

    // Produce gradients in place.
    nvte_scaled_upper_triang_masked_softmax_backward(
        output_grads_cu.data(), softmax_results_cu.data(), output_grads_cu.data(), scale_factor,
        softmax_results.stream());
}

__global__ void UpdateFP8MetaKernel(const float *amax, const float *rolled_amax_history,
                                    float *amax_history, float *scale, float *scale_inv,
                                    float margin, float fp8_max, size_t history_numel,
                                    size_t amax_numel) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= history_numel) {
        return;
    }

    amax_history[idx] = rolled_amax_history[idx];

    if (idx < amax_numel) {
        float exp = floor(log2(fp8_max / amax[idx])) - margin;
        float sf = round(powf(2.0f, abs(exp)));
        float scale_reg = scale[idx];
        sf = ((amax[idx] > 0.0f) && isfinite(amax[idx])) ? sf : scale_reg;
        scale_reg = exp < 0.0f ? 1 / sf : sf;
        scale[idx] = scale_reg;
        scale_inv[idx] = 1.0f / scale_reg;
        amax_history[idx] = 0.0f;
    }
}

void amax_and_scale_update_inplace(paddle::Tensor &amax_history,  // NOLINT
                                   paddle::Tensor &scale,         // NOLINT
                                   paddle::Tensor &scale_inv,     // NOLINT
                                   float fp8_max, float margin, const std::string &amax_compute) {
    NVTE_CHECK(amax_compute == "max" || amax_compute == "most_recent");

    paddle::Tensor amax;

    if (amax_compute == "max") {
        amax = amax_history.max({0});
    } else {
        amax = amax_history.slice(0, 1);
    }

    const auto rolled_amax_history = amax_history.roll({-1}, {0});

    auto size = amax_history.numel();
    constexpr int BLOCK_SIZE = 256;
    size_t num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    UpdateFP8MetaKernel<<<num_blocks, BLOCK_SIZE, 0, amax_history.stream()>>>(
        amax.data<float>(), rolled_amax_history.data<float>(), amax_history.data<float>(),
        scale.data<float>(), scale_inv.data<float>(), margin, fp8_max, amax_history.numel(),
        amax.numel());
    NVTE_CHECK_CUDA(hipGetLastError());
}

void update_latest_amax_history_inplace(paddle::Tensor &history,  // NOLINT
                                        const paddle::Tensor &amax) {
    // Copy amax to history[0]
    NVTE_CHECK_CUDA(hipMemcpyAsync(history.data(), amax.data(),
                                    amax.numel() * SizeOf(amax.dtype()), hipMemcpyDeviceToDevice,
                                    amax.stream()));
}

}  // namespace paddle_ext
}  // namespace transformer_engine

PD_BUILD_OP(te_gemm)
    .Inputs({"A", paddle::Optional("A_scale_inverse"), "B", paddle::Optional("B_scale_inverse"),
             paddle::Optional("bias"), "_D", paddle::Optional("_D_scale"),
             paddle::Optional("_D_amax"), paddle::Optional("_pre_gelu_out"), "_workspace"})
    .Outputs({"D", paddle::Optional("D_scale"), paddle::Optional("D_amax"),
              paddle::Optional("pre_gelu_out"), "workspace"})
    .Attrs({"A_index: int64_t", "B_index: int64_t", "D_index: int64_t", "A_type: int64_t",
            "B_type: int64_t", "D_type: int64_t", "bias_type: int64_t", "transa: bool",
            "transb: bool", "grad: bool", "workspace_size: int64_t", "accumulate: bool",
            "use_split_accumulator: bool", "math_sm_count: int64_t"})
    .SetInplaceMap({{"_D", "D"},
                    {paddle::Optional("_D_scale"), paddle::Optional("D_scale")},
                    {paddle::Optional("_D_amax"), paddle::Optional("D_amax")},
                    {paddle::Optional("_pre_gelu_out"), paddle::Optional("pre_gelu_out")},
                    {"_workspace", "workspace"}})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_gemm));

PD_BUILD_OP(cast_to_fp8)
    .Inputs({"Input", "Scale", "_Amax", "_ScaleInv"})
    .Outputs({"Output", "Amax", "ScaleInv"})
    .Attrs({"index: int64_t", "otype: int64_t"})
    .SetInplaceMap({{"_Amax", "Amax"}, {"_ScaleInv", "ScaleInv"}})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::cast_to_fp8));

PD_BUILD_OP(cast_from_fp8)
    .Inputs({"Input", "ScaleInv"})
    .Outputs({"Output"})
    .Attrs({"index: int64_t", "itype: int64_t", "otype: int64_t"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::cast_from_fp8));

PD_BUILD_OP(te_transpose)
    .Inputs({"Input"})
    .Outputs({"Output"})
    .Attrs({"otype: int64_t"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_transpose));

PD_BUILD_OP(te_cast_transpose)
    .Inputs({"Input", "Scale", "_Amax", "_ScaleInv"})
    .Outputs({"CastedOutput", "TransposedOutput", "Amax", "ScaleInv"})
    .SetInplaceMap({{"_Amax", "Amax"}, {"_ScaleInv", "ScaleInv"}})
    .Attrs({"index: int64_t", "otype: int64_t"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_cast_transpose));

PD_BUILD_OP(te_cast_transpose_bgrad)
    .Inputs({"GradOutput", "Scale", "_Amax", "_ScaleInv"})
    .Outputs({"dBias", "CastedOutput", "TransposedOutput", "Amax", "ScaleInv"})
    .SetInplaceMap({{"_Amax", "Amax"}, {"_ScaleInv", "ScaleInv"}})
    .Attrs({"index: int64_t", "otype: int64_t"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_cast_transpose_bgrad));

PD_BUILD_OP(te_gelu_fp8)
    .Inputs({"Input", "Scale", "_Amax", "_ScaleInv"})
    .Outputs({"Output", "Amax", "ScaleInv"})
    .SetInplaceMap({{"_Amax", "Amax"}, {"_ScaleInv", "ScaleInv"}})
    .Attrs({"index: int64_t", "otype: int64_t"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_gelu_fp8));

PD_BUILD_OP(te_gelu)
    .Inputs({"Input"})
    .Outputs({"Output"})
    .Attrs({"otype: int64_t"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_gelu));

PD_BUILD_OP(te_cast_transpose_bgrad_dgelu)
    .Inputs({"GradOutput", "GeluInput", "Scale", "_Amax", "_ScaleInv"})
    .Outputs({"CastedDgelu", "TransposedDgelu", "Dbias", "Amax", "ScaleInv"})
    .SetInplaceMap({{"_Amax", "Amax"}, {"_ScaleInv", "ScaleInv"}})
    .Attrs({"index: int64_t", "otype: int64_t"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_cast_transpose_bgrad_dgelu));

PD_BUILD_OP(te_layernorm_fwd_fp8)
    .Inputs({"Input", "Weight", "Bias", "Scale", "_Amax", "_ScaleInv"})
    .Outputs({"Output", "Mu", "Rsigma", "Amax", "ScaleInv"})
    .SetInplaceMap({{"_Amax", "Amax"}, {"_ScaleInv", "ScaleInv"}})
    .Attrs({"eps: float", "index: int64_t", "otype: int64_t", "sm_margin: int64_t",
            "zero_centered_gamma: bool"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_layernorm_fwd_fp8));

PD_BUILD_OP(te_layernorm_fwd)
    .Inputs({"Input", "Weight", "Bias"})
    .Outputs({"Output", "Mu", "Rsigma"})
    .Attrs({"eps: float", "otype: int64_t", "sm_margin: int64_t", "zero_centered_gamma: bool"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_layernorm_fwd));

PD_BUILD_OP(te_layernorm_bwd)
    .Inputs({"Dz", "X", "Mu", "Rsigma", "Gamma"})
    .Outputs({"Dx", "Dgamma", "Dbeta"})
    .Attrs({"sm_margin: int64_t", "zero_centered_gamma: bool"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_layernorm_bwd));

PD_BUILD_OP(te_rmsnorm_fwd)
    .Inputs({"Input", "Weight"})
    .Outputs({"Output", "InvVariance"})
    .Attrs({"eps: float", "otype: int64_t", "sm_margin: int64_t"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_rmsnorm_fwd));

PD_BUILD_OP(te_rmsnorm_fwd_fp8)
    .Inputs({"Input", "Weight", "Scale", "_Amax", "_ScaleInv"})
    .Outputs({"Output", "InvVariance", "Amax", "ScaleInv"})
    .SetInplaceMap({{"_Amax", "Amax"}, {"_ScaleInv", "ScaleInv"}})
    .Attrs({"eps: float", "index: int64_t", "otype: int64_t", "sm_margin: int64_t"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_rmsnorm_fwd_fp8));

PD_BUILD_OP(te_rmsnorm_bwd)
    .Inputs({"Dz", "X", "Rsigma", "Gamma"})
    .Outputs({"Dx", "Dgamma"})
    .Attrs({"sm_margin: int64_t"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_rmsnorm_bwd));

PD_BUILD_OP(te_fused_attn_fwd_qkvpacked)
    .Inputs({"QKV", "cu_seqlens", paddle::Optional("Bias"), "_O", paddle::Optional("_softmax_aux"),
             "rng_state"})
    .Outputs({"O", paddle::Optional("softmax_aux")})
    .Attrs({"b: int64_t", "h: int64_t", "d: int64_t", "total_seqs: int64_t", "max_seqlen: int64_t",
            "is_training: bool", "attn_scale: float", "p_dropout: float", "qkv_layout: std::string",
            "bias_type: std::string", "attn_mask_type: std::string", "qkv_type: int64_t"})
    .SetInplaceMap({{"_O", "O"},
                    {paddle::Optional("_softmax_aux"), paddle::Optional("softmax_aux")}})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_fused_attn_fwd_qkvpacked));

PD_BUILD_OP(te_fused_attn_bwd_qkvpacked)
    .Inputs({"QKV", "cu_seqlens", "O", "dO", "softmax_aux", "_dQKV", paddle::Optional("_dBias"),
             "rng_state"})
    .Outputs({"dQKV", paddle::Optional("dBias")})
    .Attrs({"b: int64_t", "h: int64_t", "d: int64_t", "total_seqs: int64_t", "max_seqlen: int64_t",
            "attn_scale: float", "p_dropout: float", "qkv_layout: std::string",
            "bias_type: std::string", "attn_mask_type: std::string", "qkv_type: int64_t"})
    .SetInplaceMap({{"_dQKV", "dQKV"}, {paddle::Optional("_dBias"), paddle::Optional("dBias")}})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_fused_attn_bwd_qkvpacked));

PD_BUILD_OP(te_fused_attn_fwd_kvpacked)
    .Inputs({"Q", "KV", "cu_seqlens_q", "cu_seqlens_kv", paddle::Optional("Bias"), "_O",
             paddle::Optional("_softmax_aux"), "rng_state"})
    .Outputs({"O", paddle::Optional("softmax_aux")})
    .Attrs({"b: int64_t", "h: int64_t", "d: int64_t", "total_seqs_q: int64_t",
            "total_seqs_kv: int64_t", "max_seqlen_q: int64_t", "max_seqlen_kv: int64_t",
            "is_training: bool", "attn_scale: float", "p_dropout: float", "qkv_layout: std::string",
            "bias_type: std::string", "attn_mask_type: std::string", "qkv_type: int64_t"})
    .SetInplaceMap({{"_O", "O"},
                    {paddle::Optional("_softmax_aux"), paddle::Optional("softmax_aux")}})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_fused_attn_fwd_kvpacked));

PD_BUILD_OP(te_fused_attn_bwd_kvpacked)
    .Inputs({"Q", "KV", "cu_seqlens_q", "cu_seqlens_kv", "O", "dO", "softmax_aux", "_dQ", "_dKV",
             paddle::Optional("_dBias"), "rng_state"})
    .Outputs({"dQ", "dKV", paddle::Optional("dBias")})
    .Attrs({"b: int64_t", "h: int64_t", "d: int64_t", "total_seqs_q: int64_t",
            "total_seqs_kv: int64_t", "max_seqlen_q: int64_t", "max_seqlen_kv: int64_t",
            "attn_scale: float", "p_dropout: float", "qkv_layout: std::string",
            "bias_type: std::string", "attn_mask_type: std::string", "qkv_type: int64_t"})
    .SetInplaceMap({{"_dQ", "dQ"},
                    {"_dKV", "dKV"},
                    {paddle::Optional("_dBias"), paddle::Optional("dBias")}})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_fused_attn_bwd_kvpacked));

PD_BUILD_OP(te_scaled_softmax_forward)
    .Inputs({"input"})
    .Outputs({"softmax_results"})
    .Attrs({"scale_factor: float"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_scaled_softmax_forward));

PD_BUILD_OP(te_scaled_softmax_backward)
    .Inputs({"out_grad_", "softmax_results"})
    .Outputs({"out_grad"})
    .Attrs({"scale_factor: float"})
    .SetInplaceMap({{"out_grad_", "out_grad"}})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_scaled_softmax_backward));

PD_BUILD_OP(te_scaled_masked_softmax_forward)
    .Inputs({"input", "mask"})
    .Outputs({"softmax_results"})
    .Attrs({"scale_factor: float"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_scaled_masked_softmax_forward));

PD_BUILD_OP(te_scaled_masked_softmax_backward)
    .Inputs({"out_grad_", "softmax_results"})
    .Outputs({"out_grad"})
    .Attrs({"scale_factor: float"})
    .SetInplaceMap({{"out_grad_", "out_grad"}})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::te_scaled_masked_softmax_backward));

PD_BUILD_OP(te_scaled_upper_triang_masked_softmax_forward)
    .Inputs({"input"})
    .Outputs({"softmax_results"})
    .Attrs({"scale_factor: float"})
    .SetKernelFn(
        PD_KERNEL(transformer_engine::paddle_ext::te_scaled_upper_triang_masked_softmax_forward));

PD_BUILD_OP(te_scaled_upper_triang_masked_softmax_backward)
    .Inputs({"out_grad_", "softmax_results"})
    .Outputs({"out_grad"})
    .Attrs({"scale_factor: float"})
    .SetInplaceMap({{"out_grad_", "out_grad"}})
    .SetKernelFn(
        PD_KERNEL(transformer_engine::paddle_ext::te_scaled_upper_triang_masked_softmax_backward));

PD_BUILD_OP(amax_and_scale_update_inplace)
    .Inputs({"_amax_history", "_scale", "_scale_inv"})
    .Outputs({"amax_history", "scale", "scale_inv"})
    .SetInplaceMap({{"_amax_history", "amax_history"},
                    {"_scale", "scale"},
                    {"_scale_inv", "scale_inv"}})
    .Attrs({"fp8_max: float", "margin: float", "amax_compute: std::string"})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::amax_and_scale_update_inplace));

PD_BUILD_OP(update_latest_amax_history_inplace)
    .Inputs({"_history", "amax"})
    .Outputs({"history"})
    .SetInplaceMap({{"_history", "history"}})
    .SetKernelFn(PD_KERNEL(transformer_engine::paddle_ext::update_latest_amax_history_inplace));
