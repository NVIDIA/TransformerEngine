#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "extensions.h"
#ifdef NVTE_WITH_USERBUFFERS
#include "comm_gemm_overlap.h"
#endif  // NVTE_WITH_USERBUFFERS

constexpr int block_size = 512;
constexpr int ctas_per_sm = 4;

// convert QKV layout to enum
NVTE_QKV_Layout get_nvte_qkv_layout(const std::string qkv_layout) {
  if (qkv_layout == "not_interleaved") {
      return NVTE_QKV_Layout::NVTE_NOT_INTERLEAVED;
  } else if (qkv_layout == "qkv_interleaved") {
      return NVTE_QKV_Layout::NVTE_QKV_INTERLEAVED;
  } else if (qkv_layout == "kv_interleaved") {
      return NVTE_QKV_Layout::NVTE_KV_INTERLEAVED;
  } else {
      NVTE_ERROR("Invalid QKV layout. \n");
  }
}

// convert bias type to enum
NVTE_Bias_Type get_nvte_bias_type(const std::string bias_type) {
  if (bias_type == "no_bias") {
      return NVTE_Bias_Type::NVTE_NO_BIAS;
  } else if (bias_type == "pre_scale_bias") {
      return NVTE_Bias_Type::NVTE_PRE_SCALE_BIAS;
  } else if (bias_type == "post_scale_bias") {
      return NVTE_Bias_Type::NVTE_POST_SCALE_BIAS;
  } else {
      NVTE_ERROR("Invalid bias type. \n");
  }
}

// convert attn mask type to enum
NVTE_Mask_Type get_nvte_mask_type(const std::string mask_type) {
  if (mask_type == "padding") {
      return NVTE_Mask_Type::NVTE_PADDING_MASK;
  } else if (mask_type == "causal") {
      return NVTE_Mask_Type::NVTE_CAUSAL_MASK;
  } else if (mask_type == "no_mask") {
      return NVTE_Mask_Type::NVTE_NO_MASK;
  } else {
      NVTE_ERROR("Invalid attention mask type. \n");
  }
}

// fast zero-fills of tensors
template <typename scalar_t>
__global__ void __launch_bounds__(block_size) mha_fill_kernel(scalar_t* out_tensor,
                const int32_t* const start_row,
                const size_t num_rows) {
  size_t row_stride = gridDim.y * blockDim.x;
  size_t row_index = blockIdx.x + static_cast<size_t>(start_row[0]);
  size_t col_index = blockIdx.y * blockDim.x + threadIdx.x;
  while (row_index < num_rows) {
    out_tensor[row_index*row_stride + col_index] = 0;
    row_index += gridDim.x;
  }
}

// fast zero-fills of tensors
void mha_fill(const at::Tensor &self, const at::Tensor &start_index) {
  auto max_tokens = self.size(0);
  auto self_2d = self.view({max_tokens, -1});
  auto fcd_size = self_2d.size(1);
  TORCH_CHECK(self.is_contiguous(), "input not contiguous");
  TORCH_CHECK(fcd_size % block_size == 0, "input size not aligned to block size");
  const int num_mp = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  uint64_t num_blk_y = (uint64_t)(fcd_size / block_size);
  uint64_t num_blk_x = (uint64_t)((num_mp * ctas_per_sm + num_blk_y - 1) / num_blk_y);
  dim3 dim_grid(num_blk_x, num_blk_y);
  dim3 dim_block(block_size);
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
          at::ScalarType::Half, at::ScalarType::BFloat16,
          self_2d.scalar_type(), "mha_fill", [&]() {
          mha_fill_kernel<<<dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                  self_2d.data_ptr<scalar_t>(),
                  static_cast<int32_t*>(start_index.data_ptr()),
                  max_tokens);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
          });
}

// extract seed and offset from PhiloxCudaState
__global__ void unpack(at::PhiloxCudaState arg, int64_t* rng_state_ptr) {
  if (arg.captured_) {
    rng_state_ptr[0] = static_cast<int64_t>(*arg.seed_.ptr);
    rng_state_ptr[1] = static_cast<int64_t>(
                    *(arg.offset_.ptr) + static_cast<int64_t>(arg.offset_intragraph_));
  } else {
    rng_state_ptr[0] = static_cast<int64_t>(arg.seed_.val);
    rng_state_ptr[1] = static_cast<int64_t>(arg.offset_.val);
  }
}

// extract PhiloxCudaState from CUDA random number generator
at::PhiloxCudaState init_philox_state(
                at::CUDAGeneratorImpl* gen,
                size_t max_seq_len,
                size_t threads_per_cta) {
  at::PhiloxCudaState philox_args;
  size_t elts_per_thread = (max_seq_len * max_seq_len + threads_per_cta - 1)/threads_per_cta;
  std::lock_guard<std::mutex> lock(gen->mutex_);
  philox_args = gen->philox_cuda_state(elts_per_thread);
  return philox_args;
}

// fused attention FWD with packed QKV
std::vector<at::Tensor> fused_attn_fwd_qkvpacked(
                size_t b, size_t max_seqlen, size_t total_seqs,
                size_t h, size_t d,
                bool is_training, float attn_scale, float p_dropout, bool set_zero,
                std::string qkv_layout, std::string bias_type, std::string attn_mask_type,
                const at::Tensor cu_seqlens,
                const at::Tensor QKV,
                const transformer_engine::DType qkv_type,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_O,
                c10::optional<at::Tensor> amax_S,
                c10::optional<at::Tensor> amax_O,
                const c10::optional<at::Tensor> Bias,
                const c10::optional<at::Generator> rng_gen) {
  using namespace transformer_engine;

  // create output tensor O
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  auto O = torch::empty({static_cast<int64_t>(total_seqs),
                  static_cast<int64_t>(h), static_cast<int64_t>(d)}, options);
  if (set_zero) {
    mha_fill(O, cu_seqlens.index({torch::indexing::Slice(-1, torch::indexing::None)}));
  }

  // construct NVTE tensors
  TensorWrapper te_QKV, te_S, te_O, te_Bias, te_cu_seqlens;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if ((!descale_QKV.has_value()) || (!scale_S.has_value()) || (!scale_O.has_value())
                    || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, scale_S, scale_O, amax_S and amax_O";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), {total_seqs, 3, h, d},
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    at::Tensor descale_S = torch::empty_like(scale_S.value());
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_S.value().data_ptr(),
                    scale_S.value().data_ptr(), descale_S.data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs, h, d},
                    qkv_type, amax_O.value().data_ptr(), scale_O.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), {total_seqs, 3, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if (Bias.has_value()) {
    auto bias_shape = Bias.value().sizes().vec();
    std::vector<size_t> shape{bias_shape.begin(), bias_shape.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), shape,
                    DType::kFloat32, nullptr, nullptr, nullptr);
  }
  te_cu_seqlens = makeTransformerEngineTensor(cu_seqlens.data_ptr(), {b+1},
                    DType::kInt32, nullptr, nullptr, nullptr);

  // convert strings to enums
  NVTE_QKV_Layout qkv_layout_enum = get_nvte_qkv_layout(qkv_layout);
  NVTE_Bias_Type bias_type_enum = get_nvte_bias_type(bias_type);
  NVTE_Mask_Type attn_mask_type_enum = get_nvte_mask_type(attn_mask_type);

  // extract random number generator seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
                  rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  size_t threads_per_cta = 128;
  at::PhiloxCudaState philox_args = init_philox_state(gen, max_seqlen, threads_per_cta);
  auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
                  philox_args, static_cast<int64_t*>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  // if training, tensors are [M, ZInv]
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd_qkvpacked(
                  te_QKV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens.data(),
                  te_rng_state.data(),
                  max_seqlen,
                  is_training, attn_scale, p_dropout,
                  qkv_layout_enum, bias_type_enum, attn_mask_type_enum,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors, rng_state]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  // nvte_aux_tensor_pack.size is 0 if inference
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    auto output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }
  if (is_training) {
    output_tensors.push_back(rng_state);
  }

  // execute the kernel
  nvte_fused_attn_fwd_qkvpacked(
                  te_QKV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens.data(),
                  te_rng_state.data(),
                  max_seqlen,
                  is_training, attn_scale, p_dropout,
                  qkv_layout_enum, bias_type_enum, attn_mask_type_enum,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, M, ZInv, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with packed QKV
std::vector<at::Tensor> fused_attn_bwd_qkvpacked(
                size_t b, size_t max_seqlen, size_t total_seqs,
                size_t h, size_t d,
                float attn_scale, float p_dropout, bool set_zero,
                std::string qkv_layout, std::string bias_type, std::string attn_mask_type,
                const at::Tensor cu_seqlens,
                const at::Tensor QKV,
                const at::Tensor O,
                const at::Tensor dO,
                const transformer_engine::DType qkv_type,
                const std::vector<at::Tensor> Aux_CTX_Tensors,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> descale_S,
                const c10::optional<at::Tensor> descale_O,
                const c10::optional<at::Tensor> descale_dO,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_dP,
                const c10::optional<at::Tensor> scale_dQKV,
                c10::optional<at::Tensor> amax_dP,
                c10::optional<at::Tensor> amax_dQKV,
                const c10::optional<at::Tensor> dBias) {
  using namespace transformer_engine;

  // create output tensor dQKV
  at::Tensor dQKV = torch::empty_like(QKV);
  if (set_zero) {
    mha_fill(dQKV, cu_seqlens.index({torch::indexing::Slice(-1, torch::indexing::None)}));
  }

  // construct NVTE tensors
  TensorWrapper te_QKV, te_O, te_dO, te_S, te_dP, te_dQKV, te_dBias;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if ((!descale_QKV.has_value()) || (!descale_S.has_value())
                    || (!descale_O.has_value()) || (!descale_dO.has_value())
                    || (!scale_S.has_value()) || (!scale_dP.has_value())
                    || (!scale_dQKV.has_value())
                    || (!amax_dP.has_value()) || (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, scale_S, scale_dP, ";
      err_tensors = err_tensors + std::string("scale_dQKV, amax_dP and amax_dQKV");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), {total_seqs, 3, h, d},
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs, h, d},
                    qkv_type, nullptr, nullptr, descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), {total_seqs, h, d},
                    qkv_type, nullptr, nullptr, descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32,
                    nullptr, scale_S.value().data_ptr(), descale_S.value().data_ptr());
    at::Tensor descale_dP = torch::empty_like(scale_dP.value());
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_dP.value().data_ptr(), scale_dP.value().data_ptr(),
                    descale_dP.data_ptr());
    te_dQKV = makeTransformerEngineTensor(dQKV.data_ptr(), {total_seqs, 3, h, d},
                    qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), {total_seqs, 3, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), {total_seqs, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQKV = makeTransformerEngineTensor(dQKV.data_ptr(), {total_seqs, 3, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if (dBias.has_value()) {
    auto bias_shape = dBias.value().sizes().vec();
    std::vector<size_t> shape{bias_shape.begin(), bias_shape.end()};
    te_dBias = makeTransformerEngineTensor(
                    dBias.value().data_ptr(), shape, DType::kFloat32,
                    nullptr, nullptr, nullptr);
  }

  // convert strings to enums
  NVTE_QKV_Layout qkv_layout_enum = get_nvte_qkv_layout(qkv_layout);
  NVTE_Bias_Type bias_type_enum = get_nvte_bias_type(bias_type);
  NVTE_Mask_Type attn_mask_type_enum = get_nvte_mask_type(attn_mask_type);

  // convert auxiliary tensors from forward into NVTETensors
  // aux_ctx_tensors are [M, ZInv, rng_state]
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create cu_seqlens tensorwrappers
  TensorWrapper te_cu_seqlens;
  te_cu_seqlens = makeTransformerEngineTensor(cu_seqlens.data_ptr(), {b+1},
                    DType::kInt32, nullptr, nullptr, nullptr);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd_qkvpacked(
                  te_QKV.data(),
                  te_dBias.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQKV.data(),
                  te_cu_seqlens.data(),
                  max_seqlen,
                  attn_scale, p_dropout,
                  qkv_layout_enum, bias_type_enum, attn_mask_type_enum,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd_qkvpacked(
                  te_QKV.data(),
                  te_dBias.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQKV.data(),
                  te_cu_seqlens.data(),
                  max_seqlen,
                  attn_scale, p_dropout,
                  qkv_layout_enum, bias_type_enum, attn_mask_type_enum,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQKV};
}

// fused attention FWD with packed KV
std::vector<at::Tensor> fused_attn_fwd_kvpacked(
                size_t b, size_t max_seqlen_q, size_t max_seqlen_kv,
                size_t total_seqs_q, size_t total_seqs_kv,
                size_t h, size_t d,
                bool is_training, float attn_scale, float p_dropout, bool set_zero,
                std::string qkv_layout, std::string bias_type, std::string attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor KV,
                const transformer_engine::DType qkv_type,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_O,
                c10::optional<at::Tensor> amax_S,
                c10::optional<at::Tensor> amax_O,
                const c10::optional<at::Tensor> Bias,
                const c10::optional<at::Generator> rng_gen) {
  using namespace transformer_engine;

  // create output tensor O
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  auto O = torch::empty({static_cast<int64_t>(total_seqs_q),
                  static_cast<int64_t>(h), static_cast<int64_t>(d)}, options);
  if (set_zero) {
    mha_fill(O, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
  }

  // construct NVTE tensors
  TensorWrapper te_Q, te_KV, te_S, te_O, te_Bias, te_cu_seqlens_q, te_cu_seqlens_kv;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if ((!descale_QKV.has_value()) || (!scale_S.has_value()) || (!scale_O.has_value())
                    || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, scale_S, scale_O, amax_S and amax_O";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), {total_seqs_kv, 2, h, d},
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    at::Tensor descale_S = torch::empty_like(scale_S.value());
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_S.value().data_ptr(),
                    scale_S.value().data_ptr(), descale_S.data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, amax_O.value().data_ptr(), scale_O.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), {total_seqs_kv, 2, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if (Bias.has_value()) {
    auto bias_shape = Bias.value().sizes().vec();
    std::vector<size_t> shape{bias_shape.begin(), bias_shape.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), shape,
                    DType::kFloat32, nullptr, nullptr, nullptr);
  }
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), {b+1},
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), {b+1},
                    DType::kInt32, nullptr, nullptr, nullptr);

  // convert strings to enums
  NVTE_QKV_Layout qkv_layout_enum = get_nvte_qkv_layout(qkv_layout);
  NVTE_Bias_Type bias_type_enum = get_nvte_bias_type(bias_type);
  NVTE_Mask_Type attn_mask_type_enum = get_nvte_mask_type(attn_mask_type);

  // extract rng seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
                  rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  size_t threads_per_cta = 128;
  at::PhiloxCudaState philox_args = init_philox_state(
                  gen, max(max_seqlen_q, max_seqlen_kv), threads_per_cta);
  auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
                  philox_args, static_cast<int64_t*>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  // if training, tensors are [M, ZInv]
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout_enum, bias_type_enum, attn_mask_type_enum,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors, rng_state]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  // nvte_aux_tensor_pack.size is 0 if inference
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    auto output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }
  if (is_training) {
    output_tensors.push_back(rng_state);
  }

  // execute the kernel
  nvte_fused_attn_fwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout_enum, bias_type_enum, attn_mask_type_enum,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, M, ZInv, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with packed KV
std::vector<at::Tensor> fused_attn_bwd_kvpacked(
                size_t b, size_t max_seqlen_q, size_t max_seqlen_kv,
                size_t total_seqs_q, size_t total_seqs_kv,
                size_t h, size_t d,
                float attn_scale, float p_dropout, bool set_zero,
                std::string qkv_layout, std::string bias_type, std::string attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor KV,
                const at::Tensor O,
                const at::Tensor dO,
                const transformer_engine::DType qkv_type,
                const std::vector<at::Tensor> Aux_CTX_Tensors,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> descale_S,
                const c10::optional<at::Tensor> descale_O,
                const c10::optional<at::Tensor> descale_dO,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_dP,
                const c10::optional<at::Tensor> scale_dQKV,
                c10::optional<at::Tensor> amax_dP,
                c10::optional<at::Tensor> amax_dQKV,
                const c10::optional<at::Tensor> dBias) {
  using namespace transformer_engine;

  // create output tensors dQ and dKV
  at::Tensor dQ = torch::empty_like(Q);
  at::Tensor dKV = torch::empty_like(KV);
  if (set_zero) {
    mha_fill(dQ, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    mha_fill(dKV, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
  }

  // construct NVTE tensors
  TensorWrapper te_Q, te_KV, te_O, te_dO, te_S, te_dP, te_dQ, te_dKV, te_dBias;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if ((!descale_QKV.has_value()) || (!descale_S.has_value())
                    || (!descale_O.has_value()) || (!descale_dO.has_value())
                    || (!scale_S.has_value()) || (!scale_dP.has_value())
                    || (!scale_dQKV.has_value())
                    || (!amax_dP.has_value()) || (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, scale_S, scale_dP, ";
      err_tensors = err_tensors + std::string("scale_dQKV, amax_dP and amax_dQKV");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), {total_seqs_kv, 2, h, d},
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr,
                    scale_S.value().data_ptr(), descale_S.value().data_ptr());
    at::Tensor descale_dP = torch::empty_like(scale_dP.value());
    te_dP = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32,
                    amax_dP.value().data_ptr(), scale_dP.value().data_ptr(),
                    descale_dP.data_ptr());
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), {total_seqs_q, h, d}, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
    te_dKV = makeTransformerEngineTensor(dKV.data_ptr(), {total_seqs_kv, 2, h, d}, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), {total_seqs_kv, 2, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_dKV = makeTransformerEngineTensor(dKV.data_ptr(), {total_seqs_kv, 2, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if (dBias.has_value()) {
    auto bias_shape = dBias.value().sizes().vec();
    std::vector<size_t> shape{bias_shape.begin(), bias_shape.end()};
    te_dBias = makeTransformerEngineTensor(
                    dBias.value().data_ptr(), shape, DType::kFloat32,
                    nullptr, nullptr, nullptr);
  }

  // create cu_seqlens tensorwrappers
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), {b+1},
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), {b+1},
                    DType::kInt32, nullptr, nullptr, nullptr);

  // convert strings to enums
  NVTE_QKV_Layout qkv_layout_enum = get_nvte_qkv_layout(qkv_layout);
  NVTE_Bias_Type bias_type_enum = get_nvte_bias_type(bias_type);
  NVTE_Mask_Type attn_mask_type_enum = get_nvte_mask_type(attn_mask_type);

  // convert auxiliary tensors from forward to NVTETensors
  // aux_ctx_tensors are [M, ZInv, rng_state]
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_dBias.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dKV.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout_enum, bias_type_enum, attn_mask_type_enum,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_dBias.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dKV.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout_enum, bias_type_enum, attn_mask_type_enum,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQ, dKV};
}

void te_gemm(at::Tensor A,
             at::Tensor A_scale_inverse,
             transformer_engine::DType A_type,
             bool transa,
             at::Tensor B,
             at::Tensor B_scale_inverse,
             transformer_engine::DType B_type,
             bool transb,
             at::Tensor D,
             at::Tensor D_scale,
             transformer_engine::DType D_type,
             at::Tensor D_amax,
             at::Tensor bias,
             transformer_engine::DType bias_type,
             at::Tensor pre_gelu_out,
             bool grad,
             at::Tensor workspace,
             size_t workspaceSize,
             bool accumulate,
             bool use_split_accumulator,
             int math_sm_count
) {
  using namespace transformer_engine;
  auto te_A = makeTransformerEngineTensor(A.data_ptr(),
                                          {static_cast<size_t>(A.size(0)),
                                           static_cast<size_t>(A.size(1))},
                                          A_type, nullptr, nullptr,
                                          A_scale_inverse.data_ptr());
  auto te_B = makeTransformerEngineTensor(B.data_ptr(),
                                          {static_cast<size_t>(B.size(0)),
                                           static_cast<size_t>(B.size(1))},
                                          B_type, nullptr, nullptr,
                                          B_scale_inverse.data_ptr());
  auto te_D = makeTransformerEngineTensor(D.data_ptr(),
                                          {static_cast<size_t>(D.size(0)),
                                           static_cast<size_t>(D.size(1))},
                                          D_type, D_amax.data_ptr(),
                                          D_scale.data_ptr(), nullptr);
  auto te_bias = makeTransformerEngineTensor(bias.data_ptr(), {static_cast<size_t>(bias.size(0))},
                                             bias_type);

  const auto gelu_shape = pre_gelu_out.data_ptr() == nullptr
                          ? std::vector<size_t>{static_cast<size_t>(pre_gelu_out.size(0))}
                          : std::vector<size_t>{static_cast<size_t>(pre_gelu_out.size(0)),
                                                static_cast<size_t>(pre_gelu_out.size(1))};
  auto te_pre_gelu_out = makeTransformerEngineTensor(pre_gelu_out.data_ptr(),
                                                     gelu_shape,
                                                     GetTransformerEngineDType(
                                                         pre_gelu_out.scalar_type()));
  auto te_workspace = makeTransformerEngineTensor(workspace.data_ptr(),
                                                  {workspaceSize},
                                                  DType::kByte);

  nvte_cublas_gemm(te_A.data(),
                   te_B.data(),
                   te_D.data(),
                   te_bias.data(),
                   te_pre_gelu_out.data(),
                   transa,
                   transb,
                   grad,
                   te_workspace.data(),
                   accumulate,
                   use_split_accumulator,
                   math_sm_count,
                   at::cuda::getCurrentCUDAStream());
}


void fused_cast_transpose(at::Tensor input,
                          at::Tensor scale,
                          at::Tensor amax,
                          at::Tensor scale_inv,
                          at::Tensor input_cast,
                          at::Tensor input_transpose,
                          transformer_engine::DType otype
) {
  using namespace transformer_engine;

  size_t M = static_cast<size_t>(input.size(0));
  size_t N = static_cast<size_t>(input.size(1));

  auto input_cu            = makeTransformerEngineTensor(input);
  auto output_cast_cu      = makeTransformerEngineTensor(input_cast.data_ptr(), {M, N}, otype,
                                                         amax.data_ptr(), scale.data_ptr(),
                                                         scale_inv.data_ptr());
  auto output_transpose_cu = makeTransformerEngineTensor(input_transpose.data_ptr(), {N, M}, otype,
                                                         amax.data_ptr(), scale.data_ptr(),
                                                         scale_inv.data_ptr());

  nvte_cast_transpose(input_cu.data(), output_cast_cu.data(), output_transpose_cu.data(),
                      at::cuda::getCurrentCUDAStream());
}


std::vector<at::Tensor> fused_cast_transpose_bgrad(at::Tensor grad_output,
                                                   at::Tensor scale,
                                                   at::Tensor amax,
                                                   at::Tensor scale_inv,
                                                   transformer_engine::DType otype
) {
  using namespace transformer_engine;

  size_t M = static_cast<size_t>(grad_output.size(0));
  size_t N = static_cast<size_t>(grad_output.size(1));

  DType grad_output_type = GetTransformerEngineDType(grad_output.scalar_type());
  auto grad_bias = allocateTorchTensor(grad_output.size(-1), grad_output_type);
  auto grad_output_cast =
            allocateTorchTensor(grad_output.size(0),
                                grad_output.size(1),
                                DType::kByte);
  auto grad_output_transpose =
            allocateTorchTensor(grad_output.size(1),
                                grad_output.size(0),
                                DType::kByte);

  auto input_cu             = makeTransformerEngineTensor(grad_output);
  auto cast_output_cu       = makeTransformerEngineTensor(grad_output_cast.data_ptr(), {M, N},
                                                          otype, amax.data_ptr(), scale.data_ptr(),
                                                          scale_inv.data_ptr());
  auto transposed_output_cu = makeTransformerEngineTensor(grad_output_transpose.data_ptr(),
                                                          {N, M}, otype, amax.data_ptr(),
                                                          scale.data_ptr(), scale_inv.data_ptr());
  auto dbias_cu             = makeTransformerEngineTensor(grad_bias);
  transformer_engine::TensorWrapper workspace;

  nvte_cast_transpose_dbias(input_cu.data(), cast_output_cu.data(),
                            transposed_output_cu.data(), dbias_cu.data(),
                            workspace.data(), at::cuda::getCurrentCUDAStream());

  // Fill workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(workspace_data.data_ptr(),
                                          workspace.shape(),
                                          workspace.dtype());

  nvte_cast_transpose_dbias(input_cu.data(), cast_output_cu.data(),
                            transposed_output_cu.data(), dbias_cu.data(),
                            workspace.data(), at::cuda::getCurrentCUDAStream());

  return {grad_bias, grad_output_cast, grad_output_transpose};
}


std::vector<at::Tensor> fused_fp8_transpose_bgrad(at::Tensor grad_output,
                                                   at::Tensor scale,
                                                   at::Tensor amax,
                                                   at::Tensor scale_inv,
                                                   transformer_engine::DType otype,
                                                   transformer_engine::DType grad_bias_type
) {
  using namespace transformer_engine;

  size_t M = static_cast<size_t>(grad_output.size(0));
  size_t N = static_cast<size_t>(grad_output.size(1));

  auto grad_bias = allocateTorchTensor(grad_output.size(-1), grad_bias_type);
  auto grad_output_transpose =
            allocateTorchTensor(grad_output.size(1),
                                grad_output.size(0),
                                DType::kByte);
  auto input_cu             = makeTransformerEngineTensor(grad_output.data_ptr(), {M, N},
                                                         otype, amax.data_ptr(), scale.data_ptr(),
                                                         scale_inv.data_ptr());
  auto transposed_output_cu = makeTransformerEngineTensor(grad_output_transpose.data_ptr(),
                                                          {N, M}, otype, amax.data_ptr(),
                                                          scale.data_ptr(), scale_inv.data_ptr());
  auto dbias_cu             = makeTransformerEngineTensor(grad_bias);
  transformer_engine::TensorWrapper workspace;

  nvte_fp8_transpose_dbias(input_cu.data(), transposed_output_cu.data(), dbias_cu.data(),
                            workspace.data(), at::cuda::getCurrentCUDAStream());

  // Fill workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(workspace_data.data_ptr(),
                                          workspace.shape(),
                                          workspace.dtype());

  nvte_fp8_transpose_dbias(input_cu.data(), transposed_output_cu.data(), dbias_cu.data(),
                            workspace.data(), at::cuda::getCurrentCUDAStream());

  return {grad_bias, grad_output_transpose};
}



std::vector<at::Tensor> fused_cast_transpose_bgrad_dgelu(at::Tensor grad_output,
                                                         at::Tensor gelu_input,
                                                         at::Tensor scale,
                                                         at::Tensor amax,
                                                         at::Tensor scale_inv,
                                                         transformer_engine::DType otype
) {
  using namespace transformer_engine;

  size_t M = static_cast<size_t>(grad_output.size(0));
  size_t N = static_cast<size_t>(grad_output.size(1));

  DType grad_output_type = GetTransformerEngineDType(grad_output.scalar_type());
  auto grad_bias = allocateTorchTensor(grad_output.size(-1), grad_output_type);
  auto dgelu =
            allocateTorchTensor(grad_output.size(0),
                                grad_output.size(1),
                                DType::kByte);
  auto dgelu_transpose =
            allocateTorchTensor(grad_output.size(1),
                                grad_output.size(0),
                                DType::kByte);

  transformer_engine::TensorWrapper workspace;
  auto gelu_input_cu        = makeTransformerEngineTensor(gelu_input);
  auto input_cu             = makeTransformerEngineTensor(grad_output);
  auto cast_output_cu       = makeTransformerEngineTensor(dgelu.data_ptr(), {M, N},
                                                          otype, amax.data_ptr(), scale.data_ptr(),
                                                          scale_inv.data_ptr());
  auto transposed_output_cu = makeTransformerEngineTensor(dgelu_transpose.data_ptr(), {N, M},
                                                          otype, amax.data_ptr(), scale.data_ptr(),
                                                          scale_inv.data_ptr());
  auto dbias_cu             = makeTransformerEngineTensor(grad_bias);

  nvte_cast_transpose_dbias_dgelu(input_cu.data(), gelu_input_cu.data(),
                                  cast_output_cu.data(), transposed_output_cu.data(),
                                  dbias_cu.data(), workspace.data(),
                                  at::cuda::getCurrentCUDAStream());

  // Fill workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(workspace_data.data_ptr(),
                                          workspace.shape(),
                                          workspace.dtype());

  nvte_cast_transpose_dbias_dgelu(input_cu.data(), gelu_input_cu.data(),
                                  cast_output_cu.data(), transposed_output_cu.data(),
                                  dbias_cu.data(), workspace.data(),
                                  at::cuda::getCurrentCUDAStream());

  return {grad_bias, dgelu, dgelu_transpose};
}


void fused_multi_cast_transpose(std::vector<at::Tensor> input_list,
                                std::vector<at::Tensor> scale_list,
                                std::vector<at::Tensor> cast_output_list,
                                std::vector<at::Tensor> transposed_output_list,
                                std::vector<at::Tensor> amax_list,
                                std::vector<at::Tensor> scale_inv_list,
                                transformer_engine::DType otype
) {
  using namespace transformer_engine;

  // Extract properties from PyTorch tensors
  std::vector<void*> input_dptr_list, scale_dptr_list,
    cast_output_dptr_list, transposed_output_dptr_list,
    amax_dptr_list, scale_inv_dptr_list;
  std::vector<std::vector<size_t>> input_shape_list, scale_shape_list,
    cast_output_shape_list, transposed_output_shape_list,
    amax_shape_list, scale_inv_shape_list;
  std::vector<transformer_engine::DType> input_type_list, scale_type_list,
    cast_output_type_list, transposed_output_type_list,
    amax_type_list, scale_inv_type_list;
  auto extract_tensor_props_skip_dtype = [](at::Tensor& tensor,
                                            std::vector<void*>& dptr_list,
                                            std::vector<std::vector<size_t>>& shape_list) {
    dptr_list.push_back(tensor.data_ptr());
    shape_list.push_back({});
    for (int d = 0; d < tensor.dim(); ++d) {
      shape_list.back().push_back(tensor.size(d));
    }
  };
  auto extract_tensor_props = [](at::Tensor& tensor,
                                 std::vector<void*>& dptr_list,
                                 std::vector<std::vector<size_t>>& shape_list,
                                 std::vector<transformer_engine::DType>& type_list) {
    dptr_list.push_back(tensor.data_ptr());
    shape_list.push_back({});
    for (int d = 0; d < tensor.dim(); ++d) {
      shape_list.back().push_back(tensor.size(d));
    }
    type_list.push_back(GetTransformerEngineDType(tensor.scalar_type()));
  };
  for (size_t tensor_id = 0; tensor_id < input_list.size(); ++tensor_id) {
    extract_tensor_props(input_list[tensor_id],
                         input_dptr_list,
                         input_shape_list,
                         input_type_list);
    extract_tensor_props(scale_list[tensor_id],
                         scale_dptr_list,
                         scale_shape_list,
                         scale_type_list);
    extract_tensor_props_skip_dtype(cast_output_list[tensor_id],
                                    cast_output_dptr_list,
                                    cast_output_shape_list);
    cast_output_type_list.push_back(otype);
    extract_tensor_props_skip_dtype(transposed_output_list[tensor_id],
                                    transposed_output_dptr_list,
                                    transposed_output_shape_list);
    transposed_output_type_list.push_back(otype);
    extract_tensor_props(amax_list[tensor_id],
                         amax_dptr_list,
                         amax_shape_list,
                         amax_type_list);
    extract_tensor_props(scale_inv_list[tensor_id],
                         scale_inv_dptr_list,
                         scale_inv_shape_list,
                         scale_inv_type_list);
  }

  transformer_engine::TensorWrapper workspace;

  // Construct TE tensors
  std::vector<NVTETensor> nvte_input_list,
    nvte_cast_output_list, nvte_transposed_output_list;
  std::vector<transformer_engine::TensorWrapper> tensor_wrappers;
  auto make_tensor = [&tensor_wrappers](void* dptr,
                                        const std::vector<size_t>& shape,
                                        transformer_engine::DType dtype,
                                        void* amax_dptr,
                                        void* scale_dptr,
                                        void* scale_inv_dptr)
    -> NVTETensor {
    tensor_wrappers.emplace_back(makeTransformerEngineTensor(dptr, shape, dtype, amax_dptr,
                                                             scale_dptr, scale_inv_dptr));
    return tensor_wrappers.back().data();
  };
  for (size_t i = 0; i < input_dptr_list.size(); ++i) {
    nvte_input_list.emplace_back(make_tensor(input_dptr_list[i],
                                             input_shape_list[i],
                                             input_type_list[i],
                                             nullptr,
                                             nullptr,
                                             nullptr));
    nvte_cast_output_list.emplace_back(make_tensor(cast_output_dptr_list[i],
                                                   cast_output_shape_list[i],
                                                   cast_output_type_list[i],
                                                   amax_dptr_list[i],
                                                   scale_dptr_list[i],
                                                   scale_inv_dptr_list[i]));
    nvte_transposed_output_list.emplace_back(make_tensor(transposed_output_dptr_list[i],
                                                         transposed_output_shape_list[i],
                                                         transposed_output_type_list[i],
                                                         amax_dptr_list[i],
                                                         scale_dptr_list[i],
                                                         scale_inv_dptr_list[i]));
  }

  // Check tensor lists
  NVTE_CHECK(nvte_cast_output_list.size() == nvte_input_list.size(),
             "Number of input and C output tensors must match");
  NVTE_CHECK(nvte_transposed_output_list.size() == nvte_input_list.size(),
             "Number of input and T output tensors must match");

  // Launch TE kernel
  nvte_multi_cast_transpose(nvte_input_list.size(),
                            nvte_input_list.data(),
                            nvte_cast_output_list.data(),
                            nvte_transposed_output_list.data(),
                            at::cuda::getCurrentCUDAStream());
}


at::Tensor fp8_transpose(at::Tensor input,
                         transformer_engine::DType otype
) {
  using namespace transformer_engine;

  size_t M = static_cast<size_t>(input.size(0));
  size_t N = static_cast<size_t>(input.size(1));

  auto output =
            allocateTorchTensor(input.size(1),
                                input.size(0),
                                DType::kByte);

  auto input_cu  = makeTransformerEngineTensor(input.data_ptr(), {M, N}, otype);
  auto output_cu = makeTransformerEngineTensor(output.data_ptr(), {N, M}, otype);

  nvte_transpose(input_cu.data(), output_cu.data(), at::cuda::getCurrentCUDAStream());

  return output;
}


at::Tensor fp8_gelu(at::Tensor input,
                    at::Tensor scale,
                    at::Tensor amax,
                    at::Tensor scale_inv,
                    transformer_engine::DType otype
) {
  using namespace transformer_engine;

  size_t M = static_cast<size_t>(input.size(0));
  size_t N = static_cast<size_t>(input.size(1));

  auto output =
            allocateTorchTensor(input.size(0),
                                input.size(1),
                                DType::kByte);

  auto input_cu =  makeTransformerEngineTensor(input);
  auto output_cu = makeTransformerEngineTensor(output.data_ptr(), {M, N}, otype,
                                               amax.data_ptr(), scale.data_ptr(),
                                               scale_inv.data_ptr());

  nvte_gelu(input_cu.data(), output_cu.data(), at::cuda::getCurrentCUDAStream());

  return output;
}


std::vector<at::Tensor> layernorm_bwd(const at::Tensor &dz,
                                      const at::Tensor &x,
                                      const at::Tensor &mu,
                                      const at::Tensor &rsigma,
                                      const at::Tensor &gamma,
                                      const int sm_margin,
                                      const bool zero_centered_gamma
) {
    auto dx = at::empty_like(x);
    auto dgamma = at::empty_like(gamma);
    auto dbeta = at::empty_like(gamma);
    transformer_engine::TensorWrapper workspace, barrier, dgamma_part, dbeta_part;

    auto dz_cu      = makeTransformerEngineTensor(dz);
    auto x_cu       = makeTransformerEngineTensor(x);
    auto mu_cu      = makeTransformerEngineTensor(mu);
    auto rsigma_cu  = makeTransformerEngineTensor(rsigma);
    auto gamma_cu   = makeTransformerEngineTensor(gamma);
    auto dx_cu      = makeTransformerEngineTensor(dx);
    auto dgamma_cu  = makeTransformerEngineTensor(dgamma);
    auto dbeta_cu   = makeTransformerEngineTensor(dbeta);

    // This call populates tensors with the required config.
    const auto bwd_fun = zero_centered_gamma ? nvte_layernorm1p_bwd : nvte_layernorm_bwd;
    bwd_fun(dz_cu.data(), x_cu.data(), mu_cu.data(), rsigma_cu.data(), gamma_cu.data(),
            dx_cu.data(), dgamma_cu.data(), dbeta_cu.data(), dgamma_part.data(),
            dbeta_part.data(), at::cuda::getCurrentCUDAStream(),
            at::cuda::getCurrentDeviceProperties()->multiProcessorCount - sm_margin,
            workspace.data(), barrier.data());

    // Alloc space for Tensors.
    auto workspace_data     = allocateSpace(workspace.shape(), workspace.dtype());
    auto barrier_data       = allocateSpace(barrier.shape(), barrier.dtype(), true);
    auto dgamma_part_data   = allocateSpace(dgamma_part.shape(), dgamma_part.dtype());
    auto dbeta_part_data    = allocateSpace(dbeta_part.shape(), dbeta_part.dtype());
    workspace   = makeTransformerEngineTensor(workspace_data.data_ptr(),
                                              workspace.shape(),
                                              workspace.dtype());
    barrier     = makeTransformerEngineTensor(barrier_data.data_ptr(),
                                              barrier.shape(),
                                              barrier.dtype());
    dgamma_part = makeTransformerEngineTensor(dgamma_part_data.data_ptr(),
                                              dgamma_part.shape(),
                                              dgamma_part.dtype());
    dbeta_part  = makeTransformerEngineTensor(dbeta_part_data.data_ptr(),
                                              dbeta_part.shape(),
                                              dbeta_part.dtype());

    // Actual call to bwd kernel.
    bwd_fun(dz_cu.data(), x_cu.data(), mu_cu.data(), rsigma_cu.data(), gamma_cu.data(),
            dx_cu.data(), dgamma_cu.data(), dbeta_cu.data(), dgamma_part.data(),
            dbeta_part.data(), at::cuda::getCurrentCUDAStream(),
            at::cuda::getCurrentDeviceProperties()->multiProcessorCount - sm_margin,
            workspace.data(), barrier.data());

    return { dx, dgamma, dbeta };
}


std::vector<at::Tensor> layernorm_fwd_fp8(const at::Tensor &input,
                                          const at::Tensor &weight,
                                          const at::Tensor &bias,
                                          float eps,
                                          at::Tensor scale,
                                          at::Tensor amax,
                                          at::Tensor scale_inv,
                                          transformer_engine::DType otype,
                                          const int sm_margin,
                                          const bool zero_centered_gamma
) {
    using namespace transformer_engine;

    size_t N = static_cast<size_t>(input.size(0));
    size_t H = static_cast<size_t>(input.size(1));

    DType itype = GetTransformerEngineDType(input.scalar_type());

    auto ln_out = at::empty_like(input, at::CUDA(GetATenDType(otype)));
    auto mu = at::empty({static_cast<int64_t>(N)}, at::CUDA(at::kFloat));
    auto rsigma = at::empty({static_cast<int64_t>(N)}, at::CUDA(at::kFloat));
    auto input_cu     = makeTransformerEngineTensor(input);
    auto gamma_cu     = makeTransformerEngineTensor(weight);
    auto beta_cu      = makeTransformerEngineTensor(bias);
    auto z_cu         = makeTransformerEngineTensor(ln_out.data_ptr(), {N, H}, otype,
                                                    amax.data_ptr(), scale.data_ptr(),
                                                    scale_inv.data_ptr());
    auto mu_cu        = makeTransformerEngineTensor(mu);
    auto rsigma_cu    = makeTransformerEngineTensor(rsigma);
    transformer_engine::TensorWrapper workspace, barrier;

    // This call populates workspace and barrier tensors with the required config
    const auto func = zero_centered_gamma ? nvte_layernorm1p_fwd : nvte_layernorm_fwd;
    func(input_cu.data(), gamma_cu.data(), beta_cu.data(), eps, z_cu.data(),
         mu_cu.data(), rsigma_cu.data(), at::cuda::getCurrentCUDAStream(),
         at::cuda::getCurrentDeviceProperties()->multiProcessorCount - sm_margin,
         workspace.data(), barrier.data());

    // Fill workspace and barrier
    auto workspace_data = allocateSpace(workspace.shape(),
                                        workspace.dtype());
    auto barrier_data = allocateSpace(barrier.shape(),
                                      barrier.dtype(),
                                      true);
    workspace = makeTransformerEngineTensor(workspace_data.data_ptr(),
                                            workspace.shape(),
                                            workspace.dtype());
    barrier   = makeTransformerEngineTensor(barrier_data.data_ptr(),
                                            barrier.shape(),
                                            barrier.dtype());

    // Actual call to fwd kernel
    func(input_cu.data(), gamma_cu.data(), beta_cu.data(), eps, z_cu.data(),
         mu_cu.data(), rsigma_cu.data(), at::cuda::getCurrentCUDAStream(),
         at::cuda::getCurrentDeviceProperties()->multiProcessorCount - sm_margin,
         workspace.data(), barrier.data());

    return {ln_out, mu, rsigma};
}


std::vector<at::Tensor> layernorm_fwd_fp8_noalloc(const at::Tensor &input,
                                                  const at::Tensor &weight,
                                                  const at::Tensor &bias,
                                                  float eps,
                                                  at::Tensor scale,
                                                  at::Tensor ln_out,
                                                  at::Tensor amax,
                                                  at::Tensor scale_inv,
                                                  transformer_engine::DType otype,
                                                  const int sm_margin,
                                                  const bool zero_centered_gamma
) {
    using namespace transformer_engine;

    size_t N = static_cast<size_t>(input.size(0));
    size_t H = static_cast<size_t>(input.size(1));

    DType itype = GetTransformerEngineDType(input.scalar_type());

    auto mu = at::empty({static_cast<int64_t>(N)}, at::CUDA(at::kFloat));
    auto rsigma = at::empty({static_cast<int64_t>(N)}, at::CUDA(at::kFloat));
    auto input_cu     = makeTransformerEngineTensor(input);
    auto gamma_cu     = makeTransformerEngineTensor(weight);
    auto beta_cu      = makeTransformerEngineTensor(bias);
    auto z_cu         = makeTransformerEngineTensor(ln_out.data_ptr(), {N, H}, otype,
                                                    amax.data_ptr(), scale.data_ptr(),
                                                    scale_inv.data_ptr());
    auto mu_cu        = makeTransformerEngineTensor(mu);
    auto rsigma_cu    = makeTransformerEngineTensor(rsigma);
    transformer_engine::TensorWrapper workspace, barrier;

    // This call populates workspace and barrier tensors with the required config
    const auto func = zero_centered_gamma ? nvte_layernorm1p_fwd : nvte_layernorm_fwd;
    func(input_cu.data(), gamma_cu.data(), beta_cu.data(), eps, z_cu.data(),
         mu_cu.data(), rsigma_cu.data(), at::cuda::getCurrentCUDAStream(),
         at::cuda::getCurrentDeviceProperties()->multiProcessorCount - sm_margin,
         workspace.data(), barrier.data());

    // Fill workspace and barrier
    auto workspace_data = allocateSpace(workspace.shape(),
                                        workspace.dtype());
    auto barrier_data = allocateSpace(barrier.shape(),
                                      barrier.dtype(),
                                      true);
    workspace = makeTransformerEngineTensor(workspace_data.data_ptr(),
                                            workspace.shape(),
                                            workspace.dtype());
    barrier   = makeTransformerEngineTensor(barrier_data.data_ptr(),
                                            barrier.shape(),
                                            barrier.dtype());

    // Actual call to fwd kernel
    func(input_cu.data(), gamma_cu.data(), beta_cu.data(), eps, z_cu.data(),
         mu_cu.data(), rsigma_cu.data(), at::cuda::getCurrentCUDAStream(),
         at::cuda::getCurrentDeviceProperties()->multiProcessorCount - sm_margin,
         workspace.data(), barrier.data());

    return {ln_out, mu, rsigma};
}


at::Tensor layernorm_fwd_fp8_inf(const at::Tensor &input,
                                 const at::Tensor &weight,
                                 const at::Tensor &bias,
                                 float eps,
                                 at::Tensor scale,
                                 at::Tensor amax,
                                 at::Tensor scale_inv,
                                 transformer_engine::DType otype,
                                 const bool zero_centered_gamma
) {
    // This is a specialized version of layernorm_fwd_fp8, optimized for inference,
    // which only returns the normalized output.
    std::vector<at::Tensor> out = layernorm_fwd_fp8(
      input, weight, bias, eps, scale, amax, scale_inv, otype, 0, zero_centered_gamma);
    return out[0];
}


std::vector<at::Tensor> layernorm_fwd(const at::Tensor &input,
                                      const at::Tensor &weight,
                                      const at::Tensor &bias,
                                      float eps,
                                      const int sm_margin,
                                      const bool zero_centered_gamma
) {
    using namespace transformer_engine;

    size_t N = static_cast<size_t>(input.size(0));
    size_t H = static_cast<size_t>(input.size(1));

    DType itype = GetTransformerEngineDType(input.scalar_type());

    auto ln_out = at::empty_like(input, at::CUDA(GetATenDType(itype)));
    auto mu = at::empty({static_cast<int64_t>(N)}, at::CUDA(at::kFloat));
    auto rsigma = at::empty({static_cast<int64_t>(N)}, at::CUDA(at::kFloat));
    auto input_cu     = makeTransformerEngineTensor(input);
    auto gamma_cu     = makeTransformerEngineTensor(weight);
    auto beta_cu      = makeTransformerEngineTensor(bias);
    auto z_cu         = makeTransformerEngineTensor(ln_out);
    auto mu_cu        = makeTransformerEngineTensor(mu);
    auto rsigma_cu    = makeTransformerEngineTensor(rsigma);
    transformer_engine::TensorWrapper workspace, barrier;

    // This call populates workspace and barrier tensors with the required config
    const auto func = zero_centered_gamma ? nvte_layernorm1p_fwd : nvte_layernorm_fwd;
    func(input_cu.data(), gamma_cu.data(), beta_cu.data(), eps, z_cu.data(),
         mu_cu.data(), rsigma_cu.data(), at::cuda::getCurrentCUDAStream(),
         at::cuda::getCurrentDeviceProperties()->multiProcessorCount - sm_margin,
         workspace.data(), barrier.data());

    // Fill workspace and barrier
    auto workspace_data = allocateSpace(workspace.shape(),
                                        workspace.dtype());
    auto barrier_data = allocateSpace(barrier.shape(),
                                      barrier.dtype(),
                                      true);
    workspace = makeTransformerEngineTensor(workspace_data.data_ptr(),
                                            workspace.shape(),
                                            workspace.dtype());
    barrier   = makeTransformerEngineTensor(barrier_data.data_ptr(),
                                            barrier.shape(),
                                            barrier.dtype());

    // Actual call to fwd kernel
    func(input_cu.data(), gamma_cu.data(), beta_cu.data(), eps, z_cu.data(),
         mu_cu.data(), rsigma_cu.data(), at::cuda::getCurrentCUDAStream(),
         at::cuda::getCurrentDeviceProperties()->multiProcessorCount - sm_margin,
         workspace.data(), barrier.data());

    return {ln_out, mu, rsigma};
}


std::vector<at::Tensor> layernorm_fwd_noalloc(const at::Tensor &input,
                                              const at::Tensor &weight,
                                              const at::Tensor &bias,
                                              at::Tensor ln_out,
                                              float eps,
                                              const int sm_margin,
                                              const bool zero_centered_gamma
) {
    using namespace transformer_engine;

    size_t N = static_cast<size_t>(input.size(0));
    size_t H = static_cast<size_t>(input.size(1));

    DType itype = GetTransformerEngineDType(input.scalar_type());

    auto mu = at::empty({static_cast<int64_t>(N)}, at::CUDA(at::kFloat));
    auto rsigma = at::empty({static_cast<int64_t>(N)}, at::CUDA(at::kFloat));
    auto input_cu     = makeTransformerEngineTensor(input);
    auto gamma_cu     = makeTransformerEngineTensor(weight);
    auto beta_cu      = makeTransformerEngineTensor(bias);
    auto z_cu         = makeTransformerEngineTensor(ln_out);
    auto mu_cu        = makeTransformerEngineTensor(mu);
    auto rsigma_cu    = makeTransformerEngineTensor(rsigma);
    transformer_engine::TensorWrapper workspace, barrier;

    // This call populates workspace and barrier tensors with the required config
    const auto func = zero_centered_gamma ? nvte_layernorm1p_fwd : nvte_layernorm_fwd;
    func(input_cu.data(), gamma_cu.data(), beta_cu.data(), eps, z_cu.data(),
         mu_cu.data(), rsigma_cu.data(), at::cuda::getCurrentCUDAStream(),
         at::cuda::getCurrentDeviceProperties()->multiProcessorCount - sm_margin,
         workspace.data(), barrier.data());

    // Fill workspace and barrier
    auto workspace_data = allocateSpace(workspace.shape(),
                                        workspace.dtype());
    auto barrier_data = allocateSpace(barrier.shape(),
                                      barrier.dtype(),
                                      true);
    workspace = makeTransformerEngineTensor(workspace_data.data_ptr(),
                                            workspace.shape(),
                                            workspace.dtype());
    barrier   = makeTransformerEngineTensor(barrier_data.data_ptr(),
                                            barrier.shape(),
                                            barrier.dtype());

    // Actual call to fwd kernel
    func(input_cu.data(), gamma_cu.data(), beta_cu.data(), eps, z_cu.data(),
         mu_cu.data(), rsigma_cu.data(), at::cuda::getCurrentCUDAStream(),
         at::cuda::getCurrentDeviceProperties()->multiProcessorCount - sm_margin,
         workspace.data(), barrier.data());

    return {ln_out, mu, rsigma};
}


at::Tensor layernorm_fwd_inf(const at::Tensor &input,
                             const at::Tensor &weight,
                             const at::Tensor &bias,
                             float eps,
                             const bool zero_centered_gamma
) {
    // This is a specialized version of layernorm_fwd, optimized for inference,
    // which only returns the normalized output.
    std::vector<at::Tensor> out = layernorm_fwd(input, weight, bias, eps, 0, zero_centered_gamma);
    return out[0];
}


at::Tensor cast_to_fp8(const at::Tensor &input,
                       const at::Tensor &scale,
                       at::Tensor amax,
                       at::Tensor scale_inv,
                       transformer_engine::DType otype
) {
    using namespace transformer_engine;
    auto input_shape = input.sizes().vec();
    std::vector<size_t> shape{input_shape.begin(), input_shape.end()};

    auto output = at::empty_like(input, at::CUDA(GetATenDType(otype)));

    auto input_cu     = makeTransformerEngineTensor(input);
    auto output_cu    = makeTransformerEngineTensor(output.data_ptr(), shape, otype,
                                                    amax.data_ptr(), scale.data_ptr(),
                                                    scale_inv.data_ptr());

    nvte_fp8_quantize(input_cu.data(), output_cu.data(),
                      at::cuda::getCurrentCUDAStream());

    return output;
}


void cast_to_fp8_noalloc(const at::Tensor &input,
                               const at::Tensor &scale,
                               at::Tensor output,
                               at::Tensor amax,
                               at::Tensor scale_inv,
                               transformer_engine::DType otype
) {
    using namespace transformer_engine;
    size_t N = static_cast<size_t>(input.size(0));
    size_t H = static_cast<size_t>(input.size(1));

    auto input_cu     = makeTransformerEngineTensor(input);
    auto output_cu    = makeTransformerEngineTensor(output.data_ptr(), {N, H}, otype,
                                                    amax.data_ptr(), scale.data_ptr(),
                                                    scale_inv.data_ptr());

    nvte_fp8_quantize(input_cu.data(), output_cu.data(),
                      at::cuda::getCurrentCUDAStream());

    return;
}


at::Tensor cast_from_fp8(const at::Tensor &input,
                         const at::Tensor &scale_inv,
                         transformer_engine::DType itype,
                         transformer_engine::DType otype
) {
    using namespace transformer_engine;
    auto input_shape = input.sizes().vec();
    std::vector<size_t> shape{input_shape.begin(), input_shape.end()};

    auto output = at::empty_like(input, at::CUDA(GetATenDType(otype)));

    auto input_cu     = makeTransformerEngineTensor(input.data_ptr(), shape, itype,
                                                    nullptr, nullptr, scale_inv.data_ptr());
    auto output_cu    = makeTransformerEngineTensor(output);

    nvte_fp8_dequantize(input_cu.data(), output_cu.data(),
                        at::cuda::getCurrentCUDAStream());

    return output;
}


at::Tensor scaled_softmax_forward(at::Tensor input,
                                  float scale_factor
) {
    using namespace transformer_engine;
    AT_ASSERTM(input.dim() == 4, "expected 4D tensor");
    AT_ASSERTM((input.scalar_type() == at::ScalarType::Half) ||
               (input.scalar_type() == at::ScalarType::BFloat16),
               "Only fp16 and bf16 are supported");

    const int batches = input.size(0);
    const int attn_heads = input.size(1);
    const int query_seq_len = input.size(2);
    const int key_seq_len = input.size(3);

    TORCH_CHECK(key_seq_len <= 4096);
    TORCH_CHECK(query_seq_len > 1);

    // Output
  auto act_options = input.options().requires_grad(false);
  auto softmax_results =
      torch::empty({batches, attn_heads, query_seq_len, key_seq_len}, act_options);

  auto input_cu = makeTransformerEngineTensor(input);
  auto softmax_results_cu = makeTransformerEngineTensor(softmax_results);

  nvte_scaled_softmax_forward(input_cu.data(), softmax_results_cu.data(), scale_factor,
                              at::cuda::getCurrentCUDAStream());

  return softmax_results;
}


at::Tensor scaled_softmax_backward(at::Tensor output_grad_,
                                   at::Tensor softmax_results_,
                                   float scale_factor
) {
    using namespace transformer_engine;

    auto output_grads = output_grad_.contiguous();
    auto softmax_results = softmax_results_.contiguous();

    AT_ASSERTM(output_grads.dim() == 4, "expected 4D tensor");
    AT_ASSERTM(softmax_results.dim() == 4, "expected 4D tensor");

    AT_ASSERTM((output_grads.scalar_type() == at::ScalarType::Half) ||
        (output_grads.scalar_type() == at::ScalarType::BFloat16),
        "Only fp16 and bf16 are supported");
    AT_ASSERTM((softmax_results.scalar_type() == at::ScalarType::Half) ||
        (softmax_results.scalar_type() == at::ScalarType::BFloat16),
        "Only fp16 and bf16 are supported");

    auto output_grads_cu = makeTransformerEngineTensor(output_grads);
    auto softmax_results_cu = makeTransformerEngineTensor(softmax_results);

    // Produce gradients in place.
    nvte_scaled_softmax_backward(
          output_grads_cu.data(), softmax_results_cu.data(), output_grads_cu.data(),
          scale_factor, at::cuda::getCurrentCUDAStream());

    return output_grads;
}


at::Tensor scaled_masked_softmax_forward(at::Tensor input,
                                         at::Tensor mask,
                                         float scale_factor
) {
    using namespace transformer_engine;

    AT_ASSERTM(input.dim() == 4, "expected 4D tensor");
    AT_ASSERTM((input.scalar_type() == at::ScalarType::Half) ||
               (input.scalar_type() == at::ScalarType::BFloat16),
               "Only fp16 and bf16 are supported");
    AT_ASSERTM(mask.dim() == 4, "expected 4D tensor");
    if (!input.is_contiguous())
        input = input.contiguous();
    if (!mask.is_contiguous())
        mask = mask.contiguous();

    const int batches = input.size(0);
    const int pad_batches = mask.size(0);
    const int attn_heads = input.size(1);
    const int query_seq_len = input.size(2);
    const int key_seq_len = input.size(3);
    TORCH_CHECK(key_seq_len <= 4096);
    TORCH_CHECK(query_seq_len > 1);
    TORCH_CHECK(pad_batches == 1 || pad_batches == batches);
    TORCH_CHECK(mask.size(1) == 1);
    TORCH_CHECK(mask.size(2) == query_seq_len);
    TORCH_CHECK(mask.size(3) == key_seq_len);

    auto act_options = input.options().requires_grad(false);
    auto softmax_results =
        torch::empty({batches, attn_heads, query_seq_len, key_seq_len}, act_options);


    auto input_cu = makeTransformerEngineTensor(input);
    auto mask_cu = makeTransformerEngineTensor(mask);
    auto softmax_results_cu = makeTransformerEngineTensor(softmax_results);

    nvte_scaled_masked_softmax_forward(
          input_cu.data(), mask_cu.data(), softmax_results_cu.data(),
          scale_factor, at::cuda::getCurrentCUDAStream());

    return softmax_results;
}


at::Tensor scaled_masked_softmax_backward(at::Tensor output_grad_,
                                          at::Tensor softmax_results_,
                                          float scale_factor
) {
    using namespace transformer_engine;

    auto output_grads = output_grad_.contiguous();
    auto softmax_results = softmax_results_.contiguous();

    AT_ASSERTM(output_grads.dim() == 4, "expected 3D tensor");
    AT_ASSERTM(softmax_results.dim() == 4, "expected 3D tensor");

    AT_ASSERTM((output_grads.scalar_type() == at::ScalarType::Half) ||
        (output_grads.scalar_type() == at::ScalarType::BFloat16),
        "Only fp16 and bf16 are supported");
    AT_ASSERTM((softmax_results.scalar_type() == at::ScalarType::Half) ||
        (softmax_results.scalar_type() == at::ScalarType::BFloat16),
        "Only fp16 and bf16 are supported");

    auto output_grads_cu = makeTransformerEngineTensor(output_grads);
    auto softmax_results_cu = makeTransformerEngineTensor(softmax_results);

    // Produce gradients in place.
    nvte_scaled_softmax_backward(
          output_grads_cu.data(), softmax_results_cu.data(), output_grads_cu.data(),
          scale_factor, at::cuda::getCurrentCUDAStream());

    return output_grads;
}


at::Tensor scaled_upper_triang_masked_softmax_forward(at::Tensor input,
                                                      float scale_factor
) {
    using namespace transformer_engine;

    AT_ASSERTM(input.dim() == 3, "expected 3D tensor");
    AT_ASSERTM((input.scalar_type() == at::ScalarType::Half) ||
               (input.scalar_type() == at::ScalarType::BFloat16),
               "Only fp16 and bf16 are supported");

    const int attn_batches = input.size(0);
    const int seq_len = input.size(1);
    TORCH_CHECK(seq_len <= 2048);

    // Output
    auto act_options = input.options().requires_grad(false);
    auto softmax_results =
        torch::empty({attn_batches, seq_len, seq_len}, act_options);

    auto input_cu = makeTransformerEngineTensor(input);
    auto softmax_results_cu = makeTransformerEngineTensor(softmax_results);

    nvte_scaled_upper_triang_masked_softmax_forward(input_cu.data(),
                                                    softmax_results_cu.data(),
                                                    scale_factor,
                                                    at::cuda::getCurrentCUDAStream());

    return softmax_results;
}


at::Tensor scaled_upper_triang_masked_softmax_backward(at::Tensor output_grads_,
                                                       at::Tensor softmax_results_,
                                                       float scale_factor
) {
    using namespace transformer_engine;

    auto output_grads = output_grads_.contiguous();
    auto softmax_results = softmax_results_.contiguous();

    AT_ASSERTM(output_grads.dim() == 3, "expected 3D tensor");
    AT_ASSERTM(softmax_results.dim() == 3, "expected 3D tensor");

    AT_ASSERTM((output_grads.scalar_type() == at::ScalarType::Half) ||
        (output_grads.scalar_type() == at::ScalarType::BFloat16),
        "Only fp16 and bf16 are supported");
    AT_ASSERTM((softmax_results.scalar_type() == at::ScalarType::Half) ||
        (softmax_results.scalar_type() == at::ScalarType::BFloat16),
        "Only fp16 and bf16 are supported");

    TORCH_CHECK(output_grads.size(1) == output_grads.size(2));

    auto output_grads_cu = makeTransformerEngineTensor(output_grads);
    auto softmax_results_cu = makeTransformerEngineTensor(softmax_results);

    // Produce gradients in place.
    nvte_scaled_upper_triang_masked_softmax_backward(output_grads_cu.data(),
                                                     softmax_results_cu.data(),
                                                     output_grads_cu.data(),
                                                     scale_factor,
                                                     at::cuda::getCurrentCUDAStream());

  return output_grads;
}


size_t get_cublasLt_version() {
    return cublasLtGetVersion();
}


bool userbuf_comm_available() {  // TODO(ksivamani) check on python side
#ifdef NVTE_WITH_USERBUFFERS
    return true;
#else
    return false;
#endif
}

void placeholder() {}  // TODO(ksivamani) clean this up


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  // Softmax functions
  m.def("scaled_softmax_forward", &scaled_softmax_forward, "Scaled Softmax FWD");
  m.def("scaled_softmax_backward", &scaled_softmax_backward, "Scaled Softmax BWD");
  m.def("scaled_masked_softmax_forward", &scaled_masked_softmax_forward,
                                                    "Scaled Masked Softmax FWD");
  m.def("scaled_masked_softmax_backward", &scaled_masked_softmax_backward,
                                                    "Scaled Masked Softmax BWD");
  m.def("scaled_upper_triang_masked_softmax_forward",
            &scaled_upper_triang_masked_softmax_forward,
            "Scaled Upper-Triangular Masked Softmax FWD");
  m.def("scaled_upper_triang_masked_softmax_backward",
            &scaled_upper_triang_masked_softmax_backward,
            "Scaled Upper-Triangular Masked Softmax BWD");

  // Other granular functions
  m.def("layernorm_fwd_fp8", &layernorm_fwd_fp8, "LN FWD FP8");
  m.def("layernorm_fwd_fp8_noalloc", &layernorm_fwd_fp8_noalloc, "LN FWD FP8");
  m.def("layernorm_bwd", &layernorm_bwd, "LN BWD");
  m.def("layernorm_fwd", &layernorm_fwd, "LN FWD");
  m.def("layernorm_fwd_noalloc", &layernorm_fwd_noalloc, "LN FWD");
  m.def("fused_cast_transpose", &fused_cast_transpose, "Fused Cast + Transpose");
  m.def("fused_cast_transpose_bgrad", &fused_cast_transpose_bgrad,
                                              "Fused Cast + Transpose + BGRAD");
  m.def("fused_fp8_transpose_bgrad", &fused_fp8_transpose_bgrad,
                                              "Fused FP8 Transpose + BGRAD");
  m.def("fused_cast_transpose_bgrad_dgelu", &fused_cast_transpose_bgrad_dgelu,
                                              "Fused Cast + Transpose + BGRAD + DGELU");
  m.def("fused_multi_cast_transpose", &fused_multi_cast_transpose,
                                              "Fused Multi-tensor Cast + Transpose");
  m.def("cast_to_fp8", &cast_to_fp8, "Cast to FP8");
  m.def("cast_to_fp8_noalloc", &cast_to_fp8_noalloc, "Cast to FP8");
  m.def("cast_from_fp8", &cast_from_fp8, "Cast from FP8");
  m.def("te_gemm", &te_gemm, "CublasLt GEMM");
  m.def("fused_attn_fwd_qkvpacked", &fused_attn_fwd_qkvpacked,
                  "Fused Attention FP8/BF16/FP16 FWD with packed QKV");
  m.def("fused_attn_bwd_qkvpacked", &fused_attn_bwd_qkvpacked,
                  "Fused Attention FP8/BF16/FP16 BWD with packed QKV");
  m.def("fused_attn_fwd_kvpacked", &fused_attn_fwd_kvpacked,
                  "Fused Attention FP8/BF16/FP16 FWD with packed KV");
  m.def("fused_attn_bwd_kvpacked", &fused_attn_bwd_kvpacked,
                  "Fused Attention FP8/BF16/FP16 BWD with packed KV");
  m.def("fp8_transpose", &fp8_transpose, "Transpose with FP8 I/O");
  m.def("fp8_gelu", &fp8_gelu, "GeLU with FP8 output");

  // Misc
  m.def("get_cublasLt_version", &get_cublasLt_version, "Get cublasLt version");
  m.def("userbuf_comm_available", &userbuf_comm_available, "If userbuf backend is available");

  // Data structures
  py::class_<transformer_engine::FP8TensorMeta>(m, "FP8TensorMeta")
    .def(py::init<>())
    .def_readwrite("scale", &transformer_engine::FP8TensorMeta::scale)
    .def_readwrite("scale_inv", &transformer_engine::FP8TensorMeta::scale_inv)
    .def_readwrite("amax_history", &transformer_engine::FP8TensorMeta::amax_history);

#ifdef NVTE_WITH_USERBUFFERS
  py::enum_<ubuf::UBOverlapAlgo>(m, "UbufOverlapAlgo")
    .value("BULK_OVERLAP_AG", ubuf::UBOverlapAlgo::BULK_OVERLAP_AG)
    .value("BULK_OVERLAP_RS", ubuf::UBOverlapAlgo::BULK_OVERLAP_RS)
    .value("SPLIT_PIPELINED_RS", ubuf::UBOverlapAlgo::SPLIT_PIPELINED_RS)
    .value("SPLIT_PIPELINED_AG", ubuf::UBOverlapAlgo::SPLIT_PIPELINED_AG);

  py::class_<ubuf::UbufCommOverlap>(m, "UbufCommOverlap")
    .def(py::init<torch::Tensor&, int, int, int, int, int, bool, int>())
    .def("bulk_overlap", &ubuf::UbufCommOverlap::bulk_overlap)
    .def("split_overlap_rs", &ubuf::UbufCommOverlap::split_overlap_rs)
    .def("copy_input_to_ubuf", &ubuf::UbufCommOverlap::copy_input_to_ubuf)
    .def("get_ubuf_output", &ubuf::UbufCommOverlap::get_ubuf_output);

  py::class_<ubuf::UbufP2PCommOverlap>(m, "UbufP2PCommOverlap")
    .def(py::init<torch::Tensor&, int, int, bool, int>())
    .def("split_overlap_ag", &ubuf::UbufP2PCommOverlap::split_overlap_ag)
    .def("copy_input_to_ubuf", &ubuf::UbufP2PCommOverlap::copy_input_to_ubuf)
    .def("get_ubuf_output", &ubuf::UbufP2PCommOverlap::get_ubuf_output);
#else  // NVTE_WITH_USERBUFFERS
  m.def("UbufOverlapAlgo", &placeholder, "Dummy function for python side annotations");
  m.def("UbufCommOverlap", &placeholder, "Dummy function for python side annotations");
  m.def("UbufP2PCommOverlap", &placeholder, "Dummy function for python side annotations");
#endif  // NVTE_WITH_USERBUFFERS

  py::enum_<transformer_engine::DType>(m, "DType", py::module_local())
    .value("kByte", transformer_engine::DType::kByte)
    .value("kInt32", transformer_engine::DType::kInt32)
    .value("kFloat32", transformer_engine::DType::kFloat32)
    .value("kFloat16", transformer_engine::DType::kFloat16)
    .value("kBFloat16", transformer_engine::DType::kBFloat16)
    .value("kFloat8E4M3", transformer_engine::DType::kFloat8E4M3)
    .value("kFloat8E5M2", transformer_engine::DType::kFloat8E5M2);

  py::enum_<transformer_engine::FP8FwdTensors>(m, "FP8FwdTensors")
    .value("GEMM1_INPUT", transformer_engine::FP8FwdTensors::GEMM1_INPUT)
    .value("GEMM1_WEIGHT", transformer_engine::FP8FwdTensors::GEMM1_WEIGHT)
    .value("GEMM1_OUTPUT", transformer_engine::FP8FwdTensors::GEMM1_OUTPUT)
    .value("GEMM2_INPUT", transformer_engine::FP8FwdTensors::GEMM2_INPUT)
    .value("GEMM2_WEIGHT", transformer_engine::FP8FwdTensors::GEMM2_WEIGHT)
    .value("GEMM2_OUTPUT", transformer_engine::FP8FwdTensors::GEMM2_OUTPUT);

  py::enum_<transformer_engine::FP8BwdTensors>(m, "FP8BwdTensors")
    .value("GRAD_OUTPUT1", transformer_engine::FP8BwdTensors::GRAD_OUTPUT1)
    .value("GRAD_INPUT1", transformer_engine::FP8BwdTensors::GRAD_INPUT1)
    .value("GRAD_OUTPUT2", transformer_engine::FP8BwdTensors::GRAD_OUTPUT2)
    .value("GRAD_INPUT2", transformer_engine::FP8BwdTensors::GRAD_INPUT2);
}
