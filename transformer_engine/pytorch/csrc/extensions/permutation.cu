#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <hipcub/hipcub.hpp>

#include "extensions.h"

using torch::Tensor;

std::tuple<Tensor, Tensor, std::vector<Tensor>> moe_permute_fwd(
    Tensor input, const transformer_engine::DType dtype, Tensor indices, int64_t num_out_tokens,
    std::vector<Tensor> workspace, int64_t max_expanded_token_num) {
  const int num_tokens = input.size(0);
  int num_cols = input.size(1);
  const int topK = indices.size(1);

  // initialize the workspace on the first run
  if (workspace.empty()) {
    auto options =
        torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA).requires_grad(false);

    Tensor sorted_indices = torch::empty(max_expanded_token_num, options);
    Tensor row_id = torch::range(0, max_expanded_token_num - 1, 1, options);
    Tensor sorted_row_id =
        torch::empty(max_expanded_token_num,
                     torch::dtype(torch::kInt32).device(torch::kCUDA).requires_grad(false));

    size_t temp_storage_bytes = 0;
    int *temp_ptr = nullptr;
    hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes, temp_ptr, temp_ptr, temp_ptr,
                                    temp_ptr, max_expanded_token_num);
    Tensor temp_storage = torch::empty(
        temp_storage_bytes, torch::dtype(torch::kInt8).device(torch::kCUDA).requires_grad(false));

    workspace.push_back(sorted_indices);
    workspace.push_back(row_id);
    workspace.push_back(sorted_row_id);
    workspace.push_back(temp_storage);
  }

  int *indices_ptr = reinterpret_cast<int *>(getDataPtr(indices, 0));
  int *sorted_indices_ptr = reinterpret_cast<int *>(getDataPtr(workspace[0], 0));
  int *row_id_ptr = reinterpret_cast<int *>(getDataPtr(workspace[1], 0));
  int *sorted_row_id_ptr = reinterpret_cast<int *>(getDataPtr(workspace[2], 0));

  void *d_temp_storage = getDataPtr(workspace[3], 0);
  size_t temp_storage_bytes = std::numeric_limits<size_t>::max();

  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, indices_ptr,
                                  sorted_indices_ptr, row_id_ptr, sorted_row_id_ptr,
                                  num_tokens * topK);

  // activations type
  at::ScalarType _st;
  if (dtype == transformer_engine::DType::kFloat8E4M3 ||
      dtype == transformer_engine::DType::kFloat8E5M2)
    _st = at::ScalarType::Float;
  else
    _st = input.scalar_type();

  // Output buffer alloc
  num_out_tokens = (num_out_tokens > 0) ? num_out_tokens : num_tokens * topK;
  Tensor permuted_output = torch::empty(
      {num_out_tokens, num_cols}, torch::dtype(_st).device(torch::kCUDA).requires_grad(false));
  Tensor row_id_map = torch::empty(
      {num_tokens * topK}, torch::dtype(torch::kInt32).device(torch::kCUDA).requires_grad(false));

  int *row_id_map_ptr = reinterpret_cast<int *>(getDataPtr(row_id_map, 0));
  auto stream = at::cuda::getCurrentCUDAStream().stream();

  void *input_ptr = getDataPtr(input, 0);
  void *permuted_output_ptr = getDataPtr(permuted_output, 0);

  if (dtype == transformer_engine::DType::kFloat8E4M3 ||
      dtype == transformer_engine::DType::kFloat8E5M2)
    num_cols *= 4;

  nvte_permute(input_ptr, permuted_output_ptr, dtype, sorted_row_id_ptr, row_id_map_ptr, nullptr,
               num_tokens, topK, num_cols, num_out_tokens, nullptr, nullptr, stream);

  return std::make_tuple(permuted_output, row_id_map, workspace);
}

Tensor moe_permute_bwd(Tensor input, const transformer_engine::DType dtype, Tensor row_id_map,
                       Tensor prob, int64_t num_tokens, int64_t topK) {
  return moe_unpermute_fwd(input, dtype, row_id_map, prob, num_tokens, topK);
}

Tensor moe_unpermute_fwd(Tensor input, const transformer_engine::DType dtype, Tensor row_id_map,
                         Tensor prob, int64_t num_tokens, int64_t topK) {
  int num_cols = input.size(1);

  // activations type
  at::ScalarType _st;
  if (dtype == transformer_engine::DType::kFloat8E4M3 ||
      dtype == transformer_engine::DType::kFloat8E5M2)
    _st = at::ScalarType::Float;
  else
    _st = input.scalar_type();

  // Output buffer alloc
  Tensor unpermuted_output = torch::empty(
      {num_tokens, num_cols}, torch::dtype(_st).device(torch::kCUDA).requires_grad(false));

  int *row_id_map_ptr = reinterpret_cast<int *>(getDataPtr(row_id_map, 0));
  float *prob_ptr = (prob.numel() > 0) ? reinterpret_cast<float *>(getDataPtr(prob, 0)) : nullptr;
  auto stream = at::cuda::getCurrentCUDAStream().stream();

  void *input_ptr = getDataPtr(input, 0);
  void *unpermuted_output_ptr = getDataPtr(unpermuted_output, 0);

  if (dtype == transformer_engine::DType::kFloat8E4M3 ||
      dtype == transformer_engine::DType::kFloat8E5M2)
    num_cols *= 4;

  nvte_unpermute(input_ptr, unpermuted_output_ptr, dtype, row_id_map_ptr, prob_ptr, num_tokens,
                 topK, num_cols, stream);

  return unpermuted_output;
}

std::tuple<Tensor, Tensor> moe_unpermute_bwd(Tensor input_bwd, Tensor input_fwd,
                                             const transformer_engine::DType dtype,
                                             Tensor row_id_map, Tensor prob) {
  const int topK = (prob.numel() > 0) ? prob.size(1) : 1;
  const int num_tokens = (prob.numel() > 0) ? prob.size(0) : row_id_map.size(0);
  int num_cols = input_bwd.size(1);

  int *row_id_map_ptr = reinterpret_cast<int *>(getDataPtr(row_id_map, 0));
  float *prob_ptr = (prob.numel() > 0) ? reinterpret_cast<float *>(getDataPtr(prob, 0)) : nullptr;

  // activations type
  at::ScalarType _st;
  if (dtype == transformer_engine::DType::kFloat8E4M3 ||
      dtype == transformer_engine::DType::kFloat8E5M2)
    _st = at::ScalarType::Float;
  else
    _st = input_bwd.scalar_type();

  // Output buffer alloc
  Tensor act_grad = torch::empty({input_fwd.size(0), num_cols},
                                 torch::dtype(_st).device(torch::kCUDA).requires_grad(false));
  Tensor prob_grad = torch::empty(
      {num_tokens, topK}, torch::dtype(torch::kFloat32).device(torch::kCUDA).requires_grad(false));
  float *prob_grad_ptr = reinterpret_cast<float *>(getDataPtr(prob_grad, 0));

  auto stream = at::cuda::getCurrentCUDAStream().stream();

  void *input_bwd_ptr = getDataPtr(input_bwd, 0);
  void *input_fwd_ptr = getDataPtr(input_fwd, 0);
  void *act_grad_ptr = getDataPtr(act_grad, 0);

  if (dtype == transformer_engine::DType::kFloat8E4M3 ||
      dtype == transformer_engine::DType::kFloat8E5M2)
    num_cols *= 4;

  nvte_permute(input_bwd_ptr, act_grad_ptr, dtype, nullptr, row_id_map_ptr, prob_ptr, num_tokens,
               topK, num_cols, 0, prob_grad_ptr, input_fwd_ptr, stream);

  return std::make_tuple(act_grad, prob_grad);
}
