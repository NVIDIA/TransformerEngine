/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "common/util/hip/hip_runtime.h"
#include "extensions.h"

void te_gemm(at::Tensor A, at::Tensor A_scale_inverse, transformer_engine::DType A_type,
             bool transa, at::Tensor B, at::Tensor B_scale_inverse,
             transformer_engine::DType B_type, bool transb, at::Tensor D, at::Tensor D_scale,
             transformer_engine::DType D_type, at::Tensor D_amax, at::Tensor bias,
             transformer_engine::DType bias_type, at::Tensor pre_gelu_out, bool grad,
             at::Tensor workspace, size_t workspaceSize, bool accumulate,
             bool use_split_accumulator, int math_sm_count) {
  using namespace transformer_engine;
  if (A.numel() == 0 || B.numel() == 0) {
    if (D.numel() != 0 && !accumulate) D.zero_();
    if (bias.numel() != 0 && grad) {
      if (B.numel() == 0) {
        bias.zero_();
      } else {
        bias.copy_(B.sum(0));
      }
    }
    if (pre_gelu_out.numel() != 0) pre_gelu_out.zero_();
    return;
  }

  A = A.contiguous();
  B = B.contiguous();

  auto te_A = makeTransformerEngineTensor(
      A.data_ptr(), {static_cast<size_t>(A.size(0)), static_cast<size_t>(A.size(1))}, A_type,
      nullptr, nullptr, A_scale_inverse.data_ptr());
  auto te_B = makeTransformerEngineTensor(
      B.data_ptr(), {static_cast<size_t>(B.size(0)), static_cast<size_t>(B.size(1))}, B_type,
      nullptr, nullptr, B_scale_inverse.data_ptr());
  auto te_D = makeTransformerEngineTensor(
      D.data_ptr(), {static_cast<size_t>(D.size(0)), static_cast<size_t>(D.size(1))}, D_type,
      D_amax.data_ptr(), D_scale.data_ptr(), nullptr);
  auto te_bias =
      makeTransformerEngineTensor(bias.data_ptr(), {static_cast<size_t>(bias.size(0))}, bias_type);

  const auto gelu_shape = pre_gelu_out.data_ptr() == nullptr
                              ? std::vector<size_t>{static_cast<size_t>(pre_gelu_out.size(0))}
                              : std::vector<size_t>{static_cast<size_t>(pre_gelu_out.size(0)),
                                                    static_cast<size_t>(pre_gelu_out.size(1))};
  auto te_pre_gelu_out = makeTransformerEngineTensor(
      pre_gelu_out.data_ptr(), gelu_shape, GetTransformerEngineDType(pre_gelu_out.scalar_type()));
  auto te_workspace =
      makeTransformerEngineTensor(workspace.data_ptr(), {workspaceSize}, DType::kByte);

  nvte_cublas_gemm(te_A.data(), te_B.data(), te_D.data(), te_bias.data(), te_pre_gelu_out.data(),
                   transa, transb, grad, te_workspace.data(), accumulate, use_split_accumulator,
                   math_sm_count, at::cuda::getCurrentCUDAStream());
}

void te_atomic_gemm(at::Tensor A, at::Tensor A_scale_inverse, transformer_engine::DType A_type,
                    bool transa, at::Tensor B, at::Tensor B_scale_inverse,
                    transformer_engine::DType B_type, bool transb, at::Tensor D, at::Tensor D_scale,
                    transformer_engine::DType D_type, at::Tensor D_amax, at::Tensor bias,
                    transformer_engine::DType bias_type, at::Tensor pre_gelu_out, bool grad,
                    at::Tensor workspace, size_t workspaceSize, bool accumulate,
                    bool use_split_accumulator, int math_sm_count, int m_split, int n_split,
                    bool gemm_producer, at::Tensor counter) {
  using namespace transformer_engine;
  auto te_A = makeTransformerEngineTensor(
      A.data_ptr(), {static_cast<size_t>(A.size(0)), static_cast<size_t>(A.size(1))}, A_type,
      nullptr, nullptr, A_scale_inverse.data_ptr());
  auto te_B = makeTransformerEngineTensor(
      B.data_ptr(), {static_cast<size_t>(B.size(0)), static_cast<size_t>(B.size(1))}, B_type,
      nullptr, nullptr, B_scale_inverse.data_ptr());
  auto te_D = makeTransformerEngineTensor(
      D.data_ptr(), {static_cast<size_t>(D.size(0)), static_cast<size_t>(D.size(1))}, D_type,
      D_amax.data_ptr(), D_scale.data_ptr(), nullptr);
  auto te_bias =
      makeTransformerEngineTensor(bias.data_ptr(), {static_cast<size_t>(bias.size(0))}, bias_type);
  auto te_counter = makeTransformerEngineTensor(
      counter.data_ptr(), {static_cast<size_t>(counter.size(0))}, DType::kInt32);

  const auto gelu_shape = pre_gelu_out.data_ptr() == nullptr
                              ? std::vector<size_t>{static_cast<size_t>(pre_gelu_out.size(0))}
                              : std::vector<size_t>{static_cast<size_t>(pre_gelu_out.size(0)),
                                                    static_cast<size_t>(pre_gelu_out.size(1))};
  auto te_pre_gelu_out = makeTransformerEngineTensor(
      pre_gelu_out.data_ptr(), gelu_shape, GetTransformerEngineDType(pre_gelu_out.scalar_type()));
  auto te_workspace =
      makeTransformerEngineTensor(workspace.data_ptr(), {workspaceSize}, DType::kByte);

  nvte_cublas_atomic_gemm(te_A.data(), te_B.data(), te_D.data(), te_bias.data(),
                          te_pre_gelu_out.data(), transa, transb, grad, te_workspace.data(),
                          accumulate, use_split_accumulator, math_sm_count, m_split, n_split,
                          gemm_producer, te_counter.data(), at::cuda::getCurrentCUDAStream());
}

void te_grouped_gemm(std::vector<at::Tensor> A, at::Tensor A_scale_inverse, int A_offset,
                     transformer_engine::DType A_type, bool transa, std::vector<at::Tensor> B,
                     at::Tensor B_scale_inverse, int B_offset, transformer_engine::DType B_type,
                     bool transb, std::vector<at::Tensor> D, int D_offset, at::Tensor D_scale,
                     transformer_engine::DType D_type, at::Tensor D_amax,
                     std::vector<at::Tensor> bias, transformer_engine::DType bias_type,
                     std::vector<at::Tensor> pre_gelu_out, bool grad,
                     std::vector<at::Tensor> workspace, size_t workspaceSize, bool accumulate,
                     bool use_split_accumulator, int math_sm_count) {
  using namespace transformer_engine;
  std::vector<NVTETensor> te_A, te_B, te_D, te_bias, te_pre_gelu_out, te_workspace;
  std::vector<transformer_engine::TensorWrapper> tensor_wrappers;
  auto make_tensor = [&tensor_wrappers](void* dptr, const std::vector<size_t>& shape,
                                        transformer_engine::DType dtype, void* amax_dptr,
                                        void* scale_dptr, void* scale_inv_dptr) -> NVTETensor {
    tensor_wrappers.emplace_back(
        makeTransformerEngineTensor(dptr, shape, dtype, amax_dptr, scale_dptr, scale_inv_dptr));
    return tensor_wrappers.back().data();
  };
  for (size_t i = 0; i < A.size(); i++) {
    if (A[i].numel() == 0 || B[i].numel() == 0) {
      if (D[i].numel() != 0 && !accumulate) D[i].zero_();
      if (bias[i].numel() != 0 && grad) {
        if (B[i].numel() == 0) {
          bias[i].zero_();
        } else {
          bias[i].copy_(B[i].sum(0));
        }
      }
      if (pre_gelu_out[i].numel() != 0) pre_gelu_out[i].zero_();
      continue;
    }

    NVTE_CHECK(A[i].is_contiguous(), "A[", i, "] must be contiguous.");
    NVTE_CHECK(B[i].is_contiguous(), "B[", i, "] must be contiguous.");
    NVTE_CHECK(D[i].is_contiguous(), "D[", i, "] must be contiguous.");

    te_A.emplace_back(make_tensor(
        A[i].data_ptr(), {static_cast<size_t>(A[i].size(0)), static_cast<size_t>(A[i].size(1))},
        A_type, nullptr, nullptr, getDataPtr(A_scale_inverse, A_offset + i)));
    te_B.emplace_back(make_tensor(
        B[i].data_ptr(), {static_cast<size_t>(B[i].size(0)), static_cast<size_t>(B[i].size(1))},
        B_type, nullptr, nullptr, getDataPtr(B_scale_inverse, B_offset + i)));
    te_D.emplace_back(make_tensor(
        D[i].data_ptr(), {static_cast<size_t>(D[i].size(0)), static_cast<size_t>(D[i].size(1))},
        D_type, getDataPtr(D_amax, D_offset + i), getDataPtr(D_scale, D_offset + i), nullptr));
    te_bias.emplace_back(make_tensor(bias[i].data_ptr(), {static_cast<size_t>(bias[i].size(0))},
                                     bias_type, nullptr, nullptr, nullptr));

    const auto gelu_shape = pre_gelu_out[i].data_ptr() == nullptr
                                ? std::vector<size_t>{static_cast<size_t>(pre_gelu_out[i].size(0))}
                                : std::vector<size_t>{static_cast<size_t>(pre_gelu_out[i].size(0)),
                                                      static_cast<size_t>(pre_gelu_out[i].size(1))};
    te_pre_gelu_out.emplace_back(make_tensor(
        pre_gelu_out[i].data_ptr(), gelu_shape,
        GetTransformerEngineDType(pre_gelu_out[i].scalar_type()), nullptr, nullptr, nullptr));
  }
  for (size_t i = 0; i < workspace.size(); i++) {
    te_workspace.emplace_back(make_tensor(workspace[i].data_ptr(), {workspaceSize}, DType::kByte,
                                          nullptr, nullptr, nullptr));
  }

  // For now, we only have multi-stream cublas backend.
  nvte_multi_stream_cublas_gemm(te_A.data(), te_B.data(), te_D.data(), te_bias.data(),
                                te_pre_gelu_out.data(), te_A.size(), transa, transb, grad,
                                te_workspace.data(), accumulate, use_split_accumulator,
                                math_sm_count, at::cuda::getCurrentCUDAStream());
}

void te_grouped_gemm_single_output(
    std::vector<at::Tensor> A, std::vector<at::Tensor> A_scale_inverse, int A_offset,
    transformer_engine::DType A_type, bool transa, std::vector<at::Tensor> B,
    at::Tensor B_scale_inverse, int B_offset, transformer_engine::DType B_type, bool transb,
    std::vector<int64_t> m_splits, at::Tensor D, int D_offset, at::Tensor D_scale,
    transformer_engine::DType D_type, at::Tensor D_amax, std::vector<at::Tensor> bias,
    transformer_engine::DType bias_type, std::vector<at::Tensor> pre_gelu_out, bool grad,
    std::vector<at::Tensor> workspace, size_t workspaceSize, bool accumulate,
    bool use_split_accumulator, int math_sm_count) {
  using namespace transformer_engine;
  std::vector<NVTETensor> te_A, te_B, te_D, te_bias, te_pre_gelu_out, te_workspace;
  std::vector<transformer_engine::TensorWrapper> tensor_wrappers;
  auto make_tensor = [&tensor_wrappers](void* dptr, const std::vector<size_t>& shape,
                                        transformer_engine::DType dtype, void* amax_dptr,
                                        void* scale_dptr, void* scale_inv_dptr) -> NVTETensor {
    tensor_wrappers.emplace_back(
        makeTransformerEngineTensor(dptr, shape, dtype, amax_dptr, scale_dptr, scale_inv_dptr));
    return tensor_wrappers.back().data();
  };
  NVTE_CHECK(D.is_contiguous(), "D must be contiguous.");
  void* d_i_ptr = reinterpret_cast<void*>(D.data_ptr());
  for (size_t i = 0; i < A.size(); i++) {
    if (m_splits[i] == 0) continue;
    NVTE_CHECK(A[i].data_ptr() != nullptr, "A[", i, "] must not be nullptr.");
    NVTE_CHECK(B[i].data_ptr() != nullptr, "B[", i, "] must not be nullptr.");
    NVTE_CHECK(A[i].is_contiguous(), "A[", i, "] must be contiguous.");
    NVTE_CHECK(B[i].is_contiguous(), "B[", i, "] must be contiguous.");
    te_A.emplace_back(make_tensor(
        A[i].data_ptr(), {static_cast<size_t>(A[i].size(0)), static_cast<size_t>(A[i].size(1))},
        A_type, nullptr, nullptr, getDataPtr(A_scale_inverse[i], A_offset)));
    te_B.emplace_back(make_tensor(
        B[i].data_ptr(), {static_cast<size_t>(B[i].size(0)), static_cast<size_t>(B[i].size(1))},
        B_type, nullptr, nullptr, getDataPtr(B_scale_inverse, B_offset + i)));
    te_D.emplace_back(make_tensor(
        d_i_ptr, {static_cast<size_t>(m_splits[i]), static_cast<size_t>(A[i].size(0))}, D_type,
        getDataPtr(D_amax, D_offset + i), getDataPtr(D_scale, D_offset + i), nullptr));
    te_bias.emplace_back(make_tensor(bias[i].data_ptr(), {static_cast<size_t>(bias[i].size(0))},
                                     bias_type, nullptr, nullptr, nullptr));

    const auto gelu_shape = pre_gelu_out[i].data_ptr() == nullptr
                                ? std::vector<size_t>{static_cast<size_t>(pre_gelu_out[i].size(0))}
                                : std::vector<size_t>{static_cast<size_t>(pre_gelu_out[i].size(0)),
                                                      static_cast<size_t>(pre_gelu_out[i].size(1))};
    te_pre_gelu_out.emplace_back(make_tensor(
        pre_gelu_out[i].data_ptr(), gelu_shape,
        GetTransformerEngineDType(pre_gelu_out[i].scalar_type()), nullptr, nullptr, nullptr));
    // Move the D pointer to the next split.
    char* char_ptr = reinterpret_cast<char*>(d_i_ptr);
    char_ptr += m_splits[i] * A[i].size(0) * D.element_size();
    d_i_ptr = reinterpret_cast<void*>(char_ptr);
  }
  for (size_t i = 0; i < workspace.size(); i++) {
    te_workspace.emplace_back(make_tensor(workspace[i].data_ptr(), {workspaceSize}, DType::kByte,
                                          nullptr, nullptr, nullptr));
  }

  // For now, we only have multi-stream cublas backend.
  nvte_multi_stream_cublas_gemm(te_A.data(), te_B.data(), te_D.data(), te_bias.data(),
                                te_pre_gelu_out.data(), te_A.size(), transa, transb, grad,
                                te_workspace.data(), accumulate, use_split_accumulator,
                                math_sm_count, at::cuda::getCurrentCUDAStream());
}
