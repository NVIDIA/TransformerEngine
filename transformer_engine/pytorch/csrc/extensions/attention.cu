#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "extensions.h"

constexpr int block_size = 512;
constexpr int ctas_per_sm = 4;

// get the fused attention backend
NVTE_Fused_Attn_Backend get_fused_attn_backend(
                const transformer_engine::DType q_dtype,
                const transformer_engine::DType kv_dtype,
                NVTE_QKV_Layout qkv_layout,
                NVTE_Bias_Type bias_type,
                NVTE_Mask_Type attn_mask_type,
                float p_dropout, size_t max_seqlen_q,
                size_t max_seqlen_kv, size_t head_dim) {
  NVTE_Fused_Attn_Backend fused_attention_backend =
          nvte_get_fused_attn_backend(
                          static_cast<NVTEDType>(q_dtype), static_cast<NVTEDType>(kv_dtype),
                          qkv_layout, bias_type, attn_mask_type,
                          p_dropout, max_seqlen_q, max_seqlen_kv, head_dim);
  return fused_attention_backend;
}

// fast zero-fills of tensors
template <typename scalar_t>
__global__ void __launch_bounds__(block_size) mha_fill_kernel(scalar_t* out_tensor,
                const int32_t* const start_row,
                const size_t num_rows) {
  size_t row_stride = gridDim.y * blockDim.x;
  size_t row_index = blockIdx.x + static_cast<size_t>(start_row[0]);
  size_t col_index = blockIdx.y * blockDim.x + threadIdx.x;
  while (row_index < num_rows) {
    out_tensor[row_index*row_stride + col_index] = 0;
    row_index += gridDim.x;
  }
}

// fast zero-fills of tensors
void mha_fill(const at::Tensor &self, const at::Tensor &start_index) {
  auto max_tokens = self.size(0);
  auto self_2d = self.view({max_tokens, -1});
  auto fcd_size = self_2d.size(1);
  TORCH_CHECK(self.is_contiguous(), "input not contiguous");
  TORCH_CHECK(fcd_size % block_size == 0, "input size not aligned to block size");
  const int num_mp = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  uint64_t num_blk_y = (uint64_t)(fcd_size / block_size);
  uint64_t num_blk_x = (uint64_t)((num_mp * ctas_per_sm + num_blk_y - 1) / num_blk_y);
  dim3 dim_grid(num_blk_x, num_blk_y);
  dim3 dim_block(block_size);
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
          at::ScalarType::Half, at::ScalarType::BFloat16,
          self_2d.scalar_type(), "mha_fill", [&]() {
          mha_fill_kernel<<<dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                  self_2d.data_ptr<scalar_t>(),
                  static_cast<int32_t*>(start_index.data_ptr()),
                  max_tokens);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
          });
}

// extract seed and offset from PhiloxCudaState
__global__ void unpack(at::PhiloxCudaState arg, int64_t* rng_state_ptr) {
  if (arg.captured_) {
    rng_state_ptr[0] = static_cast<int64_t>(*arg.seed_.ptr);
    rng_state_ptr[1] = static_cast<int64_t>(
                    *(arg.offset_.ptr) + static_cast<int64_t>(arg.offset_intragraph_));
  } else {
    rng_state_ptr[0] = static_cast<int64_t>(arg.seed_.val);
    rng_state_ptr[1] = static_cast<int64_t>(arg.offset_.val);
  }
}

// extract PhiloxCudaState from CUDA random number generator
at::PhiloxCudaState init_philox_state(
                at::CUDAGeneratorImpl* gen,
                size_t elts_per_thread) {
  at::PhiloxCudaState philox_args;
  std::lock_guard<std::mutex> lock(gen->mutex_);
  philox_args = gen->philox_cuda_state(elts_per_thread);
  return philox_args;
}

// fused attention FWD with packed QKV
std::vector<at::Tensor> fused_attn_fwd_qkvpacked(
                size_t b, size_t max_seqlen, size_t total_seqs,
                size_t h, size_t d,
                bool is_training, float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens,
                const at::Tensor QKV,
                const transformer_engine::DType qkv_type,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_O,
                c10::optional<at::Tensor> amax_S,
                c10::optional<at::Tensor> amax_O,
                const c10::optional<at::Tensor> Bias,
                const c10::optional<at::Generator> rng_gen,
                size_t rng_elts_per_thread) {
  using namespace transformer_engine;

  // create output tensor O
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  auto O = torch::empty({static_cast<int64_t>(total_seqs),
                  static_cast<int64_t>(h), static_cast<int64_t>(d)}, options);
  if (set_zero && (h * d % block_size == 0)) {
    mha_fill(O, cu_seqlens.index({torch::indexing::Slice(-1, torch::indexing::None)}));
  } else {
    O.fill_(0);
  }

  // construct NVTE tensors
  TensorWrapper te_QKV, te_S, te_O, te_Bias, te_cu_seqlens;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if ((!descale_QKV.has_value()) || (!scale_S.has_value()) || (!scale_O.has_value())
                    || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, scale_S, scale_O, amax_S and amax_O";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), {total_seqs, 3, h, d},
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    at::Tensor descale_S = torch::empty_like(scale_S.value());
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_S.value().data_ptr(),
                    scale_S.value().data_ptr(), descale_S.data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs, h, d},
                    qkv_type, amax_O.value().data_ptr(), scale_O.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), {total_seqs, 3, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if ((bias_type != NVTE_NO_BIAS) && (Bias.has_value())) {
    auto bias_shape = Bias.value().sizes().vec();
    std::vector<size_t> shape{bias_shape.begin(), bias_shape.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), shape,
                    DType::kFloat32, nullptr, nullptr, nullptr);
  }
  te_cu_seqlens = makeTransformerEngineTensor(cu_seqlens.data_ptr(), {b+1},
                    DType::kInt32, nullptr, nullptr, nullptr);

  // extract random number generator seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
                  rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  at::PhiloxCudaState philox_args = init_philox_state(gen, rng_elts_per_thread);
  auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
                  philox_args, static_cast<int64_t*>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd_qkvpacked(
                  te_QKV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens.data(),
                  te_rng_state.data(),
                  max_seqlen,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    auto output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }

  // execute the kernel
  nvte_fused_attn_fwd_qkvpacked(
                  te_QKV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens.data(),
                  te_rng_state.data(),
                  max_seqlen,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, softmax-related tensors, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with packed QKV
std::vector<at::Tensor> fused_attn_bwd_qkvpacked(
                size_t b, size_t max_seqlen, size_t total_seqs,
                size_t h, size_t d,
                float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens,
                const at::Tensor QKV,
                const at::Tensor O,
                const at::Tensor dO,
                const transformer_engine::DType qkv_type,
                const std::vector<at::Tensor> Aux_CTX_Tensors,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> descale_S,
                const c10::optional<at::Tensor> descale_O,
                const c10::optional<at::Tensor> descale_dO,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_dP,
                const c10::optional<at::Tensor> scale_dQKV,
                c10::optional<at::Tensor> amax_dP,
                c10::optional<at::Tensor> amax_dQKV) {
  using namespace transformer_engine;

  // create output tensor dQKV
  at::Tensor dQKV = torch::empty_like(QKV);
  auto max_tokens = dQKV.size(0);
  auto self_2d = dQKV.view({max_tokens, -1});
  auto fcd_size = self_2d.size(1);
  if (set_zero && (fcd_size % block_size == 0)) {
    mha_fill(dQKV, cu_seqlens.index({torch::indexing::Slice(-1, torch::indexing::None)}));
  } else {
    dQKV.fill_(0);
  }
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  at::Tensor dBias;
  TensorWrapper te_dBias;
  if (bias_type != NVTE_NO_BIAS) {
    dBias = torch::zeros({1, static_cast<int64_t>(h),
                    static_cast<int64_t>(max_seqlen),
                    static_cast<int64_t>(max_seqlen)}, options);
    te_dBias = makeTransformerEngineTensor(dBias);
  }

  // construct NVTE tensors
  TensorWrapper te_QKV, te_O, te_dO, te_S, te_dP, te_dQKV;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if ((!descale_QKV.has_value()) || (!descale_S.has_value())
                    || (!descale_O.has_value()) || (!descale_dO.has_value())
                    || (!scale_S.has_value()) || (!scale_dP.has_value())
                    || (!scale_dQKV.has_value())
                    || (!amax_dP.has_value()) || (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, scale_S, scale_dP, ";
      err_tensors = err_tensors + std::string("scale_dQKV, amax_dP and amax_dQKV");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), {total_seqs, 3, h, d},
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs, h, d},
                    qkv_type, nullptr, nullptr, descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), {total_seqs, h, d},
                    qkv_type, nullptr, nullptr, descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32,
                    nullptr, scale_S.value().data_ptr(), descale_S.value().data_ptr());
    at::Tensor descale_dP = torch::empty_like(scale_dP.value());
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_dP.value().data_ptr(), scale_dP.value().data_ptr(),
                    descale_dP.data_ptr());
    te_dQKV = makeTransformerEngineTensor(dQKV.data_ptr(), {total_seqs, 3, h, d},
                    qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), {total_seqs, 3, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), {total_seqs, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQKV = makeTransformerEngineTensor(dQKV.data_ptr(), {total_seqs, 3, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }

  // convert auxiliary tensors from forward into NVTETensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create cu_seqlens tensorwrappers
  TensorWrapper te_cu_seqlens;
  te_cu_seqlens = makeTransformerEngineTensor(cu_seqlens.data_ptr(), {b+1},
                    DType::kInt32, nullptr, nullptr, nullptr);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd_qkvpacked(
                  te_QKV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens.data(),
                  max_seqlen,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd_qkvpacked(
                  te_QKV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens.data(),
                  max_seqlen,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQKV, dBias};
}

// fused attention FWD with packed KV
std::vector<at::Tensor> fused_attn_fwd_kvpacked(
                size_t b, size_t max_seqlen_q, size_t max_seqlen_kv,
                size_t total_seqs_q, size_t total_seqs_kv,
                size_t h, size_t d,
                bool is_training, float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor KV,
                const transformer_engine::DType qkv_type,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_O,
                c10::optional<at::Tensor> amax_S,
                c10::optional<at::Tensor> amax_O,
                const c10::optional<at::Tensor> Bias,
                const c10::optional<at::Generator> rng_gen,
                size_t rng_elts_per_thread) {
  using namespace transformer_engine;

  // create output tensor O
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  auto O = torch::empty({static_cast<int64_t>(total_seqs_q),
                  static_cast<int64_t>(h), static_cast<int64_t>(d)}, options);
  if (set_zero && (h * d % block_size == 0)) {
    mha_fill(O, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
  } else {
    O.fill_(0);
  }

  // construct NVTE tensors
  TensorWrapper te_Q, te_KV, te_S, te_O, te_Bias, te_cu_seqlens_q, te_cu_seqlens_kv;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if ((!descale_QKV.has_value()) || (!scale_S.has_value()) || (!scale_O.has_value())
                    || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, scale_S, scale_O, amax_S and amax_O";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), {total_seqs_kv, 2, h, d},
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    at::Tensor descale_S = torch::empty_like(scale_S.value());
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, amax_S.value().data_ptr(),
                    scale_S.value().data_ptr(), descale_S.data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, amax_O.value().data_ptr(), scale_O.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), {total_seqs_kv, 2, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if ((bias_type != NVTE_NO_BIAS) && (Bias.has_value())) {
    auto bias_shape = Bias.value().sizes().vec();
    std::vector<size_t> shape{bias_shape.begin(), bias_shape.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), shape,
                    DType::kFloat32, nullptr, nullptr, nullptr);
  }
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), {b+1},
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), {b+1},
                    DType::kInt32, nullptr, nullptr, nullptr);

  // extract rng seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
                  rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  at::PhiloxCudaState philox_args = init_philox_state(gen, rng_elts_per_thread);
  auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
                  philox_args, static_cast<int64_t*>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    auto output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }

  // execute the kernel
  nvte_fused_attn_fwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_Bias.data(),
                  te_S.data(),
                  te_O.data(),
                  &nvte_aux_tensor_pack,
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  te_rng_state.data(),
                  max_seqlen_q, max_seqlen_kv,
                  is_training, attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, softmax-related tensors, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with packed KV
std::vector<at::Tensor> fused_attn_bwd_kvpacked(
                size_t b, size_t max_seqlen_q, size_t max_seqlen_kv,
                size_t total_seqs_q, size_t total_seqs_kv,
                size_t h, size_t d,
                float attn_scale, float p_dropout, bool set_zero,
                NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
                const at::Tensor cu_seqlens_q,
                const at::Tensor cu_seqlens_kv,
                const at::Tensor Q,
                const at::Tensor KV,
                const at::Tensor O,
                const at::Tensor dO,
                const transformer_engine::DType qkv_type,
                const std::vector<at::Tensor> Aux_CTX_Tensors,
                const c10::optional<at::Tensor> descale_QKV,
                const c10::optional<at::Tensor> descale_S,
                const c10::optional<at::Tensor> descale_O,
                const c10::optional<at::Tensor> descale_dO,
                const c10::optional<at::Tensor> scale_S,
                const c10::optional<at::Tensor> scale_dP,
                const c10::optional<at::Tensor> scale_dQKV,
                c10::optional<at::Tensor> amax_dP,
                c10::optional<at::Tensor> amax_dQKV) {
  using namespace transformer_engine;

  // create output tensors dQ and dKV
  at::Tensor dQ = torch::empty_like(Q);
  at::Tensor dKV = torch::empty_like(KV);
  auto max_tokens_q = dQ.size(0);
  auto self_2d_q = dQ.view({max_tokens_q, -1});
  auto fcd_size_q = self_2d_q.size(1);
  auto max_tokens_kv = dQ.size(0);
  auto self_2d_kv = dQ.view({max_tokens_kv, -1});
  auto fcd_size_kv = self_2d_kv.size(1);
  if (set_zero && (fcd_size_q % block_size == 0) && (fcd_size_kv % block_size == 0)) {
    mha_fill(dQ, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    mha_fill(dKV, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
  } else {
    dQ.fill_(0);
    dKV.fill_(0);
  }
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  at::Tensor dBias;
  TensorWrapper te_dBias;
  if (bias_type != NVTE_NO_BIAS) {
    dBias = torch::zeros({1, static_cast<int64_t>(h),
                    static_cast<int64_t>(max_seqlen_q),
                    static_cast<int64_t>(max_seqlen_kv)}, options);
    te_dBias = makeTransformerEngineTensor(dBias);
  }

  // construct NVTE tensors
  TensorWrapper te_Q, te_KV, te_O, te_dO, te_S, te_dP, te_dQ, te_dKV;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if ((!descale_QKV.has_value()) || (!descale_S.has_value())
                    || (!descale_O.has_value()) || (!descale_dO.has_value())
                    || (!scale_S.has_value()) || (!scale_dP.has_value())
                    || (!scale_dQKV.has_value())
                    || (!amax_dP.has_value()) || (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, scale_S, scale_dP, ";
      err_tensors = err_tensors + std::string("scale_dQKV, amax_dP and amax_dQKV");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), {total_seqs_kv, 2, h, d},
                    qkv_type, nullptr, nullptr, descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr,
                    scale_S.value().data_ptr(), descale_S.value().data_ptr());
    at::Tensor descale_dP = torch::empty_like(scale_dP.value());
    te_dP = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32,
                    amax_dP.value().data_ptr(), scale_dP.value().data_ptr(),
                    descale_dP.data_ptr());
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), {total_seqs_q, h, d}, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
    te_dKV = makeTransformerEngineTensor(dKV.data_ptr(), {total_seqs_kv, 2, h, d}, qkv_type,
                    amax_dQKV.value().data_ptr(), scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), {total_seqs_kv, 2, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0},
                    DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQ = makeTransformerEngineTensor(dQ.data_ptr(), {total_seqs_q, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
    te_dKV = makeTransformerEngineTensor(dKV.data_ptr(), {total_seqs_kv, 2, h, d},
                    qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }

  // create cu_seqlens tensorwrappers
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), {b+1},
                    DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), {b+1},
                    DType::kInt32, nullptr, nullptr, nullptr);

  // convert auxiliary tensors from forward to NVTETensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor*>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace = makeTransformerEngineTensor(
                  workspace_data.data_ptr(),
                  workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd_kvpacked(
                  te_Q.data(),
                  te_KV.data(),
                  te_O.data(),
                  te_dO.data(),
                  te_S.data(),
                  te_dP.data(),
                  &nvte_aux_tensor_pack,
                  te_dQ.data(),
                  te_dKV.data(),
                  te_dBias.data(),
                  te_cu_seqlens_q.data(),
                  te_cu_seqlens_kv.data(),
                  max_seqlen_q, max_seqlen_kv,
                  attn_scale, p_dropout,
                  qkv_layout, bias_type, attn_mask_type,
                  workspace.data(),
                  at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQ, dKV, dBias};
}

namespace flash_attention {

constexpr int warp_size = 32;
constexpr int type_size = 2;  // FP16 or BF16
constexpr int nvec = sizeof(uint64_t) / type_size;
constexpr int load_size = warp_size * nvec;
constexpr int block_size = 512;

template <typename T>
__launch_bounds__(block_size)
__global__ void prepare_kernel_fwd(const T *qkvi,
                                   T *qkv,
                                   const size_t B,
                                   const size_t S,
                                   const size_t Z,
                                   const size_t W) {
    const int warpid = (blockDim.x * blockIdx.x + threadIdx.x) / warp_size;
    const int id_in_warp = threadIdx.x % warp_size;
    const size_t offset_input = blockIdx.y * W + warpid * 3 * W * Z + id_in_warp * nvec;
    const T *my_input = qkvi + offset_input;

    const size_t s = warpid / B;
    if (s >= S) return;

    const size_t b = warpid % B;

    const size_t offset_output = blockIdx.y * B * S * Z * W +
                                 (s + b * S) * W * Z +
                                 id_in_warp * nvec;

    T *my_output = qkv + offset_output;

    for (int i = 0; i < Z; ++i) {
        uint64_t *out = reinterpret_cast<uint64_t*>(my_output + i * load_size);
        *out = *reinterpret_cast<const uint64_t*>(my_input + i * load_size * 3);
    }
}

template <typename T>
__launch_bounds__(block_size)
__global__ void prepare_kernel_bwd(const T *q, const T *k, const T *v,
                                   T *qkv, const size_t B, const size_t S,
                                   const size_t Z, const size_t W) {
    const T *input = blockIdx.y == 0 ? q : (blockIdx.y == 1 ? k : v);

    const int warpid = (blockDim.x * blockIdx.x + threadIdx.x) / warp_size;
    const int id_in_warp = threadIdx.x % warp_size;
    const size_t offset_input = warpid * W * Z + id_in_warp * nvec;
    const T *my_input = input + offset_input;

    const size_t b = warpid / S;
    if (b >= B) return;

    const size_t s = warpid % S;

    const size_t offset_output = (b + s * B) * 3 * W * Z +
                                 id_in_warp * nvec + blockIdx.y * W;

    T *my_output = qkv + offset_output;

    for (int i = 0; i < Z; ++i) {
        uint64_t *out = reinterpret_cast<uint64_t*>(my_output + i * load_size * 3);
        *out = *reinterpret_cast<const uint64_t*>(my_input + i * load_size);
    }
}

}  // namespace flash_attention

at::Tensor fa_prepare_fwd(at::Tensor qkvi) {
    NVTE_CHECK(qkvi.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(qkvi.scalar_type() == at::ScalarType::Half ||
               qkvi.scalar_type() == at::ScalarType::BFloat16);
    NVTE_CHECK(qkvi.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(qkvi.size(3) == flash_attention::load_size);
    NVTE_CHECK(qkvi.stride(3) == 1, "Wrong stride.");
    NVTE_CHECK(qkvi.stride(2) == 3 * qkvi.size(3), "Wrong stride.");
    NVTE_CHECK(qkvi.stride(1) == 3 * qkvi.size(3) * qkvi.size(2), "Wrong stride.");
    NVTE_CHECK(qkvi.stride(0) == 3 * qkvi.size(3) * qkvi.size(2) * qkvi.size(1), "Wrong stride.");

    // [s, b, n, h * 3] -> [3, b, s, n, h]
    std::vector<int64_t> shape = {3, qkvi.size(1), qkvi.size(0), qkvi.size(2), qkvi.size(3)};
    at::Tensor qkv = at::empty(shape, at::CUDA(qkvi.scalar_type()));

    size_t warps = qkvi.size(0) * qkvi.size(1);
    size_t warps_per_block = flash_attention::block_size / flash_attention::warp_size;
    size_t blocks = (warps + warps_per_block - 1) / warps_per_block;
    dim3 grid(blocks, 3);
    int threads = flash_attention::block_size;
    if (qkvi.scalar_type() == at::ScalarType::Half) {
        using dtype = at::Half;
        flash_attention::prepare_kernel_fwd<dtype><<<grid, threads, 0,
                                                     at::cuda::getCurrentCUDAStream()>>>(
            qkvi.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            shape[1],
            shape[2],
            shape[3],
            shape[4]);
    } else {
        using dtype = at::BFloat16;
        flash_attention::prepare_kernel_fwd<dtype><<<grid, threads, 0,
                                                     at::cuda::getCurrentCUDAStream()>>>(
            qkvi.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            shape[1],
            shape[2],
            shape[3],
            shape[4]);
    }

    return qkv;
}

at::Tensor fa_prepare_bwd(at::Tensor q, at::Tensor k, at::Tensor v) {
    NVTE_CHECK(q.is_contiguous());
    NVTE_CHECK(k.is_contiguous());
    NVTE_CHECK(v.is_contiguous());
    NVTE_CHECK(q.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(k.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(v.dim() == 4, "Expected 4-dim tensor.");
    NVTE_CHECK(q.scalar_type() == at::ScalarType::Half ||
               q.scalar_type() == at::ScalarType::BFloat16);
    NVTE_CHECK(k.scalar_type() == q.scalar_type());
    NVTE_CHECK(v.scalar_type() == q.scalar_type());
    NVTE_CHECK(q.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(q.size(3) == flash_attention::load_size);
    NVTE_CHECK(k.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(k.size(3) == flash_attention::load_size);
    NVTE_CHECK(v.size(3) % flash_attention::load_size == 0);
    NVTE_CHECK(v.size(3) == flash_attention::load_size);

    // 3 x [s, b, n, h] -> [b, s, n, 3 * h]

    std::vector<int64_t> shape = {q.size(1), q.size(0), q.size(2), 3 * q.size(3)};
    at::Tensor qkv = at::empty(shape, at::CUDA(q.scalar_type()));

    size_t warps = q.size(0) * q.size(1);
    size_t warps_per_block = flash_attention::block_size / flash_attention::warp_size;
    size_t blocks = (warps + warps_per_block - 1) / warps_per_block;
    dim3 grid(blocks, 3);
    int threads = flash_attention::block_size;
    if (q.scalar_type() == at::ScalarType::Half) {
        using dtype = at::Half;
        flash_attention::prepare_kernel_bwd<dtype><<<grid, threads, 0,
                                                 at::cuda::getCurrentCUDAStream()>>>(
            q.data_ptr<dtype>(),
            k.data_ptr<dtype>(),
            v.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            q.size(0),
            q.size(1),
            q.size(2),
            q.size(3));
    } else {
        using dtype = at::BFloat16;
        flash_attention::prepare_kernel_bwd<dtype><<<grid, threads, 0,
                                                 at::cuda::getCurrentCUDAStream()>>>(
            q.data_ptr<dtype>(),
            k.data_ptr<dtype>(),
            v.data_ptr<dtype>(),
            qkv.data_ptr<dtype>(),
            q.size(0),
            q.size(1),
            q.size(2),
            q.size(3));
    }

    return qkv;
}
