#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "extensions.h"

constexpr int block_size = 512;
constexpr int ctas_per_sm = 4;

// get the fused attention backend
NVTE_Fused_Attn_Backend get_fused_attn_backend(
    const transformer_engine::DType q_dtype, const transformer_engine::DType kv_dtype,
    NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
    float p_dropout, size_t num_attn_heads, size_t num_gqa_groups, size_t max_seqlen_q,
    size_t max_seqlen_kv, size_t head_dim_qk, size_t head_dim_v, int64_t window_size_left,
    int64_t window_size_right) {
  NVTE_Fused_Attn_Backend fused_attention_backend = nvte_get_fused_attn_backend(
      static_cast<NVTEDType>(q_dtype), static_cast<NVTEDType>(kv_dtype), qkv_layout, bias_type,
      attn_mask_type, p_dropout, num_attn_heads, num_gqa_groups, max_seqlen_q, max_seqlen_kv,
      head_dim_qk, head_dim_v, window_size_left, window_size_right);
  return fused_attention_backend;
}

// fast zero-fills of tensors
template <typename scalar_t>
__global__ void __launch_bounds__(block_size)
    mha_fill_kernel(scalar_t *out_tensor, const int32_t *const start_row, const size_t num_rows) {
  size_t row_stride = gridDim.y * blockDim.x;
  size_t row_index = blockIdx.x + static_cast<size_t>(start_row[0]);
  size_t col_index = blockIdx.y * blockDim.x + threadIdx.x;
  while (row_index < num_rows) {
    out_tensor[row_index * row_stride + col_index] = 0;
    row_index += gridDim.x;
  }
}

// fast zero-fills of tensors
void mha_fill(const at::Tensor &self, const at::Tensor &start_index) {
  auto max_tokens = self.size(0);
  auto self_2d = self.view({max_tokens, -1});
  auto fcd_size = self_2d.size(1);
  TORCH_CHECK(self.is_contiguous(), "input not contiguous");
  TORCH_CHECK(fcd_size % block_size == 0, "input size not aligned to block size");
  const int num_mp = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  uint64_t num_blk_y = (uint64_t)(fcd_size / block_size);
  uint64_t num_blk_x = (uint64_t)((num_mp * ctas_per_sm + num_blk_y - 1) / num_blk_y);
  dim3 dim_grid(num_blk_x, num_blk_y);
  dim3 dim_block(block_size);
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16, self_2d.scalar_type(), "mha_fill", [&]() {
        mha_fill_kernel<<<dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
            self_2d.data_ptr<scalar_t>(), static_cast<int32_t *>(start_index.data_ptr()),
            max_tokens);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

// extract seed and offset from PhiloxCudaState
__global__ void unpack(at::PhiloxCudaState arg, int64_t *rng_state_ptr) {
  if (arg.captured_) {
    rng_state_ptr[0] = static_cast<int64_t>(*arg.seed_.ptr);
    rng_state_ptr[1] =
        static_cast<int64_t>(*(arg.offset_.ptr) + static_cast<int64_t>(arg.offset_intragraph_));
  } else {
    rng_state_ptr[0] = static_cast<int64_t>(arg.seed_.val);
    rng_state_ptr[1] = static_cast<int64_t>(arg.offset_.val);
  }
}

// extract PhiloxCudaState from CUDA random number generator
at::PhiloxCudaState init_philox_state(at::CUDAGeneratorImpl *gen, size_t elts_per_thread) {
  at::PhiloxCudaState philox_args;
  std::lock_guard<std::mutex> lock(gen->mutex_);
  philox_args = gen->philox_cuda_state(elts_per_thread);
  return philox_args;
}

// fused attention FWD with packed QKV
std::vector<at::Tensor> fused_attn_fwd_qkvpacked(
    size_t max_seqlen, bool is_training, float attn_scale, float p_dropout, bool set_zero,
    NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
    const std::vector<int64_t> window_size, const at::Tensor cu_seqlens, const at::Tensor QKV,
    const transformer_engine::DType qkv_type, const c10::optional<at::Tensor> cu_seqlens_padded,
    const c10::optional<at::Tensor> descale_QKV, const int descale_QKV_offset,
    const c10::optional<at::Tensor> descale_S, const int descale_S_offset,
    const c10::optional<at::Tensor> scale_S, const int scale_S_offset,
    const c10::optional<at::Tensor> scale_O, const int scale_O_offset,
    c10::optional<at::Tensor> amax_S, const int amax_S_offset, c10::optional<at::Tensor> amax_O,
    const int amax_O_offset, const c10::optional<at::Tensor> Bias,
    const c10::optional<at::Generator> rng_gen, size_t rng_elts_per_thread) {
  using namespace transformer_engine;

  auto qkv_sizes = QKV.sizes().vec();
  std::vector<size_t> qkv_shape{qkv_sizes.begin(), qkv_sizes.end()};
  std::vector<size_t> q_shape;
  NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(qkv_layout);
  int loc_3 = 0;
  switch (layout_group) {
    case NVTE_3HD:
      loc_3 = qkv_sizes.size() - 3;
      break;
    case NVTE_H3D:
      loc_3 = qkv_sizes.size() - 2;
      break;
    default:
      NVTE_ERROR("Invalid QKV layout group.");
  }
  for (auto it = qkv_shape.begin(); it != qkv_shape.end(); ++it) {
    if (it - qkv_shape.begin() != loc_3) {
      q_shape.push_back(*it);
    }
  }
  std::vector<int64_t> o_shape{q_shape.begin(), q_shape.end()};

  // create output tensor O
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  auto O = torch::empty(o_shape, options);

  // construct NVTE tensors
  TensorWrapper te_QKV, te_S, te_O, te_Bias, te_cu_seqlens, te_cu_seqlens_padded;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto h = q_shape[q_shape.size() - 2];
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0) &&
        (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD)) {
      mha_fill(O, cu_seqlens.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      O.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!descale_S.has_value()) || (!scale_S.has_value()) ||
        (!scale_O.has_value()) || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, scale_S, scale_O, amax_S and amax_O ";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape, qkv_type, nullptr, nullptr,
                                         getDataPtr(descale_QKV.value(), descale_QKV_offset));
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32,
                                       getDataPtr(amax_S.value(), amax_S_offset),
                                       getDataPtr(scale_S.value(), scale_S_offset),
                                       getDataPtr(descale_S.value(), descale_S_offset));
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape, qkv_type,
                                       getDataPtr(amax_O.value(), amax_O_offset),
                                       getDataPtr(scale_O.value(), scale_O_offset), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    if (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD) {
      O.fill_(0);
    }
    // BF16 or FP16
    te_QKV =
        makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape, qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape, qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
    auto bias_sizes = Bias.value().sizes().vec();
    std::vector<size_t> bias_shape{bias_sizes.begin(), bias_sizes.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), bias_shape, DType::kFloat32,
                                          nullptr, nullptr, nullptr);
  }
  auto cu_seqlens_sizes = cu_seqlens.sizes().vec();
  std::vector<size_t> cu_seqlens_shape{cu_seqlens_sizes.begin(), cu_seqlens_sizes.end()};
  te_cu_seqlens = makeTransformerEngineTensor(cu_seqlens.data_ptr(), cu_seqlens_shape,
                                              DType::kInt32, nullptr, nullptr, nullptr);

  if (cu_seqlens_padded.has_value()) {
    auto cu_seqlens_padded_sizes = cu_seqlens_padded.value().sizes().vec();
    std::vector<size_t> cu_seqlens_padded_shape{cu_seqlens_padded_sizes.begin(),
                                                cu_seqlens_padded_sizes.end()};
    te_cu_seqlens_padded =
        makeTransformerEngineTensor(cu_seqlens_padded.value().data_ptr(), cu_seqlens_padded_shape,
                                    DType::kInt32, nullptr, nullptr, nullptr);
  }

  // extract random number generator seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
      rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  at::PhiloxCudaState philox_args = init_philox_state(gen, rng_elts_per_thread);
  auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
      philox_args, static_cast<int64_t *>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd_qkvpacked(
      te_QKV.data(), te_Bias.data(), te_S.data(), te_O.data(), &nvte_aux_tensor_pack,
      te_cu_seqlens.data(), te_cu_seqlens_padded.data(), te_rng_state.data(), max_seqlen,
      is_training, attn_scale, p_dropout, qkv_layout, bias_type, attn_mask_type, window_size[0],
      window_size[1], workspace.data(), at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor *>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    at::Tensor output_tensor;
    if (nvte_aux_tensor_pack.size >= 2) {
      if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
        if (i < nvte_aux_tensor_pack.size - 2) {
          output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
        } else if (i == nvte_aux_tensor_pack.size - 2) {
          output_tensor = rng_state;
        } else if (i == nvte_aux_tensor_pack.size - 1) {
          output_tensor = Bias.value();
        }
      } else {
        output_tensor = (i < nvte_aux_tensor_pack.size - 1)
                            ? allocateSpace(tensor->data.shape, tensor->data.dtype, false)
                            : rng_state;
      }
    } else {
      output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    }
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }

  // execute the kernel
  nvte_fused_attn_fwd_qkvpacked(
      te_QKV.data(), te_Bias.data(), te_S.data(), te_O.data(), &nvte_aux_tensor_pack,
      te_cu_seqlens.data(), te_cu_seqlens_padded.data(), te_rng_state.data(), max_seqlen,
      is_training, attn_scale, p_dropout, qkv_layout, bias_type, attn_mask_type, window_size[0],
      window_size[1], workspace.data(), at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, softmax-related tensors, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with packed QKV
std::vector<at::Tensor> fused_attn_bwd_qkvpacked(
    size_t max_seqlen, float attn_scale, float p_dropout, bool set_zero, NVTE_QKV_Layout qkv_layout,
    NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type, const std::vector<int64_t> window_size,
    bool deterministic, const at::Tensor cu_seqlens, const at::Tensor QKV, const at::Tensor O,
    const at::Tensor dO, const transformer_engine::DType qkv_type,
    const transformer_engine::DType dqkv_type, const std::vector<at::Tensor> Aux_CTX_Tensors,
    const c10::optional<at::Tensor> cu_seqlens_padded, const c10::optional<at::Tensor> descale_QKV,
    const c10::optional<at::Tensor> descale_S, const c10::optional<at::Tensor> descale_O,
    const c10::optional<at::Tensor> descale_dO, const c10::optional<at::Tensor> descale_dP,
    const c10::optional<at::Tensor> scale_S, const c10::optional<at::Tensor> scale_dP,
    const c10::optional<at::Tensor> scale_dQKV, c10::optional<at::Tensor> amax_dP,
    c10::optional<at::Tensor> amax_dQKV) {
  using namespace transformer_engine;

  auto qkv_sizes = QKV.sizes().vec();
  std::vector<size_t> qkv_shape{qkv_sizes.begin(), qkv_sizes.end()};
  std::vector<size_t> q_shape;
  NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(qkv_layout);
  int loc_3 = 0;
  switch (layout_group) {
    case NVTE_3HD:
      loc_3 = qkv_sizes.size() - 3;
      break;
    case NVTE_H3D:
      loc_3 = qkv_sizes.size() - 2;
      break;
    default:
      NVTE_ERROR("Invalid QKV layout group.");
  }
  for (auto it = qkv_shape.begin(); it != qkv_shape.end(); ++it) {
    if (it - qkv_shape.begin() != loc_3) {
      q_shape.push_back(*it);
    }
  }
  auto h = q_shape[q_shape.size() - 2];

  // create output tensor dQKV
  auto options = torch::TensorOptions().dtype(GetATenDType(dqkv_type)).device(torch::kCUDA);
  at::Tensor dQKV = torch::empty_like(QKV, options);

  // construct NVTE tensors
  TensorWrapper te_QKV, te_O, te_dO, te_S, te_dP, te_dQKV;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0) &&
        (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD)) {
      mha_fill(dQKV, cu_seqlens.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      dQKV.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!descale_S.has_value()) || (!descale_O.has_value()) ||
        (!descale_dO.has_value()) || (!descale_dP.has_value()) || (!scale_S.has_value()) ||
        (!scale_dP.has_value()) || (!scale_dQKV.has_value()) || (!amax_dP.has_value()) ||
        (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, descale_dO, descale_dP, ";
      err_tensors = err_tensors + std::string("scale_S, scale_dP, scale_dQKV, ");
      err_tensors = err_tensors + std::string("amax_dP and amax_dQKV ");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_QKV = makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape, qkv_type, nullptr, nullptr,
                                         descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape, qkv_type, nullptr, nullptr,
                                       descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape, dqkv_type, nullptr, nullptr,
                                        descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr,
                                       scale_S.value().data_ptr(), descale_S.value().data_ptr());
    te_dP = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, amax_dP.value().data_ptr(),
                                        scale_dP.value().data_ptr(), descale_dP.value().data_ptr());
    te_dQKV = makeTransformerEngineTensor(dQKV.data_ptr(), qkv_shape, dqkv_type,
                                          amax_dQKV.value().data_ptr(),
                                          scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    if (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD) {
      dQKV.fill_(0);
    }
    // BF16 or FP16
    te_QKV =
        makeTransformerEngineTensor(QKV.data_ptr(), qkv_shape, qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape, qkv_type, nullptr, nullptr, nullptr);
    te_dO =
        makeTransformerEngineTensor(dO.data_ptr(), q_shape, dqkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQKV = makeTransformerEngineTensor(dQKV.data_ptr(), qkv_shape, dqkv_type, nullptr, nullptr,
                                          nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }

  // convert auxiliary tensors from forward into NVTETensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor *>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create dBias the same shape as Bias
  at::Tensor dBias;
  TensorWrapper te_dBias;
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI)) {
    if (nvte_aux_tensor_pack.size >= 2) {
      std::vector<int64_t> bias_shape(Aux_CTX_Tensors[nvte_aux_tensor_pack.size - 1].sizes().vec());
      dBias = torch::empty(bias_shape, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    } else {
      dBias = torch::empty({1, static_cast<int64_t>(h), static_cast<int64_t>(max_seqlen),
                            static_cast<int64_t>(max_seqlen)},
                           options);
      te_dBias = makeTransformerEngineTensor(dBias);
    }
    if (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD) {
      dBias.fill_(0);
    }
  }

  // create cu_seqlens tensorwrappers
  auto cu_seqlens_sizes = cu_seqlens.sizes().vec();
  std::vector<size_t> cu_seqlens_shape{cu_seqlens_sizes.begin(), cu_seqlens_sizes.end()};
  TensorWrapper te_cu_seqlens = makeTransformerEngineTensor(
      cu_seqlens.data_ptr(), cu_seqlens_shape, DType::kInt32, nullptr, nullptr, nullptr);

  TensorWrapper te_cu_seqlens_padded;
  if (cu_seqlens_padded.has_value()) {
    auto cu_seqlens_padded_sizes = cu_seqlens_padded.value().sizes().vec();
    std::vector<size_t> cu_seqlens_padded_shape{cu_seqlens_padded_sizes.begin(),
                                                cu_seqlens_padded_sizes.end()};
    te_cu_seqlens_padded =
        makeTransformerEngineTensor(cu_seqlens_padded.value().data_ptr(), cu_seqlens_padded_shape,
                                    DType::kInt32, nullptr, nullptr, nullptr);
  }

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd_qkvpacked(
      te_QKV.data(), te_O.data(), te_dO.data(), te_S.data(), te_dP.data(), &nvte_aux_tensor_pack,
      te_dQKV.data(), te_dBias.data(), te_cu_seqlens.data(), te_cu_seqlens_padded.data(),
      max_seqlen, attn_scale, p_dropout, qkv_layout, bias_type, attn_mask_type, window_size[0],
      window_size[1], deterministic, workspace.data(), at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd_qkvpacked(
      te_QKV.data(), te_O.data(), te_dO.data(), te_S.data(), te_dP.data(), &nvte_aux_tensor_pack,
      te_dQKV.data(), te_dBias.data(), te_cu_seqlens.data(), te_cu_seqlens_padded.data(),
      max_seqlen, attn_scale, p_dropout, qkv_layout, bias_type, attn_mask_type, window_size[0],
      window_size[1], deterministic, workspace.data(), at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQKV, dBias};
}

// fused attention FWD with packed KV
std::vector<at::Tensor> fused_attn_fwd_kvpacked(
    size_t max_seqlen_q, size_t max_seqlen_kv, bool is_training, float attn_scale, float p_dropout,
    bool set_zero, NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type,
    NVTE_Mask_Type attn_mask_type, const std::vector<int64_t> window_size,
    const at::Tensor cu_seqlens_q, const at::Tensor cu_seqlens_kv, const at::Tensor Q,
    const at::Tensor KV, const transformer_engine::DType qkv_type,
    const c10::optional<at::Tensor> cu_seqlens_q_padded,
    const c10::optional<at::Tensor> cu_seqlens_kv_padded,
    const c10::optional<at::Tensor> descale_QKV, const int descale_QKV_offset,
    const c10::optional<at::Tensor> descale_S, const int descale_S_offset,
    const c10::optional<at::Tensor> scale_S, const int scale_S_offset,
    const c10::optional<at::Tensor> scale_O, const int scale_O_offset,
    c10::optional<at::Tensor> amax_S, const int amax_S_offset, c10::optional<at::Tensor> amax_O,
    const int amax_O_offset, const c10::optional<at::Tensor> Bias,
    const c10::optional<at::Generator> rng_gen, size_t rng_elts_per_thread) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto kv_sizes = KV.sizes().vec();
  std::vector<size_t> kv_shape{kv_sizes.begin(), kv_sizes.end()};
  std::vector<int64_t> o_shape{q_shape.begin(), q_shape.end()};

  // create output tensor O
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  auto O = torch::empty(o_shape, options);

  // construct NVTE tensors
  TensorWrapper te_Q, te_KV, te_S, te_O, te_Bias, te_cu_seqlens_q, te_cu_seqlens_kv;
  TensorWrapper te_cu_seqlens_q_padded, te_cu_seqlens_kv_padded;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto h = q_shape[q_shape.size() - 2];
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0) &&
        (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD)) {
      mha_fill(O, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      O.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!descale_S.has_value()) || (!scale_S.has_value()) ||
        (!scale_O.has_value()) || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, scale_S, scale_O, amax_S and amax_O ";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape, qkv_type, nullptr, nullptr,
                                       getDataPtr(descale_QKV.value(), descale_QKV_offset));
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), kv_shape, qkv_type, nullptr, nullptr,
                                        getDataPtr(descale_QKV.value(), descale_QKV_offset));
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32,
                                       getDataPtr(amax_S.value(), amax_S_offset),
                                       getDataPtr(scale_S.value(), scale_S_offset),
                                       getDataPtr(descale_S.value(), descale_S_offset));
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape, qkv_type,
                                       getDataPtr(amax_O.value(), amax_O_offset),
                                       getDataPtr(scale_O.value(), scale_O_offset), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    if (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD) {
      O.fill_(0);
    }
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape, qkv_type, nullptr, nullptr, nullptr);
    te_KV =
        makeTransformerEngineTensor(KV.data_ptr(), kv_shape, qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape, qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
    auto bias_sizes = Bias.value().sizes().vec();
    std::vector<size_t> bias_shape{bias_sizes.begin(), bias_sizes.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), bias_shape, DType::kFloat32,
                                          nullptr, nullptr, nullptr);
  }
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                                                DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                                                 DType::kInt32, nullptr, nullptr, nullptr);

  if ((cu_seqlens_q_padded.has_value()) && (cu_seqlens_kv_padded.has_value())) {
    auto cu_seqlens_q_padded_sizes = cu_seqlens_q_padded.value().sizes().vec();
    std::vector<size_t> cu_seqlens_q_padded_shape{cu_seqlens_q_padded_sizes.begin(),
                                                  cu_seqlens_q_padded_sizes.end()};
    auto cu_seqlens_kv_padded_sizes = cu_seqlens_kv_padded.value().sizes().vec();
    std::vector<size_t> cu_seqlens_kv_padded_shape{cu_seqlens_kv_padded_sizes.begin(),
                                                   cu_seqlens_kv_padded_sizes.end()};
    te_cu_seqlens_q_padded = makeTransformerEngineTensor(cu_seqlens_q_padded.value().data_ptr(),
                                                         cu_seqlens_q_padded_shape, DType::kInt32,
                                                         nullptr, nullptr, nullptr);
    te_cu_seqlens_kv_padded = makeTransformerEngineTensor(cu_seqlens_kv_padded.value().data_ptr(),
                                                          cu_seqlens_kv_padded_shape, DType::kInt32,
                                                          nullptr, nullptr, nullptr);
  }

  // extract rng seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
      rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  at::PhiloxCudaState philox_args = init_philox_state(gen, rng_elts_per_thread);
  auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
      philox_args, static_cast<int64_t *>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd_kvpacked(
      te_Q.data(), te_KV.data(), te_Bias.data(), te_S.data(), te_O.data(), &nvte_aux_tensor_pack,
      te_cu_seqlens_q.data(), te_cu_seqlens_kv.data(), te_cu_seqlens_q_padded.data(),
      te_cu_seqlens_kv_padded.data(), te_rng_state.data(), max_seqlen_q, max_seqlen_kv, is_training,
      attn_scale, p_dropout, qkv_layout, bias_type, attn_mask_type, window_size[0], window_size[1],
      workspace.data(), at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor *>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    at::Tensor output_tensor;
    if (nvte_aux_tensor_pack.size >= 2) {
      if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
        if (i < nvte_aux_tensor_pack.size - 2) {
          output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
        } else if (i == nvte_aux_tensor_pack.size - 2) {
          output_tensor = rng_state;
        } else if (i == nvte_aux_tensor_pack.size - 1) {
          output_tensor = Bias.value();
        }
      } else {
        output_tensor = (i < nvte_aux_tensor_pack.size - 1)
                            ? allocateSpace(tensor->data.shape, tensor->data.dtype, false)
                            : rng_state;
      }
    } else {
      output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    }
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }

  // execute the kernel
  nvte_fused_attn_fwd_kvpacked(
      te_Q.data(), te_KV.data(), te_Bias.data(), te_S.data(), te_O.data(), &nvte_aux_tensor_pack,
      te_cu_seqlens_q.data(), te_cu_seqlens_kv.data(), te_cu_seqlens_q_padded.data(),
      te_cu_seqlens_kv_padded.data(), te_rng_state.data(), max_seqlen_q, max_seqlen_kv, is_training,
      attn_scale, p_dropout, qkv_layout, bias_type, attn_mask_type, window_size[0], window_size[1],
      workspace.data(), at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, softmax-related tensors, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with packed KV
std::vector<at::Tensor> fused_attn_bwd_kvpacked(
    size_t max_seqlen_q, size_t max_seqlen_kv, float attn_scale, float p_dropout, bool set_zero,
    NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
    const std::vector<int64_t> window_size, bool deterministic, const at::Tensor cu_seqlens_q,
    const at::Tensor cu_seqlens_kv, const at::Tensor Q, const at::Tensor KV, const at::Tensor O,
    const at::Tensor dO, const transformer_engine::DType qkv_type,
    const transformer_engine::DType dqkv_type, const std::vector<at::Tensor> Aux_CTX_Tensors,
    const c10::optional<at::Tensor> cu_seqlens_q_padded,
    const c10::optional<at::Tensor> cu_seqlens_kv_padded,
    const c10::optional<at::Tensor> descale_QKV, const c10::optional<at::Tensor> descale_S,
    const c10::optional<at::Tensor> descale_O, const c10::optional<at::Tensor> descale_dO,
    const c10::optional<at::Tensor> descale_dP, const c10::optional<at::Tensor> scale_S,
    const c10::optional<at::Tensor> scale_dP, const c10::optional<at::Tensor> scale_dQKV,
    c10::optional<at::Tensor> amax_dP, c10::optional<at::Tensor> amax_dQKV) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto kv_sizes = KV.sizes().vec();
  std::vector<size_t> kv_shape{kv_sizes.begin(), kv_sizes.end()};
  std::vector<size_t> k_shape;
  for (auto i : kv_shape) {
    if (i != 2) {
      k_shape.push_back(i);
    }
  }
  auto h_q = q_shape[q_shape.size() - 2];
  auto h_kv = k_shape[k_shape.size() - 2];
  auto d = q_shape[q_shape.size() - 1];

  // create output tensors dQ and dKV
  auto options = torch::TensorOptions().dtype(GetATenDType(dqkv_type)).device(torch::kCUDA);
  at::Tensor dQ = torch::empty_like(Q, options);
  at::Tensor dKV = torch::empty_like(KV, options);

  // construct NVTE tensors
  TensorWrapper te_Q, te_KV, te_O, te_dO, te_S, te_dP, te_dQ, te_dKV;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if (set_zero && ((h_q * d) % block_size == 0) && ((h_kv * d) % block_size == 0) &&
        (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD)) {
      mha_fill(dQ, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
      mha_fill(dKV, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      dQ.fill_(0);
      dKV.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!descale_S.has_value()) || (!descale_O.has_value()) ||
        (!descale_dO.has_value()) || (!descale_dP.has_value()) || (!scale_S.has_value()) ||
        (!scale_dP.has_value()) || (!scale_dQKV.has_value()) || (!amax_dP.has_value()) ||
        (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, descale_dO, descale_dP, ";
      err_tensors = err_tensors + std::string("scale_S, scale_dP, scale_dQKV, ");
      err_tensors = err_tensors + std::string("amax_dP and amax_dQKV ");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape, qkv_type, nullptr, nullptr,
                                       descale_QKV.value().data_ptr());
    te_KV = makeTransformerEngineTensor(KV.data_ptr(), kv_shape, qkv_type, nullptr, nullptr,
                                        descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape, qkv_type, nullptr, nullptr,
                                       descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), q_shape, dqkv_type, nullptr, nullptr,
                                        descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr,
                                       scale_S.value().data_ptr(), descale_S.value().data_ptr());
    te_dP = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, amax_dP.value().data_ptr(),
                                        scale_dP.value().data_ptr(), descale_dP.value().data_ptr());
    te_dQ =
        makeTransformerEngineTensor(dQ.data_ptr(), q_shape, dqkv_type, amax_dQKV.value().data_ptr(),
                                    scale_dQKV.value().data_ptr(), nullptr);
    te_dKV = makeTransformerEngineTensor(dKV.data_ptr(), kv_shape, dqkv_type,
                                         amax_dQKV.value().data_ptr(),
                                         scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    if (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD) {
      dQ.fill_(0);
      dKV.fill_(0);
    }
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape, qkv_type, nullptr, nullptr, nullptr);
    te_KV =
        makeTransformerEngineTensor(KV.data_ptr(), kv_shape, qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape, qkv_type, nullptr, nullptr, nullptr);
    te_dO =
        makeTransformerEngineTensor(dO.data_ptr(), q_shape, dqkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQ =
        makeTransformerEngineTensor(dQ.data_ptr(), q_shape, dqkv_type, nullptr, nullptr, nullptr);
    te_dKV =
        makeTransformerEngineTensor(dKV.data_ptr(), kv_shape, dqkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }

  // create cu_seqlens tensorwrappers
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                                                DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                                                 DType::kInt32, nullptr, nullptr, nullptr);

  TensorWrapper te_cu_seqlens_q_padded, te_cu_seqlens_kv_padded;
  if ((cu_seqlens_q_padded.has_value()) && (cu_seqlens_kv_padded.has_value())) {
    auto cu_seqlens_q_padded_sizes = cu_seqlens_q_padded.value().sizes().vec();
    std::vector<size_t> cu_seqlens_q_padded_shape{cu_seqlens_q_padded_sizes.begin(),
                                                  cu_seqlens_q_padded_sizes.end()};
    auto cu_seqlens_kv_padded_sizes = cu_seqlens_kv_padded.value().sizes().vec();
    std::vector<size_t> cu_seqlens_kv_padded_shape{cu_seqlens_kv_padded_sizes.begin(),
                                                   cu_seqlens_kv_padded_sizes.end()};
    te_cu_seqlens_q_padded = makeTransformerEngineTensor(cu_seqlens_q_padded.value().data_ptr(),
                                                         cu_seqlens_q_padded_shape, DType::kInt32,
                                                         nullptr, nullptr, nullptr);
    te_cu_seqlens_kv_padded = makeTransformerEngineTensor(cu_seqlens_kv_padded.value().data_ptr(),
                                                          cu_seqlens_kv_padded_shape, DType::kInt32,
                                                          nullptr, nullptr, nullptr);
  }

  // convert auxiliary tensors from forward to NVTETensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor *>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create dBias the same shape as Bias
  at::Tensor dBias;
  TensorWrapper te_dBias;
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI)) {
    if (nvte_aux_tensor_pack.size >= 2) {
      std::vector<int64_t> bias_shape(Aux_CTX_Tensors[nvte_aux_tensor_pack.size - 1].sizes().vec());
      dBias = torch::empty(bias_shape, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    } else {
      dBias = torch::empty({1, static_cast<int64_t>(h_q), static_cast<int64_t>(max_seqlen_q),
                            static_cast<int64_t>(max_seqlen_kv)},
                           options);
      te_dBias = makeTransformerEngineTensor(dBias);
    }
    if (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD) {
      dBias.fill_(0);
    }
  }

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd_kvpacked(te_Q.data(), te_KV.data(), te_O.data(), te_dO.data(), te_S.data(),
                               te_dP.data(), &nvte_aux_tensor_pack, te_dQ.data(), te_dKV.data(),
                               te_dBias.data(), te_cu_seqlens_q.data(), te_cu_seqlens_kv.data(),
                               te_cu_seqlens_q_padded.data(), te_cu_seqlens_kv_padded.data(),
                               max_seqlen_q, max_seqlen_kv, attn_scale, p_dropout, qkv_layout,
                               bias_type, attn_mask_type, window_size[0], window_size[1],
                               deterministic, workspace.data(), at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd_kvpacked(te_Q.data(), te_KV.data(), te_O.data(), te_dO.data(), te_S.data(),
                               te_dP.data(), &nvte_aux_tensor_pack, te_dQ.data(), te_dKV.data(),
                               te_dBias.data(), te_cu_seqlens_q.data(), te_cu_seqlens_kv.data(),
                               te_cu_seqlens_q_padded.data(), te_cu_seqlens_kv_padded.data(),
                               max_seqlen_q, max_seqlen_kv, attn_scale, p_dropout, qkv_layout,
                               bias_type, attn_mask_type, window_size[0], window_size[1],
                               deterministic, workspace.data(), at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQ, dKV, dBias};
}

// fused attention FWD with separate Q, K and V tensors
std::vector<at::Tensor> fused_attn_fwd(
    size_t max_seqlen_q, size_t max_seqlen_kv, bool is_training, float attn_scale, float p_dropout,
    bool set_zero, NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type,
    NVTE_Mask_Type attn_mask_type, const std::vector<int64_t> window_size,
    const at::Tensor cu_seqlens_q, const at::Tensor cu_seqlens_kv, const at::Tensor Q,
    const at::Tensor K, const at::Tensor V, const transformer_engine::DType qkv_type,
    const c10::optional<at::Tensor> cu_seqlens_q_padded,
    const c10::optional<at::Tensor> cu_seqlens_kv_padded,
    const c10::optional<at::Tensor> descale_QKV, const int descale_QKV_offset,
    const c10::optional<at::Tensor> descale_S, const int descale_S_offset,
    const c10::optional<at::Tensor> scale_S, const int scale_S_offset,
    const c10::optional<at::Tensor> scale_O, const int scale_O_offset,
    c10::optional<at::Tensor> amax_S, const int amax_S_offset, c10::optional<at::Tensor> amax_O,
    const int amax_O_offset, const c10::optional<at::Tensor> Bias,
    const c10::optional<at::Generator> rng_gen, size_t rng_elts_per_thread) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto k_sizes = K.sizes().vec();
  std::vector<size_t> k_shape{k_sizes.begin(), k_sizes.end()};
  auto v_sizes = V.sizes().vec();
  std::vector<size_t> v_shape{v_sizes.begin(), v_sizes.end()};

  // create output tensor O
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  auto o_shape = std::vector<size_t>{q_sizes.begin(), q_sizes.end()};
  o_shape[o_shape.size() - 1] = v_sizes[v_sizes.size() - 1];
  std::vector<int64_t> o_shape_tmp{o_shape.begin(), o_shape.end()};
  auto O = torch::empty(c10::IntArrayRef(o_shape_tmp), options);

  // construct NVTE tensors
  TensorWrapper te_Q, te_K, te_V, te_S, te_O, te_Bias;
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  TensorWrapper te_cu_seqlens_q_padded, te_cu_seqlens_kv_padded;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto h = q_shape[q_shape.size() - 2];
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0) &&
        (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD)) {
      mha_fill(O, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      O.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!descale_S.has_value()) || (!scale_S.has_value()) ||
        (!scale_O.has_value()) || (!amax_S.has_value()) || (!amax_O.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, scale_S, scale_O, amax_S and amax_O ";
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape, qkv_type, nullptr, nullptr,
                                       getDataPtr(descale_QKV.value(), descale_QKV_offset));
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape, qkv_type, nullptr, nullptr,
                                       getDataPtr(descale_QKV.value(), descale_QKV_offset));
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape, qkv_type, nullptr, nullptr,
                                       getDataPtr(descale_QKV.value(), descale_QKV_offset));
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32,
                                       getDataPtr(amax_S.value(), amax_S_offset),
                                       getDataPtr(scale_S.value(), scale_S_offset),
                                       getDataPtr(descale_S.value(), descale_S_offset));
    te_O = makeTransformerEngineTensor(O.data_ptr(), q_shape, qkv_type,
                                       getDataPtr(amax_O.value(), amax_O_offset),
                                       getDataPtr(scale_O.value(), scale_O_offset), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    if (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD) {
      O.fill_(0);
    }
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape, qkv_type, nullptr, nullptr, nullptr);
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape, qkv_type, nullptr, nullptr, nullptr);
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape, qkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), o_shape, qkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
    auto bias_sizes = Bias.value().sizes().vec();
    std::vector<size_t> bias_shape{bias_sizes.begin(), bias_sizes.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), bias_shape, DType::kFloat32,
                                          nullptr, nullptr, nullptr);
  }
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                                                DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                                                 DType::kInt32, nullptr, nullptr, nullptr);

  if ((cu_seqlens_q_padded.has_value()) && (cu_seqlens_kv_padded.has_value())) {
    auto cu_seqlens_q_padded_sizes = cu_seqlens_q_padded.value().sizes().vec();
    std::vector<size_t> cu_seqlens_q_padded_shape{cu_seqlens_q_padded_sizes.begin(),
                                                  cu_seqlens_q_padded_sizes.end()};
    auto cu_seqlens_kv_padded_sizes = cu_seqlens_kv_padded.value().sizes().vec();
    std::vector<size_t> cu_seqlens_kv_padded_shape{cu_seqlens_kv_padded_sizes.begin(),
                                                   cu_seqlens_kv_padded_sizes.end()};
    te_cu_seqlens_q_padded = makeTransformerEngineTensor(cu_seqlens_q_padded.value().data_ptr(),
                                                         cu_seqlens_q_padded_shape, DType::kInt32,
                                                         nullptr, nullptr, nullptr);
    te_cu_seqlens_kv_padded = makeTransformerEngineTensor(cu_seqlens_kv_padded.value().data_ptr(),
                                                          cu_seqlens_kv_padded_shape, DType::kInt32,
                                                          nullptr, nullptr, nullptr);
  }

  // extract rng seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
      rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  at::PhiloxCudaState philox_args = init_philox_state(gen, rng_elts_per_thread);
  auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
      philox_args, static_cast<int64_t *>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd(te_Q.data(), te_K.data(), te_V.data(), te_Bias.data(), te_S.data(),
                      te_O.data(), &nvte_aux_tensor_pack, te_cu_seqlens_q.data(),
                      te_cu_seqlens_kv.data(), te_cu_seqlens_q_padded.data(),
                      te_cu_seqlens_kv_padded.data(), te_rng_state.data(), max_seqlen_q,
                      max_seqlen_kv, is_training, attn_scale, p_dropout, qkv_layout, bias_type,
                      attn_mask_type, window_size[0], window_size[1], workspace.data(),
                      at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors]
  std::vector<at::Tensor> output_tensors;
  output_tensors.push_back(O);
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor *>(nvte_aux_tensor_pack.tensors[i]);
    // allocate memory for nvte_aux_tensor_pack.tensors
    at::Tensor output_tensor;
    if (nvte_aux_tensor_pack.size >= 2) {
      if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
        if (i < nvte_aux_tensor_pack.size - 2) {
          output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
        } else if (i == nvte_aux_tensor_pack.size - 2) {
          output_tensor = rng_state;
        } else if (i == nvte_aux_tensor_pack.size - 1) {
          output_tensor = Bias.value();
        }
      } else {
        output_tensor = (i < nvte_aux_tensor_pack.size - 1)
                            ? allocateSpace(tensor->data.shape, tensor->data.dtype, false)
                            : rng_state;
      }
    } else {
      output_tensor = allocateSpace(tensor->data.shape, tensor->data.dtype, false);
    }
    output_tensors.push_back(output_tensor);
    tensor->data.dptr = output_tensor.data_ptr();
  }

  // execute the kernel
  nvte_fused_attn_fwd(te_Q.data(), te_K.data(), te_V.data(), te_Bias.data(), te_S.data(),
                      te_O.data(), &nvte_aux_tensor_pack, te_cu_seqlens_q.data(),
                      te_cu_seqlens_kv.data(), te_cu_seqlens_q_padded.data(),
                      te_cu_seqlens_kv_padded.data(), te_rng_state.data(), max_seqlen_q,
                      max_seqlen_kv, is_training, attn_scale, p_dropout, qkv_layout, bias_type,
                      attn_mask_type, window_size[0], window_size[1], workspace.data(),
                      at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, softmax-related tensors, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with separate Q, K and V
std::vector<at::Tensor> fused_attn_bwd(
    size_t max_seqlen_q, size_t max_seqlen_kv, float attn_scale, float p_dropout, bool set_zero,
    NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
    const std::vector<int64_t> window_size, bool deterministic, const at::Tensor cu_seqlens_q,
    const at::Tensor cu_seqlens_kv, const at::Tensor Q, const at::Tensor K, const at::Tensor V,
    const at::Tensor O, const at::Tensor dO, const transformer_engine::DType qkv_type,
    const transformer_engine::DType dqkv_type, const std::vector<at::Tensor> Aux_CTX_Tensors,
    const c10::optional<at::Tensor> cu_seqlens_q_padded,
    const c10::optional<at::Tensor> cu_seqlens_kv_padded,
    const c10::optional<at::Tensor> descale_QKV, const c10::optional<at::Tensor> descale_S,
    const c10::optional<at::Tensor> descale_O, const c10::optional<at::Tensor> descale_dO,
    const c10::optional<at::Tensor> descale_dP, const c10::optional<at::Tensor> scale_S,
    const c10::optional<at::Tensor> scale_dP, const c10::optional<at::Tensor> scale_dQKV,
    c10::optional<at::Tensor> amax_dP, c10::optional<at::Tensor> amax_dQKV) {
  using namespace transformer_engine;

  auto q_sizes = Q.sizes().vec();
  std::vector<size_t> q_shape{q_sizes.begin(), q_sizes.end()};
  auto k_sizes = K.sizes().vec();
  std::vector<size_t> k_shape{k_sizes.begin(), k_sizes.end()};
  auto v_sizes = V.sizes().vec();
  std::vector<size_t> v_shape{v_sizes.begin(), v_sizes.end()};
  auto h_q = q_shape[q_shape.size() - 2];
  auto h_kv = k_shape[k_shape.size() - 2];
  auto d_qk = q_shape[q_shape.size() - 1];
  auto d_v = v_shape[v_shape.size() - 1];
  auto options = torch::TensorOptions().dtype(GetATenDType(dqkv_type)).device(torch::kCUDA);
  std::vector<size_t> o_shape{q_sizes.begin(), q_sizes.end()};
  o_shape[o_shape.size() - 1] = d_v;

  at::Tensor dQ;
  at::Tensor dK;
  at::Tensor dV;
  at::Tensor dQKV, dKV;
  NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(qkv_layout);
  std::vector<int64_t> tmp_shape;
  switch (layout_group) {
    case NVTE_QKV_Layout_Group::NVTE_3HD:
      tmp_shape = std::vector<int64_t>{q_sizes.begin(), q_sizes.end()};
      tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 2, int64_t(3));
      dQKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
      dQ = dQKV.index({"...", torch::indexing::Slice(0, 1, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 3);
      dK = dQKV.index({"...", torch::indexing::Slice(1, 2, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 3);
      dV = dQKV.index({"...", torch::indexing::Slice(2, torch::indexing::None, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 3);
      break;
    case NVTE_QKV_Layout_Group::NVTE_H3D:
      tmp_shape = std::vector<int64_t>{q_sizes.begin(), q_sizes.end()};
      tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 1, int64_t(3));
      dQKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
      dQ = dQKV.index({"...", torch::indexing::Slice(0, 1, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 2);
      dK = dQKV.index({"...", torch::indexing::Slice(1, 2, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 2);
      dV = dQKV.index({"...", torch::indexing::Slice(2, torch::indexing::None, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 2);
      break;
    case NVTE_QKV_Layout_Group::NVTE_HD_2HD:
      dQ = torch::empty_like(Q, options);
      tmp_shape = std::vector<int64_t>{k_sizes.begin(), k_sizes.end()};
      tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 2, int64_t(2));
      dKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
      dK = dKV.index({"...", torch::indexing::Slice(0, 1, 1),
                      torch::indexing::Slice(0, torch::indexing::None, 1),
                      torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 3);
      dV = dKV.index({"...", torch::indexing::Slice(1, torch::indexing::None, 1),
                      torch::indexing::Slice(0, torch::indexing::None, 1),
                      torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 3);
      break;
    case NVTE_QKV_Layout_Group::NVTE_HD_H2D:
      dQ = torch::empty_like(Q, options);
      tmp_shape = std::vector<int64_t>{k_sizes.begin(), k_sizes.end()};
      tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 1, int64_t(2));
      dKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
      dK = dKV.index({"...", torch::indexing::Slice(0, 1, 1),
                      torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 2);
      dV = dKV.index({"...", torch::indexing::Slice(1, torch::indexing::None, 1),
                      torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 2);
      break;
    case NVTE_QKV_Layout_Group::NVTE_HD_HD_HD:
      dQ = torch::empty_like(Q, options);
      dK = torch::empty_like(K, options);
      dV = torch::empty_like(V, options);
      break;
    default:
      NVTE_ERROR("QKV layout not supported!");
  }

  // construct NVTE tensors
  TensorWrapper te_Q, te_K, te_V, te_O, te_dO, te_S, te_dP, te_dQ, te_dK, te_dV;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if (set_zero && ((h_q * d_qk) % block_size == 0) && ((h_kv * d_qk) % block_size == 0) &&
        dQ.is_contiguous() && dK.is_contiguous() && dV.is_contiguous() &&
        (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD)) {
      mha_fill(dQ, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
      mha_fill(dK, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
      mha_fill(dV, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      dQ.fill_(0);
      dK.fill_(0);
      dV.fill_(0);
    }
    if ((!descale_QKV.has_value()) || (!descale_S.has_value()) || (!descale_O.has_value()) ||
        (!descale_dO.has_value()) || (!descale_dP.has_value()) || (!scale_S.has_value()) ||
        (!scale_dP.has_value()) || (!scale_dQKV.has_value()) || (!amax_dP.has_value()) ||
        (!amax_dQKV.has_value())) {
      std::string err_tensors = "descale_QKV, descale_S, descale_O, descale_dO, descale_dP, ";
      err_tensors = err_tensors + std::string("scale_S, scale_dP, scale_dQKV, ");
      err_tensors = err_tensors + std::string("amax_dP and amax_dQKV ");
      NVTE_ERROR(err_tensors + std::string("are required for FP8 operation. \n"));
    }
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape, qkv_type, nullptr, nullptr,
                                       descale_QKV.value().data_ptr());
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape, qkv_type, nullptr, nullptr,
                                       descale_QKV.value().data_ptr());
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape, qkv_type, nullptr, nullptr,
                                       descale_QKV.value().data_ptr());
    te_O = makeTransformerEngineTensor(O.data_ptr(), o_shape, qkv_type, nullptr, nullptr,
                                       descale_O.value().data_ptr());
    te_dO = makeTransformerEngineTensor(dO.data_ptr(), o_shape, dqkv_type, nullptr, nullptr,
                                        descale_dO.value().data_ptr());
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr,
                                       scale_S.value().data_ptr(), descale_S.value().data_ptr());
    te_dP = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, amax_dP.value().data_ptr(),
                                        scale_dP.value().data_ptr(), descale_dP.value().data_ptr());
    te_dQ =
        makeTransformerEngineTensor(dQ.data_ptr(), q_shape, dqkv_type, amax_dQKV.value().data_ptr(),
                                    scale_dQKV.value().data_ptr(), nullptr);
    te_dK =
        makeTransformerEngineTensor(dK.data_ptr(), k_shape, dqkv_type, amax_dQKV.value().data_ptr(),
                                    scale_dQKV.value().data_ptr(), nullptr);
    te_dV =
        makeTransformerEngineTensor(dV.data_ptr(), v_shape, dqkv_type, amax_dQKV.value().data_ptr(),
                                    scale_dQKV.value().data_ptr(), nullptr);
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    if (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD) {
      dQ.fill_(0);
      dK.fill_(0);
      dV.fill_(0);
    }
    // BF16 or FP16
    te_Q = makeTransformerEngineTensor(Q.data_ptr(), q_shape, qkv_type, nullptr, nullptr, nullptr);
    te_K = makeTransformerEngineTensor(K.data_ptr(), k_shape, qkv_type, nullptr, nullptr, nullptr);
    te_V = makeTransformerEngineTensor(V.data_ptr(), v_shape, qkv_type, nullptr, nullptr, nullptr);
    te_O = makeTransformerEngineTensor(O.data_ptr(), o_shape, qkv_type, nullptr, nullptr, nullptr);
    te_dO =
        makeTransformerEngineTensor(dO.data_ptr(), o_shape, dqkv_type, nullptr, nullptr, nullptr);
    te_S = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr, nullptr, nullptr);
    te_dP = makeTransformerEngineTensor(nullptr, {0}, DType::kFloat32, nullptr, nullptr, nullptr);
    te_dQ =
        makeTransformerEngineTensor(dQ.data_ptr(), q_shape, dqkv_type, nullptr, nullptr, nullptr);
    te_dK =
        makeTransformerEngineTensor(dK.data_ptr(), k_shape, dqkv_type, nullptr, nullptr, nullptr);
    te_dV =
        makeTransformerEngineTensor(dV.data_ptr(), v_shape, dqkv_type, nullptr, nullptr, nullptr);
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }

  // create cu_seqlens tensorwrappers
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                                                DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                                                 DType::kInt32, nullptr, nullptr, nullptr);

  TensorWrapper te_cu_seqlens_q_padded, te_cu_seqlens_kv_padded;
  if ((cu_seqlens_q_padded.has_value()) && (cu_seqlens_kv_padded.has_value())) {
    auto cu_seqlens_q_padded_sizes = cu_seqlens_q_padded.value().sizes().vec();
    std::vector<size_t> cu_seqlens_q_padded_shape{cu_seqlens_q_padded_sizes.begin(),
                                                  cu_seqlens_q_padded_sizes.end()};
    auto cu_seqlens_kv_padded_sizes = cu_seqlens_kv_padded.value().sizes().vec();
    std::vector<size_t> cu_seqlens_kv_padded_shape{cu_seqlens_kv_padded_sizes.begin(),
                                                   cu_seqlens_kv_padded_sizes.end()};
    te_cu_seqlens_q_padded = makeTransformerEngineTensor(cu_seqlens_q_padded.value().data_ptr(),
                                                         cu_seqlens_q_padded_shape, DType::kInt32,
                                                         nullptr, nullptr, nullptr);
    te_cu_seqlens_kv_padded = makeTransformerEngineTensor(cu_seqlens_kv_padded.value().data_ptr(),
                                                          cu_seqlens_kv_padded_shape, DType::kInt32,
                                                          nullptr, nullptr, nullptr);
  }

  // convert auxiliary tensors from forward to NVTETensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    auto tensor = reinterpret_cast<transformer_engine::Tensor *>(nvte_aux_tensor_pack.tensors[i]);
    tensor->data.dptr = Aux_CTX_Tensors[i].data_ptr();
    std::vector<int64_t> tmp(Aux_CTX_Tensors[i].sizes().vec());
    tensor->data.shape = std::vector<size_t>(tmp.begin(), tmp.end());
    tensor->data.dtype = GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type());
  }

  // create dBias the same shape as Bias
  at::Tensor dBias;
  TensorWrapper te_dBias;
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI)) {
    if (nvte_aux_tensor_pack.size >= 2) {
      std::vector<int64_t> bias_shape(Aux_CTX_Tensors[nvte_aux_tensor_pack.size - 1].sizes().vec());
      dBias = torch::empty(bias_shape, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    } else {
      dBias = torch::empty({1, static_cast<int64_t>(h_q), static_cast<int64_t>(max_seqlen_q),
                            static_cast<int64_t>(max_seqlen_kv)},
                           options);
      te_dBias = makeTransformerEngineTensor(dBias);
    }
    if (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD) {
      dBias.fill_(0);
    }
  }

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd(te_Q.data(), te_K.data(), te_V.data(), te_O.data(), te_dO.data(), te_S.data(),
                      te_dP.data(), &nvte_aux_tensor_pack, te_dQ.data(), te_dK.data(), te_dV.data(),
                      te_dBias.data(), te_cu_seqlens_q.data(), te_cu_seqlens_kv.data(),
                      te_cu_seqlens_q_padded.data(), te_cu_seqlens_kv_padded.data(), max_seqlen_q,
                      max_seqlen_kv, attn_scale, p_dropout, qkv_layout, bias_type, attn_mask_type,
                      window_size[0], window_size[1], deterministic, workspace.data(),
                      at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd(te_Q.data(), te_K.data(), te_V.data(), te_O.data(), te_dO.data(), te_S.data(),
                      te_dP.data(), &nvte_aux_tensor_pack, te_dQ.data(), te_dK.data(), te_dV.data(),
                      te_dBias.data(), te_cu_seqlens_q.data(), te_cu_seqlens_kv.data(),
                      te_cu_seqlens_q_padded.data(), te_cu_seqlens_kv_padded.data(), max_seqlen_q,
                      max_seqlen_kv, attn_scale, p_dropout, qkv_layout, bias_type, attn_mask_type,
                      window_size[0], window_size[1], deterministic, workspace.data(),
                      at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {dQ, dK, dV, dBias};
}

namespace flash_attention {

constexpr int warp_size = 32;
constexpr int type_size = 2;  // FP16 or BF16
constexpr int nvec = sizeof(uint64_t) / type_size;
constexpr int load_size = warp_size * nvec;
constexpr int block_size = 512;

template <typename T>
__launch_bounds__(block_size) __global__
    void prepare_kernel_fwd(const T *qkvi, T *qkv, const size_t B, const size_t S, const size_t Z,
                            const size_t W) {
  const int warpid = (blockDim.x * blockIdx.x + threadIdx.x) / warp_size;
  const int id_in_warp = threadIdx.x % warp_size;
  const size_t offset_input = blockIdx.y * W + warpid * 3 * W * Z + id_in_warp * nvec;
  const T *my_input = qkvi + offset_input;

  const size_t s = warpid / B;
  if (s >= S) return;

  const size_t b = warpid % B;

  const size_t offset_output = blockIdx.y * B * S * Z * W + (s + b * S) * W * Z + id_in_warp * nvec;

  T *my_output = qkv + offset_output;

  for (int i = 0; i < Z; ++i) {
    uint64_t *out = reinterpret_cast<uint64_t *>(my_output + i * load_size);
    *out = *reinterpret_cast<const uint64_t *>(my_input + i * load_size * 3);
  }
}

template <typename T>
__launch_bounds__(block_size) __global__
    void prepare_kernel_bwd(const T *q, const T *k, const T *v, T *qkv, const size_t B,
                            const size_t S, const size_t Z, const size_t W) {
  const T *input = blockIdx.y == 0 ? q : (blockIdx.y == 1 ? k : v);

  const int warpid = (blockDim.x * blockIdx.x + threadIdx.x) / warp_size;
  const int id_in_warp = threadIdx.x % warp_size;
  const size_t offset_input = warpid * W * Z + id_in_warp * nvec;
  const T *my_input = input + offset_input;

  const size_t b = warpid / S;
  if (b >= B) return;

  const size_t s = warpid % S;

  const size_t offset_output = (b + s * B) * 3 * W * Z + id_in_warp * nvec + blockIdx.y * W;

  T *my_output = qkv + offset_output;

  for (int i = 0; i < Z; ++i) {
    uint64_t *out = reinterpret_cast<uint64_t *>(my_output + i * load_size * 3);
    *out = *reinterpret_cast<const uint64_t *>(my_input + i * load_size);
  }
}

}  // namespace flash_attention

at::Tensor fa_prepare_fwd(at::Tensor qkvi) {
  NVTE_CHECK(qkvi.dim() == 4, "Expected 4-dim tensor.");
  NVTE_CHECK(qkvi.scalar_type() == at::ScalarType::Half ||
             qkvi.scalar_type() == at::ScalarType::BFloat16);
  NVTE_CHECK(qkvi.size(3) % flash_attention::load_size == 0);
  NVTE_CHECK(qkvi.size(3) == flash_attention::load_size);
  NVTE_CHECK(qkvi.stride(3) == 1, "Wrong stride.");
  NVTE_CHECK(qkvi.stride(2) == 3 * qkvi.size(3), "Wrong stride.");
  NVTE_CHECK(qkvi.stride(1) == 3 * qkvi.size(3) * qkvi.size(2), "Wrong stride.");
  NVTE_CHECK(qkvi.stride(0) == 3 * qkvi.size(3) * qkvi.size(2) * qkvi.size(1), "Wrong stride.");

  // [s, b, n, h * 3] -> [3, b, s, n, h]
  std::vector<int64_t> shape = {3, qkvi.size(1), qkvi.size(0), qkvi.size(2), qkvi.size(3)};
  at::Tensor qkv = at::empty(shape, at::CUDA(qkvi.scalar_type()));

  size_t warps = qkvi.size(0) * qkvi.size(1);
  size_t warps_per_block = flash_attention::block_size / flash_attention::warp_size;
  size_t blocks = (warps + warps_per_block - 1) / warps_per_block;
  dim3 grid(blocks, 3);
  int threads = flash_attention::block_size;
  if (qkvi.scalar_type() == at::ScalarType::Half) {
    using dtype = at::Half;
    flash_attention::prepare_kernel_fwd<dtype>
        <<<grid, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            qkvi.data_ptr<dtype>(), qkv.data_ptr<dtype>(), shape[1], shape[2], shape[3], shape[4]);
  } else {
    using dtype = at::BFloat16;
    flash_attention::prepare_kernel_fwd<dtype>
        <<<grid, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            qkvi.data_ptr<dtype>(), qkv.data_ptr<dtype>(), shape[1], shape[2], shape[3], shape[4]);
  }

  return qkv;
}

at::Tensor fa_prepare_bwd(at::Tensor q, at::Tensor k, at::Tensor v) {
  NVTE_CHECK(q.is_contiguous());
  NVTE_CHECK(k.is_contiguous());
  NVTE_CHECK(v.is_contiguous());
  NVTE_CHECK(q.dim() == 4, "Expected 4-dim tensor.");
  NVTE_CHECK(k.dim() == 4, "Expected 4-dim tensor.");
  NVTE_CHECK(v.dim() == 4, "Expected 4-dim tensor.");
  NVTE_CHECK(q.scalar_type() == at::ScalarType::Half ||
             q.scalar_type() == at::ScalarType::BFloat16);
  NVTE_CHECK(k.scalar_type() == q.scalar_type());
  NVTE_CHECK(v.scalar_type() == q.scalar_type());
  NVTE_CHECK(q.size(3) % flash_attention::load_size == 0);
  NVTE_CHECK(q.size(3) == flash_attention::load_size);
  NVTE_CHECK(k.size(3) % flash_attention::load_size == 0);
  NVTE_CHECK(k.size(3) == flash_attention::load_size);
  NVTE_CHECK(v.size(3) % flash_attention::load_size == 0);
  NVTE_CHECK(v.size(3) == flash_attention::load_size);

  // 3 x [s, b, n, h] -> [b, s, n, 3 * h]

  std::vector<int64_t> shape = {q.size(1), q.size(0), q.size(2), 3 * q.size(3)};
  at::Tensor qkv = at::empty(shape, at::CUDA(q.scalar_type()));

  size_t warps = q.size(0) * q.size(1);
  size_t warps_per_block = flash_attention::block_size / flash_attention::warp_size;
  size_t blocks = (warps + warps_per_block - 1) / warps_per_block;
  dim3 grid(blocks, 3);
  int threads = flash_attention::block_size;
  if (q.scalar_type() == at::ScalarType::Half) {
    using dtype = at::Half;
    flash_attention::prepare_kernel_bwd<dtype>
        <<<grid, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            q.data_ptr<dtype>(), k.data_ptr<dtype>(), v.data_ptr<dtype>(), qkv.data_ptr<dtype>(),
            q.size(0), q.size(1), q.size(2), q.size(3));
  } else {
    using dtype = at::BFloat16;
    flash_attention::prepare_kernel_bwd<dtype>
        <<<grid, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            q.data_ptr<dtype>(), k.data_ptr<dtype>(), v.data_ptr<dtype>(), qkv.data_ptr<dtype>(),
            q.size(0), q.size(1), q.size(2), q.size(3));
  }

  return qkv;
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Binary search
 **************************************************************************************************/

__forceinline__ __device__ int binary_search(int target, int *array, int len) {
  int left = 1, right = len - 1;
  while (left < right) {
    int mid = (left + right) / 2;
    if (array[mid] <= target) {
      left = mid + 1;
    } else {
      right = mid;
    }
  }
  return left - 1;
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Read the half of a THD tensor
 **************************************************************************************************/

__global__ void thd_read_half_tensor_kernel(void *half, void *tensor, int *cu_seqlens, int batch,
                                            int hidden_size_in_bytes, int half_idx,
                                            int dim_size_of_token) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / 2;
  }
  __syncthreads();

  int warpid = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
  int laneid = threadIdx.x % 32;
  int num_warps = (blockDim.x * gridDim.x) / 32;
  int num_total_tokens = cu_seqlens_s[batch];
  int num_float4s_per_token = hidden_size_in_bytes / sizeof(float4);

  size_t offset = static_cast<size_t>(dim_size_of_token) * hidden_size_in_bytes;
  half = reinterpret_cast<void *>(reinterpret_cast<char *>(half) + offset / 2 * blockIdx.y);
  tensor = reinterpret_cast<void *>(reinterpret_cast<char *>(tensor) + offset * blockIdx.y);

  for (int token_id = warpid; token_id < num_total_tokens; token_id += num_warps) {
    int seqid = binary_search(token_id, cu_seqlens_s, batch + 1);

    size_t offset_in_bytes = static_cast<size_t>(token_id) * hidden_size_in_bytes;
    float4 *cur_half_token =
        reinterpret_cast<float4 *>(reinterpret_cast<char *>(half) + offset_in_bytes);

    offset_in_bytes =
        (static_cast<size_t>(token_id) + cu_seqlens_s[seqid + half_idx]) * hidden_size_in_bytes;
    float4 *cur_token =
        reinterpret_cast<float4 *>(reinterpret_cast<char *>(tensor) + offset_in_bytes);

    for (int idx = laneid; idx < num_float4s_per_token; idx += 32) {
      cur_half_token[idx] = cur_token[idx];
    }
  }
}

at::Tensor thd_read_half_tensor(const at::Tensor &tensor, const at::Tensor &cu_seqlens,
                                int half_idx) {
  NVTE_CHECK(tensor.dim() == 3 || tensor.dim() == 4);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);
  NVTE_CHECK(cu_seqlens.size(0) >= 2);

  // Shapes of q and dq are [t, h, d], so the dimension of "t" is 0
  // Shapes of kv and dkv are [2, t, h, d], so the dimension of "t" is 1
  int seq_dim = tensor.dim() == 3 ? 0 : 1;

  int batch = cu_seqlens.size(0) - 1;
  int num_heads = tensor.size(seq_dim + 1);
  int dim_per_head = tensor.size(seq_dim + 2);
  int hidden_size_in_bytes = num_heads * dim_per_head * c10::elementSize(tensor.scalar_type());

  // For 128-bits load/store
  NVTE_CHECK(hidden_size_in_bytes % 16 == 0);

  // Generate output
  std::vector<int64_t> shape(tensor.dim());
  for (size_t i = 0; i < shape.size(); i++) {
    shape[i] = tensor.size(i);
  }
  shape[seq_dim] /= 2;
  at::Tensor half = at::empty(shape, at::CUDA(tensor.scalar_type()));

  // Launch Kernel
  constexpr unsigned int block = 256;
  unsigned int grid_x = (tensor.size(seq_dim) / 2 * 32 + block - 1) / block;
  unsigned int grid_y = 1;
  for (int i = 0; i < seq_dim; i++) {
    grid_y *= tensor.size(i);
  }
  dim3 grid = {grid_x, grid_y};
  thd_read_half_tensor_kernel<<<grid, block, sizeof(int) * (batch + 1),
                                at::cuda::getCurrentCUDAStream()>>>(
      half.data_ptr(), tensor.data_ptr(), cu_seqlens.data_ptr<int>(), batch, hidden_size_in_bytes,
      half_idx, tensor.size(seq_dim));

  return half;
}

/***************************************************************************************************
 * Support THD format for Context Parallel: softmax_lse related operations
 **************************************************************************************************/

template <typename lse_dtype, bool lse_packed, typename Functor>
__global__ void thd_lse_kernel(lse_dtype *lse, float *half_lse, int *cu_seqlens, int batch,
                               int num_heads, int total_tokens) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / 2;
  }
  __syncthreads();

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;
  int num_total_tokens = cu_seqlens_s[batch];

  for (int token_id = tid; token_id < num_total_tokens; token_id += num_threads) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
    for (int head_id = blockIdx.y; head_id < num_heads; head_id += gridDim.y) {
      size_t idx, half_idx;
      if constexpr (lse_packed) {
        idx = head_id * total_tokens + token_id + cu_seqlens_s[seq_id + 1];
        half_idx = head_id * total_tokens / 2 + token_id;
      } else {
        size_t row = static_cast<size_t>(seq_id) * num_heads + head_id;
        int col = token_id - cu_seqlens_s[seq_id];
        int seq_len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];

        idx = row * total_tokens + col + seq_len;
        half_idx = row * total_tokens / 2 + col;
      }

      Functor::run(lse, half_lse, idx, half_idx);
    }
  }
}

struct LseCorrectionFunctor {
  __forceinline__ __device__ static void run(double *lse, float *half_lse, size_t idx,
                                             size_t half_idx) {
    double val = lse[idx];
    float val_per_step = half_lse[half_idx];
    double max_scale = max(val, val_per_step);
    double min_scale = min(val, val_per_step);
    lse[idx] = max_scale + log(1.0 + exp(min_scale - max_scale));
  }
};

void thd_second_half_lse_correction(at::Tensor lse, const at::Tensor &lse_per_step,
                                    const at::Tensor &cu_seqlens, bool lse_packed) {
  NVTE_CHECK(lse.scalar_type() == at::ScalarType::Double);
  NVTE_CHECK(lse_per_step.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  int batch, num_heads, total_tokens;

  if (lse_packed) {
    NVTE_CHECK(lse.dim() == 2);
    NVTE_CHECK(lse_per_step.dim() == 2);

    batch = cu_seqlens.size(0) - 1;
    num_heads = lse.size(0);
    total_tokens = lse.size(1);

    NVTE_CHECK(lse_per_step.size(0) == num_heads);
    NVTE_CHECK(lse_per_step.size(1) == total_tokens / 2);
  } else {
    NVTE_CHECK(lse.dim() == 3);
    NVTE_CHECK(lse_per_step.dim() == 3);

    batch = lse.size(0);
    num_heads = lse.size(1);
    total_tokens = lse.size(2);

    NVTE_CHECK(lse_per_step.size(0) == batch);
    NVTE_CHECK(lse_per_step.size(1) == num_heads);
    NVTE_CHECK(lse_per_step.size(2) == total_tokens / 2);
    NVTE_CHECK(cu_seqlens.size(0) == batch + 1);
  }

  constexpr unsigned int block = 256;
  unsigned int grid_x = (total_tokens / 2 + block - 1) / block;
  unsigned int grid_y = num_heads;
  dim3 grid = {grid_x, grid_y};
  if (lse_packed) {
    thd_lse_kernel<double, true, LseCorrectionFunctor>
        <<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
            lse.data_ptr<double>(), lse_per_step.data_ptr<float>(), cu_seqlens.data_ptr<int>(),
            batch, num_heads, total_tokens);
  } else {
    thd_lse_kernel<double, false, LseCorrectionFunctor>
        <<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
            lse.data_ptr<double>(), lse_per_step.data_ptr<float>(), cu_seqlens.data_ptr<int>(),
            batch, num_heads, total_tokens);
  }
}

struct ReadLseFunctor {
  __forceinline__ __device__ static void run(float *lse, float *half_lse, size_t idx,
                                             size_t half_idx) {
    half_lse[half_idx] = lse[idx];
  }
};

at::Tensor thd_read_second_half_lse(const at::Tensor &lse, const at::Tensor &cu_seqlens,
                                    bool lse_packed) {
  NVTE_CHECK(lse.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  int batch, num_heads, total_tokens;
  std::vector<int64_t> shape;

  if (lse_packed) {
    NVTE_CHECK(lse.dim() == 2);

    batch = cu_seqlens.size(0) - 1;
    num_heads = lse.size(0);
    total_tokens = lse.size(1);

    shape = {num_heads, total_tokens / 2};
  } else {
    NVTE_CHECK(lse.dim() == 3);

    batch = lse.size(0);
    num_heads = lse.size(1);
    total_tokens = lse.size(2);

    NVTE_CHECK(cu_seqlens.size(0) == batch + 1);

    shape = {batch, num_heads, total_tokens / 2};
  }

  at::Tensor half_lse = at::zeros(shape, at::CUDA(lse.scalar_type()));

  constexpr unsigned int block = 256;
  unsigned int grid_x = (total_tokens / 2 + block - 1) / block;
  unsigned int grid_y = num_heads;
  dim3 grid = {grid_x, grid_y};
  if (lse_packed) {
    thd_lse_kernel<float, true, ReadLseFunctor>
        <<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
            lse.data_ptr<float>(), half_lse.data_ptr<float>(), cu_seqlens.data_ptr<int>(), batch,
            num_heads, total_tokens);
  } else {
    thd_lse_kernel<float, false, ReadLseFunctor>
        <<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
            lse.data_ptr<float>(), half_lse.data_ptr<float>(), cu_seqlens.data_ptr<int>(), batch,
            num_heads, total_tokens);
  }

  return half_lse;
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Out correction in forward
 **************************************************************************************************/

template <typename dtype, int only_second_half, int tile_size, bool lse_packed>
__global__ void thd_out_correction_kernel(dtype *out, dtype *out_per_step, float *lse,
                                          float *lse_per_step, int *cu_seqlens, int batch,
                                          int num_heads, int dim_per_head, int lse_seqlen) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    cu_seqlens_s[i] = cu_seqlens[i] / (only_second_half + 1);
  }
  __syncthreads();

  int tile_id = (blockIdx.x * blockDim.x + threadIdx.x) / tile_size;
  int lane_id = threadIdx.x % tile_size;
  int num_tiles = (blockDim.x * gridDim.x) / tile_size;
  int num_total_tokens = cu_seqlens_s[batch];
  int num_loops_per_head = dim_per_head * sizeof(dtype) / sizeof(float4);

  for (int token_id = tile_id; token_id < num_total_tokens; token_id += num_tiles) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
    for (int head_id = blockIdx.y; head_id < num_heads; head_id += gridDim.y) {
      size_t idx, idx_per_step;

      if constexpr (lse_packed) {
        idx = head_id * lse_seqlen + token_id + cu_seqlens_s[seq_id + 1] * only_second_half;
        idx_per_step = head_id * lse_seqlen / (only_second_half + 1) + token_id;
      } else {
        size_t row = static_cast<size_t>(seq_id) * num_heads + head_id;
        int col = token_id - cu_seqlens_s[seq_id];
        int seq_len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];
        idx = row * lse_seqlen + col + seq_len * only_second_half;
        idx_per_step = row * lse_seqlen / (only_second_half + 1) + col;
      }
      float lse_corrected_exp = exp(lse_per_step[idx_per_step] - lse[idx]);

      idx = token_id + cu_seqlens_s[seq_id + 1] * only_second_half;
      idx = (idx * num_heads + head_id) * dim_per_head;
      idx_per_step = (static_cast<size_t>(token_id) * num_heads + head_id) * dim_per_head;
      dtype *cur_out = out + idx;
      dtype *cur_out_per_step = out_per_step + idx_per_step;

      for (int j = lane_id; j < num_loops_per_head; j += tile_size) {
        float4 data_per_step = reinterpret_cast<float4 *>(cur_out_per_step)[j];
        float4 data = reinterpret_cast<float4 *>(cur_out)[j];
        dtype *p_per_step = reinterpret_cast<dtype *>(&data_per_step);
        dtype *p = reinterpret_cast<dtype *>(&data);
        for (int k = 0; k < sizeof(float4) / sizeof(dtype); k++) {
          p[k] += (p_per_step[k] == 0 ? 0 : p_per_step[k] * lse_corrected_exp);
        }
        reinterpret_cast<float4 *>(cur_out)[j] = data;
      }
    }
  }
}

template <typename dtype, int only_second_half>
static void thd_out_correction_helper(at::Tensor out, const at::Tensor &out_per_step,
                                      const at::Tensor &lse, const at::Tensor &lse_per_step,
                                      const at::Tensor &cu_seqlens, bool lse_packed) {
  NVTE_CHECK(out.scalar_type() == out_per_step.scalar_type());
  NVTE_CHECK(lse.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(lse_per_step.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);

  int total_tokens = out.size(0);
  int num_heads = out.size(1);
  int dim_per_head = out.size(2);

  NVTE_CHECK(out_per_step.size(0) == total_tokens / (only_second_half + 1));
  NVTE_CHECK(out_per_step.size(1) == num_heads);
  NVTE_CHECK(out_per_step.size(2) == dim_per_head);

  int batch, lse_seqlen;
  if (lse_packed) {
    batch = cu_seqlens.size(0) - 1;
    lse_seqlen = total_tokens;

    NVTE_CHECK(lse.size(0) == num_heads);
    NVTE_CHECK(lse.size(1) == lse_seqlen);
    NVTE_CHECK(lse_per_step.size(0) == num_heads);
    NVTE_CHECK(lse_per_step.size(1) == lse_seqlen / (only_second_half + 1));
  } else {
    batch = lse.size(0);
    lse_seqlen = lse.size(2);

    NVTE_CHECK(lse.size(1) == num_heads);
    NVTE_CHECK(lse_per_step.size(0) == batch);
    NVTE_CHECK(lse_per_step.size(1) == num_heads);
    NVTE_CHECK(lse_per_step.size(2) == lse_seqlen / (only_second_half + 1));
    NVTE_CHECK(cu_seqlens.size(0) == batch + 1);
  }

  constexpr int tile = 16;
  constexpr int block = 512;
  unsigned int grid_x =
      (static_cast<size_t>(total_tokens) / (only_second_half + 1) * tile + block - 1) / block;
  dim3 grid = {grid_x, (unsigned int)num_heads};

  if (lse_packed) {
    thd_out_correction_kernel<dtype, only_second_half, tile, true>
        <<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
            out.data_ptr<dtype>(), out_per_step.data_ptr<dtype>(), lse.data_ptr<float>(),
            lse_per_step.data_ptr<float>(), cu_seqlens.data_ptr<int>(), batch, num_heads,
            dim_per_head, lse_seqlen);
  } else {
    thd_out_correction_kernel<dtype, only_second_half, tile, false>
        <<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
            out.data_ptr<dtype>(), out_per_step.data_ptr<dtype>(), lse.data_ptr<float>(),
            lse_per_step.data_ptr<float>(), cu_seqlens.data_ptr<int>(), batch, num_heads,
            dim_per_head, lse_seqlen);
  }
}

void thd_out_correction(at::Tensor out, const at::Tensor &out_per_step, const at::Tensor &lse,
                        const at::Tensor &lse_per_step, const at::Tensor &cu_seqlens,
                        bool only_second_half, bool lse_packed) {
  if (only_second_half) {
    if (out.scalar_type() == at::ScalarType::Half) {
      using dtype = at::Half;
      thd_out_correction_helper<dtype, 1>(out, out_per_step, lse, lse_per_step, cu_seqlens,
                                          lse_packed);
    } else if (out.scalar_type() == at::ScalarType::BFloat16) {
      using dtype = at::BFloat16;
      thd_out_correction_helper<dtype, 1>(out, out_per_step, lse, lse_per_step, cu_seqlens,
                                          lse_packed);
    } else if (out.scalar_type() == at::ScalarType::Float) {
      using dtype = float;
      thd_out_correction_helper<dtype, 1>(out, out_per_step, lse, lse_per_step, cu_seqlens,
                                          lse_packed);
    } else {
      NVTE_ERROR("Unsupported dtype of out\n");
    }
  } else {
    if (out.scalar_type() == at::ScalarType::Half) {
      using dtype = at::Half;
      thd_out_correction_helper<dtype, 0>(out, out_per_step, lse, lse_per_step, cu_seqlens,
                                          lse_packed);
    } else if (out.scalar_type() == at::ScalarType::BFloat16) {
      using dtype = at::BFloat16;
      thd_out_correction_helper<dtype, 0>(out, out_per_step, lse, lse_per_step, cu_seqlens,
                                          lse_packed);
    } else if (out.scalar_type() == at::ScalarType::Float) {
      using dtype = float;
      thd_out_correction_helper<dtype, 0>(out, out_per_step, lse, lse_per_step, cu_seqlens,
                                          lse_packed);
    } else {
      NVTE_ERROR("Unsupported dtype of out\n");
    }
  }
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Gradients correction in backward
 **************************************************************************************************/

template <typename dtype, typename Functor_0, typename Functor_1, int functor_idx, int group_size>
__global__ void thd_grad_correction_kernel(dtype *grad, dtype *grad_per_step, int *cu_seqlens,
                                           int batch, int hidden_size, int dim_size_of_token) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    if constexpr (functor_idx < 2) {
      cu_seqlens_s[i] = cu_seqlens[i] / 2;
    } else {
      cu_seqlens_s[i] = cu_seqlens[i];
    }
  }
  __syncthreads();

  int group_id = (blockIdx.x * blockDim.x + threadIdx.x) / group_size;
  int lane_id = threadIdx.x % group_size;
  int num_groups = (blockDim.x * gridDim.x) / group_size;
  int num_total_tokens = cu_seqlens_s[batch];
  int num_inner_loops = hidden_size * sizeof(dtype) / sizeof(float4);

  size_t offset = static_cast<size_t>(dim_size_of_token) * hidden_size;
  if constexpr (functor_idx < 2) {
    grad_per_step = grad_per_step + offset / 2 * blockIdx.y;
  } else {
    grad_per_step = grad_per_step + offset * blockIdx.y;
  }
  grad = grad + offset * blockIdx.y;

  for (int token_id = group_id; token_id < num_total_tokens; token_id += num_groups) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);

    int token_offset;
    bool is_first_half;
    if constexpr (functor_idx < 2) {
      token_offset = cu_seqlens_s[seq_id + functor_idx];
      is_first_half = (functor_idx == 0);
    } else {
      token_offset = 0;
      int len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];
      is_first_half = (token_id - cu_seqlens_s[seq_id]) < (len / 2);
    }

    dtype *token = &grad[(token_id + token_offset) * static_cast<size_t>(hidden_size)];
    dtype *token_per_step = &grad_per_step[token_id * static_cast<size_t>(hidden_size)];
    for (int idx = lane_id; idx < num_inner_loops; idx += group_size) {
      if (is_first_half) {
        Functor_0::run(token, token_per_step, idx);
      } else {
        Functor_1::run(token, token_per_step, idx);
      }
    }
  }
}

struct EmptyFunctor {
  __forceinline__ __device__ static void run(void *token, void *token_per_step, int idx) {}
};

struct CopyFunctor {
  __forceinline__ __device__ static void run(void *token, void *token_per_step, int idx) {
    reinterpret_cast<float4 *>(token)[idx] = reinterpret_cast<float4 *>(token_per_step)[idx];
  }
};

template <typename dtype>
struct AddFunctor {
  __forceinline__ __device__ static void run(dtype *token, dtype *token_per_step, int idx) {
    float4 d_ = reinterpret_cast<float4 *>(token)[idx];
    dtype *p_ = reinterpret_cast<dtype *>(&d_);

    float4 d = reinterpret_cast<float4 *>(token_per_step)[idx];
    dtype *p = reinterpret_cast<dtype *>(&d);

#pragma unroll
    for (int i = 0; i < sizeof(float4) / sizeof(dtype); i++) {
      p_[i] += p[i];
    }

    reinterpret_cast<float4 *>(token)[idx] = d_;
  }
};

template <typename dtype, typename Functor_0, typename Functor_1, int functor_idx>
static void thd_grad_correction_helper(at::Tensor grad, const at::Tensor &grad_per_step,
                                       const at::Tensor &cu_seqlens) {
  NVTE_CHECK(grad.dim() == 3 || grad.dim() == 4);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  // Shape of dq is [t, h, d], so the dimension of "t" is 0
  // Shape of dkv is [2, t, h, d], so the dimension of "t" is 1
  int seq_dim = grad.dim() == 3 ? 0 : 1;

  int total_tokens = grad.size(seq_dim);
  int num_heads = grad.size(seq_dim + 1);
  int dim_per_head = grad.size(seq_dim + 2);
  int batch = cu_seqlens.size(0) - 1;

  if constexpr (functor_idx < 2) {
    NVTE_CHECK(grad_per_step.size(seq_dim) == total_tokens / 2);
  } else {
    NVTE_CHECK(grad_per_step.size(seq_dim) == total_tokens);
  }
  NVTE_CHECK(grad_per_step.size(seq_dim + 1) == num_heads);
  NVTE_CHECK(grad_per_step.size(seq_dim + 2) == dim_per_head);

  size_t hidden_size = num_heads * dim_per_head;
  NVTE_CHECK((hidden_size * c10::elementSize(grad.scalar_type())) % 16 == 0);

  constexpr unsigned int block = 256;
  unsigned int grid_x;
  if constexpr (functor_idx < 2) {
    grid_x = (total_tokens / 2 * 32 + block - 1) / block;
  } else {
    grid_x = (total_tokens * 32 + block - 1) / block;
  }
  unsigned int grid_y = 1;
  for (int i = 0; i < seq_dim; i++) {
    grid_y *= grad.size(i);
  }
  dim3 grid = {grid_x, grid_y};

  thd_grad_correction_kernel<dtype, Functor_0, Functor_1, functor_idx, 32>
      <<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
          grad.data_ptr<dtype>(), grad_per_step.data_ptr<dtype>(), cu_seqlens.data_ptr<int>(),
          batch, hidden_size, total_tokens);
}

template <typename dtype>
static void thd_grad_dispatcher(at::Tensor grad, const at::Tensor &grad_per_step,
                                const at::Tensor &cu_seqlens, const std::string &first_half,
                                const std::string &second_half) {
  if (first_half == "add" && second_half == "none") {
    thd_grad_correction_helper<dtype, AddFunctor<dtype>, EmptyFunctor, 0>(grad, grad_per_step,
                                                                          cu_seqlens);
  } else if (first_half == "copy" && second_half == "none") {
    thd_grad_correction_helper<dtype, CopyFunctor, EmptyFunctor, 0>(grad, grad_per_step,
                                                                    cu_seqlens);
  } else if (first_half == "none" && second_half == "add") {
    thd_grad_correction_helper<dtype, EmptyFunctor, AddFunctor<dtype>, 1>(grad, grad_per_step,
                                                                          cu_seqlens);
  } else if (first_half == "none" && second_half == "copy") {
    thd_grad_correction_helper<dtype, EmptyFunctor, CopyFunctor, 1>(grad, grad_per_step,
                                                                    cu_seqlens);
  } else if (first_half == "add" && second_half == "copy") {
    thd_grad_correction_helper<dtype, AddFunctor<dtype>, CopyFunctor, 2>(grad, grad_per_step,
                                                                         cu_seqlens);
  } else if (first_half == "copy" && second_half == "add") {
    thd_grad_correction_helper<dtype, CopyFunctor, AddFunctor<dtype>, 2>(grad, grad_per_step,
                                                                         cu_seqlens);
  } else {
    NVTE_ERROR("Unsupported Functor of first half and second_half\n");
  }
}

void thd_grad_correction(at::Tensor grad, const at::Tensor &grad_per_step,
                         const at::Tensor &cu_seqlens, const std::string &first_half,
                         const std::string &second_half) {
  if (grad.scalar_type() == at::ScalarType::Half) {
    thd_grad_dispatcher<at::Half>(grad, grad_per_step, cu_seqlens, first_half, second_half);
  } else if (grad.scalar_type() == at::ScalarType::BFloat16) {
    thd_grad_dispatcher<at::BFloat16>(grad, grad_per_step, cu_seqlens, first_half, second_half);
  } else if (grad.scalar_type() == at::ScalarType::Float) {
    thd_grad_dispatcher<float>(grad, grad_per_step, cu_seqlens, first_half, second_half);
  } else {
    NVTE_ERROR("Unsupported dtype of grad\n");
  }
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Generate partitioned indices for input tokens
 **************************************************************************************************/

__global__ void thd_partition_indices_kernel(int *output, int *cu_seqlens, int batch,
                                             int total_tokens, int world_size, int rank) {
  extern __shared__ int cu_seqlens_s[];
  for (int i = threadIdx.x; i <= batch; i += blockDim.x) {
    int seqlen = cu_seqlens[i];
    // Currently we assume that each sequence length is divisible by (world_size*2) since we have
    // to distribute each sequence evenly to different GPUs.
    assert(seqlen % (world_size * 2) == 0);
    cu_seqlens_s[i] = seqlen / world_size;
  }
  __syncthreads();

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int num_threads = blockDim.x * gridDim.x;

  for (int token_id = tid; token_id < total_tokens / world_size; token_id += num_threads) {
    int seq_id = binary_search(token_id, cu_seqlens_s, batch + 1);
    int seq_len = cu_seqlens_s[seq_id + 1] - cu_seqlens_s[seq_id];
    int index = token_id - cu_seqlens_s[seq_id];
    int offset = index < seq_len / 2 ? rank : (world_size - 1) * 2 - rank;
    index += cu_seqlens_s[seq_id] * world_size + seq_len / 2 * offset;
    output[token_id] = index;
  }
}

at::Tensor thd_get_partitioned_indices(const at::Tensor &cu_seqlens, int total_tokens,
                                       int world_size, int rank) {
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);
  NVTE_CHECK(cu_seqlens.size(0) >= 2);
  NVTE_CHECK(rank >= 0 && rank < world_size);
  NVTE_CHECK(world_size > 0);
  NVTE_CHECK(total_tokens > 0 && total_tokens % (world_size * 2) == 0);

  int batch = cu_seqlens.size(0) - 1;

  std::vector<int64_t> shape = {total_tokens / world_size};
  at::Tensor output = at::empty(shape, at::CUDA(at::ScalarType::Int));

  constexpr unsigned int block = 256;
  unsigned int grid = (output.size(0) + block - 1) / block;
  thd_partition_indices_kernel<<<grid, block, sizeof(int) * (batch + 1),
                                 at::cuda::getCurrentCUDAStream()>>>(
      output.data_ptr<int>(), cu_seqlens.data_ptr<int>(), batch, total_tokens, world_size, rank);

  return output;
}
