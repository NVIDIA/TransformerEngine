#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "extensions.h"
#include "kv_cache.cuh"
#include "thd_utils.cuh"
#include "transformer_engine/transformer_engine.h"

constexpr int block_size = 512;
constexpr int ctas_per_sm = 4;

// get the fused attention backend
NVTE_Fused_Attn_Backend get_fused_attn_backend(
    const transformer_engine::DType q_dtype, const transformer_engine::DType kv_dtype,
    NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
    float p_dropout, size_t num_attn_heads, size_t num_gqa_groups, size_t max_seqlen_q,
    size_t max_seqlen_kv, size_t head_dim_qk, size_t head_dim_v, int64_t window_size_left,
    int64_t window_size_right) {
  NVTE_Fused_Attn_Backend fused_attention_backend = nvte_get_fused_attn_backend(
      static_cast<NVTEDType>(q_dtype), static_cast<NVTEDType>(kv_dtype), qkv_layout, bias_type,
      attn_mask_type, p_dropout, num_attn_heads, num_gqa_groups, max_seqlen_q, max_seqlen_kv,
      head_dim_qk, head_dim_v, window_size_left, window_size_right);
  return fused_attention_backend;
}

// fast zero-fills of tensors
template <typename scalar_t>
__global__ void __launch_bounds__(block_size)
    mha_fill_kernel(scalar_t *out_tensor, const int32_t *const start_row, const size_t num_rows) {
  size_t row_stride = gridDim.y * blockDim.x;
  size_t row_index = blockIdx.x + static_cast<size_t>(start_row[0]);
  size_t col_index = blockIdx.y * blockDim.x + threadIdx.x;
  while (row_index < num_rows) {
    out_tensor[row_index * row_stride + col_index] = 0;
    row_index += gridDim.x;
  }
}

// fast zero-fills of tensors
void mha_fill(const transformer_engine::TensorWrapper &self, const at::Tensor &start_index) {
  std::vector<size_t> shape = transformer_engine::pytorch::convertShape(self.shape());

  auto max_tokens = shape[0];
  auto fcd_size = 1;
  for (int i = 1; i <= shape.size(); i++) {
    fcd_size *= shape[i];
  }
  TORCH_CHECK(fcd_size % block_size == 0, "input size not aligned to block size");
  const int num_mp = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  uint64_t num_blk_y = (uint64_t)(fcd_size / block_size);
  uint64_t num_blk_x = (uint64_t)((num_mp * ctas_per_sm + num_blk_y - 1) / num_blk_y);
  dim3 dim_grid(num_blk_x, num_blk_y);
  dim3 dim_block(block_size);
  // trzeba jakos przekonwertowac DType na scalar_type
  at::ScalarType scalar_type = transformer_engine::pytorch::GetATenDType(self.dtype());
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16, scalar_type, "mha_fill", [&]() {
        mha_fill_kernel<<<dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
            static_cast<scalar_t *>(self.get_rowwise_data().data_ptr),
            static_cast<int32_t *>(start_index.data_ptr()), max_tokens);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

// extract seed and offset from PhiloxCudaState
__global__ void unpack(at::PhiloxCudaState arg, int64_t *rng_state_ptr) {
  if (arg.captured_) {
    rng_state_ptr[0] = static_cast<int64_t>(*arg.seed_.ptr);
    rng_state_ptr[1] =
        static_cast<int64_t>(*(arg.offset_.ptr) + static_cast<int64_t>(arg.offset_intragraph_));
  } else {
    rng_state_ptr[0] = static_cast<int64_t>(arg.seed_.val);
    rng_state_ptr[1] = static_cast<int64_t>(arg.offset_.val);
  }
}

// extract PhiloxCudaState from CUDA random number generator
at::PhiloxCudaState init_philox_state(at::CUDAGeneratorImpl *gen, size_t elts_per_thread) {
  at::PhiloxCudaState philox_args;
  std::lock_guard<std::mutex> lock(gen->mutex_);
  philox_args = gen->philox_cuda_state(elts_per_thread);
  return philox_args;
}

// fused attention FWD with separate Q, K and V tensors
std::vector<py::object> fused_attn_fwd(
    size_t max_seqlen_q, size_t max_seqlen_kv, bool is_training, float attn_scale, float p_dropout,
    bool set_zero, NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type,
    NVTE_Mask_Type attn_mask_type, const std::vector<int64_t> window_size,
    const at::Tensor cu_seqlens_q, const at::Tensor cu_seqlens_kv, const py::handle Q,
    const py::handle K, const py::handle V, const at::ScalarType fake_dtype,
    const std::optional<at::Tensor> cu_seqlens_q_padded,
    const std::optional<at::Tensor> cu_seqlens_kv_padded,
    const std::optional<at::Tensor> page_table_k, const std::optional<at::Tensor> page_table_v,
    py::handle s_quantizer, py::handle o_quantizer, const std::optional<at::Tensor> Bias,
    const std::optional<at::Generator> rng_gen, size_t rng_elts_per_thread) {
  using namespace transformer_engine;
  using namespace transformer_engine::pytorch;
  TensorWrapper te_Q, te_K, te_V, te_O, te_S;

  auto none = py::none();
  std::unique_ptr<Quantizer> S_quantizer = convert_quantizer(s_quantizer);
  std::unique_ptr<Quantizer> O_quantizer = convert_quantizer(o_quantizer);

  te_Q = makeTransformerEngineTensor(Q, none);
  te_K = makeTransformerEngineTensor(K, none);
  te_V = makeTransformerEngineTensor(V, none);

  // If qkv has FP8 dtype, fake_dtype_te is equal to the fake dtype of q, k, v - needed since torch do not have fp8 types.
  const transformer_engine::DType qkv_type = te_Q.dtype();
  const transformer_engine::DType fake_dtype_te = GetTransformerEngineDType(fake_dtype);

  std::vector<size_t> q_shape = convertShape(te_Q.shape());
  std::vector<size_t> k_shape = convertShape(te_K.shape());
  std::vector<size_t> v_shape = convertShape(te_V.shape());
  auto options = torch::TensorOptions().dtype(GetATenDType(qkv_type)).device(torch::kCUDA);
  // create output tensor O

  auto o_shape = std::vector<size_t>{q_shape.begin(), q_shape.end()};
  o_shape[o_shape.size() - 1] = v_shape[v_shape.size() - 1];
  py::object o_python, s_python;
  std::tie(te_O, o_python) = O_quantizer->create_tensor(o_shape, fake_dtype_te);
  std::tie(te_S, s_python) = S_quantizer->create_tensor({0}, DType::kFloat32);
  auto o_shape_int64 = std::vector<int64_t>{o_shape.begin(), o_shape.end()};

  // construct NVTE tensors
  TensorWrapper te_Bias;
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  TensorWrapper te_cu_seqlens_q_padded, te_cu_seqlens_kv_padded;
  TensorWrapper te_page_table_k, te_page_table_v;
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    auto h = q_shape[q_shape.size() - 2];
    auto d = q_shape[q_shape.size() - 1];
    if (set_zero && ((h * d) % block_size == 0) &&
        (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD)) {
      mha_fill(te_O, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      te_O.zero_(at::cuda::getCurrentCUDAStream());
    }
  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    if (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD) {
      te_O.zero_(at::cuda::getCurrentCUDAStream());
    }
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
    auto bias_sizes = Bias.value().sizes().vec();
    std::vector<size_t> bias_shape{bias_sizes.begin(), bias_sizes.end()};
    te_Bias = makeTransformerEngineTensor(Bias.value().data_ptr(), bias_shape, DType::kFloat32);
  }
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  te_cu_seqlens_q =
      makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape, DType::kInt32);
  te_cu_seqlens_kv =
      makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape, DType::kInt32);

  if ((cu_seqlens_q_padded.has_value()) && (cu_seqlens_kv_padded.has_value())) {
    auto cu_seqlens_q_padded_sizes = cu_seqlens_q_padded.value().sizes().vec();
    std::vector<size_t> cu_seqlens_q_padded_shape{cu_seqlens_q_padded_sizes.begin(),
                                                  cu_seqlens_q_padded_sizes.end()};
    auto cu_seqlens_kv_padded_sizes = cu_seqlens_kv_padded.value().sizes().vec();
    std::vector<size_t> cu_seqlens_kv_padded_shape{cu_seqlens_kv_padded_sizes.begin(),
                                                   cu_seqlens_kv_padded_sizes.end()};
    te_cu_seqlens_q_padded = makeTransformerEngineTensor(cu_seqlens_q_padded.value().data_ptr(),
                                                         cu_seqlens_q_padded_shape, DType::kInt32);
    te_cu_seqlens_kv_padded = makeTransformerEngineTensor(
        cu_seqlens_kv_padded.value().data_ptr(), cu_seqlens_kv_padded_shape, DType::kInt32);
  }

  if ((page_table_k.has_value()) && (page_table_v.has_value())) {
    auto page_table_k_sizes = page_table_k.value().sizes().vec();
    std::vector<size_t> page_table_k_shape{page_table_k_sizes.begin(), page_table_k_sizes.end()};
    auto page_table_v_sizes = page_table_v.value().sizes().vec();
    std::vector<size_t> page_table_v_shape{page_table_v_sizes.begin(), page_table_v_sizes.end()};
    te_page_table_k =
        makeTransformerEngineTensor(page_table_k.value().data_ptr(), page_table_k_shape,
                                    DType::kInt32, nullptr, nullptr, nullptr);
    te_page_table_v =
        makeTransformerEngineTensor(page_table_v.value().data_ptr(), page_table_v_shape,
                                    DType::kInt32, nullptr, nullptr, nullptr);
  }

  // extract rng seed and offset
  auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
      rng_gen, at::cuda::detail::getDefaultCUDAGenerator());
  at::PhiloxCudaState philox_args = init_philox_state(gen, rng_elts_per_thread);
  auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
  unpack<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
      philox_args, static_cast<int64_t *>(rng_state.data_ptr()));
  auto te_rng_state = makeTransformerEngineTensor(rng_state);

  // create auxiliary output tensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_fwd(
      te_Q.data(), te_K.data(), te_V.data(), te_Bias.data(), te_S.data(), te_O.data(),
      &nvte_aux_tensor_pack, te_cu_seqlens_q.data(), te_cu_seqlens_kv.data(),
      te_cu_seqlens_q_padded.data(), te_cu_seqlens_kv_padded.data(), te_page_table_k.data(),
      te_page_table_v.data(), te_rng_state.data(), max_seqlen_q, max_seqlen_kv, is_training,
      attn_scale, p_dropout, qkv_layout, bias_type, attn_mask_type, window_size[0], window_size[1],
      workspace.data(), at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace and auxiliary output tensors
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // output_tensors = [O, nvte_aux_tensor_pack.tensors]
  std::vector<py::object> output_tensors;
  output_tensors.push_back(o_python);
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    // allocate memory for nvte_aux_tensor_pack.tensors
    at::Tensor output_tensor;
    if (nvte_aux_tensor_pack.size >= 2) {
      if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI) && (Bias.has_value())) {
        if (i < nvte_aux_tensor_pack.size - 2) {
          NVTEShape temp_shape = nvte_tensor_shape(nvte_aux_tensor_pack.tensors[i]);
          output_tensor = allocateSpace(
              nvte_shape_to_vector(temp_shape),
              static_cast<DType>(nvte_tensor_type(nvte_aux_tensor_pack.tensors[i])), false);
        } else if (i == nvte_aux_tensor_pack.size - 2) {
          output_tensor = rng_state;
        } else if (i == nvte_aux_tensor_pack.size - 1) {
          output_tensor = Bias.value();
        }
      } else {
        NVTEShape temp_shape = nvte_tensor_shape(nvte_aux_tensor_pack.tensors[i]);
        output_tensor =
            (i < nvte_aux_tensor_pack.size - 1)
                ? allocateSpace(
                      nvte_shape_to_vector(temp_shape),
                      static_cast<DType>(nvte_tensor_type(nvte_aux_tensor_pack.tensors[i])), false)
                : rng_state;
      }
    } else {
      NVTEShape temp_shape = nvte_tensor_shape(nvte_aux_tensor_pack.tensors[i]);
      output_tensor = allocateSpace(
          nvte_shape_to_vector(temp_shape),
          static_cast<DType>(nvte_tensor_type(nvte_aux_tensor_pack.tensors[i])), false);
    }
    output_tensors.push_back(py::cast(output_tensor));
    NVTEBasicTensor temp_data = {output_tensor.data_ptr(),
                                 nvte_tensor_type(nvte_aux_tensor_pack.tensors[i]),
                                 nvte_tensor_shape(nvte_aux_tensor_pack.tensors[i])};
    nvte_set_tensor_param(&nvte_aux_tensor_pack.tensors[i], kNVTERowwiseData, &temp_data);
  }

  // execute the kernel
  nvte_fused_attn_fwd(
      te_Q.data(), te_K.data(), te_V.data(), te_Bias.data(), te_S.data(), te_O.data(),
      &nvte_aux_tensor_pack, te_cu_seqlens_q.data(), te_cu_seqlens_kv.data(),
      te_cu_seqlens_q_padded.data(), te_cu_seqlens_kv_padded.data(), te_page_table_k.data(),
      te_page_table_v.data(), te_rng_state.data(), max_seqlen_q, max_seqlen_kv, is_training,
      attn_scale, p_dropout, qkv_layout, bias_type, attn_mask_type, window_size[0], window_size[1],
      workspace.data(), at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers, but not allocated memory
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  // if training, [O, softmax-related tensors, rng_state]; if inference, [O]
  return output_tensors;
}

// fused attention BWD with separate Q, K and V
std::vector<py::object> fused_attn_bwd(
    size_t max_seqlen_q, size_t max_seqlen_kv, float attn_scale, float p_dropout, bool set_zero,
    NVTE_QKV_Layout qkv_layout, NVTE_Bias_Type bias_type, NVTE_Mask_Type attn_mask_type,
    const std::vector<int64_t> window_size, bool deterministic, const at::Tensor cu_seqlens_q,
    const at::Tensor cu_seqlens_kv, const py::handle Q, const py::handle K, const py::handle V,
    const py::handle O, const py::handle dO, const at::ScalarType fake_dtype,
    const transformer_engine::DType dqkv_type, const std::vector<at::Tensor> Aux_CTX_Tensors,
    const std::optional<at::Tensor> cu_seqlens_q_padded,
    const std::optional<at::Tensor> cu_seqlens_kv_padded, py::handle s_quantizer,
    py::handle dp_quantizer, py::handle dqkv_quantizer) {
  using namespace transformer_engine;
  using namespace transformer_engine::pytorch;
  auto none = py::none();
  TensorWrapper te_Q, te_K, te_V, te_O, te_dO, te_S, te_dP, te_dQ, te_dK, te_dV;
  te_Q = makeTransformerEngineTensor(Q, none);
  te_K = makeTransformerEngineTensor(K, none);
  te_V = makeTransformerEngineTensor(V, none);
  te_O = makeTransformerEngineTensor(O, none);
  te_dO = makeTransformerEngineTensor(dO, none);
  // qkv type from the te_Q
  std::unique_ptr<Quantizer> dQKV_quantizer = convert_quantizer(dqkv_quantizer);
  const transformer_engine::DType qkv_type = te_Q.dtype();
  const transformer_engine::DType fake_dtype_te = GetTransformerEngineDType(fake_dtype);

  py::object s_python, dp_python;
  std::unique_ptr<Quantizer> S_quantizer = convert_quantizer(s_quantizer);
  std::unique_ptr<Quantizer> dP_quantizer = convert_quantizer(dp_quantizer);
  std::tie(te_S, s_python) = S_quantizer->create_tensor({0}, DType::kFloat32);
  std::tie(te_dP, dp_python) = dP_quantizer->create_tensor({0}, DType::kFloat32);

  std::vector<size_t> q_shape = convertShape(te_Q.shape());
  std::vector<size_t> k_shape = convertShape(te_K.shape());
  std::vector<size_t> v_shape = convertShape(te_V.shape());
  auto h_q = q_shape[q_shape.size() - 2];
  auto h_kv = k_shape[k_shape.size() - 2];
  auto d_qk = q_shape[q_shape.size() - 1];
  auto d_v = v_shape[v_shape.size() - 1];
  auto options = torch::TensorOptions().dtype(GetATenDType(dqkv_type)).device(torch::kCUDA);
  std::vector<size_t> o_shape{q_shape.begin(), q_shape.end()};
  o_shape[o_shape.size() - 1] = d_v;

  at::Tensor dQ, dK, dV, dQKV, dKV;
  py::object py_dQ, py_dK, py_dV;
  NVTE_QKV_Layout_Group layout_group = nvte_get_qkv_layout_group(qkv_layout);
  std::vector<int64_t> tmp_shape;

  switch (layout_group) {
    case NVTE_QKV_Layout_Group::NVTE_3HD:
      tmp_shape = std::vector<int64_t>{q_shape.begin(), q_shape.end()};
      tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 2, int64_t(3));
      dQKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
      dQ = dQKV.index({"...", torch::indexing::Slice(0, 1, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 3);
      dK = dQKV.index({"...", torch::indexing::Slice(1, 2, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 3);
      dV = dQKV.index({"...", torch::indexing::Slice(2, torch::indexing::None, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 3);
      break;
    case NVTE_QKV_Layout_Group::NVTE_H3D:
      tmp_shape = std::vector<int64_t>{q_shape.begin(), q_shape.end()};
      tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 1, int64_t(3));
      dQKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
      dQ = dQKV.index({"...", torch::indexing::Slice(0, 1, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 2);
      dK = dQKV.index({"...", torch::indexing::Slice(1, 2, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 2);
      dV = dQKV.index({"...", torch::indexing::Slice(2, torch::indexing::None, 1),
                       torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 2);
      break;
    case NVTE_QKV_Layout_Group::NVTE_HD_2HD:
      tmp_shape = std::vector<int64_t>(q_shape.begin(), q_shape.end());
      dQ = torch::empty(tmp_shape, options);
      tmp_shape = std::vector<int64_t>{k_shape.begin(), k_shape.end()};
      tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 2, int64_t(2));
      dKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
      dK = dKV.index({"...", torch::indexing::Slice(0, 1, 1),
                      torch::indexing::Slice(0, torch::indexing::None, 1),
                      torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 3);
      dV = dKV.index({"...", torch::indexing::Slice(1, torch::indexing::None, 1),
                      torch::indexing::Slice(0, torch::indexing::None, 1),
                      torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 3);
      break;
    case NVTE_QKV_Layout_Group::NVTE_HD_H2D:
      tmp_shape = std::vector<int64_t>(q_shape.begin(), q_shape.end());
      dQ = torch::empty(tmp_shape, options);
      tmp_shape = std::vector<int64_t>{k_shape.begin(), k_shape.end()};
      tmp_shape.insert(tmp_shape.begin() + tmp_shape.size() - 1, int64_t(2));
      dKV = torch::empty(c10::IntArrayRef(tmp_shape), options);
      dK = dKV.index({"...", torch::indexing::Slice(0, 1, 1),
                      torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 2);
      dV = dKV.index({"...", torch::indexing::Slice(1, torch::indexing::None, 1),
                      torch::indexing::Slice(0, torch::indexing::None, 1)})
               .squeeze(tmp_shape.size() - 2);
      break;
    case NVTE_QKV_Layout_Group::NVTE_HD_HD_HD:
      tmp_shape = std::vector<int64_t>(q_shape.begin(), q_shape.end());
      dQ = torch::empty(tmp_shape, options);
      tmp_shape = std::vector<int64_t>(k_shape.begin(), k_shape.end());
      dK = torch::empty(tmp_shape, options);
      tmp_shape = std::vector<int64_t>(v_shape.begin(), v_shape.end());
      dV = torch::empty(tmp_shape, options);
      break;
    default:
      NVTE_ERROR("QKV layout not supported!");
  }
  std::tie(te_dQ, py_dQ) = dQKV_quantizer->create_tensor(q_shape, fake_dtype_te, dQ);
  std::tie(te_dK, py_dK) = dQKV_quantizer->create_tensor(k_shape, fake_dtype_te, dK);
  std::tie(te_dV, py_dV) = dQKV_quantizer->create_tensor(v_shape, fake_dtype_te, dV);

  // construct NVTE tensors
  if (qkv_type == DType::kFloat8E4M3 || qkv_type == DType::kFloat8E5M2) {
    // FP8
    if (set_zero && ((h_q * d_qk) % block_size == 0) && ((h_kv * d_qk) % block_size == 0) &&
        dQ.is_contiguous() && dK.is_contiguous() && dV.is_contiguous() &&
        (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD)) {
      mha_fill(te_dQ, cu_seqlens_q.index({torch::indexing::Slice(-1, torch::indexing::None)}));
      mha_fill(te_dK, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
      mha_fill(te_dV, cu_seqlens_kv.index({torch::indexing::Slice(-1, torch::indexing::None)}));
    } else {
      dQ.fill_(0);
      dK.fill_(0);
      dV.fill_(0);
    }

  } else if (qkv_type == DType::kBFloat16 || qkv_type == DType::kFloat16) {
    if (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD) {
      dQ.fill_(0);
      dK.fill_(0);
      dV.fill_(0);
    }
  } else {
    NVTE_ERROR("Fused attention only supports FP8 and BF16/FP16 data types. \n");
  }

  // create cu_seqlens tensorwrappers
  auto cu_seqlens_q_sizes = cu_seqlens_q.sizes().vec();
  std::vector<size_t> cu_seqlens_q_shape{cu_seqlens_q_sizes.begin(), cu_seqlens_q_sizes.end()};
  auto cu_seqlens_kv_sizes = cu_seqlens_kv.sizes().vec();
  std::vector<size_t> cu_seqlens_kv_shape{cu_seqlens_kv_sizes.begin(), cu_seqlens_kv_sizes.end()};
  TensorWrapper te_cu_seqlens_q, te_cu_seqlens_kv;
  te_cu_seqlens_q = makeTransformerEngineTensor(cu_seqlens_q.data_ptr(), cu_seqlens_q_shape,
                                                DType::kInt32, nullptr, nullptr, nullptr);
  te_cu_seqlens_kv = makeTransformerEngineTensor(cu_seqlens_kv.data_ptr(), cu_seqlens_kv_shape,
                                                 DType::kInt32, nullptr, nullptr, nullptr);

  TensorWrapper te_cu_seqlens_q_padded, te_cu_seqlens_kv_padded;
  if ((cu_seqlens_q_padded.has_value()) && (cu_seqlens_kv_padded.has_value())) {
    auto cu_seqlens_q_padded_sizes = cu_seqlens_q_padded.value().sizes().vec();
    std::vector<size_t> cu_seqlens_q_padded_shape{cu_seqlens_q_padded_sizes.begin(),
                                                  cu_seqlens_q_padded_sizes.end()};
    auto cu_seqlens_kv_padded_sizes = cu_seqlens_kv_padded.value().sizes().vec();
    std::vector<size_t> cu_seqlens_kv_padded_shape{cu_seqlens_kv_padded_sizes.begin(),
                                                   cu_seqlens_kv_padded_sizes.end()};
    te_cu_seqlens_q_padded = makeTransformerEngineTensor(cu_seqlens_q_padded.value().data_ptr(),
                                                         cu_seqlens_q_padded_shape, DType::kInt32);
    te_cu_seqlens_kv_padded = makeTransformerEngineTensor(
        cu_seqlens_kv_padded.value().data_ptr(), cu_seqlens_kv_padded_shape, DType::kInt32);
  }

  // convert auxiliary tensors from forward to NVTETensors
  NVTETensorPack nvte_aux_tensor_pack;
  nvte_tensor_pack_create(&nvte_aux_tensor_pack);
  nvte_aux_tensor_pack.size = Aux_CTX_Tensors.size();
  for (size_t i = 0; i < nvte_aux_tensor_pack.size; ++i) {
    const std::vector<int64_t> &signed_shape = Aux_CTX_Tensors[i].sizes().vec();
    const std::vector<size_t> tmp(signed_shape.begin(), signed_shape.end());

    NVTEBasicTensor temp_data = {
        Aux_CTX_Tensors[i].data_ptr(),
        static_cast<NVTEDType>(GetTransformerEngineDType(Aux_CTX_Tensors[i].scalar_type())),
        nvte_make_shape(tmp.data(), tmp.size())};
    nvte_set_tensor_param(&nvte_aux_tensor_pack.tensors[i], kNVTERowwiseData, &temp_data);
  }

  // create dBias the same shape as Bias
  at::Tensor dBias;
  TensorWrapper te_dBias;
  if ((bias_type != NVTE_NO_BIAS) && (bias_type != NVTE_ALIBI)) {
    if (nvte_aux_tensor_pack.size >= 2) {
      std::vector<int64_t> bias_shape(Aux_CTX_Tensors[nvte_aux_tensor_pack.size - 1].sizes().vec());
      dBias = torch::empty(bias_shape, options);
      te_dBias = makeTransformerEngineTensor(dBias);
    } else {
      dBias = torch::empty({1, static_cast<int64_t>(h_q), static_cast<int64_t>(max_seqlen_q),
                            static_cast<int64_t>(max_seqlen_kv)},
                           options);
      te_dBias = makeTransformerEngineTensor(dBias);
    }
    if (nvte_get_qkv_format(qkv_layout) == NVTE_QKV_Format::NVTE_THD) {
      dBias.fill_(0);
    }
  }

  // create workspace
  TensorWrapper workspace;

  // populate tensors with appropriate shapes and dtypes
  nvte_fused_attn_bwd(te_Q.data(), te_K.data(), te_V.data(), te_O.data(), te_dO.data(), te_S.data(),
                      te_dP.data(), &nvte_aux_tensor_pack, te_dQ.data(), te_dK.data(), te_dV.data(),
                      te_dBias.data(), te_cu_seqlens_q.data(), te_cu_seqlens_kv.data(),
                      te_cu_seqlens_q_padded.data(), te_cu_seqlens_kv_padded.data(), max_seqlen_q,
                      max_seqlen_kv, attn_scale, p_dropout, qkv_layout, bias_type, attn_mask_type,
                      window_size[0], window_size[1], deterministic, workspace.data(),
                      at::cuda::getCurrentCUDAStream());

  // allocate memory for workspace
  auto workspace_data = allocateSpace(workspace.shape(), workspace.dtype());
  workspace =
      makeTransformerEngineTensor(workspace_data.data_ptr(), workspace.shape(), workspace.dtype());

  // execute kernel
  nvte_fused_attn_bwd(te_Q.data(), te_K.data(), te_V.data(), te_O.data(), te_dO.data(), te_S.data(),
                      te_dP.data(), &nvte_aux_tensor_pack, te_dQ.data(), te_dK.data(), te_dV.data(),
                      te_dBias.data(), te_cu_seqlens_q.data(), te_cu_seqlens_kv.data(),
                      te_cu_seqlens_q_padded.data(), te_cu_seqlens_kv_padded.data(), max_seqlen_q,
                      max_seqlen_kv, attn_scale, p_dropout, qkv_layout, bias_type, attn_mask_type,
                      window_size[0], window_size[1], deterministic, workspace.data(),
                      at::cuda::getCurrentCUDAStream());

  // destroy tensor wrappers
  nvte_tensor_pack_destroy(&nvte_aux_tensor_pack);

  return {py_dQ, py_dK, py_dV, py::cast(dBias)};
}

namespace flash_attention {

constexpr int warp_size = 32;
constexpr int type_size = 2;  // FP16 or BF16
constexpr int nvec = sizeof(uint64_t) / type_size;
constexpr int load_size = warp_size * nvec;
constexpr int block_size = 512;

template <typename T>
__launch_bounds__(block_size) __global__
    void prepare_kernel_fwd(const T *qkvi, T *qkv, const size_t B, const size_t S, const size_t Z,
                            const size_t W) {
  const int warpid = (blockDim.x * blockIdx.x + threadIdx.x) / warp_size;
  const int id_in_warp = threadIdx.x % warp_size;
  const size_t offset_input = blockIdx.y * W + warpid * 3 * W * Z + id_in_warp * nvec;
  const T *my_input = qkvi + offset_input;

  const size_t s = warpid / B;
  if (s >= S) return;

  const size_t b = warpid % B;

  const size_t offset_output = blockIdx.y * B * S * Z * W + (s + b * S) * W * Z + id_in_warp * nvec;

  T *my_output = qkv + offset_output;

  for (int i = 0; i < Z; ++i) {
    uint64_t *out = reinterpret_cast<uint64_t *>(my_output + i * load_size);
    *out = *reinterpret_cast<const uint64_t *>(my_input + i * load_size * 3);
  }
}

template <typename T>
__launch_bounds__(block_size) __global__
    void prepare_kernel_bwd(const T *q, const T *k, const T *v, T *qkv, const size_t B,
                            const size_t S, const size_t Z, const size_t W) {
  const T *input = blockIdx.y == 0 ? q : (blockIdx.y == 1 ? k : v);

  const int warpid = (blockDim.x * blockIdx.x + threadIdx.x) / warp_size;
  const int id_in_warp = threadIdx.x % warp_size;
  const size_t offset_input = warpid * W * Z + id_in_warp * nvec;
  const T *my_input = input + offset_input;

  const size_t b = warpid / S;
  if (b >= B) return;

  const size_t s = warpid % S;

  const size_t offset_output = (b + s * B) * 3 * W * Z + id_in_warp * nvec + blockIdx.y * W;

  T *my_output = qkv + offset_output;

  for (int i = 0; i < Z; ++i) {
    uint64_t *out = reinterpret_cast<uint64_t *>(my_output + i * load_size * 3);
    *out = *reinterpret_cast<const uint64_t *>(my_input + i * load_size);
  }
}

}  // namespace flash_attention

at::Tensor fa_prepare_fwd(at::Tensor qkvi) {
  NVTE_CHECK(qkvi.dim() == 4, "Expected 4-dim tensor.");
  NVTE_CHECK(qkvi.scalar_type() == at::ScalarType::Half ||
             qkvi.scalar_type() == at::ScalarType::BFloat16);
  NVTE_CHECK(qkvi.size(3) % flash_attention::load_size == 0);
  NVTE_CHECK(qkvi.size(3) == flash_attention::load_size);
  NVTE_CHECK(qkvi.stride(3) == 1, "Wrong stride.");
  NVTE_CHECK(qkvi.stride(2) == 3 * qkvi.size(3), "Wrong stride.");
  NVTE_CHECK(qkvi.stride(1) == 3 * qkvi.size(3) * qkvi.size(2), "Wrong stride.");
  NVTE_CHECK(qkvi.stride(0) == 3 * qkvi.size(3) * qkvi.size(2) * qkvi.size(1), "Wrong stride.");

  // [s, b, n, h * 3] -> [3, b, s, n, h]
  std::vector<int64_t> shape = {3, qkvi.size(1), qkvi.size(0), qkvi.size(2), qkvi.size(3)};
  at::Tensor qkv = at::empty(shape, at::CUDA(qkvi.scalar_type()));

  size_t warps = qkvi.size(0) * qkvi.size(1);
  size_t warps_per_block = flash_attention::block_size / flash_attention::warp_size;
  size_t blocks = (warps + warps_per_block - 1) / warps_per_block;
  dim3 grid(blocks, 3);
  int threads = flash_attention::block_size;
  if (qkvi.scalar_type() == at::ScalarType::Half) {
    using dtype = at::Half;
    flash_attention::prepare_kernel_fwd<dtype>
        <<<grid, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            qkvi.data_ptr<dtype>(), qkv.data_ptr<dtype>(), shape[1], shape[2], shape[3], shape[4]);
  } else {
    using dtype = at::BFloat16;
    flash_attention::prepare_kernel_fwd<dtype>
        <<<grid, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            qkvi.data_ptr<dtype>(), qkv.data_ptr<dtype>(), shape[1], shape[2], shape[3], shape[4]);
  }

  return qkv;
}

at::Tensor fa_prepare_bwd(at::Tensor q, at::Tensor k, at::Tensor v) {
  NVTE_CHECK(q.is_contiguous());
  NVTE_CHECK(k.is_contiguous());
  NVTE_CHECK(v.is_contiguous());
  NVTE_CHECK(q.dim() == 4, "Expected 4-dim tensor.");
  NVTE_CHECK(k.dim() == 4, "Expected 4-dim tensor.");
  NVTE_CHECK(v.dim() == 4, "Expected 4-dim tensor.");
  NVTE_CHECK(q.scalar_type() == at::ScalarType::Half ||
             q.scalar_type() == at::ScalarType::BFloat16);
  NVTE_CHECK(k.scalar_type() == q.scalar_type());
  NVTE_CHECK(v.scalar_type() == q.scalar_type());
  NVTE_CHECK(q.size(3) % flash_attention::load_size == 0);
  NVTE_CHECK(q.size(3) == flash_attention::load_size);
  NVTE_CHECK(k.size(3) % flash_attention::load_size == 0);
  NVTE_CHECK(k.size(3) == flash_attention::load_size);
  NVTE_CHECK(v.size(3) % flash_attention::load_size == 0);
  NVTE_CHECK(v.size(3) == flash_attention::load_size);

  // 3 x [s, b, n, h] -> [b, s, n, 3 * h]

  std::vector<int64_t> shape = {q.size(1), q.size(0), q.size(2), 3 * q.size(3)};
  at::Tensor qkv = at::empty(shape, at::CUDA(q.scalar_type()));

  size_t warps = q.size(0) * q.size(1);
  size_t warps_per_block = flash_attention::block_size / flash_attention::warp_size;
  size_t blocks = (warps + warps_per_block - 1) / warps_per_block;
  dim3 grid(blocks, 3);
  int threads = flash_attention::block_size;
  if (q.scalar_type() == at::ScalarType::Half) {
    using dtype = at::Half;
    flash_attention::prepare_kernel_bwd<dtype>
        <<<grid, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            q.data_ptr<dtype>(), k.data_ptr<dtype>(), v.data_ptr<dtype>(), qkv.data_ptr<dtype>(),
            q.size(0), q.size(1), q.size(2), q.size(3));
  } else {
    using dtype = at::BFloat16;
    flash_attention::prepare_kernel_bwd<dtype>
        <<<grid, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            q.data_ptr<dtype>(), k.data_ptr<dtype>(), v.data_ptr<dtype>(), qkv.data_ptr<dtype>(),
            q.size(0), q.size(1), q.size(2), q.size(3));
  }

  return qkv;
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Read the half of a THD tensor
 **************************************************************************************************/

at::Tensor thd_read_half_tensor(const at::Tensor &tensor, const at::Tensor &cu_seqlens,
                                int half_idx) {
  NVTE_CHECK(tensor.dim() == 3 || tensor.dim() == 4);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);
  NVTE_CHECK(cu_seqlens.size(0) >= 2);

  // Shapes of q and dq are [t, h, d], so the dimension of "t" is 0
  // Shapes of kv and dkv are [2, t, h, d], so the dimension of "t" is 1
  int seq_dim = tensor.dim() == 3 ? 0 : 1;

  int batch = cu_seqlens.size(0) - 1;
  int num_heads = tensor.size(seq_dim + 1);
  int dim_per_head = tensor.size(seq_dim + 2);
  int hidden_size_in_bytes = num_heads * dim_per_head * c10::elementSize(tensor.scalar_type());

  // For 128-bits load/store
  NVTE_CHECK(hidden_size_in_bytes % 16 == 0);

  // Generate output
  std::vector<int64_t> shape(tensor.dim());
  for (size_t i = 0; i < shape.size(); i++) {
    shape[i] = tensor.size(i);
  }
  shape[seq_dim] /= 2;
  at::Tensor half = at::empty(shape, at::CUDA(tensor.scalar_type()));

  // Launch Kernel
  constexpr unsigned int block = 256;
  unsigned int grid_x = (tensor.size(seq_dim) / 2 * 32 + block - 1) / block;
  unsigned int grid_y = 1;
  for (int i = 0; i < seq_dim; i++) {
    grid_y *= tensor.size(i);
  }
  dim3 grid = {grid_x, grid_y};
  transformer_engine::fused_attn::thd_read_half_tensor_kernel<<<
      grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
      half.data_ptr(), tensor.data_ptr(), cu_seqlens.data_ptr<int>(), batch, hidden_size_in_bytes,
      half_idx, tensor.size(seq_dim));

  return half;
}

/***************************************************************************************************
 * Support THD format for Context Parallel: softmax_lse related operations
 **************************************************************************************************/

void thd_second_half_lse_correction(at::Tensor lse, const at::Tensor &lse_per_step,
                                    const at::Tensor &cu_seqlens, bool lse_packed) {
  NVTE_CHECK(lse.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(lse_per_step.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  int batch, num_heads, lse_seqlen, second_half_lse_seqlen;

  if (lse_packed) {
    NVTE_CHECK(lse.dim() == 2);
    NVTE_CHECK(lse_per_step.dim() == 2);

    batch = cu_seqlens.size(0) - 1;
    num_heads = lse.size(0);
    lse_seqlen = lse.size(1);
    second_half_lse_seqlen = lse_per_step.size(1);

    NVTE_CHECK(lse_per_step.size(0) == num_heads);
    NVTE_CHECK(second_half_lse_seqlen >= lse_seqlen / 2);
  } else {
    NVTE_CHECK(lse.dim() == 3);
    NVTE_CHECK(lse_per_step.dim() == 3);

    batch = lse.size(0);
    num_heads = lse.size(1);
    lse_seqlen = lse.size(2);
    second_half_lse_seqlen = lse_per_step.size(2);

    NVTE_CHECK(lse_per_step.size(0) == batch);
    NVTE_CHECK(lse_per_step.size(1) == num_heads);
    NVTE_CHECK(second_half_lse_seqlen == lse_seqlen / 2);
    NVTE_CHECK(cu_seqlens.size(0) == batch + 1);
  }

  constexpr unsigned int block = 256;
  unsigned int grid_x = (lse_seqlen / 2 + block - 1) / block;
  unsigned int grid_y = num_heads;
  dim3 grid = {grid_x, grid_y};

  if (lse_packed) {
    transformer_engine::fused_attn::thd_lse_kernel<true, LseCorrectionFunctor>
        <<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
            lse.data_ptr<float>(), lse_per_step.data_ptr<float>(), cu_seqlens.data_ptr<int>(),
            batch, num_heads, lse_seqlen, second_half_lse_seqlen);
  } else {
    transformer_engine::fused_attn::thd_lse_kernel<false, LseCorrectionFunctor>
        <<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
            lse.data_ptr<float>(), lse_per_step.data_ptr<float>(), cu_seqlens.data_ptr<int>(),
            batch, num_heads, lse_seqlen, second_half_lse_seqlen);
  }
}

at::Tensor thd_read_second_half_lse(const at::Tensor &lse, const at::Tensor &cu_seqlens,
                                    bool lse_packed, int second_half_lse_seqlen) {
  NVTE_CHECK(lse.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  int batch, num_heads, lse_seqlen;
  std::vector<int64_t> shape;

  if (lse_packed) {
    NVTE_CHECK(lse.dim() == 2);

    batch = cu_seqlens.size(0) - 1;
    num_heads = lse.size(0);
    lse_seqlen = lse.size(1);

    NVTE_CHECK(second_half_lse_seqlen >= lse_seqlen / 2);

    shape = {num_heads, second_half_lse_seqlen};
  } else {
    NVTE_CHECK(lse.dim() == 3);

    batch = lse.size(0);
    num_heads = lse.size(1);
    lse_seqlen = lse.size(2);

    NVTE_CHECK(cu_seqlens.size(0) == batch + 1);
    NVTE_CHECK(second_half_lse_seqlen == lse_seqlen / 2);

    shape = {batch, num_heads, second_half_lse_seqlen};
  }

  at::Tensor half_lse = at::zeros(shape, at::CUDA(lse.scalar_type()));

  constexpr unsigned int block = 256;
  unsigned int grid_x = (lse_seqlen / 2 + block - 1) / block;
  unsigned int grid_y = num_heads;
  dim3 grid = {grid_x, grid_y};

  if (lse_packed) {
    transformer_engine::fused_attn::thd_lse_kernel<true, ReadLseFunctor>
        <<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
            lse.data_ptr<float>(), half_lse.data_ptr<float>(), cu_seqlens.data_ptr<int>(), batch,
            num_heads, lse_seqlen, second_half_lse_seqlen);
  } else {
    transformer_engine::fused_attn::thd_lse_kernel<false, ReadLseFunctor>
        <<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
            lse.data_ptr<float>(), half_lse.data_ptr<float>(), cu_seqlens.data_ptr<int>(), batch,
            num_heads, lse_seqlen, second_half_lse_seqlen);
  }

  return half_lse;
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Out correction in forward
 **************************************************************************************************/

template <typename dtype, int only_second_half>
static void thd_out_correction_helper(at::Tensor out, const at::Tensor &out_per_step,
                                      const at::Tensor &lse, const at::Tensor &lse_per_step,
                                      const at::Tensor &cu_seqlens, bool lse_packed) {
  NVTE_CHECK(out.scalar_type() == out_per_step.scalar_type());
  NVTE_CHECK(lse.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(lse_per_step.scalar_type() == at::ScalarType::Float);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);

  int total_tokens = out.size(0);
  int num_heads = out.size(1);
  int dim_per_head = out.size(2);

  NVTE_CHECK(out_per_step.size(0) == total_tokens / (only_second_half + 1));
  NVTE_CHECK(out_per_step.size(1) == num_heads);
  NVTE_CHECK(out_per_step.size(2) == dim_per_head);

  int batch, lse_seqlen, lse_per_step_seqlen;
  if (lse_packed) {
    batch = cu_seqlens.size(0) - 1;
    lse_seqlen = lse.size(1);
    lse_per_step_seqlen = lse_per_step.size(1);

    NVTE_CHECK(lse.size(0) == num_heads);
    NVTE_CHECK(lse_seqlen >= total_tokens);
    NVTE_CHECK(lse_per_step.size(0) == num_heads);
    NVTE_CHECK(lse_per_step_seqlen >= lse_seqlen / (only_second_half + 1));
  } else {
    batch = lse.size(0);
    lse_seqlen = lse.size(2);
    lse_per_step_seqlen = lse_per_step.size(2);

    NVTE_CHECK(lse.size(1) == num_heads);
    NVTE_CHECK(lse_per_step.size(0) == batch);
    NVTE_CHECK(lse_per_step.size(1) == num_heads);
    NVTE_CHECK(lse_per_step_seqlen == lse_seqlen / (only_second_half + 1));
    NVTE_CHECK(cu_seqlens.size(0) == batch + 1);
  }

  constexpr int tile = 16;
  constexpr int block = 512;
  unsigned int grid_x =
      (static_cast<size_t>(total_tokens) / (only_second_half + 1) * tile + block - 1) / block;
  dim3 grid = {grid_x, (unsigned int)num_heads};

  if (lse_packed) {
    transformer_engine::fused_attn::thd_out_correction_kernel<dtype, only_second_half, tile, true>
        <<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
            out.data_ptr<dtype>(), out_per_step.data_ptr<dtype>(), lse.data_ptr<float>(),
            lse_per_step.data_ptr<float>(), cu_seqlens.data_ptr<int>(), batch, num_heads,
            dim_per_head, lse_seqlen, lse_per_step_seqlen);
  } else {
    transformer_engine::fused_attn::thd_out_correction_kernel<dtype, only_second_half, tile, false>
        <<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
            out.data_ptr<dtype>(), out_per_step.data_ptr<dtype>(), lse.data_ptr<float>(),
            lse_per_step.data_ptr<float>(), cu_seqlens.data_ptr<int>(), batch, num_heads,
            dim_per_head, lse_seqlen, lse_per_step_seqlen);
  }
}

void thd_out_correction(at::Tensor out, const at::Tensor &out_per_step, const at::Tensor &lse,
                        const at::Tensor &lse_per_step, const at::Tensor &cu_seqlens,
                        bool only_second_half, bool lse_packed) {
  if (only_second_half) {
    if (out.scalar_type() == at::ScalarType::Half) {
      using dtype = at::Half;
      thd_out_correction_helper<dtype, 1>(out, out_per_step, lse, lse_per_step, cu_seqlens,
                                          lse_packed);
    } else if (out.scalar_type() == at::ScalarType::BFloat16) {
      using dtype = at::BFloat16;
      thd_out_correction_helper<dtype, 1>(out, out_per_step, lse, lse_per_step, cu_seqlens,
                                          lse_packed);
    } else if (out.scalar_type() == at::ScalarType::Float) {
      using dtype = float;
      thd_out_correction_helper<dtype, 1>(out, out_per_step, lse, lse_per_step, cu_seqlens,
                                          lse_packed);
    } else {
      NVTE_ERROR("Unsupported dtype of out\n");
    }
  } else {
    if (out.scalar_type() == at::ScalarType::Half) {
      using dtype = at::Half;
      thd_out_correction_helper<dtype, 0>(out, out_per_step, lse, lse_per_step, cu_seqlens,
                                          lse_packed);
    } else if (out.scalar_type() == at::ScalarType::BFloat16) {
      using dtype = at::BFloat16;
      thd_out_correction_helper<dtype, 0>(out, out_per_step, lse, lse_per_step, cu_seqlens,
                                          lse_packed);
    } else if (out.scalar_type() == at::ScalarType::Float) {
      using dtype = float;
      thd_out_correction_helper<dtype, 0>(out, out_per_step, lse, lse_per_step, cu_seqlens,
                                          lse_packed);
    } else {
      NVTE_ERROR("Unsupported dtype of out\n");
    }
  }
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Gradients correction in backward
 **************************************************************************************************/

template <typename dtype, typename Functor_0, typename Functor_1, int functor_idx>
static void thd_grad_correction_helper(at::Tensor grad, const at::Tensor &grad_per_step,
                                       const at::Tensor &cu_seqlens) {
  NVTE_CHECK(grad.dim() == 3 || grad.dim() == 4);
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);

  // Shape of dq is [t, h, d], so the dimension of "t" is 0
  // Shape of dkv is [2, t, h, d], so the dimension of "t" is 1
  int seq_dim = grad.dim() == 3 ? 0 : 1;

  int total_tokens = grad.size(seq_dim);
  int num_heads = grad.size(seq_dim + 1);
  int dim_per_head = grad.size(seq_dim + 2);
  int batch = cu_seqlens.size(0) - 1;

  if constexpr (functor_idx < 2) {
    NVTE_CHECK(grad_per_step.size(seq_dim) == total_tokens / 2);
  } else {
    NVTE_CHECK(grad_per_step.size(seq_dim) == total_tokens);
  }
  NVTE_CHECK(grad_per_step.size(seq_dim + 1) == num_heads);
  NVTE_CHECK(grad_per_step.size(seq_dim + 2) == dim_per_head);

  size_t hidden_size = num_heads * dim_per_head;
  NVTE_CHECK((hidden_size * c10::elementSize(grad.scalar_type())) % 16 == 0);

  constexpr unsigned int block = 256;
  unsigned int grid_x;
  if constexpr (functor_idx < 2) {
    grid_x = (total_tokens / 2 * 32 + block - 1) / block;
  } else {
    grid_x = (total_tokens * 32 + block - 1) / block;
  }
  unsigned int grid_y = 1;
  for (int i = 0; i < seq_dim; i++) {
    grid_y *= grad.size(i);
  }
  dim3 grid = {grid_x, grid_y};

  transformer_engine::fused_attn::thd_grad_correction_kernel<dtype, Functor_0, Functor_1,
                                                             functor_idx, 32>
      <<<grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
          grad.data_ptr<dtype>(), grad_per_step.data_ptr<dtype>(), cu_seqlens.data_ptr<int>(),
          batch, hidden_size, total_tokens);
}

template <typename dtype>
static void thd_grad_dispatcher(at::Tensor grad, const at::Tensor &grad_per_step,
                                const at::Tensor &cu_seqlens, const std::string &first_half,
                                const std::string &second_half) {
  if (first_half == "add" && second_half == "none") {
    thd_grad_correction_helper<dtype, AddFunctor<dtype>, EmptyFunctor, 0>(grad, grad_per_step,
                                                                          cu_seqlens);
  } else if (first_half == "copy" && second_half == "none") {
    thd_grad_correction_helper<dtype, CopyFunctor, EmptyFunctor, 0>(grad, grad_per_step,
                                                                    cu_seqlens);
  } else if (first_half == "none" && second_half == "add") {
    thd_grad_correction_helper<dtype, EmptyFunctor, AddFunctor<dtype>, 1>(grad, grad_per_step,
                                                                          cu_seqlens);
  } else if (first_half == "none" && second_half == "copy") {
    thd_grad_correction_helper<dtype, EmptyFunctor, CopyFunctor, 1>(grad, grad_per_step,
                                                                    cu_seqlens);
  } else if (first_half == "add" && second_half == "copy") {
    thd_grad_correction_helper<dtype, AddFunctor<dtype>, CopyFunctor, 2>(grad, grad_per_step,
                                                                         cu_seqlens);
  } else if (first_half == "copy" && second_half == "add") {
    thd_grad_correction_helper<dtype, CopyFunctor, AddFunctor<dtype>, 2>(grad, grad_per_step,
                                                                         cu_seqlens);
  } else {
    NVTE_ERROR("Unsupported Functor of first half and second_half\n");
  }
}

void thd_grad_correction(at::Tensor grad, const at::Tensor &grad_per_step,
                         const at::Tensor &cu_seqlens, const std::string &first_half,
                         const std::string &second_half) {
  if (grad.scalar_type() == at::ScalarType::Half) {
    thd_grad_dispatcher<at::Half>(grad, grad_per_step, cu_seqlens, first_half, second_half);
  } else if (grad.scalar_type() == at::ScalarType::BFloat16) {
    thd_grad_dispatcher<at::BFloat16>(grad, grad_per_step, cu_seqlens, first_half, second_half);
  } else if (grad.scalar_type() == at::ScalarType::Float) {
    thd_grad_dispatcher<float>(grad, grad_per_step, cu_seqlens, first_half, second_half);
  } else {
    NVTE_ERROR("Unsupported dtype of grad\n");
  }
}

/***************************************************************************************************
 * Support THD format for Context Parallel: Generate partitioned indices for input tokens
 **************************************************************************************************/

at::Tensor thd_get_partitioned_indices(const at::Tensor &cu_seqlens, int total_tokens,
                                       int world_size, int rank) {
  NVTE_CHECK(cu_seqlens.scalar_type() == at::ScalarType::Int);
  NVTE_CHECK(cu_seqlens.dim() == 1);
  NVTE_CHECK(cu_seqlens.size(0) >= 2);
  NVTE_CHECK(rank >= 0 && rank < world_size);
  NVTE_CHECK(world_size > 0);
  NVTE_CHECK(total_tokens > 0 && total_tokens % (world_size * 2) == 0);

  int batch = cu_seqlens.size(0) - 1;

  std::vector<int64_t> shape = {total_tokens / world_size};
  at::Tensor output = at::empty(shape, at::CUDA(at::ScalarType::Int));

  constexpr unsigned int block = 256;
  unsigned int grid = (output.size(0) + block - 1) / block;
  transformer_engine::fused_attn::thd_partition_indices_kernel<<<
      grid, block, sizeof(int) * (batch + 1), at::cuda::getCurrentCUDAStream()>>>(
      output.data_ptr<int>(), cu_seqlens.data_ptr<int>(), batch, total_tokens, world_size, rank);

  return output;
}

/***************************************************************************************************
 * KV Cache: Convert a tensor from qkv_format = thd to qkv_format = bshd
 **************************************************************************************************/

template <typename scalar_t>
void convert_thd_to_bshd_launcher(at::Tensor tensor, at::Tensor new_tensor, at::Tensor cu_seqlens,
                                  int b, int max_seq_len, int h, int d) {
  transformer_engine::fused_attn::
      convert_thd_to_bshd_kernel<<<16, 256, 0, at::cuda::getCurrentCUDAStream()>>>(
          reinterpret_cast<scalar_t *>(tensor.data_ptr<scalar_t>()),
          reinterpret_cast<scalar_t *>(new_tensor.data_ptr<scalar_t>()), cu_seqlens.data_ptr<int>(),
          b, max_seq_len, h, d);
}

at::Tensor convert_thd_to_bshd(at::Tensor tensor, at::Tensor cu_seqlens, int b, int max_seq_len) {
  int h = tensor.size(1);
  int d = tensor.size(2);
  std::vector<int64_t> shape = {b, max_seq_len, h, d};
  at::Tensor new_tensor = at::zeros(shape, at::CUDA(tensor.scalar_type()));
  if (new_tensor.scalar_type() == at::ScalarType::Half) {
    using dtype = at::Half;
    convert_thd_to_bshd_launcher<dtype>(tensor, new_tensor, cu_seqlens, b, max_seq_len, h, d);
  } else if (new_tensor.scalar_type() == at::ScalarType::BFloat16) {
    using dtype = at::BFloat16;
    convert_thd_to_bshd_launcher<dtype>(tensor, new_tensor, cu_seqlens, b, max_seq_len, h, d);
  } else if (new_tensor.scalar_type() == at::ScalarType::Float) {
    using dtype = float;
    convert_thd_to_bshd_launcher<dtype>(tensor, new_tensor, cu_seqlens, b, max_seq_len, h, d);
  } else if (new_tensor.scalar_type() == at::ScalarType::Float8_e4m3fn) {
    using dtype = at::Float8_e4m3fn;
    convert_thd_to_bshd_launcher<dtype>(tensor, new_tensor, cu_seqlens, b, max_seq_len, h, d);
  } else if (new_tensor.scalar_type() == at::ScalarType::Float8_e5m2) {
    using dtype = at::Float8_e5m2;
    convert_thd_to_bshd_launcher<dtype>(tensor, new_tensor, cu_seqlens, b, max_seq_len, h, d);
  } else {
    NVTE_ERROR("Unsupported dtype for KV cache.\n");
  }
  return new_tensor;
}

/***************************************************************************************************
 * KV Cache: Convert a tensor from qkv_format = bshd to qkv_format = thd
 **************************************************************************************************/

template <typename scalar_t>
void convert_bshd_to_thd_launcher(at::Tensor tensor, at::Tensor new_tensor, at::Tensor cu_seqlens,
                                  int b, int max_seq_len, int h, int d) {
  transformer_engine::fused_attn::
      convert_bshd_to_thd_kernel<<<16, 256, 0, at::cuda::getCurrentCUDAStream()>>>(
          reinterpret_cast<scalar_t *>(tensor.data_ptr<scalar_t>()),
          reinterpret_cast<scalar_t *>(new_tensor.data_ptr<scalar_t>()), cu_seqlens.data_ptr<int>(),
          b, max_seq_len, h, d);
}

at::Tensor convert_bshd_to_thd(at::Tensor tensor, at::Tensor cu_seqlens, int t) {
  int b = tensor.size(0);
  int max_seq_len = tensor.size(1);
  int h = tensor.size(2);
  int d = tensor.size(3);
  std::vector<int64_t> shape = {t, h, d};
  at::Tensor new_tensor = at::zeros(shape, at::CUDA(tensor.scalar_type()));
  if (tensor.scalar_type() == at::ScalarType::Half) {
    using dtype = at::Half;
    convert_bshd_to_thd_launcher<dtype>(tensor, new_tensor, cu_seqlens, b, max_seq_len, h, d);
  } else if (tensor.scalar_type() == at::ScalarType::BFloat16) {
    using dtype = at::BFloat16;
    convert_bshd_to_thd_launcher<dtype>(tensor, new_tensor, cu_seqlens, b, max_seq_len, h, d);
  } else if (tensor.scalar_type() == at::ScalarType::Float) {
    using dtype = float;
    convert_bshd_to_thd_launcher<dtype>(tensor, new_tensor, cu_seqlens, b, max_seq_len, h, d);
  } else if (tensor.scalar_type() == at::ScalarType::Float8_e4m3fn) {
    using dtype = at::Float8_e4m3fn;
    convert_bshd_to_thd_launcher<dtype>(tensor, new_tensor, cu_seqlens, b, max_seq_len, h, d);
  } else if (tensor.scalar_type() == at::ScalarType::Float8_e5m2) {
    using dtype = at::Float8_e5m2;
    convert_bshd_to_thd_launcher<dtype>(tensor, new_tensor, cu_seqlens, b, max_seq_len, h, d);
  } else {
    NVTE_ERROR("Unsupported dtype for KV cache.\n");
  }
  return new_tensor;
}

/***************************************************************************************************
 * KV Cache: Copy new KV tokens to the KV cache
 *   1. new_k and new_v are in qkv_format; k_cache and v_cache are in 'bshd' format
 *   2. cu_new_lens and cu_cached_lens are in shape [b + 1]; cu_cached_lens include the added lens
 *      in current step
 *   3. Non-paged KV cache is a special case of paged KV cache, with page_table = [b, 1] and
 *      max_pages_per_seq = 1. We use the same underlying kernel for both non-paged and paged.
 *      Set is_non_paged = True/False to indicate as such.
 *   4. is_non_paged = True also re-indexes the KV cache, e.g. the initial batch indices [0, 3, 1, 2]
 *      becomes [0, 1, 1, 2]. The page_table = batch_indices.unsqueeze(1) is however unchanged.
 *      batch_indices_post can be used for monotonical indexing, i.e. [0, 1, 2, 3]. batch_indices is
 *      preserved for the next layer in the same iteration.
 *   5. Only supports same page_table for k_cache and v_cache
 *   6. Only pad_between_seqs = False when qkv_format = thd, i.e. there should be no pad tokens
 *      between sequences in new_k and new_v such as [a a a 0..0 b b 0..0 c 0..0].
 **************************************************************************************************/

template <typename scalar_t>
void copy_to_kv_cache_launcher(at::Tensor new_k, at::Tensor new_v, at::Tensor k_cache,
                               at::Tensor v_cache, at::Tensor page_table, at::Tensor cu_new_lens,
                               at::Tensor cu_cached_lens, NVTE_QKV_Format qkv_format, int h_kv,
                               int d_k, int d_v, int b, int max_ctx_len, int max_seq_len,
                               int max_pages_per_seq, bool is_non_paged) {
  if (new_k.data_ptr() != nullptr && new_v.data_ptr() != nullptr && k_cache.data_ptr() != nullptr &&
      v_cache.data_ptr() != nullptr) {
    if (is_non_paged) {
      transformer_engine::fused_attn::
          reindex_kv_cache_kernel<<<16, 256, 0, at::cuda::getCurrentCUDAStream()>>>(
              reinterpret_cast<scalar_t *>(k_cache.data_ptr<scalar_t>()),
              reinterpret_cast<scalar_t *>(v_cache.data_ptr<scalar_t>()),
              page_table.data_ptr<int>(), cu_new_lens.data_ptr<int>(),
              cu_cached_lens.data_ptr<int>(), h_kv, d_k, d_v, b, max_seq_len);
    }
    transformer_engine::fused_attn::
        copy_to_kv_cache_kernel<<<16, 256, 0, at::cuda::getCurrentCUDAStream()>>>(
            reinterpret_cast<scalar_t *>(new_k.data_ptr<scalar_t>()),
            reinterpret_cast<scalar_t *>(new_v.data_ptr<scalar_t>()),
            reinterpret_cast<scalar_t *>(k_cache.data_ptr<scalar_t>()),
            reinterpret_cast<scalar_t *>(v_cache.data_ptr<scalar_t>()), page_table.data_ptr<int>(),
            cu_new_lens.data_ptr<int>(), cu_cached_lens.data_ptr<int>(), qkv_format, h_kv, d_k, d_v,
            b, max_ctx_len, max_seq_len, max_pages_per_seq, is_non_paged);
  }
}

void copy_to_kv_cache(at::Tensor new_k, at::Tensor new_v, at::Tensor k_cache, at::Tensor v_cache,
                      at::Tensor page_table, at::Tensor cu_new_lens, at::Tensor cu_cached_lens,
                      NVTE_QKV_Format qkv_format, int b, int max_ctx_len, int max_seq_len,
                      int max_pages_per_seq, bool is_non_paged) {
  int h_kv = new_k.size(-2);
  int d_k = new_k.size(-1);
  int d_v = new_v.size(-1);
  NVTE_CHECK(k_cache.scalar_type() == v_cache.scalar_type() &&
                 new_k.scalar_type() == new_v.scalar_type() &&
                 new_k.scalar_type() == k_cache.scalar_type(),
             "new_k, new_v, k_cache and v_cache must be of the same data type.");
  NVTE_CHECK(qkv_format == NVTE_QKV_Format::NVTE_BSHD || qkv_format == NVTE_QKV_Format::NVTE_SBHD ||
                 qkv_format == NVTE_QKV_Format::NVTE_THD,
             "qkv_format must be {BSHD, SBHD, THD}.");
  if (k_cache.scalar_type() == at::ScalarType::Half) {
    using dtype = at::Half;
    copy_to_kv_cache_launcher<dtype>(new_k, new_v, k_cache, v_cache, page_table, cu_new_lens,
                                     cu_cached_lens, qkv_format, h_kv, d_k, d_v, b, max_ctx_len,
                                     max_seq_len, max_pages_per_seq, is_non_paged);

  } else if (k_cache.scalar_type() == at::ScalarType::BFloat16) {
    using dtype = at::BFloat16;
    copy_to_kv_cache_launcher<dtype>(new_k, new_v, k_cache, v_cache, page_table, cu_new_lens,
                                     cu_cached_lens, qkv_format, h_kv, d_k, d_v, b, max_ctx_len,
                                     max_seq_len, max_pages_per_seq, is_non_paged);
  } else if (k_cache.scalar_type() == at::ScalarType::Float) {
    using dtype = float;
    copy_to_kv_cache_launcher<dtype>(new_k, new_v, k_cache, v_cache, page_table, cu_new_lens,
                                     cu_cached_lens, qkv_format, h_kv, d_k, d_v, b, max_ctx_len,
                                     max_seq_len, max_pages_per_seq, is_non_paged);
  } else if (k_cache.scalar_type() == at::ScalarType::Float8_e4m3fn) {
    using dtype = at::Float8_e4m3fn;
    copy_to_kv_cache_launcher<dtype>(new_k, new_v, k_cache, v_cache, page_table, cu_new_lens,
                                     cu_cached_lens, qkv_format, h_kv, d_k, d_v, b, max_ctx_len,
                                     max_seq_len, max_pages_per_seq, is_non_paged);
  } else if (k_cache.scalar_type() == at::ScalarType::Float8_e5m2) {
    using dtype = at::Float8_e5m2;
    copy_to_kv_cache_launcher<dtype>(new_k, new_v, k_cache, v_cache, page_table, cu_new_lens,
                                     cu_cached_lens, qkv_format, h_kv, d_k, d_v, b, max_ctx_len,
                                     max_seq_len, max_pages_per_seq, is_non_paged);
  } else {
    NVTE_ERROR("Unsupported dtype for KV cache.\n");
  }
}
