#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
#include <hip/hip_fp8.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "common/utils.cuh"
#include "multi_tensor_apply.cuh"
#include "type_shim.h"

#define BLOCK_SIZE 512
#define ILP 4
#define THREADS_PER_WARP 32

typedef enum {
  ADAM_MODE_0 = 0,  // L2 regularization mode
  ADAM_MODE_1 = 1   // Decoupled weight decay mode(AdamW)
} adamMode_t;

using MATH_T = float;
using fp8e4m3 = __hip_fp8_e4m3_fnuz;
using fp8e5m2 = __hip_fp8_e5m2_fnuz;
using transformer_engine::DType;

template <typename T>
struct is_fp8 : std::false_type {};

template <>
struct is_fp8<fp8e4m3> : std::true_type {};

template <>
struct is_fp8<fp8e5m2> : std::true_type {};

template <bool is_fp8>
struct FP8Data {
  float scale;
  float *amax_ptr;
  float *scale_inv_ptr;
  float max;
  int warp_id;
};

template <>
struct FP8Data<false> {};

template <typename PARAM_T, typename GRAD_T, typename FULL_T, typename index_t>
struct AdamFunctorMaster {
  static constexpr bool is_fp8_type = is_fp8<PARAM_T>::value;

  __device__ __forceinline__ void operator()(index_t chunk_size, volatile int *noop_gmem,
                                             TensorListMetadata<5, is_fp8_type> &tl,  // NOLINT(*)
                                             const float beta1, const float beta2,
                                             const float beta1_correction,
                                             const float beta2_correction, const float epsilon,
                                             const float lr, adamMode_t mode, const float decay) {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    FP8Data<is_fp8_type> fp8_data;

    index_t tensor_loc = tl.block_to_tensor[blockIdx.x];

    // potentially use to pass in list of scalar
    // int tensor_num = tl.start_tensor_this_launch + tensor_loc;

    index_t chunk_idx = tl.block_to_chunk[blockIdx.x];
    index_t n = tl.sizes[tensor_loc];

    GRAD_T *g = reinterpret_cast<GRAD_T *>(tl.addresses[0][tensor_loc]);
    g += chunk_idx * chunk_size;

    PARAM_T *p = reinterpret_cast<PARAM_T *>(tl.addresses[1][tensor_loc]);
    p += chunk_idx * chunk_size;

    FULL_T *m = reinterpret_cast<FULL_T *>(tl.addresses[2][tensor_loc]);
    m += chunk_idx * chunk_size;

    FULL_T *v = reinterpret_cast<FULL_T *>(tl.addresses[3][tensor_loc]);
    v += chunk_idx * chunk_size;

    FULL_T *p_master = reinterpret_cast<FULL_T *>(tl.addresses[4][tensor_loc]);
    p_master += chunk_idx * chunk_size;

    n -= chunk_idx * chunk_size;

    if constexpr (is_fp8_type) {
      float *scale_ptr = reinterpret_cast<float *>(tl.fp8_meta_addresses[0][tensor_loc]);
      fp8_data.scale = scale_ptr != nullptr ? *scale_ptr : 1;
      fp8_data.amax_ptr = reinterpret_cast<float *>(tl.fp8_meta_addresses[1][tensor_loc]);
      fp8_data.scale_inv_ptr = reinterpret_cast<float *>(tl.fp8_meta_addresses[2][tensor_loc]);
      fp8_data.warp_id = threadIdx.x / THREADS_PER_WARP;
      fp8_data.max = 0;
    }

    // see note in multi_tensor_scale_kernel.cu
    for (index_t i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x * ILP) {
      MATH_T r_g[ILP];
      MATH_T r_p[ILP];
      MATH_T r_m[ILP];
      MATH_T r_v[ILP];
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          r_g[ii] = static_cast<MATH_T>(g[i]);
          r_p[ii] = static_cast<MATH_T>(p_master[i]);
          r_m[ii] = static_cast<MATH_T>(m[i]);
          r_v[ii] = static_cast<MATH_T>(v[i]);
        } else {
          r_g[ii] = MATH_T(0);
          r_p[ii] = MATH_T(0);
          r_m[ii] = MATH_T(0);
          r_v[ii] = MATH_T(0);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        if (mode == ADAM_MODE_0) {  // L2
          r_g[ii] = r_g[ii] + (decay * r_p[ii]);
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = next_m_unbiased / denom;
          r_p[ii] = r_p[ii] - (lr * update);
        } else {  // weight decay
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = (next_m_unbiased / denom) + (decay * r_p[ii]);
          r_p[ii] = r_p[ii] - (lr * update);
        }
      }

#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          p_master[i] = static_cast<FULL_T>(r_p[ii]);
          m[i] = static_cast<FULL_T>(r_m[ii]);
          v[i] = static_cast<FULL_T>(r_v[ii]);
          if constexpr (is_fp8_type) {
            __builtin_assume(fp8_data.max >= 0);
            fp8_data.max = fmaxf(fabsf(r_p[ii]), fp8_data.max);
            p[i] = static_cast<PARAM_T>(r_p[ii] * fp8_data.scale);
          } else {
            p[i] = static_cast<PARAM_T>(r_p[ii]);
          }
        }
      }
    }

    if constexpr (is_fp8_type) {
      fp8_data.max = transformer_engine::reduce_max<BLOCK_SIZE / THREADS_PER_WARP>(
          fp8_data.max, fp8_data.warp_id);
      if (threadIdx.x == 0) {
        if (fp8_data.amax_ptr != nullptr) {
          transformer_engine::atomicMaxFloat(fp8_data.amax_ptr, fp8_data.max);
        }
        if (fp8_data.scale_inv_ptr != nullptr) {
          *fp8_data.scale_inv_ptr = __frcp_rn(fp8_data.scale);
        }
      }
    }
  }
};

template <typename T, typename FULL_T, typename index_t>
struct AdamFunctor {
  __device__ __forceinline__ void operator()(index_t chunk_size, volatile int *noop_gmem,
                                             TensorListMetadata<4> &tl,  // NOLINT(*)
                                             const float beta1, const float beta2,
                                             const float beta1_correction,
                                             const float beta2_correction, const float epsilon,
                                             const float lr, adamMode_t mode, const float decay) {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    index_t tensor_loc = tl.block_to_tensor[blockIdx.x];

    // potentially use to pass in list of scalar
    // int tensor_num = tl.start_tensor_this_launch + tensor_loc;

    index_t chunk_idx = tl.block_to_chunk[blockIdx.x];
    index_t n = tl.sizes[tensor_loc];

    T *g = reinterpret_cast<T *>(tl.addresses[0][tensor_loc]);
    g += chunk_idx * chunk_size;

    T *p = reinterpret_cast<T *>(tl.addresses[1][tensor_loc]);
    p += chunk_idx * chunk_size;

    FULL_T *m = reinterpret_cast<FULL_T *>(tl.addresses[2][tensor_loc]);
    m += chunk_idx * chunk_size;

    FULL_T *v = reinterpret_cast<FULL_T *>(tl.addresses[3][tensor_loc]);
    v += chunk_idx * chunk_size;

    n -= chunk_idx * chunk_size;

    // see note in multi_tensor_scale_kernel.cu
    for (index_t i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x * ILP) {
      MATH_T r_g[ILP];
      MATH_T r_p[ILP];
      MATH_T r_m[ILP];
      MATH_T r_v[ILP];
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          r_g[ii] = g[i];
          r_p[ii] = p[i];
          r_m[ii] = m[i];
          r_v[ii] = v[i];
        } else {
          r_g[ii] = MATH_T(0);
          r_p[ii] = MATH_T(0);
          r_m[ii] = MATH_T(0);
          r_v[ii] = MATH_T(0);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        if (mode == ADAM_MODE_0) {  // L2
          r_g[ii] = r_g[ii] + (decay * r_p[ii]);
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = next_m_unbiased / denom;
          r_p[ii] = r_p[ii] - (lr * update);
        } else {  // weight decay
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = (next_m_unbiased / denom) + (decay * r_p[ii]);
          r_p[ii] = r_p[ii] - (lr * update);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          p[i] = r_p[ii];
          m[i] = r_m[ii];
          v[i] = r_v[ii];
        }
      }
    }
  }
};

template <typename T, typename FULL_T>
struct AdamCapturableFunctor {
  __device__ __forceinline__ void operator()(int chunk_size, volatile int *noop_gmem,
                                             TensorListMetadata<4> &tl,  // NOLINT(*)
                                             const float beta1, const float beta2, const int *step,
                                             const int bias_correction, const float epsilon,
                                             const float *lr, adamMode_t mode, const float decay,
                                             const float *inv_scale) {
    if (*noop_gmem == 1) return;

    float beta1_correction = 1.0f, beta2_correction = 1.0f;
    if (bias_correction == 1) {
      beta1_correction = 1 - pow(beta1, *step);
      beta2_correction = 1 - pow(beta2, *step);
    }

    int tensor_loc = tl.block_to_tensor[blockIdx.x];

    // potentially use to pass in list of scalar
    // int tensor_num = tl.start_tensor_this_launch + tensor_loc;

    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    T *g = reinterpret_cast<T *>(tl.addresses[0][tensor_loc]);
    g += chunk_idx * chunk_size;

    T *p = reinterpret_cast<T *>(tl.addresses[1][tensor_loc]);
    p += chunk_idx * chunk_size;

    FULL_T *m = reinterpret_cast<FULL_T *>(tl.addresses[2][tensor_loc]);
    m += chunk_idx * chunk_size;

    FULL_T *v = reinterpret_cast<FULL_T *>(tl.addresses[3][tensor_loc]);
    v += chunk_idx * chunk_size;

    n -= chunk_idx * chunk_size;

    // see note in multi_tensor_scale_kernel.cu
    for (int i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x * ILP) {
      MATH_T r_g[ILP];
      MATH_T r_p[ILP];
      MATH_T r_m[ILP];
      MATH_T r_v[ILP];
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          r_g[ii] = static_cast<MATH_T>(g[i]) * (*inv_scale);
          g[i] = static_cast<T>(r_g[ii]);
          r_p[ii] = static_cast<MATH_T>(p[i]);
          r_m[ii] = static_cast<MATH_T>(m[i]);
          r_v[ii] = static_cast<MATH_T>(v[i]);
        } else {
          r_g[ii] = MATH_T(0);
          r_p[ii] = MATH_T(0);
          r_m[ii] = MATH_T(0);
          r_v[ii] = MATH_T(0);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        if (mode == ADAM_MODE_0) {  // L2
          r_g[ii] = r_g[ii] + (decay * r_p[ii]);
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = next_m_unbiased / denom;
          r_p[ii] = r_p[ii] - (*lr * update);
        } else {  // weight decay
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = (next_m_unbiased / denom) + (decay * r_p[ii]);
          r_p[ii] = r_p[ii] - (*lr * update);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          p[i] = static_cast<T>(r_p[ii]);
          m[i] = static_cast<T>(r_m[ii]);
          v[i] = static_cast<T>(r_v[ii]);
        }
      }
    }
  }
};

template <typename T, typename FULL_T>
struct AdamCapturableMasterFunctor {
  __device__ __forceinline__ void operator()(int chunk_size, volatile int *noop_gmem,
                                             TensorListMetadata<5> &tl,  // NOLINT(*)
                                             const float beta1, const float beta2, const int *step,
                                             const int bias_correction, const float epsilon,
                                             const float *lr, adamMode_t mode, const float decay,
                                             const float *inv_scale) {
    if (*noop_gmem == 1) return;

    float beta1_correction = 1.0f, beta2_correction = 1.0f;
    if (bias_correction == 1) {
      beta1_correction = 1 - pow(beta1, *step);
      beta2_correction = 1 - pow(beta2, *step);
    }

    int tensor_loc = tl.block_to_tensor[blockIdx.x];

    // potentially use to pass in list of scalar
    // int tensor_num = tl.start_tensor_this_launch + tensor_loc;

    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    T *g = reinterpret_cast<T *>(tl.addresses[0][tensor_loc]);
    g += chunk_idx * chunk_size;

    T *p = reinterpret_cast<T *>(tl.addresses[1][tensor_loc]);
    p += chunk_idx * chunk_size;

    FULL_T *m = reinterpret_cast<FULL_T *>(tl.addresses[2][tensor_loc]);
    m += chunk_idx * chunk_size;

    FULL_T *v = reinterpret_cast<FULL_T *>(tl.addresses[3][tensor_loc]);
    v += chunk_idx * chunk_size;

    FULL_T *p_master = reinterpret_cast<FULL_T *>(tl.addresses[4][tensor_loc]);
    p_master += chunk_idx * chunk_size;

    n -= chunk_idx * chunk_size;

    // see note in multi_tensor_scale_kernel.cu
    for (int i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x * ILP) {
      MATH_T r_g[ILP];
      MATH_T r_p[ILP];
      MATH_T r_m[ILP];
      MATH_T r_v[ILP];
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          r_g[ii] = static_cast<MATH_T>(g[i]) * (*inv_scale);
          g[i] = static_cast<T>(r_g[ii]);
          r_p[ii] = static_cast<MATH_T>(p_master[i]);
          r_m[ii] = static_cast<MATH_T>(m[i]);
          r_v[ii] = static_cast<MATH_T>(v[i]);
        } else {
          r_g[ii] = MATH_T(0);
          r_p[ii] = MATH_T(0);
          r_m[ii] = MATH_T(0);
          r_v[ii] = MATH_T(0);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        if (mode == ADAM_MODE_0) {  // L2
          r_g[ii] = r_g[ii] + (decay * r_p[ii]);
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = next_m_unbiased / denom;
          r_p[ii] = r_p[ii] - (*lr * update);
        } else {  // weight decay
          r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
          r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
          MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
          MATH_T update = (next_m_unbiased / denom) + (decay * r_p[ii]);
          r_p[ii] = r_p[ii] - (*lr * update);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          p[i] = static_cast<T>(r_p[ii]);
          p_master[i] = static_cast<FULL_T>(r_p[ii]);
          m[i] = static_cast<FULL_T>(r_m[ii]);
          v[i] = static_cast<FULL_T>(r_v[ii]);
        }
      }
    }
  }
};

void multi_tensor_adam_cuda(int chunk_size, at::Tensor noop_flag,
                            std::vector<std::vector<at::Tensor>> tensor_lists, const float lr,
                            const float beta1, const float beta2, const float epsilon,
                            const int step, const int mode, const int bias_correction,
                            const float weight_decay) {
  using namespace at;

  // Handle bias correction mode
  float bias_correction1 = 1.0f, bias_correction2 = 1.0f;
  if (bias_correction == 1) {
    bias_correction1 = 1 - std::pow(beta1, step);
    bias_correction2 = 1 - std::pow(beta2, step);
  }

  size_t max_size = 0;
  bool requires_64bit_indexing = false;
  for (auto it = tensor_lists.begin(); it != tensor_lists.end(); it++) {
    for (auto it2 = it->begin(); it2 != it->end(); it2++) {
      if (it2->numel() > max_size) {
        max_size = it2->numel();
        if (max_size >= INT_MAX) {
          requires_64bit_indexing = true;
          break;
        }
      }
    }
    if (requires_64bit_indexing) {
      break;
    }
  }

  const auto p_in_type = tensor_lists[1][0].scalar_type();
  auto tl_size = tensor_lists.size();

  // case 4:  g, p, m, v
  // case 5:  g, p, m, v, p_master
  TORCH_CHECK(tl_size == 4 || tl_size == 5, "tensor list must contain 4 or 5");

  if (requires_64bit_indexing) {
    if (tl_size == 4) {
      // Assume single type across p,g,m1,m2 now
      DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT(
          p_in_type, 0, "adam",
          multi_tensor_apply<4>((int64_t)BLOCK_SIZE, (int64_t)chunk_size, noop_flag, tensor_lists,
                                AdamFunctor<scalar_t_0, float, int64_t>(), beta1, beta2,
                                bias_correction1, bias_correction2, epsilon, lr, (adamMode_t)mode,
                                weight_decay);)
    } else {
      // g, p, m, v, p_master
      const auto g_in_type = tensor_lists[0][0].scalar_type();
      DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT(
          p_in_type, 0, "adam",
          DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT(
              g_in_type, 1, "adam",
              multi_tensor_apply<5>((int64_t)BLOCK_SIZE, (int64_t)chunk_size, noop_flag,
                                    tensor_lists,
                                    AdamFunctorMaster<scalar_t_0, scalar_t_1, float, int64_t>(),
                                    beta1, beta2, bias_correction1, bias_correction2, epsilon, lr,
                                    (adamMode_t)mode, weight_decay);));
    }
  } else {
    if (tl_size == 4) {
      // Assume single type across p,g,m1,m2 now
      DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT(
          p_in_type, 0, "adam",
          multi_tensor_apply<4>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                                AdamFunctor<scalar_t_0, float, int32_t>(), beta1, beta2,
                                bias_correction1, bias_correction2, epsilon, lr, (adamMode_t)mode,
                                weight_decay);)
    } else {
      const auto g_in_type = tensor_lists[0][0].scalar_type();
      DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT(
          p_in_type, 0, "adam",
          DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT(
              g_in_type, 1, "adam",
              multi_tensor_apply<5>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                                    AdamFunctorMaster<scalar_t_0, scalar_t_1, float, int32_t>(),
                                    beta1, beta2, bias_correction1, bias_correction2, epsilon, lr,
                                    (adamMode_t)mode, weight_decay);));
    }
  }
  AT_CUDA_CHECK(hipGetLastError());
}

void multi_tensor_adam_fp8_cuda(int chunk_size, at::Tensor noop_flag,
                                std::vector<std::vector<at::Tensor>> tensor_lists, const float lr,
                                const float beta1, const float beta2, const float epsilon,
                                const int step, const int mode, const int bias_correction,
                                const float weight_decay, DType fp8_dtype) {
  using namespace at;

  // Handle bias correction mode
  float bias_correction1 = 1.0f, bias_correction2 = 1.0f;
  if (bias_correction == 1) {
    bias_correction1 = 1 - std::pow(beta1, step);
    bias_correction2 = 1 - std::pow(beta2, step);
  }

  size_t max_size = 0;
  bool requires_64bit_indexing = false;
  for (auto it = tensor_lists.begin(); it != tensor_lists.end(); it++) {
    for (auto it2 = it->begin(); it2 != it->end(); it2++) {
      if (it2->numel() > max_size) {
        max_size = it2->numel();
        if (max_size >= INT_MAX) {
          requires_64bit_indexing = true;
          break;
        }
      }
    }
    if (requires_64bit_indexing) {
      break;
    }
  }

  const auto g_in_type = tensor_lists[0][0].scalar_type();
  auto tl_size = tensor_lists.size();

  // case 8:  g, p_fp8, m, v, p_master, scale, amax, scale_inv
  TORCH_CHECK(tl_size == 8, "tensor list must contain 8 tensors");

  if (requires_64bit_indexing) {
    TRANSFORMER_ENGINE_TYPE_SWITCH_FP8ONLY(
        fp8_dtype, FP8_T,
        DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT(
            g_in_type, 0, "adam",
            multi_tensor_apply<5, true>(
                (int64_t)BLOCK_SIZE, (int64_t)chunk_size, noop_flag, tensor_lists,
                AdamFunctorMaster<FP8_T, scalar_t_0, float, int64_t>(), beta1, beta2,
                bias_correction1, bias_correction2, epsilon, lr, (adamMode_t)mode, weight_decay);));
  } else {
    TRANSFORMER_ENGINE_TYPE_SWITCH_FP8ONLY(
        fp8_dtype, FP8_T,
        DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT(
            g_in_type, 0, "adam",
            multi_tensor_apply<5, true>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                                        AdamFunctorMaster<FP8_T, scalar_t_0, float, int32_t>(),
                                        beta1, beta2, bias_correction1, bias_correction2, epsilon,
                                        lr, (adamMode_t)mode, weight_decay);));
  }
  AT_CUDA_CHECK(hipGetLastError());
}

void multi_tensor_adam_capturable_cuda(int chunk_size, at::Tensor noop_flag,
                                       std::vector<std::vector<at::Tensor>> tensor_lists,
                                       at::Tensor lr, const float beta1, const float beta2,
                                       const float epsilon, at::Tensor step, const int mode,
                                       const int bias_correction, const float weight_decay,
                                       at::Tensor inv_scale) {
  using namespace at;

  DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT(
      tensor_lists[0][0].scalar_type(), 0, "adam",
      multi_tensor_apply<4>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                            AdamCapturableFunctor<scalar_t_0, float>(), beta1, beta2,
                            step.data_ptr<int>(), bias_correction, epsilon, lr.data_ptr<float>(),
                            (adamMode_t)mode, weight_decay, inv_scale.data_ptr<float>());)

  AT_CUDA_CHECK(hipGetLastError());
}

void multi_tensor_adam_capturable_master_cuda(int chunk_size, at::Tensor noop_flag,
                                              std::vector<std::vector<at::Tensor>> tensor_lists,
                                              at::Tensor lr, const float beta1, const float beta2,
                                              const float epsilon, at::Tensor step, const int mode,
                                              const int bias_correction, const float weight_decay,
                                              at::Tensor inv_scale) {
  using namespace at;

  DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT(
      tensor_lists[0][0].scalar_type(), 0, "adam",
      multi_tensor_apply<5>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                            AdamCapturableMasterFunctor<scalar_t_0, float>(), beta1, beta2,
                            step.data_ptr<int>(), bias_correction, epsilon, lr.data_ptr<float>(),
                            (adamMode_t)mode, weight_decay, inv_scale.data_ptr<float>());)

  AT_CUDA_CHECK(hipGetLastError());
}
