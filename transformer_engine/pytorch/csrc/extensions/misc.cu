/*************************************************************************
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "extensions.h"

size_t get_cublasLt_version() { return cublasLtGetVersion(); }

size_t get_cudnn_version() { return hipdnnGetVersion(); }
