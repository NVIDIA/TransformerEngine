/*************************************************************************
 * Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include "scaled_upper_triang_masked_softmax_dropout.h"
#include "type_shim.h"

namespace transformer_engine {
namespace scaled_upper_triang_masked_softmax {

torch::Tensor fwd_cuda(
    torch::Tensor const& input,
    float scale_factor,
    float p_dropout,
    c10::optional<at::Generator> gen_
    ) {
  // input is a 3d tensor with dimensions [attn_batches, seq_len, seq_len]
  const int attn_batches = input.size(0);
  const int seq_len = input.size(1);
  TORCH_INTERNAL_ASSERT(seq_len <= 2048);

  // Output
  auto act_options = input.options().requires_grad(false);
  torch::Tensor softmax_results =
      torch::empty({attn_batches, seq_len, seq_len}, act_options);

  // Softmax Intermediate Result Ptr
  void* input_ptr = static_cast<void*>(input.data_ptr());
  void* softmax_results_ptr = static_cast<void*>(softmax_results.data_ptr());

  float p_keep = 1.f - p_dropout;

  DISPATCH_HALF_AND_BFLOAT(
      input.scalar_type(),
      "dispatch_scaled_upper_triang_masked_softmax_forward",
      dispatch_scaled_upper_triang_masked_softmax_forward<scalar_t, scalar_t, float>(
    reinterpret_cast<scalar_t*>(softmax_results_ptr),
    reinterpret_cast<const scalar_t*>(input_ptr),
    scale_factor,
    seq_len,
    seq_len,
    attn_batches,
    p_keep,
    gen_););
  return softmax_results;
}


torch::Tensor bwd_cuda(
    torch::Tensor const& output_grads_,
    torch::Tensor const& softmax_results_,
    float scale_factor,
    float p_dropout)  {

  auto output_grads = output_grads_.contiguous();
  auto softmax_results = softmax_results_.contiguous();

  // output grads is a 3d tensor with dimensions [attn_batches, seq_len, seq_len]
  const int attn_batches = output_grads.size(0);
  const int seq_len = output_grads.size(1);
  TORCH_INTERNAL_ASSERT(output_grads.size(1) == output_grads.size(2));

  void* output_grads_ptr = static_cast<void*>(output_grads.data_ptr());

  const float p_keep = 1.f - p_dropout;
  const float rp_keep = 1.f / p_keep;
  // Softmax Grad
  DISPATCH_HALF_AND_BFLOAT(
      output_grads_.scalar_type(),
      "dispatch_scaled_upper_triang_masked_softmax_backward",
      dispatch_scaled_upper_triang_masked_softmax_backward<scalar_t, scalar_t, float>(
        reinterpret_cast<scalar_t*>(output_grads_ptr),
        reinterpret_cast<scalar_t*>(output_grads_ptr),
        reinterpret_cast<scalar_t const*>(softmax_results.data_ptr()),
        scale_factor,
        seq_len,
        seq_len,
        attn_batches,
        p_keep,
        rp_keep););

  // backward pass is completely in-place
  return output_grads;
}

}  // end namespace scaled_upper_triang_masked_softmax
}  // end namespace transformer_engine
